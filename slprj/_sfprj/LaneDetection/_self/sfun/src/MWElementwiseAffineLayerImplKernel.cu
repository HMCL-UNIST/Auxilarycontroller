#include "hip/hip_runtime.h"
#include "MWKernelHeaders.hpp"
#include <math.h>
#include <stdio.h>
 void __global__ __launch_bounds__(1024) scale_scalar_kernel(float* 
inputBuffer, float* outputBuffer, float* puSFZkRJmyuFPfQRswDK, long int 
YGiQICncmsGZkNUyiQyg) {  for (long int idx = blockDim.x * blockIdx.x + 
threadIdx.x; idx < YGiQICncmsGZkNUyiQyg; idx += blockDim.x * gridDim.x) {  
outputBuffer[idx] = puSFZkRJmyuFPfQRswDK[0]*inputBuffer[idx]; } } void __global__ 
__launch_bounds__(1024) scale_vector_kernel(float* inputBuffer, float* 
outputBuffer, float* puSFZkRJmyuFPfQRswDK, double YNmJhGSUszJKxsodxiuV, 
double YNDVziqpDddiXQKYZZhX, long int YGiQICncmsGZkNUyiQyg) {  for 
(long int idx = blockDim.x * blockIdx.x + threadIdx.x; idx < 
YGiQICncmsGZkNUyiQyg; idx += blockDim.x * gridDim.x) { double batchIdx = 
floor(idx / YNDVziqpDddiXQKYZZhX); double i_batch = idx - (batchIdx * 
YNDVziqpDddiXQKYZZhX); double channelIdx = floor(i_batch / 
YNmJhGSUszJKxsodxiuV); outputBuffer[idx] = 
puSFZkRJmyuFPfQRswDK[static_cast<long int>(channelIdx)]*inputBuffer[idx]; } } void 
__global__ __launch_bounds__(1024) scale_matrix2d_kernel(float* inputBuffer, 
float* outputBuffer, float* puSFZkRJmyuFPfQRswDK, double 
YNmJhGSUszJKxsodxiuV, long int YGiQICncmsGZkNUyiQyg) {  for (long int 
idx = blockDim.x * blockIdx.x + threadIdx.x; idx < YGiQICncmsGZkNUyiQyg; idx += 
blockDim.x * gridDim.x) { double totalChannelIdx = floor(idx / 
YNmJhGSUszJKxsodxiuV); double i_channel = idx - (totalChannelIdx * 
YNmJhGSUszJKxsodxiuV); outputBuffer[idx] = 
puSFZkRJmyuFPfQRswDK[static_cast<long int>(i_channel)]*inputBuffer[idx]; } } void 
__global__ __launch_bounds__(1024) scale_tensor3d_kernel(float* inputBuffer, 
float* outputBuffer, float* puSFZkRJmyuFPfQRswDK, double 
YNDVziqpDddiXQKYZZhX, long int YGiQICncmsGZkNUyiQyg) {  for (long int 
idx = blockDim.x * blockIdx.x + threadIdx.x; idx < YGiQICncmsGZkNUyiQyg; idx += 
blockDim.x * gridDim.x) { double batchIdx = floor(idx / 
YNDVziqpDddiXQKYZZhX); double i_batch = idx - (batchIdx * 
YNDVziqpDddiXQKYZZhX); outputBuffer[idx] = 
puSFZkRJmyuFPfQRswDK[static_cast<long int>(i_batch)]*inputBuffer[idx]; } }  void 
__global__ __launch_bounds__(1024) offset_scalar_kernel(float* inputBuffer, 
float* outputBuffer, float* gCYwEfkibolsgZAumsuW, long int YGiQICncmsGZkNUyiQyg, 
bool ZKjSVYDDjACizBkGbqBq, int bOrQjJTNlssnrexxbHdi, int 
veFyKKHbdqBIvQLYBqfF) {  for (long int idx = blockDim.x * blockIdx.x + 
threadIdx.x; idx < YGiQICncmsGZkNUyiQyg; idx += blockDim.x * gridDim.x) { float 
out = inputBuffer[idx] + gCYwEfkibolsgZAumsuW[0]; if (ZKjSVYDDjACizBkGbqBq){ out = 
out > veFyKKHbdqBIvQLYBqfF ? veFyKKHbdqBIvQLYBqfF : out; out = out < 
bOrQjJTNlssnrexxbHdi ? bOrQjJTNlssnrexxbHdi : out; } outputBuffer[idx] = out; 
} } void __global__ __launch_bounds__(1024) offset_vector_kernel(float* 
inputBuffer, float* outputBuffer, float* gCYwEfkibolsgZAumsuW,  double 
YNmJhGSUszJKxsodxiuV, double YNDVziqpDddiXQKYZZhX, long int 
YGiQICncmsGZkNUyiQyg, bool ZKjSVYDDjACizBkGbqBq, int bOrQjJTNlssnrexxbHdi, int 
veFyKKHbdqBIvQLYBqfF) {  for (long int idx = blockDim.x * blockIdx.x + 
threadIdx.x; idx < YGiQICncmsGZkNUyiQyg; idx += blockDim.x * gridDim.x) { 
double batchIdx = floor(idx / YNDVziqpDddiXQKYZZhX); double i_batch = 
idx - (batchIdx * YNDVziqpDddiXQKYZZhX); double channelIdx = 
floor(i_batch / YNmJhGSUszJKxsodxiuV); float out = inputBuffer[idx] + 
gCYwEfkibolsgZAumsuW[static_cast<long int>(channelIdx)]; if 
(ZKjSVYDDjACizBkGbqBq){ out = out > veFyKKHbdqBIvQLYBqfF ? 
veFyKKHbdqBIvQLYBqfF : out; out = out < bOrQjJTNlssnrexxbHdi ? 
bOrQjJTNlssnrexxbHdi : out; } outputBuffer[idx] = out; } } void __global__ 
__launch_bounds__(1024) offset_matrix2d_kernel(float* inputBuffer, float* 
outputBuffer, float* gCYwEfkibolsgZAumsuW, double YNmJhGSUszJKxsodxiuV, 
long int YGiQICncmsGZkNUyiQyg, bool ZKjSVYDDjACizBkGbqBq, int 
bOrQjJTNlssnrexxbHdi, int veFyKKHbdqBIvQLYBqfF) {  for (long int idx = 
blockDim.x * blockIdx.x + threadIdx.x; idx < YGiQICncmsGZkNUyiQyg; idx += 
blockDim.x * gridDim.x) { double totalChannelIdx = floor(idx / 
YNmJhGSUszJKxsodxiuV); double i_channel = idx - (totalChannelIdx * 
YNmJhGSUszJKxsodxiuV); float out = inputBuffer[idx] + 
gCYwEfkibolsgZAumsuW[static_cast<long int>(i_channel)]; if (ZKjSVYDDjACizBkGbqBq){ 
out = out > veFyKKHbdqBIvQLYBqfF ? veFyKKHbdqBIvQLYBqfF : out; out = out < 
bOrQjJTNlssnrexxbHdi ? bOrQjJTNlssnrexxbHdi : out; } outputBuffer[idx] = out; 
} } void __global__ __launch_bounds__(1024) offset_tensor3d_kernel(float* 
inputBuffer, float* outputBuffer, float* gCYwEfkibolsgZAumsuW, double 
YNDVziqpDddiXQKYZZhX, long int YGiQICncmsGZkNUyiQyg, bool 
ZKjSVYDDjACizBkGbqBq, int bOrQjJTNlssnrexxbHdi, int veFyKKHbdqBIvQLYBqfF) {  
for (long int idx = blockDim.x * blockIdx.x + threadIdx.x; idx < 
YGiQICncmsGZkNUyiQyg; idx += blockDim.x * gridDim.x) { double batchIdx = 
floor(idx / YNDVziqpDddiXQKYZZhX); double i_batch = idx - (batchIdx * 
YNDVziqpDddiXQKYZZhX); float out = inputBuffer[idx] + 
gCYwEfkibolsgZAumsuW[static_cast<long int>(i_batch)]; if (ZKjSVYDDjACizBkGbqBq){ 
out = out > veFyKKHbdqBIvQLYBqfF ? veFyKKHbdqBIvQLYBqfF : out; out = out < 
bOrQjJTNlssnrexxbHdi ? bOrQjJTNlssnrexxbHdi : out; } outputBuffer[idx] = out; 
} } 