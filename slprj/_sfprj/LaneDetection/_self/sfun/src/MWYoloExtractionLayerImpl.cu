#include "MWYoloExtractionLayerImpl.hpp"
#include "MWTargetNetworkImpl.hpp"
#include "cnn_api.hpp"
#include "MWKernelHeaders.hpp"
#include <math.h>
#include <cassert>
 MWYoloExtractionLayerImpl::MWYoloExtractionLayerImpl(MWCNNLayer* layer, 
MWTargetNetworkImpl* ntwk_impl, int HoqiuUUuJnXGkfDodicJ) : 
MWCNNLayerImpl(layer, ntwk_impl) , 
dkLDkRwCBjeybwDHbKiE(HoqiuUUuJnXGkfDodicJ) , fSbUUBgjKRbNXrHrlOLo(3) { } 
MWYoloExtractionLayerImpl::~MWYoloExtractionLayerImpl() { } void 
MWYoloExtractionLayerImpl::propagateSize() { } void 
MWYoloExtractionLayerImpl::predict() { MWTensorBase* ipTensorBase = 
getLayer()->getInputTensor(0); MWTensorBase* opTensorBase0 = 
getLayer()->getOutputTensor(0); MWTensorBase* opTensorBase1 = 
getLayer()->getOutputTensor(1); MWTensorBase* opTensorBase2 = 
getLayer()->getOutputTensor(2); MWTensor<float>* ipTensor = 
static_cast<MWTensor<float>*>(ipTensorBase); MWTensor<float>* opTensor0 = 
static_cast<MWTensor<float>*>(opTensorBase0); MWTensor<float>* opTensor1 = 
static_cast<MWTensor<float>*>(opTensorBase1); MWTensor<float>* opTensor2 = 
static_cast<MWTensor<float>*>(opTensorBase2); assert(opTensor0->getData() != 
ipTensor->getData()); if (ipTensor->getBatchSize() == 1) { float* 
UzaGmBLFEwmwaFXebUma = ipTensor->getData(); long int 
jaqKGCwoANNDMHgAsehk = (opTensor0->getHeight()) * 
(opTensor0->getWidth()); long int jHzoRQWaHafftmrmuvHO = 
opTensor0->getChannels() * jaqKGCwoANNDMHgAsehk; 
CUDA_CALL(hipMemcpy(opTensor0->getData(), UzaGmBLFEwmwaFXebUma, 
sizeof(float) * jHzoRQWaHafftmrmuvHO, hipMemcpyDeviceToDevice)); 
long int jLyhrFjMmVnNjoeDJCwH = opTensor1->getChannels() * 
jaqKGCwoANNDMHgAsehk; CUDA_CALL(hipMemcpy(opTensor1->getData(), 
UzaGmBLFEwmwaFXebUma + jHzoRQWaHafftmrmuvHO, sizeof(float) * 
jLyhrFjMmVnNjoeDJCwH, hipMemcpyDeviceToDevice)); long int 
iwclITrbVyVrJaArrXNr = opTensor2->getChannels() * 
jaqKGCwoANNDMHgAsehk; CUDA_CALL(hipMemcpy(opTensor2->getData(), 
UzaGmBLFEwmwaFXebUma + jHzoRQWaHafftmrmuvHO + 
jLyhrFjMmVnNjoeDJCwH, sizeof(float) * 
iwclITrbVyVrJaArrXNr, hipMemcpyDeviceToDevice)); } else { int 
YOWMnLKOMqAODXiVNoGy = ipTensor->getWidth(); long int 
YNmJhGSUszJKxsodxiuV = (ipTensor->getHeight()) * 
(ipTensor->getWidth()); long int YNDVziqpDddiXQKYZZhX = 
YNmJhGSUszJKxsodxiuV * (ipTensor->getChannels()); long int 
YGiQICncmsGZkNUyiQyg = YNDVziqpDddiXQKYZZhX * ipTensor->getBatchSize(); 
long int jHzoRQWaHafftmrmuvHO = YNmJhGSUszJKxsodxiuV * 
(opTensor0->getChannels()); long int jLyhrFjMmVnNjoeDJCwH = 
YNmJhGSUszJKxsodxiuV * (opTensor1->getChannels()); long int 
iwclITrbVyVrJaArrXNr = YNmJhGSUszJKxsodxiuV * 
(opTensor2->getChannels()); long int sFIUeCwGDlfadqOrGZHC = ((YGiQICncmsGZkNUyiQyg + 
31) / 32) * 32; long int tGsvtyAVkrDznETdweDC = (sFIUeCwGDlfadqOrGZHC < 1024) 
? sFIUeCwGDlfadqOrGZHC : 1024; long int KHClOltUSuqFVVErSxVb = 
(YGiQICncmsGZkNUyiQyg + tGsvtyAVkrDznETdweDC - 1) / 
tGsvtyAVkrDznETdweDC; YoloExtractionImpl<<<KHClOltUSuqFVVErSxVb, 
tGsvtyAVkrDznETdweDC>>>( ipTensor->getData(), opTensor0->getData(), 
opTensor1->getData(), opTensor2->getData(), dkLDkRwCBjeybwDHbKiE, 
YOWMnLKOMqAODXiVNoGy, YNmJhGSUszJKxsodxiuV, 
YNDVziqpDddiXQKYZZhX, jHzoRQWaHafftmrmuvHO, 
jLyhrFjMmVnNjoeDJCwH, iwclITrbVyVrJaArrXNr, 
YGiQICncmsGZkNUyiQyg); } return; } void MWYoloExtractionLayerImpl::cleanup() { }