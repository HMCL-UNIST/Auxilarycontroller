#include "MWElementwiseAffineLayerImpl.hpp"
#include "MWTargetNetworkImpl.hpp"
#include "MWKernelHeaders.hpp"
#include "cnn_api.hpp"
#include <math.h>
#include <cassert>
#include <stdio.h>
 MWElementwiseAffineLayerImpl::MWElementwiseAffineLayerImpl(MWCNNLayer* layer, 
MWTargetNetworkImpl* ntwk_impl, int scale_H, int scale_W, int scale_C, int 
offset_H, int offset_W, int offset_C, bool isClipped, int lowerbound, int 
upperbound, const char* qWwjVYwfnvEnFKlgpqwA, const char* 
hvqKUzPqCuUJRfoNlbwW) : MWCNNLayerImpl(layer, ntwk_impl), 
puSFZkRJmyuFPfQRswDK(NULL), gCYwEfkibolsgZAumsuW(NULL), pzUAoBDvaKAtdsmkQuct(scale_H), 
qEXwbWWsnOADJeTXfRVa(scale_W), pvpNsgGssdTxeVoFIkXI(scale_C), 
hKyfKjPACkOBDvLdESxH(offset_H), hnewnpwgzKmOdualajhn(offset_W), 
hDaNSVZAofAENeIAiWEw(offset_C), ZKjSVYDDjACizBkGbqBq(isClipped), 
bOrQjJTNlssnrexxbHdi(lowerbound), veFyKKHbdqBIvQLYBqfF(upperbound) { 
CUDA_CALL(hipMalloc((void**)&puSFZkRJmyuFPfQRswDK, 
sizeof(float)*pzUAoBDvaKAtdsmkQuct*qEXwbWWsnOADJeTXfRVa*pvpNsgGssdTxeVoFIkXI)); 
CUDA_CALL(hipMalloc((void**)&gCYwEfkibolsgZAumsuW, 
sizeof(float)*hKyfKjPACkOBDvLdESxH*hnewnpwgzKmOdualajhn*hDaNSVZAofAENeIAiWEw));  
loadScale(qWwjVYwfnvEnFKlgpqwA); loadOffset(hvqKUzPqCuUJRfoNlbwW); } 
MWElementwiseAffineLayerImpl::~MWElementwiseAffineLayerImpl() { } void 
MWElementwiseAffineLayerImpl::propagateSize() { } void 
MWElementwiseAffineLayerImpl::predict() { MWTensorBase* ipTensorBase = 
getLayer()->getInputTensor(0); MWTensorBase* opTensorBase = 
getLayer()->getOutputTensor(0); MWTensor<float>* ipTensor = 
static_cast<MWTensor<float>*>(ipTensorBase); MWTensor<float>* opTensor = 
static_cast<MWTensor<float>*>(opTensorBase); int WerBmCOBWhvoFbdqfitc = 
ipTensor->getHeight(); int WmXADZOqdcQvtBUvFerh = ipTensor->getWidth(); int 
WOJynDmqVUPWjAGVIuMQ = ipTensor->getChannels(); long int 
YNmJhGSUszJKxsodxiuV = WerBmCOBWhvoFbdqfitc*WmXADZOqdcQvtBUvFerh; long 
int YNDVziqpDddiXQKYZZhX = 
YNmJhGSUszJKxsodxiuV*WOJynDmqVUPWjAGVIuMQ; long int 
YGiQICncmsGZkNUyiQyg = ipTensor->getNumElements(); long int sFIUeCwGDlfadqOrGZHC = 
((YGiQICncmsGZkNUyiQyg + 31) / 32) * 32; int tGsvtyAVkrDznETdweDC = 
(sFIUeCwGDlfadqOrGZHC < 1024) ? sFIUeCwGDlfadqOrGZHC : 1024; long int 
KHClOltUSuqFVVErSxVb = (YGiQICncmsGZkNUyiQyg + tGsvtyAVkrDznETdweDC - 
1) / tGsvtyAVkrDznETdweDC; long int qBTcAwVGZERyCjGYByPe = 
pzUAoBDvaKAtdsmkQuct * qEXwbWWsnOADJeTXfRVa * pvpNsgGssdTxeVoFIkXI; long int 
hljcfGWsvZXJZNrImpJB = hKyfKjPACkOBDvLdESxH * hnewnpwgzKmOdualajhn * 
hDaNSVZAofAENeIAiWEw; assert(qBTcAwVGZERyCjGYByPe <= YGiQICncmsGZkNUyiQyg); 
assert(hljcfGWsvZXJZNrImpJB <= YGiQICncmsGZkNUyiQyg); if (qBTcAwVGZERyCjGYByPe == 
1) { scale_scalar_kernel<<<KHClOltUSuqFVVErSxVb, 
tGsvtyAVkrDznETdweDC>>>( ipTensor->getData(),  opTensor->getData(), 
puSFZkRJmyuFPfQRswDK, YGiQICncmsGZkNUyiQyg); } else if (pzUAoBDvaKAtdsmkQuct == 1 && 
qEXwbWWsnOADJeTXfRVa == 1 && qBTcAwVGZERyCjGYByPe > 1) { 
scale_vector_kernel<<<KHClOltUSuqFVVErSxVb, tGsvtyAVkrDznETdweDC>>>( 
ipTensor->getData(),  opTensor->getData(), puSFZkRJmyuFPfQRswDK, 
YNmJhGSUszJKxsodxiuV, YNDVziqpDddiXQKYZZhX, 
YGiQICncmsGZkNUyiQyg); } else if (YNDVziqpDddiXQKYZZhX == 
qBTcAwVGZERyCjGYByPe) {  scale_tensor3d_kernel<<<KHClOltUSuqFVVErSxVb, 
tGsvtyAVkrDznETdweDC>>>( ipTensor->getData(),  opTensor->getData(), 
puSFZkRJmyuFPfQRswDK,  YNDVziqpDddiXQKYZZhX, YGiQICncmsGZkNUyiQyg); } else 
{ scale_matrix2d_kernel<<<KHClOltUSuqFVVErSxVb, 
tGsvtyAVkrDznETdweDC>>>( ipTensor->getData(),  opTensor->getData(), 
puSFZkRJmyuFPfQRswDK,  YNmJhGSUszJKxsodxiuV, YGiQICncmsGZkNUyiQyg); } if 
(hljcfGWsvZXJZNrImpJB == 1) { offset_scalar_kernel<<<KHClOltUSuqFVVErSxVb, 
tGsvtyAVkrDznETdweDC>>>( opTensor->getData(),  opTensor->getData(), 
gCYwEfkibolsgZAumsuW, YGiQICncmsGZkNUyiQyg, ZKjSVYDDjACizBkGbqBq, 
bOrQjJTNlssnrexxbHdi, veFyKKHbdqBIvQLYBqfF); } else if (hKyfKjPACkOBDvLdESxH 
== 1 && hnewnpwgzKmOdualajhn == 1 && hljcfGWsvZXJZNrImpJB > 1) { 
offset_vector_kernel<<<KHClOltUSuqFVVErSxVb, tGsvtyAVkrDznETdweDC>>>( 
opTensor->getData(),  opTensor->getData(), gCYwEfkibolsgZAumsuW, 
YNmJhGSUszJKxsodxiuV, YNDVziqpDddiXQKYZZhX, 
YGiQICncmsGZkNUyiQyg, ZKjSVYDDjACizBkGbqBq, bOrQjJTNlssnrexxbHdi, 
veFyKKHbdqBIvQLYBqfF); } else if (YNDVziqpDddiXQKYZZhX == 
hljcfGWsvZXJZNrImpJB) { offset_tensor3d_kernel<<<KHClOltUSuqFVVErSxVb, 
tGsvtyAVkrDznETdweDC>>>( opTensor->getData(),  opTensor->getData(), 
gCYwEfkibolsgZAumsuW, YNDVziqpDddiXQKYZZhX, YGiQICncmsGZkNUyiQyg, 
ZKjSVYDDjACizBkGbqBq, bOrQjJTNlssnrexxbHdi, veFyKKHbdqBIvQLYBqfF); } else { 
offset_matrix2d_kernel<<<KHClOltUSuqFVVErSxVb, 
tGsvtyAVkrDznETdweDC>>>( opTensor->getData(),  opTensor->getData(), 
gCYwEfkibolsgZAumsuW, YNmJhGSUszJKxsodxiuV, YGiQICncmsGZkNUyiQyg, 
ZKjSVYDDjACizBkGbqBq, bOrQjJTNlssnrexxbHdi, veFyKKHbdqBIvQLYBqfF); } return; 
} void MWElementwiseAffineLayerImpl::cleanup() { if (puSFZkRJmyuFPfQRswDK) { 
CUDA_FREE_CALL(puSFZkRJmyuFPfQRswDK); puSFZkRJmyuFPfQRswDK = NULL; } if 
(gCYwEfkibolsgZAumsuW) { CUDA_FREE_CALL(gCYwEfkibolsgZAumsuW); gCYwEfkibolsgZAumsuW = 
NULL; }  } void MWElementwiseAffineLayerImpl::loadScale(const char* 
qWwjVYwfnvEnFKlgpqwA) { FILE* QhTesEEIHwhNmHSeYbRR = 
MWCNNLayer::openBinaryFile(qWwjVYwfnvEnFKlgpqwA); assert(QhTesEEIHwhNmHSeYbRR); long 
int dMxIKDGTITyhdLqIHBLA = pzUAoBDvaKAtdsmkQuct*qEXwbWWsnOADJeTXfRVa*pvpNsgGssdTxeVoFIkXI; 
float* KZWeXiYFmdpQdsgidKeG = MALLOC_CALL(sizeof(float)*dMxIKDGTITyhdLqIHBLA); 
call_fread(KZWeXiYFmdpQdsgidKeG, sizeof(float), dMxIKDGTITyhdLqIHBLA, QhTesEEIHwhNmHSeYbRR, 
qWwjVYwfnvEnFKlgpqwA); CUDA_CALL(hipMemcpy(puSFZkRJmyuFPfQRswDK, 
KZWeXiYFmdpQdsgidKeG, sizeof(float)*dMxIKDGTITyhdLqIHBLA, hipMemcpyHostToDevice)); 
free(KZWeXiYFmdpQdsgidKeG); fclose(QhTesEEIHwhNmHSeYbRR);  } void 
MWElementwiseAffineLayerImpl::loadOffset(const char* hvqKUzPqCuUJRfoNlbwW) { 
FILE* QhTesEEIHwhNmHSeYbRR = MWCNNLayer::openBinaryFile(hvqKUzPqCuUJRfoNlbwW); 
assert(QhTesEEIHwhNmHSeYbRR); long int dMxIKDGTITyhdLqIHBLA = 
hKyfKjPACkOBDvLdESxH*hnewnpwgzKmOdualajhn*hDaNSVZAofAENeIAiWEw; float* 
KZWeXiYFmdpQdsgidKeG = MALLOC_CALL(sizeof(float)*dMxIKDGTITyhdLqIHBLA); 
call_fread(KZWeXiYFmdpQdsgidKeG, sizeof(float), dMxIKDGTITyhdLqIHBLA, QhTesEEIHwhNmHSeYbRR, 
hvqKUzPqCuUJRfoNlbwW); CUDA_CALL(hipMemcpy(gCYwEfkibolsgZAumsuW, 
KZWeXiYFmdpQdsgidKeG, sizeof(float)*dMxIKDGTITyhdLqIHBLA, hipMemcpyHostToDevice)); 
free(KZWeXiYFmdpQdsgidKeG); fclose(QhTesEEIHwhNmHSeYbRR);  }