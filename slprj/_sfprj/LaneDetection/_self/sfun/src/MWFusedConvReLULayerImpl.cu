#include "MWFusedConvReLULayer.hpp"
#include "MWFusedConvReLULayerImpl.hpp"
#include <cassert>
#include <stdio.h>
 MWFusedConvReLULayerImpl::MWFusedConvReLULayerImpl(MWCNNLayer* layer, 
MWTargetNetworkImpl* ntwk_impl, int filt_H, int filt_W, int numGrps, int 
numChnls, int numFilts, int FrpxvsDMwwgbpqHXWxmN, int 
FwLnexHgxHRquTKmNpoa, int ClEhcJFlvGCgiavziIag, int 
CZNYmBcNFSZWvaCklqeM, int CufLFODQDXTAPyRqYodN, int 
DCdZnqpcBnvXVgEsLBnz, int AuqaQHxmPQSyYRemQvyX, int 
AzTsxYcYjIEJsGQbeYHm, int euppfEoiaoCTcVgRPVhA, const char* 
xHViLEwTujGGrPZZgmbF, const char* JwxFdqOKggeawILBfGgg) : 
MWCNNLayerImpl(layer, ntwk_impl) , vpXxoeEhdEosLSsYXkNG(NULL) , IwKnaBoXVubIRYcxEJLH(NULL) , 
UEESbUvbMihFnquvuFij(NULL) , WprSrhAStKGxyXeoxETy(NULL) , 
GFggoMvRWucDMqzlWzCl(NULL) , HhKGcPZwrclEFnIdWerH(NULL) , 
BHuHNDGoRwGRouCxeMbw(filt_H) , BLjrjqvCcCommiXWQLjs (filt_W) , 
BuyZFXzwOMxcePIbCLfl (numGrps) , BkwhtPQUCQKchmmimoXs (numChnls) , 
BlRIQPyqJZORKENzSdYf (numFilts) , FpguQZSermqZCMRiUfML(FrpxvsDMwwgbpqHXWxmN) 
, FshVHIJMRAhtQirYPlZd(FwLnexHgxHRquTKmNpoa) , 
CTCbzQMDaLxINPbODdng(ClEhcJFlvGCgiavziIag) , 
CLOUhPjbgggWoXHTtmjC(CZNYmBcNFSZWvaCklqeM) , 
CpMjJjtGOeWOzwxpAAQP(CufLFODQDXTAPyRqYodN) , 
CqtPRJvHlGJFssiPzsOm(DCdZnqpcBnvXVgEsLBnz) , 
AdmgfUbRAfzFeYHxSnQr(AuqaQHxmPQSyYRemQvyX) , 
AwZQzUhuWVLGrWgLHRuM(AzTsxYcYjIEJsGQbeYHm) , 
fvTCtkwXgyScJYogJVFU(euppfEoiaoCTcVgRPVhA) , 
IIiwAtyrOtLzLWAUlTey((CTCbzQMDaLxINPbODdng != CLOUhPjbgggWoXHTtmjC) 
|| (CpMjJjtGOeWOzwxpAAQP != CqtPRJvHlGJFssiPzsOm)) {
#if (CUDNN_MAJOR < 6)
 throw std::runtime_error("Fused ConvReLU Layer only supported for cuDNN 6 or greater");
#else
 dJcdBfQQLhIAYHPxwQeg = ntwk_impl; 
CUDNN_CALL(hipdnnCreateConvolutionDescriptor(&NNhshzQGJHLSGjDiVerE)); 
CUDNN_CALL(hipdnnCreateFilterDescriptor(&QVgVGfoCXYiYXzPhvVPX)); 
CUDNN_CALL(hipdnnCreateTensorDescriptor(&JgLfgHrHMEMmMYTettJF));  
CUDNN_CALL(hipdnnCreateActivationDescriptor(&oJUVMnJggjhEdQLWzIUC)); 
MWTensorBase* ipTensor_conv = getLayer()->getInputTensor(0); int 
NXruhrCCiguRjAgSNDuz = CTCbzQMDaLxINPbODdng; int 
NZjOkZPwLzQsdEVkwMcX = CpMjJjtGOeWOzwxpAAQP; if 
(IIiwAtyrOtLzLWAUlTey) { NXruhrCCiguRjAgSNDuz = 0; 
NZjOkZPwLzQsdEVkwMcX = 0; UEESbUvbMihFnquvuFij = new MWTensor<float>(-1, 
-1, -1, -1, -1, NULL, getLayer(), 0); if (!UEESbUvbMihFnquvuFij) { 
MWCNNLayerImpl::throwAllocationError(__LINE__ , __FILE__); } 
CUDNN_CALL(hipdnnCreateTensorDescriptor(&XYbzSmRQGatVJtGmDZSo)); } else { 
UEESbUvbMihFnquvuFij = ipTensor_conv; } assert(UEESbUvbMihFnquvuFij != NULL); 
bUVPfnrJhLfHzOLUUrKk = CTCbzQMDaLxINPbODdng; cCXqPFPPcoHzYMDpnUxQ = 
CpMjJjtGOeWOzwxpAAQP; MWFusedConvReLULayer* fusedConvReluLayer = 
static_cast<MWFusedConvReLULayer*>(getLayer()); 
CUDNN_CALL(hipdnnSetConvolution2dDescriptor(NNhshzQGJHLSGjDiVerE, 
NXruhrCCiguRjAgSNDuz, NZjOkZPwLzQsdEVkwMcX, FpguQZSermqZCMRiUfML, 
FshVHIJMRAhtQirYPlZd, AdmgfUbRAfzFeYHxSnQr, AwZQzUhuWVLGrWgLHRuM, 
HIPDNN_CROSS_CORRELATION, HIPDNN_DATA_FLOAT));
#if (FP16_ENABLED == 1 && ( CUDNN_MAJOR > 7 || (CUDNN_MAJOR == 7 && CUDNN_MINOR >= 2) ))
 CUDNN_CALL(hipdnnSetConvolutionMathType(NNhshzQGJHLSGjDiVerE, CUDNN_TENSOR_OP_MATH_ALLOW_CONVERSION));
#endif
 if (BuyZFXzwOMxcePIbCLfl > 1){ 
CUDNN_CALL(hipdnnSetConvolutionGroupCount(NNhshzQGJHLSGjDiVerE, 
BuyZFXzwOMxcePIbCLfl)); } 
CUDNN_CALL(hipdnnSetActivationDescriptor(oJUVMnJggjhEdQLWzIUC, 
HIPDNN_ACTIVATION_RELU, HIPDNN_NOT_PROPAGATE_NAN, 0)); int 
etjQLJVQCaeAXRWYtqOl = BkwhtPQUCQKchmmimoXs*BuyZFXzwOMxcePIbCLfl; int 
fSKMHAqIghbYYgyIpNDw = BlRIQPyqJZORKENzSdYf*BuyZFXzwOMxcePIbCLfl; 
CUDNN_CALL(hipdnnSetFilter4dDescriptor(QVgVGfoCXYiYXzPhvVPX, HIPDNN_DATA_FLOAT, 
HIPDNN_TENSOR_NCHW, fSKMHAqIghbYYgyIpNDw, 
etjQLJVQCaeAXRWYtqOl/BuyZFXzwOMxcePIbCLfl, BHuHNDGoRwGRouCxeMbw, 
BLjrjqvCcCommiXWQLjs)); 
CUDNN_CALL(hipdnnSetTensor4dDescriptor(JgLfgHrHMEMmMYTettJF, HIPDNN_TENSOR_NCHW, 
HIPDNN_DATA_FLOAT, 1, fSKMHAqIghbYYgyIpNDw, 1, 1)); int weightSize = 
BkwhtPQUCQKchmmimoXs*fSKMHAqIghbYYgyIpNDw*BHuHNDGoRwGRouCxeMbw*BLjrjqvCcCommiXWQLjs; 
CUDA_CALL(hipMalloc((void**)&vpXxoeEhdEosLSsYXkNG, sizeof(float)*weightSize)); 
CUDA_CALL(hipMalloc((void**)&IwKnaBoXVubIRYcxEJLH, 
sizeof(float)*fSKMHAqIghbYYgyIpNDw)); 
loadWeights(xHViLEwTujGGrPZZgmbF); loadBias(JwxFdqOKggeawILBfGgg); createAndAddDescriptor(getLayer()->getOutputTensor(0)->getSourcePortIndex());
#endif
 } MWFusedConvReLULayerImpl::~MWFusedConvReLULayerImpl() { } void 
MWFusedConvReLULayerImpl::propagateSize() {
#if (CUDNN_MAJOR >= 6)
 MWTensorBase* ipTensor_conv = getLayer()->getInputTensor(0); int inputH; int 
inputW; if (IIiwAtyrOtLzLWAUlTey) { inputH = 
ipTensor_conv->getHeight() + CTCbzQMDaLxINPbODdng + CLOUhPjbgggWoXHTtmjC; 
inputW = ipTensor_conv->getWidth() + CpMjJjtGOeWOzwxpAAQP + 
CqtPRJvHlGJFssiPzsOm; } else { inputH = ipTensor_conv->getHeight(); inputW = 
ipTensor_conv->getWidth(); } UEESbUvbMihFnquvuFij->setHeight(inputH); 
UEESbUvbMihFnquvuFij->setWidth(inputW); 
UEESbUvbMihFnquvuFij->setChannels(ipTensor_conv->getChannels()); 
UEESbUvbMihFnquvuFij->setBatchSize(ipTensor_conv->getBatchSize()); 
UEESbUvbMihFnquvuFij->setSequenceLength(ipTensor_conv->getSequenceLength()); 
assert(UEESbUvbMihFnquvuFij->getSequenceLength() == 1); if 
(IIiwAtyrOtLzLWAUlTey) { 
CUDNN_CALL(hipdnnSetTensor4dDescriptor(XYbzSmRQGatVJtGmDZSo, HIPDNN_TENSOR_NCHW, 
HIPDNN_DATA_FLOAT, UEESbUvbMihFnquvuFij->getBatchSize(), 
UEESbUvbMihFnquvuFij->getChannels(), UEESbUvbMihFnquvuFij->getHeight(), 
UEESbUvbMihFnquvuFij->getWidth())); } else { XYbzSmRQGatVJtGmDZSo = 
MWCNNLayerImpl::getCuDNNDescriptor(UEESbUvbMihFnquvuFij); } 
assert(BkwhtPQUCQKchmmimoXs == 
UEESbUvbMihFnquvuFij->getChannels()/BuyZFXzwOMxcePIbCLfl); MWTensorBase* opTensor 
= getLayer()->getOutputTensor(0); hipdnnTensorDescriptor_t* desc = 
getDescriptor(opTensor->getSourcePortIndex()); assert(desc); 
setDescriptor<float>(*desc, static_cast<MWTensor<float>*>(opTensor));
#if (CUDNN_MAJOR < 7)
 { 
CUDNN_CALL(hipdnnGetConvolutionForwardAlgorithm(*dJcdBfQQLhIAYHPxwQeg->getCudnnHandle(), 
XYbzSmRQGatVJtGmDZSo, QVgVGfoCXYiYXzPhvVPX, NNhshzQGJHLSGjDiVerE, *desc, 
HIPDNN_CONVOLUTION_FWD_PREFER_FASTEST, 0, &NDjzAZSYJuWymuKDNZYB)); }
#else
 { const int maxAlgoCount(3); int returnedAlgoCount(-1); 
hipdnnConvolutionFwdAlgoPerf_t perf_results[maxAlgoCount]; 
CUDNN_CALL(cudnnGetConvolutionForwardAlgorithm_v7(*dJcdBfQQLhIAYHPxwQeg->getCudnnHandle(), 
XYbzSmRQGatVJtGmDZSo, QVgVGfoCXYiYXzPhvVPX, NNhshzQGJHLSGjDiVerE, *desc, 
maxAlgoCount, &returnedAlgoCount, perf_results)); NDjzAZSYJuWymuKDNZYB = 
perf_results[0].algo; }
#endif
 if (CUDNN_VERSION < 7402) fixConvAlgo(); size_t tnTPxeDjBsqLAPkJcPJX = 0; 
CUDNN_CALL(hipdnnGetConvolutionForwardWorkspaceSize(*dJcdBfQQLhIAYHPxwQeg->getCudnnHandle(), 
XYbzSmRQGatVJtGmDZSo, QVgVGfoCXYiYXzPhvVPX, NNhshzQGJHLSGjDiVerE, *desc, 
NDjzAZSYJuWymuKDNZYB, &tnTPxeDjBsqLAPkJcPJX)); if( tnTPxeDjBsqLAPkJcPJX > 
*dJcdBfQQLhIAYHPxwQeg->getProposedWorkSpaceSize() ) { 
dJcdBfQQLhIAYHPxwQeg->setProposedWorkSpaceSize(tnTPxeDjBsqLAPkJcPJX); }
#endif
 } void MWFusedConvReLULayerImpl::allocate() { MWTensorBase* ipTensor_conv = 
getLayer()->getInputTensor(0); if (IIiwAtyrOtLzLWAUlTey) { float* 
newInput; int inputH = ipTensor_conv->getHeight() + CTCbzQMDaLxINPbODdng + 
CLOUhPjbgggWoXHTtmjC; int inputW = ipTensor_conv->getWidth() + 
CpMjJjtGOeWOzwxpAAQP + CqtPRJvHlGJFssiPzsOm; int paddedSize = 
ipTensor_conv->getBatchSize() * ipTensor_conv->getChannels() * inputH * inputW; 
CUDA_CALL(hipMalloc((void**)&newInput, sizeof(float)*paddedSize)); 
CUDA_CALL(hipMemset(newInput, 0, sizeof(float)*paddedSize)); 
static_cast<MWTensor<float>*>(UEESbUvbMihFnquvuFij)->setData(newInput); } 
WprSrhAStKGxyXeoxETy = 
static_cast<MWTensor<float>*>(getLayer()->getOutputTensor(0))->getData(); 
setalpha2Ptr(getZeroPtr()); int numInputs = getLayer()->getNumInputs(); if 
(numInputs == 2) { setalpha2Ptr(getOnePtr()); WprSrhAStKGxyXeoxETy = 
static_cast<MWTensor<float>*>(getLayer()->getInputTensor(1))->getData(); } if 
(static_cast<MWTensor<float>*>(UEESbUvbMihFnquvuFij)->getData() == 
WprSrhAStKGxyXeoxETy){ int xInputTensorSize = 
getLayer()->getInputTensor(0)->getNumElements(); 
CUDA_CALL(hipMalloc((void**)&GFggoMvRWucDMqzlWzCl, sizeof(float) * 
xInputTensorSize)); } } void MWFusedConvReLULayerImpl::deallocate() { if 
(UEESbUvbMihFnquvuFij != getLayer()->getInputTensor(0)) { 
assert(IIiwAtyrOtLzLWAUlTey); 
CUDA_FREE_CALL(static_cast<MWTensor<float>*>(UEESbUvbMihFnquvuFij)->getData()); 
static_cast<MWTensor<float>*>(UEESbUvbMihFnquvuFij)->setData((float*)NULL); } if 
(GFggoMvRWucDMqzlWzCl){ CUDA_FREE_CALL(GFggoMvRWucDMqzlWzCl); 
GFggoMvRWucDMqzlWzCl = NULL;  } } void 
MWFusedConvReLULayerImpl::predict() { MWFusedConvReLULayer* fusedConvReluLayer 
= static_cast<MWFusedConvReLULayer*>(getLayer()); MWTensorBase* ipTensorBase = 
fusedConvReluLayer->getInputTensor(); MWTensorBase* opTensorBase = 
fusedConvReluLayer->getOutputTensor(); MWTensor<float>* ipTensor = 
static_cast<MWTensor<float>*>(ipTensorBase); MWTensor<float>* opTensor = 
static_cast<MWTensor<float>*>(opTensorBase); if (UEESbUvbMihFnquvuFij != 
fusedConvReluLayer->getInputTensor()) { 
CUDA_CALL(hipMemset(static_cast<MWTensor<float>*>(UEESbUvbMihFnquvuFij)->getData(), 
0, sizeof(float)*UEESbUvbMihFnquvuFij->getNumElements())); 
MWCNNLayerImpl::padInput(ipTensor->getData(), ipTensor->getHeight(), 
ipTensor->getWidth(), ipTensor->getChannels(), UEESbUvbMihFnquvuFij->getHeight(), 
UEESbUvbMihFnquvuFij->getWidth(), bUVPfnrJhLfHzOLUUrKk, cCXqPFPPcoHzYMDpnUxQ, 
static_cast<MWTensor<float>*>(UEESbUvbMihFnquvuFij)->getData(), 
ipTensor->getNumElements()); } hipdnnTensorDescriptor_t* desc = 
getDescriptor(opTensor->getSourcePortIndex()); assert(desc);
#if (CUDNN_MAJOR >= 6)
 assert(opTensor->getData() != 
static_cast<MWTensor<float>*>(UEESbUvbMihFnquvuFij)->getData() || 
(getLayer()->getNumInputs() == 2)); float* rIcMzXptfYweLArNRnBw; if 
(static_cast<MWTensor<float>*>(UEESbUvbMihFnquvuFij)->getData() == 
WprSrhAStKGxyXeoxETy){  CUDA_CALL(hipMemcpy(GFggoMvRWucDMqzlWzCl, 
static_cast<MWTensor<float>*>(UEESbUvbMihFnquvuFij)->getData(), sizeof(float) * 
opTensorBase->getNumElements(), hipMemcpyDeviceToDevice)); 
rIcMzXptfYweLArNRnBw = GFggoMvRWucDMqzlWzCl; } else { 
rIcMzXptfYweLArNRnBw = 
static_cast<MWTensor<float>*>(UEESbUvbMihFnquvuFij)->getData(); } 
CUDNN_CALL(cudnnConvolutionBiasActivationForward(*dJcdBfQQLhIAYHPxwQeg->getCudnnHandle(), 
getOnePtr(), XYbzSmRQGatVJtGmDZSo, rIcMzXptfYweLArNRnBw, 
QVgVGfoCXYiYXzPhvVPX, vpXxoeEhdEosLSsYXkNG, NNhshzQGJHLSGjDiVerE, NDjzAZSYJuWymuKDNZYB, 
dJcdBfQQLhIAYHPxwQeg->getWorkSpace(), 
*dJcdBfQQLhIAYHPxwQeg->getAllocatedWorkSpaceSize(), getalpha2Ptr(),  *desc,  
WprSrhAStKGxyXeoxETy,  JgLfgHrHMEMmMYTettJF, IwKnaBoXVubIRYcxEJLH, oJUVMnJggjhEdQLWzIUC, 
*desc, opTensor->getData()));
#endif
 } void MWFusedConvReLULayerImpl::cleanup() { 
CUDNN_CALL(hipdnnDestroyConvolutionDescriptor(NNhshzQGJHLSGjDiVerE)); 
CUDNN_CALL(hipdnnDestroyFilterDescriptor(QVgVGfoCXYiYXzPhvVPX)); 
CUDNN_CALL(hipdnnDestroyActivationDescriptor(oJUVMnJggjhEdQLWzIUC)); if 
(vpXxoeEhdEosLSsYXkNG) { CUDA_FREE_CALL(vpXxoeEhdEosLSsYXkNG); vpXxoeEhdEosLSsYXkNG = NULL; } 
CUDNN_CALL(hipdnnDestroyTensorDescriptor(JgLfgHrHMEMmMYTettJF)); if 
(IwKnaBoXVubIRYcxEJLH) { CUDA_FREE_CALL(IwKnaBoXVubIRYcxEJLH); IwKnaBoXVubIRYcxEJLH = NULL; } if 
(UEESbUvbMihFnquvuFij != getLayer()->getInputTensor(0)) { 
assert(IIiwAtyrOtLzLWAUlTey); 
CUDNN_CALL(hipdnnDestroyTensorDescriptor(XYbzSmRQGatVJtGmDZSo)); } } void 
MWFusedConvReLULayerImpl::loadWeights(const char* QMgBqCuvjnbWHWiVPEwn) { FILE* 
QhTesEEIHwhNmHSeYbRR = MWCNNLayer::openBinaryFile(QMgBqCuvjnbWHWiVPEwn); 
assert(QhTesEEIHwhNmHSeYbRR); int dMxIKDGTITyhdLqIHBLA = 
BkwhtPQUCQKchmmimoXs*BuyZFXzwOMxcePIbCLfl*BlRIQPyqJZORKENzSdYf*BHuHNDGoRwGRouCxeMbw*BLjrjqvCcCommiXWQLjs; 
 float* KZWeXiYFmdpQdsgidKeG = MALLOC_CALL(sizeof(float)*dMxIKDGTITyhdLqIHBLA); 
call_fread(KZWeXiYFmdpQdsgidKeG, sizeof(float), dMxIKDGTITyhdLqIHBLA, QhTesEEIHwhNmHSeYbRR, 
QMgBqCuvjnbWHWiVPEwn); CUDA_CALL(hipMemcpy(vpXxoeEhdEosLSsYXkNG, KZWeXiYFmdpQdsgidKeG, 
sizeof(float)*dMxIKDGTITyhdLqIHBLA, hipMemcpyHostToDevice));
#if 0
 printf("%s loaded. Size = %d. %f\n", QMgBqCuvjnbWHWiVPEwn, dMxIKDGTITyhdLqIHBLA, KZWeXiYFmdpQdsgidKeG[0]);
#endif
 free(KZWeXiYFmdpQdsgidKeG); fclose(QhTesEEIHwhNmHSeYbRR); return; } void 
MWFusedConvReLULayerImpl::loadBias(const char* QMgBqCuvjnbWHWiVPEwn) { FILE* 
QhTesEEIHwhNmHSeYbRR = MWCNNLayer::openBinaryFile(QMgBqCuvjnbWHWiVPEwn); 
assert(QhTesEEIHwhNmHSeYbRR); int dMxIKDGTITyhdLqIHBLA = 
BuyZFXzwOMxcePIbCLfl*BlRIQPyqJZORKENzSdYf;  float* KZWeXiYFmdpQdsgidKeG = 
MALLOC_CALL(sizeof(float)*dMxIKDGTITyhdLqIHBLA); call_fread(KZWeXiYFmdpQdsgidKeG, 
sizeof(float), dMxIKDGTITyhdLqIHBLA, QhTesEEIHwhNmHSeYbRR, QMgBqCuvjnbWHWiVPEwn); 
CUDA_CALL(hipMemcpy(IwKnaBoXVubIRYcxEJLH, KZWeXiYFmdpQdsgidKeG, 
sizeof(float)*dMxIKDGTITyhdLqIHBLA, hipMemcpyHostToDevice)); 
free(KZWeXiYFmdpQdsgidKeG); fclose(QhTesEEIHwhNmHSeYbRR); return; } void 
MWFusedConvReLULayerImpl::postSetup() { if (dJcdBfQQLhIAYHPxwQeg->getAutoTune()) 
{ getConvAlgoTuned(); } else { getConvAlgoWorkSpaceLimit(); } } void 
MWFusedConvReLULayerImpl::getConvAlgoTuned() { MWTensorBase* opTensorBase = 
getLayer()->getOutputTensor(0); MWTensor<float>* opTensor = 
static_cast<MWTensor<float>*>(opTensorBase); hipdnnConvolutionFwdAlgoPerf_t 
perf_results[3]; hipdnnTensorDescriptor_t* desc = 
getDescriptor(getLayer()->getOutputTensor()->getSourcePortIndex()); 
assert(desc); int returnedAlgoCount; 
CUDNN_CALL(hipdnnFindConvolutionForwardAlgorithmEx(*dJcdBfQQLhIAYHPxwQeg->getCudnnHandle(), 
XYbzSmRQGatVJtGmDZSo, 
static_cast<MWTensor<float>*>(UEESbUvbMihFnquvuFij)->getData(), 
QVgVGfoCXYiYXzPhvVPX, vpXxoeEhdEosLSsYXkNG, NNhshzQGJHLSGjDiVerE, *desc, 
opTensor->getData(), 3, &returnedAlgoCount, &perf_results[0], 
dJcdBfQQLhIAYHPxwQeg->getWorkSpace(), 
*dJcdBfQQLhIAYHPxwQeg->getAllocatedWorkSpaceSize())); NDjzAZSYJuWymuKDNZYB = 
perf_results[0].algo; if (CUDNN_VERSION < 7402) fixConvAlgo(); } void 
MWFusedConvReLULayerImpl::getConvAlgoWorkSpaceLimit() { 
hipdnnTensorDescriptor_t* desc = 
getDescriptor(getLayer()->getOutputTensor()->getSourcePortIndex()); assert(desc);
#if (CUDNN_MAJOR < 8)
 
CUDNN_CALL(hipdnnGetConvolutionForwardAlgorithm(*dJcdBfQQLhIAYHPxwQeg->getCudnnHandle(), 
XYbzSmRQGatVJtGmDZSo, QVgVGfoCXYiYXzPhvVPX, NNhshzQGJHLSGjDiVerE, *desc, 
HIPDNN_CONVOLUTION_FWD_SPECIFY_WORKSPACE_LIMIT, 
*dJcdBfQQLhIAYHPxwQeg->getAllocatedWorkSpaceSize(), &NDjzAZSYJuWymuKDNZYB));
#else
 int maxAlgoCount(-1); 
CUDNN_CALL(cudnnGetConvolutionForwardAlgorithmMaxCount(*dJcdBfQQLhIAYHPxwQeg->getCudnnHandle(), 
&maxAlgoCount)); int returnedAlgoCount(-1); 
std::vector<hipdnnConvolutionFwdAlgoPerf_t> perf_results(maxAlgoCount);  
CUDNN_CALL(cudnnGetConvolutionForwardAlgorithm_v7(*dJcdBfQQLhIAYHPxwQeg->getCudnnHandle(), 
XYbzSmRQGatVJtGmDZSo, QVgVGfoCXYiYXzPhvVPX, NNhshzQGJHLSGjDiVerE, *desc, 
maxAlgoCount, &returnedAlgoCount, &perf_results[0])); 
hipdnnConvolutionFwdAlgoPerf_t nextFastest; bool algoFound(false); for (int i = 
0; i < returnedAlgoCount; ++i) { nextFastest = perf_results[i]; if 
(nextFastest.memory <= *dJcdBfQQLhIAYHPxwQeg->getAllocatedWorkSpaceSize()) { 
NDjzAZSYJuWymuKDNZYB = nextFastest.algo; algoFound = true; break; } } assert(algoFound);
#endif
 if (CUDNN_VERSION < 7402) fixConvAlgo(); } void 
MWFusedConvReLULayerImpl::fixConvAlgo() { int inputH = 
UEESbUvbMihFnquvuFij->getHeight(); int inputW = UEESbUvbMihFnquvuFij->getWidth(); 
if (NDjzAZSYJuWymuKDNZYB == HIPDNN_CONVOLUTION_FWD_ALGO_FFT && (inputH > 64 || 
inputW > 64)) { NDjzAZSYJuWymuKDNZYB = HIPDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_GEMM; 
} }