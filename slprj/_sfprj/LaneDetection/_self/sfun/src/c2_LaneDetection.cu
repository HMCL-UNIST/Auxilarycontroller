#include "hip/hip_runtime.h"
/* Include files */

#include "LaneDetection_sfun.h"
#include "c2_LaneDetection.h"
#include "MWCudaDimUtility.hpp"
#ifdef utFree
#undef utFree
#endif

#ifdef utMalloc
#undef utMalloc
#endif

#ifdef __cplusplus

extern "C" void *utMalloc(size_t size);
extern "C" void utFree(void*);

#else

extern void *utMalloc(size_t size);
extern void utFree(void*);

#endif

#include <cstdlib>
#include <cstring>

/* Type Definitions */

/* Named Constants */
const int32_T CALL_EVENT = -1;

/* Variable Declarations */

/* Variable Definitions */
static real_T _sfTime_;

/* Function Declarations */
static void initialize_c2_LaneDetection(SFc2_LaneDetectionInstanceStruct
  *chartInstance);
static void initialize_params_c2_LaneDetection(SFc2_LaneDetectionInstanceStruct *
  chartInstance);
static void enable_c2_LaneDetection(SFc2_LaneDetectionInstanceStruct
  *chartInstance);
static void disable_c2_LaneDetection(SFc2_LaneDetectionInstanceStruct
  *chartInstance);
static void c2_do_animation_call_c2_LaneDetection
  (SFc2_LaneDetectionInstanceStruct *chartInstance);
static void ext_mode_exec_c2_LaneDetection(SFc2_LaneDetectionInstanceStruct
  *chartInstance);
static const mxArray *get_sim_state_c2_LaneDetection
  (SFc2_LaneDetectionInstanceStruct *chartInstance);
static void set_sim_state_c2_LaneDetection(SFc2_LaneDetectionInstanceStruct
  *chartInstance, const mxArray *c2_st);
static void sf_gateway_c2_LaneDetection(SFc2_LaneDetectionInstanceStruct
  *chartInstance);
static void mdl_start_c2_LaneDetection(SFc2_LaneDetectionInstanceStruct
  *chartInstance);
static void mdl_terminate_c2_LaneDetection(SFc2_LaneDetectionInstanceStruct
  *chartInstance);
static void mdl_setup_runtime_resources_c2_LaneDetection
  (SFc2_LaneDetectionInstanceStruct *chartInstance);
static void mdl_cleanup_runtime_resources_c2_LaneDetection
  (SFc2_LaneDetectionInstanceStruct *chartInstance);
static void initSimStructsc2_LaneDetection(SFc2_LaneDetectionInstanceStruct
  *chartInstance);
static void c2_eML_blk_kernel(SFc2_LaneDetectionInstanceStruct *chartInstance,
  real_T c2_b_in_1[154587], real32_T c2_b_out_1[6]);
static void c2_DeepLearningNetwork_setup(SFc2_LaneDetectionInstanceStruct
  *chartInstance, c2_trainedLaneNet0_LaneDetection0 *c2_obj);
static void c2_DeepLearningNetwork_predict(SFc2_LaneDetectionInstanceStruct
  *chartInstance, c2_trainedLaneNet0_LaneDetection0 *c2_obj, real_T
  c2_varargin_1[154587], real32_T c2_varargout_1[6]);
static void c2_emlrt_marshallIn(SFc2_LaneDetectionInstanceStruct *chartInstance,
  const mxArray *c2_b_out_1, const char_T *c2_identifier, real32_T c2_y[6]);
static void c2_b_emlrt_marshallIn(SFc2_LaneDetectionInstanceStruct
  *chartInstance, const mxArray *c2_u, const emlrtMsgIdentifier *c2_parentId,
  real32_T c2_y[6]);
static uint8_T c2_c_emlrt_marshallIn(SFc2_LaneDetectionInstanceStruct
  *chartInstance, const mxArray *c2_b_is_active_c2_LaneDetection, const char_T
  *c2_identifier);
static uint8_T c2_d_emlrt_marshallIn(SFc2_LaneDetectionInstanceStruct
  *chartInstance, const mxArray *c2_u, const emlrtMsgIdentifier *c2_parentId);
static __global__ void c2_eML_blk_kernel_kernel1(const real_T c2_b_in_1[154587],
  real_T c2_b_r_f1[154587]);
static __global__ void c2_DeepLearningNetwork_predict_kernel2(const real_T
  c2_varargin_1[154587], real32_T c2_in[154587], real_T c2_b_r_f1[154587]);
static __global__ void c2_DeepLearningNetwork_predict_kernel3(const real32_T
  c2_in[154587], c2_cell_wrap_8 c2_miniBatchT[1]);
static __global__ void c2_DeepLearningNetwork_predict_kernel4(const
  c2_cell_wrap_10 c2_outputsMiniBatch[1], real32_T c2_varargout_1[6]);
static void c2_checkCleanupCudaError(hipError_t c2_errCode, const char_T
  *c2_file, uint32_T c2_line);
static emlrtRTEInfo c2_createEmlrtInfoStruct(const char_T *c2_file, uint32_T
  c2_line);
static void init_dsm_address_info(SFc2_LaneDetectionInstanceStruct
  *chartInstance);
static void init_simulink_io_address(SFc2_LaneDetectionInstanceStruct
  *chartInstance);

/* Function Definitions */
static void initialize_c2_LaneDetection(SFc2_LaneDetectionInstanceStruct
  *chartInstance)
{
  emlrtLicenseCheckR2012b(chartInstance->c2_fEmlrtCtx,
    "distrib_computing_toolbox", 2);
  emlrtLicenseCheckR2012b(chartInstance->c2_fEmlrtCtx, "neural_network_toolbox",
    2);
  sim_mode_is_external(chartInstance->S);
  _sfTime_ = sf_get_time(chartInstance->S);
  chartInstance->c2_network_not_empty = false;
  chartInstance->c2_is_active_c2_LaneDetection = 0U;
  hipGetLastError();
  hipMalloc(&chartInstance->c2_gpu_r_f1, 1236696UL);
  hipMalloc(&chartInstance->c2_gpu_in_1, 1236696UL);
}

static void initialize_params_c2_LaneDetection(SFc2_LaneDetectionInstanceStruct *
  chartInstance)
{
}

static void enable_c2_LaneDetection(SFc2_LaneDetectionInstanceStruct
  *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
}

static void disable_c2_LaneDetection(SFc2_LaneDetectionInstanceStruct
  *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
}

static void c2_do_animation_call_c2_LaneDetection
  (SFc2_LaneDetectionInstanceStruct *chartInstance)
{
  sfDoAnimationWrapper(chartInstance->S, false, true);
  sfDoAnimationWrapper(chartInstance->S, false, false);
}

static void ext_mode_exec_c2_LaneDetection(SFc2_LaneDetectionInstanceStruct
  *chartInstance)
{
}

static const mxArray *get_sim_state_c2_LaneDetection
  (SFc2_LaneDetectionInstanceStruct *chartInstance)
{
  const mxArray *c2_b_y = NULL;
  const mxArray *c2_c_y = NULL;
  const mxArray *c2_st;
  const mxArray *c2_y = NULL;
  c2_st = NULL;
  c2_y = NULL;
  sf_mex_assign(&c2_y, sf_mex_createcellmatrix(2, 1), false);
  c2_b_y = NULL;
  sf_mex_assign(&c2_b_y, sf_mex_create("y", *chartInstance->c2_out_1, 1, 0U, 1U,
    0U, 2, 1, 6), false);
  sf_mex_setcell(c2_y, 0, c2_b_y);
  c2_c_y = NULL;
  sf_mex_assign(&c2_c_y, sf_mex_create("y",
    &chartInstance->c2_is_active_c2_LaneDetection, 3, 0U, 0U, 0U, 0), false);
  sf_mex_setcell(c2_y, 1, c2_c_y);
  sf_mex_assign(&c2_st, c2_y, false);
  return c2_st;
}

static void set_sim_state_c2_LaneDetection(SFc2_LaneDetectionInstanceStruct
  *chartInstance, const mxArray *c2_st)
{
  const mxArray *c2_u;
  c2_u = sf_mex_dup(c2_st);
  c2_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getcell(c2_u, 0)),
                      "out_1", *chartInstance->c2_out_1);
  chartInstance->c2_is_active_c2_LaneDetection = c2_c_emlrt_marshallIn
    (chartInstance, sf_mex_dup(sf_mex_getcell(c2_u, 1)),
     "is_active_c2_LaneDetection");
  sf_mex_destroy(&c2_u);
  sf_mex_destroy(&c2_st);
}

static void sf_gateway_c2_LaneDetection(SFc2_LaneDetectionInstanceStruct
  *chartInstance)
{
  int32_T c2_i;
  int32_T c2_i1;
  real32_T c2_fv[6];
  chartInstance->c2_JITTransitionAnimation[0] = 0U;
  _sfTime_ = sf_get_time(chartInstance->S);
  for (c2_i = 0; c2_i < 154587; c2_i++) {
    chartInstance->c2_dv[c2_i] = (*chartInstance->c2_in_1)[c2_i];
  }

  c2_eML_blk_kernel(chartInstance, chartInstance->c2_dv, c2_fv);
  for (c2_i1 = 0; c2_i1 < 6; c2_i1++) {
    (*chartInstance->c2_out_1)[c2_i1] = c2_fv[c2_i1];
  }

  c2_do_animation_call_c2_LaneDetection(chartInstance);
}

static void mdl_start_c2_LaneDetection(SFc2_LaneDetectionInstanceStruct
  *chartInstance)
{
}

static void mdl_terminate_c2_LaneDetection(SFc2_LaneDetectionInstanceStruct
  *chartInstance)
{
  hipError_t c2_errCode;
  hipFree(*chartInstance->c2_gpu_in_1);
  hipFree(*chartInstance->c2_gpu_r_f1);
  c2_errCode = hipGetLastError();
  if (c2_errCode != hipSuccess) {
    emlrtThinCUDAError(c2_errCode, hipGetErrorName(c2_errCode),
                       hipGetErrorString(c2_errCode), "SimGPUErrorChecks",
                       chartInstance->c2_fEmlrtCtx);
  }
}

static void mdl_setup_runtime_resources_c2_LaneDetection
  (SFc2_LaneDetectionInstanceStruct *chartInstance)
{
  setLegacyDebuggerFlag(chartInstance->S, false);
  setDebuggerFlag(chartInstance->S, false);
  sim_mode_is_external(chartInstance->S);
}

static void mdl_cleanup_runtime_resources_c2_LaneDetection
  (SFc2_LaneDetectionInstanceStruct *chartInstance)
{
}

static void initSimStructsc2_LaneDetection(SFc2_LaneDetectionInstanceStruct
  *chartInstance)
{
}

const mxArray *sf_c2_LaneDetection_get_eml_resolved_functions_info()
{
  const mxArray *c2_nameCaptureInfo = NULL;
  const char_T *c2_data[42] = {
    "789ced9dd98feb585ec7d3a31ed10d736f173d0cd38c586e0b9ea6e156924a556e7881c449aa92cabe5496ab5197e39c244e1cdbf192ed85f0366f20010f68d8"
    "d10c8c18864602346c7f03affc1d3cf18284b39caac437be4e979d731dfb67a93b6dfd5ce7fbfb9d73fa7c7c56fbde4b65dff3f97ccfb57ffef8777cbee9f3af",
    "f896d733dffa3adbfc7ec5b77be9edef6d7eabba7b7c7dd5f7fecedf2ded7fa8e9fddee69e1178054d95f50d4f0fd1c35fb68521cbd3bc529989c8272159e0c6"
    "a8bdb274580e55d8212a6fdfe49677c3e496e9e166695afe37d543cca0ac0e7d524f7ef490dbbe59e5c7f2fa6f8378df3f303f4606f971a6b3bf4ed513df3957",
    "65e99c13189a3bcf462b9968ecbc14f407fdad734510b896303d977bb484dae76d8e11da48fabc45cb68f7e6b3d57f9f7fc66ab929f15a3a9c40b7e308891944"
    "4b3ccb7773489908d2e0a5a88bf3bb16e3fcd4244e6c1fd20a47b728cdcb2ee2735aa1ab3cdde25041124424292cc26580fd5a58f4eb174cfcc2f655bebddc93",
    "533a7f3e78cf9a3f3f30f107db5fa7a8ef9cf784213aef0d19ee3c2e30ea10f18a8c2b4659154541520a3433a0bb483ea4a2c8eb3f11f19fac9ff9bccd7daed0"
    "521769696f6acf6fefc98673834a84f3e5de20ee8f0ecc17fdefe3f31fac7e7ff38bfff97cf94b4a0f5f5ed19b1aa47768bdfe7903bd339d7d30e98e2a2a956f",
    "86a36cb17417a877e862fffad18f82898e991f3e837b52e97b8517f8fea9713ed3ddebe3c476ad5ddab8906ff57dc005e0027e1eb840420fb8604ffac085c3e2"
    "fc9a499cd8aeb54b9555eb94615bdbfa0b8bfac005e0c253f4f0e5153de0823de97b850b56ebcbcf9ac489ed0f5c88d30abd1a32dcf56361d10fe003f0e1297a",
    "f8f28a1ef0c19ef4bdc207abfd869f368913dbb57629955fce36c8bbfa0b8bfac005e0c253f4f0e5153de0823de97b850b56ebcb374de2c47666b92620294865"
    "345211cf20dc2a031f800febe7810f24f4800ff6a4ef153e58ed3718e5eb99eef7b1dfa041a294cb0117800bebe7810b24f4800bf6a40f5c382cce67ba7b7d9c",
    "d83ed1dc48f1a2aa44a5aeec83f124e0027e1eb840420fb8604ffa5ee1c2bdc5383fd0dd3fc6b9b62c5724d1332401078003ebe7810324f48003f6a4ef150e58"
    "ed1f7c89f9e682d668e5d4e1aefec2a23e7001b8f0143d7c79450fb8604ffac085c3e2fc199338b17d22d162965ead52ddd15f58d4072e00179ea2872fafe801",
    "17ec49df2b5c58588cf35b267162fb98e6d836ade0f5474941a2385a96d9cecc5e7f8013c089a7e8e1cb2b7ac0097bd2f70a27acf61f0e9d77d6daa5151872f4"
    "10c1bc3370e1f179e002093de0823de97b850b0b8b711eda7fd8395f2f234c905411ca8aa4328abdfe002780134fd1c39757f48013f6a40f9c382cce43db6551",
    "42a216c36ae96a999da3e5b6060a8d69ee1df9039c004e6cebe1cb2b7ac0097bd2f70a27ac8e333dd7ddebe3c476f9e1fcd5dcea5076e0027061fd3c7081841e"
    "70c19ef4bdc2857b8b717ed530ceb545e3812afa8003c001fc3c7080841e70c09ef4bdc201abfd830f4de2c4762d36aebc61c2b6fec2a23e7001b8f0143d7c79",
    "450fb8604ffac085c3e23c747fc3860bcba9851dfd85457de00270e1297af8f28a1e70c19ef4810b87c579e87cc2920b25246b64506805e613800b0fcf031748"
    "e80117ec491fb870589c1fe9eef57162fb920b5571b9f36d0d06e0027061fd3c7081841e70c19ef4810b87c579e8773e1589e6655190d11d2d6deb2f2cea0317",
    "800b4fd1c39757f4800bf6a40f5c382c4ea37c3dd3fd8a481aaaca7affc2f21025e0027061fd3c7081841e70c19ef4bdc20552df7fde7021af2a1b30e8fc5858"
    "f403f8007c788a1ebebca2077cb0277de0c361717edd244e6c376a91810fc087f5f3c007127ac0077bd2f70a1fac8e2b1dfabd06696b6dd2b6fec2a2bedd5cf0",
    "59e4c29f99f883edaf53798781e1b18448f2e0b3fffb147870443d523cb899371a8352add9baab0493749fe9cbad5222063c38351e58ad2fdf308913db45cd7f"
    "9651a27c7b7b759273b97066910b3f34f107db9dc785bd2575f0fe78bbdab56f03278eaa478a13179162b03d977ba58b6433c167661d3552c9dd00274e8d13f7",
    "16e334fbde27b3f93a83537960b59ff02726fe60bbf378804b665d2548b557bf0eedff51f548b5ff09617a554cde95fc287cddcfb5ae6e02f3bb38f4134eaefd"
    "b73a6e74e87e679a51d831adb0022fefe82f2cea3b8d077f6ee20fb63b8f075b25a4d512183772871e291e8cab8956393c11a874466d33296ea6348603984738",
    "391edc5b8cf3a70ce35c5b36a310ae6dffbf67e20fb63baffddf94cca64640fbef0e3d52edffa87ccd15a609aa5d2f75a522f26785496a9c84f6ffd4daff85c5"
    "380ffd0ecf723f7361dde22cbfaca0ca8a30ccd03324c9f6fae3b4f9831f9bf883edcee3837189919c67fed1bfff2ebdfcf54afb4d5a8f142fa49b6c5189b523",
    "a1b490998da297dd71aa9381fec2c9f182d4be85add109adf529e5723a3f1616fd70dabad4ef9bf883edcee3c41b25b5aa3330bfe00e3d527ca0d20cea722c57"
    "8c31f5f8a010af7632dd09cc2f001f74cfede703057cd8fc3a9d0f14613ec078d371f548f121179ac9c5b252ef6bb5ba321cdf4cb24121047c38393ee0fba7c6",
    "f94c77af8f13dbc587918b75dfc1ad5cf84b137fb0dd795cd82921e2eb52810bc7d523b51ebd57caab955b3655e0a55059e2a3178af26a7b3d3a70617ffa4ee3"
    "02a9fab21cd58eee195b722a1fbcbb7f616f4901275ca647aaff30cb861896bfab2b0217bb18e7c7ddf8a41e8b0327bcc68943c79776673f61fec1b9e34b6f94",
    "148c2fb9488fd8f812dd134b73f63acd4ea9e93c298bd74a56a0800fa7c687ef5a8cf3d74ce2c4765190958224304896d767eb558412525489a76805750589d5"
    "82dff66b61d12fbb79f189455efcc4c41f6c771e2f0e29b99722a9f60ece553aae1e297ee46fee72d9215d8d4532f15cbf25de06c7374202f8716afcb03a3ff1",
    "5c77af8f13db7747379ccb09abfd8abf32f107db9dc789dd12c21505d635b9438fd478f3558dcd47b9fc4575d011e60d556c340a52ca075cf01a17becc77a237"
    "a3193bfa0b8bfa4edb3777bafba6b74a08f64dbb468f543fa15a28d5f957c97c65926ff673f2ab405251601f84f778f0a1499cd82e2325ca333d41d2e92f2cea",
    "3b8d077f6ae20fb63b8f070f2504fb1e5ca4476cdf03937a356b73d56e1ab506a31b55562bb32ceca3f61c0f0efd2eb4d6da6c5c28b373243b960756c78dfeda"
    "c41f6c77240fb64b88f0f91ac085e3ea91e24298ebcafe0b39a9e4b235a67115a7653e2ec27aa593e3c2c2629c2f4ce2c4769196e4f5d73f654a73b58bf8adb5",
    "93a55ccea99cb03a0ffd4f26fe60bbf338615662704eab9bf44871a3529c2bad2c1d1d5389cb616c30658ad93c82f125e086ee396c7f6b2b4401374e8c1b1469"
    "6ec0bcc471f54871c3df51a5cb89d8ebdee452b154313dbc8d481cac7ff51c377ed1244e6c7fb315dacc896e8ee2702a37aceea7fbc2c41f6c3f056e3c9618b9",
    "f5aec08be3ea91e2c555b6527cd54866a633a678d91ed27d7a52e5e03c0ee085ee39535e948017cbeb84785122c90b18973aae1e295edc46b9ebf22476a55cfa"
    "3ba98694e56a6abc0dfd8b93e385d5faf2b1499cd82e2365d5f62ce750b737603b951356e7bb7f60e20fb63b8f136f9614c9f3c1e1fba2c7d523c58720ad54c2",
    "d1d0fc2a47858581bfc8cc46391afa1327c70762fbafd73b78b75b9e7dc73a3996171ede7f7d40c941ffc22d7aa4f8a1a695fcf46e9251afb3a1122af5fcfe34"
    "3786f55227c78f85c5385f98c489ed7b5aa19d4384dcca8d139ef7362931582fe5263d52fbb363cd344b2955ea8aa61374ae39f7875348f501374e8d1bf8fea9",
    "717ecd244e6c5f8d8a6f35cd4ee584cf2227fec2c41f6c772027b64a08e6b5dda247aa1f2176c6b569b412192646219e91238940621485fd7827c70352ef0f5a"
    "73b4ed0e25f01db6eb732e17bc7b9ef8de9282f3c45da6476cfd93188917b2b73d75c8aae15436df0f29b9ce0d70e2d438b1b018a7ddedf3bd457fded7dd3ffa",
    "b3b6305a4e6eeb919bb776180e0c2a07a9760bce7d3dae1e290e0c26dd5145a5f2cd70942d96ee02f50e5decbb68bfddbdc1df9be5a3fe32ca47afe9fdfe13f5"
    "70fadf36d1c3f6d7d5ede6361a67045e9104ee5c91689647ed0ccd2fc7485e0e6965d7bf7b037dbbfe3ffda30fbfd122a9f793fffad6c724f5f075aaeddea1fd",
    "de49b4556ff46297d579dccf9553cacda53ce27cee69f75e587c2ffa0f83f4cf74f6d7a9c409bc177dca96509795b577eb381211df463cc322793bbf1606f971"
    "6aefd1faebd0761ddea3e13dda8d7af01e6d4ffac0932fc713abf5ee1393fcc2f636b709e225a3b6797efdefa8c8be2b8efcef13f570fa7f60a287ed0438d215",
    "d5b575bba63ce4b7566996597d8e339cf03979bf05e3f047d523c50d140ec462b9311319678afd4081123bbd70db45eb3e7d16b9f13d83f4cf7476e7cdd7890f"
    "67ec6fe7c7c2205ee867bc5d0ffa196fcf17e867bc5b3de867d8933ef0629717a4fa117b3caa2a2c27dbc5ad5f36f103db0db9b5f2e65d9c6be1ac6ab2ca867d",
    "305919c8ad17fd0de87f1c558fd83a20265dae33b5a81f5ddff9e772b0d00b556817ad17059eecf2443488f7d0fc78a6bbd7e707b66bee557976a4a21c3d443e"
    "fbfa3f7673c4b3e75eac41b2534c24fb23b0ffecb87aa4f81118cde3b369b5510976508049c53aad8b48cd45e75e003f76f941ee7c0c246adede214611a4bcaa",
    "94e9a1c8e9ce5ad8f66b61d12fbbb96275beec3f4dfcc176c7d59b35570e293ea2df9b83feca71f548addb6a4a4c33ac3651a7558c2a5c4395722ab7fd7e09bc"
    "d99fbe5779f3ab26f981ed9b062b35d43c7f0b6e5ccb9bc3d76738acdeecf0e66dc547f67c0de0cd71f548f12698e383831637ea46a84aaf920fa7b9eba2e003",
    "deb8953756ebd5cf99e407b66f1aac522e47218ed39a2c5155b6fd5858f4c369f32d7f6fe20fb63bae9eecf065a7b8e07bd92ed223355e5656e8e184a932d504"
    "ba8a24e424952d317558efe55a9e2c0ce23d343f7ec9243fb01d3750c2244bf757e7c9adc75e56c7cdd9e88fdd5cb17a1ee0bf98f883ed8eab2fbb5cd95b6c24",
    "e763a0bf725c3d527ca9fb67379de02bea82be4aa58bc1c1ed2853aac17c3ef0c5203f0efefed11b0dd56aac05e3c5b57cf967137fb0dd71f5c5882f8fc506f3"
    "2f2ed223c59748afc0d3b74995a302178559ab9bbd4cd7f804f005f8b23f3f5e98e407b66f1a2a4ae0d421bfe75dd8a97cb13aeff2af26fe60bbe3eacb0e5f0c",
    "8b8df8f984c099e3ea91e20c93efd4e74288495cbc1a8c86c560a0d6ba40d08f01ce18e4c7af98e407b6ef6db0b6ba324ee58c67bfb7f416ce6c7765802feed0"
    "2335af7fa33499543cc25f28627c1e4c8e6be34c7a7b9c1cf8b23ffd53e50bb1f3f7d70d5596e5d916ad303dfc253fa772c5eabcfe3f98f883ed8eab273b5cd1",
    "159756778027eed023f5ffbd22a692c130c3461ad2b8acc682edb930635c749e3af064972756d7257f6a921fd8be69a0d62b8e8c3e12eb732e5fac8e8ffd9b89"
    "3fd8eeb87ab3c317e3e2c30364c01b77e891e24de6b6af062e0b69aecef58b95bb40841f2b4de08d6b79b33088f748ebc81e1aac9d4f94dae88fd3c6c7dcb18e",
    "6c7fb1c13a32f7e8911a1f1bce2e2b54237fe5cfb5e6b98e9264269742d2077c712b5facd6abaf9be407b66bee51aaac08c30c3d435a332588cede5769757cec"
    "4726fe60bbe3eac9c3f930fae2827d946ed223c593347d35b84554207c9354d2f54248ca846e611fa57b79221ac46bf739634c0f3183542729706dd4f6399723",
    "1e3f676ca798a03fe21e3d52e35d0d3edcecb686acd264e5496630bda495801ff64dba961f04bf7fbe1e3429b3f3e58121abf75c9f733962b53ff263137fb0dd"
    "71f5e4a13ff26671014fdca3478a27f9f0ec3a8a2874d9ae246eb3c1503177a12660fe047862901f1f9be407b66beea5f2ab71922d98f8dccb93bf33f107db1d",
    "574f1e78a22b2ec2dff1029e1c578f144f7aa9693ecaf7fda3c43c9f0e2961999bcc86b05f057862901fdf34c90f6cd7dc2b6bbea3762abf6a9e52bce60c925d"
    "cb932f4cfcc176c7d593079eec2b2ed89fe2163d523c916f036c57be494f03a142b670a14c8bb3de18ced5772d4f48cd9768eee186094d7dceed97787cbe64a7",
    "98607ccb3d7aa4c6b54795423d58a2d237bd54a23bccb1135e2c367cc00fe0c7fefc30caff33ddafe6de72b35c6cb9592ec57704a7f2c36a3fe48726fe60bbe3"
    "eac7033f768a09ce6171991eb1efd3a707574d558d06c6c1162bcfb3c5f6655282712dd772646110afddfb4c96032568a4229e4119c47795dee6687538af787d",
    "39aebe3c8e6f19161bf453dca3478a2fb777fd5a253e8c0de779a424a4aa44453a330af8e256be58eda77ca4bbd7e707b63f2c14daec5c742a4fac8e73fdad89"
    "3fd8eeb8faa15bcf85772a023fdca1476a9c6b4e2b5ca5264523d7ea68242becabf2848af9801f6ee507f175c149442baa8496eb4d299f73fb25b02ef8cde282",
    "fe887bf448f547a886ec97ee62d52b6ada6af4aa5ce93a2066615db06b7942ea7bf6fb1aa83dc7ab38962f2f2cd69b13ff9efd21c507df5371911e29de94e249"
    "7f73fcaa381ff685eb191a6472513101df53712d6f08ae1bded928b769a91ccb175837bcafb8603ccc2d7aa4c62d4ae5666816ab4c1bd430cacd8a816eb8dddb",
    "3e170378b23f7daff2e4d0efd5af0ff0e01f5b2887f7573cfebdfa7dc505dfab77911e299e342337e3c6ddb89bacb25c6ddcbce5a56caae4039eb89527a241bc"
    "87e6c773ddbd3e3fb07db781722e47accecfff8d893fd8eeb8fab18f23f05d14d7e8911adf9adc06fad566bc12a6e7d94e2d5fcb052e151aceed722d3f48bd97",
    "8c698e6dd30ac2ab519dbe6ed8e3f3f37b8b0be6e7dda3478a27e5017d379f0af169ad36ed5ee4b2897848ad004f5ccb13abfd11a3fc3fd3fde2066ab5eb3a2a"
    "75ddca9113dfd7a82f26e2fb1a615cebb87aa438721798777b683abca85dc703bc1abb0dcbf5fa3570c4ad1cb15aaf3e31c90f6c5f0d9844f9764d732a294837",
    "34d759bdeb3a952756f733fea3893fd8eeb87ab235beb5a7b860bfbccbf4488d4bb4e6c2acd093b946fba62ede368b17b95211e6dfddcb958541bcefaa3dbf7f"
    "a23ffacbc81f7c915fb7e5aceab0c6879101d66db9458f547fe48a4997eb4c2dea47d777feb91c2cf44215da05e7acfc3f160e7e55",
    "" };

  c2_nameCaptureInfo = NULL;
  emlrtNameCaptureMxArrayR2016a(&c2_data[0], 134800U, &c2_nameCaptureInfo);
  return c2_nameCaptureInfo;
}

static void c2_eML_blk_kernel(SFc2_LaneDetectionInstanceStruct *chartInstance,
  real_T c2_b_in_1[154587], real32_T c2_b_out_1[6])
{
  hipMemcpy(chartInstance->c2_gpu_in_1, &c2_b_in_1[0], 1236696UL,
             hipMemcpyHostToDevice);
  c2_eML_blk_kernel_kernel1<<<dim3(302U, 1U, 1U), dim3(512U, 1U, 1U)>>>
    (*chartInstance->c2_gpu_in_1, *chartInstance->c2_gpu_r_f1);
  if (!chartInstance->c2_network_not_empty) {
    c2_DeepLearningNetwork_setup(chartInstance, &chartInstance->c2_network);
    chartInstance->c2_network_not_empty = true;
  }

  hipMemcpy(&chartInstance->c2_r_f1[0], chartInstance->c2_gpu_r_f1, 1236696UL,
             hipMemcpyDeviceToHost);
  c2_DeepLearningNetwork_predict(chartInstance, &chartInstance->c2_network,
    chartInstance->c2_r_f1, c2_b_out_1);
}

static void c2_DeepLearningNetwork_setup(SFc2_LaneDetectionInstanceStruct
  *chartInstance, c2_trainedLaneNet0_LaneDetection0 *c2_obj)
{
  c2_obj->setup();
}

static void c2_DeepLearningNetwork_predict(SFc2_LaneDetectionInstanceStruct
  *chartInstance, c2_trainedLaneNet0_LaneDetection0 *c2_obj, real_T
  c2_varargin_1[154587], real32_T c2_varargout_1[6])
{
  c2_cell_wrap_10 (*c2_gpu_outputsMiniBatch)[1];
  c2_cell_wrap_8 (*c2_gpu_miniBatchT)[1];
  real_T (*c2_b_gpu_r_f1)[154587];
  real_T (*c2_gpu_varargin_1)[154587];
  real32_T (*c2_gpu_in)[154587];
  real32_T (*c2_gpu_varargout_1)[6];
  hipMalloc(&c2_b_gpu_r_f1, 1236696UL);
  hipMalloc(&c2_gpu_varargout_1, 24UL);
  hipMalloc(&c2_gpu_outputsMiniBatch, 24UL);
  hipMalloc(&c2_gpu_miniBatchT, 618348UL);
  hipMalloc(&c2_gpu_in, 618348UL);
  hipMalloc(&c2_gpu_varargin_1, 1236696UL);
  hipMemcpy(c2_gpu_varargin_1, &c2_varargin_1[0], 1236696UL,
             hipMemcpyHostToDevice);
  c2_DeepLearningNetwork_predict_kernel2<<<dim3(302U, 1U, 1U), dim3(512U, 1U, 1U)>>>
    (*c2_gpu_varargin_1, *c2_gpu_in, *c2_b_gpu_r_f1);
  c2_DeepLearningNetwork_predict_kernel3<<<dim3(302U, 1U, 1U), dim3(512U, 1U, 1U)>>>
    (*c2_gpu_in, *c2_gpu_miniBatchT);
  hipMemcpy(c2_obj->getInputDataPointer(0), (*c2_gpu_miniBatchT)[0].f1,
             c2_obj->layers[0]->getOutputTensor(0)->getNumElements() * sizeof
             (real32_T), hipMemcpyDeviceToDevice);
  c2_obj->predict();
  hipMemcpy((*c2_gpu_outputsMiniBatch)[0].f1, c2_obj->getLayerOutput(17, 0),
             c2_obj->layers[17]->getOutputTensor(0)->getNumElements() * sizeof
             (real32_T), hipMemcpyDeviceToDevice);
  c2_DeepLearningNetwork_predict_kernel4<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
    (*c2_gpu_outputsMiniBatch, *c2_gpu_varargout_1);
  hipMemcpy(&c2_varargout_1[0], c2_gpu_varargout_1, 24UL,
             hipMemcpyDeviceToHost);
  hipFree(*c2_gpu_varargin_1);
  hipFree(*c2_gpu_in);
  hipFree(*c2_gpu_miniBatchT);
  hipFree(*c2_gpu_outputsMiniBatch);
  hipFree(*c2_gpu_varargout_1);
  hipFree(*c2_b_gpu_r_f1);
}

static void c2_emlrt_marshallIn(SFc2_LaneDetectionInstanceStruct *chartInstance,
  const mxArray *c2_b_out_1, const char_T *c2_identifier, real32_T c2_y[6])
{
  emlrtMsgIdentifier c2_thisId;
  c2_thisId.fIdentifier = const_cast<const char_T *>(c2_identifier);
  c2_thisId.fParent = NULL;
  c2_thisId.bParentIsCell = false;
  c2_b_emlrt_marshallIn(chartInstance, sf_mex_dup(c2_b_out_1), &c2_thisId, c2_y);
  sf_mex_destroy(&c2_b_out_1);
}

static void c2_b_emlrt_marshallIn(SFc2_LaneDetectionInstanceStruct
  *chartInstance, const mxArray *c2_u, const emlrtMsgIdentifier *c2_parentId,
  real32_T c2_y[6])
{
  int32_T c2_i;
  real32_T c2_fv[6];
  sf_mex_import(c2_parentId, sf_mex_dup(c2_u), c2_fv, 0, 1, 0U, 1, 0U, 2, 1, 6);
  for (c2_i = 0; c2_i < 6; c2_i++) {
    c2_y[c2_i] = c2_fv[c2_i];
  }

  sf_mex_destroy(&c2_u);
}

static uint8_T c2_c_emlrt_marshallIn(SFc2_LaneDetectionInstanceStruct
  *chartInstance, const mxArray *c2_b_is_active_c2_LaneDetection, const char_T
  *c2_identifier)
{
  emlrtMsgIdentifier c2_thisId;
  uint8_T c2_y;
  c2_thisId.fIdentifier = const_cast<const char_T *>(c2_identifier);
  c2_thisId.fParent = NULL;
  c2_thisId.bParentIsCell = false;
  c2_y = c2_d_emlrt_marshallIn(chartInstance, sf_mex_dup
    (c2_b_is_active_c2_LaneDetection), &c2_thisId);
  sf_mex_destroy(&c2_b_is_active_c2_LaneDetection);
  return c2_y;
}

static uint8_T c2_d_emlrt_marshallIn(SFc2_LaneDetectionInstanceStruct
  *chartInstance, const mxArray *c2_u, const emlrtMsgIdentifier *c2_parentId)
{
  uint8_T c2_b_u;
  uint8_T c2_y;
  sf_mex_import(c2_parentId, sf_mex_dup(c2_u), &c2_b_u, 1, 3, 0U, 0, 0U, 0);
  c2_y = c2_b_u;
  sf_mex_destroy(&c2_u);
  return c2_y;
}

static __global__ __launch_bounds__(512, 1) void c2_eML_blk_kernel_kernel1(const
  real_T c2_b_in_1[154587], real_T c2_b_r_f1[154587])
{
  int32_T c2_i;
  c2_i = (int32_T)mwGetGlobalThreadIndex();
  if (c2_i < 154587) {
    c2_b_r_f1[c2_i] = c2_b_in_1[c2_i];
  }
}

static __global__ __launch_bounds__(512, 1) void
  c2_DeepLearningNetwork_predict_kernel2(const real_T c2_varargin_1[154587],
  real32_T c2_in[154587], real_T c2_b_r_f1[154587])
{
  int32_T c2_i;
  c2_i = (int32_T)mwGetGlobalThreadIndex();
  if (c2_i < 154587) {
    c2_b_r_f1[c2_i] = c2_varargin_1[c2_i];
    c2_in[c2_i] = (real32_T)c2_b_r_f1[c2_i];
  }
}

static __global__ __launch_bounds__(512, 1) void
  c2_DeepLearningNetwork_predict_kernel3(const real32_T c2_in[154587],
  c2_cell_wrap_8 c2_miniBatchT[1])
{
  uint64_T c2_threadId;
  uint64_T c2_tmpIndex;
  int32_T c2_i1;
  int32_T c2_i2;
  int32_T c2_p;
  c2_threadId = mwGetGlobalThreadIndex();
  c2_i1 = (int32_T)(c2_threadId % 227UL);
  c2_tmpIndex = (c2_threadId - (uint64_T)c2_i1) / 227UL;
  c2_i2 = (int32_T)(c2_tmpIndex % 227UL);
  c2_tmpIndex = (c2_tmpIndex - (uint64_T)c2_i2) / 227UL;
  c2_p = (int32_T)c2_tmpIndex;
  if ((c2_p < 3) && (c2_i2 < 227) && (c2_i1 < 227)) {
    c2_miniBatchT[0].f1[(c2_i1 + 227 * c2_i2) + 51529 * c2_p] = c2_in[(c2_i2 +
      227 * c2_i1) + 51529 * c2_p];
  }
}

static __global__ __launch_bounds__(32, 1) void
  c2_DeepLearningNetwork_predict_kernel4(const c2_cell_wrap_10
  c2_outputsMiniBatch[1], real32_T c2_varargout_1[6])
{
  int32_T c2_i3;
  c2_i3 = (int32_T)mwGetGlobalThreadIndex();
  if (c2_i3 < 6) {
    c2_varargout_1[c2_i3] = c2_outputsMiniBatch[0].f1[c2_i3];
  }
}

void c2_trainedLaneNet0_LaneDetection0::allocate()
{
  int32_T c2_idx;
  this->targetImpl->allocate(290400, 2);
  for (c2_idx = 0; c2_idx < 18; c2_idx++) {
    this->layers[c2_idx]->allocate();
  }

  (static_cast<MWTensor<real32_T> *>(this->inputTensors[0]))->setData
    (this->layers[0]->getLayerOutput(0));
}

void c2_trainedLaneNet0_LaneDetection0::postsetup()
{
  this->targetImpl->postSetup(this->layers, this->numLayers);
}

c2_trainedLaneNet0_LaneDetection0::c2_trainedLaneNet0_LaneDetection0()
{
  this->numLayers = 18;
  this->isInitialized = false;
  this->targetImpl = 0;
  this->layers[0] = new MWInputLayer;
  this->layers[0]->setName("data");
  this->layers[1] = new MWElementwiseAffineLayer;
  this->layers[1]->setName("data_normalization");
  this->layers[1]->setInPlaceIndex(0, 0);
  this->layers[2] = new MWFusedConvReLULayer;
  this->layers[2]->setName("conv1_relu1");
  this->layers[3] = new MWNormLayer;
  this->layers[3]->setName("norm1");
  this->layers[4] = new MWMaxPoolingLayer;
  this->layers[4]->setName("pool1");
  this->layers[5] = new MWFusedConvReLULayer;
  this->layers[5]->setName("conv2_relu2");
  this->layers[6] = new MWNormLayer;
  this->layers[6]->setName("norm2");
  this->layers[7] = new MWMaxPoolingLayer;
  this->layers[7]->setName("pool2");
  this->layers[8] = new MWFusedConvReLULayer;
  this->layers[8]->setName("conv3_relu3");
  this->layers[9] = new MWFusedConvReLULayer;
  this->layers[9]->setName("conv4_relu4");
  this->layers[10] = new MWFusedConvReLULayer;
  this->layers[10]->setName("conv5_relu5");
  this->layers[11] = new MWMaxPoolingLayer;
  this->layers[11]->setName("pool5");
  this->layers[12] = new MWFCLayer;
  this->layers[12]->setName("fc6");
  this->layers[13] = new MWReLULayer;
  this->layers[13]->setName("relu6");
  this->layers[13]->setInPlaceIndex(0, 0);
  this->layers[14] = new MWFCLayer;
  this->layers[14]->setName("fcLane1");
  this->layers[15] = new MWReLULayer;
  this->layers[15]->setName("fcLane1Relu");
  this->layers[15]->setInPlaceIndex(0, 0);
  this->layers[16] = new MWFCLayer;
  this->layers[16]->setName("fcLane2");
  this->layers[17] = new MWOutputLayer;
  this->layers[17]->setName("output");
  this->layers[17]->setInPlaceIndex(0, 0);
  this->targetImpl = new MWTargetNetworkImpl;
  this->inputTensors[0] = new MWTensor<real32_T>;
  this->inputTensors[0]->setHeight(227);
  this->inputTensors[0]->setWidth(227);
  this->inputTensors[0]->setChannels(3);
  this->inputTensors[0]->setBatchSize(1);
  this->inputTensors[0]->setSequenceLength(1);
}

void c2_trainedLaneNet0_LaneDetection0::deallocate()
{
  int32_T c2_idx;
  this->targetImpl->deallocate();
  for (c2_idx = 0; c2_idx < 18; c2_idx++) {
    this->layers[c2_idx]->deallocate();
  }
}

void c2_trainedLaneNet0_LaneDetection0::setSize()
{
  int32_T c2_idx;
  for (c2_idx = 0; c2_idx < 18; c2_idx++) {
    this->layers[c2_idx]->propagateSize();
  }

  this->allocate();
  this->postsetup();
}

void c2_trainedLaneNet0_LaneDetection0::resetState()
{
}

void c2_trainedLaneNet0_LaneDetection0::setup()
{
  if (this->isInitialized) {
    this->resetState();
  } else {
    this->isInitialized = true;
    this->targetImpl->preSetup();
    this->targetImpl->setAutoTune(true);
    (static_cast<MWInputLayer *>(this->layers[0]))->createInputLayer
      (this->targetImpl, this->inputTensors[0], 227, 227, 3, 0, "", 0);
    (static_cast<MWElementwiseAffineLayer *>(this->layers[1]))
      ->createElementwiseAffineLayer(this->targetImpl, this->layers[0]
      ->getOutputTensor(0), 227, 227, 3, 227, 227, 3, false, 1, 1,
      "/home/hmcl/ADcontrol/slprj/_sfprj/LaneDetection/_self/sfun/src/cnn_trainedLaneNet0_LaneDetection0_data_scale.bin",
      "/home/hmcl/ADcontrol/slprj/_sfprj/LaneDetection/_self/sfun/src/cnn_trainedLaneNet0_LaneDetection0_data_offset.bin",
      0);
    (static_cast<MWFusedConvReLULayer *>(this->layers[2]))
      ->createFusedConvReLULayer(this->targetImpl, 1, this->layers[1]
      ->getOutputTensor(0), 11, 11, 3, 96, 4, 4, 0, 0, 0, 0, 1, 1, 1,
      "/home/hmcl/ADcontrol/slprj/_sfprj/LaneDetection/_self/sfun/src/cnn_trainedLaneNet0_LaneDetection0_conv1_w.bin",
      "/home/hmcl/ADcontrol/slprj/_sfprj/LaneDetection/_self/sfun/src/cnn_trainedLaneNet0_LaneDetection0_conv1_b.bin",
      1);
    (static_cast<MWNormLayer *>(this->layers[3]))->createNormLayer
      (this->targetImpl, this->layers[2]->getOutputTensor(0), 5, 0.0001, 0.75,
       1.0, 0);
    (static_cast<MWMaxPoolingLayer *>(this->layers[4]))->createMaxPoolingLayer
      (this->targetImpl, this->layers[3]->getOutputTensor(0), 3, 3, 2, 2, 0, 0,
       0, 0, 0, 1, 1);
    (static_cast<MWFusedConvReLULayer *>(this->layers[5]))
      ->createFusedConvReLULayer(this->targetImpl, 1, this->layers[4]
      ->getOutputTensor(0), 5, 5, 48, 128, 1, 1, 2, 2, 2, 2, 1, 1, 2,
      "/home/hmcl/ADcontrol/slprj/_sfprj/LaneDetection/_self/sfun/src/cnn_trainedLaneNet0_LaneDetection0_conv2_w.bin",
      "/home/hmcl/ADcontrol/slprj/_sfprj/LaneDetection/_self/sfun/src/cnn_trainedLaneNet0_LaneDetection0_conv2_b.bin",
      0);
    (static_cast<MWNormLayer *>(this->layers[6]))->createNormLayer
      (this->targetImpl, this->layers[5]->getOutputTensor(0), 5, 0.0001, 0.75,
       1.0, 1);
    (static_cast<MWMaxPoolingLayer *>(this->layers[7]))->createMaxPoolingLayer
      (this->targetImpl, this->layers[6]->getOutputTensor(0), 3, 3, 2, 2, 0, 0,
       0, 0, 0, 1, 0);
    (static_cast<MWFusedConvReLULayer *>(this->layers[8]))
      ->createFusedConvReLULayer(this->targetImpl, 1, this->layers[7]
      ->getOutputTensor(0), 3, 3, 256, 384, 1, 1, 1, 1, 1, 1, 1, 1, 1,
      "/home/hmcl/ADcontrol/slprj/_sfprj/LaneDetection/_self/sfun/src/cnn_trainedLaneNet0_LaneDetection0_conv3_w.bin",
      "/home/hmcl/ADcontrol/slprj/_sfprj/LaneDetection/_self/sfun/src/cnn_trainedLaneNet0_LaneDetection0_conv3_b.bin",
      1);
    (static_cast<MWFusedConvReLULayer *>(this->layers[9]))
      ->createFusedConvReLULayer(this->targetImpl, 1, this->layers[8]
      ->getOutputTensor(0), 3, 3, 192, 192, 1, 1, 1, 1, 1, 1, 1, 1, 2,
      "/home/hmcl/ADcontrol/slprj/_sfprj/LaneDetection/_self/sfun/src/cnn_trainedLaneNet0_LaneDetection0_conv4_w.bin",
      "/home/hmcl/ADcontrol/slprj/_sfprj/LaneDetection/_self/sfun/src/cnn_trainedLaneNet0_LaneDetection0_conv4_b.bin",
      0);
    (static_cast<MWFusedConvReLULayer *>(this->layers[10]))
      ->createFusedConvReLULayer(this->targetImpl, 1, this->layers[9]
      ->getOutputTensor(0), 3, 3, 192, 128, 1, 1, 1, 1, 1, 1, 1, 1, 2,
      "/home/hmcl/ADcontrol/slprj/_sfprj/LaneDetection/_self/sfun/src/cnn_trainedLaneNet0_LaneDetection0_conv5_w.bin",
      "/home/hmcl/ADcontrol/slprj/_sfprj/LaneDetection/_self/sfun/src/cnn_trainedLaneNet0_LaneDetection0_conv5_b.bin",
      1);
    (static_cast<MWMaxPoolingLayer *>(this->layers[11]))->createMaxPoolingLayer
      (this->targetImpl, this->layers[10]->getOutputTensor(0), 3, 3, 2, 2, 0, 0,
       0, 0, 0, 1, 0);
    (static_cast<MWFCLayer *>(this->layers[12]))->createFCLayer(this->targetImpl,
      this->layers[11]->getOutputTensor(0), 9216, 4096,
      "/home/hmcl/ADcontrol/slprj/_sfprj/LaneDetection/_self/sfun/src/cnn_trainedLaneNet0_LaneDetection0_fc6_w.bin",
      "/home/hmcl/ADcontrol/slprj/_sfprj/LaneDetection/_self/sfun/src/cnn_trainedLaneNet0_LaneDetection0_fc6_b.bin",
      1);
    (static_cast<MWReLULayer *>(this->layers[13]))->createReLULayer
      (this->targetImpl, this->layers[12]->getOutputTensor(0), 1);
    (static_cast<MWFCLayer *>(this->layers[14]))->createFCLayer(this->targetImpl,
      this->layers[13]->getOutputTensor(0), 4096, 16,
      "/home/hmcl/ADcontrol/slprj/_sfprj/LaneDetection/_self/sfun/src/cnn_trainedLaneNet0_LaneDetection0_fcLane1_w.bin",
      "/home/hmcl/ADcontrol/slprj/_sfprj/LaneDetection/_self/sfun/src/cnn_trainedLaneNet0_LaneDetection0_fcLane1_b.bin",
      0);
    (static_cast<MWReLULayer *>(this->layers[15]))->createReLULayer
      (this->targetImpl, this->layers[14]->getOutputTensor(0), 0);
    (static_cast<MWFCLayer *>(this->layers[16]))->createFCLayer(this->targetImpl,
      this->layers[15]->getOutputTensor(0), 16, 6,
      "/home/hmcl/ADcontrol/slprj/_sfprj/LaneDetection/_self/sfun/src/cnn_trainedLaneNet0_LaneDetection0_fcLane2_w.bin",
      "/home/hmcl/ADcontrol/slprj/_sfprj/LaneDetection/_self/sfun/src/cnn_trainedLaneNet0_LaneDetection0_fcLane2_b.bin",
      1);
    (static_cast<MWOutputLayer *>(this->layers[17]))->createOutputLayer
      (this->targetImpl, this->layers[16]->getOutputTensor(0), 1);
    this->outputTensors[0] = this->layers[17]->getOutputTensor(0);
    this->setSize();
  }
}

void c2_trainedLaneNet0_LaneDetection0::predict()
{
  int32_T c2_idx;
  for (c2_idx = 0; c2_idx < 18; c2_idx++) {
    this->layers[c2_idx]->predict();
  }
}

void c2_trainedLaneNet0_LaneDetection0::cleanup()
{
  int32_T c2_idx;
  this->deallocate();
  for (c2_idx = 0; c2_idx < 18; c2_idx++) {
    this->layers[c2_idx]->cleanup();
  }

  if (this->targetImpl) {
    this->targetImpl->cleanup();
  }
}

real32_T *c2_trainedLaneNet0_LaneDetection0::getLayerOutput(int32_T
  c2_layerIndex, int32_T c2_portIndex)
{
  return this->layers[c2_layerIndex]->getLayerOutput(c2_portIndex);
}

real32_T *c2_trainedLaneNet0_LaneDetection0::getInputDataPointer(int32_T
  c2_index)
{
  return (static_cast<MWTensor<real32_T> *>(this->inputTensors[c2_index]))
    ->getData();
}

real32_T *c2_trainedLaneNet0_LaneDetection0::getInputDataPointer()
{
  return (static_cast<MWTensor<real32_T> *>(this->inputTensors[0]))->getData();
}

real32_T *c2_trainedLaneNet0_LaneDetection0::getOutputDataPointer(int32_T
  c2_index)
{
  return (static_cast<MWTensor<real32_T> *>(this->outputTensors[c2_index]))
    ->getData();
}

real32_T *c2_trainedLaneNet0_LaneDetection0::getOutputDataPointer()
{
  return (static_cast<MWTensor<real32_T> *>(this->outputTensors[0]))->getData();
}

int32_T c2_trainedLaneNet0_LaneDetection0::getBatchSize()
{
  return this->inputTensors[0]->getBatchSize();
}

c2_trainedLaneNet0_LaneDetection0::~c2_trainedLaneNet0_LaneDetection0()
{
  int32_T c2_idx;
  this->cleanup();
  c2_checkCleanupCudaError(hipGetLastError(), __FILE__, __LINE__);
  for (c2_idx = 0; c2_idx < 18; c2_idx++) {
    delete this->layers[c2_idx];
  }

  if (this->targetImpl) {
    delete this->targetImpl;
  }

  delete this->inputTensors[0];
}

static void c2_checkCleanupCudaError(hipError_t c2_errCode, const char_T
  *c2_file, uint32_T c2_line)
{
  emlrtRTEInfo c2_rtInfo;
  if ((c2_errCode != hipSuccess) && (c2_errCode != hipErrorDeinitialized)) {
    c2_rtInfo = c2_createEmlrtInfoStruct(c2_file, c2_line);
    emlrtCUDAWarning(c2_errCode, hipGetErrorName(c2_errCode),
                     hipGetErrorString(c2_errCode), &c2_rtInfo);
  }
}

static emlrtRTEInfo c2_createEmlrtInfoStruct(const char_T *c2_file, uint32_T
  c2_line)
{
  emlrtRTEInfo c2_b_rtInfo;
  uint32_T c2_len;
  char_T *c2_brk;
  char_T *c2_fn;
  char_T *c2_pn;
  c2_len = (uint32_T)strlen(c2_file);
  c2_pn = (char_T *)calloc(c2_len + 1U, 1U);
  c2_fn = (char_T *)calloc(c2_len + 1U, 1U);
  memcpy(c2_pn, c2_file, c2_len);
  memcpy(c2_fn, c2_file, c2_len);
  c2_brk = strrchr(c2_fn, '.');
  *c2_brk = '\x00';
  c2_brk = NULL;
  c2_brk = strrchr(c2_fn, '/');
  if (c2_brk == NULL) {
    c2_brk = strrchr(c2_fn, '\\');
  }

  if (c2_brk == NULL) {
    c2_brk = c2_fn;
  } else {
    c2_brk++;
  }

  c2_b_rtInfo.lineNo = c2_line;
  c2_b_rtInfo.colNo = 0;
  c2_b_rtInfo.fName = c2_brk;
  c2_b_rtInfo.pName = c2_pn;
  return c2_b_rtInfo;
}

static void init_dsm_address_info(SFc2_LaneDetectionInstanceStruct
  *chartInstance)
{
}

static void init_simulink_io_address(SFc2_LaneDetectionInstanceStruct
  *chartInstance)
{
  chartInstance->c2_fEmlrtCtx = (void *)sfrtGetEmlrtCtx(chartInstance->S);
  chartInstance->c2_in_1 = (real_T (*)[154587])ssGetInputPortSignal_wrapper
    (chartInstance->S, 0);
  chartInstance->c2_out_1 = (real32_T (*)[6])ssGetOutputPortSignal_wrapper
    (chartInstance->S, 1);
}

#include <stdio.h>
#include <stdlib.h>
#include <string.h>

/* SFunction Glue Code */
void sf_c2_LaneDetection_get_check_sum(mxArray *plhs[])
{
  ((real_T *)mxGetPr((plhs[0])))[0] = (real_T)(2645480056U);
  ((real_T *)mxGetPr((plhs[0])))[1] = (real_T)(3584261826U);
  ((real_T *)mxGetPr((plhs[0])))[2] = (real_T)(751965951U);
  ((real_T *)mxGetPr((plhs[0])))[3] = (real_T)(2291345985U);
}

mxArray *sf_c2_LaneDetection_third_party_uses_info(void)
{
  mxArray * mxcell3p = mxCreateCellMatrix(1,1);
  mxSetCell(mxcell3p, 0, mxCreateString("dltargets.cudnn.cudnnApi"));
  return(mxcell3p);
}

mxArray *sf_c2_LaneDetection_jit_fallback_info(void)
{
  const char *infoFields[] = { "fallbackType", "fallbackReason",
    "hiddenFallbackType", "hiddenFallbackReason", "incompatibleSymbol" };

  mxArray *mxInfo = mxCreateStructMatrix(1, 1, 5, infoFields);
  mxArray *fallbackType = mxCreateString("pre");
  mxArray *fallbackReason = mxCreateString("GPUAcceleration");
  mxArray *hiddenFallbackType = mxCreateString("late");
  mxArray *hiddenFallbackReason = mxCreateString("ir_function_calls");
  mxArray *incompatibleSymbol = mxCreateString("#__setup__");
  mxSetField(mxInfo, 0, infoFields[0], fallbackType);
  mxSetField(mxInfo, 0, infoFields[1], fallbackReason);
  mxSetField(mxInfo, 0, infoFields[2], hiddenFallbackType);
  mxSetField(mxInfo, 0, infoFields[3], hiddenFallbackReason);
  mxSetField(mxInfo, 0, infoFields[4], incompatibleSymbol);
  return mxInfo;
}

mxArray *sf_c2_LaneDetection_updateBuildInfo_args_info(void)
{
  mxArray *mxBIArgs = sf_mex_decode(
    "eNpjYPT0ZQACPiDm4GRgYAPRQMzIAAGsSHxOJHGQ+h8M+NUzoal3QFLPgkU9H5J6ASg/MSUlOL+"
    "0KDnVLTMntRgi1kDAXkY0ewMI2CuDZi+I7xvumpOam5pXUp5ZnOqYlpaZl+qTWJlapJdcUAAz98"
    "MA+b+CRP97ELBXAs1eCbD/3UqLU1Oc8/PKglJ9QuF+B5n3YoD8XUCivwnZy41mL4ifnJcXn1iQC"
    "YvmYRXPgmj2CoLj2dnPDxy7nrkFOXrJpcMwf8uj2SuPJ3/DQgFkLkjTQPi/g0T/RxCwVxXNXlUC"
    "/vdOLcpLhaaF4ZQOpNHslcZRziHnhOGU/8XQ7BUD+z8ksSg9tcQvtaQ8vygb7vXhFO+SaPZKQsq"
    "90uKS/FxwhLvlFzmXuvj5wUt9BgYA8ZCiqg=="
    );
  return mxBIArgs;
}

static const mxArray *sf_get_sim_state_info_c2_LaneDetection(void)
{
  const char *infoFields[] = { "chartChecksum", "varInfo" };

  mxArray *mxInfo = mxCreateStructMatrix(1, 1, 2, infoFields);
  mxArray *mxVarInfo = sf_mex_decode(
    "eNpjYPT0ZQACPhDBxMDABqQ4IEwwYIXyGaFijHBxFri4AhCXVBakgsSLi5I9U4B0XmIumJ9YWuG"
    "Zl5YPNt+CAWE+GxbzGZHM54SKQ8AHe8r0izig62fBop8VSb8AlJ9fWhJvCAsfKE2+OxQcKNMPsT"
    "+AgD+k0PwB4mcWxycml2SWpcYnG8X7JOaluqSWpAIF8vMQ5oIAAB9fGm0="
    );
  mxArray *mxChecksum = mxCreateDoubleMatrix(1, 4, mxREAL);
  sf_c2_LaneDetection_get_check_sum(&mxChecksum);
  mxSetField(mxInfo, 0, infoFields[0], mxChecksum);
  mxSetField(mxInfo, 0, infoFields[1], mxVarInfo);
  return mxInfo;
}

static const char* sf_get_instance_specialization(void)
{
  return "soqOeZDjTPcCrKLUGh7SS3D";
}

static void sf_opaque_initialize_c2_LaneDetection(void *chartInstanceVar)
{
  initialize_params_c2_LaneDetection((SFc2_LaneDetectionInstanceStruct*)
    chartInstanceVar);
  initialize_c2_LaneDetection((SFc2_LaneDetectionInstanceStruct*)
    chartInstanceVar);
}

static void sf_opaque_enable_c2_LaneDetection(void *chartInstanceVar)
{
  enable_c2_LaneDetection((SFc2_LaneDetectionInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_disable_c2_LaneDetection(void *chartInstanceVar)
{
  disable_c2_LaneDetection((SFc2_LaneDetectionInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_gateway_c2_LaneDetection(void *chartInstanceVar)
{
  sf_gateway_c2_LaneDetection((SFc2_LaneDetectionInstanceStruct*)
    chartInstanceVar);
}

static const mxArray* sf_opaque_get_sim_state_c2_LaneDetection(SimStruct* S)
{
  return get_sim_state_c2_LaneDetection((SFc2_LaneDetectionInstanceStruct *)
    sf_get_chart_instance_ptr(S));     /* raw sim ctx */
}

static void sf_opaque_set_sim_state_c2_LaneDetection(SimStruct* S, const mxArray
  *st)
{
  set_sim_state_c2_LaneDetection((SFc2_LaneDetectionInstanceStruct*)
    sf_get_chart_instance_ptr(S), st);
}

static void sf_opaque_cleanup_runtime_resources_c2_LaneDetection(void
  *chartInstanceVar)
{
  if (chartInstanceVar!=NULL) {
    SimStruct *S = ((SFc2_LaneDetectionInstanceStruct*) chartInstanceVar)->S;
    if (sim_mode_is_rtw_gen(S) || sim_mode_is_external(S)) {
      sf_clear_rtw_identifier(S);
      unload_LaneDetection_optimization_info();
    }

    mdl_cleanup_runtime_resources_c2_LaneDetection
      ((SFc2_LaneDetectionInstanceStruct*) chartInstanceVar);
    ((SFc2_LaneDetectionInstanceStruct*) chartInstanceVar)->
      ~SFc2_LaneDetectionInstanceStruct();
    utFree(chartInstanceVar);
    if (ssGetUserData(S)!= NULL) {
      sf_free_ChartRunTimeInfo(S);
    }

    ssSetUserData(S,NULL);
  }
}

static void sf_opaque_mdl_start_c2_LaneDetection(void *chartInstanceVar)
{
  mdl_start_c2_LaneDetection((SFc2_LaneDetectionInstanceStruct*)
    chartInstanceVar);
}

static void sf_opaque_mdl_terminate_c2_LaneDetection(void *chartInstanceVar)
{
  mdl_terminate_c2_LaneDetection((SFc2_LaneDetectionInstanceStruct*)
    chartInstanceVar);
}

static void sf_opaque_init_subchart_simstructs(void *chartInstanceVar)
{
  initSimStructsc2_LaneDetection((SFc2_LaneDetectionInstanceStruct*)
    chartInstanceVar);
}

extern unsigned int sf_machine_global_initializer_called(void);
static void mdlProcessParameters_c2_LaneDetection(SimStruct *S)
{
  int i;
  for (i=0;i<ssGetNumRunTimeParams(S);i++) {
    if (ssGetSFcnParamTunable(S,i)) {
      ssUpdateDlgParamAsRunTimeParam(S,i);
    }
  }

  sf_warn_if_symbolic_dimension_param_changed(S);
  if (sf_machine_global_initializer_called()) {
    initialize_params_c2_LaneDetection((SFc2_LaneDetectionInstanceStruct*)
      sf_get_chart_instance_ptr(S));
    initSimStructsc2_LaneDetection((SFc2_LaneDetectionInstanceStruct*)
      sf_get_chart_instance_ptr(S));
  }
}

const char* sf_c2_LaneDetection_get_post_codegen_info(void)
{
  int i;
  const char* encStrCodegen [31] = {
    "eNrtXE9vG0UUd6K0ohJUPSAhVSB6QXBAOLFd4ARt7BgskiaqkyJxMbOz4+zUuzObmVm7UU/0z2f",
    "gypFjDxz4CHwETggJiTPizIE3a8dxxrt2nUDsZ3WlzWZ337z9zfs3b2bfurDS2CnAdh323/1C4S",
    "ocX4N9tdDfrgzOV0b2/vW1wvuD85+gkUiiPaJIpAsTN0Eidp9pGSaGS9EQbZlJxkWbKSYo0MZSm",
    "TxumkdJyEWnnghq+emvA06DZiCT0N+EtsTfFeExcIsTswd8alwxauqM+SZQMjkM6iE5HCJWplcN",
    "GO3oJJrUBc1MM4ktLL2ThIbHIdt6xGhDaEMAsT7F1jTEsKp5lNtN21PdPCGUURxyIjJ7GxDdZDE",
    "I2LCD2Ie/u4mBTrlkNCDKbLKAdJne5p2UpxTM5ck13PC4IEYqTsKtKKzahuPY9kLAsyN9Fk4QCG",
    "DbVIx0YsmFydd/sw493RLEC1mNeclhPrcmO0qs8h9w1mMqV27tquwyRQ7Zrsh9aCqQrUeptoZWM",
    "k5meMQeEHWXgv4083OtFyxHNwnoie1DizwylnayofcV74J4c7klUcNa5jSXSaK+svU0spTbVpdN",
    "0sKQW52KKglDnUu2L+Nt1mVhyrVGDJlM1ueaTac19/clCNiad743JIKD4gdkVSl8nqmurkOQxp1",
    "7EFjOUtJEGxlVwXhr29vjt8fJGsIw1SaUZUUBRbhmILNUvPncfK6t7oEQUJkUXhZx30KmURV0Ox",
    "G1nlQdkMmEIHLaBavRXMJIH4IuwRMONDjNJDKry2l0lNCA+TbA8JDtgNsAbYZMtA1td8Hvutwc1",
    "5imiscZWk3A6yAMbVmDOo7ZgegI2RN1JaPmIMb3tQCRAWJ4BDrYT31MUGDFtYFwwU8f7zMGRkmU",
    "4OJwE8KcOq4DyEyN2XFvvXA67r3+EuPeSTv3+MEIn5UMPoWRo/vca6uTn7sK/60M2t0ZafeG85w",
    "1p52luwH7979Ef3xIwm8b7/3zzp9/V3+7yPNfrM6WJ1wfnL99EpCHDtYds2tL++UIrrUM/m+N8L",
    "8xONfyaJd9U3u4v0er6qvtgy+CT5rNcq2vnyl4Vx28J9dv2ZEBrDG1Y0Ub/iCBseck6Q/rlv+nI",
    "3ivTpHHtcH1/vbX5xdr/+Ydt32WvK448rLnMjGtDcd+z4/j1p2Lte8/f29KP246/biZ5gEtYqML",
    "a9FSa5sIyC0MS4f5cf+c1d5ftXvVbpHbrZxz/LrsdoVLbnfR/s06ji8a/fqEOFpw6G8scD8uml/",
    "93/S/FmbLg94dnH82nBJVAx76Gdnx4DbksO2su0tip/H9yfSxQ//zlPzgO8eu7XkxkBErBhENi3",
    "drMFkwSoZFHcbqYbGl2/ZwJmmAiyxsF+3EqwgpX5EK0TqWoeyW7jN9j5nb6w9YwKld8kmXAdbP5",
    "hzrA2KYm3ZLLe8jjwu0+DdavRnxP3fwP58f/pBobTsx7ANW/N4M+J85+J/NAb9iutJueYoIGpTo",
    "qQkhxe9hxM+Qy3+I30OPH7f9EOT+i91+CE778ZHHTx+5/fvI7YciH78ocvuny2L/Hnr5o8TvIfd",
    "fD7n/esjzHw+3/5Y83PFzFD9K+ZNlkb/jvz9MwS8d/PK/wW8U4YL5lg56MIa4Te2FjYGpYJIzRR",
    "4nyTn99KmD/+mc/XRjZKV80e28hM/OyyRnPH0xBf9jB//jOeCnzpsIlOPQjHnw4vkncvljnwd6y",
    "POwGedRTxz8T+aAP/1moLXRku22Zub0RSjKuI8zjy+TZXmPiHQdhyBfB2nnxM9p+WXk4I8uJb+k",
    "Z2pGMPnpedeLF6He5STOa0pC5tS7LK6dVGayk0Wri+qhkXN5pvqhRfFHb1nGfZzrd2X/nOt387P",
    "z22fsfJHHR4T+WEL+PqzsL896uodnPb03XGdc5PHRQxE3bqPMq71lqYNBMw5uYLSTUl69Ap71fQ",
    "/j+nKZLsv6CMHin5Uz+d+PU3AeOTiPLgWnTwwZWzZ+Kbl2HLydS8o3Pp7x/ev8cPawx4ke8u9JP",
    "OT1oCjnL8jrQUvI64nLSyN/Dzl+ih4/xjqQSl6dFhL85eXBjzP/IbjH3zLy9+Bl5PXoZeTfA4x+",
    "D9lDXcflIc/fPOT+i1/+2N9/edjqW0Z/z2XW36NZwO95POT1ALjjD53t/Wns4I8vb/11WM6F0k4",
    "ocjtHP89FOc+qIK+bqhCEdRqlkToNhHLG/X04wVNXgLBurUKW5XccRvLGfwFP6aJ+",
    ""
  };

  static char newstr [2241] = "";
  newstr[0] = '\0';
  for (i = 0; i < 31; i++) {
    strcat(newstr, encStrCodegen[i]);
  }

  return newstr;
}

static void mdlSetWorkWidths_c2_LaneDetection(SimStruct *S)
{
  const char* newstr = sf_c2_LaneDetection_get_post_codegen_info();
  sf_set_work_widths(S, newstr);
  ssSetChecksum0(S,(3815622807U));
  ssSetChecksum1(S,(1617715500U));
  ssSetChecksum2(S,(503063881U));
  ssSetChecksum3(S,(3728995046U));
}

static void mdlRTW_c2_LaneDetection(SimStruct *S)
{
  if (sim_mode_is_rtw_gen(S)) {
    ssWriteRTWStrParam(S, "StateflowChartType", "Embedded MATLAB");
  }
}

static void mdlSetupRuntimeResources_c2_LaneDetection(SimStruct *S)
{
  SFc2_LaneDetectionInstanceStruct *chartInstance;
  chartInstance = (SFc2_LaneDetectionInstanceStruct *)utMalloc(sizeof
    (SFc2_LaneDetectionInstanceStruct));
  if (chartInstance==NULL) {
    sf_mex_error_message("Could not allocate memory for chart instance.");
  }

  memset(chartInstance, 0, sizeof(SFc2_LaneDetectionInstanceStruct));
  chartInstance = new (chartInstance) SFc2_LaneDetectionInstanceStruct;
  chartInstance->chartInfo.chartInstance = chartInstance;
  if (ssGetSampleTime(S, 0) == CONTINUOUS_SAMPLE_TIME && ssGetOffsetTime(S, 0) ==
      0 && ssGetNumContStates(ssGetRootSS(S)) > 0) {
    sf_error_out_about_continuous_sample_time_with_persistent_vars(S);
  }

  chartInstance->chartInfo.isEMLChart = 1;
  chartInstance->chartInfo.chartInitialized = 0;
  chartInstance->chartInfo.sFunctionGateway = sf_opaque_gateway_c2_LaneDetection;
  chartInstance->chartInfo.initializeChart =
    sf_opaque_initialize_c2_LaneDetection;
  chartInstance->chartInfo.mdlStart = sf_opaque_mdl_start_c2_LaneDetection;
  chartInstance->chartInfo.mdlTerminate =
    sf_opaque_mdl_terminate_c2_LaneDetection;
  chartInstance->chartInfo.mdlCleanupRuntimeResources =
    sf_opaque_cleanup_runtime_resources_c2_LaneDetection;
  chartInstance->chartInfo.enableChart = sf_opaque_enable_c2_LaneDetection;
  chartInstance->chartInfo.disableChart = sf_opaque_disable_c2_LaneDetection;
  chartInstance->chartInfo.getSimState =
    sf_opaque_get_sim_state_c2_LaneDetection;
  chartInstance->chartInfo.setSimState =
    sf_opaque_set_sim_state_c2_LaneDetection;
  chartInstance->chartInfo.getSimStateInfo =
    sf_get_sim_state_info_c2_LaneDetection;
  chartInstance->chartInfo.zeroCrossings = NULL;
  chartInstance->chartInfo.outputs = NULL;
  chartInstance->chartInfo.derivatives = NULL;
  chartInstance->chartInfo.mdlRTW = mdlRTW_c2_LaneDetection;
  chartInstance->chartInfo.mdlSetWorkWidths = mdlSetWorkWidths_c2_LaneDetection;
  chartInstance->chartInfo.callGetHoverDataForMsg = NULL;
  chartInstance->chartInfo.extModeExec = NULL;
  chartInstance->chartInfo.restoreLastMajorStepConfiguration = NULL;
  chartInstance->chartInfo.restoreBeforeLastMajorStepConfiguration = NULL;
  chartInstance->chartInfo.storeCurrentConfiguration = NULL;
  chartInstance->chartInfo.callAtomicSubchartUserFcn = NULL;
  chartInstance->chartInfo.callAtomicSubchartAutoFcn = NULL;
  chartInstance->chartInfo.callAtomicSubchartEventFcn = NULL;
  chartInstance->chartInfo.chartStateSetterFcn = NULL;
  chartInstance->chartInfo.chartStateGetterFcn = NULL;
  chartInstance->S = S;
  chartInstance->chartInfo.dispatchToExportedFcn = NULL;
  sf_init_ChartRunTimeInfo(S, &(chartInstance->chartInfo), false, 0,
    chartInstance->c2_JITStateAnimation,
    chartInstance->c2_JITTransitionAnimation);
  init_dsm_address_info(chartInstance);
  init_simulink_io_address(chartInstance);
  if (!sim_mode_is_rtw_gen(S)) {
  }

  mdl_setup_runtime_resources_c2_LaneDetection(chartInstance);
}

void c2_LaneDetection_method_dispatcher(SimStruct *S, int_T method, void *data)
{
  switch (method) {
   case SS_CALL_MDL_SETUP_RUNTIME_RESOURCES:
    mdlSetupRuntimeResources_c2_LaneDetection(S);
    break;

   case SS_CALL_MDL_SET_WORK_WIDTHS:
    mdlSetWorkWidths_c2_LaneDetection(S);
    break;

   case SS_CALL_MDL_PROCESS_PARAMETERS:
    mdlProcessParameters_c2_LaneDetection(S);
    break;

   default:
    /* Unhandled method */
    sf_mex_error_message("Stateflow Internal Error:\n"
                         "Error calling c2_LaneDetection_method_dispatcher.\n"
                         "Can't handle method %d.\n", method);
    break;
  }
}
