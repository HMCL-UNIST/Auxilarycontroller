#include "hip/hip_runtime.h"
/* Include files */

#include "LaneDetection_sfun.h"
#include "c2_LaneDetection.h"
#include "MWCudaDimUtility.hpp"
#ifdef utFree
#undef utFree
#endif

#ifdef utMalloc
#undef utMalloc
#endif

#ifdef __cplusplus

extern "C" void *utMalloc(size_t size);
extern "C" void utFree(void*);

#else

extern void *utMalloc(size_t size);
extern void utFree(void*);

#endif

#include <cstdlib>
#include <cstring>

/* Type Definitions */

/* Named Constants */
const int32_T CALL_EVENT = -1;

/* Variable Declarations */

/* Variable Definitions */
static real_T _sfTime_;

/* Function Declarations */
static void initialize_c2_LaneDetection(SFc2_LaneDetectionInstanceStruct
  *chartInstance);
static void initialize_params_c2_LaneDetection(SFc2_LaneDetectionInstanceStruct *
  chartInstance);
static void enable_c2_LaneDetection(SFc2_LaneDetectionInstanceStruct
  *chartInstance);
static void disable_c2_LaneDetection(SFc2_LaneDetectionInstanceStruct
  *chartInstance);
static void c2_do_animation_call_c2_LaneDetection
  (SFc2_LaneDetectionInstanceStruct *chartInstance);
static void ext_mode_exec_c2_LaneDetection(SFc2_LaneDetectionInstanceStruct
  *chartInstance);
static const mxArray *get_sim_state_c2_LaneDetection
  (SFc2_LaneDetectionInstanceStruct *chartInstance);
static void set_sim_state_c2_LaneDetection(SFc2_LaneDetectionInstanceStruct
  *chartInstance, const mxArray *c2_st);
static void sf_gateway_c2_LaneDetection(SFc2_LaneDetectionInstanceStruct
  *chartInstance);
static void mdl_start_c2_LaneDetection(SFc2_LaneDetectionInstanceStruct
  *chartInstance);
static void mdl_terminate_c2_LaneDetection(SFc2_LaneDetectionInstanceStruct
  *chartInstance);
static void mdl_setup_runtime_resources_c2_LaneDetection
  (SFc2_LaneDetectionInstanceStruct *chartInstance);
static void mdl_cleanup_runtime_resources_c2_LaneDetection
  (SFc2_LaneDetectionInstanceStruct *chartInstance);
static void initSimStructsc2_LaneDetection(SFc2_LaneDetectionInstanceStruct
  *chartInstance);
static void c2_eML_blk_kernel(SFc2_LaneDetectionInstanceStruct *chartInstance,
  real_T c2_b_in_1[154587], real32_T c2_b_out_1[6]);
static void c2_DeepLearningNetwork_setup(SFc2_LaneDetectionInstanceStruct
  *chartInstance, c2_trainedLaneNet0_LaneDetection0 *c2_obj);
static void c2_DeepLearningNetwork_predict(SFc2_LaneDetectionInstanceStruct
  *chartInstance, c2_trainedLaneNet0_LaneDetection0 *c2_obj, real_T
  c2_varargin_1[154587], real32_T c2_varargout_1[6]);
static void c2_emlrt_marshallIn(SFc2_LaneDetectionInstanceStruct *chartInstance,
  const mxArray *c2_b_out_1, const char_T *c2_identifier, real32_T c2_y[6]);
static void c2_b_emlrt_marshallIn(SFc2_LaneDetectionInstanceStruct
  *chartInstance, const mxArray *c2_u, const emlrtMsgIdentifier *c2_parentId,
  real32_T c2_y[6]);
static uint8_T c2_c_emlrt_marshallIn(SFc2_LaneDetectionInstanceStruct
  *chartInstance, const mxArray *c2_b_is_active_c2_LaneDetection, const char_T
  *c2_identifier);
static uint8_T c2_d_emlrt_marshallIn(SFc2_LaneDetectionInstanceStruct
  *chartInstance, const mxArray *c2_u, const emlrtMsgIdentifier *c2_parentId);
static __global__ void c2_eML_blk_kernel_kernel1(const real_T c2_b_in_1[154587],
  real_T c2_b_r_f1[154587]);
static __global__ void c2_DeepLearningNetwork_predict_kernel2(const real_T
  c2_varargin_1[154587], real32_T c2_in[154587], real_T c2_b_r_f1[154587]);
static __global__ void c2_DeepLearningNetwork_predict_kernel3(const real32_T
  c2_in[154587], c2_cell_wrap_8 c2_miniBatchT[1]);
static __global__ void c2_DeepLearningNetwork_predict_kernel4(const
  c2_cell_wrap_10 c2_outputsMiniBatch[1], real32_T c2_varargout_1[6]);
static void c2_checkCleanupCudaError(hipError_t c2_errCode, const char_T
  *c2_file, uint32_T c2_line);
static emlrtRTEInfo c2_createEmlrtInfoStruct(const char_T *c2_file, uint32_T
  c2_line);
static void init_dsm_address_info(SFc2_LaneDetectionInstanceStruct
  *chartInstance);
static void init_simulink_io_address(SFc2_LaneDetectionInstanceStruct
  *chartInstance);

/* Function Definitions */
static void initialize_c2_LaneDetection(SFc2_LaneDetectionInstanceStruct
  *chartInstance)
{
  emlrtLicenseCheckR2012b(chartInstance->c2_fEmlrtCtx,
    "distrib_computing_toolbox", 2);
  emlrtLicenseCheckR2012b(chartInstance->c2_fEmlrtCtx, "neural_network_toolbox",
    2);
  sim_mode_is_external(chartInstance->S);
  _sfTime_ = sf_get_time(chartInstance->S);
  chartInstance->c2_network_not_empty = false;
  chartInstance->c2_is_active_c2_LaneDetection = 0U;
  hipGetLastError();
  hipMalloc(&chartInstance->c2_gpu_r_f1, 1236696UL);
  hipMalloc(&chartInstance->c2_gpu_in_1, 1236696UL);
}

static void initialize_params_c2_LaneDetection(SFc2_LaneDetectionInstanceStruct *
  chartInstance)
{
}

static void enable_c2_LaneDetection(SFc2_LaneDetectionInstanceStruct
  *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
}

static void disable_c2_LaneDetection(SFc2_LaneDetectionInstanceStruct
  *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
}

static void c2_do_animation_call_c2_LaneDetection
  (SFc2_LaneDetectionInstanceStruct *chartInstance)
{
  sfDoAnimationWrapper(chartInstance->S, false, true);
  sfDoAnimationWrapper(chartInstance->S, false, false);
}

static void ext_mode_exec_c2_LaneDetection(SFc2_LaneDetectionInstanceStruct
  *chartInstance)
{
}

static const mxArray *get_sim_state_c2_LaneDetection
  (SFc2_LaneDetectionInstanceStruct *chartInstance)
{
  const mxArray *c2_b_y = NULL;
  const mxArray *c2_c_y = NULL;
  const mxArray *c2_st;
  const mxArray *c2_y = NULL;
  c2_st = NULL;
  c2_y = NULL;
  sf_mex_assign(&c2_y, sf_mex_createcellmatrix(2, 1), false);
  c2_b_y = NULL;
  sf_mex_assign(&c2_b_y, sf_mex_create("y", *chartInstance->c2_out_1, 1, 0U, 1U,
    0U, 2, 1, 6), false);
  sf_mex_setcell(c2_y, 0, c2_b_y);
  c2_c_y = NULL;
  sf_mex_assign(&c2_c_y, sf_mex_create("y",
    &chartInstance->c2_is_active_c2_LaneDetection, 3, 0U, 0U, 0U, 0), false);
  sf_mex_setcell(c2_y, 1, c2_c_y);
  sf_mex_assign(&c2_st, c2_y, false);
  return c2_st;
}

static void set_sim_state_c2_LaneDetection(SFc2_LaneDetectionInstanceStruct
  *chartInstance, const mxArray *c2_st)
{
  const mxArray *c2_u;
  c2_u = sf_mex_dup(c2_st);
  c2_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getcell(c2_u, 0)),
                      "out_1", *chartInstance->c2_out_1);
  chartInstance->c2_is_active_c2_LaneDetection = c2_c_emlrt_marshallIn
    (chartInstance, sf_mex_dup(sf_mex_getcell(c2_u, 1)),
     "is_active_c2_LaneDetection");
  sf_mex_destroy(&c2_u);
  sf_mex_destroy(&c2_st);
}

static void sf_gateway_c2_LaneDetection(SFc2_LaneDetectionInstanceStruct
  *chartInstance)
{
  int32_T c2_i;
  int32_T c2_i1;
  real32_T c2_fv[6];
  chartInstance->c2_JITTransitionAnimation[0] = 0U;
  _sfTime_ = sf_get_time(chartInstance->S);
  for (c2_i = 0; c2_i < 154587; c2_i++) {
    chartInstance->c2_dv[c2_i] = (*chartInstance->c2_in_1)[c2_i];
  }

  c2_eML_blk_kernel(chartInstance, chartInstance->c2_dv, c2_fv);
  for (c2_i1 = 0; c2_i1 < 6; c2_i1++) {
    (*chartInstance->c2_out_1)[c2_i1] = c2_fv[c2_i1];
  }

  c2_do_animation_call_c2_LaneDetection(chartInstance);
}

static void mdl_start_c2_LaneDetection(SFc2_LaneDetectionInstanceStruct
  *chartInstance)
{
}

static void mdl_terminate_c2_LaneDetection(SFc2_LaneDetectionInstanceStruct
  *chartInstance)
{
  hipError_t c2_errCode;
  hipFree(*chartInstance->c2_gpu_in_1);
  hipFree(*chartInstance->c2_gpu_r_f1);
  c2_errCode = hipGetLastError();
  if (c2_errCode != hipSuccess) {
    emlrtThinCUDAError(c2_errCode, hipGetErrorName(c2_errCode),
                       hipGetErrorString(c2_errCode), "SimGPUErrorChecks",
                       chartInstance->c2_fEmlrtCtx);
  }
}

static void mdl_setup_runtime_resources_c2_LaneDetection
  (SFc2_LaneDetectionInstanceStruct *chartInstance)
{
  setLegacyDebuggerFlag(chartInstance->S, false);
  setDebuggerFlag(chartInstance->S, false);
  sim_mode_is_external(chartInstance->S);
}

static void mdl_cleanup_runtime_resources_c2_LaneDetection
  (SFc2_LaneDetectionInstanceStruct *chartInstance)
{
}

static void initSimStructsc2_LaneDetection(SFc2_LaneDetectionInstanceStruct
  *chartInstance)
{
}

const mxArray *sf_c2_LaneDetection_get_eml_resolved_functions_info()
{
  const mxArray *c2_nameCaptureInfo = NULL;
  const char_T *c2_data[42] = {
    "789ced9dc993e3585ec7dd133d41374c55273d0cd34cb054079ca6a1d2763ad3652e60cb76a69dde97f45231d129cbcfb66c5992b578bb606e73830882d3b013"
    "cc30c1303411400cdbbf00072efc1d9cb81081bcbc4c5b6595dc29f9952cfd14d1ed56fc94effbfbbdf7fa7df456f9de4b65dff3f97ccfb57ffeeb777cbee9f3",
    "aff896d733dffa3adbfc7ec5b77be9edef6d7eabba7b7c7dd5f7fecedf2dedffa1e9fddee69e1178054d95f50d4f0fd1c35fb68521cbd3bc529989c8272159e0"
    "c6a8bdb274580e55d8212a6fdfe49677c3e496e9e166695afe37d543cca0ac0e7d524f7ef490dbbe59e5c7f2fa6f8378df3f303f4606f971a6b3bf4ed513df39",
    "5765e99c13189a3bcf462b9968ecbc14f407fdad734510b896303d977bb484dae76d8e11da48fabc45cb68f7e6b3d57f9f7fc66ab929f15a3a9c40b7e3088919"
    "444b3ccb7773489908d2e0a5a88bf3bb16e3fcd4244e6c1fd20a47b728cdcb2ee2735aa1ab3cdde25041124424292cc26580fd5a58f4eb174cfcc2f655bebddc",
    "93533a7f3e78cf9a3fdf37f107db5fa7a8ef9cf784213aef0d19ee3c2e30ea10f18a8c2b4659154541520a3433a0bb483ea4a2c8eb3f11f19fac9ff9bccd7dae"
    "d0521769696f6acf6fefc98673834a84f3e5de20ee8f0ecc17fdefe3f31fac7e7ff38bfff97cf94b4a0f5f5ed19b1aa47768bdfe7903bd339d7d30e98e2a2a95",
    "6f86a36cb17417a877e862fffad18f82898e991f3e837b52e97b8517f8fea9713ed3ddebe3c476ad5ddab8906ff57dc005e0027e1eb840420fb8604ffac085c3"
    "e2fc9a499cd8aeb54b9555eb94615bdbfa0b8bfac005e0c253f4f0e5153de0823de97b850b56ebcbcf9ac489ed0f5c88d30abd1a32dcf56361d10fe003f0e129",
    "7af8f28a1ef0c19ef4bdc207abfd869f368913dbb57629955fce36c8bbfa0b8bfac005e0c253f4f0e5153de0823de97b850b56ebcb374de2c47666b926202948"
    "65345211cf20dc2a031f800febe7810f24f4800ff6a4ef153e58ed3718e5eb99eef7b1dfa041a294cb0117800bebe7810b24f4800bf6a40f5c382cce67ba7b7d",
    "9cd83ed1dc48f1a2aa44a5aeec83f124e0027e1eb840420fb8604ffa5ee1c2bdc5383fd0dd3fc6b9b62c5724d1332401078003ebe7810324f48003f6a4ef150e"
    "58ed1f7c89f9e682d668e5d4e1aefec2a23e7001b8f0143d7c79450fb8604ffac085c3e2fc199338b17d22d162965ead52ddd15f58d4072e00179ea2872fafe8",
    "0117ec49df2b5c58588cf35b267162fb98e6d836ade0f5474941a2385a96d9cecc5e7f8013c089a7e8e1cb2b7ac0097bd2f70a27acf61f0e9d77d6daa5151872"
    "f410c1bc3370e1f179e002093de0823de97b850b0b8b711eda7fd8395f2f234c905411ca8aa4328abdfe002780134fd1c39757f48013f6a40f9c382cce43db65",
    "5142a216c36ae96a999da3e5b6060a8d69ee1df9039c004e6cebe1cb2b7ac0097bd2f70a27ac8e333dd7ddebe3c476f9e1fcd5dcea5076e0027061fd3c708184"
    "1e70c19ef4bdc2857b8b717ed530ceb545e3812afa8003c001fc3c7080841e70c09ef4bdc201abfd830f4de2c4762d36aebc61c2b6fec2a23e7001b8f0143d7c",
    "79450fb8604ffac085c3e23c747fc3860bcba9851dfd85457de00270e1297af8f28a1e70c19ef4810b87c579e87cc2920b25246b64506805e613800b0fcf0317"
    "48e80117ec491fb870589c1fe9eef57162fb920b5571b9f36d0d06e0027061fd3c7081841e70c19ef4810b87c579e8773e1589e6655190d11d2d6deb2f2cea03",
    "17800b4fd1c39757f4800bf6a40f5c382c4ea37c3dd3fd8a481aaaca7affc2f21025e0027061fd3c7081841e70c19ef4bdc20552df7fde7021af2a1b30e8fc58"
    "58f403f8007c788a1ebebca2077cb0277de0c361717edd244e6c376a91810fc087f5f3c007127ac0077bd2f70a1fac8e2b1dfabd06696b6dd2b6fec2a2bedd5c",
    "f059e4c29f98f883edaf53798781e1b18448f2e0b3fffb147870443d523cb899371a8352add9baab0493749fe9cbad5222063c38351e58ad2fdf308913db45cd"
    "7f9651a27c7b7b759273b97066910b3f34f107db9dc785bd2575f0fe78bbdab56f03278eaa478a13179162b03d977ba58b6433c167661d3552c9dd00274e8d13",
    "f716e334fbde27b3f93a83537960b59ff04726fe60bbf378804b665d2548b557bf0eedff51f548b5ff09617a554cde95fc287cddcfb5ae6e02f3bb38f4134eae"
    "fdb73a6e74e87e679a51d831adb0022fefe82f2cea3b8d077f6ae20fb63b8f075b25a4d512183772871e291e8cab8956393c11a874466d33296ea63486039847",
    "38391edc5b8cf3a70ce35c5b36a310ae6dffbf67e20fb63baffddf94cca64640fbef0e3d52edffa87ccd15a609aa5d2f75a522f26785496a9c84f6ffd4daff85"
    "c5380ffd0ecf723f7361dde22cbfaca0ca8a30ccd03324c9f6fae3b4f9831f9bf883edcee3837189919c67fed1bffe2ebdfcf54afb4d5a8f142fa49b6c5189b5",
    "23a1b490998da297dd71aa9381fec2c9f182d4be85add109adf529e5723a3f1616fd70dabad4bf32f107db9dc789374a6a5567607ec11d7aa4f840a519d4e558"
    "ae1863eaf141215eed64ba13985f003ee89edbcf070af8b0f9753a1f28c27c80f1a6e3ea91e2432e34938b65a5ded76a756538be9964834208f870727cc0f74f",
    "8df399ee5e1f27b68b0f2317ebbe835bb9f0e726fe60bbf3b8b05342c4d7a502178eab476a3d7aaf94572bb76caac04ba1b2c4472f14e5d5f67a74e0c2fef49d"
    "c60552f56539aa1ddd33b6e4543e7877ffc2de92024eb84c8f54ff61960d312c7f5757042e7631ce8fbbf1493d16074e788d13878e2fedce7ec2fc8373c797de",
    "2829185f72911eb1f125ba2796e6ec759a9d52d3795216af95ac40011f4e8d0fdfb518e7af99c489eda2202b054960902cafcfd6ab0825a4a8124fd10aea0a12"
    "ab05bfedd7c2a25f76f3e2138bbcf889893fd8ee3c5e1c52722f4552ed1d9cab745c3d52fcc8dfdce5b243ba1a8b64e2b97e4bbc0d8e6f8404f0e3d4f861757e",
    "e2b9ee5e1f27b6ef8e6e38971356fb157f61e20fb63b8f13bb25842b0aac6b72871ea9f1e6ab1a9b8f72f98beaa023cc1baad86814a4940fb8e0352e7c99ef44"
    "6f463376f41716f59db66fee74f74d6f9510ec9b768d1ea97e42b550aaf3af92f9ca24dfece7e45781a4a2c03e08eff1e0439338b15d464a94677a82a4d35f58",
    "d4771a0ffed8c41f6c771e0f1e4a08f63db8488fd8be0726f56ad6e6aadd346a0d4637aaac566659d847ed391e1cfa5d68adb5d9b85066e748762c0fac8e1bfd"
    "a5893fd8ee481e6c9710e1f335800bc7d523c58530d795fd177252c9656b4ce32a4ecb7c5c84f54a27c78585c5385f98c489ed222dc9ebaf7fca94e66a17f15b",
    "6b274bb99c533961751efa1f4cfcc176e771c2acc4e09c5637e991e246a538575a593a3aa61297c3d860ca14b37904e34bc00ddd73d8fed65688026e9c183728"
    "d2dc807989e3ea91e286bfa34a9713b1d7bdc9a562a9627a781b913858ffea396efca2499cd8fe662bb49913dd1cc5e1546e58dd4ff785893fd87e0adc782c31",
    "72eb5d8117c7d523c58bab6ca5f8aa91cc4c674cf1b23da4fbf4a4cac1791cc00bdd73a6bc28012f96d709f1a2449217302e755c3d52bcb88d72d7e549ec4ab9"
    "f477520d29cbd5d4781bfa1727c70babf5e5639338b15d46caaaed59cea16e6fc0762a27acce777fdfc41f6c771e27de2c2992e783c3f7458fab478a0f415aa9",
    "84a3a1f9558e0a0b037f91998d7234f4274e8e0fc4f65faf77f06eb73cfb8e75722c2f3cbcfffa809283fe855bf448f1434d2bf9e9dd24a35e67432554eaf9fd"
    "696e0ceba54e8e1f0b8b71be308913dbf7b4423b8708b9951b273cef6d5262b05eca4d7aa4f667c79a699652aad4154d27e85c73ee0fa790ea036e9c1a37f0fd",
    "53e3fc9a499cd8be1a15df6a9a9dca099f454efc99893fd8ee404e6c9510cc6bbb458f543f42ec8c6bd36825324c8c423c23471281c4280afbf14e8e07a4de1f"
    "b4e668db1d4ae03b6cd7e75c2e78f73cf1bd2505e789bb4c8fd8fa2731122f646f7bea9055c3a96cbe1f52729d1be0c4a9716261314ebbdbe77b8bfebcafbb7f",
    "f4676d61b49cdcd623376fed301c18540e52ed169cfb7a5c3d521c184cbaa38a4ae59be1285b2cdd05ea1dbad877d17ebb7b83bf37cb47fd65948f5ed3fbfd27"
    "eae1f4bf6da287edafabdbcd6d34ce08bc2209dcb922d12c8fda199a5f8e91bc1cd2caae7ff706fa76fd7ffa871f7ea34552ef27fff9ad8f49eae1eb54dbbd43",
    "fbbd9368abdee8c52eabf3b89f2ba7949b4b79c4f9dcd3eebdb0f85ef46f06e99fe9ecaf538913782ffa942da12e2b6befd6712422be8d788645f2767e2d0cf2"
    "e3d4dea3f5d7a1ed3abc47c37bb41bf5e03dda9ef481275f8e2756ebdd2726f985ed6d6e13c44b466df3fcfadf51917d571cf9df27eae1f4ffc0440fdb0970a4",
    "2baa6beb764d79c86fadd22cb3fa1c6738e173f27e0bc6e18faa478a1b281c88c572632632ce14fb810225767ae1b68bd67dfa2c72e37b06e99fe9ecce9baf13"
    "1fced8dfce8f8541bcd0cf78bb1ef433de9e2fd0cf78b77ad0cfb0277de0c52e2f48f523f6785455584eb68b5bbf6ce207b61b726be5cdbb38d7c259d564950d",
    "fb60b232905b2ffa1bd0ff38aa1eb175404cba5c676a513fbabef3cfe560a117aad02e5a2f0a3cd9e5896810efa1f9f14c77afcf0f6cd7dcabf2ec4845397a88"
    "7cf6f57fece68867cfbd588364a79848f64760ffd971f548f123309ac767d36aa312eca000938a755a17919a8bcebd007eecf283dcf91848d4bcbd438c224879",
    "5529d34391d39db5b0edd7c2a25f7673c5ea7cd9bf9bf883ed8eab376bae1c527c44bf3707fd95e3ea915ab7d5949866586da24eab1855b8862ae5546efbfd12"
    "78b33f7daff2e6574df203db370d566aa879fe16dcb8963787afcf7058bdd9e1cddb8a8fecf91ac09be3ea91e24d30c707072d6ed48d50955e251f4e73d745c1",
    "07bc712b6facd6ab9f33c90f6cdf3458a55c8e421ca73559a2aa6cfbb1b0e887d3e65bfed6c41f6c775c3dd9e1cb4e71c1f7b25da4476abcacacd0c3095365aa"
    "09741549c8492a5b62eab0decbb53c5918c47b687efc92497e603b6ea0844996eeafce935b8fbdac8e9bb3d11fbbb962f53cc07f32f107db1d575f76b9b2b7d8",
    "48cec7407fe5b87aa4f852f7cf6e3ac157d4057d954a178383db51a65483f97ce08b417e1cfcfda3371aaad5580bc68b6bf9f28f26fe60bbe3ea8b115f1e8b0d"
    "e65f5ca4478a2f915e81a76f932a47052e0ab356377b99aef109e00bf0657f7ebc30c90f6cdf345494c0a9437ecfbbb053f96275dee59f4dfcc176c7d5971dbe",
    "18161bf1f3098133c7d523c51926dfa9cf851093b87835180d8bc140ad7581a01f039c31c88f5f31c90f6cdfdb606d75659cca19cf7e6fe92d9cd9eeca005fdc"
    "a1476a5eff466932a97884bf50c4f83c981cd7c699f4f63839f0657ffaa7ca1762e7efaf1baa2ccbb32d5a617af84b7e4ee58ad579fdbf33f107db1d574f76b8",
    "a22b2eadee004fdca147eaff7b454c258361868d34a471598d05db7361c6b8e83c75e0c92e4facae4bfed4243fb07dd340ad571c197d24d6e75cbe581d1ffb17"
    "137fb0dd71f566872fc6c58707c88037eed023c59bcc6d5f0d5c16d25c9deb172b7781083f569ac01bd7f2666110ef91d6913d34583b9f28b5d11fa78d8fb963",
    "1dd9fe62837564eed123353e369c5d56a846feca9f6bcd731d25c94c2e85a40ff8e256be58ad575f37c90f6cd7dca354591186197a86b4664a109dbdafd2eaf8"
    "d88f4cfcc176c7d59387f361f4c505fb28dda4478a2769fa6a708ba840f826a9a4eb85909409ddc23e4af7f2443488d7ee73c6981e6206a94e52e0daa8ed732e",
    "473c7eced84e31417fc43d7aa4c6bb1a7cb8d96d0d59a5c9ca93cc607a492b013fec9b742d3f087eff7c3d685266e7cb034356efb93ee772c46a7fe4c726fe60"
    "bbe3eac9437fe4cde2029eb8478f144ff2e1d9751451e8b25d49dc6683a162ee424dc0fc09f0c4203f3e36c90f6cd7dc4be557e3245b30f1b997277f63e20fb6",
    "3bae9e3cf044575c84bfe3053c39ae1e299ef452d37c94effb4789793e1d52c23237990d61bf0af0c4203fbe69921fd8aeb957d67c47ed547ed53ca578cd1924"
    "bb96275f98f883ed8eab270f3cd9575cb03fc52d7aa47822df06d8ae7c939e0642856ce142991667bd319cabef5a9e909a2fd1dcc30d139afa9cdb2ff1f87cc9",
    "4e31c1f8967bf4488d6b8f2a857ab044a56f7aa94477986327bc586cf8801fc08ffdf96194ff67ba5fcdbde566b9d872b35c8aef084ee587d57ec80f4dfcc176"
    "c7d58f077eec14139cc3e2323d62dfa74f0fae9aaa1a0d8c832d569e678bedcba404e35aaee5c8c2205ebbf7992c074ad048453c833288ef2abdcdd1ea705ef1",
    "fa725c7d791cdf322c36e8a7b8478f145f6eeffab54a7c181bcef348494855898a746614f0c5ad7cb1da4ff94877afcf0f6c7f5828b4d9b9e8549e581de7fa6b"
    "137fb0dd71f543b79e0bef54047eb8438fd438d79c56b84a4d8a46aed5d14856d857e50915f3013fdcca0fe2eb8293885654092dd79b523ee7f64b605df09bc5",
    "05fd11f7e891ea8f500dd92fddc5aa57d4b4d5e855b9d27540ccc2ba60d7f284d4f7ecf735507b8e57712c5f5e58ac3727fe3dfb438a0fbea7e2223d52bc29c5"
    "93fee6f855713eec0bd73334c8e4a26202bea7e25ade105c37bcb3516ed35239962fb06e785f71c178985bf4488d5b94cacdd02c569936a861949b1503dd70bb",
    "b77d2e06f0647ffa5ee5c9a1dfab5f1fe0c13fb6500eefaf78fc7bf5fb8a0bbe57ef223d523c69466ec68dbb71375965b9dab879cb4bd954c9073c712b4f4483"
    "780fcd8fe7ba7b7d7e60fb6e03e55c8e589d9fff81893fd8eeb8fab18f23f05d14d7e8911adf9adc06fad566bc12a6e7d94e2d5fcb052e151aceed722d3f48bd",
    "978c698e6dd30ac2ab519dbe6ed8e3f3f37b8b0be6e7dda3478a27e5017d379f0af169ad36ed5ee4b2897848ad004f5ccb13abfd11a3fc3fd3fde2066ab5eb3a"
    "2a75ddca9113dfd7a82f26e2fb1a615cebb87aa438721798777b683abca85dc703bc1abb0dcbf5fa3570c4ad1cb15aaf3e31c90f6c5f0d9844f9764d732a2948",
    "3734d759bdeb3a952756f733febd893fd8eeb87ab235beb5a7b860bfbccbf4488d4bb4e6c2acd093b946fba62ede368b17b95211e6dfddcb958541bcefaa3dbf"
    "7fa23ffacbc81f7c915fb7e5aceab0c6879101d66db9458f547fe48a4997eb4c2dea47d777feb91c2cf44215da05e7acfc3f9e707ec9",
    "" };

  c2_nameCaptureInfo = NULL;
  emlrtNameCaptureMxArrayR2016a(&c2_data[0], 134800U, &c2_nameCaptureInfo);
  return c2_nameCaptureInfo;
}

static void c2_eML_blk_kernel(SFc2_LaneDetectionInstanceStruct *chartInstance,
  real_T c2_b_in_1[154587], real32_T c2_b_out_1[6])
{
  hipMemcpy(chartInstance->c2_gpu_in_1, &c2_b_in_1[0], 1236696UL,
             hipMemcpyHostToDevice);
  c2_eML_blk_kernel_kernel1<<<dim3(302U, 1U, 1U), dim3(512U, 1U, 1U)>>>
    (*chartInstance->c2_gpu_in_1, *chartInstance->c2_gpu_r_f1);
  if (!chartInstance->c2_network_not_empty) {
    c2_DeepLearningNetwork_setup(chartInstance, &chartInstance->c2_network);
    chartInstance->c2_network_not_empty = true;
  }

  hipMemcpy(&chartInstance->c2_r_f1[0], chartInstance->c2_gpu_r_f1, 1236696UL,
             hipMemcpyDeviceToHost);
  c2_DeepLearningNetwork_predict(chartInstance, &chartInstance->c2_network,
    chartInstance->c2_r_f1, c2_b_out_1);
}

static void c2_DeepLearningNetwork_setup(SFc2_LaneDetectionInstanceStruct
  *chartInstance, c2_trainedLaneNet0_LaneDetection0 *c2_obj)
{
  c2_obj->setup();
}

static void c2_DeepLearningNetwork_predict(SFc2_LaneDetectionInstanceStruct
  *chartInstance, c2_trainedLaneNet0_LaneDetection0 *c2_obj, real_T
  c2_varargin_1[154587], real32_T c2_varargout_1[6])
{
  c2_cell_wrap_10 (*c2_gpu_outputsMiniBatch)[1];
  c2_cell_wrap_8 (*c2_gpu_miniBatchT)[1];
  real_T (*c2_b_gpu_r_f1)[154587];
  real_T (*c2_gpu_varargin_1)[154587];
  real32_T (*c2_gpu_in)[154587];
  real32_T (*c2_gpu_varargout_1)[6];
  hipMalloc(&c2_b_gpu_r_f1, 1236696UL);
  hipMalloc(&c2_gpu_varargout_1, 24UL);
  hipMalloc(&c2_gpu_outputsMiniBatch, 24UL);
  hipMalloc(&c2_gpu_miniBatchT, 618348UL);
  hipMalloc(&c2_gpu_in, 618348UL);
  hipMalloc(&c2_gpu_varargin_1, 1236696UL);
  hipMemcpy(c2_gpu_varargin_1, &c2_varargin_1[0], 1236696UL,
             hipMemcpyHostToDevice);
  c2_DeepLearningNetwork_predict_kernel2<<<dim3(302U, 1U, 1U), dim3(512U, 1U, 1U)>>>
    (*c2_gpu_varargin_1, *c2_gpu_in, *c2_b_gpu_r_f1);
  c2_DeepLearningNetwork_predict_kernel3<<<dim3(302U, 1U, 1U), dim3(512U, 1U, 1U)>>>
    (*c2_gpu_in, *c2_gpu_miniBatchT);
  hipMemcpy(c2_obj->getInputDataPointer(0), (*c2_gpu_miniBatchT)[0].f1,
             c2_obj->layers[0]->getOutputTensor(0)->getNumElements() * sizeof
             (real32_T), hipMemcpyDeviceToDevice);
  c2_obj->predict();
  hipMemcpy((*c2_gpu_outputsMiniBatch)[0].f1, c2_obj->getLayerOutput(17, 0),
             c2_obj->layers[17]->getOutputTensor(0)->getNumElements() * sizeof
             (real32_T), hipMemcpyDeviceToDevice);
  c2_DeepLearningNetwork_predict_kernel4<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
    (*c2_gpu_outputsMiniBatch, *c2_gpu_varargout_1);
  hipMemcpy(&c2_varargout_1[0], c2_gpu_varargout_1, 24UL,
             hipMemcpyDeviceToHost);
  hipFree(*c2_gpu_varargin_1);
  hipFree(*c2_gpu_in);
  hipFree(*c2_gpu_miniBatchT);
  hipFree(*c2_gpu_outputsMiniBatch);
  hipFree(*c2_gpu_varargout_1);
  hipFree(*c2_b_gpu_r_f1);
}

static void c2_emlrt_marshallIn(SFc2_LaneDetectionInstanceStruct *chartInstance,
  const mxArray *c2_b_out_1, const char_T *c2_identifier, real32_T c2_y[6])
{
  emlrtMsgIdentifier c2_thisId;
  c2_thisId.fIdentifier = const_cast<const char_T *>(c2_identifier);
  c2_thisId.fParent = NULL;
  c2_thisId.bParentIsCell = false;
  c2_b_emlrt_marshallIn(chartInstance, sf_mex_dup(c2_b_out_1), &c2_thisId, c2_y);
  sf_mex_destroy(&c2_b_out_1);
}

static void c2_b_emlrt_marshallIn(SFc2_LaneDetectionInstanceStruct
  *chartInstance, const mxArray *c2_u, const emlrtMsgIdentifier *c2_parentId,
  real32_T c2_y[6])
{
  int32_T c2_i;
  real32_T c2_fv[6];
  sf_mex_import(c2_parentId, sf_mex_dup(c2_u), c2_fv, 0, 1, 0U, 1, 0U, 2, 1, 6);
  for (c2_i = 0; c2_i < 6; c2_i++) {
    c2_y[c2_i] = c2_fv[c2_i];
  }

  sf_mex_destroy(&c2_u);
}

static uint8_T c2_c_emlrt_marshallIn(SFc2_LaneDetectionInstanceStruct
  *chartInstance, const mxArray *c2_b_is_active_c2_LaneDetection, const char_T
  *c2_identifier)
{
  emlrtMsgIdentifier c2_thisId;
  uint8_T c2_y;
  c2_thisId.fIdentifier = const_cast<const char_T *>(c2_identifier);
  c2_thisId.fParent = NULL;
  c2_thisId.bParentIsCell = false;
  c2_y = c2_d_emlrt_marshallIn(chartInstance, sf_mex_dup
    (c2_b_is_active_c2_LaneDetection), &c2_thisId);
  sf_mex_destroy(&c2_b_is_active_c2_LaneDetection);
  return c2_y;
}

static uint8_T c2_d_emlrt_marshallIn(SFc2_LaneDetectionInstanceStruct
  *chartInstance, const mxArray *c2_u, const emlrtMsgIdentifier *c2_parentId)
{
  uint8_T c2_b_u;
  uint8_T c2_y;
  sf_mex_import(c2_parentId, sf_mex_dup(c2_u), &c2_b_u, 1, 3, 0U, 0, 0U, 0);
  c2_y = c2_b_u;
  sf_mex_destroy(&c2_u);
  return c2_y;
}

static __global__ __launch_bounds__(512, 1) void c2_eML_blk_kernel_kernel1(const
  real_T c2_b_in_1[154587], real_T c2_b_r_f1[154587])
{
  int32_T c2_i;
  c2_i = (int32_T)mwGetGlobalThreadIndex();
  if (c2_i < 154587) {
    c2_b_r_f1[c2_i] = c2_b_in_1[c2_i];
  }
}

static __global__ __launch_bounds__(512, 1) void
  c2_DeepLearningNetwork_predict_kernel2(const real_T c2_varargin_1[154587],
  real32_T c2_in[154587], real_T c2_b_r_f1[154587])
{
  int32_T c2_i;
  c2_i = (int32_T)mwGetGlobalThreadIndex();
  if (c2_i < 154587) {
    c2_b_r_f1[c2_i] = c2_varargin_1[c2_i];
    c2_in[c2_i] = (real32_T)c2_b_r_f1[c2_i];
  }
}

static __global__ __launch_bounds__(512, 1) void
  c2_DeepLearningNetwork_predict_kernel3(const real32_T c2_in[154587],
  c2_cell_wrap_8 c2_miniBatchT[1])
{
  uint64_T c2_threadId;
  uint64_T c2_tmpIndex;
  int32_T c2_i1;
  int32_T c2_i2;
  int32_T c2_p;
  c2_threadId = mwGetGlobalThreadIndex();
  c2_i1 = (int32_T)(c2_threadId % 227UL);
  c2_tmpIndex = (c2_threadId - (uint64_T)c2_i1) / 227UL;
  c2_i2 = (int32_T)(c2_tmpIndex % 227UL);
  c2_tmpIndex = (c2_tmpIndex - (uint64_T)c2_i2) / 227UL;
  c2_p = (int32_T)c2_tmpIndex;
  if ((c2_p < 3) && (c2_i2 < 227) && (c2_i1 < 227)) {
    c2_miniBatchT[0].f1[(c2_i1 + 227 * c2_i2) + 51529 * c2_p] = c2_in[(c2_i2 +
      227 * c2_i1) + 51529 * c2_p];
  }
}

static __global__ __launch_bounds__(32, 1) void
  c2_DeepLearningNetwork_predict_kernel4(const c2_cell_wrap_10
  c2_outputsMiniBatch[1], real32_T c2_varargout_1[6])
{
  int32_T c2_i3;
  c2_i3 = (int32_T)mwGetGlobalThreadIndex();
  if (c2_i3 < 6) {
    c2_varargout_1[c2_i3] = c2_outputsMiniBatch[0].f1[c2_i3];
  }
}

void c2_trainedLaneNet0_LaneDetection0::allocate()
{
  int32_T c2_idx;
  this->targetImpl->allocate(290400, 2);
  for (c2_idx = 0; c2_idx < 18; c2_idx++) {
    this->layers[c2_idx]->allocate();
  }

  (static_cast<MWTensor<real32_T> *>(this->inputTensors[0]))->setData
    (this->layers[0]->getLayerOutput(0));
}

void c2_trainedLaneNet0_LaneDetection0::postsetup()
{
  this->targetImpl->postSetup(this->layers, this->numLayers);
}

c2_trainedLaneNet0_LaneDetection0::c2_trainedLaneNet0_LaneDetection0()
{
  this->numLayers = 18;
  this->isInitialized = false;
  this->targetImpl = 0;
  this->layers[0] = new MWInputLayer;
  this->layers[0]->setName("data");
  this->layers[1] = new MWElementwiseAffineLayer;
  this->layers[1]->setName("data_normalization");
  this->layers[1]->setInPlaceIndex(0, 0);
  this->layers[2] = new MWFusedConvReLULayer;
  this->layers[2]->setName("conv1_relu1");
  this->layers[3] = new MWNormLayer;
  this->layers[3]->setName("norm1");
  this->layers[4] = new MWMaxPoolingLayer;
  this->layers[4]->setName("pool1");
  this->layers[5] = new MWFusedConvReLULayer;
  this->layers[5]->setName("conv2_relu2");
  this->layers[6] = new MWNormLayer;
  this->layers[6]->setName("norm2");
  this->layers[7] = new MWMaxPoolingLayer;
  this->layers[7]->setName("pool2");
  this->layers[8] = new MWFusedConvReLULayer;
  this->layers[8]->setName("conv3_relu3");
  this->layers[9] = new MWFusedConvReLULayer;
  this->layers[9]->setName("conv4_relu4");
  this->layers[10] = new MWFusedConvReLULayer;
  this->layers[10]->setName("conv5_relu5");
  this->layers[11] = new MWMaxPoolingLayer;
  this->layers[11]->setName("pool5");
  this->layers[12] = new MWFCLayer;
  this->layers[12]->setName("fc6");
  this->layers[13] = new MWReLULayer;
  this->layers[13]->setName("relu6");
  this->layers[13]->setInPlaceIndex(0, 0);
  this->layers[14] = new MWFCLayer;
  this->layers[14]->setName("fcLane1");
  this->layers[15] = new MWReLULayer;
  this->layers[15]->setName("fcLane1Relu");
  this->layers[15]->setInPlaceIndex(0, 0);
  this->layers[16] = new MWFCLayer;
  this->layers[16]->setName("fcLane2");
  this->layers[17] = new MWOutputLayer;
  this->layers[17]->setName("output");
  this->layers[17]->setInPlaceIndex(0, 0);
  this->targetImpl = new MWTargetNetworkImpl;
  this->inputTensors[0] = new MWTensor<real32_T>;
  this->inputTensors[0]->setHeight(227);
  this->inputTensors[0]->setWidth(227);
  this->inputTensors[0]->setChannels(3);
  this->inputTensors[0]->setBatchSize(1);
  this->inputTensors[0]->setSequenceLength(1);
}

void c2_trainedLaneNet0_LaneDetection0::deallocate()
{
  int32_T c2_idx;
  this->targetImpl->deallocate();
  for (c2_idx = 0; c2_idx < 18; c2_idx++) {
    this->layers[c2_idx]->deallocate();
  }
}

void c2_trainedLaneNet0_LaneDetection0::setSize()
{
  int32_T c2_idx;
  for (c2_idx = 0; c2_idx < 18; c2_idx++) {
    this->layers[c2_idx]->propagateSize();
  }

  this->allocate();
  this->postsetup();
}

void c2_trainedLaneNet0_LaneDetection0::resetState()
{
}

void c2_trainedLaneNet0_LaneDetection0::setup()
{
  if (this->isInitialized) {
    this->resetState();
  } else {
    this->isInitialized = true;
    this->targetImpl->preSetup();
    this->targetImpl->setAutoTune(true);
    (static_cast<MWInputLayer *>(this->layers[0]))->createInputLayer
      (this->targetImpl, this->inputTensors[0], 227, 227, 3, 0, "", 0);
    (static_cast<MWElementwiseAffineLayer *>(this->layers[1]))
      ->createElementwiseAffineLayer(this->targetImpl, this->layers[0]
      ->getOutputTensor(0), 227, 227, 3, 227, 227, 3, false, 1, 1,
      "/home/hmcl/ADcontrol/slprj/_sfprj/LaneDetection/_self/sfun/src/cnn_trainedLaneNet0_LaneDetection0_data_scale.bin",
      "/home/hmcl/ADcontrol/slprj/_sfprj/LaneDetection/_self/sfun/src/cnn_trainedLaneNet0_LaneDetection0_data_offset.bin",
      0);
    (static_cast<MWFusedConvReLULayer *>(this->layers[2]))
      ->createFusedConvReLULayer(this->targetImpl, 1, this->layers[1]
      ->getOutputTensor(0), 11, 11, 3, 96, 4, 4, 0, 0, 0, 0, 1, 1, 1,
      "/home/hmcl/ADcontrol/slprj/_sfprj/LaneDetection/_self/sfun/src/cnn_trainedLaneNet0_LaneDetection0_conv1_w.bin",
      "/home/hmcl/ADcontrol/slprj/_sfprj/LaneDetection/_self/sfun/src/cnn_trainedLaneNet0_LaneDetection0_conv1_b.bin",
      1);
    (static_cast<MWNormLayer *>(this->layers[3]))->createNormLayer
      (this->targetImpl, this->layers[2]->getOutputTensor(0), 5, 0.0001, 0.75,
       1.0, 0);
    (static_cast<MWMaxPoolingLayer *>(this->layers[4]))->createMaxPoolingLayer
      (this->targetImpl, this->layers[3]->getOutputTensor(0), 3, 3, 2, 2, 0, 0,
       0, 0, 0, 1, 1);
    (static_cast<MWFusedConvReLULayer *>(this->layers[5]))
      ->createFusedConvReLULayer(this->targetImpl, 1, this->layers[4]
      ->getOutputTensor(0), 5, 5, 48, 128, 1, 1, 2, 2, 2, 2, 1, 1, 2,
      "/home/hmcl/ADcontrol/slprj/_sfprj/LaneDetection/_self/sfun/src/cnn_trainedLaneNet0_LaneDetection0_conv2_w.bin",
      "/home/hmcl/ADcontrol/slprj/_sfprj/LaneDetection/_self/sfun/src/cnn_trainedLaneNet0_LaneDetection0_conv2_b.bin",
      0);
    (static_cast<MWNormLayer *>(this->layers[6]))->createNormLayer
      (this->targetImpl, this->layers[5]->getOutputTensor(0), 5, 0.0001, 0.75,
       1.0, 1);
    (static_cast<MWMaxPoolingLayer *>(this->layers[7]))->createMaxPoolingLayer
      (this->targetImpl, this->layers[6]->getOutputTensor(0), 3, 3, 2, 2, 0, 0,
       0, 0, 0, 1, 0);
    (static_cast<MWFusedConvReLULayer *>(this->layers[8]))
      ->createFusedConvReLULayer(this->targetImpl, 1, this->layers[7]
      ->getOutputTensor(0), 3, 3, 256, 384, 1, 1, 1, 1, 1, 1, 1, 1, 1,
      "/home/hmcl/ADcontrol/slprj/_sfprj/LaneDetection/_self/sfun/src/cnn_trainedLaneNet0_LaneDetection0_conv3_w.bin",
      "/home/hmcl/ADcontrol/slprj/_sfprj/LaneDetection/_self/sfun/src/cnn_trainedLaneNet0_LaneDetection0_conv3_b.bin",
      1);
    (static_cast<MWFusedConvReLULayer *>(this->layers[9]))
      ->createFusedConvReLULayer(this->targetImpl, 1, this->layers[8]
      ->getOutputTensor(0), 3, 3, 192, 192, 1, 1, 1, 1, 1, 1, 1, 1, 2,
      "/home/hmcl/ADcontrol/slprj/_sfprj/LaneDetection/_self/sfun/src/cnn_trainedLaneNet0_LaneDetection0_conv4_w.bin",
      "/home/hmcl/ADcontrol/slprj/_sfprj/LaneDetection/_self/sfun/src/cnn_trainedLaneNet0_LaneDetection0_conv4_b.bin",
      0);
    (static_cast<MWFusedConvReLULayer *>(this->layers[10]))
      ->createFusedConvReLULayer(this->targetImpl, 1, this->layers[9]
      ->getOutputTensor(0), 3, 3, 192, 128, 1, 1, 1, 1, 1, 1, 1, 1, 2,
      "/home/hmcl/ADcontrol/slprj/_sfprj/LaneDetection/_self/sfun/src/cnn_trainedLaneNet0_LaneDetection0_conv5_w.bin",
      "/home/hmcl/ADcontrol/slprj/_sfprj/LaneDetection/_self/sfun/src/cnn_trainedLaneNet0_LaneDetection0_conv5_b.bin",
      1);
    (static_cast<MWMaxPoolingLayer *>(this->layers[11]))->createMaxPoolingLayer
      (this->targetImpl, this->layers[10]->getOutputTensor(0), 3, 3, 2, 2, 0, 0,
       0, 0, 0, 1, 0);
    (static_cast<MWFCLayer *>(this->layers[12]))->createFCLayer(this->targetImpl,
      this->layers[11]->getOutputTensor(0), 9216, 4096,
      "/home/hmcl/ADcontrol/slprj/_sfprj/LaneDetection/_self/sfun/src/cnn_trainedLaneNet0_LaneDetection0_fc6_w.bin",
      "/home/hmcl/ADcontrol/slprj/_sfprj/LaneDetection/_self/sfun/src/cnn_trainedLaneNet0_LaneDetection0_fc6_b.bin",
      1);
    (static_cast<MWReLULayer *>(this->layers[13]))->createReLULayer
      (this->targetImpl, this->layers[12]->getOutputTensor(0), 1);
    (static_cast<MWFCLayer *>(this->layers[14]))->createFCLayer(this->targetImpl,
      this->layers[13]->getOutputTensor(0), 4096, 16,
      "/home/hmcl/ADcontrol/slprj/_sfprj/LaneDetection/_self/sfun/src/cnn_trainedLaneNet0_LaneDetection0_fcLane1_w.bin",
      "/home/hmcl/ADcontrol/slprj/_sfprj/LaneDetection/_self/sfun/src/cnn_trainedLaneNet0_LaneDetection0_fcLane1_b.bin",
      0);
    (static_cast<MWReLULayer *>(this->layers[15]))->createReLULayer
      (this->targetImpl, this->layers[14]->getOutputTensor(0), 0);
    (static_cast<MWFCLayer *>(this->layers[16]))->createFCLayer(this->targetImpl,
      this->layers[15]->getOutputTensor(0), 16, 6,
      "/home/hmcl/ADcontrol/slprj/_sfprj/LaneDetection/_self/sfun/src/cnn_trainedLaneNet0_LaneDetection0_fcLane2_w.bin",
      "/home/hmcl/ADcontrol/slprj/_sfprj/LaneDetection/_self/sfun/src/cnn_trainedLaneNet0_LaneDetection0_fcLane2_b.bin",
      1);
    (static_cast<MWOutputLayer *>(this->layers[17]))->createOutputLayer
      (this->targetImpl, this->layers[16]->getOutputTensor(0), 1);
    this->outputTensors[0] = this->layers[17]->getOutputTensor(0);
    this->setSize();
  }
}

void c2_trainedLaneNet0_LaneDetection0::predict()
{
  int32_T c2_idx;
  for (c2_idx = 0; c2_idx < 18; c2_idx++) {
    this->layers[c2_idx]->predict();
  }
}

void c2_trainedLaneNet0_LaneDetection0::cleanup()
{
  int32_T c2_idx;
  this->deallocate();
  for (c2_idx = 0; c2_idx < 18; c2_idx++) {
    this->layers[c2_idx]->cleanup();
  }

  if (this->targetImpl) {
    this->targetImpl->cleanup();
  }
}

real32_T *c2_trainedLaneNet0_LaneDetection0::getLayerOutput(int32_T
  c2_layerIndex, int32_T c2_portIndex)
{
  return this->layers[c2_layerIndex]->getLayerOutput(c2_portIndex);
}

real32_T *c2_trainedLaneNet0_LaneDetection0::getInputDataPointer(int32_T
  c2_index)
{
  return (static_cast<MWTensor<real32_T> *>(this->inputTensors[c2_index]))
    ->getData();
}

real32_T *c2_trainedLaneNet0_LaneDetection0::getInputDataPointer()
{
  return (static_cast<MWTensor<real32_T> *>(this->inputTensors[0]))->getData();
}

real32_T *c2_trainedLaneNet0_LaneDetection0::getOutputDataPointer(int32_T
  c2_index)
{
  return (static_cast<MWTensor<real32_T> *>(this->outputTensors[c2_index]))
    ->getData();
}

real32_T *c2_trainedLaneNet0_LaneDetection0::getOutputDataPointer()
{
  return (static_cast<MWTensor<real32_T> *>(this->outputTensors[0]))->getData();
}

int32_T c2_trainedLaneNet0_LaneDetection0::getBatchSize()
{
  return this->inputTensors[0]->getBatchSize();
}

c2_trainedLaneNet0_LaneDetection0::~c2_trainedLaneNet0_LaneDetection0()
{
  int32_T c2_idx;
  this->cleanup();
  c2_checkCleanupCudaError(hipGetLastError(), __FILE__, __LINE__);
  for (c2_idx = 0; c2_idx < 18; c2_idx++) {
    delete this->layers[c2_idx];
  }

  if (this->targetImpl) {
    delete this->targetImpl;
  }

  delete this->inputTensors[0];
}

static void c2_checkCleanupCudaError(hipError_t c2_errCode, const char_T
  *c2_file, uint32_T c2_line)
{
  emlrtRTEInfo c2_rtInfo;
  if ((c2_errCode != hipSuccess) && (c2_errCode != hipErrorDeinitialized)) {
    c2_rtInfo = c2_createEmlrtInfoStruct(c2_file, c2_line);
    emlrtCUDAWarning(c2_errCode, hipGetErrorName(c2_errCode),
                     hipGetErrorString(c2_errCode), &c2_rtInfo);
  }
}

static emlrtRTEInfo c2_createEmlrtInfoStruct(const char_T *c2_file, uint32_T
  c2_line)
{
  emlrtRTEInfo c2_b_rtInfo;
  uint32_T c2_len;
  char_T *c2_brk;
  char_T *c2_fn;
  char_T *c2_pn;
  c2_len = (uint32_T)strlen(c2_file);
  c2_pn = (char_T *)calloc(c2_len + 1U, 1U);
  c2_fn = (char_T *)calloc(c2_len + 1U, 1U);
  memcpy(c2_pn, c2_file, c2_len);
  memcpy(c2_fn, c2_file, c2_len);
  c2_brk = strrchr(c2_fn, '.');
  *c2_brk = '\x00';
  c2_brk = NULL;
  c2_brk = strrchr(c2_fn, '/');
  if (c2_brk == NULL) {
    c2_brk = strrchr(c2_fn, '\\');
  }

  if (c2_brk == NULL) {
    c2_brk = c2_fn;
  } else {
    c2_brk++;
  }

  c2_b_rtInfo.lineNo = c2_line;
  c2_b_rtInfo.colNo = 0;
  c2_b_rtInfo.fName = c2_brk;
  c2_b_rtInfo.pName = c2_pn;
  return c2_b_rtInfo;
}

static void init_dsm_address_info(SFc2_LaneDetectionInstanceStruct
  *chartInstance)
{
}

static void init_simulink_io_address(SFc2_LaneDetectionInstanceStruct
  *chartInstance)
{
  chartInstance->c2_fEmlrtCtx = (void *)sfrtGetEmlrtCtx(chartInstance->S);
  chartInstance->c2_in_1 = (real_T (*)[154587])ssGetInputPortSignal_wrapper
    (chartInstance->S, 0);
  chartInstance->c2_out_1 = (real32_T (*)[6])ssGetOutputPortSignal_wrapper
    (chartInstance->S, 1);
}

#include <stdio.h>
#include <stdlib.h>
#include <string.h>

/* SFunction Glue Code */
void sf_c2_LaneDetection_get_check_sum(mxArray *plhs[])
{
  ((real_T *)mxGetPr((plhs[0])))[0] = (real_T)(2645480056U);
  ((real_T *)mxGetPr((plhs[0])))[1] = (real_T)(3584261826U);
  ((real_T *)mxGetPr((plhs[0])))[2] = (real_T)(751965951U);
  ((real_T *)mxGetPr((plhs[0])))[3] = (real_T)(2291345985U);
}

mxArray *sf_c2_LaneDetection_third_party_uses_info(void)
{
  mxArray * mxcell3p = mxCreateCellMatrix(1,1);
  mxSetCell(mxcell3p, 0, mxCreateString("dltargets.cudnn.cudnnApi"));
  return(mxcell3p);
}

mxArray *sf_c2_LaneDetection_jit_fallback_info(void)
{
  const char *infoFields[] = { "fallbackType", "fallbackReason",
    "hiddenFallbackType", "hiddenFallbackReason", "incompatibleSymbol" };

  mxArray *mxInfo = mxCreateStructMatrix(1, 1, 5, infoFields);
  mxArray *fallbackType = mxCreateString("pre");
  mxArray *fallbackReason = mxCreateString("GPUAcceleration");
  mxArray *hiddenFallbackType = mxCreateString("late");
  mxArray *hiddenFallbackReason = mxCreateString("ir_function_calls");
  mxArray *incompatibleSymbol = mxCreateString("#__setup__");
  mxSetField(mxInfo, 0, infoFields[0], fallbackType);
  mxSetField(mxInfo, 0, infoFields[1], fallbackReason);
  mxSetField(mxInfo, 0, infoFields[2], hiddenFallbackType);
  mxSetField(mxInfo, 0, infoFields[3], hiddenFallbackReason);
  mxSetField(mxInfo, 0, infoFields[4], incompatibleSymbol);
  return mxInfo;
}

mxArray *sf_c2_LaneDetection_updateBuildInfo_args_info(void)
{
  mxArray *mxBIArgs = sf_mex_decode(
    "eNpjYPT0ZQACPiDm4GRgYAPRQMzIAAGsSHxOJHGQ+h8M+NUzoal3QFLPgkU9H5J6ASg/MSUlOL+"
    "0KDnVLTMntRgi1kDAXkY0ewMI2CuDZi+I7xvumpOam5pXUp5ZnOqYlpaZl+qTWJlapJdcUAAz98"
    "MA+b+CRP97ELBXAs1eCbD/3UqLU1Oc8/PKglJ9QuF+B5n3YoD8XUCivwnZy41mL4ifnJcXn1iQC"
    "YvmYRXPgmj2CoLj2dnPDxy7nrkFOXrJpcMwf8uj2SuPJ3/DQgFkLkjTQPi/g0T/RxCwVxXNXlUC"
    "/vdOLcpLhaaF4ZQOpNHslcZRziHnhOGU/8XQ7BUD+z8ksSg9tcQvtaQ8vygb7vXhFO+SaPZKQsq"
    "90uKS/FxwhLvlFzmXuvj5wUt9BgYA8ZCiqg=="
    );
  return mxBIArgs;
}

static const mxArray *sf_get_sim_state_info_c2_LaneDetection(void)
{
  const char *infoFields[] = { "chartChecksum", "varInfo" };

  mxArray *mxInfo = mxCreateStructMatrix(1, 1, 2, infoFields);
  mxArray *mxVarInfo = sf_mex_decode(
    "eNpjYPT0ZQACPhDBxMDABqQ4IEwwYIXyGaFijHBxFri4AhCXVBakgsSLi5I9U4B0XmIumJ9YWuG"
    "Zl5YPNt+CAWE+GxbzGZHM54SKQ8AHe8r0izig62fBop8VSb8AlJ9fWhJvCAsfKE2+OxQcKNMPsT"
    "+AgD+k0PwB4mcWxycml2SWpcYnG8X7JOaluqSWpAIF8vMQ5oIAAB9fGm0="
    );
  mxArray *mxChecksum = mxCreateDoubleMatrix(1, 4, mxREAL);
  sf_c2_LaneDetection_get_check_sum(&mxChecksum);
  mxSetField(mxInfo, 0, infoFields[0], mxChecksum);
  mxSetField(mxInfo, 0, infoFields[1], mxVarInfo);
  return mxInfo;
}

static const char* sf_get_instance_specialization(void)
{
  return "soqOeZDjTPcCrKLUGh7SS3D";
}

static void sf_opaque_initialize_c2_LaneDetection(void *chartInstanceVar)
{
  initialize_params_c2_LaneDetection((SFc2_LaneDetectionInstanceStruct*)
    chartInstanceVar);
  initialize_c2_LaneDetection((SFc2_LaneDetectionInstanceStruct*)
    chartInstanceVar);
}

static void sf_opaque_enable_c2_LaneDetection(void *chartInstanceVar)
{
  enable_c2_LaneDetection((SFc2_LaneDetectionInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_disable_c2_LaneDetection(void *chartInstanceVar)
{
  disable_c2_LaneDetection((SFc2_LaneDetectionInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_gateway_c2_LaneDetection(void *chartInstanceVar)
{
  sf_gateway_c2_LaneDetection((SFc2_LaneDetectionInstanceStruct*)
    chartInstanceVar);
}

static const mxArray* sf_opaque_get_sim_state_c2_LaneDetection(SimStruct* S)
{
  return get_sim_state_c2_LaneDetection((SFc2_LaneDetectionInstanceStruct *)
    sf_get_chart_instance_ptr(S));     /* raw sim ctx */
}

static void sf_opaque_set_sim_state_c2_LaneDetection(SimStruct* S, const mxArray
  *st)
{
  set_sim_state_c2_LaneDetection((SFc2_LaneDetectionInstanceStruct*)
    sf_get_chart_instance_ptr(S), st);
}

static void sf_opaque_cleanup_runtime_resources_c2_LaneDetection(void
  *chartInstanceVar)
{
  if (chartInstanceVar!=NULL) {
    SimStruct *S = ((SFc2_LaneDetectionInstanceStruct*) chartInstanceVar)->S;
    if (sim_mode_is_rtw_gen(S) || sim_mode_is_external(S)) {
      sf_clear_rtw_identifier(S);
      unload_LaneDetection_optimization_info();
    }

    mdl_cleanup_runtime_resources_c2_LaneDetection
      ((SFc2_LaneDetectionInstanceStruct*) chartInstanceVar);
    ((SFc2_LaneDetectionInstanceStruct*) chartInstanceVar)->
      ~SFc2_LaneDetectionInstanceStruct();
    utFree(chartInstanceVar);
    if (ssGetUserData(S)!= NULL) {
      sf_free_ChartRunTimeInfo(S);
    }

    ssSetUserData(S,NULL);
  }
}

static void sf_opaque_mdl_start_c2_LaneDetection(void *chartInstanceVar)
{
  mdl_start_c2_LaneDetection((SFc2_LaneDetectionInstanceStruct*)
    chartInstanceVar);
}

static void sf_opaque_mdl_terminate_c2_LaneDetection(void *chartInstanceVar)
{
  mdl_terminate_c2_LaneDetection((SFc2_LaneDetectionInstanceStruct*)
    chartInstanceVar);
}

static void sf_opaque_init_subchart_simstructs(void *chartInstanceVar)
{
  initSimStructsc2_LaneDetection((SFc2_LaneDetectionInstanceStruct*)
    chartInstanceVar);
}

extern unsigned int sf_machine_global_initializer_called(void);
static void mdlProcessParameters_c2_LaneDetection(SimStruct *S)
{
  int i;
  for (i=0;i<ssGetNumRunTimeParams(S);i++) {
    if (ssGetSFcnParamTunable(S,i)) {
      ssUpdateDlgParamAsRunTimeParam(S,i);
    }
  }

  sf_warn_if_symbolic_dimension_param_changed(S);
  if (sf_machine_global_initializer_called()) {
    initialize_params_c2_LaneDetection((SFc2_LaneDetectionInstanceStruct*)
      sf_get_chart_instance_ptr(S));
    initSimStructsc2_LaneDetection((SFc2_LaneDetectionInstanceStruct*)
      sf_get_chart_instance_ptr(S));
  }
}

const char* sf_c2_LaneDetection_get_post_codegen_info(void)
{
  int i;
  const char* encStrCodegen [22] = {
    "eNrtV81v40QUd6ruipVgVSEkpBUfe0FwQKQfy8cJ2uYDItJthdNF4hKm4+d4NuMZd2actH8JV44",
    "cOfBH8Cdw4oLEGXHmwBvbTVPHTpqNSBO0kVx37N9783uf8+xUWkcO/h7iFb3jOPfx/gpeG076u5",
    "etK2NX+nzTeT9b/4JCIg5PiCKhdqb+BAnhG9CSx4ZJ0RK+LIQx4YMCQREbSWXKtGkWxpyJfjMW1",
    "OrT3waMBm4gY+4doizxjgW/RG1RbE5QT50poKYJ4JlAybgXNDnpjRgrM6wFQPs6DqeZoMG4cWRp",
    "6aOYGxZxaFwAbQltCDLW19xcQwzUzEWpmdZS7V4BZRhxRkShtQHRLkToYAOnkYd/j2ODRuVhNCD",
    "KHEJABqDbrJ/olALyOpnGF2dMECMVI7wR8poVnOR2wpHPkfSAT3EIcjtUQPqRZMKUx99toqUNQc",
    "441OEs7pVrc+E8tsF/xmAIqtRvfk0OQJEeHIvSTROHNC6SaI2yZBJmWAjPiDqgGD8NXmn2YuZol",
    "2CcoIMSZTBIjGzpjmIDdG+ptjhs2cycVTJxmAZbz4Il2hoDmBaFkbYmFTXCuS6FdWTUhgHwRGud",
    "GDIdlmotxmnNvI5EB9v0Lq+GWDAMfAarSeGxwnANcoCk7zzFxnITSWNtZFjD5K2325OvJ2EtYUD",
    "5hEJRF1CEaUCfJe4t1+YxbWOPQGRlEnpF4DRDZqEc7ceiPpSqjz6Z0kSuTbARLQWGuoexxEo41V",
    "g002A2lrNwlNAAPNtgGIcjLBvEFvhE29Z2gHU3YOayDpoqFhVENcaqwzbUsAl1GcGp6As5FE0lQ",
    "zfr8WkUsDNgDw8xBp2kxgRFVUwbbBfsensPAJOSKMFE7xDbnLpsIsnCiNlzb9u5PvdevcW5dyWX",
    "v38wpqdSoMcZu+f3fbAxfd8N/K+Sye2Pyb2W22czJ2dxW3j98Gv4x4eEf99675+3//y79vsi+/+",
    "8Md+c8DBbv3XVkEcFNpjIa4v9aozXZoH+N8f0b2VrLc+P4bv6884Jramv26dfBp+67l49jc8Mvh",
    "s5vlfPH9uTAbMxyWNFW142wNg1idNj3er/bIzv/Rn+eJA9T39/fbGY/Bv7efkif93L+cuuZWy6O",
    "7n8fXEej/cXk0/3P5lhx6OcHY+SOaBLbHeBLt3ttonA2cJAcsxP1ue8+f5S7qXcKstVXvD8Wrac",
    "s2S5Re2b9xxfNfz2lD7q5PBbK2zHovPVf43/zZlvDno3W38++iSqBYx7BdNx9hpnWL/o7f8kT21",
    "RTsO/nsP/OGM+kLm8tutqIEOoBiHl1YM6fiwYJXlV80g9r3a1b283hgZ8CNyv2g+vKo58VSpE1+",
    "C3nwDP4p6C2b45ZWx3fWof7HaHH50xsQY8z27NM8zxDJfCE5UO9jJvOivOcyfz5qrz3F0Tnjtzx",
    "T3K8YyWwtMjhnQ1JRws1dX255M1ifvemvD8eE360u5cPO/yPNqZ49y8y7ivR34+uRH3n2bwPM/x",
    "PF9e/5S+r8FkXG873/VzfPtLytNPRtFfdZ7DuXjebd2nPv0XWQnEqA==",
    ""
  };

  static char newstr [1557] = "";
  newstr[0] = '\0';
  for (i = 0; i < 22; i++) {
    strcat(newstr, encStrCodegen[i]);
  }

  return newstr;
}

static void mdlSetWorkWidths_c2_LaneDetection(SimStruct *S)
{
  const char* newstr = sf_c2_LaneDetection_get_post_codegen_info();
  sf_set_work_widths(S, newstr);
  ssSetChecksum0(S,(3815622807U));
  ssSetChecksum1(S,(1617715500U));
  ssSetChecksum2(S,(503063881U));
  ssSetChecksum3(S,(3728995046U));
}

static void mdlRTW_c2_LaneDetection(SimStruct *S)
{
  if (sim_mode_is_rtw_gen(S)) {
    ssWriteRTWStrParam(S, "StateflowChartType", "Embedded MATLAB");
  }
}

static void mdlSetupRuntimeResources_c2_LaneDetection(SimStruct *S)
{
  SFc2_LaneDetectionInstanceStruct *chartInstance;
  chartInstance = (SFc2_LaneDetectionInstanceStruct *)utMalloc(sizeof
    (SFc2_LaneDetectionInstanceStruct));
  if (chartInstance==NULL) {
    sf_mex_error_message("Could not allocate memory for chart instance.");
  }

  memset(chartInstance, 0, sizeof(SFc2_LaneDetectionInstanceStruct));
  chartInstance = new (chartInstance) SFc2_LaneDetectionInstanceStruct;
  chartInstance->chartInfo.chartInstance = chartInstance;
  if (ssGetSampleTime(S, 0) == CONTINUOUS_SAMPLE_TIME && ssGetOffsetTime(S, 0) ==
      0 && ssGetNumContStates(ssGetRootSS(S)) > 0) {
    sf_error_out_about_continuous_sample_time_with_persistent_vars(S);
  }

  chartInstance->chartInfo.isEMLChart = 1;
  chartInstance->chartInfo.chartInitialized = 0;
  chartInstance->chartInfo.sFunctionGateway = sf_opaque_gateway_c2_LaneDetection;
  chartInstance->chartInfo.initializeChart =
    sf_opaque_initialize_c2_LaneDetection;
  chartInstance->chartInfo.mdlStart = sf_opaque_mdl_start_c2_LaneDetection;
  chartInstance->chartInfo.mdlTerminate =
    sf_opaque_mdl_terminate_c2_LaneDetection;
  chartInstance->chartInfo.mdlCleanupRuntimeResources =
    sf_opaque_cleanup_runtime_resources_c2_LaneDetection;
  chartInstance->chartInfo.enableChart = sf_opaque_enable_c2_LaneDetection;
  chartInstance->chartInfo.disableChart = sf_opaque_disable_c2_LaneDetection;
  chartInstance->chartInfo.getSimState =
    sf_opaque_get_sim_state_c2_LaneDetection;
  chartInstance->chartInfo.setSimState =
    sf_opaque_set_sim_state_c2_LaneDetection;
  chartInstance->chartInfo.getSimStateInfo =
    sf_get_sim_state_info_c2_LaneDetection;
  chartInstance->chartInfo.zeroCrossings = NULL;
  chartInstance->chartInfo.outputs = NULL;
  chartInstance->chartInfo.derivatives = NULL;
  chartInstance->chartInfo.mdlRTW = mdlRTW_c2_LaneDetection;
  chartInstance->chartInfo.mdlSetWorkWidths = mdlSetWorkWidths_c2_LaneDetection;
  chartInstance->chartInfo.callGetHoverDataForMsg = NULL;
  chartInstance->chartInfo.extModeExec = NULL;
  chartInstance->chartInfo.restoreLastMajorStepConfiguration = NULL;
  chartInstance->chartInfo.restoreBeforeLastMajorStepConfiguration = NULL;
  chartInstance->chartInfo.storeCurrentConfiguration = NULL;
  chartInstance->chartInfo.callAtomicSubchartUserFcn = NULL;
  chartInstance->chartInfo.callAtomicSubchartAutoFcn = NULL;
  chartInstance->chartInfo.callAtomicSubchartEventFcn = NULL;
  chartInstance->chartInfo.chartStateSetterFcn = NULL;
  chartInstance->chartInfo.chartStateGetterFcn = NULL;
  chartInstance->S = S;
  chartInstance->chartInfo.dispatchToExportedFcn = NULL;
  sf_init_ChartRunTimeInfo(S, &(chartInstance->chartInfo), false, 0,
    chartInstance->c2_JITStateAnimation,
    chartInstance->c2_JITTransitionAnimation);
  init_dsm_address_info(chartInstance);
  init_simulink_io_address(chartInstance);
  if (!sim_mode_is_rtw_gen(S)) {
  }

  mdl_setup_runtime_resources_c2_LaneDetection(chartInstance);
}

void c2_LaneDetection_method_dispatcher(SimStruct *S, int_T method, void *data)
{
  switch (method) {
   case SS_CALL_MDL_SETUP_RUNTIME_RESOURCES:
    mdlSetupRuntimeResources_c2_LaneDetection(S);
    break;

   case SS_CALL_MDL_SET_WORK_WIDTHS:
    mdlSetWorkWidths_c2_LaneDetection(S);
    break;

   case SS_CALL_MDL_PROCESS_PARAMETERS:
    mdlProcessParameters_c2_LaneDetection(S);
    break;

   default:
    /* Unhandled method */
    sf_mex_error_message("Stateflow Internal Error:\n"
                         "Error calling c2_LaneDetection_method_dispatcher.\n"
                         "Can't handle method %d.\n", method);
    break;
  }
}
