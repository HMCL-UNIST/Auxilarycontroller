#include "MWTargetNetworkImpl.hpp"
#include "MWCNNLayerImpl.hpp"
#include <cassert>
#include <cmath>
#include <algorithm>
 void MWTargetNetworkImpl::allocate(int BufSize, int numBufsToAlloc) { numBufs 
= numBufsToAlloc; for(int i = 0; i < numBufs; i++) { float *memPtr = 0; 
CUDA_CALL(hipMalloc((void**)&memPtr, sizeof(float)*BufSize)); 
memBuffer.push_back(memPtr); }  } void 
MWTargetNetworkImpl::allocatePermuteBuffers(int bufSize, int numBufsToAlloc) { 
for (int i = 0; i < numBufsToAlloc; i++) { float* memPtr = 0; 
CUDA_CALL(hipMalloc((void**)&memPtr, sizeof(float) * bufSize)); 
mbKaFvmHqfBiTISNPGKJ.push_back(memPtr); } } void 
MWTargetNetworkImpl::preSetup() {  NbunkIVaMPVYgAQHXXYd = new 
hipblasHandle_t; if(!NbunkIVaMPVYgAQHXXYd) { 
MWCNNLayerImpl::throwAllocationError(__LINE__ , __FILE__); } 
hipblasCreate(NbunkIVaMPVYgAQHXXYd); NldNILHvuQqQPSAHXxdT = new 
hipdnnHandle_t; if(!NldNILHvuQqQPSAHXxdT) { 
MWCNNLayerImpl::throwAllocationError(__LINE__ , __FILE__); } 
hipdnnCreate(NldNILHvuQqQPSAHXxdT);  } void 
MWTargetNetworkImpl::postSetup(MWCNNLayer* layers[],int numLayers) { if 
(*getProposedWorkSpaceSize() > *getAllocatedWorkSpaceSize()) { if 
(xcusoQxPPodcHwVviCWI) { destroyWorkSpace(xcusoQxPPodcHwVviCWI); } 
createWorkSpace(xcusoQxPPodcHwVviCWI); while ((!xcusoQxPPodcHwVviCWI) && 
(*getProposedWorkSpaceSize() > 0)) { 
setProposedWorkSpaceSize(MWTargetNetworkImpl::getNextProposedWorkSpaceSize(*getProposedWorkSpaceSize())); 
createWorkSpace(xcusoQxPPodcHwVviCWI); } } for (int i = 0; i < numLayers; i++) 
{ layers[i]->postSetup();  }  } size_t 
MWTargetNetworkImpl::getNextProposedWorkSpaceSize(size_t failedWorkSpaceSize) { 
assert(failedWorkSpaceSize > 0); return failedWorkSpaceSize/2; } void 
MWTargetNetworkImpl::createWorkSpace(float*& xkUNToJIgvoLoUQuzKRF) { 
hipError_t rlQsibXJSWJVnUVpdNeL = hipMalloc((void**)&xkUNToJIgvoLoUQuzKRF, 
*getProposedWorkSpaceSize()); if (rlQsibXJSWJVnUVpdNeL != hipSuccess) { 
xkUNToJIgvoLoUQuzKRF = NULL; setAllocatedWorkSpaceSize(0);  
rlQsibXJSWJVnUVpdNeL = hipGetLastError();  } else { 
setAllocatedWorkSpaceSize(*getProposedWorkSpaceSize()); } } void 
MWTargetNetworkImpl::destroyWorkSpace(float*& xkUNToJIgvoLoUQuzKRF) { 
CUDA_FREE_CALL(xkUNToJIgvoLoUQuzKRF); xkUNToJIgvoLoUQuzKRF = NULL; 
setAllocatedWorkSpaceSize(0);  } void 
MWTargetNetworkImpl::setProposedWorkSpaceSize(size_t wss) { 
nNULvWnBXnnWdpEkHPAH = wss;  } size_t* 
MWTargetNetworkImpl::getProposedWorkSpaceSize() { return 
&nNULvWnBXnnWdpEkHPAH; } void 
MWTargetNetworkImpl::setAllocatedWorkSpaceSize(size_t wss) { 
GvDXGhRLfipwBoRPoGfI = wss;  } size_t* 
MWTargetNetworkImpl::getAllocatedWorkSpaceSize() { return 
&GvDXGhRLfipwBoRPoGfI; } float* 
MWTargetNetworkImpl::getWorkSpace() { return xcusoQxPPodcHwVviCWI; } float* 
MWTargetNetworkImpl::getPermuteBuffer(int bufIndex) { return 
mbKaFvmHqfBiTISNPGKJ[bufIndex]; } hipblasHandle_t* 
MWTargetNetworkImpl::getCublasHandle() { return NbunkIVaMPVYgAQHXXYd; } 
hipdnnHandle_t* MWTargetNetworkImpl::getCudnnHandle() { return 
NldNILHvuQqQPSAHXxdT; } void MWTargetNetworkImpl::setAutoTune(bool 
autotune) { MW_autoTune = autotune; } bool MWTargetNetworkImpl::getAutoTune() 
const { return MW_autoTune; } void MWTargetNetworkImpl::deallocate() { for(int 
i = 0; i < memBuffer.size(); i++) { float *memPtr = memBuffer[i]; if(memPtr) { 
CUDA_FREE_CALL(memPtr); }  } memBuffer.clear(); for(int i = 0; i < 
mbKaFvmHqfBiTISNPGKJ.size(); i++) { float *memPtr = 
mbKaFvmHqfBiTISNPGKJ[i]; if(memPtr) { CUDA_FREE_CALL(memPtr); } } 
mbKaFvmHqfBiTISNPGKJ.clear(); } void MWTargetNetworkImpl::cleanup() { if 
(xcusoQxPPodcHwVviCWI) { destroyWorkSpace(xcusoQxPPodcHwVviCWI); } if 
(NbunkIVaMPVYgAQHXXYd) { hipblasDestroy(*NbunkIVaMPVYgAQHXXYd); delete 
NbunkIVaMPVYgAQHXXYd; } if (NldNILHvuQqQPSAHXxdT) {
#if (CUDNN_MAJOR < 8) 
 hipdnnDestroy(*NldNILHvuQqQPSAHXxdT);
#else
 hipError_t hipError_t = hipPeekAtLastError(); if (hipError_t != 
hipErrorDeinitialized) { hipdnnDestroy(*NldNILHvuQqQPSAHXxdT); }
#endif
 delete NldNILHvuQqQPSAHXxdT; } } float * 
MWTargetNetworkImpl::getBufferPtr(int bufferIndex) { return 
memBuffer[bufferIndex];  } MWTargetNetworkImpl::~MWTargetNetworkImpl() { }