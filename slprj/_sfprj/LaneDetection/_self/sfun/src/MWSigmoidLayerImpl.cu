#include "MWSigmoidLayer.hpp"
#include "MWSigmoidLayerImpl.hpp"
#include "MWTargetNetworkImpl.hpp"
#include <stdarg.h>
#include <cassert>
 MWSigmoidLayerImpl::MWSigmoidLayerImpl(MWCNNLayer* layer, MWTargetNetworkImpl* 
ntwk_impl) : MWCNNLayerImpl(layer, ntwk_impl)  { 
CUDNN_CALL(hipdnnCreateActivationDescriptor(&rkzbRnJPJHmyWmkoOrFj)); 
createAndAddDescriptor(getLayer()->getOutputTensor()->getSourcePortIndex()); } 
MWSigmoidLayerImpl::~MWSigmoidLayerImpl() { } void 
MWSigmoidLayerImpl::propagateSize() { MWTensorBase* opTensor = 
getLayer()->getOutputTensor(0); hipdnnTensorDescriptor_t* desc = 
getDescriptor(opTensor->getSourcePortIndex()); assert(desc); 
setDescriptor<float>(*desc, static_cast<MWTensor<float>*>(opTensor)); 
CUDNN_CALL(hipdnnSetActivationDescriptor(rkzbRnJPJHmyWmkoOrFj, 
HIPDNN_ACTIVATION_SIGMOID,  HIPDNN_NOT_PROPAGATE_NAN, 0));  } void 
MWSigmoidLayerImpl::predict() { MWSigmoidLayer* SigmoidLayer = 
static_cast<MWSigmoidLayer*>(getLayer()); MWTensorBase* ipTensorBase = 
SigmoidLayer->getInputTensor(0);  MWTensorBase* opTensorBase = 
SigmoidLayer->getOutputTensor(0); MWTensor<float>* ipTensor = 
static_cast<MWTensor<float>*>(ipTensorBase); MWTensor<float>* opTensor = 
static_cast<MWTensor<float>*>(opTensorBase); hipdnnTensorDescriptor_t* desc = 
getDescriptor(opTensor->getSourcePortIndex()); assert(desc); 
hipdnnTensorDescriptor_t ipDesc = 
MWCNNLayerImpl::getCuDNNDescriptor(ipTensorBase); 
CUDNN_CALL(hipdnnActivationForward(*dJcdBfQQLhIAYHPxwQeg->getCudnnHandle(), 
rkzbRnJPJHmyWmkoOrFj,  getOnePtr(), ipDesc, ipTensor->getData(), 
getZeroPtr(), *desc, opTensor->getData())); } void 
MWSigmoidLayerImpl::cleanup() { MWSigmoidLayer* SigmoidLayer = 
static_cast<MWSigmoidLayer*>(getLayer()); 
CUDNN_CALL(hipdnnDestroyActivationDescriptor(rkzbRnJPJHmyWmkoOrFj)); }