#include "MWYoloSoftmaxLayer.hpp"
#include "MWYoloSoftmaxLayerImpl.hpp"
#include "MWTargetNetworkImpl.hpp"
 MWYoloSoftmaxLayerImpl::MWYoloSoftmaxLayerImpl(MWCNNLayer* layer, 
MWTargetNetworkImpl* ntwk_impl, int HoqiuUUuJnXGkfDodicJ) : 
MWCNNLayerImpl(layer, ntwk_impl) , 
dkLDkRwCBjeybwDHbKiE(HoqiuUUuJnXGkfDodicJ) { 
tCfVGVGaqfGdJypAKQqq = new hipdnnTensorDescriptor_t; if 
(!tCfVGVGaqfGdJypAKQqq) { 
MWCNNLayerImpl::throwAllocationError(__LINE__ , __FILE__); }  
CUDNN_CALL(hipdnnCreateTensorDescriptor(tCfVGVGaqfGdJypAKQqq)); 
} MWYoloSoftmaxLayerImpl::~MWYoloSoftmaxLayerImpl() { } void 
MWYoloSoftmaxLayerImpl::propagateSize() { MWTensorBase* ipTensor = 
getLayer()->getInputTensor(0); MWTensorBase* opTensor = 
getLayer()->getOutputTensor(0); int eUSuiwvLvXVXrpUkgBVu = 
ipTensor->getChannels() / dkLDkRwCBjeybwDHbKiE; 
CUDNN_CALL(hipdnnSetTensor4dDescriptor(*tCfVGVGaqfGdJypAKQqq, 
HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, opTensor->getBatchSize(), 
eUSuiwvLvXVXrpUkgBVu, dkLDkRwCBjeybwDHbKiE, opTensor->getWidth() * 
opTensor->getHeight())); } void MWYoloSoftmaxLayerImpl::predict() { 
MWTensorBase* ipTensorBase = getLayer()->getInputTensor(0); MWTensorBase* 
opTensorBase = getLayer()->getOutputTensor(0); MWTensor<float>* ipTensor = 
static_cast<MWTensor<float>*>(ipTensorBase); MWTensor<float>* opTensor = 
static_cast<MWTensor<float>*>(opTensorBase); CUDNN_CALL(hipdnnSoftmaxForward( 
*dJcdBfQQLhIAYHPxwQeg->getCudnnHandle(), HIPDNN_SOFTMAX_ACCURATE, 
HIPDNN_SOFTMAX_MODE_CHANNEL, getOnePtr(), *tCfVGVGaqfGdJypAKQqq, 
ipTensor->getData(), getZeroPtr(), *tCfVGVGaqfGdJypAKQqq, 
opTensor->getData())); } void MWYoloSoftmaxLayerImpl::cleanup() { 
CUDNN_CALL(hipdnnDestroyTensorDescriptor(*tCfVGVGaqfGdJypAKQqq)); 
}