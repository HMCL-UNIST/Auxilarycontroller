#include "hip/hip_runtime.h"
/* Include files */

#include "LaneDetection_sfun.h"
#include "c5_LaneDetection.h"
#include <string.h>
#include "MWCudaDimUtility.hpp"
#include "MWLaunchParametersUtilities.hpp"
#include "MWShuffleUtility.h"
#include "MWSortFunctors.h"
#include "MWSortWithIndexUtility.h"
#include "mwmathutil.h"
#ifdef utFree
#undef utFree
#endif

#ifdef utMalloc
#undef utMalloc
#endif

#ifdef __cplusplus

extern "C" void *utMalloc(size_t size);
extern "C" void utFree(void*);

#else

extern void *utMalloc(size_t size);
extern void utFree(void*);

#endif

#include <cstdlib>
#include <cstring>

/* Type Definitions */

/* Named Constants */
const int32_T CALL_EVENT = -1;

/* Variable Declarations */

/* Variable Definitions */
static real_T _sfTime_;

/* Function Declarations */
static void initialize_c5_LaneDetection(SFc5_LaneDetectionInstanceStruct
  *chartInstance);
static void initialize_params_c5_LaneDetection(SFc5_LaneDetectionInstanceStruct *
  chartInstance);
static void enable_c5_LaneDetection(SFc5_LaneDetectionInstanceStruct
  *chartInstance);
static void disable_c5_LaneDetection(SFc5_LaneDetectionInstanceStruct
  *chartInstance);
static void c5_do_animation_call_c5_LaneDetection
  (SFc5_LaneDetectionInstanceStruct *chartInstance);
static void ext_mode_exec_c5_LaneDetection(SFc5_LaneDetectionInstanceStruct
  *chartInstance);
static const mxArray *get_sim_state_c5_LaneDetection
  (SFc5_LaneDetectionInstanceStruct *chartInstance);
static void set_sim_state_c5_LaneDetection(SFc5_LaneDetectionInstanceStruct
  *chartInstance, const mxArray *c5_st);
static void sf_gateway_c5_LaneDetection(SFc5_LaneDetectionInstanceStruct
  *chartInstance);
static void mdl_start_c5_LaneDetection(SFc5_LaneDetectionInstanceStruct
  *chartInstance);
static void mdl_terminate_c5_LaneDetection(SFc5_LaneDetectionInstanceStruct
  *chartInstance);
static void mdl_setup_runtime_resources_c5_LaneDetection
  (SFc5_LaneDetectionInstanceStruct *chartInstance);
static void mdl_cleanup_runtime_resources_c5_LaneDetection
  (SFc5_LaneDetectionInstanceStruct *chartInstance);
static void initSimStructsc5_LaneDetection(SFc5_LaneDetectionInstanceStruct
  *chartInstance);
static void c5_eML_blk_kernel(SFc5_LaneDetectionInstanceStruct *chartInstance,
  real32_T c5_b_In[921600], real_T c5_b_bboxes_data[], int32_T c5_bboxes_size[2],
  real32_T c5_b_scores_data[], int32_T c5_scores_size[2]);
static void c5_DeepLearningNetwork_setup(SFc5_LaneDetectionInstanceStruct
  *chartInstance, c5_yolov2ResNet50VehicleExample0_LaneDetection0 *c5_obj);
static real32_T c5_callFcn(SFc5_LaneDetectionInstanceStruct *chartInstance,
  real32_T c5_input1, real32_T c5_input2);
static real32_T c5_b_callFcn(SFc5_LaneDetectionInstanceStruct *chartInstance,
  real32_T c5_input1, real32_T c5_input2);
static void c5_DeepLearningNetwork_activations(SFc5_LaneDetectionInstanceStruct *
  chartInstance, c5_yolov2ResNet50VehicleExample0_LaneDetection0 *c5_obj,
  real32_T c5_varargin_1[150528], real32_T c5_b_out[4704]);
static void c5_indexShapeCheck(SFc5_LaneDetectionInstanceStruct *chartInstance,
  int32_T c5_matrixSize, int32_T c5_indexSize[2]);
static void c5_emlrt_marshallIn(SFc5_LaneDetectionInstanceStruct *chartInstance,
  const mxArray *c5_bboxes, const char_T *c5_identifier, real_T c5_y_data[],
  int32_T c5_y_size[2]);
static void c5_b_emlrt_marshallIn(SFc5_LaneDetectionInstanceStruct
  *chartInstance, const mxArray *c5_u, const emlrtMsgIdentifier *c5_parentId,
  real_T c5_y_data[], int32_T c5_y_size[2]);
static void c5_c_emlrt_marshallIn(SFc5_LaneDetectionInstanceStruct
  *chartInstance, const mxArray *c5_scores, const char_T *c5_identifier,
  real32_T c5_y_data[], int32_T c5_y_size[2]);
static void c5_d_emlrt_marshallIn(SFc5_LaneDetectionInstanceStruct
  *chartInstance, const mxArray *c5_u, const emlrtMsgIdentifier *c5_parentId,
  real32_T c5_y_data[], int32_T c5_y_size[2]);
static uint8_T c5_e_emlrt_marshallIn(SFc5_LaneDetectionInstanceStruct
  *chartInstance, const mxArray *c5_b_is_active_c5_LaneDetection, const char_T
  *c5_identifier);
static uint8_T c5_f_emlrt_marshallIn(SFc5_LaneDetectionInstanceStruct
  *chartInstance, const mxArray *c5_u, const emlrtMsgIdentifier *c5_parentId);
static void c5_emxConvertDynamicMatrixFromEmx_(SFc5_LaneDetectionInstanceStruct *
  chartInstance, c5_emxArray_real_T_20x4 *c5_emx, real_T c5_data[80], int32_T
  c5_size[2]);
static void c5_b_emxConvertDynamicMatrixFromEmx_
  (SFc5_LaneDetectionInstanceStruct *chartInstance, c5_emxArray_real32_T_20x1
   *c5_emx, real32_T c5_data[20], int32_T c5_size[2]);
static __global__ void c5_coder_reduce0(const real32_T c5_inputVar[150528],
  real32_T *c5_outputVar);
static __device__ real32_T c5_threadGroupReduction(real32_T c5_val, uint32_T
  c5_lane, uint32_T c5_mask);
static __device__ real32_T c5_shflDown1(real32_T c5_in1, uint32_T c5_offset,
  uint32_T c5_mask);
static __device__ real32_T c5_workGroupReduction(real32_T c5_val, uint32_T
  c5_mask, uint32_T c5_numActiveWarps);
static __device__ real32_T c5_b_threadGroupReduction(real32_T c5_val, uint32_T
  c5_lane, uint32_T c5_mask);
static __device__ real32_T c5_b_workGroupReduction(real32_T c5_val, uint32_T
  c5_mask, uint32_T c5_numActiveWarps);
static __device__ real32_T c5_atomicOpreal32_T(real32_T *c5_address, real32_T
  c5_value);
static __device__ real32_T c5_b_atomicOpreal32_T(real32_T *c5_address, real32_T
  c5_value);
static __global__ void c5_eML_blk_kernel_kernel1(int16_T c5_aux1[960]);
static __global__ void c5_eML_blk_kernel_kernel2(int16_T c5_aux2[1280]);
static __global__ void c5_eML_blk_kernel_kernel3(const int16_T c5_aux1[960],
  real_T c5_rowWeights[2016], int16_T c5_ipRowIndices[2016]);
static __global__ void c5_eML_blk_kernel_kernel4(const int16_T c5_aux2[1280],
  real_T c5_colWeights[2688], int16_T c5_ipColIndices[2688]);
static __global__ void c5_eML_blk_kernel_kernel5(const real_T c5_rowWeights[2016],
  real_T c5_rowWeightsTotal[224]);
static __global__ void c5_eML_blk_kernel_kernel6(const real_T c5_rowWeights[2016],
  const int32_T c5_xoffset, real_T c5_rowWeightsTotal[224]);
static __global__ void c5_eML_blk_kernel_kernel7(const real_T c5_colWeights[2688],
  real_T c5_colWeightsTotal[224]);
static __global__ void c5_eML_blk_kernel_kernel8(const real_T c5_colWeights[2688],
  const int32_T c5_xoffset, real_T c5_colWeightsTotal[224]);
static __global__ void c5_eML_blk_kernel_kernel9(const real_T
  c5_colWeightsTotal[224], const real_T c5_colWeights[2688], const real32_T
  c5_b_In[921600], const int16_T c5_ipColIndices[2688], real32_T
  c5_partialResize[322560]);
static __global__ void c5_eML_blk_kernel_kernel10(const real_T
  c5_rowWeightsTotal[224], const real_T c5_rowWeights[2016], const real32_T
  c5_partialResize[322560], const int16_T c5_ipRowIndices[2016], real32_T
  c5_b_out[150528]);
static __global__ void c5_eML_blk_kernel_kernel11(real32_T c5_b_out[150528],
  real32_T c5_outVal[2]);
static __global__ void c5_eML_blk_kernel_kernel12(const real32_T c5_y, const
  real32_T c5_outVal, real32_T c5_b_out[150528], real32_T c5_c_out[150528]);
static __global__ void c5_eML_blk_kernel_kernel13(const int8_T c5_dv[8], real_T
  c5_anchors[8]);
static __global__ void c5_eML_blk_kernel_kernel14(const real_T c5_dv1[4], real_T
  c5_anchors[4], real_T c5_b_anchors[8]);
static __global__ void c5_eML_blk_kernel_kernel15(const real_T c5_anchors[8],
  const real32_T c5_tmpFeatureMap[4704], real32_T c5_boxOut[4704]);
static __global__ void c5_eML_blk_kernel_kernel16(const real32_T c5_boxOut[4704],
  boolean_T c5_bv[784]);
static __global__ void c5_eML_blk_kernel_kernel17(const real32_T c5_boxOut[4704],
  const int16_T c5_ii_data[784], const int32_T c5_thresholdedPrediction_size[2],
  const int32_T c5_ii_size[1], real32_T c5_thresholdedPrediction_data[4704]);
static __global__ void c5_eML_blk_kernel_kernel18(const real32_T
  c5_thresholdedPrediction_data[4704], const int32_T
  c5_thresholdedPrediction_size[2], const int32_T c5_bboxesX1Y1X2Y2_size[2],
  const int32_T c5_i7, real_T c5_bboxesX1Y1X2Y2_data[3136]);
static __global__ void c5_eML_blk_kernel_kernel19(const real_T
  c5_bboxesX1Y1X2Y2_data[3136], const int32_T c5_i10, real_T c5_x1_data[784]);
static __global__ void c5_eML_blk_kernel_kernel20(const real_T
  c5_bboxesX1Y1X2Y2_data[3136], const int32_T c5_bboxesX1Y1X2Y2_size[2], const
  int32_T c5_i14, real_T c5_y1_data[784]);
static __global__ void c5_eML_blk_kernel_kernel21(const real_T
  c5_bboxesX1Y1X2Y2_data[3136], const int32_T c5_bboxesX1Y1X2Y2_size[2], const
  int32_T c5_i17, real_T c5_x2_data[784]);
static __global__ void c5_eML_blk_kernel_kernel22(const real_T
  c5_bboxesX1Y1X2Y2_data[3136], const int32_T c5_bboxesX1Y1X2Y2_size[2], const
  int32_T c5_i19, real_T c5_y2_data[784]);
static __global__ void c5_eML_blk_kernel_kernel23(const int32_T c5_end, real_T
  c5_x1_data[784]);
static __global__ void c5_eML_blk_kernel_kernel24(const int32_T c5_end, real_T
  c5_y1_data[784]);
static __global__ void c5_eML_blk_kernel_kernel25(const int32_T c5_end, real_T
  c5_x2_data[784]);
static __global__ void c5_eML_blk_kernel_kernel26(const int32_T c5_end, real_T
  c5_y2_data[784]);
static __global__ void c5_eML_blk_kernel_kernel27(const real_T c5_x1_data[784],
  const int32_T c5_loop_ub, real_T c5_bboxesX1Y1X2Y2_data[3136]);
static __global__ void c5_eML_blk_kernel_kernel28(const real_T c5_y1_data[784],
  const int32_T c5_bboxesX1Y1X2Y2_size[2], const int32_T c5_loop_ub, real_T
  c5_bboxesX1Y1X2Y2_data[3136]);
static __global__ void c5_eML_blk_kernel_kernel29(const real_T c5_x2_data[784],
  const int32_T c5_bboxesX1Y1X2Y2_size[2], const int32_T c5_loop_ub, real_T
  c5_bboxesX1Y1X2Y2_data[3136]);
static __global__ void c5_eML_blk_kernel_kernel30(const real_T c5_y2_data[784],
  const int32_T c5_bboxesX1Y1X2Y2_size[2], const int32_T c5_loop_ub, real_T
  c5_bboxesX1Y1X2Y2_data[3136]);
static __global__ void c5_eML_blk_kernel_kernel31(const real_T
  c5_bboxesX1Y1X2Y2_data[3136], const int32_T c5_loop_ub, real_T
  c5_bboxPred_data[3136]);
static __global__ void c5_eML_blk_kernel_kernel32(const real_T
  c5_bboxesX1Y1X2Y2_data[3136], const int32_T c5_bboxesX1Y1X2Y2_size[2], const
  int32_T c5_bboxPred_size[2], const int32_T c5_loop_ub, real_T
  c5_bboxPred_data[3136]);
static __global__ void c5_eML_blk_kernel_kernel33(const real_T
  c5_bboxesX1Y1X2Y2_data[3136], const int32_T c5_bboxesX1Y1X2Y2_size[2], const
  int32_T c5_bboxPred_size[2], const int32_T c5_loop_ub, real_T
  c5_bboxPred_data[3136]);
static __global__ void c5_eML_blk_kernel_kernel34(const real_T
  c5_bboxesX1Y1X2Y2_data[3136], const int32_T c5_bboxesX1Y1X2Y2_size[2], const
  int32_T c5_bboxPred_size[2], const int32_T c5_loop_ub, real_T
  c5_bboxPred_data[3136]);
static __global__ void c5_eML_blk_kernel_kernel35(const int32_T c5_nx, real_T
  c5_bboxPred_data[3136]);
static __global__ void c5_eML_blk_kernel_kernel36(const real_T c5_bboxPred_data
  [3136], const int32_T c5_bboxPred_size[2], const int32_T c5_i32, real_T
  c5_b_bboxPred_data[784]);
static __global__ void c5_eML_blk_kernel_kernel37(const real_T c5_bboxPred_data
  [784], const int32_T c5_bboxPred_size[2], const int32_T c5_b_bboxPred_size[1],
  real_T c5_b_bboxPred_data[3136]);
static __global__ void c5_eML_blk_kernel_kernel38(const real_T c5_bboxPred_data
  [3136], const int32_T c5_bboxPred_size[2], const int32_T c5_i35, real_T
  c5_b_bboxPred_data[784]);
static __global__ void c5_eML_blk_kernel_kernel39(const real_T c5_bboxPred_data
  [784], const int32_T c5_bboxPred_size[2], const int32_T c5_b_bboxPred_size[1],
  real_T c5_b_bboxPred_data[3136]);
static __global__ void c5_eML_blk_kernel_kernel40(const real_T c5_bboxPred_data
  [3136], const int32_T c5_bboxPred_size[2], const int32_T c5_i, const int32_T
  c5_b_bboxPred_size[2], const int32_T c5_count, real_T c5_b_bboxPred_data[3136]);
static __global__ void c5_eML_blk_kernel_kernel41(const real32_T
  c5_thresholdedPrediction_data[4704], const int32_T
  c5_thresholdedPrediction_size[2], const int32_T c5_i, const real_T c5_count,
  real32_T c5_classPred_data[784], real32_T c5_scorePred_data[784]);
static __global__ void c5_eML_blk_kernel_kernel42(const int32_T c5_i38, const
  int32_T c5_i39, int32_T c5_idx_data[784]);
static __global__ void c5_eML_blk_kernel_kernel43(const int32_T
  c5_bboxPred_size[2], boolean_T c5_b_data[784]);
static __global__ void c5_eML_blk_kernel_kernel44(const int32_T c5_b_size[2],
  const boolean_T c5_b_data[784], int32_T c5_i3, int32_T *c5_n);
static __device__ int32_T c5_c_threadGroupReduction(int32_T c5_val, uint32_T
  c5_lane, uint32_T c5_mask);
static __device__ int32_T c5_b_shflDown1(int32_T c5_in1, uint32_T c5_offset,
  uint32_T c5_mask);
static __device__ int32_T c5_c_workGroupReduction(int32_T c5_val, uint32_T
  c5_mask, uint32_T c5_numActiveWarps);
static __global__ void c5_eML_blk_kernel_kernel45(const int32_T *c5_n, const
  int32_T c5_bboxPred_size[2], int32_T *c5_nrows);
static __global__ void c5_eML_blk_kernel_kernel46(const int32_T
  c5_bboxPred_size[2], const int32_T *c5_nrows, const int32_T c5_idx_data[784],
  real_T c5_bboxPred_data[3136]);
static __global__ void c5_eML_blk_kernel_kernel47(const real_T c5_bboxPred_data
  [3136], const int32_T c5_bboxPred_size[2], const int32_T c5_b_bboxPred_size[2],
  const int32_T c5_i4, real_T c5_b_bboxPred_data[3136]);
static __global__ void c5_eML_blk_kernel_kernel48(const real_T c5_bboxPred_data
  [3136], const int32_T c5_bboxPred_size[2], real_T c5_b_bboxPred_data[3136]);
static __global__ void c5_eML_blk_kernel_kernel49(const int32_T c5_i45, const
  int32_T c5_i47, int32_T c5_idx_data[784]);
static __global__ void c5_eML_blk_kernel_kernel50(const int32_T
  c5_scorePred_size[1], boolean_T c5_b_data[784]);
static __global__ void c5_eML_blk_kernel_kernel51(const int32_T c5_b_size[2],
  const boolean_T c5_b_data[784], int32_T c5_i6, int32_T *c5_n);
static __global__ void c5_eML_blk_kernel_kernel52(const int32_T c5_i50, const
  int32_T c5_i51, int32_T c5_idx_data[784]);
static __global__ void c5_eML_blk_kernel_kernel53(const int32_T
  c5_classPred_size[1], boolean_T c5_b_data[784]);
static __global__ void c5_eML_blk_kernel_kernel54(const int32_T c5_b_size[2],
  const boolean_T c5_b_data[784], int32_T c5_i9, int32_T *c5_n);
static __global__ void c5_eML_blk_kernel_kernel55(const real32_T
  c5_scorePred_data[784], const int32_T c5_scorePred_size[1], real32_T
  c5_out_data[784]);
static __global__ void c5_eML_blk_kernel_kernel56(const uint32_T c5_dv2[2],
  real_T c5_x1_data[784]);
static __global__ void c5_eML_blk_kernel_kernel57(const real_T c5_bboxPred_data
  [3136], const int32_T c5_bboxPred_size[2], const real_T c5_x1_data[784], const
  int32_T c5_bboxesX1Y1X2Y2_size[2], const int32_T c5_x1_size[1], real_T
  c5_bboxesX1Y1X2Y2_data[3136]);
static __global__ void c5_eML_blk_kernel_kernel58(const real32_T
  c5_classPred_data[784], const real_T c5_x1_data[784], const int32_T
  c5_x1_size[1], real_T c5_y1_data[784]);
static __global__ void c5_eML_blk_kernel_kernel59(const int32_T c5_x1_size[1],
  boolean_T c5_selectedIndex_data[784]);
static __global__ void c5_eML_blk_kernel_kernel60(const real_T
  c5_bboxesX1Y1X2Y2_data[3136], const int32_T c5_bboxesX1Y1X2Y2_size[2], const
  int32_T c5_i61, real_T c5_area_data[784]);
static __global__ void c5_eML_blk_kernel_kernel61(const int32_T
  c5_bboxesX1Y1X2Y2_size[2], const real_T c5_bboxesX1Y1X2Y2_data[3136], const
  int32_T c5_i64, real_T c5_x2_data[784]);
static __global__ void c5_eML_blk_kernel_kernel62(const real_T
  c5_bboxesX1Y1X2Y2_data[3136], const int32_T c5_bboxesX1Y1X2Y2_size[2], const
  int32_T c5_i66, real_T c5_y2_data[784]);
static __global__ void c5_eML_blk_kernel_kernel63(const int32_T c5_i12, const
  int32_T c5_iv[2], boolean_T c5_selectedIndex_data[784]);
static __global__ void c5_eML_blk_kernel_kernel64(const boolean_T
  c5_selectedIndex_data[784], const real_T c5_x1_data[784], const int32_T
  c5_selectedIndex_size[1], boolean_T c5_index_data[784]);
static __global__ void c5_eML_blk_kernel_kernel65(const real_T c5_bboxPred_data
  [3136], const int32_T c5_bboxPred_size[2], const int16_T c5_iv1_data[784],
  const int32_T c5_bboxesX1Y1X2Y2_size[2], const int32_T c5_iv1_size[1], real_T
  c5_bboxesX1Y1X2Y2_data[3136]);
static __global__ void c5_eML_blk_kernel_kernel66(const real32_T
  c5_scorePred_data[784], const int32_T c5_scorePred_size[1], real32_T
  c5_b_scores_data[784]);
static __global__ void c5_eML_blk_kernel_kernel67(const real_T
  c5_bboxesX1Y1X2Y2_data[3136], const int32_T c5_bboxesX1Y1X2Y2_size[2], real_T
  c5_b_bboxes_data[]);
static __global__ void c5_eML_blk_kernel_kernel68(const real32_T
  c5_b_scores_data[784], const int32_T c5_scores_size[1], real32_T
  c5_c_scores_data[]);
static __global__ void c5_DeepLearningNetwork_activations_kernel69(const
  real32_T c5_varargin_1[150528], c5_cell_wrap_18 *c5_r);
static __global__ void c5_DeepLearningNetwork_activations_kernel70(const
  c5_cell_wrap_18 *c5_r, c5_cell_wrap_18 c5_miniBatchT[1]);
static __global__ void c5_DeepLearningNetwork_activations_kernel71(const
  real32_T c5_outMiniBatch[4704], real32_T c5_b_out[4704]);
static __device__ real32_T c5_callFcn_device(real32_T c5_input1, real32_T
  c5_input2);
static __device__ real32_T c5_b_callFcn_device(real32_T c5_input1, real32_T
  c5_input2);
static void c5_checkCleanupCudaError(hipError_t c5_errCode, const char_T
  *c5_file, uint32_T c5_line);
static emlrtRTEInfo c5_createEmlrtInfoStruct(const char_T *c5_file, uint32_T
  c5_line);
static void init_dsm_address_info(SFc5_LaneDetectionInstanceStruct
  *chartInstance);
static void init_simulink_io_address(SFc5_LaneDetectionInstanceStruct
  *chartInstance);

/* Function Definitions */
static void initialize_c5_LaneDetection(SFc5_LaneDetectionInstanceStruct
  *chartInstance)
{
  emlrtLicenseCheckR2012b(chartInstance->c5_fEmlrtCtx,
    "distrib_computing_toolbox", 2);
  emlrtLicenseCheckR2012b(chartInstance->c5_fEmlrtCtx, "neural_network_toolbox",
    2);
  emlrtLicenseCheckR2012b(chartInstance->c5_fEmlrtCtx,
    "video_and_image_blockset", 2);
  sim_mode_is_external(chartInstance->S);
  _sfTime_ = sf_get_time(chartInstance->S);
  chartInstance->c5_yolodetector_not_empty = false;
  chartInstance->c5_is_active_c5_LaneDetection = 0U;
  hipGetLastError();
  hipMalloc(&chartInstance->c5_d_gpu_loop_ub, 4UL);
  hipMalloc(&chartInstance->c5_gpu_i7, 4UL);
  hipMalloc(&chartInstance->c5_c_gpu_bboxPred_size, 4UL);
  hipMalloc(&chartInstance->c5_d_gpu_end, 4UL);
  hipMalloc(&chartInstance->c5_gpu_aux2, 2560UL);
  hipMalloc(&chartInstance->c5_gpu_classPred_size, 4UL);
  hipMalloc(&chartInstance->c5_b_gpu_xoffset, 4UL);
  hipMalloc(&chartInstance->c5_gpu_bboxPred_data, 6272UL);
  hipMalloc(&chartInstance->c5_gpu_end, 4UL);
  hipMalloc(&chartInstance->c5_d_gpu_bboxPred_data, 6272UL);
  hipMalloc(&chartInstance->c5_gpu_i47, 4UL);
  hipMalloc(&chartInstance->c5_gpu_rowWeights, 16128UL);
  hipMalloc(&chartInstance->c5_gpu_i64, 4UL);
  hipMalloc(&chartInstance->c5_gpu_bboxesX1Y1X2Y2_data, 25088UL);
  hipMalloc(&chartInstance->c5_gpu_idx_data, 3136UL);
  hipMalloc(&chartInstance->c5_gpu_i9, 4UL);
  hipMalloc(&chartInstance->c5_b_gpu_scores_data, 3136UL);
  hipMalloc(&chartInstance->c5_gpu_bv, 784UL);
  hipMalloc(&chartInstance->c5_gpu_selectedIndex_size, 4UL);
  hipMalloc(&chartInstance->c5_gpu_outVal, 4UL);
  hipMalloc(&chartInstance->c5_gpu_selectedIndex_data, 784UL);
  hipMalloc(&chartInstance->c5_c_gpu_end, 4UL);
  hipMalloc(&chartInstance->c5_gpu_boxOut, 18816UL);
  hipMalloc(&chartInstance->c5_h_gpu_loop_ub, 4UL);
  hipMalloc(&chartInstance->c5_gpu_i35, 4UL);
  hipMalloc(&chartInstance->c5_gpu_scores_data, 20U * sizeof(real32_T));
  hipMalloc(&chartInstance->c5_gpu_count, 4UL);
  hipMalloc(&chartInstance->c5_gpu_i61, 4UL);
  hipMalloc(&chartInstance->c5_gpu_b_size, 8UL);
  hipMalloc(&chartInstance->c5_gpu_i10, 4UL);
  hipMalloc(&chartInstance->c5_b_gpu_end, 4UL);
  hipMalloc(&chartInstance->c5_b_gpu_anchors, 32UL);
  hipMalloc(&chartInstance->c5_gpu_i, 4UL);
  hipMalloc(&chartInstance->c5_gpu_ii_size, 4UL);
  hipMalloc(&chartInstance->c5_gpu_colWeightsTotal, 1792UL);
  hipMalloc(&chartInstance->c5_c_gpu_n, 4UL);
  hipMalloc(&chartInstance->c5_gpu_loop_ub, 4UL);
  hipMalloc(&chartInstance->c5_gpu_i51, 4UL);
  hipMalloc(&chartInstance->c5_b_gpu_count, 8UL);
  hipMalloc(&chartInstance->c5_gpu_nx, 4UL);
  hipMalloc(&chartInstance->c5_gpu_i50, 4UL);
  hipMalloc(&chartInstance->c5_gpu_scores_size, 4UL);
  hipMalloc(&chartInstance->c5_gpu_iv, 8UL);
  hipMalloc(&chartInstance->c5_c_gpu_bboxPred_data, 25088UL);
  hipMalloc(&chartInstance->c5_gpu_index_data, 784UL);
  hipMalloc(&chartInstance->c5_gpu_b_data, 784UL);
  hipMalloc(&chartInstance->c5_gpu_In, 3686400UL);
  hipMalloc(&chartInstance->c5_b_gpu_outVal, 8UL);
  hipMalloc(&chartInstance->c5_gpu_dv, 8UL);
  hipMalloc(&chartInstance->c5_gpu_area_data, 6272UL);
  hipMalloc(&chartInstance->c5_gpu_nrows, 4UL);
  hipMalloc(&chartInstance->c5_gpu_bboxes_data, 80U * sizeof(real_T));
  hipMalloc(&chartInstance->c5_gpu_y1_data, 6272UL);
  hipMalloc(&chartInstance->c5_gpu_i17, 4UL);
  hipMalloc(&chartInstance->c5_gpu_iv1_data, 1568UL);
  hipMalloc(&chartInstance->c5_gpu_i39, 4UL);
  hipMalloc(&chartInstance->c5_gpu_x2_data, 6272UL);
  hipMalloc(&chartInstance->c5_gpu_i38, 4UL);
  hipMalloc(&chartInstance->c5_b_gpu_out, 602112UL);
  hipMalloc(&chartInstance->c5_gpu_classPred_data, 3136UL);
  hipMalloc(&chartInstance->c5_gpu_rowWeightsTotal, 1792UL);
  hipMalloc(&chartInstance->c5_gpu_i32, 4UL);
  hipMalloc(&chartInstance->c5_gpu_dv2, 8UL);
  hipMalloc(&chartInstance->c5_gpu_ipRowIndices, 4032UL);
  hipMalloc(&chartInstance->c5_gpu_i14, 4UL);
  hipMalloc(&chartInstance->c5_gpu_i66, 4UL);
  hipMalloc(&chartInstance->c5_gpu_x1_data, 6272UL);
  hipMalloc(&chartInstance->c5_gpu_colWeights, 21504UL);
  hipMalloc(&chartInstance->c5_b_gpu_bboxPred_data, 25088UL);
  hipMalloc(&chartInstance->c5_b_gpu_loop_ub, 4UL);
  hipMalloc(&chartInstance->c5_e_gpu_bboxPred_size, 8UL);
  hipMalloc(&chartInstance->c5_gpu_anchors, 64UL);
  hipMalloc(&chartInstance->c5_gpu_y, 4UL);
  hipMalloc(&chartInstance->c5_gpu_scorePred_data, 3136UL);
  hipMalloc(&chartInstance->c5_gpu_bboxesX1Y1X2Y2_size, 8UL);
  hipMalloc(&chartInstance->c5_gpu_ii_data, 1568UL);
  hipMalloc(&chartInstance->c5_gpu_thresholdedPrediction_data, 18816UL);
  hipMalloc(&chartInstance->c5_gpu_bboxPred_size, 8UL);
  hipMalloc(&chartInstance->c5_gpu_i19, 4UL);
  hipMalloc(&chartInstance->c5_d_gpu_bboxPred_size, 8UL);
  hipMalloc(&chartInstance->c5_g_gpu_loop_ub, 4UL);
  hipMalloc(&chartInstance->c5_gpu_partialResize, 1290240UL);
  hipMalloc(&chartInstance->c5_b_gpu_n, 4UL);
  hipMalloc(&chartInstance->c5_gpu_xoffset, 4UL);
  hipMalloc(&chartInstance->c5_gpu_i45, 4UL);
  hipMalloc(&chartInstance->c5_gpu_thresholdedPrediction_size, 8UL);
  hipMalloc(&chartInstance->c5_e_gpu_bboxPred_data, 25088UL);
  hipMalloc(&chartInstance->c5_gpu_aux1, 1920UL);
  hipMalloc(&chartInstance->c5_gpu_x1_size, 4UL);
  hipMalloc(&chartInstance->c5_gpu_dv1, 32UL);
  hipMalloc(&chartInstance->c5_gpu_out_data, 3136UL);
  hipMalloc(&chartInstance->c5_gpu_scorePred_size, 4UL);
  hipMalloc(&chartInstance->c5_gpu_i6, 4UL);
  hipMalloc(&chartInstance->c5_gpu_i3, 4UL);
  hipMalloc(&chartInstance->c5_gpu_n, 4UL);
  hipMalloc(&chartInstance->c5_f_gpu_loop_ub, 4UL);
  hipMalloc(&chartInstance->c5_gpu_ipColIndices, 5376UL);
  hipMalloc(&chartInstance->c5_c_gpu_loop_ub, 4UL);
  hipMalloc(&chartInstance->c5_b_gpu_bboxPred_size, 4UL);
  hipMalloc(&chartInstance->c5_gpu_y2_data, 6272UL);
  hipMalloc(&chartInstance->c5_gpu_i12, 4UL);
  hipMalloc(&chartInstance->c5_gpu_out, 602112UL);
  hipMalloc(&chartInstance->c5_gpu_iv1_size, 4UL);
  hipMalloc(&chartInstance->c5_e_gpu_loop_ub, 4UL);
  hipMalloc(&chartInstance->c5_gpu_i4, 4UL);
  hipMalloc(&chartInstance->c5_gpu_tmpFeatureMap, 18816UL);
}

static void initialize_params_c5_LaneDetection(SFc5_LaneDetectionInstanceStruct *
  chartInstance)
{
}

static void enable_c5_LaneDetection(SFc5_LaneDetectionInstanceStruct
  *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
}

static void disable_c5_LaneDetection(SFc5_LaneDetectionInstanceStruct
  *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
}

static void c5_do_animation_call_c5_LaneDetection
  (SFc5_LaneDetectionInstanceStruct *chartInstance)
{
  sfDoAnimationWrapper(chartInstance->S, false, true);
  sfDoAnimationWrapper(chartInstance->S, false, false);
}

static void ext_mode_exec_c5_LaneDetection(SFc5_LaneDetectionInstanceStruct
  *chartInstance)
{
}

static const mxArray *get_sim_state_c5_LaneDetection
  (SFc5_LaneDetectionInstanceStruct *chartInstance)
{
  const mxArray *c5_b_y = NULL;
  const mxArray *c5_c_y = NULL;
  const mxArray *c5_d_y = NULL;
  const mxArray *c5_st;
  const mxArray *c5_y = NULL;
  c5_st = NULL;
  c5_y = NULL;
  sf_mex_assign(&c5_y, sf_mex_createcellmatrix(3, 1), false);
  c5_b_y = NULL;
  sf_mex_assign(&c5_b_y, sf_mex_create("y", chartInstance->c5_bboxes_data, 0, 0U,
    1U, 0U, 2, (*chartInstance->c5_bboxes_sizes)[0],
    (*chartInstance->c5_bboxes_sizes)[1]), false);
  sf_mex_setcell(c5_y, 0, c5_b_y);
  c5_c_y = NULL;
  sf_mex_assign(&c5_c_y, sf_mex_create("y", chartInstance->c5_scores_data, 1, 0U,
    1U, 0U, 2, (*chartInstance->c5_scores_sizes)[0], 1), false);
  sf_mex_setcell(c5_y, 1, c5_c_y);
  c5_d_y = NULL;
  sf_mex_assign(&c5_d_y, sf_mex_create("y",
    &chartInstance->c5_is_active_c5_LaneDetection, 3, 0U, 0U, 0U, 0), false);
  sf_mex_setcell(c5_y, 2, c5_d_y);
  sf_mex_assign(&c5_st, c5_y, false);
  return c5_st;
}

static void set_sim_state_c5_LaneDetection(SFc5_LaneDetectionInstanceStruct
  *chartInstance, const mxArray *c5_st)
{
  const mxArray *c5_u;
  c5_u = sf_mex_dup(c5_st);
  c5_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getcell(c5_u, 0)),
                      "bboxes", *chartInstance->c5_bboxes_data,
                      *chartInstance->c5_bboxes_sizes);
  c5_c_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getcell(c5_u, 1)),
                        "scores", *chartInstance->c5_scores_data,
                        *chartInstance->c5_scores_sizes);
  chartInstance->c5_is_active_c5_LaneDetection = c5_e_emlrt_marshallIn
    (chartInstance, sf_mex_dup(sf_mex_getcell(c5_u, 2)),
     "is_active_c5_LaneDetection");
  sf_mex_destroy(&c5_u);
  sf_mex_destroy(&c5_st);
}

static void sf_gateway_c5_LaneDetection(SFc5_LaneDetectionInstanceStruct
  *chartInstance)
{
  c5_emxArray_real32_T_20x1 c5_scores;
  c5_emxArray_real_T_20x4 c5_bboxes;
  int32_T c5_i;
  chartInstance->c5_JITTransitionAnimation[0] = 0U;
  _sfTime_ = sf_get_time(chartInstance->S);
  for (c5_i = 0; c5_i < 921600; c5_i++) {
    chartInstance->c5_fv[c5_i] = (*chartInstance->c5_In)[c5_i];
  }

  c5_eML_blk_kernel(chartInstance, chartInstance->c5_fv, c5_bboxes.data,
                    c5_bboxes.size, c5_scores.data, c5_scores.size);
  c5_emxConvertDynamicMatrixFromEmx_(chartInstance, &c5_bboxes,
    *chartInstance->c5_bboxes_data, *chartInstance->c5_bboxes_sizes);
  c5_b_emxConvertDynamicMatrixFromEmx_(chartInstance, &c5_scores,
    *chartInstance->c5_scores_data, *chartInstance->c5_scores_sizes);
  c5_do_animation_call_c5_LaneDetection(chartInstance);
}

static void mdl_start_c5_LaneDetection(SFc5_LaneDetectionInstanceStruct
  *chartInstance)
{
}

static void mdl_terminate_c5_LaneDetection(SFc5_LaneDetectionInstanceStruct
  *chartInstance)
{
  hipError_t c5_errCode;
  hipFree(*chartInstance->c5_gpu_out_data);
  hipFree(chartInstance->c5_gpu_i9);
  hipFree(*chartInstance->c5_gpu_x1_data);
  hipFree(*chartInstance->c5_gpu_y2_data);
  hipFree(*chartInstance->c5_b_gpu_scores_data);
  hipFree(chartInstance->c5_gpu_i17);
  hipFree(chartInstance->c5_d_gpu_end);
  hipFree(*chartInstance->c5_b_gpu_out);
  hipFree(chartInstance->c5_gpu_i32);
  hipFree(chartInstance->c5_gpu_i39);
  hipFree(chartInstance->c5_gpu_loop_ub);
  hipFree(*chartInstance->c5_gpu_anchors);
  hipFree(*chartInstance->c5_gpu_ipRowIndices);
  hipFree(chartInstance->c5_gpu_i38);
  hipFree(*chartInstance->c5_gpu_bboxesX1Y1X2Y2_data);
  hipFree(*chartInstance->c5_gpu_thresholdedPrediction_data);
  hipFree(chartInstance->c5_gpu_scores_data);
  hipFree(chartInstance->c5_b_gpu_n);
  hipFree(*chartInstance->c5_gpu_iv);
  hipFree(chartInstance->c5_gpu_i10);
  hipFree(chartInstance->c5_gpu_i12);
  hipFree(chartInstance->c5_c_gpu_end);
  hipFree(*chartInstance->c5_gpu_aux1);
  hipFree(*chartInstance->c5_b_gpu_bboxPred_data);
  hipFree(*chartInstance->c5_gpu_thresholdedPrediction_size);
  hipFree(chartInstance->c5_gpu_n);
  hipFree(*chartInstance->c5_gpu_bv);
  hipFree(*chartInstance->c5_gpu_iv1_size);
  hipFree(*chartInstance->c5_gpu_ii_data);
  hipFree(chartInstance->c5_b_gpu_xoffset);
  hipFree(chartInstance->c5_g_gpu_loop_ub);
  hipFree(chartInstance->c5_gpu_i35);
  hipFree(chartInstance->c5_gpu_bboxes_data);
  hipFree(*chartInstance->c5_gpu_aux2);
  hipFree(chartInstance->c5_gpu_xoffset);
  hipFree(chartInstance->c5_gpu_i50);
  hipFree(chartInstance->c5_gpu_outVal);
  hipFree(*chartInstance->c5_gpu_out);
  hipFree(*chartInstance->c5_gpu_colWeightsTotal);
  hipFree(*chartInstance->c5_gpu_colWeights);
  hipFree(*chartInstance->c5_gpu_index_data);
  hipFree(*chartInstance->c5_gpu_ii_size);
  hipFree(*chartInstance->c5_gpu_scorePred_data);
  hipFree(*chartInstance->c5_gpu_dv);
  hipFree(chartInstance->c5_gpu_nrows);
  hipFree(chartInstance->c5_gpu_nx);
  hipFree(*chartInstance->c5_gpu_boxOut);
  hipFree(chartInstance->c5_h_gpu_loop_ub);
  hipFree(*chartInstance->c5_gpu_rowWeights);
  hipFree(*chartInstance->c5_gpu_dv1);
  hipFree(chartInstance->c5_gpu_i14);
  hipFree(*chartInstance->c5_gpu_bboxesX1Y1X2Y2_size);
  hipFree(*chartInstance->c5_c_gpu_bboxPred_data);
  hipFree(chartInstance->c5_gpu_count);
  hipFree(*chartInstance->c5_gpu_scores_size);
  hipFree(*chartInstance->c5_gpu_selectedIndex_size);
  hipFree(*chartInstance->c5_b_gpu_bboxPred_size);
  hipFree(*chartInstance->c5_gpu_dv2);
  hipFree(chartInstance->c5_gpu_i51);
  hipFree(*chartInstance->c5_e_gpu_bboxPred_size);
  hipFree(chartInstance->c5_gpu_i4);
  hipFree(chartInstance->c5_b_gpu_count);
  hipFree(*chartInstance->c5_gpu_area_data);
  hipFree(*chartInstance->c5_gpu_ipColIndices);
  hipFree(*chartInstance->c5_gpu_iv1_data);
  hipFree(chartInstance->c5_gpu_i19);
  hipFree(*chartInstance->c5_gpu_scorePred_size);
  hipFree(*chartInstance->c5_gpu_classPred_size);
  hipFree(*chartInstance->c5_gpu_rowWeightsTotal);
  hipFree(*chartInstance->c5_gpu_tmpFeatureMap);
  hipFree(*chartInstance->c5_gpu_bboxPred_data);
  hipFree(*chartInstance->c5_d_gpu_bboxPred_data);
  hipFree(chartInstance->c5_d_gpu_loop_ub);
  hipFree(chartInstance->c5_gpu_i6);
  hipFree(*chartInstance->c5_gpu_bboxPred_size);
  hipFree(chartInstance->c5_gpu_i64);
  hipFree(*chartInstance->c5_gpu_y1_data);
  hipFree(chartInstance->c5_gpu_i66);
  hipFree(chartInstance->c5_gpu_end);
  hipFree(*chartInstance->c5_d_gpu_bboxPred_size);
  hipFree(chartInstance->c5_gpu_i47);
  hipFree(chartInstance->c5_gpu_i3);
  hipFree(*chartInstance->c5_gpu_partialResize);
  hipFree(*chartInstance->c5_gpu_x1_size);
  hipFree(chartInstance->c5_c_gpu_n);
  hipFree(*chartInstance->c5_gpu_idx_data);
  hipFree(chartInstance->c5_gpu_i);
  hipFree(*chartInstance->c5_e_gpu_bboxPred_data);
  hipFree(chartInstance->c5_b_gpu_loop_ub);
  hipFree(chartInstance->c5_b_gpu_end);
  hipFree(*chartInstance->c5_c_gpu_bboxPred_size);
  hipFree(*chartInstance->c5_gpu_b_data);
  hipFree(chartInstance->c5_e_gpu_loop_ub);
  hipFree(*chartInstance->c5_gpu_x2_data);
  hipFree(*chartInstance->c5_gpu_classPred_data);
  hipFree(chartInstance->c5_gpu_i7);
  hipFree(*chartInstance->c5_b_gpu_anchors);
  hipFree(chartInstance->c5_f_gpu_loop_ub);
  hipFree(*chartInstance->c5_b_gpu_outVal);
  hipFree(chartInstance->c5_gpu_i61);
  hipFree(*chartInstance->c5_gpu_b_size);
  hipFree(chartInstance->c5_gpu_i45);
  hipFree(*chartInstance->c5_gpu_In);
  hipFree(chartInstance->c5_c_gpu_loop_ub);
  hipFree(chartInstance->c5_gpu_y);
  hipFree(*chartInstance->c5_gpu_selectedIndex_data);
  c5_errCode = hipGetLastError();
  if (c5_errCode != hipSuccess) {
    emlrtThinCUDAError(c5_errCode, hipGetErrorName(c5_errCode),
                       hipGetErrorString(c5_errCode), "SimGPUErrorChecks",
                       chartInstance->c5_fEmlrtCtx);
  }
}

static void mdl_setup_runtime_resources_c5_LaneDetection
  (SFc5_LaneDetectionInstanceStruct *chartInstance)
{
  setLegacyDebuggerFlag(chartInstance->S, false);
  setDebuggerFlag(chartInstance->S, false);
  sim_mode_is_external(chartInstance->S);
}

static void mdl_cleanup_runtime_resources_c5_LaneDetection
  (SFc5_LaneDetectionInstanceStruct *chartInstance)
{
}

static void initSimStructsc5_LaneDetection(SFc5_LaneDetectionInstanceStruct
  *chartInstance)
{
}

const mxArray *sf_c5_LaneDetection_get_eml_resolved_functions_info()
{
  const mxArray *c5_nameCaptureInfo = NULL;
  const char_T *c5_data[45] = {
    "789ced9dc973ebc87dc739ae7132763c338a27138f5359de547289277922a985922f3109921229eea4c4e555ea09049b24441000b170bb84b9b92a871c928353"
    "d9173b7179c9a4ca4965ad5cf327e43fc835e5532eae0ab8b444e2110f1c35d90f6afe50358307fda0fefe7a517fd0bbefad44fa2d9fcff79ef5df1ffd8ecff7",
    "5f5ffb9c6f7abdeb9b5f078bfbe77cab97ddfed6e25eb53de3ebf3beb7577e6f6aff434beff717cf82221b6868cc1f64be8bee7fb3a1744599978dd248453e0d"
    "e98ad4478d99a5294aa824765171f921337deac6974cf70f53d3f4df5c1b099da2d9f5696dfdc14369f961961ed3ebd621be6fbba487fdb2a787fd3dac3779a4",
    "1e0effab2e7ad8de476d5190d0cb06329060888afc72a448cacb7ed0e6cf2da13f3fe5e8cfdca28b724b420f7adf22d43b72d45bb5bf88fdf6615be9a2c37657"
    "900ec3d169f9d314e9d0315d9e7767bfe7962eef6fe8a7fdfef0fe3bb3fb8ffef3ed9734f5f0b52f7a4387f0362d673fefa07760b39ba55c878ba3782e71153b",
    "e713b562af9c6fc51efcc8b9e8b8f9e17378a615fe7f3bfcfea6e9d87308ffc0667f91a8587fb1a6ae1d4a8ac04b87e97029158e1c1682fea0bf7e68288a5457"
    "86877a9bd750e3b021094a03692febbc8e561f3e99fdfbf013d1a28d265be1480adf8822a4a610afc9565d9441c640d13acf555b3c49cbcb072ef1c4f6997fcf",
    "abd954b61f5cf8b2e2c72da11f6fdb9e1ffc985b042b0597f5fe9750eff71cf556ed2f12dc72851c5504b38b6443c7195d345555d18c1c2f74f816d237c9787d"
    "fe2b2afe95f93b2f1bd24b83d75ac80a7b511a56527b91f3b4eaa55fffc9c750cfef508ff4eff64307bd039bfda6d4baeb9d1f5f5db5e315a1de3ff5378ee249",
    "1f3bf5fcbed703df248cffc72ef1c7f62e6f487c9db3fc692139637d949a325f97504e5354a41922c26d14daed853997d690d2e6cf3b6f91f9f31d177fb0fd4d"
    "96936fac498643878f085af5ded73ffd317064877ab4da0b9d41ab5732b96c2d1416f3859b40a5c9e7ef2e8023ac7004b72b1e1bff776dcff6f863bbe5cfc281",
    "6cfdce07bc005ee0f7811734f48017db091f784116ff2fb9c41fdb2d7f4a33af52627d597f42a80fbc005e3c460f5ffba207bcd84ef8fbce0bd272f4b32ef1c7"
    "f67b5e4479839f0dcdfb56fc9810fa01dc006e3c460f5ffba207dcd84ef8fbce0dd276c6175de28fed963f89ec743463319001bc005eccdf075ed0d0035e6c27",
    "fc7de7056939fa8a4bfcb15d98ced58d2b5a11f54c240b08d7d6c00de0c6fc7de0060d3de0c676c2df776e90b6339cd2fbc0767f686758f0286432c00be0c5fc"
    "7de0050d3de0c576c2075e90c5ff5ddbb33dfed83eb06aa284ac9a46586be93ee897025ee0f7811734f48017db097fdf79714b18ff776ccf0ff19f5ba633a4f8",
    "11d2800fc087f9fbc0071a7ac087ed84bfef7c206d4f7c8671ee9ce56cc6ecfa56f42784fac00be0c563f4f0b52f7ac08bed840fbc208bffcfb8c41fdb071aaf"
    "a6f9d96cda15fd09a13ef00278f1183d7ced8b1ef0623be1ef3b2f2684f1ff0597f8637b9f97c4066fe0f9507145e3245ed7c5e6c8b7557f801fc08fc7e8e16b",
    "5ff4801fdb097fdff941daded874bcdbf267068c0cdf4530de0dbc78781f7841430f78b19df0f79d1713c2f86fdade58d99730a50c9056528a86660a8b2ddb81"
    "1fc08ff9fbc00f1a7ac08fed840ffc208bffa6f5b5aa21d5726f36c5b6288ed174590687fabcb4b0033f801ff3f7811f34f4801fdb097fdff941da5ff59eedd9",
    "1e7f6cd7eff7b3cdcc0e87025e002fe6ef032f68e8012fb613febef3e29630fe9f778cffdc6271c2547dc007e0037e1ff840430ff8b09df0f79d0fa4ed892fb8"
    "c41fdb055e928a0b562ceb4f08f58117c08bc7e8e16b5ff48017db091f784116ff4dd7672c78311dba58d19f10ea032f80178fd1c3d7bee8012fb6133ef0822c",
    "fe9b8e574c795140ba450c833760bc027871ff3ef082861ef0623be1032fc8e2ffbeedd91e7f6c9ff2e25a9daee89b03037801bc98bf0fbca0a107bcd84ef8c0"
    "0bb2f86f7a5eaba1f1b2ae2a3abae1b565fd09a13ef00278f1183d7ced8b1ef0623be1032fc8e2ef94de07b6bb8ab4ae69ccd75f4c3799025e002fe6ef032f68",
    "e8012fb613febef382d6f9de0b5e644d63018cc5cf811bc08df9fbc00d1a7ac08ded840fdc208bff072ef1c776a79a1ab801dc98bf0fdca0a107dcd84ef8fbce"
    "0dd2fea94dcfcbd096e64a2deb4f08f5b7cd0b1f212ffeccc51f6c7f91c87a0c180f394493139ffce463e0c40ef56871e2725cad760ae55afda6148cf377c29d",
    "5e2fc422c0095638415a8e3e74893fb6ab966ba26084e5c6f26c29eff2e2809017df75f107dbbdc78bb539f51c7f4fd0aaefbe06fcd8a91e2d7e1c9de7838db1"
    "de2e1cc56b3139356a9ae7a5cc25f083157edc12c6dfeddc5661713a86573941daaef813177fb0dd7b9cc039332f10b4eab1df002eec548f161762caf0341fbf",
    "29f851e8e22e533fbd0c8c6fa2d0ae60860ba4fd4f9baeefe60543ecf386a8c8fa8afe8450df6b9cf873177fb0dd7b9c58ca21ab8c40ff131b7ab438d1bf8ed5"
    "8ba181c225536643484823a3daedc03805339cb8258cff4f3bc67f6e59f46630cb853f76f107dbbdc78545ce6c581e800b4f438f16177ac50b29378c718d4aa1",
    "a5e5913fad0c12fd387081152e4c08e3bfe9f948d3f5dbb9794d343dd9c2d40da59be247489b37273ccb0dd2f1891fbaf883eddee386738ed11cdffefebffd2e"
    "3fbdef4bbd4e5b8f1647b4cb74de8834ce8f934a6ad40b9fb4fa89660ada17cc7084d6ba8ba55e0eab562a64328b9f7b951fa4f367bfede20fb67b8f1fafe4d4",
    "acc4c0f8051b7ab4b8c12505d49244291f112ad14e2e7add4cb506307e01dc58dc1fc70d0eb8b1b87b9d1b1c656e40bfd56ef5687123733cd2f345a3726795ea"
    "52b77f3948079563e00633dc5009e3ffaeedd91e7f6c57ef7b40e66d0d5679f1972efe60bbf778b19243d4e7cf022f76ab476bde7cbb90354b576222276bc745",
    "4d0e1f19c6d9f2bc79e0c5faf09f0a2f6895a369af79784d1f9557b9b1bfeb2fd6e614f083313d5aed8d51fa5810e59b8aa14891a37eb6df8a0e2a9128f003f8"
    "31bf36eda75a1d7585f10deff653bd9253d04fc5901eb57e2abead16c6e245521c72c3715c572f8cb4c2013758e1c63709e3ff6b2ef1c77655d18d9ca60848d7",
    "e77b12969402324c4de67803b5144db4aaab65bf26847e6d9b231f1172e49f5dfcc176ef7164939c7baed2aa0761dfa9ddead1e24af6f22693eef2d791f35434"
    "735757af82fd4b25065c61852ba4e31fefd99eedf1c7f6d55e12eff283b41df2572efe60bbf7f8b19a43b8030be659b1a147abdffab42c66c352f6e8bad354c6",
    "5553ad56735ac207bc005eccafcf720ef8a25764457f42a8efb5f5804f779df8520ec13a7166f468b52bae73858a7c16cf9606d9da5d463f0bc40d03d6710027"
    "707cbfe0127f6cd791119685b6a22d9e59e5c49fbaf883eddee3c47d0ec1ba0d86f4a8addb101267a38674dd4aa27aa77769ea6669948675e3c089c57dd373bf",
    "ad5a68e140511c23ddb39c20ed7ffa6b177fb0dd939c58ce21cafb8c002f76ab478b1721a9a5fb8ff4b891499785ea6994d7e5a80af3a798e1c58430fecf5ce2"
    "8fed2aafe9f3535c75ce72aa85e4a5399e854cc6abfc201dfffe918b3fd8ee3d7eb8e518ec7bcb921e2d9e94f263a39ee6c37d2e76d28d7486423e9d45d04f05",
    "3c59dc9fb9c41fdb5f5b3b71c09327c6138e364f60dc63b77ab478e26f9adac9406db72e338948229fec5e9d6b12ccd3059e2ceebfe8127f6c7fb5765a8cc52e"
    "b624f12a4f48d7097eeae20fb63f059e3ce418bd79b9c091ddead1e2c869ba943fabc653c391903f6974f93b7e702dc1be24c091c59d982305e0c8f47a421c29",
    "d0e408f46fed568f1647aec2d245711039354efccd44554b4b6533da80f608331c212d475f76893fb6ebc898d549d3b1dbe505e75ee507e938fb775cfcc176ef"
    "f1e3d59ca2b90f3b9c13bb5b3d5adc08f24629143e1e9f66b890d2f1e785512fc343fb83196e505b6f3e5fb1bc5c23addbf6cab31cd9e3f5e61be41cb44758d1",
    "a3c5153369648737839479913e2ea042dbef4f4a7d98bfc50c572684f17fe6127f6c5f533bad6cb2c42a4f9ef078bb4b8ec1fc2d96f468ad478fd49222675c73"
    "a73c1fe333b5b13f9440a60f78c20a4f54c2f87fc925fed83eeb755faab2bdca0f1f213ffec2c51f6cf7203f967208c6d359d1a3d5ee509bfdf2305c3aefc67a",
    "c7b2a09fc702b15e18d61932c3095adf1b963fcb9513a7c84db1e5f32e2ff677dff6b53905fbb633a6476d3e967a1ecda5afda665734438974f6eed8c8342f81"
    "1facf0634218ff6dd7dbb784febc6d7b7ef0676e11ac445ad6a3375eee314cacf919cdf172d84777b77ab4f8d019b47a2593cbd64261315fb809549a7cfe8ea1",
    "7584b70ebfef968ef6cb291df74def5b8fd4c3e19fb9e861fb8bebe5ea361c1514d9d014e970a4484a3f5840ba55e79df86f505b1424141bf25d5542cfbbbc41"
    "ebeff57fbef8617d7aa7a5f78dbf7916a4a9872fd6ebbfe1d9b8df6bd423e96645abd7c4ec695fd0820ced23fe8cf0fbe8df1dc23fb0d95f24624fe0fbe863b1",
    "805aa26e202d8a542437902c88485f4eaf89437a3cb5ef69fbb569fd0edfd3f03dcda21e7c4f6f277ce0c967e30969b9fbc825bdb0bd212d22f15c301bb23cff"
    "7f5815df1447feef917a38fc3f70d1c3760a1c69a9e6dcba5c52eed3db2a34d3a43ec4094e797fc0df827efa9dead1e2060a0522914c5f38efa7f277811ca736",
    "dba10643f34b7d84dc78bae711f00f93cf97ce23a0c585351e5d1ba2a46fabbdf3cb2e7e60bb637b67e6cd9b583fedad62324b86751f193303bd7943bf093cd9"
    "a91eb5715f2159ac08e5b01f5ddcf8c77a30d73e2ef10ccd1b029eacf2447588efa6e9f1aeedd99e1ed86eb9772d8b3d1365f82ef26dafdf6cdb1cd9db75d473",
    "90ac6413cd7e2c589fb05b3d5afc08f4c6d1d1f0ba5a0a36514048449af5a3f33243eba8811fabfca0b7ae1aa996b7374830142d6b1ac5d9c8edea1adde97b5e"
    "e50a69ffe77fb8f883ed9e2b3773ae6c927d54cf5382f6ca6ef568cd4faf69422d64d650b39e0f1b52d5d432a6b4fc7d09bc591ffebef2e6575dd203db171556",
    "a26b79fe1adc30cb9bcdc7db3c566e5678f3baeca3bbfe1a78b35b3d5abc0966e460a72ef55ae75ca95dca8692d2455ef1016f58e50d69b9fa3997f4c0f64585"
    "55c8643824495695a59ab383c2bdca17d2f1961fb8f883ed9e2b272b7c59c92e380f96213d5afd654583ef0e846be13a864ecf637a9c4b17840a8cdf33cb9389",
    "437c374d8f5f72490f6cc715943248f377b3fd86e67d2fb3ed887cdee50ae97e51ffe4e20fb67baebcac72656db6d11c8f81f6ca6ef568f1a5e21f5d368367dc"
    "117f9a48e6839dab5eaa5086f17ce08b437a6c7ccec62b15d5acaf05e38559befca38b3fd8eeb9f2e2c497876c83f11786f468f1e5bc9d93f9abb8297181a3dc",
    "a8de4a9f24cb3243eb20812fdbe5cb3397f4c0f64545c52992d995d77c0b7b952fa4e32effe2e20fb67baebcacf0c531dba8ef47059cd9ad1e2dce08d96665ac"
    "1c0bb1a3b34eaf9b0f06caf52304ed18e08c437afc8a4b7a60fbda0a6ba929e355ceeced391dafe1cc725306f8c2861ead71fd4ba32624a2e7f291a146c7c178",
    "bfdc4f2597fbc9812febc37faa7ca1b60ff3bca24a8bb258e70da18d4f80f22a5748c7f5ffdec51f6cf75c3959e18a2dbb96cacdad43fc81274f438fd6dfbda1"
    "26e2c190209e57b57ed18c041b63652430b47f2ef064bbf3923f76490f6c5f5450f319474e870bfabccb17d2feb17f75f107db3d576e56f8e29c7db8830c78c3",
    "861e2ddea4aeeeccc0492e2955a4bb7ce926702ef78d1af08659de4c1ce2bba37964f715d6ca11763eef7206e69139671bcc2363478f56ff58777452e2aad953"
    "7fa63ece348db8303851e23ee00bab7c212d571fb8a407b65bee71a66e28dd143f425635a5a8de5e5749da3ff67d177fb0dd73e5e47e7f187b76c13a4a96f468",
    "f124c99f76ae1017085dc68d642577aca58eaf601d25bb3c511de2bbed7dc68436123a89665c911aa8e1f32e47f67c9fb1956c82f6083b7ab4fabbaa72a8d6aa"
    "7745a326ea83546778c21b013fac9b64961f14cfbb9d779a14c5f174c390d977aecfbb1c216d8ffcd0c51f6cf75c39b96f8fbc9a5dc01376f468f1241b1a5d84",
    "11874e1aa5d8553a789ccf1c9931183f019e38a4c7975dd203db2df712d9593fc9124c7cecf2e47b2efe60bbe7cac93d4f6cd945f95c16e0c96ef568f1a49d18"
    "66c3f29dbf171b6793c746489706a32eac57019e38a4c7575cd203db2df78a96efa891c8ceaaa784dc1005a433cb934f5dfcc176cf95937b9eaccb2e589fc28a",
    "1e2d9ee85701b1a55f268781e35c3a77640cf3a3761ff6d5679627b4c64b2cf770c584a665d9ab1cd9f3f192956c82fe2d76f468f56bf74ab94ab0c0252fdb89"
    "58ab9b1107b29aaffa801fc08ff5e9e194fe07b6bbe5de74b15c64ba582e213715aff283b41df25d177fb0dd73e5e39e1f2bd904fbb030a647edbce164e7b466",
    "9ae1403f5817f5713adf38896bd0afc52c47260ef1ddf63a93694709ea994816500ac92da3bdd85a1df62b9e5f9e2b2f0ffd5b8ed906ed1476f468f1e5eae6ae"
    "5c8a7623dd71161931ed5ae3ce9b230ef8c22a5f48db29efdb9eede981edf71385162b17bdca13d27eaebf73f107db3d573e6cf3b9f04a45e0071b7ab4fab9c6",
    "bc2195ca5af8fcc2ecf574433c2b0eb8880ff8c12a3fa8cf0b8e23de3035349d6f6a7d96789623302ff8d5ec82f6083b7ab4da235c55f76b3791eb536e58afb6"
    "afa5c245404dc3bc60667942eb3cfb7515d49aed553ccb976784e5e6899f67bf49f6c1792a0ce9d1e24d211af7d7fa67f971f74eb918a14e2a135663709e0ab3",
    "bca1386f7865a1dca2a6f22c5f60def0baec82fe3056f468f55b148ab5e351a434ac72ddb034ca075aa1467b795f0ce0c9faf0f795279b9e573fdfc0437ea8a1"
    "3cde5ed9f3f3ead765179c57cf901e2d9ed4ce2ffbd59b7e2b7e2d4ae57eed4ad6d289820f78c22a4f5487f86e9a1eefd99eede981edab15947739423a3effb7",
    "2efe60bbe7cac73a8ec0b928cce8d1eadf1a5c05eeae6bd152881fa79be56c3913383178d8b78b597ed0fa2ee9f392d8e00d8467a37a7ddef09e8fcfafcd2e18"
    "9f67478f164f8a1dfe663c54a2c37279d83acaa463d163b3043c619627a4ed11a7f43fb0dd7105355b751dd65aac72e489af6bb46713f5758dd0afb55b3d5a1c",
    "b9098c5b6d34ec1e952fa201d98c5c85f44ae50238c22a4748cbd5472ee981edb30e93b0dc285b4ec515ed92979ab36f5daff284743de33fb8f883ed9e2b274b"
    "fd5b6bb20bd6cb33a647ab5fa23e5646b9b62e551b9715f5aa963fca14f230fece2e57260ef17d53f5f9ed23fdb15f4efee08bfebc2d6f1587393e9c0c306f8b",
    "153d5aed91532159ac08e5b01f5ddcf8c77a30d73e2ef1b0cf0a70c3213dbeea921ed8eec88d85fd4d7183defe8e1e2b0e6b7e46731ce4eb9ffe1878b1433d5a"
    "bce80c5abd92c9656ba1b0982fdc042a4d3e7fc750ff1569fdf06d87f00f6c764ff322ae689c6ddee6c421dec08dd7eb01375e9f2ec08d37ab07dcd84ef8c08d",
    "dd7003faa95eaf07fd54ebd307faa9e8e8413f1559f8ff0fa87abd35", "" };

  c5_nameCaptureInfo = NULL;
  emlrtNameCaptureMxArrayR2016a(&c5_data[0], 142048U, &c5_nameCaptureInfo);
  return c5_nameCaptureInfo;
}

static void c5_eML_blk_kernel(SFc5_LaneDetectionInstanceStruct *chartInstance,
  real32_T c5_b_In[921600], real_T c5_b_bboxes_data[], int32_T c5_bboxes_size[2],
  real32_T c5_b_scores_data[], int32_T c5_scores_size[2])
{
  static real_T c5_dv1[4] = { 3.6875, 1.375, 1.8125, 6.8125 };

  static int8_T c5_dv[8] = { 59, 22, 29, 109, 43, 18, 23, 84 };

  c5_yolov2ResNet50VehicleExample0_LaneDetection0 *c5_iobj_0;
  c5_yolov2ResNet50VehicleExample0_LaneDetection0 *c5_this_Network;
  dim3 c5_ab_block;
  dim3 c5_ab_grid;
  dim3 c5_b_block;
  dim3 c5_b_grid;
  dim3 c5_bb_block;
  dim3 c5_bb_grid;
  dim3 c5_block;
  dim3 c5_c_block;
  dim3 c5_c_grid;
  dim3 c5_cb_block;
  dim3 c5_cb_grid;
  dim3 c5_d_block;
  dim3 c5_d_grid;
  dim3 c5_db_block;
  dim3 c5_db_grid;
  dim3 c5_e_block;
  dim3 c5_e_grid;
  dim3 c5_eb_block;
  dim3 c5_eb_grid;
  dim3 c5_f_block;
  dim3 c5_f_grid;
  dim3 c5_fb_block;
  dim3 c5_fb_grid;
  dim3 c5_g_block;
  dim3 c5_g_grid;
  dim3 c5_gb_block;
  dim3 c5_gb_grid;
  dim3 c5_grid;
  dim3 c5_h_block;
  dim3 c5_h_grid;
  dim3 c5_hb_block;
  dim3 c5_hb_grid;
  dim3 c5_i_block;
  dim3 c5_i_grid;
  dim3 c5_ib_block;
  dim3 c5_ib_grid;
  dim3 c5_j_block;
  dim3 c5_j_grid;
  dim3 c5_jb_block;
  dim3 c5_jb_grid;
  dim3 c5_k_block;
  dim3 c5_k_grid;
  dim3 c5_kb_block;
  dim3 c5_kb_grid;
  dim3 c5_l_block;
  dim3 c5_l_grid;
  dim3 c5_lb_block;
  dim3 c5_lb_grid;
  dim3 c5_m_block;
  dim3 c5_m_grid;
  dim3 c5_mb_block;
  dim3 c5_mb_grid;
  dim3 c5_n_block;
  dim3 c5_n_grid;
  dim3 c5_nb_block;
  dim3 c5_nb_grid;
  dim3 c5_o_block;
  dim3 c5_o_grid;
  dim3 c5_ob_block;
  dim3 c5_ob_grid;
  dim3 c5_p_block;
  dim3 c5_p_grid;
  dim3 c5_pb_block;
  dim3 c5_pb_grid;
  dim3 c5_q_block;
  dim3 c5_q_grid;
  dim3 c5_qb_block;
  dim3 c5_qb_grid;
  dim3 c5_r_block;
  dim3 c5_r_grid;
  dim3 c5_rb_block;
  dim3 c5_rb_grid;
  dim3 c5_s_block;
  dim3 c5_s_grid;
  dim3 c5_sb_block;
  dim3 c5_sb_grid;
  dim3 c5_t_block;
  dim3 c5_t_grid;
  dim3 c5_tb_block;
  dim3 c5_tb_grid;
  dim3 c5_u_block;
  dim3 c5_u_grid;
  dim3 c5_ub_block;
  dim3 c5_ub_grid;
  dim3 c5_v_block;
  dim3 c5_v_grid;
  dim3 c5_vb_block;
  dim3 c5_vb_grid;
  dim3 c5_w_block;
  dim3 c5_w_grid;
  dim3 c5_wb_block;
  dim3 c5_wb_grid;
  dim3 c5_x_block;
  dim3 c5_x_grid;
  dim3 c5_y_block;
  dim3 c5_y_grid;
  real_T c5_b_bboxPred_data[3136];
  real_T c5_bboxPred_data[3136];
  real_T c5_bboxesX1Y1X2Y2_data[3136];
  real_T c5_area_data[784];
  real_T c5_x2_data[784];
  real_T c5_y1_data[784];
  real_T c5_y2_data[784];
  real_T c5_areaOfIntersect;
  real_T c5_count;
  real_T c5_height;
  real_T c5_width;
  int32_T c5_idx_data[784];
  int32_T c5_b_size[2];
  int32_T c5_bboxPred_size[2];
  int32_T c5_bboxesX1Y1X2Y2_size[2];
  int32_T c5_d_bboxPred_size[2];
  int32_T c5_e_bboxPred_size[2];
  int32_T c5_inDims[2];
  int32_T c5_iv[2];
  int32_T c5_iv1[2];
  int32_T c5_thresholdedPrediction_size[2];
  int32_T c5_b_bboxPred_size[1];
  int32_T c5_b_scores_size[1];
  int32_T c5_c_bboxPred_size[1];
  int32_T c5_classPred_size[1];
  int32_T c5_ii_size[1];
  int32_T c5_index_size[1];
  int32_T c5_iv1_size[1];
  int32_T c5_scorePred_size[1];
  int32_T c5_selectedIndex_size[1];
  int32_T c5_x1_size[1];
  int32_T c5_x2_size[1];
  int32_T c5_y1_size[1];
  int32_T c5_y2_size[1];
  int32_T c5_b_end;
  int32_T c5_b_i;
  int32_T c5_b_i14;
  int32_T c5_b_i7;
  int32_T c5_b_j;
  int32_T c5_b_k;
  int32_T c5_b_k0;
  int32_T c5_b_n;
  int32_T c5_b_nx;
  int32_T c5_b_nxin;
  int32_T c5_b_partialTrueCount;
  int32_T c5_b_trueCount;
  int32_T c5_c_i;
  int32_T c5_c_j;
  int32_T c5_c_k;
  int32_T c5_c_n;
  int32_T c5_c_partialTrueCount;
  int32_T c5_c_trueCount;
  int32_T c5_currentBox;
  int32_T c5_d_i;
  int32_T c5_d_k;
  int32_T c5_e_i;
  int32_T c5_e_k;
  int32_T c5_end;
  int32_T c5_f_i;
  int32_T c5_f_k;
  int32_T c5_g_i;
  int32_T c5_g_k;
  int32_T c5_h_i;
  int32_T c5_h_k;
  int32_T c5_i;
  int32_T c5_i1;
  int32_T c5_i10;
  int32_T c5_i11;
  int32_T c5_i12;
  int32_T c5_i13;
  int32_T c5_i14;
  int32_T c5_i17;
  int32_T c5_i19;
  int32_T c5_i25;
  int32_T c5_i26;
  int32_T c5_i27;
  int32_T c5_i3;
  int32_T c5_i32;
  int32_T c5_i35;
  int32_T c5_i39;
  int32_T c5_i4;
  int32_T c5_i47;
  int32_T c5_i5;
  int32_T c5_i51;
  int32_T c5_i6;
  int32_T c5_i61;
  int32_T c5_i64;
  int32_T c5_i66;
  int32_T c5_i7;
  int32_T c5_i8;
  int32_T c5_i9;
  int32_T c5_i_i;
  int32_T c5_idx;
  int32_T c5_ii;
  int32_T c5_j;
  int32_T c5_j_i;
  int32_T c5_k;
  int32_T c5_k0;
  int32_T c5_n;
  int32_T c5_nrows;
  int32_T c5_nrowx;
  int32_T c5_nx;
  int32_T c5_nxin;
  int32_T c5_nxout;
  int32_T c5_partialTrueCount;
  int32_T c5_sortDim;
  int32_T c5_trueCount;
  real32_T c5_tmpFeatureMap[4704];
  real32_T c5_c_scores_data[784];
  real32_T c5_classPred_data[784];
  real32_T c5_scorePred_data[784];
  real32_T c5_outVal[2];
  real32_T c5_y;
  uint32_T c5_dv2[2];
  int16_T c5_ii_data[784];
  int16_T c5_iv1_data[784];
  boolean_T c5_b_data[784];
  boolean_T c5_bv[784];
  boolean_T c5_index_data[784];
  boolean_T c5_selectedIndex_data[784];
  boolean_T c5_ab_validLaunchParams;
  boolean_T c5_area_data_dirtyOnGpu;
  boolean_T c5_b_bboxPred_data_dirtyOnGpu;
  boolean_T c5_b_bboxPred_size_dirtyOnCpu;
  boolean_T c5_b_data_dirtyOnCpu;
  boolean_T c5_b_data_dirtyOnGpu;
  boolean_T c5_b_n_dirtyOnGpu;
  boolean_T c5_b_validLaunchParams;
  boolean_T c5_bb_validLaunchParams;
  boolean_T c5_bboxPred_data_dirtyOnCpu;
  boolean_T c5_bboxPred_data_dirtyOnGpu;
  boolean_T c5_bboxPred_size_dirtyOnCpu;
  boolean_T c5_bboxesX1Y1X2Y2_data_dirtyOnGpu;
  boolean_T c5_bboxesX1Y1X2Y2_size_dirtyOnCpu;
  boolean_T c5_bboxes_data_dirtyOnGpu;
  boolean_T c5_bv_dirtyOnGpu;
  boolean_T c5_c_bboxPred_size_dirtyOnCpu;
  boolean_T c5_c_validLaunchParams;
  boolean_T c5_cb_validLaunchParams;
  boolean_T c5_classPred_data_dirtyOnCpu;
  boolean_T c5_classPred_data_dirtyOnGpu;
  boolean_T c5_d_validLaunchParams;
  boolean_T c5_db_validLaunchParams;
  boolean_T c5_e_validLaunchParams;
  boolean_T c5_eb_validLaunchParams;
  boolean_T c5_exitg1;
  boolean_T c5_f_validLaunchParams;
  boolean_T c5_fb_validLaunchParams;
  boolean_T c5_g_validLaunchParams;
  boolean_T c5_gb_validLaunchParams;
  boolean_T c5_guard1 = false;
  boolean_T c5_h_validLaunchParams;
  boolean_T c5_hb_validLaunchParams;
  boolean_T c5_i_validLaunchParams;
  boolean_T c5_ib_validLaunchParams;
  boolean_T c5_idx_data_dirtyOnGpu;
  boolean_T c5_ii_data_dirtyOnCpu;
  boolean_T c5_index_data_dirtyOnGpu;
  boolean_T c5_iv1_data_dirtyOnCpu;
  boolean_T c5_j_validLaunchParams;
  boolean_T c5_jb_validLaunchParams;
  boolean_T c5_k_validLaunchParams;
  boolean_T c5_kb_validLaunchParams;
  boolean_T c5_l_validLaunchParams;
  boolean_T c5_lb_validLaunchParams;
  boolean_T c5_m_validLaunchParams;
  boolean_T c5_mb_validLaunchParams;
  boolean_T c5_n_dirtyOnCpu;
  boolean_T c5_n_dirtyOnGpu;
  boolean_T c5_n_validLaunchParams;
  boolean_T c5_nb_validLaunchParams;
  boolean_T c5_nrows_dirtyOnGpu;
  boolean_T c5_o_validLaunchParams;
  boolean_T c5_ob_validLaunchParams;
  boolean_T c5_p_validLaunchParams;
  boolean_T c5_pb_validLaunchParams;
  boolean_T c5_q_validLaunchParams;
  boolean_T c5_qb_validLaunchParams;
  boolean_T c5_r_validLaunchParams;
  boolean_T c5_rb_validLaunchParams;
  boolean_T c5_s_validLaunchParams;
  boolean_T c5_sb_validLaunchParams;
  boolean_T c5_scorePred_data_dirtyOnCpu;
  boolean_T c5_scorePred_data_dirtyOnGpu;
  boolean_T c5_scores_data_dirtyOnCpu;
  boolean_T c5_scores_data_dirtyOnGpu;
  boolean_T c5_selectedIndex_data_dirtyOnCpu;
  boolean_T c5_selectedIndex_data_dirtyOnGpu;
  boolean_T c5_t_validLaunchParams;
  boolean_T c5_tb_validLaunchParams;
  boolean_T c5_thresholdedPrediction_size_dirtyOnCpu;
  boolean_T c5_u_validLaunchParams;
  boolean_T c5_ub_validLaunchParams;
  boolean_T c5_v_validLaunchParams;
  boolean_T c5_validLaunchParams;
  boolean_T c5_vb_validLaunchParams;
  boolean_T c5_w_validLaunchParams;
  boolean_T c5_wb_validLaunchParams;
  boolean_T c5_x1_size_dirtyOnCpu;
  boolean_T c5_x2_data_dirtyOnGpu;
  boolean_T c5_x_validLaunchParams;
  boolean_T c5_y1_data_dirtyOnGpu;
  boolean_T c5_y2_data_dirtyOnGpu;
  boolean_T c5_y_validLaunchParams;
  c5_scores_data_dirtyOnCpu = false;
  c5_iv1_data_dirtyOnCpu = false;
  c5_selectedIndex_data_dirtyOnCpu = false;
  c5_b_data_dirtyOnCpu = false;
  c5_scorePred_data_dirtyOnCpu = false;
  c5_classPred_data_dirtyOnCpu = false;
  c5_bboxPred_data_dirtyOnCpu = false;
  c5_ii_data_dirtyOnCpu = false;
  c5_scores_data_dirtyOnGpu = false;
  c5_bboxes_data_dirtyOnGpu = false;
  c5_index_data_dirtyOnGpu = false;
  c5_area_data_dirtyOnGpu = false;
  c5_selectedIndex_data_dirtyOnGpu = false;
  c5_b_data_dirtyOnGpu = false;
  c5_idx_data_dirtyOnGpu = false;
  c5_scorePred_data_dirtyOnGpu = false;
  c5_classPred_data_dirtyOnGpu = false;
  c5_bboxPred_data_dirtyOnGpu = false;
  c5_b_bboxPred_data_dirtyOnGpu = false;
  c5_y2_data_dirtyOnGpu = false;
  c5_x2_data_dirtyOnGpu = false;
  c5_y1_data_dirtyOnGpu = false;
  c5_bboxesX1Y1X2Y2_data_dirtyOnGpu = false;
  if (!chartInstance->c5_yolodetector_not_empty) {
    c5_iobj_0 = &chartInstance->c5_gobj_0;
    c5_DeepLearningNetwork_setup(chartInstance, &chartInstance->c5_gobj_0);
    chartInstance->c5_yolodetector.Network = c5_iobj_0;
    chartInstance->c5_yolodetector_not_empty = true;
  }

  c5_this_Network = chartInstance->c5_yolodetector.Network;
  c5_eML_blk_kernel_kernel1<<<dim3(2U, 1U, 1U), dim3(512U, 1U, 1U)>>>
    (*chartInstance->c5_gpu_aux1);
  c5_eML_blk_kernel_kernel2<<<dim3(3U, 1U, 1U), dim3(512U, 1U, 1U)>>>
    (*chartInstance->c5_gpu_aux2);
  c5_eML_blk_kernel_kernel3<<<dim3(4U, 1U, 1U), dim3(512U, 1U, 1U)>>>
    (*chartInstance->c5_gpu_aux1, *chartInstance->c5_gpu_rowWeights,
     *chartInstance->c5_gpu_ipRowIndices);
  c5_eML_blk_kernel_kernel4<<<dim3(6U, 1U, 1U), dim3(512U, 1U, 1U)>>>
    (*chartInstance->c5_gpu_aux2, *chartInstance->c5_gpu_colWeights,
     *chartInstance->c5_gpu_ipColIndices);
  c5_eML_blk_kernel_kernel5<<<dim3(1U, 1U, 1U), dim3(224U, 1U, 1U)>>>
    (*chartInstance->c5_gpu_rowWeights, *chartInstance->c5_gpu_rowWeightsTotal);
  for (c5_k = 0; c5_k < 8; c5_k++) {
    c5_eML_blk_kernel_kernel6<<<dim3(1U, 1U, 1U), dim3(224U, 1U, 1U)>>>
      (*chartInstance->c5_gpu_rowWeights, (c5_k + 1) * 224,
       *chartInstance->c5_gpu_rowWeightsTotal);
  }

  c5_eML_blk_kernel_kernel7<<<dim3(1U, 1U, 1U), dim3(224U, 1U, 1U)>>>
    (*chartInstance->c5_gpu_colWeights, *chartInstance->c5_gpu_colWeightsTotal);
  for (c5_b_k = 0; c5_b_k < 11; c5_b_k++) {
    c5_eML_blk_kernel_kernel8<<<dim3(1U, 1U, 1U), dim3(224U, 1U, 1U)>>>
      (*chartInstance->c5_gpu_colWeights, (c5_b_k + 1) * 224,
       *chartInstance->c5_gpu_colWeightsTotal);
  }

  hipMemcpy(chartInstance->c5_gpu_In, &c5_b_In[0], 3686400UL,
             hipMemcpyHostToDevice);
  c5_eML_blk_kernel_kernel9<<<dim3(630U, 1U, 1U), dim3(512U, 1U, 1U)>>>
    (*chartInstance->c5_gpu_colWeightsTotal, *chartInstance->c5_gpu_colWeights, *
     chartInstance->c5_gpu_In, *chartInstance->c5_gpu_ipColIndices,
     *chartInstance->c5_gpu_partialResize);
  c5_eML_blk_kernel_kernel10<<<dim3(294U, 1U, 1U), dim3(512U, 1U, 1U)>>>
    (*chartInstance->c5_gpu_rowWeightsTotal, *chartInstance->c5_gpu_rowWeights, *
     chartInstance->c5_gpu_partialResize, *chartInstance->c5_gpu_ipRowIndices,
     *chartInstance->c5_gpu_out);
  c5_eML_blk_kernel_kernel11<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
    (*chartInstance->c5_gpu_out, *chartInstance->c5_b_gpu_outVal);
  c5_coder_reduce0<<<dim3(294U, 1U, 1U), dim3(512U, 1U, 1U)>>>
    (*chartInstance->c5_gpu_out, *chartInstance->c5_b_gpu_outVal);
  hipMemcpy(&c5_outVal[0], chartInstance->c5_b_gpu_outVal, 8UL,
             hipMemcpyDeviceToHost);
  c5_y = c5_outVal[1] - c5_outVal[0];
  c5_eML_blk_kernel_kernel12<<<dim3(294U, 1U, 1U), dim3(512U, 1U, 1U)>>>(c5_y,
    c5_outVal[0], *chartInstance->c5_gpu_out, *chartInstance->c5_b_gpu_out);
  hipMemcpy(&chartInstance->c5_out[0], chartInstance->c5_b_gpu_out, 602112UL,
             hipMemcpyDeviceToHost);
  c5_DeepLearningNetwork_activations(chartInstance, c5_this_Network,
    chartInstance->c5_out, c5_tmpFeatureMap);
  hipMemcpy(chartInstance->c5_gpu_dv, &c5_dv[0], 8UL, hipMemcpyHostToDevice);
  c5_eML_blk_kernel_kernel13<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
    (*chartInstance->c5_gpu_dv, *chartInstance->c5_gpu_anchors);
  hipMemcpy(chartInstance->c5_gpu_dv1, &c5_dv1[0], 32UL, hipMemcpyHostToDevice);
  c5_eML_blk_kernel_kernel14<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
    (*chartInstance->c5_gpu_dv1, *chartInstance->c5_b_gpu_anchors,
     *chartInstance->c5_gpu_anchors);
  hipMemcpy(chartInstance->c5_gpu_tmpFeatureMap, &c5_tmpFeatureMap[0], 18816UL,
             hipMemcpyHostToDevice);
  c5_eML_blk_kernel_kernel15<<<dim3(2U, 1U, 1U), dim3(512U, 1U, 1U)>>>
    (*chartInstance->c5_gpu_anchors, *chartInstance->c5_gpu_tmpFeatureMap,
     *chartInstance->c5_gpu_boxOut);
  c5_eML_blk_kernel_kernel16<<<dim3(2U, 1U, 1U), dim3(512U, 1U, 1U)>>>
    (*chartInstance->c5_gpu_boxOut, *chartInstance->c5_gpu_bv);
  c5_bv_dirtyOnGpu = true;
  c5_trueCount = 0;
  for (c5_i = 0; c5_i < 784; c5_i++) {
    if (c5_bv_dirtyOnGpu) {
      hipMemcpy(&c5_bv[0], chartInstance->c5_gpu_bv, 784UL,
                 hipMemcpyDeviceToHost);
      c5_bv_dirtyOnGpu = false;
    }

    if (c5_bv[c5_i]) {
      c5_trueCount++;
    }
  }

  c5_ii_size[0] = c5_trueCount;
  c5_partialTrueCount = 0;
  for (c5_b_i = 0; c5_b_i < 784; c5_b_i++) {
    if (c5_bv_dirtyOnGpu) {
      hipMemcpy(&c5_bv[0], chartInstance->c5_gpu_bv, 784UL,
                 hipMemcpyDeviceToHost);
      c5_bv_dirtyOnGpu = false;
    }

    if (c5_bv[c5_b_i]) {
      c5_ii_data[c5_partialTrueCount] = (int16_T)(c5_b_i + 1);
      c5_ii_data_dirtyOnCpu = true;
      c5_partialTrueCount++;
    }
  }

  c5_thresholdedPrediction_size[0] = c5_trueCount;
  c5_thresholdedPrediction_size[1] = 6;
  c5_thresholdedPrediction_size_dirtyOnCpu = true;
  c5_validLaunchParams = mwGetLaunchParameters((real_T)(((int64_T)(c5_ii_size[0]
    - 1) + 1L) * 6L), &c5_grid, &c5_block, 1024U, 65535U);
  if (c5_validLaunchParams) {
    if (c5_ii_data_dirtyOnCpu) {
      hipMemcpy(chartInstance->c5_gpu_ii_data, &c5_ii_data[0], 1568UL,
                 hipMemcpyHostToDevice);
    }

    hipMemcpy(chartInstance->c5_gpu_thresholdedPrediction_size,
               &c5_thresholdedPrediction_size[0], 8UL, hipMemcpyHostToDevice);
    c5_thresholdedPrediction_size_dirtyOnCpu = false;
    hipMemcpy(chartInstance->c5_gpu_ii_size, &c5_ii_size[0], 4UL,
               hipMemcpyHostToDevice);
    c5_eML_blk_kernel_kernel17<<<c5_grid, c5_block>>>
      (*chartInstance->c5_gpu_boxOut, *chartInstance->c5_gpu_ii_data,
       *chartInstance->c5_gpu_thresholdedPrediction_size,
       *chartInstance->c5_gpu_ii_size,
       *chartInstance->c5_gpu_thresholdedPrediction_data);
  }

  if (c5_ii_size[0] != 0) {
    c5_i7 = c5_ii_size[0] - 1;
    c5_bboxesX1Y1X2Y2_size[0] = c5_ii_size[0];
    c5_bboxesX1Y1X2Y2_size[1] = 4;
    c5_bboxesX1Y1X2Y2_size_dirtyOnCpu = true;
    c5_b_validLaunchParams = mwGetLaunchParameters((real_T)(((int64_T)c5_i7 + 1L)
      * 4L), &c5_b_grid, &c5_b_block, 1024U, 65535U);
    if (c5_b_validLaunchParams) {
      if (c5_thresholdedPrediction_size_dirtyOnCpu) {
        hipMemcpy(chartInstance->c5_gpu_thresholdedPrediction_size,
                   &c5_thresholdedPrediction_size[0], 8UL,
                   hipMemcpyHostToDevice);
        c5_thresholdedPrediction_size_dirtyOnCpu = false;
      }

      hipMemcpy(chartInstance->c5_gpu_bboxesX1Y1X2Y2_size,
                 &c5_bboxesX1Y1X2Y2_size[0], 8UL, hipMemcpyHostToDevice);
      c5_bboxesX1Y1X2Y2_size_dirtyOnCpu = false;
      c5_eML_blk_kernel_kernel18<<<c5_b_grid, c5_b_block>>>
        (*chartInstance->c5_gpu_thresholdedPrediction_data,
         *chartInstance->c5_gpu_thresholdedPrediction_size,
         *chartInstance->c5_gpu_bboxesX1Y1X2Y2_size, c5_i7,
         *chartInstance->c5_gpu_bboxesX1Y1X2Y2_data);
      c5_bboxesX1Y1X2Y2_data_dirtyOnGpu = true;
    }

    c5_i10 = c5_bboxesX1Y1X2Y2_size[0] - 1;
    c5_x1_size[0] = c5_bboxesX1Y1X2Y2_size[0];
    c5_d_validLaunchParams = mwGetLaunchParameters((real_T)((int64_T)c5_i10 + 1L),
      &c5_d_grid, &c5_d_block, 1024U, 65535U);
    if (c5_d_validLaunchParams) {
      c5_eML_blk_kernel_kernel19<<<c5_d_grid, c5_d_block>>>
        (*chartInstance->c5_gpu_bboxesX1Y1X2Y2_data, c5_i10,
         *chartInstance->c5_gpu_x1_data);
    }

    c5_i14 = c5_bboxesX1Y1X2Y2_size[0] - 1;
    c5_y1_size[0] = c5_bboxesX1Y1X2Y2_size[0];
    c5_f_validLaunchParams = mwGetLaunchParameters((real_T)((int64_T)c5_i14 + 1L),
      &c5_f_grid, &c5_f_block, 1024U, 65535U);
    if (c5_f_validLaunchParams) {
      if (c5_bboxesX1Y1X2Y2_size_dirtyOnCpu) {
        hipMemcpy(chartInstance->c5_gpu_bboxesX1Y1X2Y2_size,
                   &c5_bboxesX1Y1X2Y2_size[0], 8UL, hipMemcpyHostToDevice);
        c5_bboxesX1Y1X2Y2_size_dirtyOnCpu = false;
      }

      c5_eML_blk_kernel_kernel20<<<c5_f_grid, c5_f_block>>>
        (*chartInstance->c5_gpu_bboxesX1Y1X2Y2_data,
         *chartInstance->c5_gpu_bboxesX1Y1X2Y2_size, c5_i14,
         *chartInstance->c5_gpu_y1_data);
      c5_y1_data_dirtyOnGpu = true;
    }

    c5_i17 = c5_bboxesX1Y1X2Y2_size[0] - 1;
    c5_x2_size[0] = c5_bboxesX1Y1X2Y2_size[0];
    c5_g_validLaunchParams = mwGetLaunchParameters((real_T)((int64_T)c5_i17 + 1L),
      &c5_g_grid, &c5_g_block, 1024U, 65535U);
    if (c5_g_validLaunchParams) {
      if (c5_bboxesX1Y1X2Y2_size_dirtyOnCpu) {
        hipMemcpy(chartInstance->c5_gpu_bboxesX1Y1X2Y2_size,
                   &c5_bboxesX1Y1X2Y2_size[0], 8UL, hipMemcpyHostToDevice);
        c5_bboxesX1Y1X2Y2_size_dirtyOnCpu = false;
      }

      c5_eML_blk_kernel_kernel21<<<c5_g_grid, c5_g_block>>>
        (*chartInstance->c5_gpu_bboxesX1Y1X2Y2_data,
         *chartInstance->c5_gpu_bboxesX1Y1X2Y2_size, c5_i17,
         *chartInstance->c5_gpu_x2_data);
      c5_x2_data_dirtyOnGpu = true;
    }

    c5_i19 = c5_bboxesX1Y1X2Y2_size[0] - 1;
    c5_y2_size[0] = c5_bboxesX1Y1X2Y2_size[0];
    c5_h_validLaunchParams = mwGetLaunchParameters((real_T)((int64_T)c5_i19 + 1L),
      &c5_h_grid, &c5_h_block, 1024U, 65535U);
    if (c5_h_validLaunchParams) {
      if (c5_bboxesX1Y1X2Y2_size_dirtyOnCpu) {
        hipMemcpy(chartInstance->c5_gpu_bboxesX1Y1X2Y2_size,
                   &c5_bboxesX1Y1X2Y2_size[0], 8UL, hipMemcpyHostToDevice);
      }

      c5_eML_blk_kernel_kernel22<<<c5_h_grid, c5_h_block>>>
        (*chartInstance->c5_gpu_bboxesX1Y1X2Y2_data,
         *chartInstance->c5_gpu_bboxesX1Y1X2Y2_size, c5_i19,
         *chartInstance->c5_gpu_y2_data);
      c5_y2_data_dirtyOnGpu = true;
    }

    c5_i_validLaunchParams = mwGetLaunchParameters((real_T)((int64_T)
      (c5_bboxesX1Y1X2Y2_size[0] - 1) + 1L), &c5_i_grid, &c5_i_block, 1024U,
      65535U);
    if (c5_i_validLaunchParams) {
      c5_eML_blk_kernel_kernel23<<<c5_i_grid, c5_i_block>>>
        (c5_bboxesX1Y1X2Y2_size[0], *chartInstance->c5_gpu_x1_data);
    }

    c5_j_validLaunchParams = mwGetLaunchParameters((real_T)((int64_T)
      (c5_bboxesX1Y1X2Y2_size[0] - 1) + 1L), &c5_j_grid, &c5_j_block, 1024U,
      65535U);
    if (c5_j_validLaunchParams) {
      c5_eML_blk_kernel_kernel24<<<c5_j_grid, c5_j_block>>>
        (c5_bboxesX1Y1X2Y2_size[0], *chartInstance->c5_gpu_y1_data);
      c5_y1_data_dirtyOnGpu = true;
    }

    c5_k_validLaunchParams = mwGetLaunchParameters((real_T)((int64_T)
      (c5_bboxesX1Y1X2Y2_size[0] - 1) + 1L), &c5_k_grid, &c5_k_block, 1024U,
      65535U);
    if (c5_k_validLaunchParams) {
      c5_eML_blk_kernel_kernel25<<<c5_k_grid, c5_k_block>>>
        (c5_bboxesX1Y1X2Y2_size[0], *chartInstance->c5_gpu_x2_data);
      c5_x2_data_dirtyOnGpu = true;
    }

    c5_l_validLaunchParams = mwGetLaunchParameters((real_T)((int64_T)
      (c5_bboxesX1Y1X2Y2_size[0] - 1) + 1L), &c5_l_grid, &c5_l_block, 1024U,
      65535U);
    if (c5_l_validLaunchParams) {
      c5_eML_blk_kernel_kernel26<<<c5_l_grid, c5_l_block>>>
        (c5_bboxesX1Y1X2Y2_size[0], *chartInstance->c5_gpu_y2_data);
      c5_y2_data_dirtyOnGpu = true;
    }

    c5_bboxesX1Y1X2Y2_size[0] = c5_x1_size[0];
    c5_bboxesX1Y1X2Y2_size[1] = 4;
    c5_bboxesX1Y1X2Y2_size_dirtyOnCpu = true;
    c5_m_validLaunchParams = mwGetLaunchParameters((real_T)((int64_T)
      (c5_x1_size[0] - 1) + 1L), &c5_m_grid, &c5_m_block, 1024U, 65535U);
    if (c5_m_validLaunchParams) {
      c5_eML_blk_kernel_kernel27<<<c5_m_grid, c5_m_block>>>
        (*chartInstance->c5_gpu_x1_data, c5_x1_size[0] - 1,
         *chartInstance->c5_gpu_bboxesX1Y1X2Y2_data);
      c5_bboxesX1Y1X2Y2_data_dirtyOnGpu = true;
    }

    c5_n_validLaunchParams = mwGetLaunchParameters((real_T)((int64_T)
      (c5_y1_size[0] - 1) + 1L), &c5_n_grid, &c5_n_block, 1024U, 65535U);
    if (c5_n_validLaunchParams) {
      hipMemcpy(chartInstance->c5_gpu_bboxesX1Y1X2Y2_size,
                 &c5_bboxesX1Y1X2Y2_size[0], 8UL, hipMemcpyHostToDevice);
      c5_bboxesX1Y1X2Y2_size_dirtyOnCpu = false;
      c5_eML_blk_kernel_kernel28<<<c5_n_grid, c5_n_block>>>
        (*chartInstance->c5_gpu_y1_data,
         *chartInstance->c5_gpu_bboxesX1Y1X2Y2_size, c5_y1_size[0] - 1,
         *chartInstance->c5_gpu_bboxesX1Y1X2Y2_data);
      c5_bboxesX1Y1X2Y2_data_dirtyOnGpu = true;
    }

    c5_o_validLaunchParams = mwGetLaunchParameters((real_T)((int64_T)
      (c5_x2_size[0] - 1) + 1L), &c5_o_grid, &c5_o_block, 1024U, 65535U);
    if (c5_o_validLaunchParams) {
      if (c5_bboxesX1Y1X2Y2_size_dirtyOnCpu) {
        hipMemcpy(chartInstance->c5_gpu_bboxesX1Y1X2Y2_size,
                   &c5_bboxesX1Y1X2Y2_size[0], 8UL, hipMemcpyHostToDevice);
        c5_bboxesX1Y1X2Y2_size_dirtyOnCpu = false;
      }

      c5_eML_blk_kernel_kernel29<<<c5_o_grid, c5_o_block>>>
        (*chartInstance->c5_gpu_x2_data,
         *chartInstance->c5_gpu_bboxesX1Y1X2Y2_size, c5_x2_size[0] - 1,
         *chartInstance->c5_gpu_bboxesX1Y1X2Y2_data);
      c5_bboxesX1Y1X2Y2_data_dirtyOnGpu = true;
    }

    c5_p_validLaunchParams = mwGetLaunchParameters((real_T)((int64_T)
      (c5_y2_size[0] - 1) + 1L), &c5_p_grid, &c5_p_block, 1024U, 65535U);
    if (c5_p_validLaunchParams) {
      if (c5_bboxesX1Y1X2Y2_size_dirtyOnCpu) {
        hipMemcpy(chartInstance->c5_gpu_bboxesX1Y1X2Y2_size,
                   &c5_bboxesX1Y1X2Y2_size[0], 8UL, hipMemcpyHostToDevice);
        c5_bboxesX1Y1X2Y2_size_dirtyOnCpu = false;
      }

      c5_eML_blk_kernel_kernel30<<<c5_p_grid, c5_p_block>>>
        (*chartInstance->c5_gpu_y2_data,
         *chartInstance->c5_gpu_bboxesX1Y1X2Y2_size, c5_y2_size[0] - 1,
         *chartInstance->c5_gpu_bboxesX1Y1X2Y2_data);
      c5_bboxesX1Y1X2Y2_data_dirtyOnGpu = true;
    }

    c5_i25 = c5_bboxesX1Y1X2Y2_size[0];
    c5_i26 = c5_bboxesX1Y1X2Y2_size[0];
    c5_i27 = c5_bboxesX1Y1X2Y2_size[0];
    c5_bboxPred_size[0] = c5_bboxesX1Y1X2Y2_size[0];
    c5_bboxPred_size[1] = 4;
    c5_bboxPred_size_dirtyOnCpu = true;
    c5_q_validLaunchParams = mwGetLaunchParameters((real_T)((int64_T)
      (c5_bboxesX1Y1X2Y2_size[0] - 1) + 1L), &c5_q_grid, &c5_q_block, 1024U,
      65535U);
    if (c5_q_validLaunchParams) {
      c5_eML_blk_kernel_kernel31<<<c5_q_grid, c5_q_block>>>
        (*chartInstance->c5_gpu_bboxesX1Y1X2Y2_data, c5_bboxesX1Y1X2Y2_size[0] -
         1, *chartInstance->c5_c_gpu_bboxPred_data);
      c5_b_bboxPred_data_dirtyOnGpu = true;
    }

    c5_r_validLaunchParams = mwGetLaunchParameters((real_T)((int64_T)(c5_i25 - 1)
      + 1L), &c5_r_grid, &c5_r_block, 1024U, 65535U);
    if (c5_r_validLaunchParams) {
      if (c5_bboxesX1Y1X2Y2_size_dirtyOnCpu) {
        hipMemcpy(chartInstance->c5_gpu_bboxesX1Y1X2Y2_size,
                   &c5_bboxesX1Y1X2Y2_size[0], 8UL, hipMemcpyHostToDevice);
        c5_bboxesX1Y1X2Y2_size_dirtyOnCpu = false;
      }

      hipMemcpy(chartInstance->c5_d_gpu_bboxPred_size, &c5_bboxPred_size[0],
                 8UL, hipMemcpyHostToDevice);
      c5_bboxPred_size_dirtyOnCpu = false;
      c5_eML_blk_kernel_kernel32<<<c5_r_grid, c5_r_block>>>
        (*chartInstance->c5_gpu_bboxesX1Y1X2Y2_data,
         *chartInstance->c5_gpu_bboxesX1Y1X2Y2_size,
         *chartInstance->c5_d_gpu_bboxPred_size, c5_i25 - 1,
         *chartInstance->c5_c_gpu_bboxPred_data);
      c5_b_bboxPred_data_dirtyOnGpu = true;
    }

    c5_s_validLaunchParams = mwGetLaunchParameters((real_T)((int64_T)(c5_i26 - 1)
      + 1L), &c5_s_grid, &c5_s_block, 1024U, 65535U);
    if (c5_s_validLaunchParams) {
      if (c5_bboxesX1Y1X2Y2_size_dirtyOnCpu) {
        hipMemcpy(chartInstance->c5_gpu_bboxesX1Y1X2Y2_size,
                   &c5_bboxesX1Y1X2Y2_size[0], 8UL, hipMemcpyHostToDevice);
        c5_bboxesX1Y1X2Y2_size_dirtyOnCpu = false;
      }

      if (c5_bboxPred_size_dirtyOnCpu) {
        hipMemcpy(chartInstance->c5_d_gpu_bboxPred_size, &c5_bboxPred_size[0],
                   8UL, hipMemcpyHostToDevice);
        c5_bboxPred_size_dirtyOnCpu = false;
      }

      c5_eML_blk_kernel_kernel33<<<c5_s_grid, c5_s_block>>>
        (*chartInstance->c5_gpu_bboxesX1Y1X2Y2_data,
         *chartInstance->c5_gpu_bboxesX1Y1X2Y2_size,
         *chartInstance->c5_d_gpu_bboxPred_size, c5_i26 - 1,
         *chartInstance->c5_c_gpu_bboxPred_data);
      c5_b_bboxPred_data_dirtyOnGpu = true;
    }

    c5_t_validLaunchParams = mwGetLaunchParameters((real_T)((int64_T)(c5_i27 - 1)
      + 1L), &c5_t_grid, &c5_t_block, 1024U, 65535U);
    if (c5_t_validLaunchParams) {
      if (c5_bboxesX1Y1X2Y2_size_dirtyOnCpu) {
        hipMemcpy(chartInstance->c5_gpu_bboxesX1Y1X2Y2_size,
                   &c5_bboxesX1Y1X2Y2_size[0], 8UL, hipMemcpyHostToDevice);
      }

      if (c5_bboxPred_size_dirtyOnCpu) {
        hipMemcpy(chartInstance->c5_d_gpu_bboxPred_size, &c5_bboxPred_size[0],
                   8UL, hipMemcpyHostToDevice);
        c5_bboxPred_size_dirtyOnCpu = false;
      }

      c5_eML_blk_kernel_kernel34<<<c5_t_grid, c5_t_block>>>
        (*chartInstance->c5_gpu_bboxesX1Y1X2Y2_data,
         *chartInstance->c5_gpu_bboxesX1Y1X2Y2_size,
         *chartInstance->c5_d_gpu_bboxPred_size, c5_i27 - 1,
         *chartInstance->c5_c_gpu_bboxPred_data);
      c5_b_bboxPred_data_dirtyOnGpu = true;
    }

    c5_nx = c5_bboxPred_size[0] << 2;
    c5_u_validLaunchParams = mwGetLaunchParameters((real_T)((int64_T)(c5_nx - 1)
      + 1L), &c5_u_grid, &c5_u_block, 1024U, 65535U);
    if (c5_u_validLaunchParams) {
      c5_eML_blk_kernel_kernel35<<<c5_u_grid, c5_u_block>>>(c5_nx,
        *chartInstance->c5_c_gpu_bboxPred_data);
      c5_b_bboxPred_data_dirtyOnGpu = true;
    }

    c5_i32 = c5_bboxPred_size[0] - 1;
    c5_b_bboxPred_size[0] = c5_bboxPred_size[0];
    c5_v_validLaunchParams = mwGetLaunchParameters((real_T)((int64_T)c5_i32 + 1L),
      &c5_v_grid, &c5_v_block, 1024U, 65535U);
    if (c5_v_validLaunchParams) {
      if (c5_bboxPred_size_dirtyOnCpu) {
        hipMemcpy(chartInstance->c5_d_gpu_bboxPred_size, &c5_bboxPred_size[0],
                   8UL, hipMemcpyHostToDevice);
        c5_bboxPred_size_dirtyOnCpu = false;
      }

      c5_eML_blk_kernel_kernel36<<<c5_v_grid, c5_v_block>>>
        (*chartInstance->c5_c_gpu_bboxPred_data,
         *chartInstance->c5_d_gpu_bboxPred_size, c5_i32,
         *chartInstance->c5_gpu_bboxPred_data);
    }

    c5_w_validLaunchParams = mwGetLaunchParameters((real_T)((int64_T)
      (c5_b_bboxPred_size[0] - 1) + 1L), &c5_w_grid, &c5_w_block, 1024U, 65535U);
    if (c5_w_validLaunchParams) {
      if (c5_bboxPred_size_dirtyOnCpu) {
        hipMemcpy(chartInstance->c5_d_gpu_bboxPred_size, &c5_bboxPred_size[0],
                   8UL, hipMemcpyHostToDevice);
        c5_bboxPred_size_dirtyOnCpu = false;
      }

      hipMemcpy(chartInstance->c5_c_gpu_bboxPred_size, &c5_b_bboxPred_size[0],
                 4UL, hipMemcpyHostToDevice);
      c5_eML_blk_kernel_kernel37<<<c5_w_grid, c5_w_block>>>
        (*chartInstance->c5_gpu_bboxPred_data,
         *chartInstance->c5_d_gpu_bboxPred_size,
         *chartInstance->c5_c_gpu_bboxPred_size,
         *chartInstance->c5_c_gpu_bboxPred_data);
      c5_b_bboxPred_data_dirtyOnGpu = true;
    }

    c5_i35 = c5_bboxPred_size[0] - 1;
    c5_c_bboxPred_size[0] = c5_bboxPred_size[0];
    c5_x_validLaunchParams = mwGetLaunchParameters((real_T)((int64_T)c5_i35 + 1L),
      &c5_x_grid, &c5_x_block, 1024U, 65535U);
    if (c5_x_validLaunchParams) {
      if (c5_bboxPred_size_dirtyOnCpu) {
        hipMemcpy(chartInstance->c5_d_gpu_bboxPred_size, &c5_bboxPred_size[0],
                   8UL, hipMemcpyHostToDevice);
        c5_bboxPred_size_dirtyOnCpu = false;
      }

      c5_eML_blk_kernel_kernel38<<<c5_x_grid, c5_x_block>>>
        (*chartInstance->c5_c_gpu_bboxPred_data,
         *chartInstance->c5_d_gpu_bboxPred_size, c5_i35,
         *chartInstance->c5_d_gpu_bboxPred_data);
    }

    c5_y_validLaunchParams = mwGetLaunchParameters((real_T)((int64_T)
      (c5_c_bboxPred_size[0] - 1) + 1L), &c5_y_grid, &c5_y_block, 1024U, 65535U);
    if (c5_y_validLaunchParams) {
      if (c5_bboxPred_size_dirtyOnCpu) {
        hipMemcpy(chartInstance->c5_d_gpu_bboxPred_size, &c5_bboxPred_size[0],
                   8UL, hipMemcpyHostToDevice);
        c5_bboxPred_size_dirtyOnCpu = false;
      }

      hipMemcpy(chartInstance->c5_b_gpu_bboxPred_size, &c5_c_bboxPred_size[0],
                 4UL, hipMemcpyHostToDevice);
      c5_eML_blk_kernel_kernel39<<<c5_y_grid, c5_y_block>>>
        (*chartInstance->c5_d_gpu_bboxPred_data,
         *chartInstance->c5_d_gpu_bboxPred_size,
         *chartInstance->c5_b_gpu_bboxPred_size,
         *chartInstance->c5_c_gpu_bboxPred_data);
      c5_b_bboxPred_data_dirtyOnGpu = true;
    }

    c5_count = 0.0;
    c5_d_bboxPred_size[0] = c5_bboxPred_size[0];
    c5_d_bboxPred_size[1] = 4;
    c5_b_bboxPred_size_dirtyOnCpu = true;
    c5_scorePred_size[0] = c5_bboxPred_size[0];
    c5_classPred_size[0] = c5_bboxPred_size[0];
    c5_c_i = c5_bboxPred_size[0];
    for (c5_d_i = 0; c5_d_i < c5_c_i; c5_d_i++) {
      if (c5_b_bboxPred_data_dirtyOnGpu) {
        hipMemcpy(&c5_bboxPred_data[0], chartInstance->c5_c_gpu_bboxPred_data,
                   25088UL, hipMemcpyDeviceToHost);
        c5_b_bboxPred_data_dirtyOnGpu = false;
      }

      if ((c5_bboxPred_data[c5_d_i + c5_bboxPred_size[0] * 3] >= 1.0) &&
          (c5_bboxPred_data[c5_d_i + (c5_bboxPred_size[0] << 1)] >= 1.0) &&
          (c5_bboxPred_data[c5_d_i + c5_bboxPred_size[0] * 3] <= 480.0) &&
          (c5_bboxPred_data[c5_d_i + (c5_bboxPred_size[0] << 1)] <= 640.0)) {
        c5_count++;
        if (c5_bboxPred_size_dirtyOnCpu) {
          hipMemcpy(chartInstance->c5_d_gpu_bboxPred_size, &c5_bboxPred_size[0],
                     8UL, hipMemcpyHostToDevice);
          c5_bboxPred_size_dirtyOnCpu = false;
        }

        if (c5_b_bboxPred_size_dirtyOnCpu) {
          hipMemcpy(chartInstance->c5_gpu_bboxPred_size, &c5_d_bboxPred_size[0],
                     8UL, hipMemcpyHostToDevice);
          c5_b_bboxPred_size_dirtyOnCpu = false;
        }

        c5_eML_blk_kernel_kernel40<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
          (*chartInstance->c5_c_gpu_bboxPred_data,
           *chartInstance->c5_d_gpu_bboxPred_size, c5_d_i,
           *chartInstance->c5_gpu_bboxPred_size, (int32_T)c5_count - 1,
           *chartInstance->c5_b_gpu_bboxPred_data);
        c5_bboxPred_data_dirtyOnGpu = true;
        if (c5_thresholdedPrediction_size_dirtyOnCpu) {
          hipMemcpy(chartInstance->c5_gpu_thresholdedPrediction_size,
                     &c5_thresholdedPrediction_size[0], 8UL,
                     hipMemcpyHostToDevice);
          c5_thresholdedPrediction_size_dirtyOnCpu = false;
        }

        c5_eML_blk_kernel_kernel41<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
          (*chartInstance->c5_gpu_thresholdedPrediction_data,
           *chartInstance->c5_gpu_thresholdedPrediction_size, c5_d_i, c5_count, *
           chartInstance->c5_gpu_classPred_data,
           *chartInstance->c5_gpu_scorePred_data);
        c5_scorePred_data_dirtyOnGpu = true;
        c5_classPred_data_dirtyOnGpu = true;
      }
    }

    c5_i39 = c5_d_bboxPred_size[0];
    c5_ab_validLaunchParams = mwGetLaunchParameters((real_T)((int64_T)(c5_i39 -
      (int32_T)(c5_count + 1.0)) + 1L), &c5_ab_grid, &c5_ab_block, 1024U, 65535U);
    if (c5_ab_validLaunchParams) {
      c5_eML_blk_kernel_kernel42<<<c5_ab_grid, c5_ab_block>>>((int32_T)(c5_count
        + 1.0), c5_i39, *chartInstance->c5_gpu_idx_data);
      c5_idx_data_dirtyOnGpu = true;
    }

    c5_nrowx = c5_d_bboxPred_size[0];
    if ((c5_d_bboxPred_size[0] - (int32_T)(c5_count + 1.0)) + 1 == 1) {
      c5_nrows = c5_d_bboxPred_size[0] - 1;
      c5_nrows_dirtyOnGpu = false;
      if (c5_b_bboxPred_size_dirtyOnCpu) {
        hipMemcpy(chartInstance->c5_gpu_bboxPred_size, &c5_d_bboxPred_size[0],
                   8UL, hipMemcpyHostToDevice);
      }

      hipMemcpy(chartInstance->c5_gpu_nrows, &c5_nrows, 4UL,
                 hipMemcpyHostToDevice);
      c5_eML_blk_kernel_kernel46<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
        (*chartInstance->c5_gpu_bboxPred_size, chartInstance->c5_gpu_nrows,
         *chartInstance->c5_gpu_idx_data, *chartInstance->c5_b_gpu_bboxPred_data);
    } else {
      c5_b_size[0] = 1;
      c5_b_size[1] = c5_d_bboxPred_size[0];
      c5_bb_validLaunchParams = mwGetLaunchParameters((real_T)((int64_T)
        (c5_d_bboxPred_size[0] - 1) + 1L), &c5_bb_grid, &c5_bb_block, 1024U,
        65535U);
      if (c5_bb_validLaunchParams) {
        if (c5_b_bboxPred_size_dirtyOnCpu) {
          hipMemcpy(chartInstance->c5_gpu_bboxPred_size, &c5_d_bboxPred_size[0],
                     8UL, hipMemcpyHostToDevice);
          c5_b_bboxPred_size_dirtyOnCpu = false;
        }

        c5_eML_blk_kernel_kernel43<<<c5_bb_grid, c5_bb_block>>>
          (*chartInstance->c5_gpu_bboxPred_size, *chartInstance->c5_gpu_b_data);
        c5_b_data_dirtyOnGpu = true;
      }

      c5_i1 = (c5_d_bboxPred_size[0] - (int32_T)(c5_count + 1.0)) + 1;
      for (c5_c_k = 0; c5_c_k < c5_i1; c5_c_k++) {
        if (c5_idx_data_dirtyOnGpu) {
          hipMemcpy(&c5_idx_data[0], chartInstance->c5_gpu_idx_data, 3136UL,
                     hipMemcpyDeviceToHost);
          c5_idx_data_dirtyOnGpu = false;
        }

        if (c5_b_data_dirtyOnGpu) {
          hipMemcpy(&c5_b_data[0], chartInstance->c5_gpu_b_data, 784UL,
                     hipMemcpyDeviceToHost);
          c5_b_data_dirtyOnGpu = false;
        }

        c5_b_data[c5_idx_data[c5_c_k] - 1] = true;
        c5_b_data_dirtyOnCpu = true;
      }

      c5_n = 0;
      c5_n_dirtyOnCpu = true;
      c5_i3 = c5_b_size[1];
      c5_cb_validLaunchParams = mwGetLaunchParameters((real_T)((int64_T)
        (c5_b_size[1] - 1) + 1L), &c5_cb_grid, &c5_cb_block, 1024U, 65535U);
      if (c5_cb_validLaunchParams) {
        if (c5_b_data_dirtyOnCpu) {
          hipMemcpy(chartInstance->c5_gpu_b_data, &c5_b_data[0], 784UL,
                     hipMemcpyHostToDevice);
          c5_b_data_dirtyOnCpu = false;
        }

        hipMemcpy(chartInstance->c5_gpu_b_size, &c5_b_size[0], 8UL,
                   hipMemcpyHostToDevice);
        hipMemcpy(chartInstance->c5_b_gpu_n, &c5_n, 4UL, hipMemcpyHostToDevice);
        c5_eML_blk_kernel_kernel44<<<c5_cb_grid, c5_cb_block>>>
          (*chartInstance->c5_gpu_b_size, *chartInstance->c5_gpu_b_data, c5_i3,
           chartInstance->c5_b_gpu_n);
        c5_n_dirtyOnCpu = false;
      }

      if (c5_b_bboxPred_size_dirtyOnCpu) {
        hipMemcpy(chartInstance->c5_gpu_bboxPred_size, &c5_d_bboxPred_size[0],
                   8UL, hipMemcpyHostToDevice);
      }

      if (c5_n_dirtyOnCpu) {
        hipMemcpy(chartInstance->c5_b_gpu_n, &c5_n, 4UL, hipMemcpyHostToDevice);
      }

      c5_eML_blk_kernel_kernel45<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
        (chartInstance->c5_b_gpu_n, *chartInstance->c5_gpu_bboxPred_size,
         chartInstance->c5_gpu_nrows);
      c5_nrows_dirtyOnGpu = true;
      c5_e_i = 0;
      for (c5_d_k = 0; c5_d_k < c5_nrowx; c5_d_k++) {
        c5_guard1 = false;
        if (c5_d_k + 1 > c5_b_size[1]) {
          c5_guard1 = true;
        } else {
          if (c5_b_data_dirtyOnGpu) {
            hipMemcpy(&c5_b_data[0], chartInstance->c5_gpu_b_data, 784UL,
                       hipMemcpyDeviceToHost);
            c5_b_data_dirtyOnGpu = false;
          }

          if (!c5_b_data[c5_d_k]) {
            c5_guard1 = true;
          }
        }

        if (c5_guard1) {
          for (c5_j = 0; c5_j < 4; c5_j++) {
            if (c5_bboxPred_data_dirtyOnGpu) {
              hipMemcpy(&c5_b_bboxPred_data[0],
                         chartInstance->c5_b_gpu_bboxPred_data, 25088UL,
                         hipMemcpyDeviceToHost);
              c5_bboxPred_data_dirtyOnGpu = false;
            }

            c5_b_bboxPred_data[c5_e_i + c5_d_bboxPred_size[0] * c5_j] =
              c5_b_bboxPred_data[c5_d_k + c5_d_bboxPred_size[0] * c5_j];
            c5_bboxPred_data_dirtyOnCpu = true;
          }

          c5_e_i++;
        }
      }
    }

    if (c5_nrows_dirtyOnGpu) {
      hipMemcpy(&c5_nrows, chartInstance->c5_gpu_nrows, 4UL,
                 hipMemcpyDeviceToHost);
    }

    if (1 > c5_nrows) {
      c5_i4 = -1;
    } else {
      c5_i4 = c5_nrows - 1;
    }

    c5_e_bboxPred_size[0] = c5_i4 + 1;
    c5_e_bboxPred_size[1] = 4;
    c5_c_bboxPred_size_dirtyOnCpu = true;
    c5_db_validLaunchParams = mwGetLaunchParameters((real_T)(((int64_T)c5_i4 +
      1L) * 4L), &c5_db_grid, &c5_db_block, 1024U, 65535U);
    if (c5_db_validLaunchParams) {
      if (c5_bboxPred_data_dirtyOnCpu) {
        hipMemcpy(chartInstance->c5_b_gpu_bboxPred_data, &c5_b_bboxPred_data[0],
                   25088UL, hipMemcpyHostToDevice);
        c5_bboxPred_data_dirtyOnCpu = false;
      }

      hipMemcpy(chartInstance->c5_e_gpu_bboxPred_size, &c5_e_bboxPred_size[0],
                 8UL, hipMemcpyHostToDevice);
      c5_c_bboxPred_size_dirtyOnCpu = false;
      c5_eML_blk_kernel_kernel47<<<c5_db_grid, c5_db_block>>>
        (*chartInstance->c5_b_gpu_bboxPred_data,
         *chartInstance->c5_gpu_bboxPred_size,
         *chartInstance->c5_e_gpu_bboxPred_size, c5_i4,
         *chartInstance->c5_e_gpu_bboxPred_data);
    }

    c5_d_bboxPred_size[0] = c5_e_bboxPred_size[0];
    c5_d_bboxPred_size[1] = 4;
    c5_b_bboxPred_size_dirtyOnCpu = true;
    c5_eb_validLaunchParams = mwGetLaunchParameters((real_T)((int64_T)
      (c5_e_bboxPred_size[0] * 4 - 1) + 1L), &c5_eb_grid, &c5_eb_block, 1024U,
      65535U);
    if (c5_eb_validLaunchParams) {
      if (c5_c_bboxPred_size_dirtyOnCpu) {
        hipMemcpy(chartInstance->c5_e_gpu_bboxPred_size, &c5_e_bboxPred_size[0],
                   8UL, hipMemcpyHostToDevice);
      }

      c5_eML_blk_kernel_kernel48<<<c5_eb_grid, c5_eb_block>>>
        (*chartInstance->c5_e_gpu_bboxPred_data,
         *chartInstance->c5_e_gpu_bboxPred_size,
         *chartInstance->c5_b_gpu_bboxPred_data);
      c5_bboxPred_data_dirtyOnCpu = false;
    }

    c5_i47 = c5_scorePred_size[0];
    c5_fb_validLaunchParams = mwGetLaunchParameters((real_T)((int64_T)(c5_i47 -
      (int32_T)(c5_count + 1.0)) + 1L), &c5_fb_grid, &c5_fb_block, 1024U, 65535U);
    if (c5_fb_validLaunchParams) {
      c5_eML_blk_kernel_kernel49<<<c5_fb_grid, c5_fb_block>>>((int32_T)(c5_count
        + 1.0), c5_i47, *chartInstance->c5_gpu_idx_data);
      c5_idx_data_dirtyOnGpu = true;
    }

    c5_nxin = c5_scorePred_size[0];
    c5_b_size[0] = 1;
    c5_b_size[1] = c5_scorePred_size[0];
    c5_gb_validLaunchParams = mwGetLaunchParameters((real_T)((int64_T)
      (c5_scorePred_size[0] - 1) + 1L), &c5_gb_grid, &c5_gb_block, 1024U, 65535U);
    if (c5_gb_validLaunchParams) {
      hipMemcpy(chartInstance->c5_gpu_scorePred_size, &c5_scorePred_size[0],
                 4UL, hipMemcpyHostToDevice);
      c5_eML_blk_kernel_kernel50<<<c5_gb_grid, c5_gb_block>>>
        (*chartInstance->c5_gpu_scorePred_size, *chartInstance->c5_gpu_b_data);
      c5_b_data_dirtyOnCpu = false;
      c5_b_data_dirtyOnGpu = true;
    }

    c5_i5 = (c5_scorePred_size[0] - (int32_T)(c5_count + 1.0)) + 1;
    for (c5_e_k = 0; c5_e_k < c5_i5; c5_e_k++) {
      if (c5_idx_data_dirtyOnGpu) {
        hipMemcpy(&c5_idx_data[0], chartInstance->c5_gpu_idx_data, 3136UL,
                   hipMemcpyDeviceToHost);
        c5_idx_data_dirtyOnGpu = false;
      }

      if (c5_b_data_dirtyOnGpu) {
        hipMemcpy(&c5_b_data[0], chartInstance->c5_gpu_b_data, 784UL,
                   hipMemcpyDeviceToHost);
        c5_b_data_dirtyOnGpu = false;
      }

      c5_b_data[c5_idx_data[c5_e_k] - 1] = true;
      c5_b_data_dirtyOnCpu = true;
    }

    c5_b_n = 0;
    c5_n_dirtyOnGpu = false;
    c5_i6 = c5_b_size[1];
    c5_hb_validLaunchParams = mwGetLaunchParameters((real_T)((int64_T)
      (c5_b_size[1] - 1) + 1L), &c5_hb_grid, &c5_hb_block, 1024U, 65535U);
    if (c5_hb_validLaunchParams) {
      if (c5_b_data_dirtyOnCpu) {
        hipMemcpy(chartInstance->c5_gpu_b_data, &c5_b_data[0], 784UL,
                   hipMemcpyHostToDevice);
        c5_b_data_dirtyOnCpu = false;
      }

      hipMemcpy(chartInstance->c5_gpu_b_size, &c5_b_size[0], 8UL,
                 hipMemcpyHostToDevice);
      hipMemcpy(chartInstance->c5_gpu_n, &c5_b_n, 4UL, hipMemcpyHostToDevice);
      c5_eML_blk_kernel_kernel51<<<c5_hb_grid, c5_hb_block>>>
        (*chartInstance->c5_gpu_b_size, *chartInstance->c5_gpu_b_data, c5_i6,
         chartInstance->c5_gpu_n);
      c5_n_dirtyOnGpu = true;
    }

    if (c5_n_dirtyOnGpu) {
      hipMemcpy(&c5_b_n, chartInstance->c5_gpu_n, 4UL, hipMemcpyDeviceToHost);
    }

    c5_nxout = c5_scorePred_size[0] - c5_b_n;
    c5_k0 = -1;
    for (c5_f_k = 0; c5_f_k < c5_nxin; c5_f_k++) {
      c5_guard1 = false;
      if (c5_f_k + 1 > c5_b_size[1]) {
        c5_guard1 = true;
      } else {
        if (c5_b_data_dirtyOnGpu) {
          hipMemcpy(&c5_b_data[0], chartInstance->c5_gpu_b_data, 784UL,
                     hipMemcpyDeviceToHost);
          c5_b_data_dirtyOnGpu = false;
        }

        if (!c5_b_data[c5_f_k]) {
          c5_guard1 = true;
        }
      }

      if (c5_guard1) {
        c5_k0++;
        if (c5_scorePred_data_dirtyOnGpu) {
          hipMemcpy(&c5_scorePred_data[0], chartInstance->c5_gpu_scorePred_data,
                     3136UL, hipMemcpyDeviceToHost);
          c5_scorePred_data_dirtyOnGpu = false;
        }

        c5_scorePred_data[c5_k0] = c5_scorePred_data[c5_f_k];
        c5_scorePred_data_dirtyOnCpu = true;
      }
    }

    if (1 > c5_nxout) {
      c5_b_i7 = 0;
    } else {
      c5_b_i7 = c5_nxout;
    }

    c5_scorePred_size[0] = c5_b_i7;
    c5_i51 = c5_classPred_size[0];
    c5_ib_validLaunchParams = mwGetLaunchParameters((real_T)((int64_T)(c5_i51 -
      (int32_T)(c5_count + 1.0)) + 1L), &c5_ib_grid, &c5_ib_block, 1024U, 65535U);
    if (c5_ib_validLaunchParams) {
      c5_eML_blk_kernel_kernel52<<<c5_ib_grid, c5_ib_block>>>((int32_T)(c5_count
        + 1.0), c5_i51, *chartInstance->c5_gpu_idx_data);
      c5_idx_data_dirtyOnGpu = true;
    }

    c5_b_nxin = c5_classPred_size[0];
    c5_b_size[0] = 1;
    c5_b_size[1] = c5_classPred_size[0];
    c5_jb_validLaunchParams = mwGetLaunchParameters((real_T)((int64_T)
      (c5_classPred_size[0] - 1) + 1L), &c5_jb_grid, &c5_jb_block, 1024U, 65535U);
    if (c5_jb_validLaunchParams) {
      hipMemcpy(chartInstance->c5_gpu_classPred_size, &c5_classPred_size[0],
                 4UL, hipMemcpyHostToDevice);
      c5_eML_blk_kernel_kernel53<<<c5_jb_grid, c5_jb_block>>>
        (*chartInstance->c5_gpu_classPred_size, *chartInstance->c5_gpu_b_data);
      c5_b_data_dirtyOnCpu = false;
      c5_b_data_dirtyOnGpu = true;
    }

    c5_i8 = (c5_classPred_size[0] - (int32_T)(c5_count + 1.0)) + 1;
    for (c5_g_k = 0; c5_g_k < c5_i8; c5_g_k++) {
      if (c5_idx_data_dirtyOnGpu) {
        hipMemcpy(&c5_idx_data[0], chartInstance->c5_gpu_idx_data, 3136UL,
                   hipMemcpyDeviceToHost);
        c5_idx_data_dirtyOnGpu = false;
      }

      if (c5_b_data_dirtyOnGpu) {
        hipMemcpy(&c5_b_data[0], chartInstance->c5_gpu_b_data, 784UL,
                   hipMemcpyDeviceToHost);
        c5_b_data_dirtyOnGpu = false;
      }

      c5_b_data[c5_idx_data[c5_g_k] - 1] = true;
      c5_b_data_dirtyOnCpu = true;
    }

    c5_c_n = 0;
    c5_b_n_dirtyOnGpu = false;
    c5_i9 = c5_b_size[1];
    c5_kb_validLaunchParams = mwGetLaunchParameters((real_T)((int64_T)
      (c5_b_size[1] - 1) + 1L), &c5_kb_grid, &c5_kb_block, 1024U, 65535U);
    if (c5_kb_validLaunchParams) {
      if (c5_b_data_dirtyOnCpu) {
        hipMemcpy(chartInstance->c5_gpu_b_data, &c5_b_data[0], 784UL,
                   hipMemcpyHostToDevice);
      }

      hipMemcpy(chartInstance->c5_gpu_b_size, &c5_b_size[0], 8UL,
                 hipMemcpyHostToDevice);
      hipMemcpy(chartInstance->c5_c_gpu_n, &c5_c_n, 4UL, hipMemcpyHostToDevice);
      c5_eML_blk_kernel_kernel54<<<c5_kb_grid, c5_kb_block>>>
        (*chartInstance->c5_gpu_b_size, *chartInstance->c5_gpu_b_data, c5_i9,
         chartInstance->c5_c_gpu_n);
      c5_b_n_dirtyOnGpu = true;
    }

    if (c5_b_n_dirtyOnGpu) {
      hipMemcpy(&c5_c_n, chartInstance->c5_c_gpu_n, 4UL, hipMemcpyDeviceToHost);
    }

    c5_b_k0 = -1;
    for (c5_h_k = 0; c5_h_k < c5_b_nxin; c5_h_k++) {
      c5_guard1 = false;
      if (c5_h_k + 1 > c5_b_size[1]) {
        c5_guard1 = true;
      } else {
        if (c5_b_data_dirtyOnGpu) {
          hipMemcpy(&c5_b_data[0], chartInstance->c5_gpu_b_data, 784UL,
                     hipMemcpyDeviceToHost);
          c5_b_data_dirtyOnGpu = false;
        }

        if (!c5_b_data[c5_h_k]) {
          c5_guard1 = true;
        }
      }

      if (c5_guard1) {
        c5_b_k0++;
        if (c5_classPred_data_dirtyOnGpu) {
          hipMemcpy(&c5_classPred_data[0], chartInstance->c5_gpu_classPred_data,
                     3136UL, hipMemcpyDeviceToHost);
          c5_classPred_data_dirtyOnGpu = false;
        }

        c5_classPred_data[c5_b_k0] = c5_classPred_data[c5_h_k];
        c5_classPred_data_dirtyOnCpu = true;
      }
    }

    if (c5_d_bboxPred_size[0] == 0) {
      c5_bboxesX1Y1X2Y2_size[0] = 0;
      c5_bboxesX1Y1X2Y2_size[1] = 4;
      c5_bboxesX1Y1X2Y2_size_dirtyOnCpu = true;
      c5_b_scores_size[0] = c5_scorePred_size[0];
      c5_lb_validLaunchParams = mwGetLaunchParameters((real_T)((int64_T)
        (c5_scorePred_size[0] - 1) + 1L), &c5_lb_grid, &c5_lb_block, 1024U,
        65535U);
      if (c5_lb_validLaunchParams) {
        if (c5_scorePred_data_dirtyOnCpu) {
          hipMemcpy(chartInstance->c5_gpu_scorePred_data, &c5_scorePred_data[0],
                     3136UL, hipMemcpyHostToDevice);
        }

        hipMemcpy(chartInstance->c5_gpu_scorePred_size, &c5_scorePred_size[0],
                   4UL, hipMemcpyHostToDevice);
        c5_eML_blk_kernel_kernel66<<<c5_lb_grid, c5_lb_block>>>
          (*chartInstance->c5_gpu_scorePred_data,
           *chartInstance->c5_gpu_scorePred_size,
           *chartInstance->c5_b_gpu_scores_data);
      }
    } else {
      c5_x1_size[0] = c5_b_i7;
      c5_x1_size_dirtyOnCpu = true;
      if (c5_b_i7 != 0) {
        c5_sortDim = 2;
        if (c5_b_i7 != 1) {
          c5_sortDim = 1;
        }

        c5_inDims[0] = c5_b_i7;
        c5_inDims[1] = 1;
        c5_nb_validLaunchParams = mwGetLaunchParameters((real_T)((int64_T)
          (c5_scorePred_size[0] - 1) + 1L), &c5_nb_grid, &c5_nb_block, 1024U,
          65535U);
        if (c5_nb_validLaunchParams) {
          if (c5_scorePred_data_dirtyOnCpu) {
            hipMemcpy(chartInstance->c5_gpu_scorePred_data, &c5_scorePred_data
                       [0], 3136UL, hipMemcpyHostToDevice);
          }

          hipMemcpy(chartInstance->c5_gpu_scorePred_size, &c5_scorePred_size[0],
                     4UL, hipMemcpyHostToDevice);
          c5_eML_blk_kernel_kernel55<<<c5_nb_grid, c5_nb_block>>>
            (*chartInstance->c5_gpu_scorePred_data,
             *chartInstance->c5_gpu_scorePred_size,
             *chartInstance->c5_gpu_out_data);
        }

        c5_dv2[0] = (uint32_T)c5_b_i7;
        c5_x1_size[0] = c5_b_i7;
        c5_pb_validLaunchParams = mwGetLaunchParameters((real_T)((int64_T)
          ((int32_T)c5_dv2[0] - 1) + 1L), &c5_pb_grid, &c5_pb_block, 1024U,
          65535U);
        if (c5_pb_validLaunchParams) {
          hipMemcpy(chartInstance->c5_gpu_dv2, &c5_dv2[0], 8UL,
                     hipMemcpyHostToDevice);
          c5_eML_blk_kernel_kernel56<<<c5_pb_grid, c5_pb_block>>>
            (*chartInstance->c5_gpu_dv2, *chartInstance->c5_gpu_x1_data);
        }

        thrustSortImplWithIndex(&(*chartInstance->c5_gpu_out_data)[0],
          &(*chartInstance->c5_gpu_x1_data)[0], 2, &c5_inDims[0], c5_sortDim,
          'd', false);
      }

      c5_bboxesX1Y1X2Y2_size[0] = c5_x1_size[0];
      c5_bboxesX1Y1X2Y2_size[1] = 4;
      c5_bboxesX1Y1X2Y2_size_dirtyOnCpu = true;
      c5_mb_validLaunchParams = mwGetLaunchParameters((real_T)(((int64_T)
        (c5_x1_size[0] - 1) + 1L) * 4L), &c5_mb_grid, &c5_mb_block, 1024U,
        65535U);
      if (c5_mb_validLaunchParams) {
        hipMemcpy(chartInstance->c5_gpu_bboxesX1Y1X2Y2_size,
                   &c5_bboxesX1Y1X2Y2_size[0], 8UL, hipMemcpyHostToDevice);
        c5_bboxesX1Y1X2Y2_size_dirtyOnCpu = false;
        hipMemcpy(chartInstance->c5_gpu_bboxPred_size, &c5_d_bboxPred_size[0],
                   8UL, hipMemcpyHostToDevice);
        c5_b_bboxPred_size_dirtyOnCpu = false;
        if (c5_bboxPred_data_dirtyOnCpu) {
          hipMemcpy(chartInstance->c5_b_gpu_bboxPred_data, &c5_b_bboxPred_data
                     [0], 25088UL, hipMemcpyHostToDevice);
          c5_bboxPred_data_dirtyOnCpu = false;
        }

        hipMemcpy(chartInstance->c5_gpu_x1_size, &c5_x1_size[0], 4UL,
                   hipMemcpyHostToDevice);
        c5_x1_size_dirtyOnCpu = false;
        c5_eML_blk_kernel_kernel57<<<c5_mb_grid, c5_mb_block>>>
          (*chartInstance->c5_b_gpu_bboxPred_data,
           *chartInstance->c5_gpu_bboxPred_size, *chartInstance->c5_gpu_x1_data,
           *chartInstance->c5_gpu_bboxesX1Y1X2Y2_size,
           *chartInstance->c5_gpu_x1_size,
           *chartInstance->c5_gpu_bboxesX1Y1X2Y2_data);
        c5_bboxesX1Y1X2Y2_data_dirtyOnGpu = true;
      }

      c5_ob_validLaunchParams = mwGetLaunchParameters((real_T)((int64_T)
        (c5_x1_size[0] - 1) + 1L), &c5_ob_grid, &c5_ob_block, 1024U, 65535U);
      if (c5_ob_validLaunchParams) {
        if (c5_classPred_data_dirtyOnCpu) {
          hipMemcpy(chartInstance->c5_gpu_classPred_data, &c5_classPred_data[0],
                     3136UL, hipMemcpyHostToDevice);
        }

        if (c5_x1_size_dirtyOnCpu) {
          hipMemcpy(chartInstance->c5_gpu_x1_size, &c5_x1_size[0], 4UL,
                     hipMemcpyHostToDevice);
          c5_x1_size_dirtyOnCpu = false;
        }

        c5_eML_blk_kernel_kernel58<<<c5_ob_grid, c5_ob_block>>>
          (*chartInstance->c5_gpu_classPred_data, *chartInstance->c5_gpu_x1_data,
           *chartInstance->c5_gpu_x1_size, *chartInstance->c5_gpu_y1_data);
        c5_y1_data_dirtyOnGpu = true;
      }

      c5_selectedIndex_size[0] = c5_x1_size[0];
      c5_qb_validLaunchParams = mwGetLaunchParameters((real_T)((int64_T)
        (c5_x1_size[0] - 1) + 1L), &c5_qb_grid, &c5_qb_block, 1024U, 65535U);
      if (c5_qb_validLaunchParams) {
        if (c5_x1_size_dirtyOnCpu) {
          hipMemcpy(chartInstance->c5_gpu_x1_size, &c5_x1_size[0], 4UL,
                     hipMemcpyHostToDevice);
        }

        c5_eML_blk_kernel_kernel59<<<c5_qb_grid, c5_qb_block>>>
          (*chartInstance->c5_gpu_x1_size,
           *chartInstance->c5_gpu_selectedIndex_data);
        c5_selectedIndex_data_dirtyOnGpu = true;
      }

      c5_i61 = c5_x1_size[0] - 1;
      c5_rb_validLaunchParams = mwGetLaunchParameters((real_T)((int64_T)c5_i61 +
        1L), &c5_rb_grid, &c5_rb_block, 1024U, 65535U);
      if (c5_rb_validLaunchParams) {
        if (c5_bboxesX1Y1X2Y2_size_dirtyOnCpu) {
          hipMemcpy(chartInstance->c5_gpu_bboxesX1Y1X2Y2_size,
                     &c5_bboxesX1Y1X2Y2_size[0], 8UL, hipMemcpyHostToDevice);
          c5_bboxesX1Y1X2Y2_size_dirtyOnCpu = false;
        }

        c5_eML_blk_kernel_kernel60<<<c5_rb_grid, c5_rb_block>>>
          (*chartInstance->c5_gpu_bboxesX1Y1X2Y2_data,
           *chartInstance->c5_gpu_bboxesX1Y1X2Y2_size, c5_i61,
           *chartInstance->c5_gpu_area_data);
        c5_area_data_dirtyOnGpu = true;
      }

      c5_i64 = c5_x1_size[0] - 1;
      c5_sb_validLaunchParams = mwGetLaunchParameters((real_T)((int64_T)c5_i64 +
        1L), &c5_sb_grid, &c5_sb_block, 1024U, 65535U);
      if (c5_sb_validLaunchParams) {
        if (c5_bboxesX1Y1X2Y2_size_dirtyOnCpu) {
          hipMemcpy(chartInstance->c5_gpu_bboxesX1Y1X2Y2_size,
                     &c5_bboxesX1Y1X2Y2_size[0], 8UL, hipMemcpyHostToDevice);
          c5_bboxesX1Y1X2Y2_size_dirtyOnCpu = false;
        }

        c5_eML_blk_kernel_kernel61<<<c5_sb_grid, c5_sb_block>>>
          (*chartInstance->c5_gpu_bboxesX1Y1X2Y2_size,
           *chartInstance->c5_gpu_bboxesX1Y1X2Y2_data, c5_i64,
           *chartInstance->c5_gpu_x2_data);
        c5_x2_data_dirtyOnGpu = true;
      }

      c5_i66 = c5_x1_size[0] - 1;
      c5_tb_validLaunchParams = mwGetLaunchParameters((real_T)((int64_T)c5_i66 +
        1L), &c5_tb_grid, &c5_tb_block, 1024U, 65535U);
      if (c5_tb_validLaunchParams) {
        if (c5_bboxesX1Y1X2Y2_size_dirtyOnCpu) {
          hipMemcpy(chartInstance->c5_gpu_bboxesX1Y1X2Y2_size,
                     &c5_bboxesX1Y1X2Y2_size[0], 8UL, hipMemcpyHostToDevice);
        }

        c5_eML_blk_kernel_kernel62<<<c5_tb_grid, c5_tb_block>>>
          (*chartInstance->c5_gpu_bboxesX1Y1X2Y2_data,
           *chartInstance->c5_gpu_bboxesX1Y1X2Y2_size, c5_i66,
           *chartInstance->c5_gpu_y2_data);
        c5_y2_data_dirtyOnGpu = true;
      }

      c5_currentBox = -1;
      c5_i11 = c5_x1_size[0];
      for (c5_f_i = 0; c5_f_i < c5_i11; c5_f_i++) {
        c5_currentBox = c5_f_i;
        if (c5_y1_data_dirtyOnGpu) {
          hipMemcpy(&c5_y1_data[0], chartInstance->c5_gpu_y1_data, 6272UL,
                     hipMemcpyDeviceToHost);
          c5_y1_data_dirtyOnGpu = false;
        }

        if (muDoubleScalarIsNaN(c5_y1_data[c5_f_i])) {
          if (c5_selectedIndex_data_dirtyOnGpu) {
            hipMemcpy(&c5_selectedIndex_data[0],
                       chartInstance->c5_gpu_selectedIndex_data, 784UL,
                       hipMemcpyDeviceToHost);
            c5_selectedIndex_data_dirtyOnGpu = false;
          }

          c5_selectedIndex_data[c5_f_i] = false;
          c5_selectedIndex_data_dirtyOnCpu = true;
        } else {
          if (c5_selectedIndex_data_dirtyOnGpu) {
            hipMemcpy(&c5_selectedIndex_data[0],
                       chartInstance->c5_gpu_selectedIndex_data, 784UL,
                       hipMemcpyDeviceToHost);
            c5_selectedIndex_data_dirtyOnGpu = false;
          }

          if (c5_selectedIndex_data[c5_f_i]) {
            c5_b_i14 = (c5_x1_size[0] - c5_f_i) - 2;
            for (c5_b_j = 0; c5_b_j <= c5_b_i14; c5_b_j++) {
              c5_c_j = (c5_f_i + c5_b_j) + 1;
              if (c5_selectedIndex_data[c5_c_j] && (!(c5_y1_data[c5_c_j] !=
                    c5_y1_data[c5_f_i]))) {
                if (c5_bboxesX1Y1X2Y2_data_dirtyOnGpu) {
                  hipMemcpy(&c5_bboxesX1Y1X2Y2_data[0],
                             chartInstance->c5_gpu_bboxesX1Y1X2Y2_data, 25088UL,
                             hipMemcpyDeviceToHost);
                  c5_bboxesX1Y1X2Y2_data_dirtyOnGpu = false;
                }

                if (c5_x2_data_dirtyOnGpu) {
                  hipMemcpy(&c5_x2_data[0], chartInstance->c5_gpu_x2_data,
                             6272UL, hipMemcpyDeviceToHost);
                  c5_x2_data_dirtyOnGpu = false;
                }

                c5_width = muDoubleScalarMin(c5_x2_data[c5_f_i],
                  c5_x2_data[c5_c_j]) - muDoubleScalarMax
                  (c5_bboxesX1Y1X2Y2_data[c5_f_i], c5_bboxesX1Y1X2Y2_data[c5_c_j]);
                if (!(c5_width <= 0.0)) {
                  if (c5_y2_data_dirtyOnGpu) {
                    hipMemcpy(&c5_y2_data[0], chartInstance->c5_gpu_y2_data,
                               6272UL, hipMemcpyDeviceToHost);
                    c5_y2_data_dirtyOnGpu = false;
                  }

                  c5_height = muDoubleScalarMin(c5_y2_data[c5_f_i],
                    c5_y2_data[c5_c_j]) - muDoubleScalarMax
                    (c5_bboxesX1Y1X2Y2_data[c5_f_i + c5_bboxesX1Y1X2Y2_size[0]],
                     c5_bboxesX1Y1X2Y2_data[c5_c_j + c5_bboxesX1Y1X2Y2_size[0]]);
                  if (!(c5_height <= 0.0)) {
                    c5_areaOfIntersect = c5_width * c5_height;
                    if (c5_area_data_dirtyOnGpu) {
                      hipMemcpy(&c5_area_data[0],
                                 chartInstance->c5_gpu_area_data, 6272UL,
                                 hipMemcpyDeviceToHost);
                      c5_area_data_dirtyOnGpu = false;
                    }

                    if (c5_areaOfIntersect / ((c5_area_data[c5_f_i] +
                          c5_area_data[c5_c_j]) - c5_areaOfIntersect) > 0.5) {
                      c5_selectedIndex_data[c5_c_j] = false;
                      c5_selectedIndex_data_dirtyOnCpu = true;
                    }
                  }
                }
              }
            }
          }
        }
      }

      if (c5_currentBox + 2 > c5_selectedIndex_size[0]) {
        c5_i12 = 0;
        c5_i13 = 0;
      } else {
        c5_i12 = c5_currentBox + 1;
        c5_i13 = c5_selectedIndex_size[0];
      }

      c5_iv[1] = c5_i13 - c5_i12;
      c5_ub_validLaunchParams = mwGetLaunchParameters((real_T)((int64_T)(c5_iv[1]
        - 1) + 1L), &c5_ub_grid, &c5_ub_block, 1024U, 65535U);
      if (c5_ub_validLaunchParams) {
        if (c5_selectedIndex_data_dirtyOnCpu) {
          hipMemcpy(chartInstance->c5_gpu_selectedIndex_data,
                     &c5_selectedIndex_data[0], 784UL, hipMemcpyHostToDevice);
          c5_selectedIndex_data_dirtyOnCpu = false;
        }

        hipMemcpy(chartInstance->c5_gpu_iv, &c5_iv[0], 8UL,
                   hipMemcpyHostToDevice);
        c5_eML_blk_kernel_kernel63<<<c5_ub_grid, c5_ub_block>>>(c5_i12,
          *chartInstance->c5_gpu_iv, *chartInstance->c5_gpu_selectedIndex_data);
      }

      c5_index_size[0] = c5_selectedIndex_size[0];
      c5_vb_validLaunchParams = mwGetLaunchParameters((real_T)((int64_T)
        (c5_selectedIndex_size[0] - 1) + 1L), &c5_vb_grid, &c5_vb_block, 1024U,
        65535U);
      if (c5_vb_validLaunchParams) {
        if (c5_selectedIndex_data_dirtyOnCpu) {
          hipMemcpy(chartInstance->c5_gpu_selectedIndex_data,
                     &c5_selectedIndex_data[0], 784UL, hipMemcpyHostToDevice);
        }

        hipMemcpy(chartInstance->c5_gpu_selectedIndex_size,
                   &c5_selectedIndex_size[0], 4UL, hipMemcpyHostToDevice);
        c5_eML_blk_kernel_kernel64<<<c5_vb_grid, c5_vb_block>>>
          (*chartInstance->c5_gpu_selectedIndex_data,
           *chartInstance->c5_gpu_x1_data,
           *chartInstance->c5_gpu_selectedIndex_size,
           *chartInstance->c5_gpu_index_data);
        c5_index_data_dirtyOnGpu = true;
      }

      c5_end = c5_index_size[0] - 1;
      c5_b_trueCount = 0;
      for (c5_g_i = 0; c5_g_i <= c5_end; c5_g_i++) {
        if (c5_index_data_dirtyOnGpu) {
          hipMemcpy(&c5_index_data[0], chartInstance->c5_gpu_index_data, 784UL,
                     hipMemcpyDeviceToHost);
          c5_index_data_dirtyOnGpu = false;
        }

        if (c5_index_data[c5_g_i]) {
          c5_b_trueCount++;
        }
      }

      c5_iv1_size[0] = c5_b_trueCount;
      c5_b_partialTrueCount = 0;
      for (c5_h_i = 0; c5_h_i <= c5_end; c5_h_i++) {
        if (c5_index_data_dirtyOnGpu) {
          hipMemcpy(&c5_index_data[0], chartInstance->c5_gpu_index_data, 784UL,
                     hipMemcpyDeviceToHost);
          c5_index_data_dirtyOnGpu = false;
        }

        if (c5_index_data[c5_h_i]) {
          c5_iv1_data[c5_b_partialTrueCount] = (int16_T)(c5_h_i + 1);
          c5_iv1_data_dirtyOnCpu = true;
          c5_b_partialTrueCount++;
        }
      }

      c5_bboxesX1Y1X2Y2_size[0] = c5_b_trueCount;
      c5_bboxesX1Y1X2Y2_size[1] = 4;
      c5_bboxesX1Y1X2Y2_size_dirtyOnCpu = true;
      c5_wb_validLaunchParams = mwGetLaunchParameters((real_T)(((int64_T)
        (c5_iv1_size[0] - 1) + 1L) * 4L), &c5_wb_grid, &c5_wb_block, 1024U,
        65535U);
      if (c5_wb_validLaunchParams) {
        hipMemcpy(chartInstance->c5_gpu_bboxesX1Y1X2Y2_size,
                   &c5_bboxesX1Y1X2Y2_size[0], 8UL, hipMemcpyHostToDevice);
        c5_bboxesX1Y1X2Y2_size_dirtyOnCpu = false;
        if (c5_b_bboxPred_size_dirtyOnCpu) {
          hipMemcpy(chartInstance->c5_gpu_bboxPred_size, &c5_d_bboxPred_size[0],
                     8UL, hipMemcpyHostToDevice);
        }

        if (c5_bboxPred_data_dirtyOnCpu) {
          hipMemcpy(chartInstance->c5_b_gpu_bboxPred_data, &c5_b_bboxPred_data
                     [0], 25088UL, hipMemcpyHostToDevice);
        }

        if (c5_iv1_data_dirtyOnCpu) {
          hipMemcpy(chartInstance->c5_gpu_iv1_data, &c5_iv1_data[0], 1568UL,
                     hipMemcpyHostToDevice);
        }

        hipMemcpy(chartInstance->c5_gpu_iv1_size, &c5_iv1_size[0], 4UL,
                   hipMemcpyHostToDevice);
        c5_eML_blk_kernel_kernel65<<<c5_wb_grid, c5_wb_block>>>
          (*chartInstance->c5_b_gpu_bboxPred_data,
           *chartInstance->c5_gpu_bboxPred_size, *chartInstance->c5_gpu_iv1_data,
           *chartInstance->c5_gpu_bboxesX1Y1X2Y2_size,
           *chartInstance->c5_gpu_iv1_size,
           *chartInstance->c5_gpu_bboxesX1Y1X2Y2_data);
      }

      c5_b_end = c5_index_size[0] - 1;
      c5_c_trueCount = 0;
      for (c5_i_i = 0; c5_i_i <= c5_b_end; c5_i_i++) {
        if (c5_index_data_dirtyOnGpu) {
          hipMemcpy(&c5_index_data[0], chartInstance->c5_gpu_index_data, 784UL,
                     hipMemcpyDeviceToHost);
          c5_index_data_dirtyOnGpu = false;
        }

        if (c5_index_data[c5_i_i]) {
          c5_c_trueCount++;
        }
      }

      c5_b_scores_size[0] = c5_c_trueCount;
      c5_c_partialTrueCount = 0;
      for (c5_j_i = 0; c5_j_i <= c5_b_end; c5_j_i++) {
        if (c5_index_data_dirtyOnGpu) {
          hipMemcpy(&c5_index_data[0], chartInstance->c5_gpu_index_data, 784UL,
                     hipMemcpyDeviceToHost);
          c5_index_data_dirtyOnGpu = false;
        }

        if (c5_index_data[c5_j_i]) {
          if (c5_scorePred_data_dirtyOnGpu) {
            hipMemcpy(&c5_scorePred_data[0],
                       chartInstance->c5_gpu_scorePred_data, 3136UL,
                       hipMemcpyDeviceToHost);
            c5_scorePred_data_dirtyOnGpu = false;
          }

          c5_c_scores_data[c5_c_partialTrueCount] = c5_scorePred_data[c5_j_i];
          c5_scores_data_dirtyOnCpu = true;
          c5_c_partialTrueCount++;
        }
      }

      c5_b_nx = c5_index_size[0];
      c5_idx = 0;
      c5_ii = 1;
      c5_exitg1 = false;
      while ((!c5_exitg1) && (c5_ii <= c5_b_nx)) {
        if (c5_index_data_dirtyOnGpu) {
          hipMemcpy(&c5_index_data[0], chartInstance->c5_gpu_index_data, 784UL,
                     hipMemcpyDeviceToHost);
          c5_index_data_dirtyOnGpu = false;
        }

        if (c5_index_data[c5_ii - 1]) {
          c5_idx++;
          if (c5_idx >= c5_b_nx) {
            c5_exitg1 = true;
          } else {
            c5_ii++;
          }
        } else {
          c5_ii++;
        }
      }

      if (c5_index_size[0] != 1) {
        c5_iv1[0] = 1;
        if (1 > c5_idx) {
          c5_iv1[1] = 0;
        } else {
          c5_iv1[1] = c5_idx;
        }

        c5_indexShapeCheck(chartInstance, c5_index_size[0], c5_iv1);
      }
    }
  } else {
    c5_bboxesX1Y1X2Y2_size[0] = 0;
    c5_bboxesX1Y1X2Y2_size[1] = 4;
    c5_bboxesX1Y1X2Y2_size_dirtyOnCpu = true;
    c5_b_scores_size[0] = 0;
  }

  c5_bboxes_size[0] = c5_bboxesX1Y1X2Y2_size[0];
  c5_bboxes_size[1] = 4;
  c5_c_validLaunchParams = mwGetLaunchParameters((real_T)((int64_T)
    (c5_bboxesX1Y1X2Y2_size[0] * 4 - 1) + 1L), &c5_c_grid, &c5_c_block, 1024U,
    65535U);
  if (c5_c_validLaunchParams) {
    if (c5_bboxesX1Y1X2Y2_size_dirtyOnCpu) {
      hipMemcpy(chartInstance->c5_gpu_bboxesX1Y1X2Y2_size,
                 &c5_bboxesX1Y1X2Y2_size[0], 8UL, hipMemcpyHostToDevice);
    }

    c5_eML_blk_kernel_kernel67<<<c5_c_grid, c5_c_block>>>
      (*chartInstance->c5_gpu_bboxesX1Y1X2Y2_data,
       *chartInstance->c5_gpu_bboxesX1Y1X2Y2_size,
       chartInstance->c5_gpu_bboxes_data);
    c5_bboxes_data_dirtyOnGpu = true;
  }

  c5_scores_size[0] = c5_b_scores_size[0];
  c5_scores_size[1] = 1;
  c5_e_validLaunchParams = mwGetLaunchParameters((real_T)((int64_T)
    (c5_b_scores_size[0] - 1) + 1L), &c5_e_grid, &c5_e_block, 1024U, 65535U);
  if (c5_e_validLaunchParams) {
    if (c5_scores_data_dirtyOnCpu) {
      hipMemcpy(chartInstance->c5_b_gpu_scores_data, &c5_c_scores_data[0],
                 3136UL, hipMemcpyHostToDevice);
    }

    hipMemcpy(chartInstance->c5_gpu_scores_size, &c5_b_scores_size[0], 4UL,
               hipMemcpyHostToDevice);
    c5_eML_blk_kernel_kernel68<<<c5_e_grid, c5_e_block>>>
      (*chartInstance->c5_b_gpu_scores_data, *chartInstance->c5_gpu_scores_size,
       chartInstance->c5_gpu_scores_data);
    c5_scores_data_dirtyOnGpu = true;
  }

  if (c5_bboxes_data_dirtyOnGpu) {
    hipMemcpy(&c5_b_bboxes_data[0], chartInstance->c5_gpu_bboxes_data,
               (uint32_T)(c5_bboxes_size[0] * 4) * sizeof(real_T),
               hipMemcpyDeviceToHost);
  }

  if (c5_scores_data_dirtyOnGpu) {
    hipMemcpy(&c5_b_scores_data[0], chartInstance->c5_gpu_scores_data,
               (uint32_T)c5_scores_size[0] * sizeof(real32_T),
               hipMemcpyDeviceToHost);
  }
}

static void c5_DeepLearningNetwork_setup(SFc5_LaneDetectionInstanceStruct
  *chartInstance, c5_yolov2ResNet50VehicleExample0_LaneDetection0 *c5_obj)
{
  c5_obj->setup();
}

static real32_T c5_callFcn(SFc5_LaneDetectionInstanceStruct *chartInstance,
  real32_T c5_input1, real32_T c5_input2)
{
  return muSingleScalarMin(c5_input1, c5_input2);
}

static real32_T c5_b_callFcn(SFc5_LaneDetectionInstanceStruct *chartInstance,
  real32_T c5_input1, real32_T c5_input2)
{
  return muSingleScalarMax(c5_input1, c5_input2);
}

static void c5_DeepLearningNetwork_activations(SFc5_LaneDetectionInstanceStruct *
  chartInstance, c5_yolov2ResNet50VehicleExample0_LaneDetection0 *c5_obj,
  real32_T c5_varargin_1[150528], real32_T c5_b_out[4704])
{
  c5_cell_wrap_18 (*c5_gpu_miniBatchT)[1];
  c5_cell_wrap_18 *c5_gpu_r;
  real32_T (*c5_gpu_varargin_1)[150528];
  real32_T (*c5_c_gpu_out)[4704];
  real32_T (*c5_gpu_outMiniBatch)[4704];
  hipMalloc(&c5_c_gpu_out, 18816UL);
  hipMalloc(&c5_gpu_outMiniBatch, 18816UL);
  hipMalloc(&c5_gpu_miniBatchT, 602112UL);
  hipMalloc(&c5_gpu_r, 602112UL);
  hipMalloc(&c5_gpu_varargin_1, 602112UL);
  hipMemcpy(c5_gpu_varargin_1, &c5_varargin_1[0], 602112UL,
             hipMemcpyHostToDevice);
  c5_DeepLearningNetwork_activations_kernel69<<<dim3(294U, 1U, 1U), dim3(512U,
    1U, 1U)>>>(*c5_gpu_varargin_1, c5_gpu_r);
  c5_DeepLearningNetwork_activations_kernel70<<<dim3(294U, 1U, 1U), dim3(512U,
    1U, 1U)>>>(c5_gpu_r, *c5_gpu_miniBatchT);
  hipMemcpy(c5_obj->getInputDataPointer(0), (*c5_gpu_miniBatchT)[0].f1,
             c5_obj->layers[0]->getOutputTensor(0)->getNumElements() * sizeof
             (real32_T), hipMemcpyDeviceToDevice);
  c5_obj->activations(56);
  hipMemcpy(*c5_gpu_outMiniBatch, c5_obj->getLayerOutput(56, 0), c5_obj->
             layers[56]->getOutputTensor(0)->getNumElements() * sizeof(real32_T),
             hipMemcpyDeviceToDevice);
  c5_DeepLearningNetwork_activations_kernel71<<<dim3(10U, 1U, 1U), dim3(512U, 1U,
    1U)>>>(*c5_gpu_outMiniBatch, *c5_c_gpu_out);
  hipMemcpy(&c5_b_out[0], c5_c_gpu_out, 18816UL, hipMemcpyDeviceToHost);
  hipFree(*c5_gpu_varargin_1);
  hipFree(c5_gpu_r);
  hipFree(*c5_gpu_miniBatchT);
  hipFree(*c5_gpu_outMiniBatch);
  hipFree(*c5_c_gpu_out);
}

static void c5_indexShapeCheck(SFc5_LaneDetectionInstanceStruct *chartInstance,
  int32_T c5_matrixSize, int32_T c5_indexSize[2])
{
}

static void c5_emlrt_marshallIn(SFc5_LaneDetectionInstanceStruct *chartInstance,
  const mxArray *c5_bboxes, const char_T *c5_identifier, real_T c5_y_data[],
  int32_T c5_y_size[2])
{
  emlrtMsgIdentifier c5_thisId;
  c5_thisId.fIdentifier = const_cast<const char_T *>(c5_identifier);
  c5_thisId.fParent = NULL;
  c5_thisId.bParentIsCell = false;
  c5_b_emlrt_marshallIn(chartInstance, sf_mex_dup(c5_bboxes), &c5_thisId,
                        c5_y_data, c5_y_size);
  sf_mex_destroy(&c5_bboxes);
}

static void c5_b_emlrt_marshallIn(SFc5_LaneDetectionInstanceStruct
  *chartInstance, const mxArray *c5_u, const emlrtMsgIdentifier *c5_parentId,
  real_T c5_y_data[], int32_T c5_y_size[2])
{
  real_T c5_dv_data[80];
  int32_T c5_dv_size[2];
  int32_T c5_i;
  int32_T c5_i1;
  int32_T c5_i2;
  uint32_T c5_uv[2];
  boolean_T c5_bv[2];
  for (c5_i = 0; c5_i < 2; c5_i++) {
    c5_uv[c5_i] = (uint32_T)(-16 * c5_i) + 20U;
  }

  c5_dv_size[0] = sf_mex_get_dimension(c5_u, 0);
  c5_dv_size[1] = sf_mex_get_dimension(c5_u, 1);
  for (c5_i1 = 0; c5_i1 < 2; c5_i1++) {
    c5_bv[c5_i1] = true;
  }

  sf_mex_import_vs(c5_parentId, sf_mex_dup(c5_u), &c5_dv_data, 1, 0, 0U, 1, 0U,
                   2, c5_bv, c5_uv, c5_dv_size);
  c5_y_size[0] = c5_dv_size[0];
  c5_y_size[1] = c5_dv_size[1];
  for (c5_i2 = 0; c5_i2 < c5_dv_size[0] * c5_dv_size[1]; c5_i2++) {
    c5_y_data[c5_i2] = c5_dv_data[c5_i2];
  }

  sf_mex_destroy(&c5_u);
}

static void c5_c_emlrt_marshallIn(SFc5_LaneDetectionInstanceStruct
  *chartInstance, const mxArray *c5_scores, const char_T *c5_identifier,
  real32_T c5_y_data[], int32_T c5_y_size[2])
{
  emlrtMsgIdentifier c5_thisId;
  c5_thisId.fIdentifier = const_cast<const char_T *>(c5_identifier);
  c5_thisId.fParent = NULL;
  c5_thisId.bParentIsCell = false;
  c5_d_emlrt_marshallIn(chartInstance, sf_mex_dup(c5_scores), &c5_thisId,
                        c5_y_data, c5_y_size);
  sf_mex_destroy(&c5_scores);
}

static void c5_d_emlrt_marshallIn(SFc5_LaneDetectionInstanceStruct
  *chartInstance, const mxArray *c5_u, const emlrtMsgIdentifier *c5_parentId,
  real32_T c5_y_data[], int32_T c5_y_size[2])
{
  static boolean_T c5_b_bv[2] = { true, false };

  int32_T c5_fv_size[2];
  int32_T c5_b_i;
  int32_T c5_i;
  int32_T c5_i1;
  real32_T c5_fv_data[20];
  uint32_T c5_uv[2];
  boolean_T c5_bv[2];
  for (c5_i = 0; c5_i < 2; c5_i++) {
    c5_uv[c5_i] = (uint32_T)(-19 * c5_i) + 20U;
  }

  c5_fv_size[0] = sf_mex_get_dimension(c5_u, 0);
  c5_fv_size[1] = sf_mex_get_dimension(c5_u, 1);
  for (c5_b_i = 0; c5_b_i < 2; c5_b_i++) {
    c5_bv[c5_b_i] = c5_b_bv[c5_b_i];
  }

  sf_mex_import_vs(c5_parentId, sf_mex_dup(c5_u), &c5_fv_data, 0, 1, 0U, 1, 0U,
                   2, c5_bv, c5_uv, c5_fv_size);
  c5_y_size[0] = c5_fv_size[0];
  c5_y_size[1] = 1;
  for (c5_i1 = 0; c5_i1 < c5_fv_size[0] * c5_fv_size[1]; c5_i1++) {
    c5_y_data[c5_i1] = c5_fv_data[c5_i1];
  }

  sf_mex_destroy(&c5_u);
}

static uint8_T c5_e_emlrt_marshallIn(SFc5_LaneDetectionInstanceStruct
  *chartInstance, const mxArray *c5_b_is_active_c5_LaneDetection, const char_T
  *c5_identifier)
{
  emlrtMsgIdentifier c5_thisId;
  uint8_T c5_y;
  c5_thisId.fIdentifier = const_cast<const char_T *>(c5_identifier);
  c5_thisId.fParent = NULL;
  c5_thisId.bParentIsCell = false;
  c5_y = c5_f_emlrt_marshallIn(chartInstance, sf_mex_dup
    (c5_b_is_active_c5_LaneDetection), &c5_thisId);
  sf_mex_destroy(&c5_b_is_active_c5_LaneDetection);
  return c5_y;
}

static uint8_T c5_f_emlrt_marshallIn(SFc5_LaneDetectionInstanceStruct
  *chartInstance, const mxArray *c5_u, const emlrtMsgIdentifier *c5_parentId)
{
  uint8_T c5_b_u;
  uint8_T c5_y;
  sf_mex_import(c5_parentId, sf_mex_dup(c5_u), &c5_b_u, 1, 3, 0U, 0, 0U, 0);
  c5_y = c5_b_u;
  sf_mex_destroy(&c5_u);
  return c5_y;
}

static void c5_emxConvertDynamicMatrixFromEmx_(SFc5_LaneDetectionInstanceStruct *
  chartInstance, c5_emxArray_real_T_20x4 *c5_emx, real_T c5_data[80], int32_T
  c5_size[2])
{
  memcpy((void *)c5_data, &c5_emx->data, sizeof(real_T) * (uint32_T)
         (c5_emx->size[0] * c5_emx->size[1]));
  memcpy((void *)c5_size, &c5_emx->size, sizeof(int32_T) << 1);
}

static void c5_b_emxConvertDynamicMatrixFromEmx_
  (SFc5_LaneDetectionInstanceStruct *chartInstance, c5_emxArray_real32_T_20x1
   *c5_emx, real32_T c5_data[20], int32_T c5_size[2])
{
  memcpy((void *)c5_data, &c5_emx->data, sizeof(real32_T) * (uint32_T)
         (c5_emx->size[0] * c5_emx->size[1]));
  memcpy((void *)c5_size, &c5_emx->size, sizeof(int32_T) << 1);
}

static __global__ __launch_bounds__(1024, 1) void c5_coder_reduce0(const
  real32_T c5_inputVar[150528], real32_T *c5_outputVar)
{
  real32_T c5_tmpRed0;
  real32_T c5_tmpRed1;
  uint32_T c5_blockStride;
  uint32_T c5_idx;
  uint32_T c5_m;
  uint32_T c5_mask;
  uint32_T c5_numActiveThreads;
  uint32_T c5_numActiveWarps;
  uint32_T c5_thBlkId;
  uint32_T c5_threadId;
  uint32_T c5_threadStride;
  c5_threadStride = (uint32_T)mwGetTotalThreadsLaunched();
  c5_threadId = (uint32_T)mwGetGlobalThreadIndex();
  c5_thBlkId = (uint32_T)mwGetThreadIndexWithinBlock();
  c5_blockStride = (uint32_T)mwGetThreadsPerBlock();
  c5_tmpRed0 = 0.0F;
  c5_tmpRed1 = 0.0F;
  c5_numActiveThreads = c5_blockStride;
  if (mwIsLastBlock()) {
    c5_m = 150527U % c5_blockStride;
    if (c5_m > 0U) {
      c5_numActiveThreads = c5_m;
    }
  }

  c5_numActiveWarps = ((c5_numActiveThreads + warpSize) - 1U) / warpSize;
  if (c5_threadId <= 150526U) {
    c5_tmpRed0 = c5_inputVar[c5_threadId];
    c5_tmpRed1 = c5_tmpRed0;
  }

  c5_mask = __ballot_sync(MAX_uint32_T, c5_threadId <= 150526U);
  for (c5_idx = c5_threadId + c5_threadStride; c5_idx <= 150526U; c5_idx +=
       c5_threadStride) {
    c5_tmpRed0 = c5_b_callFcn_device(c5_tmpRed0, c5_inputVar[c5_idx]);
    c5_tmpRed1 = c5_callFcn_device(c5_tmpRed1, c5_inputVar[c5_idx]);
  }

  c5_tmpRed0 = c5_workGroupReduction(c5_tmpRed0, c5_mask, c5_numActiveWarps);
  c5_tmpRed1 = c5_b_workGroupReduction(c5_tmpRed1, c5_mask, c5_numActiveWarps);
  if (c5_thBlkId == 0U) {
    c5_atomicOpreal32_T(&c5_outputVar[0], c5_tmpRed0);
    c5_b_atomicOpreal32_T(&c5_outputVar[1], c5_tmpRed1);
  }
}

static __device__ real32_T c5_threadGroupReduction(real32_T c5_val, uint32_T
  c5_lane, uint32_T c5_mask)
{
  real32_T c5_other;
  uint32_T c5_activeSize;
  uint32_T c5_offset;
  c5_activeSize = __popc(c5_mask);
  c5_offset = (c5_activeSize + 1U) / 2U;
  while (c5_activeSize > 1U) {
    c5_other = c5_shflDown1(c5_val, c5_offset, c5_mask);
    if (c5_lane + c5_offset < c5_activeSize) {
      c5_val = c5_b_callFcn_device(c5_val, c5_other);
    }

    c5_activeSize = c5_offset;
    c5_offset = (c5_offset + 1U) / 2U;
  }

  return c5_val;
}

static __device__ real32_T c5_shflDown1(real32_T c5_in1, uint32_T c5_offset,
  uint32_T c5_mask)
{
  int32_T *c5_tmp;
  c5_tmp = (int32_T *)&c5_in1;
  *c5_tmp = __shfl_down_sync(c5_mask, *c5_tmp, c5_offset);
  return *(real32_T *)c5_tmp;
}

static __device__ real32_T c5_workGroupReduction(real32_T c5_val, uint32_T
  c5_mask, uint32_T c5_numActiveWarps)
{
  __shared__ real32_T c5_shared[32];
  uint32_T c5_lane;
  uint32_T c5_thBlkId;
  uint32_T c5_widx;
  c5_thBlkId = (uint32_T)mwGetThreadIndexWithinBlock();
  c5_lane = c5_thBlkId % warpSize;
  c5_widx = c5_thBlkId / warpSize;
  c5_val = c5_threadGroupReduction(c5_val, c5_lane, c5_mask);
  if (c5_lane == 0U) {
    c5_shared[c5_widx] = c5_val;
  }

  __syncthreads();
  c5_mask = __ballot_sync(MAX_uint32_T, c5_lane < c5_numActiveWarps);
  c5_val = c5_shared[c5_lane];
  if (c5_widx == 0U) {
    c5_val = c5_threadGroupReduction(c5_val, c5_lane, c5_mask);
  }

  return c5_val;
}

static __device__ real32_T c5_b_threadGroupReduction(real32_T c5_val, uint32_T
  c5_lane, uint32_T c5_mask)
{
  real32_T c5_other;
  uint32_T c5_activeSize;
  uint32_T c5_offset;
  c5_activeSize = __popc(c5_mask);
  c5_offset = (c5_activeSize + 1U) / 2U;
  while (c5_activeSize > 1U) {
    c5_other = c5_shflDown1(c5_val, c5_offset, c5_mask);
    if (c5_lane + c5_offset < c5_activeSize) {
      c5_val = c5_callFcn_device(c5_val, c5_other);
    }

    c5_activeSize = c5_offset;
    c5_offset = (c5_offset + 1U) / 2U;
  }

  return c5_val;
}

static __device__ real32_T c5_b_workGroupReduction(real32_T c5_val, uint32_T
  c5_mask, uint32_T c5_numActiveWarps)
{
  __shared__ real32_T c5_shared[32];
  uint32_T c5_lane;
  uint32_T c5_thBlkId;
  uint32_T c5_widx;
  c5_thBlkId = (uint32_T)mwGetThreadIndexWithinBlock();
  c5_lane = c5_thBlkId % warpSize;
  c5_widx = c5_thBlkId / warpSize;
  c5_val = c5_b_threadGroupReduction(c5_val, c5_lane, c5_mask);
  if (c5_lane == 0U) {
    c5_shared[c5_widx] = c5_val;
  }

  __syncthreads();
  c5_mask = __ballot_sync(MAX_uint32_T, c5_lane < c5_numActiveWarps);
  c5_val = c5_shared[c5_lane];
  if (c5_widx == 0U) {
    c5_val = c5_b_threadGroupReduction(c5_val, c5_lane, c5_mask);
  }

  return c5_val;
}

static __device__ real32_T c5_atomicOpreal32_T(real32_T *c5_address, real32_T
  c5_value)
{
  uint32_T c5_assumed;
  uint32_T c5_old;
  uint32_T *c5_address_as_up;
  c5_address_as_up = (uint32_T *)c5_address;
  c5_old = *c5_address_as_up;
  do {
    c5_assumed = c5_old;
    c5_old = atomicCAS(c5_address_as_up, c5_old, __float_as_uint
                       (c5_b_callFcn_device(c5_value, __uint_as_float(c5_old))));
  } while (c5_assumed != c5_old);

  return __uint_as_float(c5_old);
}

static __device__ real32_T c5_b_atomicOpreal32_T(real32_T *c5_address, real32_T
  c5_value)
{
  uint32_T c5_assumed;
  uint32_T c5_old;
  uint32_T *c5_address_as_up;
  c5_address_as_up = (uint32_T *)c5_address;
  c5_old = *c5_address_as_up;
  do {
    c5_assumed = c5_old;
    c5_old = atomicCAS(c5_address_as_up, c5_old, __float_as_uint
                       (c5_callFcn_device(c5_value, __uint_as_float(c5_old))));
  } while (c5_assumed != c5_old);

  return __uint_as_float(c5_old);
}

static __global__ __launch_bounds__(512, 1) void c5_eML_blk_kernel_kernel1
  (int16_T c5_aux1[960])
{
  int32_T c5_i;
  c5_i = (int32_T)mwGetGlobalThreadIndex();
  if (c5_i < 960) {
    if (c5_i + 1 <= 480) {
      c5_aux1[c5_i] = (int16_T)(c5_i + 1);
    } else {
      c5_aux1[c5_i] = (int16_T)(960 - c5_i);
    }
  }
}

static __global__ __launch_bounds__(512, 1) void c5_eML_blk_kernel_kernel2
  (int16_T c5_aux2[1280])
{
  int32_T c5_i;
  c5_i = (int32_T)mwGetGlobalThreadIndex();
  if (c5_i < 1280) {
    if (c5_i + 1 <= 640) {
      c5_aux2[c5_i] = (int16_T)(c5_i + 1);
    } else {
      c5_aux2[c5_i] = (int16_T)(1280 - c5_i);
    }
  }
}

static __global__ __launch_bounds__(512, 1) void c5_eML_blk_kernel_kernel3(const
  int16_T c5_aux1[960], real_T c5_rowWeights[2016], int16_T c5_ipRowIndices[2016])
{
  real_T c5_absx;
  real_T c5_absx2;
  real_T c5_absx3;
  real_T c5_ipRowIdx;
  uint64_T c5_threadId;
  int32_T c5_k;
  int32_T c5_l;
  int32_T c5_oldIdx;
  int32_T c5_rowIdx;
  int32_T c5_rowIndices;
  c5_threadId = mwGetGlobalThreadIndex();
  c5_k = (int32_T)(c5_threadId % 9UL);
  c5_rowIdx = (int32_T)((c5_threadId - (uint64_T)c5_k) / 9UL);
  if ((c5_rowIdx < 224) && (c5_k < 9)) {
    c5_ipRowIdx = ((real_T)c5_rowIdx + 1.0) / 0.46666666666666667 +
      -0.5714285714285714;
    c5_rowIndices = (int32_T)floor(c5_ipRowIdx - 4.2857142857142856);
    c5_absx = fabs(0.46666666666666667 * (c5_ipRowIdx - ((real_T)(c5_rowIndices
      + c5_k) + 1.0)));
    c5_absx2 = c5_absx * c5_absx;
    c5_absx3 = pow(c5_absx, 3.0);
    c5_oldIdx = (c5_rowIndices + c5_k) + 1;
    if (c5_oldIdx - 1 == 0) {
      c5_l = 0;
    } else {
      c5_l = (int32_T)fmod((real_T)c5_oldIdx - 1.0, 960.0);
      if ((c5_l != 0) && (c5_oldIdx - 1 < 0)) {
        c5_l += 960;
      }
    }

    c5_ipRowIndices[c5_rowIdx + 224 * c5_k] = c5_aux1[c5_l];
    c5_rowWeights[c5_rowIdx + 224 * c5_k] = 0.46666666666666667 * (((1.5 *
      c5_absx3 - 2.5 * c5_absx2) + 1.0) * (real_T)(c5_absx <= 1.0) + (((-0.5 *
      c5_absx3 + 2.5 * c5_absx2) - 4.0 * c5_absx) + 2.0) * (real_T)((1.0 <
      c5_absx) && (c5_absx <= 2.0)));
  }
}

static __global__ __launch_bounds__(512, 1) void c5_eML_blk_kernel_kernel4(const
  int16_T c5_aux2[1280], real_T c5_colWeights[2688], int16_T c5_ipColIndices
  [2688])
{
  real_T c5_absx;
  real_T c5_absx2;
  real_T c5_absx3;
  real_T c5_ipColIdx;
  uint64_T c5_threadId;
  int32_T c5_colIdx;
  int32_T c5_colIndices;
  int32_T c5_k;
  int32_T c5_l;
  int32_T c5_oldIdx;
  c5_threadId = mwGetGlobalThreadIndex();
  c5_k = (int32_T)(c5_threadId % 12UL);
  c5_colIdx = (int32_T)((c5_threadId - (uint64_T)c5_k) / 12UL);
  if ((c5_colIdx < 224) && (c5_k < 12)) {
    c5_ipColIdx = ((real_T)c5_colIdx + 1.0) / 0.35 + -0.9285714285714286;
    c5_colIndices = (int32_T)floor(c5_ipColIdx - 5.7142857142857144);
    c5_absx = fabs(0.35 * (c5_ipColIdx - ((real_T)(c5_colIndices + c5_k) + 1.0)));
    c5_absx2 = c5_absx * c5_absx;
    c5_absx3 = pow(c5_absx, 3.0);
    c5_oldIdx = (c5_colIndices + c5_k) + 1;
    if (c5_oldIdx - 1 == 0) {
      c5_l = 0;
    } else {
      c5_l = (int32_T)fmod((real_T)c5_oldIdx - 1.0, 1280.0);
      if ((c5_l != 0) && (c5_oldIdx - 1 < 0)) {
        c5_l += 1280;
      }
    }

    c5_ipColIndices[c5_colIdx + 224 * c5_k] = c5_aux2[c5_l];
    c5_colWeights[c5_colIdx + 224 * c5_k] = 0.35 * (((1.5 * c5_absx3 - 2.5 *
      c5_absx2) + 1.0) * (real_T)(c5_absx <= 1.0) + (((-0.5 * c5_absx3 + 2.5 *
      c5_absx2) - 4.0 * c5_absx) + 2.0) * (real_T)((1.0 < c5_absx) && (c5_absx <=
      2.0)));
  }
}

static __global__ __launch_bounds__(224, 1) void c5_eML_blk_kernel_kernel5(const
  real_T c5_rowWeights[2016], real_T c5_rowWeightsTotal[224])
{
  int32_T c5_j;
  c5_j = (int32_T)mwGetGlobalThreadIndex();
  if (c5_j < 224) {
    c5_rowWeightsTotal[c5_j] = c5_rowWeights[c5_j];
  }
}

static __global__ __launch_bounds__(224, 1) void c5_eML_blk_kernel_kernel6(const
  real_T c5_rowWeights[2016], const int32_T c5_xoffset, real_T
  c5_rowWeightsTotal[224])
{
  int32_T c5_j;
  c5_j = (int32_T)mwGetGlobalThreadIndex();
  if (c5_j < 224) {
    c5_rowWeightsTotal[c5_j] += c5_rowWeights[c5_xoffset + c5_j];
  }
}

static __global__ __launch_bounds__(224, 1) void c5_eML_blk_kernel_kernel7(const
  real_T c5_colWeights[2688], real_T c5_colWeightsTotal[224])
{
  int32_T c5_j;
  c5_j = (int32_T)mwGetGlobalThreadIndex();
  if (c5_j < 224) {
    c5_colWeightsTotal[c5_j] = c5_colWeights[c5_j];
  }
}

static __global__ __launch_bounds__(224, 1) void c5_eML_blk_kernel_kernel8(const
  real_T c5_colWeights[2688], const int32_T c5_xoffset, real_T
  c5_colWeightsTotal[224])
{
  int32_T c5_j;
  c5_j = (int32_T)mwGetGlobalThreadIndex();
  if (c5_j < 224) {
    c5_colWeightsTotal[c5_j] += c5_colWeights[c5_xoffset + c5_j];
  }
}

static __global__ __launch_bounds__(512, 1) void c5_eML_blk_kernel_kernel9(const
  real_T c5_colWeightsTotal[224], const real_T c5_colWeights[2688], const
  real32_T c5_b_In[921600], const int16_T c5_ipColIndices[2688], real32_T
  c5_partialResize[322560])
{
  real_T c5_sumVal;
  uint64_T c5_threadId;
  uint64_T c5_tmpIndex;
  int32_T c5_colIdx;
  int32_T c5_dimIdx;
  int32_T c5_l;
  int32_T c5_rowIdx;
  c5_threadId = mwGetGlobalThreadIndex();
  c5_dimIdx = (int32_T)(c5_threadId % 3UL);
  c5_tmpIndex = (c5_threadId - (uint64_T)c5_dimIdx) / 3UL;
  c5_colIdx = (int32_T)(c5_tmpIndex % 224UL);
  c5_tmpIndex = (c5_tmpIndex - (uint64_T)c5_colIdx) / 224UL;
  c5_rowIdx = (int32_T)c5_tmpIndex;
  if ((c5_rowIdx < 480) && (c5_colIdx < 224) && (c5_dimIdx < 3)) {
    c5_sumVal = 0.0;
    for (c5_l = 0; c5_l < 12; c5_l++) {
      c5_sumVal += (real_T)c5_b_In[(c5_rowIdx + 480 * ((int32_T)
        c5_ipColIndices[c5_colIdx + 224 * c5_l] - 1)) + 307200 * c5_dimIdx] *
        (c5_colWeights[c5_colIdx + 224 * c5_l] / c5_colWeightsTotal[c5_colIdx]);
    }

    c5_partialResize[(c5_rowIdx + 480 * c5_colIdx) + 107520 * c5_dimIdx] =
      (real32_T)c5_sumVal;
  }
}

static __global__ __launch_bounds__(512, 1) void c5_eML_blk_kernel_kernel10(
  const real_T c5_rowWeightsTotal[224], const real_T c5_rowWeights[2016], const
  real32_T c5_partialResize[322560], const int16_T c5_ipRowIndices[2016],
  real32_T c5_b_out[150528])
{
  real_T c5_sumVal;
  uint64_T c5_threadId;
  uint64_T c5_tmpIndex;
  int32_T c5_colIdx;
  int32_T c5_dimIdx;
  int32_T c5_l;
  int32_T c5_rowIdx;
  c5_threadId = mwGetGlobalThreadIndex();
  c5_dimIdx = (int32_T)(c5_threadId % 3UL);
  c5_tmpIndex = (c5_threadId - (uint64_T)c5_dimIdx) / 3UL;
  c5_rowIdx = (int32_T)(c5_tmpIndex % 224UL);
  c5_tmpIndex = (c5_tmpIndex - (uint64_T)c5_rowIdx) / 224UL;
  c5_colIdx = (int32_T)c5_tmpIndex;
  if ((c5_colIdx < 224) && (c5_rowIdx < 224) && (c5_dimIdx < 3)) {
    c5_sumVal = 0.0;
    for (c5_l = 0; c5_l < 9; c5_l++) {
      c5_sumVal += (real_T)c5_partialResize[(((int32_T)c5_ipRowIndices[c5_rowIdx
        + 224 * c5_l] + 480 * c5_colIdx) + 107520 * c5_dimIdx) - 1] *
        (c5_rowWeights[c5_rowIdx + 224 * c5_l] / c5_rowWeightsTotal[c5_rowIdx]);
    }

    c5_b_out[(c5_rowIdx + 224 * c5_colIdx) + 50176 * c5_dimIdx] = (real32_T)
      c5_sumVal;
  }
}

static __global__ __launch_bounds__(32, 1) void c5_eML_blk_kernel_kernel11
  (real32_T c5_b_out[150528], real32_T c5_outVal[2])
{
  int32_T c5_indV;
  c5_indV = (int32_T)mwGetGlobalThreadIndex();
  if (c5_indV < 2) {
    c5_outVal[c5_indV] = c5_b_out[150527];
    c5_outVal[c5_indV] = c5_b_out[150527];
  }
}

static __global__ __launch_bounds__(512, 1) void c5_eML_blk_kernel_kernel12(
  const real32_T c5_y, const real32_T c5_outVal, real32_T c5_b_out[150528],
  real32_T c5_c_out[150528])
{
  int32_T c5_i2;
  c5_i2 = (int32_T)mwGetGlobalThreadIndex();
  if (c5_i2 < 150528) {
    c5_c_out[c5_i2] = (c5_b_out[c5_i2] - c5_outVal) / c5_y;
  }
}

static __global__ __launch_bounds__(32, 1) void c5_eML_blk_kernel_kernel13(const
  int8_T c5_dv[8], real_T c5_anchors[8])
{
  int32_T c5_i3;
  c5_i3 = (int32_T)mwGetGlobalThreadIndex();
  if (c5_i3 < 8) {
    c5_anchors[c5_i3] = (real_T)c5_dv[c5_i3];
  }
}

static __global__ __launch_bounds__(32, 1) void c5_eML_blk_kernel_kernel14(const
  real_T c5_dv1[4], real_T c5_anchors[4], real_T c5_b_anchors[8])
{
  int32_T c5_i4;
  c5_i4 = (int32_T)mwGetGlobalThreadIndex();
  if (c5_i4 < 4) {
    c5_b_anchors[c5_i4] = c5_dv1[c5_i4];
    c5_anchors[c5_i4] = c5_b_anchors[c5_i4 + 4] / 16.0;
    c5_b_anchors[c5_i4 + 4] = c5_anchors[c5_i4];
  }
}

static __global__ __launch_bounds__(512, 1) void c5_eML_blk_kernel_kernel15(
  const real_T c5_anchors[8], const real32_T c5_tmpFeatureMap[4704], real32_T
  c5_boxOut[4704])
{
  uint64_T c5_threadId;
  uint64_T c5_tmpIndex;
  int32_T c5_anchorIdx;
  int32_T c5_colIdx;
  int32_T c5_ind;
  int32_T c5_rowIdx;
  real32_T c5_bh;
  real32_T c5_bw;
  real32_T c5_cx;
  real32_T c5_cy;
  c5_threadId = mwGetGlobalThreadIndex();
  c5_rowIdx = (int32_T)(c5_threadId % 14UL);
  c5_tmpIndex = (c5_threadId - (uint64_T)c5_rowIdx) / 14UL;
  c5_colIdx = (int32_T)(c5_tmpIndex % 14UL);
  c5_tmpIndex = (c5_tmpIndex - (uint64_T)c5_colIdx) / 14UL;
  c5_anchorIdx = (int32_T)c5_tmpIndex;
  if ((c5_anchorIdx < 4) && (c5_colIdx < 14) && (c5_rowIdx < 14)) {
    c5_ind = ((((c5_rowIdx * 14) << 2) + (c5_colIdx << 2)) + c5_anchorIdx) + 1;
    c5_cx = (c5_tmpFeatureMap[((c5_rowIdx + 14 * c5_colIdx) + 196 * c5_anchorIdx)
             + 784] + (real32_T)c5_colIdx) * 16.0F;
    c5_cy = (c5_tmpFeatureMap[((c5_rowIdx + 14 * c5_colIdx) + 196 * c5_anchorIdx)
             + 1568] + (real32_T)c5_rowIdx) * 16.0F;
    c5_bw = c5_tmpFeatureMap[((c5_rowIdx + 14 * c5_colIdx) + 196 * c5_anchorIdx)
      + 2352] * (real32_T)c5_anchors[c5_anchorIdx + 4] * 16.0F;
    c5_bh = c5_tmpFeatureMap[((c5_rowIdx + 14 * c5_colIdx) + 196 * c5_anchorIdx)
      + 3136] * (real32_T)c5_anchors[c5_anchorIdx] * 16.0F;
    c5_boxOut[c5_ind - 1] = c5_cx - c5_bw / 2.0F;
    c5_boxOut[c5_ind + 783] = c5_cy - c5_bh / 2.0F;
    c5_boxOut[c5_ind + 1567] = c5_cx + c5_bw / 2.0F;
    c5_boxOut[c5_ind + 2351] = c5_cy + c5_bh / 2.0F;
    c5_boxOut[c5_ind + 3135] = c5_tmpFeatureMap[(c5_rowIdx + 14 * c5_colIdx) +
      196 * c5_anchorIdx] * c5_tmpFeatureMap[((c5_rowIdx + 14 * c5_colIdx) + 196
      * c5_anchorIdx) + 3920];
    c5_boxOut[c5_ind + 3919] = 1.0F;
  }
}

static __global__ __launch_bounds__(512, 1) void c5_eML_blk_kernel_kernel16(
  const real32_T c5_boxOut[4704], boolean_T c5_bv[784])
{
  int32_T c5_i5;
  c5_i5 = (int32_T)mwGetGlobalThreadIndex();
  if (c5_i5 < 784) {
    c5_bv[c5_i5] = (boolean_T)((real_T)c5_boxOut[c5_i5 + 3136] >= 0.2);
  }
}

static __global__ __launch_bounds__(1024, 1) void c5_eML_blk_kernel_kernel17(
  const real32_T c5_boxOut[4704], const int16_T c5_ii_data[784], const int32_T
  c5_thresholdedPrediction_size[2], const int32_T c5_ii_size[1], real32_T
  c5_thresholdedPrediction_data[4704])
{
  int64_T c5_loopEnd;
  uint64_T c5_idx;
  uint64_T c5_threadId;
  uint64_T c5_threadStride;
  uint64_T c5_tmpIndex;
  int32_T c5_i6;
  int32_T c5_i9;
  c5_threadId = mwGetGlobalThreadIndex();
  c5_threadStride = mwGetTotalThreadsLaunched();
  c5_loopEnd = ((int64_T)(c5_ii_size[0] - 1) + 1L) * 6L - 1L;
  for (c5_idx = c5_threadId; c5_idx <= (uint64_T)c5_loopEnd; c5_idx +=
       c5_threadStride) {
    c5_i9 = (int32_T)(c5_idx % ((uint64_T)(c5_ii_size[0] - 1) + 1UL));
    c5_tmpIndex = (c5_idx - (uint64_T)c5_i9) / ((uint64_T)(c5_ii_size[0] - 1) +
      1UL);
    c5_i6 = (int32_T)c5_tmpIndex;
    c5_thresholdedPrediction_data[c5_i9 + c5_thresholdedPrediction_size[0] *
      c5_i6] = c5_boxOut[((int32_T)c5_ii_data[c5_i9] + 784 * c5_i6) - 1];
  }
}

static __global__ __launch_bounds__(1024, 1) void c5_eML_blk_kernel_kernel18(
  const real32_T c5_thresholdedPrediction_data[4704], const int32_T
  c5_thresholdedPrediction_size[2], const int32_T c5_bboxesX1Y1X2Y2_size[2],
  const int32_T c5_i7, real_T c5_bboxesX1Y1X2Y2_data[3136])
{
  int64_T c5_loopEnd;
  uint64_T c5_idx;
  uint64_T c5_threadId;
  uint64_T c5_threadStride;
  int32_T c5_i13;
  int32_T c5_i8;
  c5_threadId = mwGetGlobalThreadIndex();
  c5_threadStride = mwGetTotalThreadsLaunched();
  c5_loopEnd = ((int64_T)c5_i7 + 1L) * 4L - 1L;
  for (c5_idx = c5_threadId; c5_idx <= (uint64_T)c5_loopEnd; c5_idx +=
       c5_threadStride) {
    c5_i13 = (int32_T)(c5_idx % ((uint64_T)c5_i7 + 1UL));
    c5_i8 = (int32_T)((c5_idx - (uint64_T)c5_i13) / ((uint64_T)c5_i7 + 1UL));
    c5_bboxesX1Y1X2Y2_data[c5_i13 + c5_bboxesX1Y1X2Y2_size[0] * c5_i8] = (real_T)
      c5_thresholdedPrediction_data[c5_i13 + c5_thresholdedPrediction_size[0] *
      c5_i8];
  }
}

static __global__ __launch_bounds__(1024, 1) void c5_eML_blk_kernel_kernel19(
  const real_T c5_bboxesX1Y1X2Y2_data[3136], const int32_T c5_i10, real_T
  c5_x1_data[784])
{
  int64_T c5_loopEnd;
  uint64_T c5_idx;
  uint64_T c5_threadId;
  uint64_T c5_threadStride;
  int32_T c5_i12;
  c5_threadId = mwGetGlobalThreadIndex();
  c5_threadStride = mwGetTotalThreadsLaunched();
  c5_loopEnd = (int64_T)c5_i10;
  for (c5_idx = c5_threadId; c5_idx <= (uint64_T)c5_loopEnd; c5_idx +=
       c5_threadStride) {
    c5_i12 = (int32_T)c5_idx;
    c5_x1_data[c5_i12] = c5_bboxesX1Y1X2Y2_data[c5_i12];
  }
}

static __global__ __launch_bounds__(1024, 1) void c5_eML_blk_kernel_kernel20(
  const real_T c5_bboxesX1Y1X2Y2_data[3136], const int32_T
  c5_bboxesX1Y1X2Y2_size[2], const int32_T c5_i14, real_T c5_y1_data[784])
{
  int64_T c5_loopEnd;
  uint64_T c5_idx;
  uint64_T c5_threadId;
  uint64_T c5_threadStride;
  int32_T c5_i16;
  c5_threadId = mwGetGlobalThreadIndex();
  c5_threadStride = mwGetTotalThreadsLaunched();
  c5_loopEnd = (int64_T)c5_i14;
  for (c5_idx = c5_threadId; c5_idx <= (uint64_T)c5_loopEnd; c5_idx +=
       c5_threadStride) {
    c5_i16 = (int32_T)c5_idx;
    c5_y1_data[c5_i16] = c5_bboxesX1Y1X2Y2_data[c5_i16 + c5_bboxesX1Y1X2Y2_size
      [0]];
  }
}

static __global__ __launch_bounds__(1024, 1) void c5_eML_blk_kernel_kernel21(
  const real_T c5_bboxesX1Y1X2Y2_data[3136], const int32_T
  c5_bboxesX1Y1X2Y2_size[2], const int32_T c5_i17, real_T c5_x2_data[784])
{
  int64_T c5_loopEnd;
  uint64_T c5_idx;
  uint64_T c5_threadId;
  uint64_T c5_threadStride;
  int32_T c5_i18;
  c5_threadId = mwGetGlobalThreadIndex();
  c5_threadStride = mwGetTotalThreadsLaunched();
  c5_loopEnd = (int64_T)c5_i17;
  for (c5_idx = c5_threadId; c5_idx <= (uint64_T)c5_loopEnd; c5_idx +=
       c5_threadStride) {
    c5_i18 = (int32_T)c5_idx;
    c5_x2_data[c5_i18] = c5_bboxesX1Y1X2Y2_data[c5_i18 +
      (c5_bboxesX1Y1X2Y2_size[0] << 1)];
  }
}

static __global__ __launch_bounds__(1024, 1) void c5_eML_blk_kernel_kernel22(
  const real_T c5_bboxesX1Y1X2Y2_data[3136], const int32_T
  c5_bboxesX1Y1X2Y2_size[2], const int32_T c5_i19, real_T c5_y2_data[784])
{
  int64_T c5_loopEnd;
  uint64_T c5_idx;
  uint64_T c5_threadId;
  uint64_T c5_threadStride;
  int32_T c5_i20;
  c5_threadId = mwGetGlobalThreadIndex();
  c5_threadStride = mwGetTotalThreadsLaunched();
  c5_loopEnd = (int64_T)c5_i19;
  for (c5_idx = c5_threadId; c5_idx <= (uint64_T)c5_loopEnd; c5_idx +=
       c5_threadStride) {
    c5_i20 = (int32_T)c5_idx;
    c5_y2_data[c5_i20] = c5_bboxesX1Y1X2Y2_data[c5_i20 + c5_bboxesX1Y1X2Y2_size
      [0] * 3];
  }
}

static __global__ __launch_bounds__(1024, 1) void c5_eML_blk_kernel_kernel23(
  const int32_T c5_end, real_T c5_x1_data[784])
{
  int64_T c5_loopEnd;
  uint64_T c5_idx;
  uint64_T c5_threadId;
  uint64_T c5_threadStride;
  int32_T c5_i;
  c5_threadId = mwGetGlobalThreadIndex();
  c5_threadStride = mwGetTotalThreadsLaunched();
  c5_loopEnd = (int64_T)(c5_end - 1);
  for (c5_idx = c5_threadId; c5_idx <= (uint64_T)c5_loopEnd; c5_idx +=
       c5_threadStride) {
    c5_i = (int32_T)c5_idx;
    if (c5_x1_data[c5_i] < 1.0) {
      c5_x1_data[c5_i] = 1.0;
    }
  }
}

static __global__ __launch_bounds__(1024, 1) void c5_eML_blk_kernel_kernel24(
  const int32_T c5_end, real_T c5_y1_data[784])
{
  int64_T c5_loopEnd;
  uint64_T c5_idx;
  uint64_T c5_threadId;
  uint64_T c5_threadStride;
  int32_T c5_i;
  c5_threadId = mwGetGlobalThreadIndex();
  c5_threadStride = mwGetTotalThreadsLaunched();
  c5_loopEnd = (int64_T)(c5_end - 1);
  for (c5_idx = c5_threadId; c5_idx <= (uint64_T)c5_loopEnd; c5_idx +=
       c5_threadStride) {
    c5_i = (int32_T)c5_idx;
    if (c5_y1_data[c5_i] < 1.0) {
      c5_y1_data[c5_i] = 1.0;
    }
  }
}

static __global__ __launch_bounds__(1024, 1) void c5_eML_blk_kernel_kernel25(
  const int32_T c5_end, real_T c5_x2_data[784])
{
  int64_T c5_loopEnd;
  uint64_T c5_idx;
  uint64_T c5_threadId;
  uint64_T c5_threadStride;
  int32_T c5_i;
  c5_threadId = mwGetGlobalThreadIndex();
  c5_threadStride = mwGetTotalThreadsLaunched();
  c5_loopEnd = (int64_T)(c5_end - 1);
  for (c5_idx = c5_threadId; c5_idx <= (uint64_T)c5_loopEnd; c5_idx +=
       c5_threadStride) {
    c5_i = (int32_T)c5_idx;
    if (c5_x2_data[c5_i] > 224.0) {
      c5_x2_data[c5_i] = 224.0;
    }
  }
}

static __global__ __launch_bounds__(1024, 1) void c5_eML_blk_kernel_kernel26(
  const int32_T c5_end, real_T c5_y2_data[784])
{
  int64_T c5_loopEnd;
  uint64_T c5_idx;
  uint64_T c5_threadId;
  uint64_T c5_threadStride;
  int32_T c5_i;
  c5_threadId = mwGetGlobalThreadIndex();
  c5_threadStride = mwGetTotalThreadsLaunched();
  c5_loopEnd = (int64_T)(c5_end - 1);
  for (c5_idx = c5_threadId; c5_idx <= (uint64_T)c5_loopEnd; c5_idx +=
       c5_threadStride) {
    c5_i = (int32_T)c5_idx;
    if (c5_y2_data[c5_i] > 224.0) {
      c5_y2_data[c5_i] = 224.0;
    }
  }
}

static __global__ __launch_bounds__(1024, 1) void c5_eML_blk_kernel_kernel27(
  const real_T c5_x1_data[784], const int32_T c5_loop_ub, real_T
  c5_bboxesX1Y1X2Y2_data[3136])
{
  int64_T c5_loopEnd;
  uint64_T c5_idx;
  uint64_T c5_threadId;
  uint64_T c5_threadStride;
  int32_T c5_i21;
  c5_threadId = mwGetGlobalThreadIndex();
  c5_threadStride = mwGetTotalThreadsLaunched();
  c5_loopEnd = (int64_T)c5_loop_ub;
  for (c5_idx = c5_threadId; c5_idx <= (uint64_T)c5_loopEnd; c5_idx +=
       c5_threadStride) {
    c5_i21 = (int32_T)c5_idx;
    c5_bboxesX1Y1X2Y2_data[c5_i21] = c5_x1_data[c5_i21];
  }
}

static __global__ __launch_bounds__(1024, 1) void c5_eML_blk_kernel_kernel28(
  const real_T c5_y1_data[784], const int32_T c5_bboxesX1Y1X2Y2_size[2], const
  int32_T c5_loop_ub, real_T c5_bboxesX1Y1X2Y2_data[3136])
{
  int64_T c5_loopEnd;
  uint64_T c5_idx;
  uint64_T c5_threadId;
  uint64_T c5_threadStride;
  int32_T c5_i22;
  c5_threadId = mwGetGlobalThreadIndex();
  c5_threadStride = mwGetTotalThreadsLaunched();
  c5_loopEnd = (int64_T)c5_loop_ub;
  for (c5_idx = c5_threadId; c5_idx <= (uint64_T)c5_loopEnd; c5_idx +=
       c5_threadStride) {
    c5_i22 = (int32_T)c5_idx;
    c5_bboxesX1Y1X2Y2_data[c5_i22 + c5_bboxesX1Y1X2Y2_size[0]] =
      c5_y1_data[c5_i22];
  }
}

static __global__ __launch_bounds__(1024, 1) void c5_eML_blk_kernel_kernel29(
  const real_T c5_x2_data[784], const int32_T c5_bboxesX1Y1X2Y2_size[2], const
  int32_T c5_loop_ub, real_T c5_bboxesX1Y1X2Y2_data[3136])
{
  int64_T c5_loopEnd;
  uint64_T c5_idx;
  uint64_T c5_threadId;
  uint64_T c5_threadStride;
  int32_T c5_i23;
  c5_threadId = mwGetGlobalThreadIndex();
  c5_threadStride = mwGetTotalThreadsLaunched();
  c5_loopEnd = (int64_T)c5_loop_ub;
  for (c5_idx = c5_threadId; c5_idx <= (uint64_T)c5_loopEnd; c5_idx +=
       c5_threadStride) {
    c5_i23 = (int32_T)c5_idx;
    c5_bboxesX1Y1X2Y2_data[c5_i23 + (c5_bboxesX1Y1X2Y2_size[0] << 1)] =
      c5_x2_data[c5_i23];
  }
}

static __global__ __launch_bounds__(1024, 1) void c5_eML_blk_kernel_kernel30(
  const real_T c5_y2_data[784], const int32_T c5_bboxesX1Y1X2Y2_size[2], const
  int32_T c5_loop_ub, real_T c5_bboxesX1Y1X2Y2_data[3136])
{
  int64_T c5_loopEnd;
  uint64_T c5_idx;
  uint64_T c5_threadId;
  uint64_T c5_threadStride;
  int32_T c5_i24;
  c5_threadId = mwGetGlobalThreadIndex();
  c5_threadStride = mwGetTotalThreadsLaunched();
  c5_loopEnd = (int64_T)c5_loop_ub;
  for (c5_idx = c5_threadId; c5_idx <= (uint64_T)c5_loopEnd; c5_idx +=
       c5_threadStride) {
    c5_i24 = (int32_T)c5_idx;
    c5_bboxesX1Y1X2Y2_data[c5_i24 + c5_bboxesX1Y1X2Y2_size[0] * 3] =
      c5_y2_data[c5_i24];
  }
}

static __global__ __launch_bounds__(1024, 1) void c5_eML_blk_kernel_kernel31(
  const real_T c5_bboxesX1Y1X2Y2_data[3136], const int32_T c5_loop_ub, real_T
  c5_bboxPred_data[3136])
{
  int64_T c5_loopEnd;
  uint64_T c5_idx;
  uint64_T c5_threadId;
  uint64_T c5_threadStride;
  int32_T c5_i28;
  c5_threadId = mwGetGlobalThreadIndex();
  c5_threadStride = mwGetTotalThreadsLaunched();
  c5_loopEnd = (int64_T)c5_loop_ub;
  for (c5_idx = c5_threadId; c5_idx <= (uint64_T)c5_loopEnd; c5_idx +=
       c5_threadStride) {
    c5_i28 = (int32_T)c5_idx;
    c5_bboxPred_data[c5_i28] = ((c5_bboxesX1Y1X2Y2_data[c5_i28] - 0.5) *
      2.8571428571428572 + -0.9285714285714286) + 0.5;
  }
}

static __global__ __launch_bounds__(1024, 1) void c5_eML_blk_kernel_kernel32(
  const real_T c5_bboxesX1Y1X2Y2_data[3136], const int32_T
  c5_bboxesX1Y1X2Y2_size[2], const int32_T c5_bboxPred_size[2], const int32_T
  c5_loop_ub, real_T c5_bboxPred_data[3136])
{
  int64_T c5_loopEnd;
  uint64_T c5_idx;
  uint64_T c5_threadId;
  uint64_T c5_threadStride;
  int32_T c5_i29;
  c5_threadId = mwGetGlobalThreadIndex();
  c5_threadStride = mwGetTotalThreadsLaunched();
  c5_loopEnd = (int64_T)c5_loop_ub;
  for (c5_idx = c5_threadId; c5_idx <= (uint64_T)c5_loopEnd; c5_idx +=
       c5_threadStride) {
    c5_i29 = (int32_T)c5_idx;
    c5_bboxPred_data[c5_i29 + c5_bboxPred_size[0]] =
      ((c5_bboxesX1Y1X2Y2_data[c5_i29 + c5_bboxesX1Y1X2Y2_size[0]] - 0.5) *
       2.1428571428571428 + -0.5714285714285714) + 0.5;
  }
}

static __global__ __launch_bounds__(1024, 1) void c5_eML_blk_kernel_kernel33(
  const real_T c5_bboxesX1Y1X2Y2_data[3136], const int32_T
  c5_bboxesX1Y1X2Y2_size[2], const int32_T c5_bboxPred_size[2], const int32_T
  c5_loop_ub, real_T c5_bboxPred_data[3136])
{
  int64_T c5_loopEnd;
  uint64_T c5_idx;
  uint64_T c5_threadId;
  uint64_T c5_threadStride;
  int32_T c5_i30;
  c5_threadId = mwGetGlobalThreadIndex();
  c5_threadStride = mwGetTotalThreadsLaunched();
  c5_loopEnd = (int64_T)c5_loop_ub;
  for (c5_idx = c5_threadId; c5_idx <= (uint64_T)c5_loopEnd; c5_idx +=
       c5_threadStride) {
    c5_i30 = (int32_T)c5_idx;
    c5_bboxPred_data[c5_i30 + (c5_bboxPred_size[0] << 1)] =
      ((c5_bboxesX1Y1X2Y2_data[c5_i30 + (c5_bboxesX1Y1X2Y2_size[0] << 1)] + 0.5)
       * 2.8571428571428572 + -0.9285714285714286) - 0.5;
  }
}

static __global__ __launch_bounds__(1024, 1) void c5_eML_blk_kernel_kernel34(
  const real_T c5_bboxesX1Y1X2Y2_data[3136], const int32_T
  c5_bboxesX1Y1X2Y2_size[2], const int32_T c5_bboxPred_size[2], const int32_T
  c5_loop_ub, real_T c5_bboxPred_data[3136])
{
  int64_T c5_loopEnd;
  uint64_T c5_idx;
  uint64_T c5_threadId;
  uint64_T c5_threadStride;
  int32_T c5_i31;
  c5_threadId = mwGetGlobalThreadIndex();
  c5_threadStride = mwGetTotalThreadsLaunched();
  c5_loopEnd = (int64_T)c5_loop_ub;
  for (c5_idx = c5_threadId; c5_idx <= (uint64_T)c5_loopEnd; c5_idx +=
       c5_threadStride) {
    c5_i31 = (int32_T)c5_idx;
    c5_bboxPred_data[c5_i31 + c5_bboxPred_size[0] * 3] =
      ((c5_bboxesX1Y1X2Y2_data[c5_i31 + c5_bboxesX1Y1X2Y2_size[0] * 3] + 0.5) *
       2.1428571428571428 + -0.5714285714285714) - 0.5;
  }
}

static __global__ __launch_bounds__(1024, 1) void c5_eML_blk_kernel_kernel35(
  const int32_T c5_nx, real_T c5_bboxPred_data[3136])
{
  int64_T c5_loopEnd;
  uint64_T c5_idx;
  uint64_T c5_threadId;
  uint64_T c5_threadStride;
  int32_T c5_k;
  c5_threadId = mwGetGlobalThreadIndex();
  c5_threadStride = mwGetTotalThreadsLaunched();
  c5_loopEnd = (int64_T)(c5_nx - 1);
  for (c5_idx = c5_threadId; c5_idx <= (uint64_T)c5_loopEnd; c5_idx +=
       c5_threadStride) {
    c5_k = (int32_T)c5_idx;
    c5_bboxPred_data[c5_k] = floor(c5_bboxPred_data[c5_k]);
  }
}

static __global__ __launch_bounds__(1024, 1) void c5_eML_blk_kernel_kernel36(
  const real_T c5_bboxPred_data[3136], const int32_T c5_bboxPred_size[2], const
  int32_T c5_i32, real_T c5_b_bboxPred_data[784])
{
  int64_T c5_loopEnd;
  uint64_T c5_idx;
  uint64_T c5_threadId;
  uint64_T c5_threadStride;
  int32_T c5_i33;
  c5_threadId = mwGetGlobalThreadIndex();
  c5_threadStride = mwGetTotalThreadsLaunched();
  c5_loopEnd = (int64_T)c5_i32;
  for (c5_idx = c5_threadId; c5_idx <= (uint64_T)c5_loopEnd; c5_idx +=
       c5_threadStride) {
    c5_i33 = (int32_T)c5_idx;
    c5_b_bboxPred_data[c5_i33] = (c5_bboxPred_data[c5_i33 + (c5_bboxPred_size[0]
      << 1)] - c5_bboxPred_data[c5_i33]) + 1.0;
  }
}

static __global__ __launch_bounds__(1024, 1) void c5_eML_blk_kernel_kernel37(
  const real_T c5_bboxPred_data[784], const int32_T c5_bboxPred_size[2], const
  int32_T c5_b_bboxPred_size[1], real_T c5_b_bboxPred_data[3136])
{
  int64_T c5_loopEnd;
  uint64_T c5_idx;
  uint64_T c5_threadId;
  uint64_T c5_threadStride;
  int32_T c5_i34;
  c5_threadId = mwGetGlobalThreadIndex();
  c5_threadStride = mwGetTotalThreadsLaunched();
  c5_loopEnd = (int64_T)(c5_b_bboxPred_size[0] - 1);
  for (c5_idx = c5_threadId; c5_idx <= (uint64_T)c5_loopEnd; c5_idx +=
       c5_threadStride) {
    c5_i34 = (int32_T)c5_idx;
    c5_b_bboxPred_data[c5_i34 + (c5_bboxPred_size[0] << 1)] =
      c5_bboxPred_data[c5_i34];
  }
}

static __global__ __launch_bounds__(1024, 1) void c5_eML_blk_kernel_kernel38(
  const real_T c5_bboxPred_data[3136], const int32_T c5_bboxPred_size[2], const
  int32_T c5_i35, real_T c5_b_bboxPred_data[784])
{
  int64_T c5_loopEnd;
  uint64_T c5_idx;
  uint64_T c5_threadId;
  uint64_T c5_threadStride;
  int32_T c5_i36;
  c5_threadId = mwGetGlobalThreadIndex();
  c5_threadStride = mwGetTotalThreadsLaunched();
  c5_loopEnd = (int64_T)c5_i35;
  for (c5_idx = c5_threadId; c5_idx <= (uint64_T)c5_loopEnd; c5_idx +=
       c5_threadStride) {
    c5_i36 = (int32_T)c5_idx;
    c5_b_bboxPred_data[c5_i36] = (c5_bboxPred_data[c5_i36 + c5_bboxPred_size[0] *
      3] - c5_bboxPred_data[c5_i36 + c5_bboxPred_size[0]]) + 1.0;
  }
}

static __global__ __launch_bounds__(1024, 1) void c5_eML_blk_kernel_kernel39(
  const real_T c5_bboxPred_data[784], const int32_T c5_bboxPred_size[2], const
  int32_T c5_b_bboxPred_size[1], real_T c5_b_bboxPred_data[3136])
{
  int64_T c5_loopEnd;
  uint64_T c5_idx;
  uint64_T c5_threadId;
  uint64_T c5_threadStride;
  int32_T c5_i37;
  c5_threadId = mwGetGlobalThreadIndex();
  c5_threadStride = mwGetTotalThreadsLaunched();
  c5_loopEnd = (int64_T)(c5_b_bboxPred_size[0] - 1);
  for (c5_idx = c5_threadId; c5_idx <= (uint64_T)c5_loopEnd; c5_idx +=
       c5_threadStride) {
    c5_i37 = (int32_T)c5_idx;
    c5_b_bboxPred_data[c5_i37 + c5_bboxPred_size[0] * 3] =
      c5_bboxPred_data[c5_i37];
  }
}

static __global__ __launch_bounds__(32, 1) void c5_eML_blk_kernel_kernel40(const
  real_T c5_bboxPred_data[3136], const int32_T c5_bboxPred_size[2], const
  int32_T c5_i, const int32_T c5_b_bboxPred_size[2], const int32_T c5_count,
  real_T c5_b_bboxPred_data[3136])
{
  int32_T c5_i41;
  c5_i41 = (int32_T)mwGetGlobalThreadIndex();
  if (c5_i41 < 4) {
    c5_b_bboxPred_data[c5_count + c5_b_bboxPred_size[0] * c5_i41] =
      c5_bboxPred_data[c5_i + c5_bboxPred_size[0] * c5_i41];
  }
}

static __global__ __launch_bounds__(32, 1) void c5_eML_blk_kernel_kernel41(const
  real32_T c5_thresholdedPrediction_data[4704], const int32_T
  c5_thresholdedPrediction_size[2], const int32_T c5_i, const real_T c5_count,
  real32_T c5_classPred_data[784], real32_T c5_scorePred_data[784])
{
  int32_T c5_tmpIdx;
  c5_tmpIdx = (int32_T)mwGetGlobalThreadIndex();
  if (c5_tmpIdx < 1) {
    c5_scorePred_data[(int32_T)c5_count - 1] =
      c5_thresholdedPrediction_data[c5_i + (c5_thresholdedPrediction_size[0] <<
      2)];
    c5_classPred_data[(int32_T)c5_count - 1] =
      c5_thresholdedPrediction_data[c5_i + c5_thresholdedPrediction_size[0] * 5];
  }
}

static __global__ __launch_bounds__(1024, 1) void c5_eML_blk_kernel_kernel42(
  const int32_T c5_i38, const int32_T c5_i39, int32_T c5_idx_data[784])
{
  int64_T c5_loopEnd;
  uint64_T c5_idx;
  uint64_T c5_threadId;
  uint64_T c5_threadStride;
  int32_T c5_i40;
  c5_threadId = mwGetGlobalThreadIndex();
  c5_threadStride = mwGetTotalThreadsLaunched();
  c5_loopEnd = (int64_T)(c5_i39 - c5_i38);
  for (c5_idx = c5_threadId; c5_idx <= (uint64_T)c5_loopEnd; c5_idx +=
       c5_threadStride) {
    c5_i40 = (int32_T)c5_idx;
    c5_idx_data[c5_i40] = c5_i38 + c5_i40;
  }
}

static __global__ __launch_bounds__(1024, 1) void c5_eML_blk_kernel_kernel43(
  const int32_T c5_bboxPred_size[2], boolean_T c5_b_data[784])
{
  int64_T c5_loopEnd;
  uint64_T c5_idx;
  uint64_T c5_threadId;
  uint64_T c5_threadStride;
  int32_T c5_i42;
  c5_threadId = mwGetGlobalThreadIndex();
  c5_threadStride = mwGetTotalThreadsLaunched();
  c5_loopEnd = (int64_T)(c5_bboxPred_size[0] - 1);
  for (c5_idx = c5_threadId; c5_idx <= (uint64_T)c5_loopEnd; c5_idx +=
       c5_threadStride) {
    c5_i42 = (int32_T)c5_idx;
    c5_b_data[c5_i42] = false;
  }
}

static __global__ __launch_bounds__(1024, 1) void c5_eML_blk_kernel_kernel44(
  const int32_T c5_b_size[2], const boolean_T c5_b_data[784], int32_T c5_i3,
  int32_T *c5_n)
{
  int64_T c5_loopEnd;
  int32_T c5_tmpRed0;
  uint32_T c5_blockStride;
  uint32_T c5_idx;
  uint32_T c5_m;
  uint32_T c5_mask;
  uint32_T c5_numActiveThreads;
  uint32_T c5_numActiveWarps;
  uint32_T c5_thBlkId;
  uint32_T c5_threadId;
  uint32_T c5_threadStride;
  c5_threadStride = (uint32_T)mwGetTotalThreadsLaunched();
  c5_threadId = (uint32_T)mwGetGlobalThreadIndex();
  c5_thBlkId = (uint32_T)mwGetThreadIndexWithinBlock();
  c5_blockStride = (uint32_T)mwGetThreadsPerBlock();
  c5_tmpRed0 = 0;
  c5_loopEnd = (int64_T)(c5_i3 - 1);
  c5_numActiveThreads = c5_blockStride;
  if (mwIsLastBlock()) {
    c5_m = ((int64_T)(c5_i3 - 1) + 1L) % (int64_T)c5_blockStride;
    if (c5_m > 0U) {
      c5_numActiveThreads = c5_m;
    }
  }

  c5_numActiveWarps = (uint32_T)(c5_numActiveThreads + ((int64_T)warpSize - 1L))
    / warpSize;
  if ((int64_T)c5_threadId <= c5_loopEnd) {
    c5_tmpRed0 = (int32_T)c5_b_data[c5_threadId];
  }

  c5_mask = __ballot_sync(MAX_uint32_T, (int64_T)c5_threadId <= c5_loopEnd);
  for (c5_idx = c5_threadId + c5_threadStride; c5_idx <= (uint32_T)c5_loopEnd;
       c5_idx += c5_threadStride) {
    c5_tmpRed0 += (int32_T)c5_b_data[c5_idx];
  }

  c5_tmpRed0 = c5_c_workGroupReduction(c5_tmpRed0, c5_mask, c5_numActiveWarps);
  if (c5_thBlkId == 0U) {
    atomicAdd(&c5_n[0], c5_tmpRed0);
  }
}

static __device__ int32_T c5_c_threadGroupReduction(int32_T c5_val, uint32_T
  c5_lane, uint32_T c5_mask)
{
  int32_T c5_other;
  uint32_T c5_activeSize;
  uint32_T c5_offset;
  c5_activeSize = __popc(c5_mask);
  c5_offset = (c5_activeSize + 1U) / 2U;
  while (c5_activeSize > 1U) {
    c5_other = c5_b_shflDown1(c5_val, c5_offset, c5_mask);
    if (c5_lane + c5_offset < c5_activeSize) {
      c5_val += c5_other;
    }

    c5_activeSize = c5_offset;
    c5_offset = (c5_offset + 1U) / 2U;
  }

  return c5_val;
}

static __device__ int32_T c5_b_shflDown1(int32_T c5_in1, uint32_T c5_offset,
  uint32_T c5_mask)
{
  c5_in1 = __shfl_down_sync(c5_mask, c5_in1, c5_offset);
  return c5_in1;
}

static __device__ int32_T c5_c_workGroupReduction(int32_T c5_val, uint32_T
  c5_mask, uint32_T c5_numActiveWarps)
{
  __shared__ int32_T c5_shared[32];
  uint32_T c5_lane;
  uint32_T c5_thBlkId;
  uint32_T c5_widx;
  c5_thBlkId = (uint32_T)mwGetThreadIndexWithinBlock();
  c5_lane = c5_thBlkId % warpSize;
  c5_widx = c5_thBlkId / warpSize;
  c5_val = c5_c_threadGroupReduction(c5_val, c5_lane, c5_mask);
  if (c5_lane == 0U) {
    c5_shared[c5_widx] = c5_val;
  }

  __syncthreads();
  c5_mask = __ballot_sync(MAX_uint32_T, c5_lane < c5_numActiveWarps);
  c5_val = c5_shared[c5_lane];
  if (c5_widx == 0U) {
    c5_val = c5_c_threadGroupReduction(c5_val, c5_lane, c5_mask);
  }

  return c5_val;
}

static __global__ __launch_bounds__(32, 1) void c5_eML_blk_kernel_kernel45(const
  int32_T *c5_n, const int32_T c5_bboxPred_size[2], int32_T *c5_nrows)
{
  int32_T c5_tmpIdx;
  c5_tmpIdx = (int32_T)mwGetGlobalThreadIndex();
  if (c5_tmpIdx < 1) {
    *c5_nrows = c5_bboxPred_size[0] - *c5_n;
  }
}

static __global__ __launch_bounds__(32, 1) void c5_eML_blk_kernel_kernel46(const
  int32_T c5_bboxPred_size[2], const int32_T *c5_nrows, const int32_T
  c5_idx_data[784], real_T c5_bboxPred_data[3136])
{
  int32_T c5_b_i;
  int32_T c5_i;
  int32_T c5_i2;
  int32_T c5_j;
  c5_j = (int32_T)mwGetGlobalThreadIndex();
  if (c5_j < 4) {
    c5_i2 = c5_idx_data[0];
    for (c5_i = 0; c5_i <= *c5_nrows - c5_i2; c5_i++) {
      c5_b_i = c5_i2 + c5_i;
      c5_bboxPred_data[(c5_b_i + c5_bboxPred_size[0] * c5_j) - 1] =
        c5_bboxPred_data[c5_b_i + c5_bboxPred_size[0] * c5_j];
    }
  }
}

static __global__ __launch_bounds__(1024, 1) void c5_eML_blk_kernel_kernel47(
  const real_T c5_bboxPred_data[3136], const int32_T c5_bboxPred_size[2], const
  int32_T c5_b_bboxPred_size[2], const int32_T c5_i4, real_T c5_b_bboxPred_data
  [3136])
{
  int64_T c5_loopEnd;
  uint64_T c5_idx;
  uint64_T c5_threadId;
  uint64_T c5_threadStride;
  int32_T c5_i43;
  int32_T c5_i46;
  c5_threadId = mwGetGlobalThreadIndex();
  c5_threadStride = mwGetTotalThreadsLaunched();
  c5_loopEnd = ((int64_T)c5_i4 + 1L) * 4L - 1L;
  for (c5_idx = c5_threadId; c5_idx <= (uint64_T)c5_loopEnd; c5_idx +=
       c5_threadStride) {
    c5_i46 = (int32_T)(c5_idx % ((uint64_T)c5_i4 + 1UL));
    c5_i43 = (int32_T)((c5_idx - (uint64_T)c5_i46) / ((uint64_T)c5_i4 + 1UL));
    c5_b_bboxPred_data[c5_i46 + c5_b_bboxPred_size[0] * c5_i43] =
      c5_bboxPred_data[c5_i46 + c5_bboxPred_size[0] * c5_i43];
  }
}

static __global__ __launch_bounds__(1024, 1) void c5_eML_blk_kernel_kernel48(
  const real_T c5_bboxPred_data[3136], const int32_T c5_bboxPred_size[2], real_T
  c5_b_bboxPred_data[3136])
{
  int64_T c5_loopEnd;
  uint64_T c5_idx;
  uint64_T c5_threadId;
  uint64_T c5_threadStride;
  int32_T c5_i44;
  c5_threadId = mwGetGlobalThreadIndex();
  c5_threadStride = mwGetTotalThreadsLaunched();
  c5_loopEnd = (int64_T)(c5_bboxPred_size[0] * 4 - 1);
  for (c5_idx = c5_threadId; c5_idx <= (uint64_T)c5_loopEnd; c5_idx +=
       c5_threadStride) {
    c5_i44 = (int32_T)c5_idx;
    c5_b_bboxPred_data[c5_i44] = c5_bboxPred_data[c5_i44];
  }
}

static __global__ __launch_bounds__(1024, 1) void c5_eML_blk_kernel_kernel49(
  const int32_T c5_i45, const int32_T c5_i47, int32_T c5_idx_data[784])
{
  int64_T c5_loopEnd;
  uint64_T c5_idx;
  uint64_T c5_threadId;
  uint64_T c5_threadStride;
  int32_T c5_i48;
  c5_threadId = mwGetGlobalThreadIndex();
  c5_threadStride = mwGetTotalThreadsLaunched();
  c5_loopEnd = (int64_T)(c5_i47 - c5_i45);
  for (c5_idx = c5_threadId; c5_idx <= (uint64_T)c5_loopEnd; c5_idx +=
       c5_threadStride) {
    c5_i48 = (int32_T)c5_idx;
    c5_idx_data[c5_i48] = c5_i45 + c5_i48;
  }
}

static __global__ __launch_bounds__(1024, 1) void c5_eML_blk_kernel_kernel50(
  const int32_T c5_scorePred_size[1], boolean_T c5_b_data[784])
{
  int64_T c5_loopEnd;
  uint64_T c5_idx;
  uint64_T c5_threadId;
  uint64_T c5_threadStride;
  int32_T c5_i49;
  c5_threadId = mwGetGlobalThreadIndex();
  c5_threadStride = mwGetTotalThreadsLaunched();
  c5_loopEnd = (int64_T)(c5_scorePred_size[0] - 1);
  for (c5_idx = c5_threadId; c5_idx <= (uint64_T)c5_loopEnd; c5_idx +=
       c5_threadStride) {
    c5_i49 = (int32_T)c5_idx;
    c5_b_data[c5_i49] = false;
  }
}

static __global__ __launch_bounds__(1024, 1) void c5_eML_blk_kernel_kernel51(
  const int32_T c5_b_size[2], const boolean_T c5_b_data[784], int32_T c5_i6,
  int32_T *c5_n)
{
  int64_T c5_loopEnd;
  int32_T c5_tmpRed0;
  uint32_T c5_blockStride;
  uint32_T c5_idx;
  uint32_T c5_m;
  uint32_T c5_mask;
  uint32_T c5_numActiveThreads;
  uint32_T c5_numActiveWarps;
  uint32_T c5_thBlkId;
  uint32_T c5_threadId;
  uint32_T c5_threadStride;
  c5_threadStride = (uint32_T)mwGetTotalThreadsLaunched();
  c5_threadId = (uint32_T)mwGetGlobalThreadIndex();
  c5_thBlkId = (uint32_T)mwGetThreadIndexWithinBlock();
  c5_blockStride = (uint32_T)mwGetThreadsPerBlock();
  c5_tmpRed0 = 0;
  c5_loopEnd = (int64_T)(c5_i6 - 1);
  c5_numActiveThreads = c5_blockStride;
  if (mwIsLastBlock()) {
    c5_m = ((int64_T)(c5_i6 - 1) + 1L) % (int64_T)c5_blockStride;
    if (c5_m > 0U) {
      c5_numActiveThreads = c5_m;
    }
  }

  c5_numActiveWarps = (uint32_T)(c5_numActiveThreads + ((int64_T)warpSize - 1L))
    / warpSize;
  if ((int64_T)c5_threadId <= c5_loopEnd) {
    c5_tmpRed0 = (int32_T)c5_b_data[c5_threadId];
  }

  c5_mask = __ballot_sync(MAX_uint32_T, (int64_T)c5_threadId <= c5_loopEnd);
  for (c5_idx = c5_threadId + c5_threadStride; c5_idx <= (uint32_T)c5_loopEnd;
       c5_idx += c5_threadStride) {
    c5_tmpRed0 += (int32_T)c5_b_data[c5_idx];
  }

  c5_tmpRed0 = c5_c_workGroupReduction(c5_tmpRed0, c5_mask, c5_numActiveWarps);
  if (c5_thBlkId == 0U) {
    atomicAdd(&c5_n[0], c5_tmpRed0);
  }
}

static __global__ __launch_bounds__(1024, 1) void c5_eML_blk_kernel_kernel52(
  const int32_T c5_i50, const int32_T c5_i51, int32_T c5_idx_data[784])
{
  int64_T c5_loopEnd;
  uint64_T c5_idx;
  uint64_T c5_threadId;
  uint64_T c5_threadStride;
  int32_T c5_i52;
  c5_threadId = mwGetGlobalThreadIndex();
  c5_threadStride = mwGetTotalThreadsLaunched();
  c5_loopEnd = (int64_T)(c5_i51 - c5_i50);
  for (c5_idx = c5_threadId; c5_idx <= (uint64_T)c5_loopEnd; c5_idx +=
       c5_threadStride) {
    c5_i52 = (int32_T)c5_idx;
    c5_idx_data[c5_i52] = c5_i50 + c5_i52;
  }
}

static __global__ __launch_bounds__(1024, 1) void c5_eML_blk_kernel_kernel53(
  const int32_T c5_classPred_size[1], boolean_T c5_b_data[784])
{
  int64_T c5_loopEnd;
  uint64_T c5_idx;
  uint64_T c5_threadId;
  uint64_T c5_threadStride;
  int32_T c5_i53;
  c5_threadId = mwGetGlobalThreadIndex();
  c5_threadStride = mwGetTotalThreadsLaunched();
  c5_loopEnd = (int64_T)(c5_classPred_size[0] - 1);
  for (c5_idx = c5_threadId; c5_idx <= (uint64_T)c5_loopEnd; c5_idx +=
       c5_threadStride) {
    c5_i53 = (int32_T)c5_idx;
    c5_b_data[c5_i53] = false;
  }
}

static __global__ __launch_bounds__(1024, 1) void c5_eML_blk_kernel_kernel54(
  const int32_T c5_b_size[2], const boolean_T c5_b_data[784], int32_T c5_i9,
  int32_T *c5_n)
{
  int64_T c5_loopEnd;
  int32_T c5_tmpRed0;
  uint32_T c5_blockStride;
  uint32_T c5_idx;
  uint32_T c5_m;
  uint32_T c5_mask;
  uint32_T c5_numActiveThreads;
  uint32_T c5_numActiveWarps;
  uint32_T c5_thBlkId;
  uint32_T c5_threadId;
  uint32_T c5_threadStride;
  c5_threadStride = (uint32_T)mwGetTotalThreadsLaunched();
  c5_threadId = (uint32_T)mwGetGlobalThreadIndex();
  c5_thBlkId = (uint32_T)mwGetThreadIndexWithinBlock();
  c5_blockStride = (uint32_T)mwGetThreadsPerBlock();
  c5_tmpRed0 = 0;
  c5_loopEnd = (int64_T)(c5_i9 - 1);
  c5_numActiveThreads = c5_blockStride;
  if (mwIsLastBlock()) {
    c5_m = ((int64_T)(c5_i9 - 1) + 1L) % (int64_T)c5_blockStride;
    if (c5_m > 0U) {
      c5_numActiveThreads = c5_m;
    }
  }

  c5_numActiveWarps = (uint32_T)(c5_numActiveThreads + ((int64_T)warpSize - 1L))
    / warpSize;
  if ((int64_T)c5_threadId <= c5_loopEnd) {
    c5_tmpRed0 = (int32_T)c5_b_data[c5_threadId];
  }

  c5_mask = __ballot_sync(MAX_uint32_T, (int64_T)c5_threadId <= c5_loopEnd);
  for (c5_idx = c5_threadId + c5_threadStride; c5_idx <= (uint32_T)c5_loopEnd;
       c5_idx += c5_threadStride) {
    c5_tmpRed0 += (int32_T)c5_b_data[c5_idx];
  }

  c5_tmpRed0 = c5_c_workGroupReduction(c5_tmpRed0, c5_mask, c5_numActiveWarps);
  if (c5_thBlkId == 0U) {
    atomicAdd(&c5_n[0], c5_tmpRed0);
  }
}

static __global__ __launch_bounds__(1024, 1) void c5_eML_blk_kernel_kernel55(
  const real32_T c5_scorePred_data[784], const int32_T c5_scorePred_size[1],
  real32_T c5_out_data[784])
{
  int64_T c5_loopEnd;
  uint64_T c5_idx;
  uint64_T c5_threadId;
  uint64_T c5_threadStride;
  int32_T c5_i58;
  c5_threadId = mwGetGlobalThreadIndex();
  c5_threadStride = mwGetTotalThreadsLaunched();
  c5_loopEnd = (int64_T)(c5_scorePred_size[0] - 1);
  for (c5_idx = c5_threadId; c5_idx <= (uint64_T)c5_loopEnd; c5_idx +=
       c5_threadStride) {
    c5_i58 = (int32_T)c5_idx;
    c5_out_data[c5_i58] = c5_scorePred_data[c5_i58];
  }
}

static __global__ __launch_bounds__(1024, 1) void c5_eML_blk_kernel_kernel56(
  const uint32_T c5_dv2[2], real_T c5_x1_data[784])
{
  int64_T c5_loopEnd;
  uint64_T c5_idx;
  uint64_T c5_threadId;
  uint64_T c5_threadStride;
  int32_T c5_i62;
  c5_threadId = mwGetGlobalThreadIndex();
  c5_threadStride = mwGetTotalThreadsLaunched();
  c5_loopEnd = (int64_T)((int32_T)c5_dv2[0] - 1);
  for (c5_idx = c5_threadId; c5_idx <= (uint64_T)c5_loopEnd; c5_idx +=
       c5_threadStride) {
    c5_i62 = (int32_T)c5_idx;
    c5_x1_data[c5_i62] = 0.0;
  }
}

static __global__ __launch_bounds__(1024, 1) void c5_eML_blk_kernel_kernel57(
  const real_T c5_bboxPred_data[3136], const int32_T c5_bboxPred_size[2], const
  real_T c5_x1_data[784], const int32_T c5_bboxesX1Y1X2Y2_size[2], const int32_T
  c5_x1_size[1], real_T c5_bboxesX1Y1X2Y2_data[3136])
{
  int64_T c5_loopEnd;
  uint64_T c5_idx;
  uint64_T c5_threadId;
  uint64_T c5_threadStride;
  uint64_T c5_tmpIndex;
  int32_T c5_i56;
  int32_T c5_i59;
  c5_threadId = mwGetGlobalThreadIndex();
  c5_threadStride = mwGetTotalThreadsLaunched();
  c5_loopEnd = ((int64_T)(c5_x1_size[0] - 1) + 1L) * 4L - 1L;
  for (c5_idx = c5_threadId; c5_idx <= (uint64_T)c5_loopEnd; c5_idx +=
       c5_threadStride) {
    c5_i59 = (int32_T)(c5_idx % ((uint64_T)(c5_x1_size[0] - 1) + 1UL));
    c5_tmpIndex = (c5_idx - (uint64_T)c5_i59) / ((uint64_T)(c5_x1_size[0] - 1) +
      1UL);
    c5_i56 = (int32_T)c5_tmpIndex;
    c5_bboxesX1Y1X2Y2_data[c5_i59 + c5_bboxesX1Y1X2Y2_size[0] * c5_i56] =
      c5_bboxPred_data[((int32_T)c5_x1_data[c5_i59] + c5_bboxPred_size[0] *
                        c5_i56) - 1];
  }
}

static __global__ __launch_bounds__(1024, 1) void c5_eML_blk_kernel_kernel58(
  const real32_T c5_classPred_data[784], const real_T c5_x1_data[784], const
  int32_T c5_x1_size[1], real_T c5_y1_data[784])
{
  int64_T c5_loopEnd;
  uint64_T c5_idx;
  uint64_T c5_threadId;
  uint64_T c5_threadStride;
  int32_T c5_i57;
  c5_threadId = mwGetGlobalThreadIndex();
  c5_threadStride = mwGetTotalThreadsLaunched();
  c5_loopEnd = (int64_T)(c5_x1_size[0] - 1);
  for (c5_idx = c5_threadId; c5_idx <= (uint64_T)c5_loopEnd; c5_idx +=
       c5_threadStride) {
    c5_i57 = (int32_T)c5_idx;
    c5_y1_data[c5_i57] = (real_T)c5_classPred_data[(int32_T)c5_x1_data[c5_i57] -
      1];
  }
}

static __global__ __launch_bounds__(1024, 1) void c5_eML_blk_kernel_kernel59(
  const int32_T c5_x1_size[1], boolean_T c5_selectedIndex_data[784])
{
  int64_T c5_loopEnd;
  uint64_T c5_idx;
  uint64_T c5_threadId;
  uint64_T c5_threadStride;
  int32_T c5_i60;
  c5_threadId = mwGetGlobalThreadIndex();
  c5_threadStride = mwGetTotalThreadsLaunched();
  c5_loopEnd = (int64_T)(c5_x1_size[0] - 1);
  for (c5_idx = c5_threadId; c5_idx <= (uint64_T)c5_loopEnd; c5_idx +=
       c5_threadStride) {
    c5_i60 = (int32_T)c5_idx;
    c5_selectedIndex_data[c5_i60] = true;
  }
}

static __global__ __launch_bounds__(1024, 1) void c5_eML_blk_kernel_kernel60(
  const real_T c5_bboxesX1Y1X2Y2_data[3136], const int32_T
  c5_bboxesX1Y1X2Y2_size[2], const int32_T c5_i61, real_T c5_area_data[784])
{
  int64_T c5_loopEnd;
  uint64_T c5_idx;
  uint64_T c5_threadId;
  uint64_T c5_threadStride;
  int32_T c5_i63;
  c5_threadId = mwGetGlobalThreadIndex();
  c5_threadStride = mwGetTotalThreadsLaunched();
  c5_loopEnd = (int64_T)c5_i61;
  for (c5_idx = c5_threadId; c5_idx <= (uint64_T)c5_loopEnd; c5_idx +=
       c5_threadStride) {
    c5_i63 = (int32_T)c5_idx;
    c5_area_data[c5_i63] = c5_bboxesX1Y1X2Y2_data[c5_i63 +
      (c5_bboxesX1Y1X2Y2_size[0] << 1)] * c5_bboxesX1Y1X2Y2_data[c5_i63 +
      c5_bboxesX1Y1X2Y2_size[0] * 3];
  }
}

static __global__ __launch_bounds__(1024, 1) void c5_eML_blk_kernel_kernel61(
  const int32_T c5_bboxesX1Y1X2Y2_size[2], const real_T c5_bboxesX1Y1X2Y2_data
  [3136], const int32_T c5_i64, real_T c5_x2_data[784])
{
  int64_T c5_loopEnd;
  uint64_T c5_idx;
  uint64_T c5_threadId;
  uint64_T c5_threadStride;
  int32_T c5_i65;
  c5_threadId = mwGetGlobalThreadIndex();
  c5_threadStride = mwGetTotalThreadsLaunched();
  c5_loopEnd = (int64_T)c5_i64;
  for (c5_idx = c5_threadId; c5_idx <= (uint64_T)c5_loopEnd; c5_idx +=
       c5_threadStride) {
    c5_i65 = (int32_T)c5_idx;
    c5_x2_data[c5_i65] = c5_bboxesX1Y1X2Y2_data[c5_i65] +
      c5_bboxesX1Y1X2Y2_data[c5_i65 + (c5_bboxesX1Y1X2Y2_size[0] << 1)];
  }
}

static __global__ __launch_bounds__(1024, 1) void c5_eML_blk_kernel_kernel62(
  const real_T c5_bboxesX1Y1X2Y2_data[3136], const int32_T
  c5_bboxesX1Y1X2Y2_size[2], const int32_T c5_i66, real_T c5_y2_data[784])
{
  int64_T c5_loopEnd;
  uint64_T c5_idx;
  uint64_T c5_threadId;
  uint64_T c5_threadStride;
  int32_T c5_i67;
  c5_threadId = mwGetGlobalThreadIndex();
  c5_threadStride = mwGetTotalThreadsLaunched();
  c5_loopEnd = (int64_T)c5_i66;
  for (c5_idx = c5_threadId; c5_idx <= (uint64_T)c5_loopEnd; c5_idx +=
       c5_threadStride) {
    c5_i67 = (int32_T)c5_idx;
    c5_y2_data[c5_i67] = c5_bboxesX1Y1X2Y2_data[c5_i67 + c5_bboxesX1Y1X2Y2_size
      [0]] + c5_bboxesX1Y1X2Y2_data[c5_i67 + c5_bboxesX1Y1X2Y2_size[0] * 3];
  }
}

static __global__ __launch_bounds__(1024, 1) void c5_eML_blk_kernel_kernel63(
  const int32_T c5_i12, const int32_T c5_iv[2], boolean_T c5_selectedIndex_data
  [784])
{
  int64_T c5_loopEnd;
  uint64_T c5_idx;
  uint64_T c5_threadId;
  uint64_T c5_threadStride;
  int32_T c5_i68;
  c5_threadId = mwGetGlobalThreadIndex();
  c5_threadStride = mwGetTotalThreadsLaunched();
  c5_loopEnd = (int64_T)(c5_iv[1] - 1);
  for (c5_idx = c5_threadId; c5_idx <= (uint64_T)c5_loopEnd; c5_idx +=
       c5_threadStride) {
    c5_i68 = (int32_T)c5_idx;
    c5_selectedIndex_data[c5_i12 + c5_i68] = false;
  }
}

static __global__ __launch_bounds__(1024, 1) void c5_eML_blk_kernel_kernel64(
  const boolean_T c5_selectedIndex_data[784], const real_T c5_x1_data[784],
  const int32_T c5_selectedIndex_size[1], boolean_T c5_index_data[784])
{
  int64_T c5_loopEnd;
  uint64_T c5_idx;
  uint64_T c5_threadId;
  uint64_T c5_threadStride;
  int32_T c5_i69;
  c5_threadId = mwGetGlobalThreadIndex();
  c5_threadStride = mwGetTotalThreadsLaunched();
  c5_loopEnd = (int64_T)(c5_selectedIndex_size[0] - 1);
  for (c5_idx = c5_threadId; c5_idx <= (uint64_T)c5_loopEnd; c5_idx +=
       c5_threadStride) {
    c5_i69 = (int32_T)c5_idx;
    c5_index_data[(int32_T)c5_x1_data[c5_i69] - 1] =
      c5_selectedIndex_data[c5_i69];
  }
}

static __global__ __launch_bounds__(1024, 1) void c5_eML_blk_kernel_kernel65(
  const real_T c5_bboxPred_data[3136], const int32_T c5_bboxPred_size[2], const
  int16_T c5_iv1_data[784], const int32_T c5_bboxesX1Y1X2Y2_size[2], const
  int32_T c5_iv1_size[1], real_T c5_bboxesX1Y1X2Y2_data[3136])
{
  int64_T c5_loopEnd;
  uint64_T c5_idx;
  uint64_T c5_threadId;
  uint64_T c5_threadStride;
  uint64_T c5_tmpIndex;
  int32_T c5_i70;
  int32_T c5_i71;
  c5_threadId = mwGetGlobalThreadIndex();
  c5_threadStride = mwGetTotalThreadsLaunched();
  c5_loopEnd = ((int64_T)(c5_iv1_size[0] - 1) + 1L) * 4L - 1L;
  for (c5_idx = c5_threadId; c5_idx <= (uint64_T)c5_loopEnd; c5_idx +=
       c5_threadStride) {
    c5_i71 = (int32_T)(c5_idx % ((uint64_T)(c5_iv1_size[0] - 1) + 1UL));
    c5_tmpIndex = (c5_idx - (uint64_T)c5_i71) / ((uint64_T)(c5_iv1_size[0] - 1)
      + 1UL);
    c5_i70 = (int32_T)c5_tmpIndex;
    c5_bboxesX1Y1X2Y2_data[c5_i71 + c5_bboxesX1Y1X2Y2_size[0] * c5_i70] =
      c5_bboxPred_data[((int32_T)c5_iv1_data[c5_i71] + c5_bboxPred_size[0] *
                        c5_i70) - 1];
  }
}

static __global__ __launch_bounds__(1024, 1) void c5_eML_blk_kernel_kernel66(
  const real32_T c5_scorePred_data[784], const int32_T c5_scorePred_size[1],
  real32_T c5_b_scores_data[784])
{
  int64_T c5_loopEnd;
  uint64_T c5_idx;
  uint64_T c5_threadId;
  uint64_T c5_threadStride;
  int32_T c5_i55;
  c5_threadId = mwGetGlobalThreadIndex();
  c5_threadStride = mwGetTotalThreadsLaunched();
  c5_loopEnd = (int64_T)(c5_scorePred_size[0] - 1);
  for (c5_idx = c5_threadId; c5_idx <= (uint64_T)c5_loopEnd; c5_idx +=
       c5_threadStride) {
    c5_i55 = (int32_T)c5_idx;
    c5_b_scores_data[c5_i55] = c5_scorePred_data[c5_i55];
  }
}

static __global__ __launch_bounds__(1024, 1) void c5_eML_blk_kernel_kernel67(
  const real_T c5_bboxesX1Y1X2Y2_data[3136], const int32_T
  c5_bboxesX1Y1X2Y2_size[2], real_T c5_b_bboxes_data[])
{
  int64_T c5_loopEnd;
  uint64_T c5_idx;
  uint64_T c5_threadId;
  uint64_T c5_threadStride;
  int32_T c5_i11;
  c5_threadId = mwGetGlobalThreadIndex();
  c5_threadStride = mwGetTotalThreadsLaunched();
  c5_loopEnd = (int64_T)(c5_bboxesX1Y1X2Y2_size[0] * 4 - 1);
  for (c5_idx = c5_threadId; c5_idx <= (uint64_T)c5_loopEnd; c5_idx +=
       c5_threadStride) {
    c5_i11 = (int32_T)c5_idx;
    c5_b_bboxes_data[c5_i11] = c5_bboxesX1Y1X2Y2_data[c5_i11];
  }
}

static __global__ __launch_bounds__(1024, 1) void c5_eML_blk_kernel_kernel68(
  const real32_T c5_b_scores_data[784], const int32_T c5_scores_size[1],
  real32_T c5_c_scores_data[])
{
  int64_T c5_loopEnd;
  uint64_T c5_idx;
  uint64_T c5_threadId;
  uint64_T c5_threadStride;
  int32_T c5_i15;
  c5_threadId = mwGetGlobalThreadIndex();
  c5_threadStride = mwGetTotalThreadsLaunched();
  c5_loopEnd = (int64_T)(c5_scores_size[0] - 1);
  for (c5_idx = c5_threadId; c5_idx <= (uint64_T)c5_loopEnd; c5_idx +=
       c5_threadStride) {
    c5_i15 = (int32_T)c5_idx;
    c5_c_scores_data[c5_i15] = c5_b_scores_data[c5_i15];
  }
}

static __global__ __launch_bounds__(512, 1) void
  c5_DeepLearningNetwork_activations_kernel69(const real32_T c5_varargin_1
  [150528], c5_cell_wrap_18 *c5_r)
{
  int32_T c5_i;
  c5_i = (int32_T)mwGetGlobalThreadIndex();
  if (c5_i < 150528) {
    c5_r->f1[c5_i] = c5_varargin_1[c5_i];
  }
}

static __global__ __launch_bounds__(512, 1) void
  c5_DeepLearningNetwork_activations_kernel70(const c5_cell_wrap_18 *c5_r,
  c5_cell_wrap_18 c5_miniBatchT[1])
{
  uint64_T c5_threadId;
  uint64_T c5_tmpIndex;
  int32_T c5_i1;
  int32_T c5_i2;
  int32_T c5_p;
  c5_threadId = mwGetGlobalThreadIndex();
  c5_i1 = (int32_T)(c5_threadId % 224UL);
  c5_tmpIndex = (c5_threadId - (uint64_T)c5_i1) / 224UL;
  c5_i2 = (int32_T)(c5_tmpIndex % 224UL);
  c5_tmpIndex = (c5_tmpIndex - (uint64_T)c5_i2) / 224UL;
  c5_p = (int32_T)c5_tmpIndex;
  if ((c5_p < 3) && (c5_i2 < 224) && (c5_i1 < 224)) {
    c5_miniBatchT[0].f1[(c5_i1 + 224 * c5_i2) + 50176 * c5_p] = c5_r->f1[(c5_i2
      + 224 * c5_i1) + 50176 * c5_p];
  }
}

static __global__ __launch_bounds__(512, 1) void
  c5_DeepLearningNetwork_activations_kernel71(const real32_T c5_outMiniBatch
  [4704], real32_T c5_b_out[4704])
{
  uint64_T c5_threadId;
  uint64_T c5_tmpIndex;
  int32_T c5_i3;
  int32_T c5_i4;
  int32_T c5_p;
  c5_threadId = mwGetGlobalThreadIndex();
  c5_i3 = (int32_T)(c5_threadId % 14UL);
  c5_tmpIndex = (c5_threadId - (uint64_T)c5_i3) / 14UL;
  c5_i4 = (int32_T)(c5_tmpIndex % 14UL);
  c5_tmpIndex = (c5_tmpIndex - (uint64_T)c5_i4) / 14UL;
  c5_p = (int32_T)c5_tmpIndex;
  if ((c5_p < 24) && (c5_i4 < 14) && (c5_i3 < 14)) {
    c5_b_out[(c5_i3 + 14 * c5_i4) + 196 * c5_p] = c5_outMiniBatch[(c5_i4 + 14 *
      c5_i3) + 196 * c5_p];
  }
}

static __device__ real32_T c5_callFcn_device(real32_T c5_input1, real32_T
  c5_input2)
{
  return fmaxf(c5_input1, c5_input2);
}

static __device__ real32_T c5_b_callFcn_device(real32_T c5_input1, real32_T
  c5_input2)
{
  return fminf(c5_input1, c5_input2);
}

void c5_yolov2ResNet50VehicleExample0_LaneDetection0::allocate()
{
  int32_T c5_idx;
  this->targetImpl->allocate(802816, 4);
  for (c5_idx = 0; c5_idx < 57; c5_idx++) {
    this->layers[c5_idx]->allocate();
  }

  (static_cast<MWTensor<real32_T> *>(this->inputTensors[0]))->setData
    (this->layers[0]->getLayerOutput(0));
}

void c5_yolov2ResNet50VehicleExample0_LaneDetection0::postsetup()
{
  this->targetImpl->postSetup(this->layers, this->numLayers);
}

c5_yolov2ResNet50VehicleExample0_LaneDetection0::
  c5_yolov2ResNet50VehicleExample0_LaneDetection0()
{
  this->numLayers = 57;
  this->isInitialized = false;
  this->targetImpl = 0;
  this->layers[0] = new MWInputLayer;
  this->layers[0]->setName("input_1");
  this->layers[1] = new MWElementwiseAffineLayer;
  this->layers[1]->setName("input_1_normalization");
  this->layers[1]->setInPlaceIndex(0, 0);
  this->layers[2] = new MWFusedConvReLULayer;
  this->layers[2]->setName("conv1_activation_1_relu");
  this->layers[3] = new MWMaxPoolingLayer;
  this->layers[3]->setName("max_pooling2d_1");
  this->layers[4] = new MWFusedConvReLULayer;
  this->layers[4]->setName("res2a_branch2a_activation_2_relu");
  this->layers[5] = new MWFusedConvReLULayer;
  this->layers[5]->setName("res2a_branch2b_activation_3_relu");
  this->layers[6] = new MWConvLayer;
  this->layers[6]->setName("res2a_branch1");
  this->layers[7] = new MWFusedConvReLULayer;
  this->layers[7]->setName("res2a_branch2c_activation_4_relu");
  this->layers[7]->setInPlaceIndex(0, 1);
  this->layers[8] = new MWFusedConvReLULayer;
  this->layers[8]->setName("res2b_branch2a_activation_5_relu");
  this->layers[9] = new MWFusedConvReLULayer;
  this->layers[9]->setName("res2b_branch2b_activation_6_relu");
  this->layers[10] = new MWFusedConvReLULayer;
  this->layers[10]->setName("res2b_branch2c_activation_7_relu");
  this->layers[10]->setInPlaceIndex(0, 1);
  this->layers[11] = new MWFusedConvReLULayer;
  this->layers[11]->setName("res2c_branch2a_activation_8_relu");
  this->layers[12] = new MWFusedConvReLULayer;
  this->layers[12]->setName("res2c_branch2b_activation_9_relu");
  this->layers[13] = new MWMaxPoolingLayer;
  this->layers[13]->setName("downsample_add_3");
  this->layers[14] = new MWFusedConvReLULayer;
  this->layers[14]->setName("res2c_branch2c_activation_10_relu");
  this->layers[14]->setInPlaceIndex(0, 1);
  this->layers[15] = new MWFusedConvReLULayer;
  this->layers[15]->setName("res3a_branch2a_activation_11_relu");
  this->layers[16] = new MWFusedConvReLULayer;
  this->layers[16]->setName("res3a_branch2b_activation_12_relu");
  this->layers[17] = new MWConvLayer;
  this->layers[17]->setName("res3a_branch1");
  this->layers[18] = new MWFusedConvReLULayer;
  this->layers[18]->setName("res3a_branch2c_activation_13_relu");
  this->layers[18]->setInPlaceIndex(0, 1);
  this->layers[19] = new MWFusedConvReLULayer;
  this->layers[19]->setName("res3b_branch2a_activation_14_relu");
  this->layers[20] = new MWFusedConvReLULayer;
  this->layers[20]->setName("res3b_branch2b_activation_15_relu");
  this->layers[21] = new MWFusedConvReLULayer;
  this->layers[21]->setName("res3b_branch2c_activation_16_relu");
  this->layers[21]->setInPlaceIndex(0, 1);
  this->layers[22] = new MWFusedConvReLULayer;
  this->layers[22]->setName("res3c_branch2a_activation_17_relu");
  this->layers[23] = new MWFusedConvReLULayer;
  this->layers[23]->setName("res3c_branch2b_activation_18_relu");
  this->layers[24] = new MWFusedConvReLULayer;
  this->layers[24]->setName("res3c_branch2c_activation_19_relu");
  this->layers[24]->setInPlaceIndex(0, 1);
  this->layers[25] = new MWFusedConvReLULayer;
  this->layers[25]->setName("res3d_branch2a_activation_20_relu");
  this->layers[26] = new MWFusedConvReLULayer;
  this->layers[26]->setName("res3d_branch2b_activation_21_relu");
  this->layers[27] = new MWMaxPoolingLayer;
  this->layers[27]->setName("downsample_add_7");
  this->layers[28] = new MWFusedConvReLULayer;
  this->layers[28]->setName("res3d_branch2c_activation_22_relu");
  this->layers[28]->setInPlaceIndex(0, 1);
  this->layers[29] = new MWFusedConvReLULayer;
  this->layers[29]->setName("res4a_branch2a_activation_23_relu");
  this->layers[30] = new MWFusedConvReLULayer;
  this->layers[30]->setName("res4a_branch2b_activation_24_relu");
  this->layers[31] = new MWConvLayer;
  this->layers[31]->setName("res4a_branch1");
  this->layers[32] = new MWFusedConvReLULayer;
  this->layers[32]->setName("res4a_branch2c_activation_25_relu");
  this->layers[32]->setInPlaceIndex(0, 1);
  this->layers[33] = new MWFusedConvReLULayer;
  this->layers[33]->setName("res4b_branch2a_activation_26_relu");
  this->layers[34] = new MWFusedConvReLULayer;
  this->layers[34]->setName("res4b_branch2b_activation_27_relu");
  this->layers[35] = new MWFusedConvReLULayer;
  this->layers[35]->setName("res4b_branch2c_activation_28_relu");
  this->layers[35]->setInPlaceIndex(0, 1);
  this->layers[36] = new MWFusedConvReLULayer;
  this->layers[36]->setName("res4c_branch2a_activation_29_relu");
  this->layers[37] = new MWFusedConvReLULayer;
  this->layers[37]->setName("res4c_branch2b_activation_30_relu");
  this->layers[38] = new MWFusedConvReLULayer;
  this->layers[38]->setName("res4c_branch2c_activation_31_relu");
  this->layers[38]->setInPlaceIndex(0, 1);
  this->layers[39] = new MWFusedConvReLULayer;
  this->layers[39]->setName("res4d_branch2a_activation_32_relu");
  this->layers[40] = new MWFusedConvReLULayer;
  this->layers[40]->setName("res4d_branch2b_activation_33_relu");
  this->layers[41] = new MWFusedConvReLULayer;
  this->layers[41]->setName("res4d_branch2c_activation_34_relu");
  this->layers[41]->setInPlaceIndex(0, 1);
  this->layers[42] = new MWFusedConvReLULayer;
  this->layers[42]->setName("res4e_branch2a_activation_35_relu");
  this->layers[43] = new MWFusedConvReLULayer;
  this->layers[43]->setName("res4e_branch2b_activation_36_relu");
  this->layers[44] = new MWFusedConvReLULayer;
  this->layers[44]->setName("res4e_branch2c_activation_37_relu");
  this->layers[44]->setInPlaceIndex(0, 1);
  this->layers[45] = new MWFusedConvReLULayer;
  this->layers[45]->setName("res4f_branch2a_activation_38_relu");
  this->layers[46] = new MWFusedConvReLULayer;
  this->layers[46]->setName("res4f_branch2b_activation_39_relu");
  this->layers[47] = new MWFusedConvReLULayer;
  this->layers[47]->setName("res4f_branch2c_activation_40_relu");
  this->layers[47]->setInPlaceIndex(0, 1);
  this->layers[48] = new MWFusedConvReLULayer;
  this->layers[48]->setName("yolov2Conv1_yolov2Relu1");
  this->layers[49] = new MWFusedConvReLULayer;
  this->layers[49]->setName("yolov2Conv2_yolov2Relu2");
  this->layers[50] = new MWConvLayer;
  this->layers[50]->setName("yolov2ClassConv");
  this->layers[51] = new MWYoloExtractionLayer;
  this->layers[51]->setName("YOLOv2ExtractionLayer");
  this->layers[52] = new MWSigmoidLayer;
  this->layers[52]->setName("YOLOSigmoidLayer");
  this->layers[53] = new MWExponentialLayer;
  this->layers[53]->setName("YOLOv2ExponentialLayer");
  this->layers[54] = new MWYoloSoftmaxLayer;
  this->layers[54]->setName("YOLOv2SoftmaxLayer");
  this->layers[55] = new MWConcatenationLayer;
  this->layers[55]->setName("YOLOv2ConcatenationLayer");
  this->layers[56] = new MWOutputLayer;
  this->layers[56]->setName("yolov2OutputLayer");
  this->layers[56]->setInPlaceIndex(0, 0);
  this->targetImpl = new MWTargetNetworkImpl;
  this->inputTensors[0] = new MWTensor<real32_T>;
  this->inputTensors[0]->setHeight(224);
  this->inputTensors[0]->setWidth(224);
  this->inputTensors[0]->setChannels(3);
  this->inputTensors[0]->setBatchSize(1);
  this->inputTensors[0]->setSequenceLength(1);
}

void c5_yolov2ResNet50VehicleExample0_LaneDetection0::deallocate()
{
  int32_T c5_idx;
  this->targetImpl->deallocate();
  for (c5_idx = 0; c5_idx < 57; c5_idx++) {
    this->layers[c5_idx]->deallocate();
  }
}

void c5_yolov2ResNet50VehicleExample0_LaneDetection0::setSize()
{
  int32_T c5_idx;
  for (c5_idx = 0; c5_idx < 57; c5_idx++) {
    this->layers[c5_idx]->propagateSize();
  }

  this->allocate();
  this->postsetup();
}

void c5_yolov2ResNet50VehicleExample0_LaneDetection0::resetState()
{
}

void c5_yolov2ResNet50VehicleExample0_LaneDetection0::setup()
{
  if (this->isInitialized) {
    this->resetState();
  } else {
    this->isInitialized = true;
    this->targetImpl->preSetup();
    this->targetImpl->setAutoTune(true);
    (static_cast<MWInputLayer *>(this->layers[0]))->createInputLayer
      (this->targetImpl, this->inputTensors[0], 224, 224, 3, 0, "", 0);
    (static_cast<MWElementwiseAffineLayer *>(this->layers[1]))
      ->createElementwiseAffineLayer(this->targetImpl, this->layers[0]
      ->getOutputTensor(0), 1, 1, 3, 1, 1, 3, false, 1, 1,
      "/home/hmcl/ADcontrol/slprj/_sfprj/LaneDetection/_self/sfun/src/cnn_yolov2ResNet50VehicleExample0_LaneDetection0_input_1_scale.bi"
      "n",
      "/home/hmcl/ADcontrol/slprj/_sfprj/LaneDetection/_self/sfun/src/cnn_yolov2ResNet50VehicleExample0_LaneDetection0_input_1_offset.b"
      "in", 0);
    (static_cast<MWFusedConvReLULayer *>(this->layers[2]))
      ->createFusedConvReLULayer(this->targetImpl, 1, this->layers[1]
      ->getOutputTensor(0), 7, 7, 3, 64, 2, 2, 3, 3, 3, 3, 1, 1, 1,
      "/home/hmcl/ADcontrol/slprj/_sfprj/LaneDetection/_self/sfun/src/cnn_yolov2ResNet50VehicleExample0_LaneDetection0_conv1_w.bin",
      "/home/hmcl/ADcontrol/slprj/_sfprj/LaneDetection/_self/sfun/src/cnn_yolov2ResNet50VehicleExample0_LaneDetection0_conv1_b.bin",
      1);
    (static_cast<MWMaxPoolingLayer *>(this->layers[3]))->createMaxPoolingLayer
      (this->targetImpl, this->layers[2]->getOutputTensor(0), 3, 3, 2, 2, 0, 0,
       0, 0, 0, 1, 0);
    (static_cast<MWFusedConvReLULayer *>(this->layers[4]))
      ->createFusedConvReLULayer(this->targetImpl, 1, this->layers[3]
      ->getOutputTensor(0), 1, 1, 64, 64, 1, 1, 0, 0, 0, 0, 1, 1, 1,
      "/home/hmcl/ADcontrol/slprj/_sfprj/LaneDetection/_self/sfun/src/cnn_yolov2ResNet50VehicleExample0_LaneDetection0_res2a_branch2a_w"
      ".bin",
      "/home/hmcl/ADcontrol/slprj/_sfprj/LaneDetection/_self/sfun/src/cnn_yolov2ResNet50VehicleExample0_LaneDetection0_res2a_branch2a_b"
      ".bin", 1);
    (static_cast<MWFusedConvReLULayer *>(this->layers[5]))
      ->createFusedConvReLULayer(this->targetImpl, 1, this->layers[4]
      ->getOutputTensor(0), 3, 3, 64, 64, 1, 1, 1, 1, 1, 1, 1, 1, 1,
      "/home/hmcl/ADcontrol/slprj/_sfprj/LaneDetection/_self/sfun/src/cnn_yolov2ResNet50VehicleExample0_LaneDetection0_res2a_branch2b_w"
      ".bin",
      "/home/hmcl/ADcontrol/slprj/_sfprj/LaneDetection/_self/sfun/src/cnn_yolov2ResNet50VehicleExample0_LaneDetection0_res2a_branch2b_b"
      ".bin", 2);
    (static_cast<MWConvLayer *>(this->layers[6]))->createConvLayer
      (this->targetImpl, this->layers[3]->getOutputTensor(0), 1, 1, 64, 256, 1,
       1, 0, 0, 0, 0, 1, 1, 1,
       "/home/hmcl/ADcontrol/slprj/_sfprj/LaneDetection/_self/sfun/src/cnn_yolov2ResNet50VehicleExample0_LaneDetection0_res2a_branch1_w."
       "bin",
       "/home/hmcl/ADcontrol/slprj/_sfprj/LaneDetection/_self/sfun/src/cnn_yolov2ResNet50VehicleExample0_LaneDetection0_res2a_branch1_b."
       "bin", 1);
    (static_cast<MWFusedConvReLULayer *>(this->layers[7]))
      ->createFusedConvReLULayer(this->targetImpl, 2, this->layers[5]
      ->getOutputTensor(0), this->layers[6]->getOutputTensor(0), 1, 1, 64, 256,
      1, 1, 0, 0, 0, 0, 1, 1, 1,
      "/home/hmcl/ADcontrol/slprj/_sfprj/LaneDetection/_self/sfun/src/cnn_yolov2ResNet50VehicleExample0_LaneDetection0_res2a_branch2c_w"
      ".bin",
      "/home/hmcl/ADcontrol/slprj/_sfprj/LaneDetection/_self/sfun/src/cnn_yolov2ResNet50VehicleExample0_LaneDetection0_res2a_branch2c_b"
      ".bin", 1);
    (static_cast<MWFusedConvReLULayer *>(this->layers[8]))
      ->createFusedConvReLULayer(this->targetImpl, 1, this->layers[7]
      ->getOutputTensor(0), 1, 1, 256, 64, 1, 1, 0, 0, 0, 0, 1, 1, 1,
      "/home/hmcl/ADcontrol/slprj/_sfprj/LaneDetection/_self/sfun/src/cnn_yolov2ResNet50VehicleExample0_LaneDetection0_res2b_branch2a_w"
      ".bin",
      "/home/hmcl/ADcontrol/slprj/_sfprj/LaneDetection/_self/sfun/src/cnn_yolov2ResNet50VehicleExample0_LaneDetection0_res2b_branch2a_b"
      ".bin", 0);
    (static_cast<MWFusedConvReLULayer *>(this->layers[9]))
      ->createFusedConvReLULayer(this->targetImpl, 1, this->layers[8]
      ->getOutputTensor(0), 3, 3, 64, 64, 1, 1, 1, 1, 1, 1, 1, 1, 1,
      "/home/hmcl/ADcontrol/slprj/_sfprj/LaneDetection/_self/sfun/src/cnn_yolov2ResNet50VehicleExample0_LaneDetection0_res2b_branch2b_w"
      ".bin",
      "/home/hmcl/ADcontrol/slprj/_sfprj/LaneDetection/_self/sfun/src/cnn_yolov2ResNet50VehicleExample0_LaneDetection0_res2b_branch2b_b"
      ".bin", 2);
    (static_cast<MWFusedConvReLULayer *>(this->layers[10]))
      ->createFusedConvReLULayer(this->targetImpl, 2, this->layers[9]
      ->getOutputTensor(0), this->layers[7]->getOutputTensor(0), 1, 1, 64, 256,
      1, 1, 0, 0, 0, 0, 1, 1, 1,
      "/home/hmcl/ADcontrol/slprj/_sfprj/LaneDetection/_self/sfun/src/cnn_yolov2ResNet50VehicleExample0_LaneDetection0_res2b_branch2c_w"
      ".bin",
      "/home/hmcl/ADcontrol/slprj/_sfprj/LaneDetection/_self/sfun/src/cnn_yolov2ResNet50VehicleExample0_LaneDetection0_res2b_branch2c_b"
      ".bin", 1);
    (static_cast<MWFusedConvReLULayer *>(this->layers[11]))
      ->createFusedConvReLULayer(this->targetImpl, 1, this->layers[10]
      ->getOutputTensor(0), 1, 1, 256, 64, 1, 1, 0, 0, 0, 0, 1, 1, 1,
      "/home/hmcl/ADcontrol/slprj/_sfprj/LaneDetection/_self/sfun/src/cnn_yolov2ResNet50VehicleExample0_LaneDetection0_res2c_branch2a_w"
      ".bin",
      "/home/hmcl/ADcontrol/slprj/_sfprj/LaneDetection/_self/sfun/src/cnn_yolov2ResNet50VehicleExample0_LaneDetection0_res2c_branch2a_b"
      ".bin", 0);
    (static_cast<MWFusedConvReLULayer *>(this->layers[12]))
      ->createFusedConvReLULayer(this->targetImpl, 1, this->layers[11]
      ->getOutputTensor(0), 3, 3, 64, 64, 2, 2, 1, 1, 1, 1, 1, 1, 1,
      "/home/hmcl/ADcontrol/slprj/_sfprj/LaneDetection/_self/sfun/src/cnn_yolov2ResNet50VehicleExample0_LaneDetection0_res2c_branch2b_w"
      ".bin",
      "/home/hmcl/ADcontrol/slprj/_sfprj/LaneDetection/_self/sfun/src/cnn_yolov2ResNet50VehicleExample0_LaneDetection0_res2c_branch2b_b"
      ".bin", 2);
    (static_cast<MWMaxPoolingLayer *>(this->layers[13]))->createMaxPoolingLayer
      (this->targetImpl, this->layers[10]->getOutputTensor(0), 1, 1, 2, 2, 0, 0,
       0, 0, 0, 1, 0);
    (static_cast<MWFusedConvReLULayer *>(this->layers[14]))
      ->createFusedConvReLULayer(this->targetImpl, 2, this->layers[12]
      ->getOutputTensor(0), this->layers[13]->getOutputTensor(0), 1, 1, 64, 256,
      1, 1, 0, 0, 0, 0, 1, 1, 1,
      "/home/hmcl/ADcontrol/slprj/_sfprj/LaneDetection/_self/sfun/src/cnn_yolov2ResNet50VehicleExample0_LaneDetection0_res2c_branch2c_w"
      ".bin",
      "/home/hmcl/ADcontrol/slprj/_sfprj/LaneDetection/_self/sfun/src/cnn_yolov2ResNet50VehicleExample0_LaneDetection0_res2c_branch2c_b"
      ".bin", 0);
    (static_cast<MWFusedConvReLULayer *>(this->layers[15]))
      ->createFusedConvReLULayer(this->targetImpl, 1, this->layers[14]
      ->getOutputTensor(0), 1, 1, 256, 128, 1, 1, 0, 0, 0, 0, 1, 1, 1,
      "/home/hmcl/ADcontrol/slprj/_sfprj/LaneDetection/_self/sfun/src/cnn_yolov2ResNet50VehicleExample0_LaneDetection0_res3a_branch2a_w"
      ".bin",
      "/home/hmcl/ADcontrol/slprj/_sfprj/LaneDetection/_self/sfun/src/cnn_yolov2ResNet50VehicleExample0_LaneDetection0_res3a_branch2a_b"
      ".bin", 1);
    (static_cast<MWFusedConvReLULayer *>(this->layers[16]))
      ->createFusedConvReLULayer(this->targetImpl, 1, this->layers[15]
      ->getOutputTensor(0), 3, 3, 128, 128, 1, 1, 1, 1, 1, 1, 1, 1, 1,
      "/home/hmcl/ADcontrol/slprj/_sfprj/LaneDetection/_self/sfun/src/cnn_yolov2ResNet50VehicleExample0_LaneDetection0_res3a_branch2b_w"
      ".bin",
      "/home/hmcl/ADcontrol/slprj/_sfprj/LaneDetection/_self/sfun/src/cnn_yolov2ResNet50VehicleExample0_LaneDetection0_res3a_branch2b_b"
      ".bin", 2);
    (static_cast<MWConvLayer *>(this->layers[17]))->createConvLayer
      (this->targetImpl, this->layers[14]->getOutputTensor(0), 1, 1, 256, 512, 1,
       1, 0, 0, 0, 0, 1, 1, 1,
       "/home/hmcl/ADcontrol/slprj/_sfprj/LaneDetection/_self/sfun/src/cnn_yolov2ResNet50VehicleExample0_LaneDetection0_res3a_branch1_w."
       "bin",
       "/home/hmcl/ADcontrol/slprj/_sfprj/LaneDetection/_self/sfun/src/cnn_yolov2ResNet50VehicleExample0_LaneDetection0_res3a_branch1_b."
       "bin", 1);
    (static_cast<MWFusedConvReLULayer *>(this->layers[18]))
      ->createFusedConvReLULayer(this->targetImpl, 2, this->layers[16]
      ->getOutputTensor(0), this->layers[17]->getOutputTensor(0), 1, 1, 128, 512,
      1, 1, 0, 0, 0, 0, 1, 1, 1,
      "/home/hmcl/ADcontrol/slprj/_sfprj/LaneDetection/_self/sfun/src/cnn_yolov2ResNet50VehicleExample0_LaneDetection0_res3a_branch2c_w"
      ".bin",
      "/home/hmcl/ADcontrol/slprj/_sfprj/LaneDetection/_self/sfun/src/cnn_yolov2ResNet50VehicleExample0_LaneDetection0_res3a_branch2c_b"
      ".bin", 1);
    (static_cast<MWFusedConvReLULayer *>(this->layers[19]))
      ->createFusedConvReLULayer(this->targetImpl, 1, this->layers[18]
      ->getOutputTensor(0), 1, 1, 512, 128, 1, 1, 0, 0, 0, 0, 1, 1, 1,
      "/home/hmcl/ADcontrol/slprj/_sfprj/LaneDetection/_self/sfun/src/cnn_yolov2ResNet50VehicleExample0_LaneDetection0_res3b_branch2a_w"
      ".bin",
      "/home/hmcl/ADcontrol/slprj/_sfprj/LaneDetection/_self/sfun/src/cnn_yolov2ResNet50VehicleExample0_LaneDetection0_res3b_branch2a_b"
      ".bin", 0);
    (static_cast<MWFusedConvReLULayer *>(this->layers[20]))
      ->createFusedConvReLULayer(this->targetImpl, 1, this->layers[19]
      ->getOutputTensor(0), 3, 3, 128, 128, 1, 1, 1, 1, 1, 1, 1, 1, 1,
      "/home/hmcl/ADcontrol/slprj/_sfprj/LaneDetection/_self/sfun/src/cnn_yolov2ResNet50VehicleExample0_LaneDetection0_res3b_branch2b_w"
      ".bin",
      "/home/hmcl/ADcontrol/slprj/_sfprj/LaneDetection/_self/sfun/src/cnn_yolov2ResNet50VehicleExample0_LaneDetection0_res3b_branch2b_b"
      ".bin", 2);
    (static_cast<MWFusedConvReLULayer *>(this->layers[21]))
      ->createFusedConvReLULayer(this->targetImpl, 2, this->layers[20]
      ->getOutputTensor(0), this->layers[18]->getOutputTensor(0), 1, 1, 128, 512,
      1, 1, 0, 0, 0, 0, 1, 1, 1,
      "/home/hmcl/ADcontrol/slprj/_sfprj/LaneDetection/_self/sfun/src/cnn_yolov2ResNet50VehicleExample0_LaneDetection0_res3b_branch2c_w"
      ".bin",
      "/home/hmcl/ADcontrol/slprj/_sfprj/LaneDetection/_self/sfun/src/cnn_yolov2ResNet50VehicleExample0_LaneDetection0_res3b_branch2c_b"
      ".bin", 1);
    (static_cast<MWFusedConvReLULayer *>(this->layers[22]))
      ->createFusedConvReLULayer(this->targetImpl, 1, this->layers[21]
      ->getOutputTensor(0), 1, 1, 512, 128, 1, 1, 0, 0, 0, 0, 1, 1, 1,
      "/home/hmcl/ADcontrol/slprj/_sfprj/LaneDetection/_self/sfun/src/cnn_yolov2ResNet50VehicleExample0_LaneDetection0_res3c_branch2a_w"
      ".bin",
      "/home/hmcl/ADcontrol/slprj/_sfprj/LaneDetection/_self/sfun/src/cnn_yolov2ResNet50VehicleExample0_LaneDetection0_res3c_branch2a_b"
      ".bin", 0);
    (static_cast<MWFusedConvReLULayer *>(this->layers[23]))
      ->createFusedConvReLULayer(this->targetImpl, 1, this->layers[22]
      ->getOutputTensor(0), 3, 3, 128, 128, 1, 1, 1, 1, 1, 1, 1, 1, 1,
      "/home/hmcl/ADcontrol/slprj/_sfprj/LaneDetection/_self/sfun/src/cnn_yolov2ResNet50VehicleExample0_LaneDetection0_res3c_branch2b_w"
      ".bin",
      "/home/hmcl/ADcontrol/slprj/_sfprj/LaneDetection/_self/sfun/src/cnn_yolov2ResNet50VehicleExample0_LaneDetection0_res3c_branch2b_b"
      ".bin", 2);
    (static_cast<MWFusedConvReLULayer *>(this->layers[24]))
      ->createFusedConvReLULayer(this->targetImpl, 2, this->layers[23]
      ->getOutputTensor(0), this->layers[21]->getOutputTensor(0), 1, 1, 128, 512,
      1, 1, 0, 0, 0, 0, 1, 1, 1,
      "/home/hmcl/ADcontrol/slprj/_sfprj/LaneDetection/_self/sfun/src/cnn_yolov2ResNet50VehicleExample0_LaneDetection0_res3c_branch2c_w"
      ".bin",
      "/home/hmcl/ADcontrol/slprj/_sfprj/LaneDetection/_self/sfun/src/cnn_yolov2ResNet50VehicleExample0_LaneDetection0_res3c_branch2c_b"
      ".bin", 1);
    (static_cast<MWFusedConvReLULayer *>(this->layers[25]))
      ->createFusedConvReLULayer(this->targetImpl, 1, this->layers[24]
      ->getOutputTensor(0), 1, 1, 512, 128, 1, 1, 0, 0, 0, 0, 1, 1, 1,
      "/home/hmcl/ADcontrol/slprj/_sfprj/LaneDetection/_self/sfun/src/cnn_yolov2ResNet50VehicleExample0_LaneDetection0_res3d_branch2a_w"
      ".bin",
      "/home/hmcl/ADcontrol/slprj/_sfprj/LaneDetection/_self/sfun/src/cnn_yolov2ResNet50VehicleExample0_LaneDetection0_res3d_branch2a_b"
      ".bin", 0);
    (static_cast<MWFusedConvReLULayer *>(this->layers[26]))
      ->createFusedConvReLULayer(this->targetImpl, 1, this->layers[25]
      ->getOutputTensor(0), 3, 3, 128, 128, 2, 2, 1, 1, 1, 1, 1, 1, 1,
      "/home/hmcl/ADcontrol/slprj/_sfprj/LaneDetection/_self/sfun/src/cnn_yolov2ResNet50VehicleExample0_LaneDetection0_res3d_branch2b_w"
      ".bin",
      "/home/hmcl/ADcontrol/slprj/_sfprj/LaneDetection/_self/sfun/src/cnn_yolov2ResNet50VehicleExample0_LaneDetection0_res3d_branch2b_b"
      ".bin", 2);
    (static_cast<MWMaxPoolingLayer *>(this->layers[27]))->createMaxPoolingLayer
      (this->targetImpl, this->layers[24]->getOutputTensor(0), 1, 1, 2, 2, 0, 0,
       0, 0, 0, 1, 0);
    (static_cast<MWFusedConvReLULayer *>(this->layers[28]))
      ->createFusedConvReLULayer(this->targetImpl, 2, this->layers[26]
      ->getOutputTensor(0), this->layers[27]->getOutputTensor(0), 1, 1, 128, 512,
      1, 1, 0, 0, 0, 0, 1, 1, 1,
      "/home/hmcl/ADcontrol/slprj/_sfprj/LaneDetection/_self/sfun/src/cnn_yolov2ResNet50VehicleExample0_LaneDetection0_res3d_branch2c_w"
      ".bin",
      "/home/hmcl/ADcontrol/slprj/_sfprj/LaneDetection/_self/sfun/src/cnn_yolov2ResNet50VehicleExample0_LaneDetection0_res3d_branch2c_b"
      ".bin", 0);
    (static_cast<MWFusedConvReLULayer *>(this->layers[29]))
      ->createFusedConvReLULayer(this->targetImpl, 1, this->layers[28]
      ->getOutputTensor(0), 1, 1, 512, 256, 1, 1, 0, 0, 0, 0, 1, 1, 1,
      "/home/hmcl/ADcontrol/slprj/_sfprj/LaneDetection/_self/sfun/src/cnn_yolov2ResNet50VehicleExample0_LaneDetection0_res4a_branch2a_w"
      ".bin",
      "/home/hmcl/ADcontrol/slprj/_sfprj/LaneDetection/_self/sfun/src/cnn_yolov2ResNet50VehicleExample0_LaneDetection0_res4a_branch2a_b"
      ".bin", 1);
    (static_cast<MWFusedConvReLULayer *>(this->layers[30]))
      ->createFusedConvReLULayer(this->targetImpl, 1, this->layers[29]
      ->getOutputTensor(0), 3, 3, 256, 256, 1, 1, 1, 1, 1, 1, 1, 1, 1,
      "/home/hmcl/ADcontrol/slprj/_sfprj/LaneDetection/_self/sfun/src/cnn_yolov2ResNet50VehicleExample0_LaneDetection0_res4a_branch2b_w"
      ".bin",
      "/home/hmcl/ADcontrol/slprj/_sfprj/LaneDetection/_self/sfun/src/cnn_yolov2ResNet50VehicleExample0_LaneDetection0_res4a_branch2b_b"
      ".bin", 2);
    (static_cast<MWConvLayer *>(this->layers[31]))->createConvLayer
      (this->targetImpl, this->layers[28]->getOutputTensor(0), 1, 1, 512, 1024,
       1, 1, 0, 0, 0, 0, 1, 1, 1,
       "/home/hmcl/ADcontrol/slprj/_sfprj/LaneDetection/_self/sfun/src/cnn_yolov2ResNet50VehicleExample0_LaneDetection0_res4a_branch1_w."
       "bin",
       "/home/hmcl/ADcontrol/slprj/_sfprj/LaneDetection/_self/sfun/src/cnn_yolov2ResNet50VehicleExample0_LaneDetection0_res4a_branch1_b."
       "bin", 1);
    (static_cast<MWFusedConvReLULayer *>(this->layers[32]))
      ->createFusedConvReLULayer(this->targetImpl, 2, this->layers[30]
      ->getOutputTensor(0), this->layers[31]->getOutputTensor(0), 1, 1, 256,
      1024, 1, 1, 0, 0, 0, 0, 1, 1, 1,
      "/home/hmcl/ADcontrol/slprj/_sfprj/LaneDetection/_self/sfun/src/cnn_yolov2ResNet50VehicleExample0_LaneDetection0_res4a_branch2c_w"
      ".bin",
      "/home/hmcl/ADcontrol/slprj/_sfprj/LaneDetection/_self/sfun/src/cnn_yolov2ResNet50VehicleExample0_LaneDetection0_res4a_branch2c_b"
      ".bin", 1);
    (static_cast<MWFusedConvReLULayer *>(this->layers[33]))
      ->createFusedConvReLULayer(this->targetImpl, 1, this->layers[32]
      ->getOutputTensor(0), 1, 1, 1024, 256, 1, 1, 0, 0, 0, 0, 1, 1, 1,
      "/home/hmcl/ADcontrol/slprj/_sfprj/LaneDetection/_self/sfun/src/cnn_yolov2ResNet50VehicleExample0_LaneDetection0_res4b_branch2a_w"
      ".bin",
      "/home/hmcl/ADcontrol/slprj/_sfprj/LaneDetection/_self/sfun/src/cnn_yolov2ResNet50VehicleExample0_LaneDetection0_res4b_branch2a_b"
      ".bin", 0);
    (static_cast<MWFusedConvReLULayer *>(this->layers[34]))
      ->createFusedConvReLULayer(this->targetImpl, 1, this->layers[33]
      ->getOutputTensor(0), 3, 3, 256, 256, 1, 1, 1, 1, 1, 1, 1, 1, 1,
      "/home/hmcl/ADcontrol/slprj/_sfprj/LaneDetection/_self/sfun/src/cnn_yolov2ResNet50VehicleExample0_LaneDetection0_res4b_branch2b_w"
      ".bin",
      "/home/hmcl/ADcontrol/slprj/_sfprj/LaneDetection/_self/sfun/src/cnn_yolov2ResNet50VehicleExample0_LaneDetection0_res4b_branch2b_b"
      ".bin", 2);
    (static_cast<MWFusedConvReLULayer *>(this->layers[35]))
      ->createFusedConvReLULayer(this->targetImpl, 2, this->layers[34]
      ->getOutputTensor(0), this->layers[32]->getOutputTensor(0), 1, 1, 256,
      1024, 1, 1, 0, 0, 0, 0, 1, 1, 1,
      "/home/hmcl/ADcontrol/slprj/_sfprj/LaneDetection/_self/sfun/src/cnn_yolov2ResNet50VehicleExample0_LaneDetection0_res4b_branch2c_w"
      ".bin",
      "/home/hmcl/ADcontrol/slprj/_sfprj/LaneDetection/_self/sfun/src/cnn_yolov2ResNet50VehicleExample0_LaneDetection0_res4b_branch2c_b"
      ".bin", 1);
    (static_cast<MWFusedConvReLULayer *>(this->layers[36]))
      ->createFusedConvReLULayer(this->targetImpl, 1, this->layers[35]
      ->getOutputTensor(0), 1, 1, 1024, 256, 1, 1, 0, 0, 0, 0, 1, 1, 1,
      "/home/hmcl/ADcontrol/slprj/_sfprj/LaneDetection/_self/sfun/src/cnn_yolov2ResNet50VehicleExample0_LaneDetection0_res4c_branch2a_w"
      ".bin",
      "/home/hmcl/ADcontrol/slprj/_sfprj/LaneDetection/_self/sfun/src/cnn_yolov2ResNet50VehicleExample0_LaneDetection0_res4c_branch2a_b"
      ".bin", 0);
    (static_cast<MWFusedConvReLULayer *>(this->layers[37]))
      ->createFusedConvReLULayer(this->targetImpl, 1, this->layers[36]
      ->getOutputTensor(0), 3, 3, 256, 256, 1, 1, 1, 1, 1, 1, 1, 1, 1,
      "/home/hmcl/ADcontrol/slprj/_sfprj/LaneDetection/_self/sfun/src/cnn_yolov2ResNet50VehicleExample0_LaneDetection0_res4c_branch2b_w"
      ".bin",
      "/home/hmcl/ADcontrol/slprj/_sfprj/LaneDetection/_self/sfun/src/cnn_yolov2ResNet50VehicleExample0_LaneDetection0_res4c_branch2b_b"
      ".bin", 2);
    (static_cast<MWFusedConvReLULayer *>(this->layers[38]))
      ->createFusedConvReLULayer(this->targetImpl, 2, this->layers[37]
      ->getOutputTensor(0), this->layers[35]->getOutputTensor(0), 1, 1, 256,
      1024, 1, 1, 0, 0, 0, 0, 1, 1, 1,
      "/home/hmcl/ADcontrol/slprj/_sfprj/LaneDetection/_self/sfun/src/cnn_yolov2ResNet50VehicleExample0_LaneDetection0_res4c_branch2c_w"
      ".bin",
      "/home/hmcl/ADcontrol/slprj/_sfprj/LaneDetection/_self/sfun/src/cnn_yolov2ResNet50VehicleExample0_LaneDetection0_res4c_branch2c_b"
      ".bin", 1);
    (static_cast<MWFusedConvReLULayer *>(this->layers[39]))
      ->createFusedConvReLULayer(this->targetImpl, 1, this->layers[38]
      ->getOutputTensor(0), 1, 1, 1024, 256, 1, 1, 0, 0, 0, 0, 1, 1, 1,
      "/home/hmcl/ADcontrol/slprj/_sfprj/LaneDetection/_self/sfun/src/cnn_yolov2ResNet50VehicleExample0_LaneDetection0_res4d_branch2a_w"
      ".bin",
      "/home/hmcl/ADcontrol/slprj/_sfprj/LaneDetection/_self/sfun/src/cnn_yolov2ResNet50VehicleExample0_LaneDetection0_res4d_branch2a_b"
      ".bin", 0);
    (static_cast<MWFusedConvReLULayer *>(this->layers[40]))
      ->createFusedConvReLULayer(this->targetImpl, 1, this->layers[39]
      ->getOutputTensor(0), 3, 3, 256, 256, 1, 1, 1, 1, 1, 1, 1, 1, 1,
      "/home/hmcl/ADcontrol/slprj/_sfprj/LaneDetection/_self/sfun/src/cnn_yolov2ResNet50VehicleExample0_LaneDetection0_res4d_branch2b_w"
      ".bin",
      "/home/hmcl/ADcontrol/slprj/_sfprj/LaneDetection/_self/sfun/src/cnn_yolov2ResNet50VehicleExample0_LaneDetection0_res4d_branch2b_b"
      ".bin", 2);
    (static_cast<MWFusedConvReLULayer *>(this->layers[41]))
      ->createFusedConvReLULayer(this->targetImpl, 2, this->layers[40]
      ->getOutputTensor(0), this->layers[38]->getOutputTensor(0), 1, 1, 256,
      1024, 1, 1, 0, 0, 0, 0, 1, 1, 1,
      "/home/hmcl/ADcontrol/slprj/_sfprj/LaneDetection/_self/sfun/src/cnn_yolov2ResNet50VehicleExample0_LaneDetection0_res4d_branch2c_w"
      ".bin",
      "/home/hmcl/ADcontrol/slprj/_sfprj/LaneDetection/_self/sfun/src/cnn_yolov2ResNet50VehicleExample0_LaneDetection0_res4d_branch2c_b"
      ".bin", 1);
    (static_cast<MWFusedConvReLULayer *>(this->layers[42]))
      ->createFusedConvReLULayer(this->targetImpl, 1, this->layers[41]
      ->getOutputTensor(0), 1, 1, 1024, 256, 1, 1, 0, 0, 0, 0, 1, 1, 1,
      "/home/hmcl/ADcontrol/slprj/_sfprj/LaneDetection/_self/sfun/src/cnn_yolov2ResNet50VehicleExample0_LaneDetection0_res4e_branch2a_w"
      ".bin",
      "/home/hmcl/ADcontrol/slprj/_sfprj/LaneDetection/_self/sfun/src/cnn_yolov2ResNet50VehicleExample0_LaneDetection0_res4e_branch2a_b"
      ".bin", 0);
    (static_cast<MWFusedConvReLULayer *>(this->layers[43]))
      ->createFusedConvReLULayer(this->targetImpl, 1, this->layers[42]
      ->getOutputTensor(0), 3, 3, 256, 256, 1, 1, 1, 1, 1, 1, 1, 1, 1,
      "/home/hmcl/ADcontrol/slprj/_sfprj/LaneDetection/_self/sfun/src/cnn_yolov2ResNet50VehicleExample0_LaneDetection0_res4e_branch2b_w"
      ".bin",
      "/home/hmcl/ADcontrol/slprj/_sfprj/LaneDetection/_self/sfun/src/cnn_yolov2ResNet50VehicleExample0_LaneDetection0_res4e_branch2b_b"
      ".bin", 2);
    (static_cast<MWFusedConvReLULayer *>(this->layers[44]))
      ->createFusedConvReLULayer(this->targetImpl, 2, this->layers[43]
      ->getOutputTensor(0), this->layers[41]->getOutputTensor(0), 1, 1, 256,
      1024, 1, 1, 0, 0, 0, 0, 1, 1, 1,
      "/home/hmcl/ADcontrol/slprj/_sfprj/LaneDetection/_self/sfun/src/cnn_yolov2ResNet50VehicleExample0_LaneDetection0_res4e_branch2c_w"
      ".bin",
      "/home/hmcl/ADcontrol/slprj/_sfprj/LaneDetection/_self/sfun/src/cnn_yolov2ResNet50VehicleExample0_LaneDetection0_res4e_branch2c_b"
      ".bin", 1);
    (static_cast<MWFusedConvReLULayer *>(this->layers[45]))
      ->createFusedConvReLULayer(this->targetImpl, 1, this->layers[44]
      ->getOutputTensor(0), 1, 1, 1024, 256, 1, 1, 0, 0, 0, 0, 1, 1, 1,
      "/home/hmcl/ADcontrol/slprj/_sfprj/LaneDetection/_self/sfun/src/cnn_yolov2ResNet50VehicleExample0_LaneDetection0_res4f_branch2a_w"
      ".bin",
      "/home/hmcl/ADcontrol/slprj/_sfprj/LaneDetection/_self/sfun/src/cnn_yolov2ResNet50VehicleExample0_LaneDetection0_res4f_branch2a_b"
      ".bin", 0);
    (static_cast<MWFusedConvReLULayer *>(this->layers[46]))
      ->createFusedConvReLULayer(this->targetImpl, 1, this->layers[45]
      ->getOutputTensor(0), 3, 3, 256, 256, 1, 1, 1, 1, 1, 1, 1, 1, 1,
      "/home/hmcl/ADcontrol/slprj/_sfprj/LaneDetection/_self/sfun/src/cnn_yolov2ResNet50VehicleExample0_LaneDetection0_res4f_branch2b_w"
      ".bin",
      "/home/hmcl/ADcontrol/slprj/_sfprj/LaneDetection/_self/sfun/src/cnn_yolov2ResNet50VehicleExample0_LaneDetection0_res4f_branch2b_b"
      ".bin", 2);
    (static_cast<MWFusedConvReLULayer *>(this->layers[47]))
      ->createFusedConvReLULayer(this->targetImpl, 2, this->layers[46]
      ->getOutputTensor(0), this->layers[44]->getOutputTensor(0), 1, 1, 256,
      1024, 1, 1, 0, 0, 0, 0, 1, 1, 1,
      "/home/hmcl/ADcontrol/slprj/_sfprj/LaneDetection/_self/sfun/src/cnn_yolov2ResNet50VehicleExample0_LaneDetection0_res4f_branch2c_w"
      ".bin",
      "/home/hmcl/ADcontrol/slprj/_sfprj/LaneDetection/_self/sfun/src/cnn_yolov2ResNet50VehicleExample0_LaneDetection0_res4f_branch2c_b"
      ".bin", 1);
    (static_cast<MWFusedConvReLULayer *>(this->layers[48]))
      ->createFusedConvReLULayer(this->targetImpl, 1, this->layers[47]
      ->getOutputTensor(0), 3, 3, 1024, 1024, 1, 1, 1, 1, 1, 1, 1, 1, 1,
      "/home/hmcl/ADcontrol/slprj/_sfprj/LaneDetection/_self/sfun/src/cnn_yolov2ResNet50VehicleExample0_LaneDetection0_yolov2Conv1_w.bi"
      "n",
      "/home/hmcl/ADcontrol/slprj/_sfprj/LaneDetection/_self/sfun/src/cnn_yolov2ResNet50VehicleExample0_LaneDetection0_yolov2Conv1_b.bi"
      "n", 0);
    (static_cast<MWFusedConvReLULayer *>(this->layers[49]))
      ->createFusedConvReLULayer(this->targetImpl, 1, this->layers[48]
      ->getOutputTensor(0), 3, 3, 1024, 1024, 1, 1, 1, 1, 1, 1, 1, 1, 1,
      "/home/hmcl/ADcontrol/slprj/_sfprj/LaneDetection/_self/sfun/src/cnn_yolov2ResNet50VehicleExample0_LaneDetection0_yolov2Conv2_w.bi"
      "n",
      "/home/hmcl/ADcontrol/slprj/_sfprj/LaneDetection/_self/sfun/src/cnn_yolov2ResNet50VehicleExample0_LaneDetection0_yolov2Conv2_b.bi"
      "n", 1);
    (static_cast<MWConvLayer *>(this->layers[50]))->createConvLayer
      (this->targetImpl, this->layers[49]->getOutputTensor(0), 1, 1, 1024, 24, 1,
       1, 0, 0, 0, 0, 1, 1, 1,
       "/home/hmcl/ADcontrol/slprj/_sfprj/LaneDetection/_self/sfun/src/cnn_yolov2ResNet50VehicleExample0_LaneDetection0_yolov2ClassConv_"
       "w.bin",
       "/home/hmcl/ADcontrol/slprj/_sfprj/LaneDetection/_self/sfun/src/cnn_yolov2ResNet50VehicleExample0_LaneDetection0_yolov2ClassConv_"
       "b.bin", 0);
    (static_cast<MWYoloExtractionLayer *>(this->layers[51]))
      ->createYoloExtractionLayer(this->targetImpl, this->layers[50]
      ->getOutputTensor(0), 4, 1, 2, 3);
    (static_cast<MWSigmoidLayer *>(this->layers[52]))->createSigmoidLayer
      (this->targetImpl, this->layers[51]->getOutputTensor(0), 0);
    (static_cast<MWExponentialLayer *>(this->layers[53]))
      ->createExponentialLayer(this->targetImpl, this->layers[51]
      ->getOutputTensor(1), 1);
    (static_cast<MWYoloSoftmaxLayer *>(this->layers[54]))
      ->createYoloSoftmaxLayer(this->targetImpl, this->layers[51]
      ->getOutputTensor(2), 4, 2);
    (static_cast<MWConcatenationLayer *>(this->layers[55]))
      ->createConcatenationLayer(this->targetImpl, 3, this->layers[52]
      ->getOutputTensor(0), this->layers[53]->getOutputTensor(0), this->layers
      [54]->getOutputTensor(0), 3, 3);
    (static_cast<MWOutputLayer *>(this->layers[56]))->createOutputLayer
      (this->targetImpl, this->layers[55]->getOutputTensor(0), 3);
    this->outputTensors[0] = this->layers[56]->getOutputTensor(0);
    this->setSize();
  }
}

void c5_yolov2ResNet50VehicleExample0_LaneDetection0::predict()
{
  int32_T c5_idx;
  for (c5_idx = 0; c5_idx < 57; c5_idx++) {
    this->layers[c5_idx]->predict();
  }
}

void c5_yolov2ResNet50VehicleExample0_LaneDetection0::activations(int32_T
  c5_layerIdx)
{
  int32_T c5_idx;
  for (c5_idx = 0; c5_idx <= c5_layerIdx; c5_idx++) {
    this->layers[c5_idx]->predict();
  }
}

void c5_yolov2ResNet50VehicleExample0_LaneDetection0::cleanup()
{
  int32_T c5_idx;
  this->deallocate();
  for (c5_idx = 0; c5_idx < 57; c5_idx++) {
    this->layers[c5_idx]->cleanup();
  }

  if (this->targetImpl) {
    this->targetImpl->cleanup();
  }
}

real32_T *c5_yolov2ResNet50VehicleExample0_LaneDetection0::getLayerOutput
  (int32_T c5_layerIndex, int32_T c5_portIndex)
{
  return this->layers[c5_layerIndex]->getLayerOutput(c5_portIndex);
}

real32_T *c5_yolov2ResNet50VehicleExample0_LaneDetection0::getInputDataPointer
  (int32_T c5_index)
{
  return (static_cast<MWTensor<real32_T> *>(this->inputTensors[c5_index]))
    ->getData();
}

real32_T *c5_yolov2ResNet50VehicleExample0_LaneDetection0::getInputDataPointer()
{
  return (static_cast<MWTensor<real32_T> *>(this->inputTensors[0]))->getData();
}

real32_T *c5_yolov2ResNet50VehicleExample0_LaneDetection0::getOutputDataPointer
  (int32_T c5_index)
{
  return (static_cast<MWTensor<real32_T> *>(this->outputTensors[c5_index]))
    ->getData();
}

real32_T *c5_yolov2ResNet50VehicleExample0_LaneDetection0::getOutputDataPointer()
{
  return (static_cast<MWTensor<real32_T> *>(this->outputTensors[0]))->getData();
}

int32_T c5_yolov2ResNet50VehicleExample0_LaneDetection0::getBatchSize()
{
  return this->inputTensors[0]->getBatchSize();
}

c5_yolov2ResNet50VehicleExample0_LaneDetection0::
  ~c5_yolov2ResNet50VehicleExample0_LaneDetection0()
{
  int32_T c5_idx;
  this->cleanup();
  c5_checkCleanupCudaError(hipGetLastError(), __FILE__, __LINE__);
  for (c5_idx = 0; c5_idx < 57; c5_idx++) {
    delete this->layers[c5_idx];
  }

  if (this->targetImpl) {
    delete this->targetImpl;
  }

  delete this->inputTensors[0];
}

static void c5_checkCleanupCudaError(hipError_t c5_errCode, const char_T
  *c5_file, uint32_T c5_line)
{
  emlrtRTEInfo c5_rtInfo;
  if ((c5_errCode != hipSuccess) && (c5_errCode != hipErrorDeinitialized)) {
    c5_rtInfo = c5_createEmlrtInfoStruct(c5_file, c5_line);
    emlrtCUDAWarning(c5_errCode, hipGetErrorName(c5_errCode),
                     hipGetErrorString(c5_errCode), &c5_rtInfo);
  }
}

static emlrtRTEInfo c5_createEmlrtInfoStruct(const char_T *c5_file, uint32_T
  c5_line)
{
  emlrtRTEInfo c5_b_rtInfo;
  uint32_T c5_len;
  char_T *c5_brk;
  char_T *c5_fn;
  char_T *c5_pn;
  c5_len = (uint32_T)strlen(c5_file);
  c5_pn = (char_T *)calloc(c5_len + 1U, 1U);
  c5_fn = (char_T *)calloc(c5_len + 1U, 1U);
  memcpy(c5_pn, c5_file, c5_len);
  memcpy(c5_fn, c5_file, c5_len);
  c5_brk = strrchr(c5_fn, '.');
  *c5_brk = '\x00';
  c5_brk = NULL;
  c5_brk = strrchr(c5_fn, '/');
  if (c5_brk == NULL) {
    c5_brk = strrchr(c5_fn, '\\');
  }

  if (c5_brk == NULL) {
    c5_brk = c5_fn;
  } else {
    c5_brk++;
  }

  c5_b_rtInfo.lineNo = c5_line;
  c5_b_rtInfo.colNo = 0;
  c5_b_rtInfo.fName = c5_brk;
  c5_b_rtInfo.pName = c5_pn;
  return c5_b_rtInfo;
}

static void init_dsm_address_info(SFc5_LaneDetectionInstanceStruct
  *chartInstance)
{
}

static void init_simulink_io_address(SFc5_LaneDetectionInstanceStruct
  *chartInstance)
{
  chartInstance->c5_fEmlrtCtx = (void *)sfrtGetEmlrtCtx(chartInstance->S);
  chartInstance->c5_bboxes_data = (real_T (*)[80])ssGetOutputPortSignal_wrapper
    (chartInstance->S, 1);
  chartInstance->c5_bboxes_sizes = (int32_T (*)[2])
    ssGetCurrentOutputPortDimensions_wrapper(chartInstance->S, 1);
  chartInstance->c5_In = (real32_T (*)[921600])ssGetInputPortSignal_wrapper
    (chartInstance->S, 0);
  chartInstance->c5_scores_data = (real32_T (*)[20])
    ssGetOutputPortSignal_wrapper(chartInstance->S, 2);
  chartInstance->c5_scores_sizes = (int32_T (*)[2])
    ssGetCurrentOutputPortDimensions_wrapper(chartInstance->S, 2);
}

#include <stdio.h>
#include <stdlib.h>
#include <string.h>

/* SFunction Glue Code */
void sf_c5_LaneDetection_get_check_sum(mxArray *plhs[])
{
  ((real_T *)mxGetPr((plhs[0])))[0] = (real_T)(955709936U);
  ((real_T *)mxGetPr((plhs[0])))[1] = (real_T)(3991420490U);
  ((real_T *)mxGetPr((plhs[0])))[2] = (real_T)(2490454663U);
  ((real_T *)mxGetPr((plhs[0])))[3] = (real_T)(2264674843U);
}

mxArray *sf_c5_LaneDetection_third_party_uses_info(void)
{
  mxArray * mxcell3p = mxCreateCellMatrix(1,1);
  mxSetCell(mxcell3p, 0, mxCreateString("dltargets.cudnn.cudnnApi"));
  return(mxcell3p);
}

mxArray *sf_c5_LaneDetection_jit_fallback_info(void)
{
  const char *infoFields[] = { "fallbackType", "fallbackReason",
    "hiddenFallbackType", "hiddenFallbackReason", "incompatibleSymbol" };

  mxArray *mxInfo = mxCreateStructMatrix(1, 1, 5, infoFields);
  mxArray *fallbackType = mxCreateString("pre");
  mxArray *fallbackReason = mxCreateString("GPUAcceleration");
  mxArray *hiddenFallbackType = mxCreateString("late");
  mxArray *hiddenFallbackReason = mxCreateString("ir_function_calls");
  mxArray *incompatibleSymbol = mxCreateString("#__setup__");
  mxSetField(mxInfo, 0, infoFields[0], fallbackType);
  mxSetField(mxInfo, 0, infoFields[1], fallbackReason);
  mxSetField(mxInfo, 0, infoFields[2], hiddenFallbackType);
  mxSetField(mxInfo, 0, infoFields[3], hiddenFallbackReason);
  mxSetField(mxInfo, 0, infoFields[4], incompatibleSymbol);
  return mxInfo;
}

mxArray *sf_c5_LaneDetection_updateBuildInfo_args_info(void)
{
  mxArray *mxBIArgs = sf_mex_decode(
    "eNrlWMFqwkAQjaUthYJWQatYEASvfkOLVSrVUGpL21NZklWWJrsh2Wh667Gf18/osccem0QTN2N"
    "q8CRsAiHsMtnHy8y8mYlSGI4V/yr691dNUY7954l/F5TldSSsz4X9wP5X2W5/AOwvBfvDFPuiYH"
    "+2WiNdnzDX1vCAGNhZ7n1k4BYA7l0G7gXADdbjp76BTUz5gjj4ajolFI/QO7a7mmVF5/7sib+3I"
    "/+bDNw6wK2H/Aeug/Ueo/N7PHqMuQfnfe+Jt7Uj7yzcU4AbrDVKX5FFIjdL5ecywC2Hfu6paujd"
    "oWkZXc2VML9bALe1Jb+jrxCcG7y0D/6fO/J/zsDtANxOBv9bbFO8igWZ4qAJcJv/6JyYCTLlfw3"
    "g1kL+D8ieYa5ivmD2W0xdJr83AG5jqXuuw5kZOnzA7J57raqx6uehvvshryGOKeKEUSnrewngli"
    "Le83UrJ39+9z2LUV/oCTKEDlYm3hWAWwl5T8jMZERPdO150LUXZrC+x22kJRJbQl1Lj/eA/4RNu"
    "Yk8EO9y9zGbei5rH5Oe77GuJwYZ+fMd6nvu5rgU/usJRqa4rwLc6kadW/te/v9zKXVOCH2Z5vc2"
    "wG1v4y8O77npd8R6n9S/P7KUnlA="
    );
  return mxBIArgs;
}

static const mxArray *sf_get_sim_state_info_c5_LaneDetection(void)
{
  const char *infoFields[] = { "chartChecksum", "varInfo" };

  mxArray *mxInfo = mxCreateStructMatrix(1, 1, 2, infoFields);
  mxArray *mxVarInfo = sf_mex_decode(
    "eNpjYPT0ZQACPiA+wcTAwAakOYCYiQECWKF8RiBmhtIQcRa4uAIQl1QWpILEi4uSPVOAdF5iLpi"
    "fWFrhmZeWDzbfggFhPhsW8xmRzOeEikPAB3vK9Is4oOtnwaKfDUm/AJSflJRfkVoMCR9YOA2cPx"
    "TI9kdxcn4R9fyB6Q7S9EPsDyDgDyk0f4D4mcXxicklmWWp8cmm8T6JeakuqSWpQIH8PIS5IAAA5"
    "lYhew=="
    );
  mxArray *mxChecksum = mxCreateDoubleMatrix(1, 4, mxREAL);
  sf_c5_LaneDetection_get_check_sum(&mxChecksum);
  mxSetField(mxInfo, 0, infoFields[0], mxChecksum);
  mxSetField(mxInfo, 0, infoFields[1], mxVarInfo);
  return mxInfo;
}

static const char* sf_get_instance_specialization(void)
{
  return "ssN2SKAuylfcyN4by9KRKKE";
}

static void sf_opaque_initialize_c5_LaneDetection(void *chartInstanceVar)
{
  initialize_params_c5_LaneDetection((SFc5_LaneDetectionInstanceStruct*)
    chartInstanceVar);
  initialize_c5_LaneDetection((SFc5_LaneDetectionInstanceStruct*)
    chartInstanceVar);
}

static void sf_opaque_enable_c5_LaneDetection(void *chartInstanceVar)
{
  enable_c5_LaneDetection((SFc5_LaneDetectionInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_disable_c5_LaneDetection(void *chartInstanceVar)
{
  disable_c5_LaneDetection((SFc5_LaneDetectionInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_gateway_c5_LaneDetection(void *chartInstanceVar)
{
  sf_gateway_c5_LaneDetection((SFc5_LaneDetectionInstanceStruct*)
    chartInstanceVar);
}

static const mxArray* sf_opaque_get_sim_state_c5_LaneDetection(SimStruct* S)
{
  return get_sim_state_c5_LaneDetection((SFc5_LaneDetectionInstanceStruct *)
    sf_get_chart_instance_ptr(S));     /* raw sim ctx */
}

static void sf_opaque_set_sim_state_c5_LaneDetection(SimStruct* S, const mxArray
  *st)
{
  set_sim_state_c5_LaneDetection((SFc5_LaneDetectionInstanceStruct*)
    sf_get_chart_instance_ptr(S), st);
}

static void sf_opaque_cleanup_runtime_resources_c5_LaneDetection(void
  *chartInstanceVar)
{
  if (chartInstanceVar!=NULL) {
    SimStruct *S = ((SFc5_LaneDetectionInstanceStruct*) chartInstanceVar)->S;
    if (sim_mode_is_rtw_gen(S) || sim_mode_is_external(S)) {
      sf_clear_rtw_identifier(S);
      unload_LaneDetection_optimization_info();
    }

    mdl_cleanup_runtime_resources_c5_LaneDetection
      ((SFc5_LaneDetectionInstanceStruct*) chartInstanceVar);
    ((SFc5_LaneDetectionInstanceStruct*) chartInstanceVar)->
      ~SFc5_LaneDetectionInstanceStruct();
    utFree(chartInstanceVar);
    if (ssGetUserData(S)!= NULL) {
      sf_free_ChartRunTimeInfo(S);
    }

    ssSetUserData(S,NULL);
  }
}

static void sf_opaque_mdl_start_c5_LaneDetection(void *chartInstanceVar)
{
  mdl_start_c5_LaneDetection((SFc5_LaneDetectionInstanceStruct*)
    chartInstanceVar);
}

static void sf_opaque_mdl_terminate_c5_LaneDetection(void *chartInstanceVar)
{
  mdl_terminate_c5_LaneDetection((SFc5_LaneDetectionInstanceStruct*)
    chartInstanceVar);
}

static void sf_opaque_init_subchart_simstructs(void *chartInstanceVar)
{
  initSimStructsc5_LaneDetection((SFc5_LaneDetectionInstanceStruct*)
    chartInstanceVar);
}

extern unsigned int sf_machine_global_initializer_called(void);
static void mdlProcessParameters_c5_LaneDetection(SimStruct *S)
{
  int i;
  for (i=0;i<ssGetNumRunTimeParams(S);i++) {
    if (ssGetSFcnParamTunable(S,i)) {
      ssUpdateDlgParamAsRunTimeParam(S,i);
    }
  }

  sf_warn_if_symbolic_dimension_param_changed(S);
  if (sf_machine_global_initializer_called()) {
    initialize_params_c5_LaneDetection((SFc5_LaneDetectionInstanceStruct*)
      sf_get_chart_instance_ptr(S));
    initSimStructsc5_LaneDetection((SFc5_LaneDetectionInstanceStruct*)
      sf_get_chart_instance_ptr(S));
  }
}

const char* sf_c5_LaneDetection_get_post_codegen_info(void)
{
  int i;
  const char* encStrCodegen [32] = {
    "eNrtXM1v40QUT7vLipVgtQcE0grE3kBCIm2SIpAQbJsPEfVjS9Mtx2hmPKmH2GPXM04a7Yn9+D8",
    "47pEDB/4EjnBA4j9A3Lhw542Tpq5jJ+t2afKijZSmdt48/97nvBk/p7DS3C3A6w68X/BC4RZ8vg",
    "nv1cLw9cboeCX2Hp6/WfhodPwzDJKhu08C4qrC1JckLj/gynNCLTzZlB0vlUzIDg+4ZEDre4HO4",
    "qaEGzpCdhuhZIaf+s4WzG7ZXuhYWzCWWA+lMwBufqj3gU9NBJzpBueWtgMvPLYbDjkeIw50v2pz",
    "1lWhO00ExXUr9A0stRs6WvgOr59y1pRKE0CszrG1NNG8qk8zxTSSqtYZoef6jiAyVVqbqBb3QcG",
    "aP/It+Psw1CBUkozZJNBb3CY9rnZEN+LpSZ7kKRR8QYUk2gsEcequUzUDJ7HtO4Bn17O4M0UhgG",
    "0r4KTre0LqbPu3GiBpXRLq8Bqn4XE2txY/CY3xjwTv8yBTb52q1+MBOeYPZeZFI4XUTyNrjb1kk",
    "kwLlx+RYJOB/RS3Mr0XPEe1CNiJH8KILDIeCdlUh4HogXozuYVu03jmrJAJ3aGx1SyyiFu9x6dZ",
    "YcytwWSVOI7KJDv0/B3e407EtUY0mU425JpOp5SwDj1QsHHv7GgIpQDDj8iqnrREqrl6CYIo7+x",
    "BYrlIyUKlPbcKzlvb2Zn8epKsKTUPOoTxtCwQEKE46CxSbzY3SyhjeyAEVDqCl0Y89JBZVAXVCW",
    "Wt7wVd0MmUJHIugrFoJqGrjsGWEAmPFATNNDJjy1l0jDCbWybBCIfvQtgAbYpOlEltmxB3PaEHN",
    "a5YIPwUq4YQdZCG6sahBj5/JLvS68tG4LmtUY4fWgEyA+RwF2xwGMWYZMBKKA3pQpxf3uIcnJIE",
    "UsjjLUhzwaABIFMtZua9tcL5vPfWS8x7Z+OSnx/H+Kyk8CnEPpPXvb06/bqr8N/KaNyD2Li3E9e",
    "5mRhn6O7C++gP9sHWb+TfL//6/cW3737y91Wuf3ojX51wZ3T8/llCHgdYb8KvDe03MVw3U/i/F+",
    "N/d3Ss1F6ptb0ZDpwOG+xV6OCL7YPt7XrE79fV6XhvJPCenb9vZgbwxsiPA9a0RgWMOSbhcFo3/",
    "D+P4b01Qx+3R+eHr3++vtr4dx4kx6fp61ZCX+aYUu/URETcf+cnx/1Ly6GYF7w6OSZx5Bs/vP7+",
    "DDnuJeS4F9UzbWKyJG+zjfYOkVAjaR6VK5N5Jm/cvh73etwij1u55Dx82XGrV5z3r2vcVeXLW48",
    "sGv3alDxaSNDfXWA5rlon/t/0fxby1XMfjo6/Gi/tqrZwrJQqf/Q11OKdtG+XxE/9g+n0foL+lx",
    "n1wQ8JvzbHRdtzedF2mVPcrMGiRweeU1SOH3xfbKuO+bhQNMBJ7nSKZgFZhNK1yKRsDzzH65UOu",
    "NrjemPtiNuCma2raDtj7WLNsTYihjV2r9Smn1Ih0eJfb/dz4n+ewP98fvgdopQRYiwDVvw0B/5n",
    "CfzP5oAfFhWVTpsGRDK7xM5dCCl+ihE/R67/MX6KHj9u/yHI4xe7/xCc/mMhz58Wcv+3kPsPQz5",
    "/MeT+z5bF/yl6/aPET5HHL0UevxR5/UNxx2+J4s6fcfwo9U+WRf+J+P1xBn4vgd97Nfh1QITklq",
    "EDCSYQd5g5sT5yFUx6ZsjzJLlknD5N4H865zhdj+2UL7qfl/D5eZlkzKc/zcD/OIH/8Rzws8SdC",
    "JTzUM46ePHiE7n+sa8DKfI6LOc66kkC/5M54I+efWivt71OR3F9fiMUZd7HWceXybLcR0S6j0OQ",
    "74N0MvLnrPrSTeB3r6W+ZBd6RjDF6WX3ixeh3+UszytGHJ7od1lcP6nk8pNF64vqo9FzOVf/0KL",
    "EI12WeR/n/l3ZuuT+3fz8fOOCny/y/IgwHkvI74eVreXZT6d49tP7433GRZ4fKYq8sYGyrqbL0g",
    "eDZh5cx+gnpax+BTz7+xTj/nKZLcv+CMESn5UL9d+LGThPEjhPrgWnRTSZ2DZ+Kb12E3i711Rvf",
    "Jbz/uv8cPax54k+8udJKPJ+UJTrF+T9oCXk/cTlpdE/RY6focePsQ+kktWnhQR/eXnw46x/CO75",
    "t4z8PngZeT96GfnzAPHnIfuo+7go8vqNIo9f/PrHfv+LYutvif+eS97fo1nA53ko8n4A3PmH5bt",
    "/6ifw+9e3/zpu50LpJwy5n6Nf56JcZ1WQ901VCMI+jVKsTwOhnnE/H07w9BUg7FurkGX5HYdY3f",
    "gf5Lio9w==",
    ""
  };

  static char newstr [2261] = "";
  newstr[0] = '\0';
  for (i = 0; i < 32; i++) {
    strcat(newstr, encStrCodegen[i]);
  }

  return newstr;
}

static void mdlSetWorkWidths_c5_LaneDetection(SimStruct *S)
{
  const char* newstr = sf_c5_LaneDetection_get_post_codegen_info();
  sf_set_work_widths(S, newstr);
  ssSetChecksum0(S,(493081942U));
  ssSetChecksum1(S,(4116828226U));
  ssSetChecksum2(S,(2832328508U));
  ssSetChecksum3(S,(3911915089U));
}

static void mdlRTW_c5_LaneDetection(SimStruct *S)
{
  if (sim_mode_is_rtw_gen(S)) {
    ssWriteRTWStrParam(S, "StateflowChartType", "Embedded MATLAB");
  }
}

static void mdlSetupRuntimeResources_c5_LaneDetection(SimStruct *S)
{
  SFc5_LaneDetectionInstanceStruct *chartInstance;
  chartInstance = (SFc5_LaneDetectionInstanceStruct *)utMalloc(sizeof
    (SFc5_LaneDetectionInstanceStruct));
  if (chartInstance==NULL) {
    sf_mex_error_message("Could not allocate memory for chart instance.");
  }

  memset(chartInstance, 0, sizeof(SFc5_LaneDetectionInstanceStruct));
  chartInstance = new (chartInstance) SFc5_LaneDetectionInstanceStruct;
  chartInstance->chartInfo.chartInstance = chartInstance;
  if (ssGetSampleTime(S, 0) == CONTINUOUS_SAMPLE_TIME && ssGetOffsetTime(S, 0) ==
      0 && ssGetNumContStates(ssGetRootSS(S)) > 0) {
    sf_error_out_about_continuous_sample_time_with_persistent_vars(S);
  }

  chartInstance->chartInfo.isEMLChart = 1;
  chartInstance->chartInfo.chartInitialized = 0;
  chartInstance->chartInfo.sFunctionGateway = sf_opaque_gateway_c5_LaneDetection;
  chartInstance->chartInfo.initializeChart =
    sf_opaque_initialize_c5_LaneDetection;
  chartInstance->chartInfo.mdlStart = sf_opaque_mdl_start_c5_LaneDetection;
  chartInstance->chartInfo.mdlTerminate =
    sf_opaque_mdl_terminate_c5_LaneDetection;
  chartInstance->chartInfo.mdlCleanupRuntimeResources =
    sf_opaque_cleanup_runtime_resources_c5_LaneDetection;
  chartInstance->chartInfo.enableChart = sf_opaque_enable_c5_LaneDetection;
  chartInstance->chartInfo.disableChart = sf_opaque_disable_c5_LaneDetection;
  chartInstance->chartInfo.getSimState =
    sf_opaque_get_sim_state_c5_LaneDetection;
  chartInstance->chartInfo.setSimState =
    sf_opaque_set_sim_state_c5_LaneDetection;
  chartInstance->chartInfo.getSimStateInfo =
    sf_get_sim_state_info_c5_LaneDetection;
  chartInstance->chartInfo.zeroCrossings = NULL;
  chartInstance->chartInfo.outputs = NULL;
  chartInstance->chartInfo.derivatives = NULL;
  chartInstance->chartInfo.mdlRTW = mdlRTW_c5_LaneDetection;
  chartInstance->chartInfo.mdlSetWorkWidths = mdlSetWorkWidths_c5_LaneDetection;
  chartInstance->chartInfo.callGetHoverDataForMsg = NULL;
  chartInstance->chartInfo.extModeExec = NULL;
  chartInstance->chartInfo.restoreLastMajorStepConfiguration = NULL;
  chartInstance->chartInfo.restoreBeforeLastMajorStepConfiguration = NULL;
  chartInstance->chartInfo.storeCurrentConfiguration = NULL;
  chartInstance->chartInfo.callAtomicSubchartUserFcn = NULL;
  chartInstance->chartInfo.callAtomicSubchartAutoFcn = NULL;
  chartInstance->chartInfo.callAtomicSubchartEventFcn = NULL;
  chartInstance->chartInfo.chartStateSetterFcn = NULL;
  chartInstance->chartInfo.chartStateGetterFcn = NULL;
  chartInstance->S = S;
  chartInstance->chartInfo.dispatchToExportedFcn = NULL;
  sf_init_ChartRunTimeInfo(S, &(chartInstance->chartInfo), false, 0,
    chartInstance->c5_JITStateAnimation,
    chartInstance->c5_JITTransitionAnimation);
  init_dsm_address_info(chartInstance);
  init_simulink_io_address(chartInstance);
  if (!sim_mode_is_rtw_gen(S)) {
  }

  mdl_setup_runtime_resources_c5_LaneDetection(chartInstance);
}

void c5_LaneDetection_method_dispatcher(SimStruct *S, int_T method, void *data)
{
  switch (method) {
   case SS_CALL_MDL_SETUP_RUNTIME_RESOURCES:
    mdlSetupRuntimeResources_c5_LaneDetection(S);
    break;

   case SS_CALL_MDL_SET_WORK_WIDTHS:
    mdlSetWorkWidths_c5_LaneDetection(S);
    break;

   case SS_CALL_MDL_PROCESS_PARAMETERS:
    mdlProcessParameters_c5_LaneDetection(S);
    break;

   default:
    /* Unhandled method */
    sf_mex_error_message("Stateflow Internal Error:\n"
                         "Error calling c5_LaneDetection_method_dispatcher.\n"
                         "Can't handle method %d.\n", method);
    break;
  }
}
