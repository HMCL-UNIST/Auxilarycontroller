#include "hip/hip_runtime.h"
/* Include files */

#include "LaneDetection_sfun.h"
#include "c5_LaneDetection.h"
#include <string.h>
#include "MWCudaDimUtility.hpp"
#include "MWLaunchParametersUtilities.hpp"
#include "MWShuffleUtility.h"
#include "MWSortFunctors.h"
#include "MWSortWithIndexUtility.h"
#include "mwmathutil.h"
#ifdef utFree
#undef utFree
#endif

#ifdef utMalloc
#undef utMalloc
#endif

#ifdef __cplusplus

extern "C" void *utMalloc(size_t size);
extern "C" void utFree(void*);

#else

extern void *utMalloc(size_t size);
extern void utFree(void*);

#endif

#include <cstdlib>
#include <cstring>

/* Type Definitions */

/* Named Constants */
const int32_T CALL_EVENT = -1;

/* Variable Declarations */

/* Variable Definitions */
static real_T _sfTime_;

/* Function Declarations */
static void initialize_c5_LaneDetection(SFc5_LaneDetectionInstanceStruct
  *chartInstance);
static void initialize_params_c5_LaneDetection(SFc5_LaneDetectionInstanceStruct *
  chartInstance);
static void enable_c5_LaneDetection(SFc5_LaneDetectionInstanceStruct
  *chartInstance);
static void disable_c5_LaneDetection(SFc5_LaneDetectionInstanceStruct
  *chartInstance);
static void c5_do_animation_call_c5_LaneDetection
  (SFc5_LaneDetectionInstanceStruct *chartInstance);
static void ext_mode_exec_c5_LaneDetection(SFc5_LaneDetectionInstanceStruct
  *chartInstance);
static const mxArray *get_sim_state_c5_LaneDetection
  (SFc5_LaneDetectionInstanceStruct *chartInstance);
static void set_sim_state_c5_LaneDetection(SFc5_LaneDetectionInstanceStruct
  *chartInstance, const mxArray *c5_st);
static void sf_gateway_c5_LaneDetection(SFc5_LaneDetectionInstanceStruct
  *chartInstance);
static void mdl_start_c5_LaneDetection(SFc5_LaneDetectionInstanceStruct
  *chartInstance);
static void mdl_terminate_c5_LaneDetection(SFc5_LaneDetectionInstanceStruct
  *chartInstance);
static void mdl_setup_runtime_resources_c5_LaneDetection
  (SFc5_LaneDetectionInstanceStruct *chartInstance);
static void mdl_cleanup_runtime_resources_c5_LaneDetection
  (SFc5_LaneDetectionInstanceStruct *chartInstance);
static void initSimStructsc5_LaneDetection(SFc5_LaneDetectionInstanceStruct
  *chartInstance);
static void c5_eML_blk_kernel(SFc5_LaneDetectionInstanceStruct *chartInstance,
  real32_T c5_b_In[1229760], real_T c5_b_bboxes_data[], int32_T c5_bboxes_size[2],
  real32_T c5_b_scores_data[], int32_T c5_scores_size[2]);
static void c5_DeepLearningNetwork_setup(SFc5_LaneDetectionInstanceStruct
  *chartInstance, c5_yolov2ResNet50VehicleExample0_LaneDetection0 *c5_obj);
static real32_T c5_callFcn(SFc5_LaneDetectionInstanceStruct *chartInstance,
  real32_T c5_input1, real32_T c5_input2);
static real32_T c5_b_callFcn(SFc5_LaneDetectionInstanceStruct *chartInstance,
  real32_T c5_input1, real32_T c5_input2);
static void c5_DeepLearningNetwork_activations(SFc5_LaneDetectionInstanceStruct *
  chartInstance, c5_yolov2ResNet50VehicleExample0_LaneDetection0 *c5_obj,
  real32_T c5_varargin_1[150528], real32_T c5_b_out[4704]);
static void c5_indexShapeCheck(SFc5_LaneDetectionInstanceStruct *chartInstance,
  int32_T c5_matrixSize, int32_T c5_indexSize[2]);
static void c5_emlrt_marshallIn(SFc5_LaneDetectionInstanceStruct *chartInstance,
  const mxArray *c5_bboxes, const char_T *c5_identifier, real_T c5_y_data[],
  int32_T c5_y_size[2]);
static void c5_b_emlrt_marshallIn(SFc5_LaneDetectionInstanceStruct
  *chartInstance, const mxArray *c5_u, const emlrtMsgIdentifier *c5_parentId,
  real_T c5_y_data[], int32_T c5_y_size[2]);
static void c5_c_emlrt_marshallIn(SFc5_LaneDetectionInstanceStruct
  *chartInstance, const mxArray *c5_scores, const char_T *c5_identifier,
  real32_T c5_y_data[], int32_T c5_y_size[2]);
static void c5_d_emlrt_marshallIn(SFc5_LaneDetectionInstanceStruct
  *chartInstance, const mxArray *c5_u, const emlrtMsgIdentifier *c5_parentId,
  real32_T c5_y_data[], int32_T c5_y_size[2]);
static uint8_T c5_e_emlrt_marshallIn(SFc5_LaneDetectionInstanceStruct
  *chartInstance, const mxArray *c5_b_is_active_c5_LaneDetection, const char_T
  *c5_identifier);
static uint8_T c5_f_emlrt_marshallIn(SFc5_LaneDetectionInstanceStruct
  *chartInstance, const mxArray *c5_u, const emlrtMsgIdentifier *c5_parentId);
static void c5_emxConvertDynamicMatrixFromEmx_(SFc5_LaneDetectionInstanceStruct *
  chartInstance, c5_emxArray_real_T_20x4 *c5_emx, real_T c5_data[80], int32_T
  c5_size[2]);
static void c5_b_emxConvertDynamicMatrixFromEmx_
  (SFc5_LaneDetectionInstanceStruct *chartInstance, c5_emxArray_real32_T_20x1
   *c5_emx, real32_T c5_data[20], int32_T c5_size[2]);
static __global__ void c5_coder_reduce0(const real32_T c5_inputVar[150528],
  real32_T *c5_outputVar);
static __device__ real32_T c5_threadGroupReduction(real32_T c5_val, uint32_T
  c5_lane, uint32_T c5_mask);
static __device__ real32_T c5_shflDown1(real32_T c5_in1, uint32_T c5_offset,
  uint32_T c5_mask);
static __device__ real32_T c5_workGroupReduction(real32_T c5_val, uint32_T
  c5_mask, uint32_T c5_numActiveWarps);
static __device__ real32_T c5_b_threadGroupReduction(real32_T c5_val, uint32_T
  c5_lane, uint32_T c5_mask);
static __device__ real32_T c5_b_workGroupReduction(real32_T c5_val, uint32_T
  c5_mask, uint32_T c5_numActiveWarps);
static __device__ real32_T c5_atomicOpreal32_T(real32_T *c5_address, real32_T
  c5_value);
static __device__ real32_T c5_b_atomicOpreal32_T(real32_T *c5_address, real32_T
  c5_value);
static __global__ void c5_eML_blk_kernel_kernel1(int16_T c5_aux1[960]);
static __global__ void c5_eML_blk_kernel_kernel2(int16_T c5_aux2[1708]);
static __global__ void c5_eML_blk_kernel_kernel3(const int16_T c5_aux1[960],
  real_T c5_rowWeights[2016], int16_T c5_ipRowIndices[2016]);
static __global__ void c5_eML_blk_kernel_kernel4(const int16_T c5_aux2[1708],
  real_T c5_colWeights[3584], int16_T c5_ipColIndices[3584]);
static __global__ void c5_eML_blk_kernel_kernel5(const real_T c5_rowWeights[2016],
  real_T c5_rowWeightsTotal[224]);
static __global__ void c5_eML_blk_kernel_kernel6(const real_T c5_rowWeights[2016],
  const int32_T c5_xoffset, real_T c5_rowWeightsTotal[224]);
static __global__ void c5_eML_blk_kernel_kernel7(const real_T c5_colWeights[3584],
  real_T c5_colWeightsTotal[224]);
static __global__ void c5_eML_blk_kernel_kernel8(const real_T c5_colWeights[3584],
  const int32_T c5_xoffset, real_T c5_colWeightsTotal[224]);
static __global__ void c5_eML_blk_kernel_kernel9(const real_T
  c5_colWeightsTotal[224], const real_T c5_colWeights[3584], const real32_T
  c5_b_In[1229760], const int16_T c5_ipColIndices[3584], real32_T
  c5_partialResize[322560]);
static __global__ void c5_eML_blk_kernel_kernel10(const real_T
  c5_rowWeightsTotal[224], const real_T c5_rowWeights[2016], const real32_T
  c5_partialResize[322560], const int16_T c5_ipRowIndices[2016], real32_T
  c5_b_out[150528]);
static __global__ void c5_eML_blk_kernel_kernel11(real32_T c5_b_out[150528],
  real32_T c5_outVal[2]);
static __global__ void c5_eML_blk_kernel_kernel12(const real32_T c5_y, const
  real32_T c5_outVal, real32_T c5_b_out[150528], real32_T c5_c_out[150528]);
static __global__ void c5_eML_blk_kernel_kernel13(const int8_T c5_dv[8], real_T
  c5_anchors[8]);
static __global__ void c5_eML_blk_kernel_kernel14(const real_T c5_dv1[4], real_T
  c5_anchors[4], real_T c5_b_anchors[8]);
static __global__ void c5_eML_blk_kernel_kernel15(const real_T c5_anchors[8],
  const real32_T c5_tmpFeatureMap[4704], real32_T c5_boxOut[4704]);
static __global__ void c5_eML_blk_kernel_kernel16(const real32_T c5_boxOut[4704],
  boolean_T c5_bv[784]);
static __global__ void c5_eML_blk_kernel_kernel17(const real32_T c5_boxOut[4704],
  const int16_T c5_ii_data[784], const int32_T c5_thresholdedPrediction_size[2],
  const int32_T c5_ii_size[1], real32_T c5_thresholdedPrediction_data[4704]);
static __global__ void c5_eML_blk_kernel_kernel18(const real32_T
  c5_thresholdedPrediction_data[4704], const int32_T
  c5_thresholdedPrediction_size[2], const int32_T c5_bboxesX1Y1X2Y2_size[2],
  const int32_T c5_i7, real_T c5_bboxesX1Y1X2Y2_data[3136]);
static __global__ void c5_eML_blk_kernel_kernel19(const real_T
  c5_bboxesX1Y1X2Y2_data[3136], const int32_T c5_i10, real_T c5_x1_data[784]);
static __global__ void c5_eML_blk_kernel_kernel20(const real_T
  c5_bboxesX1Y1X2Y2_data[3136], const int32_T c5_bboxesX1Y1X2Y2_size[2], const
  int32_T c5_i14, real_T c5_y1_data[784]);
static __global__ void c5_eML_blk_kernel_kernel21(const real_T
  c5_bboxesX1Y1X2Y2_data[3136], const int32_T c5_bboxesX1Y1X2Y2_size[2], const
  int32_T c5_i17, real_T c5_x2_data[784]);
static __global__ void c5_eML_blk_kernel_kernel22(const real_T
  c5_bboxesX1Y1X2Y2_data[3136], const int32_T c5_bboxesX1Y1X2Y2_size[2], const
  int32_T c5_i19, real_T c5_y2_data[784]);
static __global__ void c5_eML_blk_kernel_kernel23(const int32_T c5_end, real_T
  c5_x1_data[784]);
static __global__ void c5_eML_blk_kernel_kernel24(const int32_T c5_end, real_T
  c5_y1_data[784]);
static __global__ void c5_eML_blk_kernel_kernel25(const int32_T c5_end, real_T
  c5_x2_data[784]);
static __global__ void c5_eML_blk_kernel_kernel26(const int32_T c5_end, real_T
  c5_y2_data[784]);
static __global__ void c5_eML_blk_kernel_kernel27(const real_T c5_x1_data[784],
  const int32_T c5_loop_ub, real_T c5_bboxesX1Y1X2Y2_data[3136]);
static __global__ void c5_eML_blk_kernel_kernel28(const real_T c5_y1_data[784],
  const int32_T c5_bboxesX1Y1X2Y2_size[2], const int32_T c5_loop_ub, real_T
  c5_bboxesX1Y1X2Y2_data[3136]);
static __global__ void c5_eML_blk_kernel_kernel29(const real_T c5_x2_data[784],
  const int32_T c5_bboxesX1Y1X2Y2_size[2], const int32_T c5_loop_ub, real_T
  c5_bboxesX1Y1X2Y2_data[3136]);
static __global__ void c5_eML_blk_kernel_kernel30(const real_T c5_y2_data[784],
  const int32_T c5_bboxesX1Y1X2Y2_size[2], const int32_T c5_loop_ub, real_T
  c5_bboxesX1Y1X2Y2_data[3136]);
static __global__ void c5_eML_blk_kernel_kernel31(const real_T
  c5_bboxesX1Y1X2Y2_data[3136], const int32_T c5_loop_ub, real_T
  c5_bboxPred_data[3136]);
static __global__ void c5_eML_blk_kernel_kernel32(const real_T
  c5_bboxesX1Y1X2Y2_data[3136], const int32_T c5_bboxesX1Y1X2Y2_size[2], const
  int32_T c5_bboxPred_size[2], const int32_T c5_loop_ub, real_T
  c5_bboxPred_data[3136]);
static __global__ void c5_eML_blk_kernel_kernel33(const real_T
  c5_bboxesX1Y1X2Y2_data[3136], const int32_T c5_bboxesX1Y1X2Y2_size[2], const
  int32_T c5_bboxPred_size[2], const int32_T c5_loop_ub, real_T
  c5_bboxPred_data[3136]);
static __global__ void c5_eML_blk_kernel_kernel34(const real_T
  c5_bboxesX1Y1X2Y2_data[3136], const int32_T c5_bboxesX1Y1X2Y2_size[2], const
  int32_T c5_bboxPred_size[2], const int32_T c5_loop_ub, real_T
  c5_bboxPred_data[3136]);
static __global__ void c5_eML_blk_kernel_kernel35(const int32_T c5_nx, real_T
  c5_bboxPred_data[3136]);
static __global__ void c5_eML_blk_kernel_kernel36(const real_T c5_bboxPred_data
  [3136], const int32_T c5_bboxPred_size[2], const int32_T c5_i32, real_T
  c5_b_bboxPred_data[784]);
static __global__ void c5_eML_blk_kernel_kernel37(const real_T c5_bboxPred_data
  [784], const int32_T c5_bboxPred_size[2], const int32_T c5_b_bboxPred_size[1],
  real_T c5_b_bboxPred_data[3136]);
static __global__ void c5_eML_blk_kernel_kernel38(const real_T c5_bboxPred_data
  [3136], const int32_T c5_bboxPred_size[2], const int32_T c5_i35, real_T
  c5_b_bboxPred_data[784]);
static __global__ void c5_eML_blk_kernel_kernel39(const real_T c5_bboxPred_data
  [784], const int32_T c5_bboxPred_size[2], const int32_T c5_b_bboxPred_size[1],
  real_T c5_b_bboxPred_data[3136]);
static __global__ void c5_eML_blk_kernel_kernel40(const real_T c5_bboxPred_data
  [3136], const int32_T c5_bboxPred_size[2], const int32_T c5_i, const int32_T
  c5_b_bboxPred_size[2], const int32_T c5_count, real_T c5_b_bboxPred_data[3136]);
static __global__ void c5_eML_blk_kernel_kernel41(const real32_T
  c5_thresholdedPrediction_data[4704], const int32_T
  c5_thresholdedPrediction_size[2], const int32_T c5_i, const real_T c5_count,
  real32_T c5_classPred_data[784], real32_T c5_scorePred_data[784]);
static __global__ void c5_eML_blk_kernel_kernel42(const int32_T c5_i38, const
  int32_T c5_i39, int32_T c5_idx_data[784]);
static __global__ void c5_eML_blk_kernel_kernel43(const int32_T
  c5_bboxPred_size[2], boolean_T c5_b_data[784]);
static __global__ void c5_eML_blk_kernel_kernel44(const int32_T c5_b_size[2],
  const boolean_T c5_b_data[784], int32_T c5_i3, int32_T *c5_n);
static __device__ int32_T c5_c_threadGroupReduction(int32_T c5_val, uint32_T
  c5_lane, uint32_T c5_mask);
static __device__ int32_T c5_b_shflDown1(int32_T c5_in1, uint32_T c5_offset,
  uint32_T c5_mask);
static __device__ int32_T c5_c_workGroupReduction(int32_T c5_val, uint32_T
  c5_mask, uint32_T c5_numActiveWarps);
static __global__ void c5_eML_blk_kernel_kernel45(const int32_T *c5_n, const
  int32_T c5_bboxPred_size[2], int32_T *c5_nrows);
static __global__ void c5_eML_blk_kernel_kernel46(const int32_T
  c5_bboxPred_size[2], const int32_T *c5_nrows, const int32_T c5_idx_data[784],
  real_T c5_bboxPred_data[3136]);
static __global__ void c5_eML_blk_kernel_kernel47(const real_T c5_bboxPred_data
  [3136], const int32_T c5_bboxPred_size[2], const int32_T c5_b_bboxPred_size[2],
  const int32_T c5_i4, real_T c5_b_bboxPred_data[3136]);
static __global__ void c5_eML_blk_kernel_kernel48(const real_T c5_bboxPred_data
  [3136], const int32_T c5_bboxPred_size[2], real_T c5_b_bboxPred_data[3136]);
static __global__ void c5_eML_blk_kernel_kernel49(const int32_T c5_i45, const
  int32_T c5_i47, int32_T c5_idx_data[784]);
static __global__ void c5_eML_blk_kernel_kernel50(const int32_T
  c5_scorePred_size[1], boolean_T c5_b_data[784]);
static __global__ void c5_eML_blk_kernel_kernel51(const int32_T c5_b_size[2],
  const boolean_T c5_b_data[784], int32_T c5_i6, int32_T *c5_n);
static __global__ void c5_eML_blk_kernel_kernel52(const int32_T c5_i50, const
  int32_T c5_i51, int32_T c5_idx_data[784]);
static __global__ void c5_eML_blk_kernel_kernel53(const int32_T
  c5_classPred_size[1], boolean_T c5_b_data[784]);
static __global__ void c5_eML_blk_kernel_kernel54(const int32_T c5_b_size[2],
  const boolean_T c5_b_data[784], int32_T c5_i9, int32_T *c5_n);
static __global__ void c5_eML_blk_kernel_kernel55(const real32_T
  c5_scorePred_data[784], const int32_T c5_scorePred_size[1], real32_T
  c5_out_data[784]);
static __global__ void c5_eML_blk_kernel_kernel56(const uint32_T c5_dv2[2],
  real_T c5_x1_data[784]);
static __global__ void c5_eML_blk_kernel_kernel57(const real_T c5_bboxPred_data
  [3136], const int32_T c5_bboxPred_size[2], const real_T c5_x1_data[784], const
  int32_T c5_bboxesX1Y1X2Y2_size[2], const int32_T c5_x1_size[1], real_T
  c5_bboxesX1Y1X2Y2_data[3136]);
static __global__ void c5_eML_blk_kernel_kernel58(const real32_T
  c5_classPred_data[784], const real_T c5_x1_data[784], const int32_T
  c5_x1_size[1], real_T c5_y1_data[784]);
static __global__ void c5_eML_blk_kernel_kernel59(const int32_T c5_x1_size[1],
  boolean_T c5_selectedIndex_data[784]);
static __global__ void c5_eML_blk_kernel_kernel60(const real_T
  c5_bboxesX1Y1X2Y2_data[3136], const int32_T c5_bboxesX1Y1X2Y2_size[2], const
  int32_T c5_i61, real_T c5_area_data[784]);
static __global__ void c5_eML_blk_kernel_kernel61(const int32_T
  c5_bboxesX1Y1X2Y2_size[2], const real_T c5_bboxesX1Y1X2Y2_data[3136], const
  int32_T c5_i64, real_T c5_x2_data[784]);
static __global__ void c5_eML_blk_kernel_kernel62(const real_T
  c5_bboxesX1Y1X2Y2_data[3136], const int32_T c5_bboxesX1Y1X2Y2_size[2], const
  int32_T c5_i66, real_T c5_y2_data[784]);
static __global__ void c5_eML_blk_kernel_kernel63(const int32_T c5_i12, const
  int32_T c5_iv[2], boolean_T c5_selectedIndex_data[784]);
static __global__ void c5_eML_blk_kernel_kernel64(const boolean_T
  c5_selectedIndex_data[784], const real_T c5_x1_data[784], const int32_T
  c5_selectedIndex_size[1], boolean_T c5_index_data[784]);
static __global__ void c5_eML_blk_kernel_kernel65(const real_T c5_bboxPred_data
  [3136], const int32_T c5_bboxPred_size[2], const int16_T c5_iv1_data[784],
  const int32_T c5_bboxesX1Y1X2Y2_size[2], const int32_T c5_iv1_size[1], real_T
  c5_bboxesX1Y1X2Y2_data[3136]);
static __global__ void c5_eML_blk_kernel_kernel66(const real32_T
  c5_scorePred_data[784], const int32_T c5_scorePred_size[1], real32_T
  c5_b_scores_data[784]);
static __global__ void c5_eML_blk_kernel_kernel67(const real_T
  c5_bboxesX1Y1X2Y2_data[3136], const int32_T c5_bboxesX1Y1X2Y2_size[2], real_T
  c5_b_bboxes_data[]);
static __global__ void c5_eML_blk_kernel_kernel68(const real32_T
  c5_b_scores_data[784], const int32_T c5_scores_size[1], real32_T
  c5_c_scores_data[]);
static __global__ void c5_DeepLearningNetwork_activations_kernel69(const
  real32_T c5_varargin_1[150528], c5_cell_wrap_18 *c5_r);
static __global__ void c5_DeepLearningNetwork_activations_kernel70(const
  c5_cell_wrap_18 *c5_r, c5_cell_wrap_18 c5_miniBatchT[1]);
static __global__ void c5_DeepLearningNetwork_activations_kernel71(const
  real32_T c5_outMiniBatch[4704], real32_T c5_b_out[4704]);
static __device__ real32_T c5_callFcn_device(real32_T c5_input1, real32_T
  c5_input2);
static __device__ real32_T c5_b_callFcn_device(real32_T c5_input1, real32_T
  c5_input2);
static void c5_checkCleanupCudaError(hipError_t c5_errCode, const char_T
  *c5_file, uint32_T c5_line);
static emlrtRTEInfo c5_createEmlrtInfoStruct(const char_T *c5_file, uint32_T
  c5_line);
static void init_dsm_address_info(SFc5_LaneDetectionInstanceStruct
  *chartInstance);
static void init_simulink_io_address(SFc5_LaneDetectionInstanceStruct
  *chartInstance);

/* Function Definitions */
static void initialize_c5_LaneDetection(SFc5_LaneDetectionInstanceStruct
  *chartInstance)
{
  emlrtLicenseCheckR2012b(chartInstance->c5_fEmlrtCtx,
    "distrib_computing_toolbox", 2);
  emlrtLicenseCheckR2012b(chartInstance->c5_fEmlrtCtx, "neural_network_toolbox",
    2);
  emlrtLicenseCheckR2012b(chartInstance->c5_fEmlrtCtx,
    "video_and_image_blockset", 2);
  sim_mode_is_external(chartInstance->S);
  _sfTime_ = sf_get_time(chartInstance->S);
  chartInstance->c5_yolodetector_not_empty = false;
  chartInstance->c5_is_active_c5_LaneDetection = 0U;
  hipGetLastError();
  hipMalloc(&chartInstance->c5_h_gpu_loop_ub, 4UL);
  hipMalloc(&chartInstance->c5_gpu_scores_data, 3136UL);
  hipMalloc(&chartInstance->c5_gpu_loop_ub, 4UL);
  hipMalloc(&chartInstance->c5_b_gpu_out, 602112UL);
  hipMalloc(&chartInstance->c5_gpu_x1_size, 4UL);
  hipMalloc(&chartInstance->c5_gpu_bv, 784UL);
  hipMalloc(&chartInstance->c5_gpu_x2_data, 6272UL);
  hipMalloc(&chartInstance->c5_gpu_area_data, 6272UL);
  hipMalloc(&chartInstance->c5_gpu_y1_data, 6272UL);
  hipMalloc(&chartInstance->c5_gpu_i38, 4UL);
  hipMalloc(&chartInstance->c5_gpu_out_data, 3136UL);
  hipMalloc(&chartInstance->c5_gpu_ipRowIndices, 4032UL);
  hipMalloc(&chartInstance->c5_c_gpu_end, 4UL);
  hipMalloc(&chartInstance->c5_gpu_x1_data, 6272UL);
  hipMalloc(&chartInstance->c5_gpu_outVal, 8UL);
  hipMalloc(&chartInstance->c5_b_gpu_bboxPred_size, 4UL);
  hipMalloc(&chartInstance->c5_gpu_rowWeightsTotal, 1792UL);
  hipMalloc(&chartInstance->c5_gpu_i35, 4UL);
  hipMalloc(&chartInstance->c5_gpu_In, 4919040UL);
  hipMalloc(&chartInstance->c5_gpu_dv, 8UL);
  hipMalloc(&chartInstance->c5_gpu_ii_data, 1568UL);
  hipMalloc(&chartInstance->c5_gpu_anchors, 64UL);
  hipMalloc(&chartInstance->c5_gpu_i61, 4UL);
  hipMalloc(&chartInstance->c5_gpu_scorePred_size, 4UL);
  hipMalloc(&chartInstance->c5_gpu_thresholdedPrediction_data, 18816UL);
  hipMalloc(&chartInstance->c5_gpu_count, 4UL);
  hipMalloc(&chartInstance->c5_c_gpu_n, 4UL);
  hipMalloc(&chartInstance->c5_c_gpu_loop_ub, 4UL);
  hipMalloc(&chartInstance->c5_gpu_ii_size, 4UL);
  hipMalloc(&chartInstance->c5_gpu_b_data, 784UL);
  hipMalloc(&chartInstance->c5_gpu_n, 4UL);
  hipMalloc(&chartInstance->c5_gpu_out, 602112UL);
  hipMalloc(&chartInstance->c5_gpu_selectedIndex_size, 4UL);
  hipMalloc(&chartInstance->c5_gpu_i14, 4UL);
  hipMalloc(&chartInstance->c5_gpu_ipColIndices, 7168UL);
  hipMalloc(&chartInstance->c5_gpu_classPred_size, 4UL);
  hipMalloc(&chartInstance->c5_gpu_i12, 4UL);
  hipMalloc(&chartInstance->c5_gpu_i10, 4UL);
  hipMalloc(&chartInstance->c5_d_gpu_end, 4UL);
  hipMalloc(&chartInstance->c5_gpu_thresholdedPrediction_size, 8UL);
  hipMalloc(&chartInstance->c5_d_gpu_bboxPred_size, 8UL);
  hipMalloc(&chartInstance->c5_b_gpu_n, 4UL);
  hipMalloc(&chartInstance->c5_gpu_i17, 4UL);
  hipMalloc(&chartInstance->c5_gpu_y2_data, 6272UL);
  hipMalloc(&chartInstance->c5_gpu_end, 4UL);
  hipMalloc(&chartInstance->c5_c_gpu_bboxPred_data, 25088UL);
  hipMalloc(&chartInstance->c5_gpu_b_size, 8UL);
  hipMalloc(&chartInstance->c5_gpu_i4, 4UL);
  hipMalloc(&chartInstance->c5_gpu_rowWeights, 16128UL);
  hipMalloc(&chartInstance->c5_gpu_partialResize, 1290240UL);
  hipMalloc(&chartInstance->c5_d_gpu_bboxPred_data, 6272UL);
  hipMalloc(&chartInstance->c5_gpu_i, 4UL);
  hipMalloc(&chartInstance->c5_gpu_i50, 4UL);
  hipMalloc(&chartInstance->c5_gpu_iv1_data, 1568UL);
  hipMalloc(&chartInstance->c5_gpu_bboxPred_size, 8UL);
  hipMalloc(&chartInstance->c5_gpu_aux1, 1920UL);
  hipMalloc(&chartInstance->c5_gpu_xoffset, 4UL);
  hipMalloc(&chartInstance->c5_gpu_i47, 4UL);
  hipMalloc(&chartInstance->c5_gpu_iv1_size, 4UL);
  hipMalloc(&chartInstance->c5_gpu_dv1, 32UL);
  hipMalloc(&chartInstance->c5_b_gpu_bboxPred_data, 25088UL);
  hipMalloc(&chartInstance->c5_gpu_i9, 4UL);
  hipMalloc(&chartInstance->c5_gpu_bboxPred_data, 6272UL);
  hipMalloc(&chartInstance->c5_gpu_idx_data, 3136UL);
  hipMalloc(&chartInstance->c5_gpu_index_data, 784UL);
  hipMalloc(&chartInstance->c5_d_gpu_loop_ub, 4UL);
  hipMalloc(&chartInstance->c5_b_gpu_count, 8UL);
  hipMalloc(&chartInstance->c5_b_gpu_outVal, 4UL);
  hipMalloc(&chartInstance->c5_gpu_i66, 4UL);
  hipMalloc(&chartInstance->c5_e_gpu_loop_ub, 4UL);
  hipMalloc(&chartInstance->c5_gpu_nrows, 4UL);
  hipMalloc(&chartInstance->c5_b_gpu_loop_ub, 4UL);
  hipMalloc(&chartInstance->c5_gpu_i7, 4UL);
  hipMalloc(&chartInstance->c5_g_gpu_loop_ub, 4UL);
  hipMalloc(&chartInstance->c5_gpu_colWeights, 28672UL);
  hipMalloc(&chartInstance->c5_gpu_scores_size, 4UL);
  hipMalloc(&chartInstance->c5_gpu_nx, 4UL);
  hipMalloc(&chartInstance->c5_gpu_bboxes_data, 80U * sizeof(real_T));
  hipMalloc(&chartInstance->c5_gpu_classPred_data, 3136UL);
  hipMalloc(&chartInstance->c5_gpu_i64, 4UL);
  hipMalloc(&chartInstance->c5_f_gpu_loop_ub, 4UL);
  hipMalloc(&chartInstance->c5_gpu_i32, 4UL);
  hipMalloc(&chartInstance->c5_gpu_i19, 4UL);
  hipMalloc(&chartInstance->c5_gpu_aux2, 3416UL);
  hipMalloc(&chartInstance->c5_e_gpu_bboxPred_data, 25088UL);
  hipMalloc(&chartInstance->c5_gpu_i3, 4UL);
  hipMalloc(&chartInstance->c5_gpu_i51, 4UL);
  hipMalloc(&chartInstance->c5_b_gpu_xoffset, 4UL);
  hipMalloc(&chartInstance->c5_gpu_dv2, 8UL);
  hipMalloc(&chartInstance->c5_gpu_scorePred_data, 3136UL);
  hipMalloc(&chartInstance->c5_b_gpu_end, 4UL);
  hipMalloc(&chartInstance->c5_gpu_colWeightsTotal, 1792UL);
  hipMalloc(&chartInstance->c5_gpu_selectedIndex_data, 784UL);
  hipMalloc(&chartInstance->c5_gpu_boxOut, 18816UL);
  hipMalloc(&chartInstance->c5_gpu_y, 4UL);
  hipMalloc(&chartInstance->c5_c_gpu_bboxPred_size, 8UL);
  hipMalloc(&chartInstance->c5_gpu_iv, 8UL);
  hipMalloc(&chartInstance->c5_b_gpu_anchors, 32UL);
  hipMalloc(&chartInstance->c5_e_gpu_bboxPred_size, 4UL);
  hipMalloc(&chartInstance->c5_gpu_bboxesX1Y1X2Y2_size, 8UL);
  hipMalloc(&chartInstance->c5_gpu_tmpFeatureMap, 18816UL);
  hipMalloc(&chartInstance->c5_b_gpu_scores_data, 20U * sizeof(real32_T));
  hipMalloc(&chartInstance->c5_gpu_i39, 4UL);
  hipMalloc(&chartInstance->c5_gpu_i45, 4UL);
  hipMalloc(&chartInstance->c5_gpu_i6, 4UL);
  hipMalloc(&chartInstance->c5_gpu_bboxesX1Y1X2Y2_data, 25088UL);
}

static void initialize_params_c5_LaneDetection(SFc5_LaneDetectionInstanceStruct *
  chartInstance)
{
}

static void enable_c5_LaneDetection(SFc5_LaneDetectionInstanceStruct
  *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
}

static void disable_c5_LaneDetection(SFc5_LaneDetectionInstanceStruct
  *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
}

static void c5_do_animation_call_c5_LaneDetection
  (SFc5_LaneDetectionInstanceStruct *chartInstance)
{
  sfDoAnimationWrapper(chartInstance->S, false, true);
  sfDoAnimationWrapper(chartInstance->S, false, false);
}

static void ext_mode_exec_c5_LaneDetection(SFc5_LaneDetectionInstanceStruct
  *chartInstance)
{
}

static const mxArray *get_sim_state_c5_LaneDetection
  (SFc5_LaneDetectionInstanceStruct *chartInstance)
{
  const mxArray *c5_b_y = NULL;
  const mxArray *c5_c_y = NULL;
  const mxArray *c5_d_y = NULL;
  const mxArray *c5_st;
  const mxArray *c5_y = NULL;
  c5_st = NULL;
  c5_y = NULL;
  sf_mex_assign(&c5_y, sf_mex_createcellmatrix(3, 1), false);
  c5_b_y = NULL;
  sf_mex_assign(&c5_b_y, sf_mex_create("y", chartInstance->c5_bboxes_data, 0, 0U,
    1U, 0U, 2, (*chartInstance->c5_bboxes_sizes)[0],
    (*chartInstance->c5_bboxes_sizes)[1]), false);
  sf_mex_setcell(c5_y, 0, c5_b_y);
  c5_c_y = NULL;
  sf_mex_assign(&c5_c_y, sf_mex_create("y", chartInstance->c5_scores_data, 1, 0U,
    1U, 0U, 2, (*chartInstance->c5_scores_sizes)[0], 1), false);
  sf_mex_setcell(c5_y, 1, c5_c_y);
  c5_d_y = NULL;
  sf_mex_assign(&c5_d_y, sf_mex_create("y",
    &chartInstance->c5_is_active_c5_LaneDetection, 3, 0U, 0U, 0U, 0), false);
  sf_mex_setcell(c5_y, 2, c5_d_y);
  sf_mex_assign(&c5_st, c5_y, false);
  return c5_st;
}

static void set_sim_state_c5_LaneDetection(SFc5_LaneDetectionInstanceStruct
  *chartInstance, const mxArray *c5_st)
{
  const mxArray *c5_u;
  c5_u = sf_mex_dup(c5_st);
  c5_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getcell(c5_u, 0)),
                      "bboxes", *chartInstance->c5_bboxes_data,
                      *chartInstance->c5_bboxes_sizes);
  c5_c_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getcell(c5_u, 1)),
                        "scores", *chartInstance->c5_scores_data,
                        *chartInstance->c5_scores_sizes);
  chartInstance->c5_is_active_c5_LaneDetection = c5_e_emlrt_marshallIn
    (chartInstance, sf_mex_dup(sf_mex_getcell(c5_u, 2)),
     "is_active_c5_LaneDetection");
  sf_mex_destroy(&c5_u);
  sf_mex_destroy(&c5_st);
}

static void sf_gateway_c5_LaneDetection(SFc5_LaneDetectionInstanceStruct
  *chartInstance)
{
  c5_emxArray_real32_T_20x1 c5_scores;
  c5_emxArray_real_T_20x4 c5_bboxes;
  int32_T c5_i;
  chartInstance->c5_JITTransitionAnimation[0] = 0U;
  _sfTime_ = sf_get_time(chartInstance->S);
  for (c5_i = 0; c5_i < 1229760; c5_i++) {
    chartInstance->c5_fv[c5_i] = (*chartInstance->c5_In)[c5_i];
  }

  c5_eML_blk_kernel(chartInstance, chartInstance->c5_fv, c5_bboxes.data,
                    c5_bboxes.size, c5_scores.data, c5_scores.size);
  c5_emxConvertDynamicMatrixFromEmx_(chartInstance, &c5_bboxes,
    *chartInstance->c5_bboxes_data, *chartInstance->c5_bboxes_sizes);
  c5_b_emxConvertDynamicMatrixFromEmx_(chartInstance, &c5_scores,
    *chartInstance->c5_scores_data, *chartInstance->c5_scores_sizes);
  c5_do_animation_call_c5_LaneDetection(chartInstance);
}

static void mdl_start_c5_LaneDetection(SFc5_LaneDetectionInstanceStruct
  *chartInstance)
{
}

static void mdl_terminate_c5_LaneDetection(SFc5_LaneDetectionInstanceStruct
  *chartInstance)
{
  hipError_t c5_errCode;
  hipFree(*chartInstance->c5_gpu_aux1);
  hipFree(*chartInstance->c5_gpu_iv);
  hipFree(*chartInstance->c5_gpu_index_data);
  hipFree(chartInstance->c5_gpu_i39);
  hipFree(*chartInstance->c5_gpu_aux2);
  hipFree(*chartInstance->c5_gpu_In);
  hipFree(chartInstance->c5_gpu_i32);
  hipFree(*chartInstance->c5_b_gpu_out);
  hipFree(chartInstance->c5_g_gpu_loop_ub);
  hipFree(*chartInstance->c5_gpu_iv1_size);
  hipFree(*chartInstance->c5_e_gpu_bboxPred_size);
  hipFree(chartInstance->c5_f_gpu_loop_ub);
  hipFree(*chartInstance->c5_gpu_rowWeightsTotal);
  hipFree(*chartInstance->c5_gpu_scorePred_size);
  hipFree(chartInstance->c5_h_gpu_loop_ub);
  hipFree(*chartInstance->c5_gpu_bboxPred_data);
  hipFree(*chartInstance->c5_gpu_rowWeights);
  hipFree(*chartInstance->c5_c_gpu_bboxPred_size);
  hipFree(*chartInstance->c5_gpu_tmpFeatureMap);
  hipFree(*chartInstance->c5_gpu_selectedIndex_size);
  hipFree(chartInstance->c5_gpu_i64);
  hipFree(*chartInstance->c5_e_gpu_bboxPred_data);
  hipFree(*chartInstance->c5_gpu_selectedIndex_data);
  hipFree(chartInstance->c5_gpu_i38);
  hipFree(*chartInstance->c5_gpu_classPred_size);
  hipFree(*chartInstance->c5_d_gpu_bboxPred_data);
  hipFree(chartInstance->c5_gpu_i9);
  hipFree(*chartInstance->c5_gpu_ii_size);
  hipFree(chartInstance->c5_gpu_end);
  hipFree(*chartInstance->c5_gpu_thresholdedPrediction_data);
  hipFree(*chartInstance->c5_gpu_colWeights);
  hipFree(chartInstance->c5_gpu_i);
  hipFree(*chartInstance->c5_d_gpu_bboxPred_size);
  hipFree(chartInstance->c5_b_gpu_count);
  hipFree(*chartInstance->c5_gpu_colWeightsTotal);
  hipFree(chartInstance->c5_gpu_nx);
  hipFree(chartInstance->c5_gpu_n);
  hipFree(chartInstance->c5_d_gpu_loop_ub);
  hipFree(chartInstance->c5_gpu_i66);
  hipFree(*chartInstance->c5_gpu_out_data);
  hipFree(*chartInstance->c5_gpu_out);
  hipFree(*chartInstance->c5_gpu_ipColIndices);
  hipFree(chartInstance->c5_gpu_count);
  hipFree(chartInstance->c5_gpu_i35);
  hipFree(chartInstance->c5_gpu_i19);
  hipFree(chartInstance->c5_gpu_nrows);
  hipFree(*chartInstance->c5_gpu_scores_data);
  hipFree(*chartInstance->c5_gpu_b_size);
  hipFree(*chartInstance->c5_gpu_ipRowIndices);
  hipFree(*chartInstance->c5_gpu_x1_data);
  hipFree(*chartInstance->c5_gpu_ii_data);
  hipFree(*chartInstance->c5_gpu_scores_size);
  hipFree(*chartInstance->c5_gpu_dv);
  hipFree(chartInstance->c5_b_gpu_loop_ub);
  hipFree(chartInstance->c5_gpu_i12);
  hipFree(*chartInstance->c5_gpu_partialResize);
  hipFree(*chartInstance->c5_gpu_classPred_data);
  hipFree(*chartInstance->c5_gpu_dv1);
  hipFree(chartInstance->c5_gpu_i50);
  hipFree(*chartInstance->c5_gpu_x1_size);
  hipFree(chartInstance->c5_b_gpu_scores_data);
  hipFree(chartInstance->c5_gpu_i14);
  hipFree(chartInstance->c5_gpu_i6);
  hipFree(chartInstance->c5_gpu_xoffset);
  hipFree(*chartInstance->c5_gpu_dv2);
  hipFree(*chartInstance->c5_gpu_thresholdedPrediction_size);
  hipFree(*chartInstance->c5_gpu_bboxesX1Y1X2Y2_data);
  hipFree(chartInstance->c5_gpu_i7);
  hipFree(*chartInstance->c5_gpu_bv);
  hipFree(*chartInstance->c5_gpu_anchors);
  hipFree(chartInstance->c5_b_gpu_outVal);
  hipFree(*chartInstance->c5_gpu_scorePred_data);
  hipFree(chartInstance->c5_c_gpu_n);
  hipFree(*chartInstance->c5_c_gpu_bboxPred_data);
  hipFree(*chartInstance->c5_gpu_idx_data);
  hipFree(*chartInstance->c5_gpu_b_data);
  hipFree(chartInstance->c5_gpu_i10);
  hipFree(chartInstance->c5_gpu_i3);
  hipFree(chartInstance->c5_gpu_loop_ub);
  hipFree(chartInstance->c5_b_gpu_end);
  hipFree(chartInstance->c5_d_gpu_end);
  hipFree(chartInstance->c5_b_gpu_xoffset);
  hipFree(chartInstance->c5_b_gpu_n);
  hipFree(chartInstance->c5_c_gpu_end);
  hipFree(chartInstance->c5_gpu_bboxes_data);
  hipFree(*chartInstance->c5_gpu_bboxPred_size);
  hipFree(chartInstance->c5_gpu_i47);
  hipFree(chartInstance->c5_e_gpu_loop_ub);
  hipFree(*chartInstance->c5_gpu_iv1_data);
  hipFree(*chartInstance->c5_gpu_area_data);
  hipFree(*chartInstance->c5_b_gpu_bboxPred_size);
  hipFree(chartInstance->c5_c_gpu_loop_ub);
  hipFree(*chartInstance->c5_gpu_outVal);
  hipFree(*chartInstance->c5_b_gpu_anchors);
  hipFree(chartInstance->c5_gpu_i61);
  hipFree(chartInstance->c5_gpu_i4);
  hipFree(*chartInstance->c5_b_gpu_bboxPred_data);
  hipFree(*chartInstance->c5_gpu_y2_data);
  hipFree(*chartInstance->c5_gpu_boxOut);
  hipFree(chartInstance->c5_gpu_i51);
  hipFree(chartInstance->c5_gpu_i45);
  hipFree(*chartInstance->c5_gpu_y1_data);
  hipFree(chartInstance->c5_gpu_i17);
  hipFree(chartInstance->c5_gpu_y);
  hipFree(*chartInstance->c5_gpu_bboxesX1Y1X2Y2_size);
  hipFree(*chartInstance->c5_gpu_x2_data);
  c5_errCode = hipGetLastError();
  if (c5_errCode != hipSuccess) {
    emlrtThinCUDAError(c5_errCode, hipGetErrorName(c5_errCode),
                       hipGetErrorString(c5_errCode), "SimGPUErrorChecks",
                       chartInstance->c5_fEmlrtCtx);
  }
}

static void mdl_setup_runtime_resources_c5_LaneDetection
  (SFc5_LaneDetectionInstanceStruct *chartInstance)
{
  setLegacyDebuggerFlag(chartInstance->S, false);
  setDebuggerFlag(chartInstance->S, false);
  sim_mode_is_external(chartInstance->S);
}

static void mdl_cleanup_runtime_resources_c5_LaneDetection
  (SFc5_LaneDetectionInstanceStruct *chartInstance)
{
}

static void initSimStructsc5_LaneDetection(SFc5_LaneDetectionInstanceStruct
  *chartInstance)
{
}

const mxArray *sf_c5_LaneDetection_get_eml_resolved_functions_info()
{
  const mxArray *c5_nameCaptureInfo = NULL;
  const char_T *c5_data[45] = {
    "789ced9dc96f2bc97dc739c638193b9e19c59389c7419637882ff1244f24b550f225269ba4448a3b297179089e9acd22d962b3bbd90bb74b989b811c72c8c540"
    "f6c54e8c38f6047082acc835a75c72c97f906be0532e06d25c4a22fbb15f735464bd56f1d7c04cbfd6af55df5f2daa4fd7ee7b2b917ecbe7f3bd67fdf75fbfed",
    "f3fdc7d73fe79b5eeffae6d7c1e2fe39dfea65b7bfb5b8576dcff8fabcefed95df9bdaffd3d2fbbdc5b3a0c8061a1af30799efa2fbdf6c285d51e665a3345291"
    "4f43ba22f5516366698a122a895d545c7ec84c9fbaf125d3fdc3d434fd37d74642a768767d5a5b7ff0505a7e98a5c7f4ba7588efdb2ee961bfece9617f0feb4d",
    "1ea987c3ffaa8b1eb6f7515b1424f4b2810c2418a222bf1c2992f2b21fb4f9734be8cf4f39fa33b7e8a2dc92d083deb709f58e1cf556ed2f62bf75d856bae8b0"
    "dd15a4c370745afe34453a744c97e7ddd9efb9a5cbfb1bfa69bf3fbcffceecfea37f7ffb254d3d7ced8bded021bc4dcbd9cf3be81dd8ec6629d7e1e2289e4b5c",
    "c5cef944add82be75bb1073f722e3a6e7ef81c9e6985ffdf0ebfbf693af61cc23fb0d95f242ad65faca96b879222f0d2613a5c4a85238785a03fe8af1f1a8a22"
    "d595e1a1dee635d4386c4882d240dacb3aafa3d5874f66ff3efc44b468a3c9563892c237a208a929c46bb2551765903150b4ce73d5164fd2f2f2814b3cb17de6",
    "dff36a3695ed0717beacf8714be8c7dbb6e7073fe616c14ac165bdff25d4fb5d47bd55fb8b04b75c214715c1ec22d9d07146174d55553423c70b1dbe85f44d32"
    "5e9fff8a8a7f65fececb86f4d2e0b516b2c25e948695d45ee43cad7ae9d77ef231d4f33bd423fdbbfdd041efc066bf29b5ee7ae7c75757ed7845a8f74ffd8da3",
    "78d2c74e3dbfeff5c0b708e3ffb14bfcb1bdcb1b125fe72c7f5a48ce581fa5a6ccd72594d31415698688701b85767b61cea535a4b4f9f3ce5b64fe7cd7c51f6c"
    "7f93e5e49b6b92e1d0e1238256bdf78d4f7f0c1cd9a11eadf64267d0ea954c2e5b0b85c57ce1265069f2f9bb0be0082b1cc1ed8ac7c6ff5ddbb33dfed86ef9b3",
    "70205bbff3012f8017f87de0050d3de0c576c2075e90c5ff4b2ef1c776cb9fd2ccab94585fd69f10ea032f80178fd1c3d7bee8012fb613febef382b41cfdac4b"
    "fcb1fd9e1751dee06743f3be153f26847e0037801b8fd1c3d7bee80137b613febe7383b49df14597f863bbe54f223b1dcd580c64002f8017f3f7811734f48017",
    "db097fdf79415a8ebee2127f6c17a67375e38a56443d13c902c2b5357003b8317f1fb841430fb8b19df0f79d1ba4ed0ca7f43eb0dd1fda19163c0a990cf00278"
    "317f1f7841430f78b19df0811764f17fd7f66c8f3fb60fac9a2821aba611d65aba0ffaa58017f87de0050d3de0c576c2df775edc12c6ff1ddbf343fce796e90c",
    "297e8434e003f061fe3ef081861ef0613be1ef3b1f48db139f619c3b67399b31bbbe15fd09a13ef00278f1183d7ced8b1ef0623be1032fc8e2ff332ef1c7f681"
    "c6ab697e369b76457f42a80fbc005e3c460f5ffba207bcd84ef8fbce8b0961fc7fc125fed8dee725b1c11b783e545cd13889d775b139f26dd51fe007f0e3317a",
    "f8da173de0c776c2df777e90b637361defb6fc990123c377118c77032f1ede075ed0d0035e6c27fc7de7c58430fe9bb63756f6254c2903a49594a2a199c262cb"
    "76e007f063fe3ef083861ef0633be1033fc8e2bf697dad6a48b5dc9b4db12d8a63345d96c1a13e2f2decc00fe0c7fc7de0070d3de0c776c2df777e90f657bd67",
    "7bb6c71fdbf5fbfd6c33b3c3a18017c08bf9fbc00b1a7ac08bed84bfefbcb8258cffe71de33fb7589c30551ff001f880df073ed0d0033e6c27fc7de703697be2"
    "0b2ef1c7768197a4e28215cbfa13427de005f0e2317af8da173de0c576c2075e90c57fd3f5190b5e4c872e56f42784fac00be0c563f4f0b52f7ac08bed840fbc",
    "208bffa6e315535e14906e11c3e00d18af005edcbf0fbca0a107bcd84ef8c00bb2f8bf6f7bb6c71fdba7bcb856a72bfae6c0005e002fe6ef032f68e8012fb613"
    "3ef0822cfe9b9ed76a68bcacab8a8e6e786d597f42a80fbc005e3c460f5ffba207bcd84ef8c00bb2f83ba5f781edae22ad6b1af3f517d34da68017c08bf9fbc0",
    "0b1a7ac08bed84bfefbca075bef7821759d3580063f173e0067063fe3e7083861e70633be10337c8e2ff814bfcb1dda9a6066e0037e6ef033768e80137b613fe"
    "be7383b47f6ad3f332b4a5b952cbfa1342fd6df3c247c88b3f76f107db5f24b21e03c6430ed1e4c4273ff91838b1433d5a9cb81c57ab9d42b956bf2905e3fc9d",
    "70a7d70bb1087082154e9096a30f5de28fedaae59a281861b9b13c5bcabbbc3820e4c5f75cfcc176eff1626d4e3dc7df13b4eabbaf033f76aa478b1f47e7f960"
    "63acb70b47f15a4c4e8d9ae679297309fc60851fb784f1773bb755589c8ee1554e90b62bfed0c51f6cf71e2770cecc0b04ad7aecd7810b3bd5a3c58598323ccd",
    "c76f0a7e14bab8cbd44f2f03e39b28b42b98e10269ffd3a6ebbb79c110fbbc212ab2bea23f21d4f71a27fec4c51f6cf71e279672c82a23d0ffc4861e2d4ef4af"
    "63f56268a070c994d91012d2c8a8763b304ec10c276e09e3ffd38ef19f5b16bd19cc72e10f5cfcc176ef716191331b9607e0c2d3d0a3c5855ef142ca0d635ca3",
    "52686979e44f2b83443f0e5c60850b13c2f86f7a3ed274fd766e5e134d4fb6307543e9a6f811d2e6cd09cf7283747ce2072efe60bbf7b8e19c6334c7b7bfff2f"
    "bfc34feffb52afd3d6a3c511ed329d37228df3e3a4921af5c227ad7ea29982f605331ca1b5ee62a997c3aa950a99cce2e75ee507e9fcd9efb8f883eddee3c72b",
    "39352b31307ec1861e2d6e704901b52451ca47844ab4938b5e3753ad018c5f003716f7c77183036e2cee5ee70647991bd06fb55b3d5adcc81c8ff47cd1a8dc59"
    "a5bad4ed5f0ed241e518b8c10c3754c2f8bf6b7bb6c71fdbd5fb1e90795b83555efc998b3fd8ee3d5eace410f5f9b3c08bddead19a37df2e64cdd29598c8c9da",
    "715193c3478671b63c6f1e78b13efca7c20b5ae568da6b1e5ed347e5556eeceffa8bb53905fc604c8f567b63943e1644f9a6622852e4a89fedb7a2834a240afc"
    "007eccaf4dfba956475d617cc3bbfd54afe414f45331a447ad9f8a6fab85b178911487dc701cd7d50b23ad70c00d56b8f12dc2f87fcd25fed8ae2aba91d31401",
    "e9fa7c4fc292524086a9c91c6fa096a2895675b5ecd784d0af6d73e423428efca38b3fd8ee3d8e6c9273cf555af520ec3bb55b3d5a5cc95ede64d25dfe3a729e"
    "8a66eeeaea55b07fa9c4802bac708574fce33ddbb33dfed8beda4be25d7e90b643fedcc51f6cf71e3f5673087760c13c2b36f468f55b9f96c56c58ca1e5d779a",
    "cab86aaad56a4e4bf88017c08bf9f559ce015ff48aace84f08f5bdb61ef0e9ae135fca215827ce8c1ead76c575ae5091cfe2d9d2205bbbcbe86781b861c03a0e"
    "e0048eef175ce28fed3a32c2b2d056b4c533ab9cf823177fb0dd7b9cb8cf2158b7c1901eb5751b42e26cd490ae5b4954eff42e4ddd2c8dd2b06e1c38b1b86f7a",
    "eeb7550b2d1c288a63a47b9613a4fd4f7fe1e20fb67b9213cb3944799f11e0c56ef568f12224b574ff911e3732e9b2503d8df2ba1c5561fe1433bc9810c6ff99"
    "4bfcb15de5357d7e8aabce594eb590bc34c7b390c978951fa4e3df3f72f107dbbdc70fb71c837d6f59d2a3c593527e6cd4d37cb8cfc54eba91ce50c8a7b308fa",
    "a980278bfb3397f863fb6b6b270e78f2c478c2d1e6098c7bec568f164ffc4d533b19a8edd665261149e493ddab734d8279bac093c5fd175de28fedafd64e8bb1"
    "d8c596245ee509e93ac14f5dfcc1f6a7c093871ca3372f1738b25b3d5a1c394d97f267d5786a3812f2278d2e7fc70fae25d8970438b2b81373a4001c995e4f88",
    "23059a1c81feadddead1e2c85558ba280e22a7c689bf99a86a69a96c461bd01e618623a4e5e8cb2ef1c7761d19b33a693a76bbbce0dcabfc201d67ffae8b3fd8"
    "ee3d7ebc9a5334f76187736277ab478b1b41de2885c2c7e3d30c17523afebc30ea6578687f30c30d6aebcde72b96976ba475db5e7996237bbcde7c839c83f608",
    "2b7ab4b862268decf06690322fd2c7055468fbfd49a90ff3b798e1ca8430fecf5ce28fed6b6aa7954d9658e5c9131e6f77c93198bfc5921eadf5e8915a52e48c"
    "6bee94e7637ca636f68712c8f4014f58e1894a18ff2fb9c41fdb67bdee4b55b657f9e123e4c79fbaf883ed1ee4c7520ec1783a2b7ab4da1d6ab35f1e864be7dd",
    "58ef5816f4f35820d60bc33a43663841eb7bc3f267b972e214b929b67cdee5c5feeedbbe36a760df76c6f4a8cdc752cfa3b9f455dbec8a662891cede1d1b99e6"
    "25f083157e4c08e3bfed7afb96d09fb76dcf0ffecc2d829548cb7af4c6cb3d8689353fa3395e0efbe8ee568f161f3a8356af6472d95a282ce60b37814a93cfdf",
    "31b48ef0d6e1f7ddd2d17e39a5e3bee97dfb917a38fc33173d6c7f71bd5cdd86a382221b9a221d8e1449e9070b48b7eabc13ff0d6a8b82846243beab4ae87997"
    "3768fdbdfecf173fac4fefb4f4bef997cf8234f5f0c57afd373c1bf77b8d7a24ddac68f59a983ded0b5a90a17dc49f117e1ffdab43f80736fb8b44ec097c1f7d",
    "2c16504bd40da445918ae406920511e9cbe93571488fa7f63d6dbf36addfe17b1abea759d483efe9ed840f3cf96c3c212d771fb9a417b637a445249e0b664396"
    "e7ff0fabe29be2c8ff3d520f87fffb2e7ad84e81232dd59c5b974bca7d7a5b85669ad48738c129ef0ff89bd04fbf533d5adc40a1402492e90be7fd54fe2e90e3",
    "d4663bd460687ea98f901b4ff73c02fe61f2f9d27904b4b8b0c6a36b4394f46db5777ed9c50f6c776cefccbc7913eba7bd554c66c9b0ee236366a0376fe83780"
    "273bd5a336ee2b248b15a11cf6a38b1bff580fe6dac7259ea17943c093559ea80ef1dd343ddeb53ddbd303db2df7ae65b167a20cdf45beedf59b6d9b237bbb8e",
    "7a0e92956ca2d98f05eb1376ab478b1f81de383a1a5e574bc1260a088948b37e745e66681d35f063951ff4d65523d5f2f6060986a2654da3381bb95d5da33b7d"
    "cfab5c21edfffc37177fb0dd73e566ce954db28fea794ad05ed9ad1eadf9e9354da885cc1a6ad6f36143aa9a5ac69496bf2f8137ebc3df57defcaa4b7a60fba2",
    "c24a742dcf5f831b6679b3f9789bc7cacd0a6f5e977d74d75f036f76ab478b37c18c1cecd4a55eeb9c2bb54bd95052bac82b3ee00dabbc212d573fe7921ed8be"
    "a8b00a990c8724c9aab254737650b857f9423adef2b72efe60bbe7cac90a5f56b20bce8365488f567f59d1e0bb03e15ab88ea1d3f3981ee7d205a102e3f7ccf2",
    "64e210df4dd3e3975cd203db7105a50cd2fcdd6cbfa179dfcb6c3b229f77b942ba5fd43fb8f883ed9e2b2fab5c599b6d34c763a0bdb25b3d5a7ca9f84797cde0"
    "1977c49f2692f960e7aa972a94613c1ff8e2901e1b9fb3f14a4535eb6bc17861962f7fefe20fb67baebc38f1e521db60fc85213d5a7c396fe764fe2a6e4a5ce0",
    "2837aab7d227c9b2ccd03a48e0cb76f9f2cc253db07d5151718a6476e535dfc25ee50be9b8cb3fb9f883ed9e2b2f2b7c71cc36eafb51016776ab478b3342b659"
    "192bc742ece8acd3ebe6838172fd08413b0638e3901ebfe2921ed8beb6c25a6aca7895337b7b4ec76b38b3dc9401beb0a1476b5cffd2a80989e8b97c64a8d171",
    "30de2ff753c9e57e72e0cbfaf09f2a5fa8edc33cafa8d2a22cd6794368e313a0bcca15d271fd1fbaf883ed9e2b272b5cb165d752b9b975883ff0e469e8d1fabb"
    "37d4443c1812c4f3aad62f9a916063ac8c0486f6cf059e6c775ef2c72ee981ed8b0a6a3ee3c8e970419f77f942da3ff6cf2efe60bbe7cacd0a5f9cb30f779001",
    "6fd8d0a3c59bd4d59d1938c925a58a74972fdd04cee5be5103de30cb9b89437c77348fecbec25a39c2cee75dcec03c32e76c837964ece8d1ea1feb8e4e4a5c35"
    "7beacfd4c799a6111706274adc077c61952fa4e5ea0397f4c076cb3dced40da59be247c8aaa614d5dbeb2a49fbc7beefe20fb67bae9cdcef0f63cf2e5847c992",
    "1e2d9e24f9d3ce15e202a1cbb891ace48eb5d4f115aca3649727aa437cb7bdcf98d0464227d18c2b5203357cdee5c89eef33b6924dd01e61478f567f57550ed5"
    "5af5ae68d4447d90ea0c4f7823e0877593ccf283e279b7f34e93a2389e6e1832fbcef5799723a4ed911fb8f883ed9e2b27f7ed9157b30b78c28e1e2d9e6443a3",
    "8b30e2d049a314bb4a078ff399233306e327c01387f4f8b24b7a60bbe55e223beb275982898f5d9efc8d8b3fd8eeb97272cf135b76513e970578b25b3d5a3c69"
    "2786d9b07ce7efc5c6d9e4b111d2a5c1a80beb5580270ee9f11597f4c076cbbda2e53b6a24b2b3ea2921374401e9ccf2e453177fb0dd73e5e49e27ebb20bd6a7",
    "b0a2478b27fa55406ce997c961e03897ce1d19c3fca8dd877df599e509adf112cb3d5c31a16959f62a47f67cbc64259ba07f8b1d3d5afddabd52ae122c70c9cb"
    "7622d6ea66c481ace6ab3ee007f0637d7a38a5ff81ed6eb9375d2c17992e964bc84dc5abfc206d877ccfc51f6cf75cf9b8e7c74a36c13e2c8ce9513b6f38d939",
    "ad996638d00fd6457d9cce374ee21af46b31cb9189437cb7bdce64da51827a269205944272cb682fb65687fd8ae797e7cacb43ff9663b6413b851d3d5a7cb9ba"
    "b92b97a2dd48779c45464cbbd6b8f3e68803beb0ca17d276cafbb6677b7a60fbfd44a1c5ca45aff284b49febaf5dfcc176cf950fdb7c2ebc5211f8c1861ead7e",
    "ae316f48a5b2163ebf307b3ddd10cf8a032ee2037eb0ca0feaf382e388374c0d4de79b5a9f259ee508cc0b7e35bba03dc28e1eadf60857d5fdda4de4fa941bd6"
    "abed6ba9701150d3302f98599ed03acf7e5d05b5667b15cff2e51961b979e2e7d96f927d709e0a437ab4785388c6fdb5fe597edcbd532e46a893ca84d5189ca7",
    "c22c6f28ce1b5e5928b7a8a93ccb179837bc2ebba03f8c153d5afd168562ed7814290dab5c372c8df28156a8d15ede170378b23efc7de5c9a6e7d5cf37f0901f"
    "6a288fb757f6fcbcfa75d905e7d533a4478b27b5f3cb7ef5a6df8a5f8b52b95fbb92b574a2e0039eb0ca13d521be9ba6c77bb6677b7a60fb6a05e55d8e908ecf",
    "ff958b3fd8eeb9f2b18e23702e0a337ab4fab7065781bbeb5ab414e2c7e966395bce044e0c1ef6ed62961fb4be4bfabc24367803e1d9a85e9f37bce7e3f36bb3"
    "0bc6e7d9d1a3c5936287bf190f95e8b05c1eb68e32e958f4d82c014f98e509697bc429fd0f6c775c41cd565d87b516ab1c79e2eb1aedd9447d5d23f46bed568f",
    "16476e02e3561b0dbb47e58b684036235721bd52b9008eb0ca11d272f5914b7a60fbacc3242c37ca96537145bbe4a5e6ec5bd7ab3c215dcff8772efe60bbe7ca"
    "c952ffd69aec82f5f28ce9d1ea97a88f9551aead4bd5c66545bdaae58f32853c8cbfb3cb9589437cdf547d7efb487fec97933ff8a23f6fcb5bc5618e0f2703cc",
    "db62458f567be45448162b4239ec471737feb11eccb58f4b3cecb302dc70488fafbaa407b63b7263617f53dca0b7bfa3c78ac39a9fd11c07f9c6a73f065eec50"
    "8f162f3a8356af6472d95a282ce60b37814a93cfdf31d47f455a3f7cc721fc039bddd3bc882b1a679bb73971883770e3f57ac08dd7a70b70e3cdea0137b6133e",
    "706337dc807eaad7eb413fd5faf4817e2a3a7ad04f4516feff0305eebda9", "" };

  c5_nameCaptureInfo = NULL;
  emlrtNameCaptureMxArrayR2016a(&c5_data[0], 142048U, &c5_nameCaptureInfo);
  return c5_nameCaptureInfo;
}

static void c5_eML_blk_kernel(SFc5_LaneDetectionInstanceStruct *chartInstance,
  real32_T c5_b_In[1229760], real_T c5_b_bboxes_data[], int32_T c5_bboxes_size[2],
  real32_T c5_b_scores_data[], int32_T c5_scores_size[2])
{
  static real_T c5_dv1[4] = { 3.6875, 1.375, 1.8125, 6.8125 };

  static int8_T c5_dv[8] = { 59, 22, 29, 109, 43, 18, 23, 84 };

  c5_yolov2ResNet50VehicleExample0_LaneDetection0 *c5_iobj_0;
  c5_yolov2ResNet50VehicleExample0_LaneDetection0 *c5_this_Network;
  dim3 c5_ab_block;
  dim3 c5_ab_grid;
  dim3 c5_b_block;
  dim3 c5_b_grid;
  dim3 c5_bb_block;
  dim3 c5_bb_grid;
  dim3 c5_block;
  dim3 c5_c_block;
  dim3 c5_c_grid;
  dim3 c5_cb_block;
  dim3 c5_cb_grid;
  dim3 c5_d_block;
  dim3 c5_d_grid;
  dim3 c5_db_block;
  dim3 c5_db_grid;
  dim3 c5_e_block;
  dim3 c5_e_grid;
  dim3 c5_eb_block;
  dim3 c5_eb_grid;
  dim3 c5_f_block;
  dim3 c5_f_grid;
  dim3 c5_fb_block;
  dim3 c5_fb_grid;
  dim3 c5_g_block;
  dim3 c5_g_grid;
  dim3 c5_gb_block;
  dim3 c5_gb_grid;
  dim3 c5_grid;
  dim3 c5_h_block;
  dim3 c5_h_grid;
  dim3 c5_hb_block;
  dim3 c5_hb_grid;
  dim3 c5_i_block;
  dim3 c5_i_grid;
  dim3 c5_ib_block;
  dim3 c5_ib_grid;
  dim3 c5_j_block;
  dim3 c5_j_grid;
  dim3 c5_jb_block;
  dim3 c5_jb_grid;
  dim3 c5_k_block;
  dim3 c5_k_grid;
  dim3 c5_kb_block;
  dim3 c5_kb_grid;
  dim3 c5_l_block;
  dim3 c5_l_grid;
  dim3 c5_lb_block;
  dim3 c5_lb_grid;
  dim3 c5_m_block;
  dim3 c5_m_grid;
  dim3 c5_mb_block;
  dim3 c5_mb_grid;
  dim3 c5_n_block;
  dim3 c5_n_grid;
  dim3 c5_nb_block;
  dim3 c5_nb_grid;
  dim3 c5_o_block;
  dim3 c5_o_grid;
  dim3 c5_ob_block;
  dim3 c5_ob_grid;
  dim3 c5_p_block;
  dim3 c5_p_grid;
  dim3 c5_pb_block;
  dim3 c5_pb_grid;
  dim3 c5_q_block;
  dim3 c5_q_grid;
  dim3 c5_qb_block;
  dim3 c5_qb_grid;
  dim3 c5_r_block;
  dim3 c5_r_grid;
  dim3 c5_rb_block;
  dim3 c5_rb_grid;
  dim3 c5_s_block;
  dim3 c5_s_grid;
  dim3 c5_sb_block;
  dim3 c5_sb_grid;
  dim3 c5_t_block;
  dim3 c5_t_grid;
  dim3 c5_tb_block;
  dim3 c5_tb_grid;
  dim3 c5_u_block;
  dim3 c5_u_grid;
  dim3 c5_ub_block;
  dim3 c5_ub_grid;
  dim3 c5_v_block;
  dim3 c5_v_grid;
  dim3 c5_vb_block;
  dim3 c5_vb_grid;
  dim3 c5_w_block;
  dim3 c5_w_grid;
  dim3 c5_wb_block;
  dim3 c5_wb_grid;
  dim3 c5_x_block;
  dim3 c5_x_grid;
  dim3 c5_y_block;
  dim3 c5_y_grid;
  real_T c5_b_bboxPred_data[3136];
  real_T c5_bboxPred_data[3136];
  real_T c5_bboxesX1Y1X2Y2_data[3136];
  real_T c5_area_data[784];
  real_T c5_x2_data[784];
  real_T c5_y1_data[784];
  real_T c5_y2_data[784];
  real_T c5_areaOfIntersect;
  real_T c5_count;
  real_T c5_height;
  real_T c5_width;
  int32_T c5_idx_data[784];
  int32_T c5_b_size[2];
  int32_T c5_bboxPred_size[2];
  int32_T c5_bboxesX1Y1X2Y2_size[2];
  int32_T c5_d_bboxPred_size[2];
  int32_T c5_e_bboxPred_size[2];
  int32_T c5_inDims[2];
  int32_T c5_iv[2];
  int32_T c5_iv1[2];
  int32_T c5_thresholdedPrediction_size[2];
  int32_T c5_b_bboxPred_size[1];
  int32_T c5_b_scores_size[1];
  int32_T c5_c_bboxPred_size[1];
  int32_T c5_classPred_size[1];
  int32_T c5_ii_size[1];
  int32_T c5_index_size[1];
  int32_T c5_iv1_size[1];
  int32_T c5_scorePred_size[1];
  int32_T c5_selectedIndex_size[1];
  int32_T c5_x1_size[1];
  int32_T c5_x2_size[1];
  int32_T c5_y1_size[1];
  int32_T c5_y2_size[1];
  int32_T c5_b_end;
  int32_T c5_b_i;
  int32_T c5_b_i14;
  int32_T c5_b_i7;
  int32_T c5_b_j;
  int32_T c5_b_k;
  int32_T c5_b_k0;
  int32_T c5_b_n;
  int32_T c5_b_nx;
  int32_T c5_b_nxin;
  int32_T c5_b_partialTrueCount;
  int32_T c5_b_trueCount;
  int32_T c5_c_i;
  int32_T c5_c_j;
  int32_T c5_c_k;
  int32_T c5_c_n;
  int32_T c5_c_partialTrueCount;
  int32_T c5_c_trueCount;
  int32_T c5_currentBox;
  int32_T c5_d_i;
  int32_T c5_d_k;
  int32_T c5_e_i;
  int32_T c5_e_k;
  int32_T c5_end;
  int32_T c5_f_i;
  int32_T c5_f_k;
  int32_T c5_g_i;
  int32_T c5_g_k;
  int32_T c5_h_i;
  int32_T c5_h_k;
  int32_T c5_i;
  int32_T c5_i1;
  int32_T c5_i10;
  int32_T c5_i11;
  int32_T c5_i12;
  int32_T c5_i13;
  int32_T c5_i14;
  int32_T c5_i17;
  int32_T c5_i19;
  int32_T c5_i25;
  int32_T c5_i26;
  int32_T c5_i27;
  int32_T c5_i3;
  int32_T c5_i32;
  int32_T c5_i35;
  int32_T c5_i39;
  int32_T c5_i4;
  int32_T c5_i47;
  int32_T c5_i5;
  int32_T c5_i51;
  int32_T c5_i6;
  int32_T c5_i61;
  int32_T c5_i64;
  int32_T c5_i66;
  int32_T c5_i7;
  int32_T c5_i8;
  int32_T c5_i9;
  int32_T c5_i_i;
  int32_T c5_idx;
  int32_T c5_ii;
  int32_T c5_j;
  int32_T c5_j_i;
  int32_T c5_k;
  int32_T c5_k0;
  int32_T c5_n;
  int32_T c5_nrows;
  int32_T c5_nrowx;
  int32_T c5_nx;
  int32_T c5_nxin;
  int32_T c5_nxout;
  int32_T c5_partialTrueCount;
  int32_T c5_sortDim;
  int32_T c5_trueCount;
  real32_T c5_tmpFeatureMap[4704];
  real32_T c5_c_scores_data[784];
  real32_T c5_classPred_data[784];
  real32_T c5_scorePred_data[784];
  real32_T c5_outVal[2];
  real32_T c5_y;
  uint32_T c5_dv2[2];
  int16_T c5_ii_data[784];
  int16_T c5_iv1_data[784];
  boolean_T c5_b_data[784];
  boolean_T c5_bv[784];
  boolean_T c5_index_data[784];
  boolean_T c5_selectedIndex_data[784];
  boolean_T c5_ab_validLaunchParams;
  boolean_T c5_area_data_dirtyOnGpu;
  boolean_T c5_b_bboxPred_data_dirtyOnGpu;
  boolean_T c5_b_bboxPred_size_dirtyOnCpu;
  boolean_T c5_b_data_dirtyOnCpu;
  boolean_T c5_b_data_dirtyOnGpu;
  boolean_T c5_b_n_dirtyOnGpu;
  boolean_T c5_b_validLaunchParams;
  boolean_T c5_bb_validLaunchParams;
  boolean_T c5_bboxPred_data_dirtyOnCpu;
  boolean_T c5_bboxPred_data_dirtyOnGpu;
  boolean_T c5_bboxPred_size_dirtyOnCpu;
  boolean_T c5_bboxesX1Y1X2Y2_data_dirtyOnGpu;
  boolean_T c5_bboxesX1Y1X2Y2_size_dirtyOnCpu;
  boolean_T c5_bboxes_data_dirtyOnGpu;
  boolean_T c5_bv_dirtyOnGpu;
  boolean_T c5_c_bboxPred_size_dirtyOnCpu;
  boolean_T c5_c_validLaunchParams;
  boolean_T c5_cb_validLaunchParams;
  boolean_T c5_classPred_data_dirtyOnCpu;
  boolean_T c5_classPred_data_dirtyOnGpu;
  boolean_T c5_d_validLaunchParams;
  boolean_T c5_db_validLaunchParams;
  boolean_T c5_e_validLaunchParams;
  boolean_T c5_eb_validLaunchParams;
  boolean_T c5_exitg1;
  boolean_T c5_f_validLaunchParams;
  boolean_T c5_fb_validLaunchParams;
  boolean_T c5_g_validLaunchParams;
  boolean_T c5_gb_validLaunchParams;
  boolean_T c5_guard1 = false;
  boolean_T c5_h_validLaunchParams;
  boolean_T c5_hb_validLaunchParams;
  boolean_T c5_i_validLaunchParams;
  boolean_T c5_ib_validLaunchParams;
  boolean_T c5_idx_data_dirtyOnGpu;
  boolean_T c5_ii_data_dirtyOnCpu;
  boolean_T c5_index_data_dirtyOnGpu;
  boolean_T c5_iv1_data_dirtyOnCpu;
  boolean_T c5_j_validLaunchParams;
  boolean_T c5_jb_validLaunchParams;
  boolean_T c5_k_validLaunchParams;
  boolean_T c5_kb_validLaunchParams;
  boolean_T c5_l_validLaunchParams;
  boolean_T c5_lb_validLaunchParams;
  boolean_T c5_m_validLaunchParams;
  boolean_T c5_mb_validLaunchParams;
  boolean_T c5_n_dirtyOnCpu;
  boolean_T c5_n_dirtyOnGpu;
  boolean_T c5_n_validLaunchParams;
  boolean_T c5_nb_validLaunchParams;
  boolean_T c5_nrows_dirtyOnGpu;
  boolean_T c5_o_validLaunchParams;
  boolean_T c5_ob_validLaunchParams;
  boolean_T c5_p_validLaunchParams;
  boolean_T c5_pb_validLaunchParams;
  boolean_T c5_q_validLaunchParams;
  boolean_T c5_qb_validLaunchParams;
  boolean_T c5_r_validLaunchParams;
  boolean_T c5_rb_validLaunchParams;
  boolean_T c5_s_validLaunchParams;
  boolean_T c5_sb_validLaunchParams;
  boolean_T c5_scorePred_data_dirtyOnCpu;
  boolean_T c5_scorePred_data_dirtyOnGpu;
  boolean_T c5_scores_data_dirtyOnCpu;
  boolean_T c5_scores_data_dirtyOnGpu;
  boolean_T c5_selectedIndex_data_dirtyOnCpu;
  boolean_T c5_selectedIndex_data_dirtyOnGpu;
  boolean_T c5_t_validLaunchParams;
  boolean_T c5_tb_validLaunchParams;
  boolean_T c5_thresholdedPrediction_size_dirtyOnCpu;
  boolean_T c5_u_validLaunchParams;
  boolean_T c5_ub_validLaunchParams;
  boolean_T c5_v_validLaunchParams;
  boolean_T c5_validLaunchParams;
  boolean_T c5_vb_validLaunchParams;
  boolean_T c5_w_validLaunchParams;
  boolean_T c5_wb_validLaunchParams;
  boolean_T c5_x1_size_dirtyOnCpu;
  boolean_T c5_x2_data_dirtyOnGpu;
  boolean_T c5_x_validLaunchParams;
  boolean_T c5_y1_data_dirtyOnGpu;
  boolean_T c5_y2_data_dirtyOnGpu;
  boolean_T c5_y_validLaunchParams;
  c5_scores_data_dirtyOnCpu = false;
  c5_iv1_data_dirtyOnCpu = false;
  c5_selectedIndex_data_dirtyOnCpu = false;
  c5_b_data_dirtyOnCpu = false;
  c5_scorePred_data_dirtyOnCpu = false;
  c5_classPred_data_dirtyOnCpu = false;
  c5_bboxPred_data_dirtyOnCpu = false;
  c5_ii_data_dirtyOnCpu = false;
  c5_scores_data_dirtyOnGpu = false;
  c5_bboxes_data_dirtyOnGpu = false;
  c5_index_data_dirtyOnGpu = false;
  c5_area_data_dirtyOnGpu = false;
  c5_selectedIndex_data_dirtyOnGpu = false;
  c5_b_data_dirtyOnGpu = false;
  c5_idx_data_dirtyOnGpu = false;
  c5_scorePred_data_dirtyOnGpu = false;
  c5_classPred_data_dirtyOnGpu = false;
  c5_bboxPred_data_dirtyOnGpu = false;
  c5_b_bboxPred_data_dirtyOnGpu = false;
  c5_y2_data_dirtyOnGpu = false;
  c5_x2_data_dirtyOnGpu = false;
  c5_y1_data_dirtyOnGpu = false;
  c5_bboxesX1Y1X2Y2_data_dirtyOnGpu = false;
  if (!chartInstance->c5_yolodetector_not_empty) {
    c5_iobj_0 = &chartInstance->c5_gobj_0;
    c5_DeepLearningNetwork_setup(chartInstance, &chartInstance->c5_gobj_0);
    chartInstance->c5_yolodetector.Network = c5_iobj_0;
    chartInstance->c5_yolodetector_not_empty = true;
  }

  c5_this_Network = chartInstance->c5_yolodetector.Network;
  c5_eML_blk_kernel_kernel1<<<dim3(2U, 1U, 1U), dim3(512U, 1U, 1U)>>>
    (*chartInstance->c5_gpu_aux1);
  c5_eML_blk_kernel_kernel2<<<dim3(4U, 1U, 1U), dim3(512U, 1U, 1U)>>>
    (*chartInstance->c5_gpu_aux2);
  c5_eML_blk_kernel_kernel3<<<dim3(4U, 1U, 1U), dim3(512U, 1U, 1U)>>>
    (*chartInstance->c5_gpu_aux1, *chartInstance->c5_gpu_rowWeights,
     *chartInstance->c5_gpu_ipRowIndices);
  c5_eML_blk_kernel_kernel4<<<dim3(7U, 1U, 1U), dim3(512U, 1U, 1U)>>>
    (*chartInstance->c5_gpu_aux2, *chartInstance->c5_gpu_colWeights,
     *chartInstance->c5_gpu_ipColIndices);
  c5_eML_blk_kernel_kernel5<<<dim3(1U, 1U, 1U), dim3(224U, 1U, 1U)>>>
    (*chartInstance->c5_gpu_rowWeights, *chartInstance->c5_gpu_rowWeightsTotal);
  for (c5_k = 0; c5_k < 8; c5_k++) {
    c5_eML_blk_kernel_kernel6<<<dim3(1U, 1U, 1U), dim3(224U, 1U, 1U)>>>
      (*chartInstance->c5_gpu_rowWeights, (c5_k + 1) * 224,
       *chartInstance->c5_gpu_rowWeightsTotal);
  }

  c5_eML_blk_kernel_kernel7<<<dim3(1U, 1U, 1U), dim3(224U, 1U, 1U)>>>
    (*chartInstance->c5_gpu_colWeights, *chartInstance->c5_gpu_colWeightsTotal);
  for (c5_b_k = 0; c5_b_k < 15; c5_b_k++) {
    c5_eML_blk_kernel_kernel8<<<dim3(1U, 1U, 1U), dim3(224U, 1U, 1U)>>>
      (*chartInstance->c5_gpu_colWeights, (c5_b_k + 1) * 224,
       *chartInstance->c5_gpu_colWeightsTotal);
  }

  hipMemcpy(chartInstance->c5_gpu_In, &c5_b_In[0], 4919040UL,
             hipMemcpyHostToDevice);
  c5_eML_blk_kernel_kernel9<<<dim3(630U, 1U, 1U), dim3(512U, 1U, 1U)>>>
    (*chartInstance->c5_gpu_colWeightsTotal, *chartInstance->c5_gpu_colWeights, *
     chartInstance->c5_gpu_In, *chartInstance->c5_gpu_ipColIndices,
     *chartInstance->c5_gpu_partialResize);
  c5_eML_blk_kernel_kernel10<<<dim3(294U, 1U, 1U), dim3(512U, 1U, 1U)>>>
    (*chartInstance->c5_gpu_rowWeightsTotal, *chartInstance->c5_gpu_rowWeights, *
     chartInstance->c5_gpu_partialResize, *chartInstance->c5_gpu_ipRowIndices,
     *chartInstance->c5_gpu_out);
  c5_eML_blk_kernel_kernel11<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
    (*chartInstance->c5_gpu_out, *chartInstance->c5_gpu_outVal);
  c5_coder_reduce0<<<dim3(294U, 1U, 1U), dim3(512U, 1U, 1U)>>>
    (*chartInstance->c5_gpu_out, *chartInstance->c5_gpu_outVal);
  hipMemcpy(&c5_outVal[0], chartInstance->c5_gpu_outVal, 8UL,
             hipMemcpyDeviceToHost);
  c5_y = c5_outVal[1] - c5_outVal[0];
  c5_eML_blk_kernel_kernel12<<<dim3(294U, 1U, 1U), dim3(512U, 1U, 1U)>>>(c5_y,
    c5_outVal[0], *chartInstance->c5_gpu_out, *chartInstance->c5_b_gpu_out);
  hipMemcpy(&chartInstance->c5_out[0], chartInstance->c5_b_gpu_out, 602112UL,
             hipMemcpyDeviceToHost);
  c5_DeepLearningNetwork_activations(chartInstance, c5_this_Network,
    chartInstance->c5_out, c5_tmpFeatureMap);
  hipMemcpy(chartInstance->c5_gpu_dv, &c5_dv[0], 8UL, hipMemcpyHostToDevice);
  c5_eML_blk_kernel_kernel13<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
    (*chartInstance->c5_gpu_dv, *chartInstance->c5_gpu_anchors);
  hipMemcpy(chartInstance->c5_gpu_dv1, &c5_dv1[0], 32UL, hipMemcpyHostToDevice);
  c5_eML_blk_kernel_kernel14<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
    (*chartInstance->c5_gpu_dv1, *chartInstance->c5_b_gpu_anchors,
     *chartInstance->c5_gpu_anchors);
  hipMemcpy(chartInstance->c5_gpu_tmpFeatureMap, &c5_tmpFeatureMap[0], 18816UL,
             hipMemcpyHostToDevice);
  c5_eML_blk_kernel_kernel15<<<dim3(2U, 1U, 1U), dim3(512U, 1U, 1U)>>>
    (*chartInstance->c5_gpu_anchors, *chartInstance->c5_gpu_tmpFeatureMap,
     *chartInstance->c5_gpu_boxOut);
  c5_eML_blk_kernel_kernel16<<<dim3(2U, 1U, 1U), dim3(512U, 1U, 1U)>>>
    (*chartInstance->c5_gpu_boxOut, *chartInstance->c5_gpu_bv);
  c5_bv_dirtyOnGpu = true;
  c5_trueCount = 0;
  for (c5_i = 0; c5_i < 784; c5_i++) {
    if (c5_bv_dirtyOnGpu) {
      hipMemcpy(&c5_bv[0], chartInstance->c5_gpu_bv, 784UL,
                 hipMemcpyDeviceToHost);
      c5_bv_dirtyOnGpu = false;
    }

    if (c5_bv[c5_i]) {
      c5_trueCount++;
    }
  }

  c5_ii_size[0] = c5_trueCount;
  c5_partialTrueCount = 0;
  for (c5_b_i = 0; c5_b_i < 784; c5_b_i++) {
    if (c5_bv_dirtyOnGpu) {
      hipMemcpy(&c5_bv[0], chartInstance->c5_gpu_bv, 784UL,
                 hipMemcpyDeviceToHost);
      c5_bv_dirtyOnGpu = false;
    }

    if (c5_bv[c5_b_i]) {
      c5_ii_data[c5_partialTrueCount] = (int16_T)(c5_b_i + 1);
      c5_ii_data_dirtyOnCpu = true;
      c5_partialTrueCount++;
    }
  }

  c5_thresholdedPrediction_size[0] = c5_trueCount;
  c5_thresholdedPrediction_size[1] = 6;
  c5_thresholdedPrediction_size_dirtyOnCpu = true;
  c5_validLaunchParams = mwGetLaunchParameters((real_T)(((int64_T)(c5_ii_size[0]
    - 1) + 1L) * 6L), &c5_grid, &c5_block, 1024U, 65535U);
  if (c5_validLaunchParams) {
    if (c5_ii_data_dirtyOnCpu) {
      hipMemcpy(chartInstance->c5_gpu_ii_data, &c5_ii_data[0], 1568UL,
                 hipMemcpyHostToDevice);
    }

    hipMemcpy(chartInstance->c5_gpu_thresholdedPrediction_size,
               &c5_thresholdedPrediction_size[0], 8UL, hipMemcpyHostToDevice);
    c5_thresholdedPrediction_size_dirtyOnCpu = false;
    hipMemcpy(chartInstance->c5_gpu_ii_size, &c5_ii_size[0], 4UL,
               hipMemcpyHostToDevice);
    c5_eML_blk_kernel_kernel17<<<c5_grid, c5_block>>>
      (*chartInstance->c5_gpu_boxOut, *chartInstance->c5_gpu_ii_data,
       *chartInstance->c5_gpu_thresholdedPrediction_size,
       *chartInstance->c5_gpu_ii_size,
       *chartInstance->c5_gpu_thresholdedPrediction_data);
  }

  if (c5_ii_size[0] != 0) {
    c5_i7 = c5_ii_size[0] - 1;
    c5_bboxesX1Y1X2Y2_size[0] = c5_ii_size[0];
    c5_bboxesX1Y1X2Y2_size[1] = 4;
    c5_bboxesX1Y1X2Y2_size_dirtyOnCpu = true;
    c5_b_validLaunchParams = mwGetLaunchParameters((real_T)(((int64_T)c5_i7 + 1L)
      * 4L), &c5_b_grid, &c5_b_block, 1024U, 65535U);
    if (c5_b_validLaunchParams) {
      if (c5_thresholdedPrediction_size_dirtyOnCpu) {
        hipMemcpy(chartInstance->c5_gpu_thresholdedPrediction_size,
                   &c5_thresholdedPrediction_size[0], 8UL,
                   hipMemcpyHostToDevice);
        c5_thresholdedPrediction_size_dirtyOnCpu = false;
      }

      hipMemcpy(chartInstance->c5_gpu_bboxesX1Y1X2Y2_size,
                 &c5_bboxesX1Y1X2Y2_size[0], 8UL, hipMemcpyHostToDevice);
      c5_bboxesX1Y1X2Y2_size_dirtyOnCpu = false;
      c5_eML_blk_kernel_kernel18<<<c5_b_grid, c5_b_block>>>
        (*chartInstance->c5_gpu_thresholdedPrediction_data,
         *chartInstance->c5_gpu_thresholdedPrediction_size,
         *chartInstance->c5_gpu_bboxesX1Y1X2Y2_size, c5_i7,
         *chartInstance->c5_gpu_bboxesX1Y1X2Y2_data);
      c5_bboxesX1Y1X2Y2_data_dirtyOnGpu = true;
    }

    c5_i10 = c5_bboxesX1Y1X2Y2_size[0] - 1;
    c5_x1_size[0] = c5_bboxesX1Y1X2Y2_size[0];
    c5_d_validLaunchParams = mwGetLaunchParameters((real_T)((int64_T)c5_i10 + 1L),
      &c5_d_grid, &c5_d_block, 1024U, 65535U);
    if (c5_d_validLaunchParams) {
      c5_eML_blk_kernel_kernel19<<<c5_d_grid, c5_d_block>>>
        (*chartInstance->c5_gpu_bboxesX1Y1X2Y2_data, c5_i10,
         *chartInstance->c5_gpu_x1_data);
    }

    c5_i14 = c5_bboxesX1Y1X2Y2_size[0] - 1;
    c5_y1_size[0] = c5_bboxesX1Y1X2Y2_size[0];
    c5_f_validLaunchParams = mwGetLaunchParameters((real_T)((int64_T)c5_i14 + 1L),
      &c5_f_grid, &c5_f_block, 1024U, 65535U);
    if (c5_f_validLaunchParams) {
      if (c5_bboxesX1Y1X2Y2_size_dirtyOnCpu) {
        hipMemcpy(chartInstance->c5_gpu_bboxesX1Y1X2Y2_size,
                   &c5_bboxesX1Y1X2Y2_size[0], 8UL, hipMemcpyHostToDevice);
        c5_bboxesX1Y1X2Y2_size_dirtyOnCpu = false;
      }

      c5_eML_blk_kernel_kernel20<<<c5_f_grid, c5_f_block>>>
        (*chartInstance->c5_gpu_bboxesX1Y1X2Y2_data,
         *chartInstance->c5_gpu_bboxesX1Y1X2Y2_size, c5_i14,
         *chartInstance->c5_gpu_y1_data);
      c5_y1_data_dirtyOnGpu = true;
    }

    c5_i17 = c5_bboxesX1Y1X2Y2_size[0] - 1;
    c5_x2_size[0] = c5_bboxesX1Y1X2Y2_size[0];
    c5_g_validLaunchParams = mwGetLaunchParameters((real_T)((int64_T)c5_i17 + 1L),
      &c5_g_grid, &c5_g_block, 1024U, 65535U);
    if (c5_g_validLaunchParams) {
      if (c5_bboxesX1Y1X2Y2_size_dirtyOnCpu) {
        hipMemcpy(chartInstance->c5_gpu_bboxesX1Y1X2Y2_size,
                   &c5_bboxesX1Y1X2Y2_size[0], 8UL, hipMemcpyHostToDevice);
        c5_bboxesX1Y1X2Y2_size_dirtyOnCpu = false;
      }

      c5_eML_blk_kernel_kernel21<<<c5_g_grid, c5_g_block>>>
        (*chartInstance->c5_gpu_bboxesX1Y1X2Y2_data,
         *chartInstance->c5_gpu_bboxesX1Y1X2Y2_size, c5_i17,
         *chartInstance->c5_gpu_x2_data);
      c5_x2_data_dirtyOnGpu = true;
    }

    c5_i19 = c5_bboxesX1Y1X2Y2_size[0] - 1;
    c5_y2_size[0] = c5_bboxesX1Y1X2Y2_size[0];
    c5_h_validLaunchParams = mwGetLaunchParameters((real_T)((int64_T)c5_i19 + 1L),
      &c5_h_grid, &c5_h_block, 1024U, 65535U);
    if (c5_h_validLaunchParams) {
      if (c5_bboxesX1Y1X2Y2_size_dirtyOnCpu) {
        hipMemcpy(chartInstance->c5_gpu_bboxesX1Y1X2Y2_size,
                   &c5_bboxesX1Y1X2Y2_size[0], 8UL, hipMemcpyHostToDevice);
      }

      c5_eML_blk_kernel_kernel22<<<c5_h_grid, c5_h_block>>>
        (*chartInstance->c5_gpu_bboxesX1Y1X2Y2_data,
         *chartInstance->c5_gpu_bboxesX1Y1X2Y2_size, c5_i19,
         *chartInstance->c5_gpu_y2_data);
      c5_y2_data_dirtyOnGpu = true;
    }

    c5_i_validLaunchParams = mwGetLaunchParameters((real_T)((int64_T)
      (c5_bboxesX1Y1X2Y2_size[0] - 1) + 1L), &c5_i_grid, &c5_i_block, 1024U,
      65535U);
    if (c5_i_validLaunchParams) {
      c5_eML_blk_kernel_kernel23<<<c5_i_grid, c5_i_block>>>
        (c5_bboxesX1Y1X2Y2_size[0], *chartInstance->c5_gpu_x1_data);
    }

    c5_j_validLaunchParams = mwGetLaunchParameters((real_T)((int64_T)
      (c5_bboxesX1Y1X2Y2_size[0] - 1) + 1L), &c5_j_grid, &c5_j_block, 1024U,
      65535U);
    if (c5_j_validLaunchParams) {
      c5_eML_blk_kernel_kernel24<<<c5_j_grid, c5_j_block>>>
        (c5_bboxesX1Y1X2Y2_size[0], *chartInstance->c5_gpu_y1_data);
      c5_y1_data_dirtyOnGpu = true;
    }

    c5_k_validLaunchParams = mwGetLaunchParameters((real_T)((int64_T)
      (c5_bboxesX1Y1X2Y2_size[0] - 1) + 1L), &c5_k_grid, &c5_k_block, 1024U,
      65535U);
    if (c5_k_validLaunchParams) {
      c5_eML_blk_kernel_kernel25<<<c5_k_grid, c5_k_block>>>
        (c5_bboxesX1Y1X2Y2_size[0], *chartInstance->c5_gpu_x2_data);
      c5_x2_data_dirtyOnGpu = true;
    }

    c5_l_validLaunchParams = mwGetLaunchParameters((real_T)((int64_T)
      (c5_bboxesX1Y1X2Y2_size[0] - 1) + 1L), &c5_l_grid, &c5_l_block, 1024U,
      65535U);
    if (c5_l_validLaunchParams) {
      c5_eML_blk_kernel_kernel26<<<c5_l_grid, c5_l_block>>>
        (c5_bboxesX1Y1X2Y2_size[0], *chartInstance->c5_gpu_y2_data);
      c5_y2_data_dirtyOnGpu = true;
    }

    c5_bboxesX1Y1X2Y2_size[0] = c5_x1_size[0];
    c5_bboxesX1Y1X2Y2_size[1] = 4;
    c5_bboxesX1Y1X2Y2_size_dirtyOnCpu = true;
    c5_m_validLaunchParams = mwGetLaunchParameters((real_T)((int64_T)
      (c5_x1_size[0] - 1) + 1L), &c5_m_grid, &c5_m_block, 1024U, 65535U);
    if (c5_m_validLaunchParams) {
      c5_eML_blk_kernel_kernel27<<<c5_m_grid, c5_m_block>>>
        (*chartInstance->c5_gpu_x1_data, c5_x1_size[0] - 1,
         *chartInstance->c5_gpu_bboxesX1Y1X2Y2_data);
      c5_bboxesX1Y1X2Y2_data_dirtyOnGpu = true;
    }

    c5_n_validLaunchParams = mwGetLaunchParameters((real_T)((int64_T)
      (c5_y1_size[0] - 1) + 1L), &c5_n_grid, &c5_n_block, 1024U, 65535U);
    if (c5_n_validLaunchParams) {
      hipMemcpy(chartInstance->c5_gpu_bboxesX1Y1X2Y2_size,
                 &c5_bboxesX1Y1X2Y2_size[0], 8UL, hipMemcpyHostToDevice);
      c5_bboxesX1Y1X2Y2_size_dirtyOnCpu = false;
      c5_eML_blk_kernel_kernel28<<<c5_n_grid, c5_n_block>>>
        (*chartInstance->c5_gpu_y1_data,
         *chartInstance->c5_gpu_bboxesX1Y1X2Y2_size, c5_y1_size[0] - 1,
         *chartInstance->c5_gpu_bboxesX1Y1X2Y2_data);
      c5_bboxesX1Y1X2Y2_data_dirtyOnGpu = true;
    }

    c5_o_validLaunchParams = mwGetLaunchParameters((real_T)((int64_T)
      (c5_x2_size[0] - 1) + 1L), &c5_o_grid, &c5_o_block, 1024U, 65535U);
    if (c5_o_validLaunchParams) {
      if (c5_bboxesX1Y1X2Y2_size_dirtyOnCpu) {
        hipMemcpy(chartInstance->c5_gpu_bboxesX1Y1X2Y2_size,
                   &c5_bboxesX1Y1X2Y2_size[0], 8UL, hipMemcpyHostToDevice);
        c5_bboxesX1Y1X2Y2_size_dirtyOnCpu = false;
      }

      c5_eML_blk_kernel_kernel29<<<c5_o_grid, c5_o_block>>>
        (*chartInstance->c5_gpu_x2_data,
         *chartInstance->c5_gpu_bboxesX1Y1X2Y2_size, c5_x2_size[0] - 1,
         *chartInstance->c5_gpu_bboxesX1Y1X2Y2_data);
      c5_bboxesX1Y1X2Y2_data_dirtyOnGpu = true;
    }

    c5_p_validLaunchParams = mwGetLaunchParameters((real_T)((int64_T)
      (c5_y2_size[0] - 1) + 1L), &c5_p_grid, &c5_p_block, 1024U, 65535U);
    if (c5_p_validLaunchParams) {
      if (c5_bboxesX1Y1X2Y2_size_dirtyOnCpu) {
        hipMemcpy(chartInstance->c5_gpu_bboxesX1Y1X2Y2_size,
                   &c5_bboxesX1Y1X2Y2_size[0], 8UL, hipMemcpyHostToDevice);
        c5_bboxesX1Y1X2Y2_size_dirtyOnCpu = false;
      }

      c5_eML_blk_kernel_kernel30<<<c5_p_grid, c5_p_block>>>
        (*chartInstance->c5_gpu_y2_data,
         *chartInstance->c5_gpu_bboxesX1Y1X2Y2_size, c5_y2_size[0] - 1,
         *chartInstance->c5_gpu_bboxesX1Y1X2Y2_data);
      c5_bboxesX1Y1X2Y2_data_dirtyOnGpu = true;
    }

    c5_i25 = c5_bboxesX1Y1X2Y2_size[0];
    c5_i26 = c5_bboxesX1Y1X2Y2_size[0];
    c5_i27 = c5_bboxesX1Y1X2Y2_size[0];
    c5_bboxPred_size[0] = c5_bboxesX1Y1X2Y2_size[0];
    c5_bboxPred_size[1] = 4;
    c5_bboxPred_size_dirtyOnCpu = true;
    c5_q_validLaunchParams = mwGetLaunchParameters((real_T)((int64_T)
      (c5_bboxesX1Y1X2Y2_size[0] - 1) + 1L), &c5_q_grid, &c5_q_block, 1024U,
      65535U);
    if (c5_q_validLaunchParams) {
      c5_eML_blk_kernel_kernel31<<<c5_q_grid, c5_q_block>>>
        (*chartInstance->c5_gpu_bboxesX1Y1X2Y2_data, c5_bboxesX1Y1X2Y2_size[0] -
         1, *chartInstance->c5_e_gpu_bboxPred_data);
      c5_b_bboxPred_data_dirtyOnGpu = true;
    }

    c5_r_validLaunchParams = mwGetLaunchParameters((real_T)((int64_T)(c5_i25 - 1)
      + 1L), &c5_r_grid, &c5_r_block, 1024U, 65535U);
    if (c5_r_validLaunchParams) {
      if (c5_bboxesX1Y1X2Y2_size_dirtyOnCpu) {
        hipMemcpy(chartInstance->c5_gpu_bboxesX1Y1X2Y2_size,
                   &c5_bboxesX1Y1X2Y2_size[0], 8UL, hipMemcpyHostToDevice);
        c5_bboxesX1Y1X2Y2_size_dirtyOnCpu = false;
      }

      hipMemcpy(chartInstance->c5_c_gpu_bboxPred_size, &c5_bboxPred_size[0],
                 8UL, hipMemcpyHostToDevice);
      c5_bboxPred_size_dirtyOnCpu = false;
      c5_eML_blk_kernel_kernel32<<<c5_r_grid, c5_r_block>>>
        (*chartInstance->c5_gpu_bboxesX1Y1X2Y2_data,
         *chartInstance->c5_gpu_bboxesX1Y1X2Y2_size,
         *chartInstance->c5_c_gpu_bboxPred_size, c5_i25 - 1,
         *chartInstance->c5_e_gpu_bboxPred_data);
      c5_b_bboxPred_data_dirtyOnGpu = true;
    }

    c5_s_validLaunchParams = mwGetLaunchParameters((real_T)((int64_T)(c5_i26 - 1)
      + 1L), &c5_s_grid, &c5_s_block, 1024U, 65535U);
    if (c5_s_validLaunchParams) {
      if (c5_bboxesX1Y1X2Y2_size_dirtyOnCpu) {
        hipMemcpy(chartInstance->c5_gpu_bboxesX1Y1X2Y2_size,
                   &c5_bboxesX1Y1X2Y2_size[0], 8UL, hipMemcpyHostToDevice);
        c5_bboxesX1Y1X2Y2_size_dirtyOnCpu = false;
      }

      if (c5_bboxPred_size_dirtyOnCpu) {
        hipMemcpy(chartInstance->c5_c_gpu_bboxPred_size, &c5_bboxPred_size[0],
                   8UL, hipMemcpyHostToDevice);
        c5_bboxPred_size_dirtyOnCpu = false;
      }

      c5_eML_blk_kernel_kernel33<<<c5_s_grid, c5_s_block>>>
        (*chartInstance->c5_gpu_bboxesX1Y1X2Y2_data,
         *chartInstance->c5_gpu_bboxesX1Y1X2Y2_size,
         *chartInstance->c5_c_gpu_bboxPred_size, c5_i26 - 1,
         *chartInstance->c5_e_gpu_bboxPred_data);
      c5_b_bboxPred_data_dirtyOnGpu = true;
    }

    c5_t_validLaunchParams = mwGetLaunchParameters((real_T)((int64_T)(c5_i27 - 1)
      + 1L), &c5_t_grid, &c5_t_block, 1024U, 65535U);
    if (c5_t_validLaunchParams) {
      if (c5_bboxesX1Y1X2Y2_size_dirtyOnCpu) {
        hipMemcpy(chartInstance->c5_gpu_bboxesX1Y1X2Y2_size,
                   &c5_bboxesX1Y1X2Y2_size[0], 8UL, hipMemcpyHostToDevice);
      }

      if (c5_bboxPred_size_dirtyOnCpu) {
        hipMemcpy(chartInstance->c5_c_gpu_bboxPred_size, &c5_bboxPred_size[0],
                   8UL, hipMemcpyHostToDevice);
        c5_bboxPred_size_dirtyOnCpu = false;
      }

      c5_eML_blk_kernel_kernel34<<<c5_t_grid, c5_t_block>>>
        (*chartInstance->c5_gpu_bboxesX1Y1X2Y2_data,
         *chartInstance->c5_gpu_bboxesX1Y1X2Y2_size,
         *chartInstance->c5_c_gpu_bboxPred_size, c5_i27 - 1,
         *chartInstance->c5_e_gpu_bboxPred_data);
      c5_b_bboxPred_data_dirtyOnGpu = true;
    }

    c5_nx = c5_bboxPred_size[0] << 2;
    c5_u_validLaunchParams = mwGetLaunchParameters((real_T)((int64_T)(c5_nx - 1)
      + 1L), &c5_u_grid, &c5_u_block, 1024U, 65535U);
    if (c5_u_validLaunchParams) {
      c5_eML_blk_kernel_kernel35<<<c5_u_grid, c5_u_block>>>(c5_nx,
        *chartInstance->c5_e_gpu_bboxPred_data);
      c5_b_bboxPred_data_dirtyOnGpu = true;
    }

    c5_i32 = c5_bboxPred_size[0] - 1;
    c5_b_bboxPred_size[0] = c5_bboxPred_size[0];
    c5_v_validLaunchParams = mwGetLaunchParameters((real_T)((int64_T)c5_i32 + 1L),
      &c5_v_grid, &c5_v_block, 1024U, 65535U);
    if (c5_v_validLaunchParams) {
      if (c5_bboxPred_size_dirtyOnCpu) {
        hipMemcpy(chartInstance->c5_c_gpu_bboxPred_size, &c5_bboxPred_size[0],
                   8UL, hipMemcpyHostToDevice);
        c5_bboxPred_size_dirtyOnCpu = false;
      }

      c5_eML_blk_kernel_kernel36<<<c5_v_grid, c5_v_block>>>
        (*chartInstance->c5_e_gpu_bboxPred_data,
         *chartInstance->c5_c_gpu_bboxPred_size, c5_i32,
         *chartInstance->c5_d_gpu_bboxPred_data);
    }

    c5_w_validLaunchParams = mwGetLaunchParameters((real_T)((int64_T)
      (c5_b_bboxPred_size[0] - 1) + 1L), &c5_w_grid, &c5_w_block, 1024U, 65535U);
    if (c5_w_validLaunchParams) {
      if (c5_bboxPred_size_dirtyOnCpu) {
        hipMemcpy(chartInstance->c5_c_gpu_bboxPred_size, &c5_bboxPred_size[0],
                   8UL, hipMemcpyHostToDevice);
        c5_bboxPred_size_dirtyOnCpu = false;
      }

      hipMemcpy(chartInstance->c5_b_gpu_bboxPred_size, &c5_b_bboxPred_size[0],
                 4UL, hipMemcpyHostToDevice);
      c5_eML_blk_kernel_kernel37<<<c5_w_grid, c5_w_block>>>
        (*chartInstance->c5_d_gpu_bboxPred_data,
         *chartInstance->c5_c_gpu_bboxPred_size,
         *chartInstance->c5_b_gpu_bboxPred_size,
         *chartInstance->c5_e_gpu_bboxPred_data);
      c5_b_bboxPred_data_dirtyOnGpu = true;
    }

    c5_i35 = c5_bboxPred_size[0] - 1;
    c5_c_bboxPred_size[0] = c5_bboxPred_size[0];
    c5_x_validLaunchParams = mwGetLaunchParameters((real_T)((int64_T)c5_i35 + 1L),
      &c5_x_grid, &c5_x_block, 1024U, 65535U);
    if (c5_x_validLaunchParams) {
      if (c5_bboxPred_size_dirtyOnCpu) {
        hipMemcpy(chartInstance->c5_c_gpu_bboxPred_size, &c5_bboxPred_size[0],
                   8UL, hipMemcpyHostToDevice);
        c5_bboxPred_size_dirtyOnCpu = false;
      }

      c5_eML_blk_kernel_kernel38<<<c5_x_grid, c5_x_block>>>
        (*chartInstance->c5_e_gpu_bboxPred_data,
         *chartInstance->c5_c_gpu_bboxPred_size, c5_i35,
         *chartInstance->c5_gpu_bboxPred_data);
    }

    c5_y_validLaunchParams = mwGetLaunchParameters((real_T)((int64_T)
      (c5_c_bboxPred_size[0] - 1) + 1L), &c5_y_grid, &c5_y_block, 1024U, 65535U);
    if (c5_y_validLaunchParams) {
      if (c5_bboxPred_size_dirtyOnCpu) {
        hipMemcpy(chartInstance->c5_c_gpu_bboxPred_size, &c5_bboxPred_size[0],
                   8UL, hipMemcpyHostToDevice);
        c5_bboxPred_size_dirtyOnCpu = false;
      }

      hipMemcpy(chartInstance->c5_e_gpu_bboxPred_size, &c5_c_bboxPred_size[0],
                 4UL, hipMemcpyHostToDevice);
      c5_eML_blk_kernel_kernel39<<<c5_y_grid, c5_y_block>>>
        (*chartInstance->c5_gpu_bboxPred_data,
         *chartInstance->c5_c_gpu_bboxPred_size,
         *chartInstance->c5_e_gpu_bboxPred_size,
         *chartInstance->c5_e_gpu_bboxPred_data);
      c5_b_bboxPred_data_dirtyOnGpu = true;
    }

    c5_count = 0.0;
    c5_d_bboxPred_size[0] = c5_bboxPred_size[0];
    c5_d_bboxPred_size[1] = 4;
    c5_b_bboxPred_size_dirtyOnCpu = true;
    c5_scorePred_size[0] = c5_bboxPred_size[0];
    c5_classPred_size[0] = c5_bboxPred_size[0];
    c5_c_i = c5_bboxPred_size[0];
    for (c5_d_i = 0; c5_d_i < c5_c_i; c5_d_i++) {
      if (c5_b_bboxPred_data_dirtyOnGpu) {
        hipMemcpy(&c5_bboxPred_data[0], chartInstance->c5_e_gpu_bboxPred_data,
                   25088UL, hipMemcpyDeviceToHost);
        c5_b_bboxPred_data_dirtyOnGpu = false;
      }

      if ((c5_bboxPred_data[c5_d_i + c5_bboxPred_size[0] * 3] >= 1.0) &&
          (c5_bboxPred_data[c5_d_i + (c5_bboxPred_size[0] << 1)] >= 1.0) &&
          (c5_bboxPred_data[c5_d_i + c5_bboxPred_size[0] * 3] <= 480.0) &&
          (c5_bboxPred_data[c5_d_i + (c5_bboxPred_size[0] << 1)] <= 854.0)) {
        c5_count++;
        if (c5_bboxPred_size_dirtyOnCpu) {
          hipMemcpy(chartInstance->c5_c_gpu_bboxPred_size, &c5_bboxPred_size[0],
                     8UL, hipMemcpyHostToDevice);
          c5_bboxPred_size_dirtyOnCpu = false;
        }

        if (c5_b_bboxPred_size_dirtyOnCpu) {
          hipMemcpy(chartInstance->c5_gpu_bboxPred_size, &c5_d_bboxPred_size[0],
                     8UL, hipMemcpyHostToDevice);
          c5_b_bboxPred_size_dirtyOnCpu = false;
        }

        c5_eML_blk_kernel_kernel40<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
          (*chartInstance->c5_e_gpu_bboxPred_data,
           *chartInstance->c5_c_gpu_bboxPred_size, c5_d_i,
           *chartInstance->c5_gpu_bboxPred_size, (int32_T)c5_count - 1,
           *chartInstance->c5_c_gpu_bboxPred_data);
        c5_bboxPred_data_dirtyOnGpu = true;
        if (c5_thresholdedPrediction_size_dirtyOnCpu) {
          hipMemcpy(chartInstance->c5_gpu_thresholdedPrediction_size,
                     &c5_thresholdedPrediction_size[0], 8UL,
                     hipMemcpyHostToDevice);
          c5_thresholdedPrediction_size_dirtyOnCpu = false;
        }

        c5_eML_blk_kernel_kernel41<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
          (*chartInstance->c5_gpu_thresholdedPrediction_data,
           *chartInstance->c5_gpu_thresholdedPrediction_size, c5_d_i, c5_count, *
           chartInstance->c5_gpu_classPred_data,
           *chartInstance->c5_gpu_scorePred_data);
        c5_scorePred_data_dirtyOnGpu = true;
        c5_classPred_data_dirtyOnGpu = true;
      }
    }

    c5_i39 = c5_d_bboxPred_size[0];
    c5_ab_validLaunchParams = mwGetLaunchParameters((real_T)((int64_T)(c5_i39 -
      (int32_T)(c5_count + 1.0)) + 1L), &c5_ab_grid, &c5_ab_block, 1024U, 65535U);
    if (c5_ab_validLaunchParams) {
      c5_eML_blk_kernel_kernel42<<<c5_ab_grid, c5_ab_block>>>((int32_T)(c5_count
        + 1.0), c5_i39, *chartInstance->c5_gpu_idx_data);
      c5_idx_data_dirtyOnGpu = true;
    }

    c5_nrowx = c5_d_bboxPred_size[0];
    if ((c5_d_bboxPred_size[0] - (int32_T)(c5_count + 1.0)) + 1 == 1) {
      c5_nrows = c5_d_bboxPred_size[0] - 1;
      c5_nrows_dirtyOnGpu = false;
      if (c5_b_bboxPred_size_dirtyOnCpu) {
        hipMemcpy(chartInstance->c5_gpu_bboxPred_size, &c5_d_bboxPred_size[0],
                   8UL, hipMemcpyHostToDevice);
      }

      hipMemcpy(chartInstance->c5_gpu_nrows, &c5_nrows, 4UL,
                 hipMemcpyHostToDevice);
      c5_eML_blk_kernel_kernel46<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
        (*chartInstance->c5_gpu_bboxPred_size, chartInstance->c5_gpu_nrows,
         *chartInstance->c5_gpu_idx_data, *chartInstance->c5_c_gpu_bboxPred_data);
    } else {
      c5_b_size[0] = 1;
      c5_b_size[1] = c5_d_bboxPred_size[0];
      c5_bb_validLaunchParams = mwGetLaunchParameters((real_T)((int64_T)
        (c5_d_bboxPred_size[0] - 1) + 1L), &c5_bb_grid, &c5_bb_block, 1024U,
        65535U);
      if (c5_bb_validLaunchParams) {
        if (c5_b_bboxPred_size_dirtyOnCpu) {
          hipMemcpy(chartInstance->c5_gpu_bboxPred_size, &c5_d_bboxPred_size[0],
                     8UL, hipMemcpyHostToDevice);
          c5_b_bboxPred_size_dirtyOnCpu = false;
        }

        c5_eML_blk_kernel_kernel43<<<c5_bb_grid, c5_bb_block>>>
          (*chartInstance->c5_gpu_bboxPred_size, *chartInstance->c5_gpu_b_data);
        c5_b_data_dirtyOnGpu = true;
      }

      c5_i1 = (c5_d_bboxPred_size[0] - (int32_T)(c5_count + 1.0)) + 1;
      for (c5_c_k = 0; c5_c_k < c5_i1; c5_c_k++) {
        if (c5_idx_data_dirtyOnGpu) {
          hipMemcpy(&c5_idx_data[0], chartInstance->c5_gpu_idx_data, 3136UL,
                     hipMemcpyDeviceToHost);
          c5_idx_data_dirtyOnGpu = false;
        }

        if (c5_b_data_dirtyOnGpu) {
          hipMemcpy(&c5_b_data[0], chartInstance->c5_gpu_b_data, 784UL,
                     hipMemcpyDeviceToHost);
          c5_b_data_dirtyOnGpu = false;
        }

        c5_b_data[c5_idx_data[c5_c_k] - 1] = true;
        c5_b_data_dirtyOnCpu = true;
      }

      c5_n = 0;
      c5_n_dirtyOnCpu = true;
      c5_i3 = c5_b_size[1];
      c5_cb_validLaunchParams = mwGetLaunchParameters((real_T)((int64_T)
        (c5_b_size[1] - 1) + 1L), &c5_cb_grid, &c5_cb_block, 1024U, 65535U);
      if (c5_cb_validLaunchParams) {
        if (c5_b_data_dirtyOnCpu) {
          hipMemcpy(chartInstance->c5_gpu_b_data, &c5_b_data[0], 784UL,
                     hipMemcpyHostToDevice);
          c5_b_data_dirtyOnCpu = false;
        }

        hipMemcpy(chartInstance->c5_gpu_b_size, &c5_b_size[0], 8UL,
                   hipMemcpyHostToDevice);
        hipMemcpy(chartInstance->c5_gpu_n, &c5_n, 4UL, hipMemcpyHostToDevice);
        c5_eML_blk_kernel_kernel44<<<c5_cb_grid, c5_cb_block>>>
          (*chartInstance->c5_gpu_b_size, *chartInstance->c5_gpu_b_data, c5_i3,
           chartInstance->c5_gpu_n);
        c5_n_dirtyOnCpu = false;
      }

      if (c5_b_bboxPred_size_dirtyOnCpu) {
        hipMemcpy(chartInstance->c5_gpu_bboxPred_size, &c5_d_bboxPred_size[0],
                   8UL, hipMemcpyHostToDevice);
      }

      if (c5_n_dirtyOnCpu) {
        hipMemcpy(chartInstance->c5_gpu_n, &c5_n, 4UL, hipMemcpyHostToDevice);
      }

      c5_eML_blk_kernel_kernel45<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
        (chartInstance->c5_gpu_n, *chartInstance->c5_gpu_bboxPred_size,
         chartInstance->c5_gpu_nrows);
      c5_nrows_dirtyOnGpu = true;
      c5_e_i = 0;
      for (c5_d_k = 0; c5_d_k < c5_nrowx; c5_d_k++) {
        c5_guard1 = false;
        if (c5_d_k + 1 > c5_b_size[1]) {
          c5_guard1 = true;
        } else {
          if (c5_b_data_dirtyOnGpu) {
            hipMemcpy(&c5_b_data[0], chartInstance->c5_gpu_b_data, 784UL,
                       hipMemcpyDeviceToHost);
            c5_b_data_dirtyOnGpu = false;
          }

          if (!c5_b_data[c5_d_k]) {
            c5_guard1 = true;
          }
        }

        if (c5_guard1) {
          for (c5_j = 0; c5_j < 4; c5_j++) {
            if (c5_bboxPred_data_dirtyOnGpu) {
              hipMemcpy(&c5_b_bboxPred_data[0],
                         chartInstance->c5_c_gpu_bboxPred_data, 25088UL,
                         hipMemcpyDeviceToHost);
              c5_bboxPred_data_dirtyOnGpu = false;
            }

            c5_b_bboxPred_data[c5_e_i + c5_d_bboxPred_size[0] * c5_j] =
              c5_b_bboxPred_data[c5_d_k + c5_d_bboxPred_size[0] * c5_j];
            c5_bboxPred_data_dirtyOnCpu = true;
          }

          c5_e_i++;
        }
      }
    }

    if (c5_nrows_dirtyOnGpu) {
      hipMemcpy(&c5_nrows, chartInstance->c5_gpu_nrows, 4UL,
                 hipMemcpyDeviceToHost);
    }

    if (1 > c5_nrows) {
      c5_i4 = -1;
    } else {
      c5_i4 = c5_nrows - 1;
    }

    c5_e_bboxPred_size[0] = c5_i4 + 1;
    c5_e_bboxPred_size[1] = 4;
    c5_c_bboxPred_size_dirtyOnCpu = true;
    c5_db_validLaunchParams = mwGetLaunchParameters((real_T)(((int64_T)c5_i4 +
      1L) * 4L), &c5_db_grid, &c5_db_block, 1024U, 65535U);
    if (c5_db_validLaunchParams) {
      if (c5_bboxPred_data_dirtyOnCpu) {
        hipMemcpy(chartInstance->c5_c_gpu_bboxPred_data, &c5_b_bboxPred_data[0],
                   25088UL, hipMemcpyHostToDevice);
        c5_bboxPred_data_dirtyOnCpu = false;
      }

      hipMemcpy(chartInstance->c5_d_gpu_bboxPred_size, &c5_e_bboxPred_size[0],
                 8UL, hipMemcpyHostToDevice);
      c5_c_bboxPred_size_dirtyOnCpu = false;
      c5_eML_blk_kernel_kernel47<<<c5_db_grid, c5_db_block>>>
        (*chartInstance->c5_c_gpu_bboxPred_data,
         *chartInstance->c5_gpu_bboxPred_size,
         *chartInstance->c5_d_gpu_bboxPred_size, c5_i4,
         *chartInstance->c5_b_gpu_bboxPred_data);
    }

    c5_d_bboxPred_size[0] = c5_e_bboxPred_size[0];
    c5_d_bboxPred_size[1] = 4;
    c5_b_bboxPred_size_dirtyOnCpu = true;
    c5_eb_validLaunchParams = mwGetLaunchParameters((real_T)((int64_T)
      (c5_e_bboxPred_size[0] * 4 - 1) + 1L), &c5_eb_grid, &c5_eb_block, 1024U,
      65535U);
    if (c5_eb_validLaunchParams) {
      if (c5_c_bboxPred_size_dirtyOnCpu) {
        hipMemcpy(chartInstance->c5_d_gpu_bboxPred_size, &c5_e_bboxPred_size[0],
                   8UL, hipMemcpyHostToDevice);
      }

      c5_eML_blk_kernel_kernel48<<<c5_eb_grid, c5_eb_block>>>
        (*chartInstance->c5_b_gpu_bboxPred_data,
         *chartInstance->c5_d_gpu_bboxPred_size,
         *chartInstance->c5_c_gpu_bboxPred_data);
      c5_bboxPred_data_dirtyOnCpu = false;
    }

    c5_i47 = c5_scorePred_size[0];
    c5_fb_validLaunchParams = mwGetLaunchParameters((real_T)((int64_T)(c5_i47 -
      (int32_T)(c5_count + 1.0)) + 1L), &c5_fb_grid, &c5_fb_block, 1024U, 65535U);
    if (c5_fb_validLaunchParams) {
      c5_eML_blk_kernel_kernel49<<<c5_fb_grid, c5_fb_block>>>((int32_T)(c5_count
        + 1.0), c5_i47, *chartInstance->c5_gpu_idx_data);
      c5_idx_data_dirtyOnGpu = true;
    }

    c5_nxin = c5_scorePred_size[0];
    c5_b_size[0] = 1;
    c5_b_size[1] = c5_scorePred_size[0];
    c5_gb_validLaunchParams = mwGetLaunchParameters((real_T)((int64_T)
      (c5_scorePred_size[0] - 1) + 1L), &c5_gb_grid, &c5_gb_block, 1024U, 65535U);
    if (c5_gb_validLaunchParams) {
      hipMemcpy(chartInstance->c5_gpu_scorePred_size, &c5_scorePred_size[0],
                 4UL, hipMemcpyHostToDevice);
      c5_eML_blk_kernel_kernel50<<<c5_gb_grid, c5_gb_block>>>
        (*chartInstance->c5_gpu_scorePred_size, *chartInstance->c5_gpu_b_data);
      c5_b_data_dirtyOnCpu = false;
      c5_b_data_dirtyOnGpu = true;
    }

    c5_i5 = (c5_scorePred_size[0] - (int32_T)(c5_count + 1.0)) + 1;
    for (c5_e_k = 0; c5_e_k < c5_i5; c5_e_k++) {
      if (c5_idx_data_dirtyOnGpu) {
        hipMemcpy(&c5_idx_data[0], chartInstance->c5_gpu_idx_data, 3136UL,
                   hipMemcpyDeviceToHost);
        c5_idx_data_dirtyOnGpu = false;
      }

      if (c5_b_data_dirtyOnGpu) {
        hipMemcpy(&c5_b_data[0], chartInstance->c5_gpu_b_data, 784UL,
                   hipMemcpyDeviceToHost);
        c5_b_data_dirtyOnGpu = false;
      }

      c5_b_data[c5_idx_data[c5_e_k] - 1] = true;
      c5_b_data_dirtyOnCpu = true;
    }

    c5_b_n = 0;
    c5_n_dirtyOnGpu = false;
    c5_i6 = c5_b_size[1];
    c5_hb_validLaunchParams = mwGetLaunchParameters((real_T)((int64_T)
      (c5_b_size[1] - 1) + 1L), &c5_hb_grid, &c5_hb_block, 1024U, 65535U);
    if (c5_hb_validLaunchParams) {
      if (c5_b_data_dirtyOnCpu) {
        hipMemcpy(chartInstance->c5_gpu_b_data, &c5_b_data[0], 784UL,
                   hipMemcpyHostToDevice);
        c5_b_data_dirtyOnCpu = false;
      }

      hipMemcpy(chartInstance->c5_gpu_b_size, &c5_b_size[0], 8UL,
                 hipMemcpyHostToDevice);
      hipMemcpy(chartInstance->c5_b_gpu_n, &c5_b_n, 4UL, hipMemcpyHostToDevice);
      c5_eML_blk_kernel_kernel51<<<c5_hb_grid, c5_hb_block>>>
        (*chartInstance->c5_gpu_b_size, *chartInstance->c5_gpu_b_data, c5_i6,
         chartInstance->c5_b_gpu_n);
      c5_n_dirtyOnGpu = true;
    }

    if (c5_n_dirtyOnGpu) {
      hipMemcpy(&c5_b_n, chartInstance->c5_b_gpu_n, 4UL, hipMemcpyDeviceToHost);
    }

    c5_nxout = c5_scorePred_size[0] - c5_b_n;
    c5_k0 = -1;
    for (c5_f_k = 0; c5_f_k < c5_nxin; c5_f_k++) {
      c5_guard1 = false;
      if (c5_f_k + 1 > c5_b_size[1]) {
        c5_guard1 = true;
      } else {
        if (c5_b_data_dirtyOnGpu) {
          hipMemcpy(&c5_b_data[0], chartInstance->c5_gpu_b_data, 784UL,
                     hipMemcpyDeviceToHost);
          c5_b_data_dirtyOnGpu = false;
        }

        if (!c5_b_data[c5_f_k]) {
          c5_guard1 = true;
        }
      }

      if (c5_guard1) {
        c5_k0++;
        if (c5_scorePred_data_dirtyOnGpu) {
          hipMemcpy(&c5_scorePred_data[0], chartInstance->c5_gpu_scorePred_data,
                     3136UL, hipMemcpyDeviceToHost);
          c5_scorePred_data_dirtyOnGpu = false;
        }

        c5_scorePred_data[c5_k0] = c5_scorePred_data[c5_f_k];
        c5_scorePred_data_dirtyOnCpu = true;
      }
    }

    if (1 > c5_nxout) {
      c5_b_i7 = 0;
    } else {
      c5_b_i7 = c5_nxout;
    }

    c5_scorePred_size[0] = c5_b_i7;
    c5_i51 = c5_classPred_size[0];
    c5_ib_validLaunchParams = mwGetLaunchParameters((real_T)((int64_T)(c5_i51 -
      (int32_T)(c5_count + 1.0)) + 1L), &c5_ib_grid, &c5_ib_block, 1024U, 65535U);
    if (c5_ib_validLaunchParams) {
      c5_eML_blk_kernel_kernel52<<<c5_ib_grid, c5_ib_block>>>((int32_T)(c5_count
        + 1.0), c5_i51, *chartInstance->c5_gpu_idx_data);
      c5_idx_data_dirtyOnGpu = true;
    }

    c5_b_nxin = c5_classPred_size[0];
    c5_b_size[0] = 1;
    c5_b_size[1] = c5_classPred_size[0];
    c5_jb_validLaunchParams = mwGetLaunchParameters((real_T)((int64_T)
      (c5_classPred_size[0] - 1) + 1L), &c5_jb_grid, &c5_jb_block, 1024U, 65535U);
    if (c5_jb_validLaunchParams) {
      hipMemcpy(chartInstance->c5_gpu_classPred_size, &c5_classPred_size[0],
                 4UL, hipMemcpyHostToDevice);
      c5_eML_blk_kernel_kernel53<<<c5_jb_grid, c5_jb_block>>>
        (*chartInstance->c5_gpu_classPred_size, *chartInstance->c5_gpu_b_data);
      c5_b_data_dirtyOnCpu = false;
      c5_b_data_dirtyOnGpu = true;
    }

    c5_i8 = (c5_classPred_size[0] - (int32_T)(c5_count + 1.0)) + 1;
    for (c5_g_k = 0; c5_g_k < c5_i8; c5_g_k++) {
      if (c5_idx_data_dirtyOnGpu) {
        hipMemcpy(&c5_idx_data[0], chartInstance->c5_gpu_idx_data, 3136UL,
                   hipMemcpyDeviceToHost);
        c5_idx_data_dirtyOnGpu = false;
      }

      if (c5_b_data_dirtyOnGpu) {
        hipMemcpy(&c5_b_data[0], chartInstance->c5_gpu_b_data, 784UL,
                   hipMemcpyDeviceToHost);
        c5_b_data_dirtyOnGpu = false;
      }

      c5_b_data[c5_idx_data[c5_g_k] - 1] = true;
      c5_b_data_dirtyOnCpu = true;
    }

    c5_c_n = 0;
    c5_b_n_dirtyOnGpu = false;
    c5_i9 = c5_b_size[1];
    c5_kb_validLaunchParams = mwGetLaunchParameters((real_T)((int64_T)
      (c5_b_size[1] - 1) + 1L), &c5_kb_grid, &c5_kb_block, 1024U, 65535U);
    if (c5_kb_validLaunchParams) {
      if (c5_b_data_dirtyOnCpu) {
        hipMemcpy(chartInstance->c5_gpu_b_data, &c5_b_data[0], 784UL,
                   hipMemcpyHostToDevice);
      }

      hipMemcpy(chartInstance->c5_gpu_b_size, &c5_b_size[0], 8UL,
                 hipMemcpyHostToDevice);
      hipMemcpy(chartInstance->c5_c_gpu_n, &c5_c_n, 4UL, hipMemcpyHostToDevice);
      c5_eML_blk_kernel_kernel54<<<c5_kb_grid, c5_kb_block>>>
        (*chartInstance->c5_gpu_b_size, *chartInstance->c5_gpu_b_data, c5_i9,
         chartInstance->c5_c_gpu_n);
      c5_b_n_dirtyOnGpu = true;
    }

    if (c5_b_n_dirtyOnGpu) {
      hipMemcpy(&c5_c_n, chartInstance->c5_c_gpu_n, 4UL, hipMemcpyDeviceToHost);
    }

    c5_b_k0 = -1;
    for (c5_h_k = 0; c5_h_k < c5_b_nxin; c5_h_k++) {
      c5_guard1 = false;
      if (c5_h_k + 1 > c5_b_size[1]) {
        c5_guard1 = true;
      } else {
        if (c5_b_data_dirtyOnGpu) {
          hipMemcpy(&c5_b_data[0], chartInstance->c5_gpu_b_data, 784UL,
                     hipMemcpyDeviceToHost);
          c5_b_data_dirtyOnGpu = false;
        }

        if (!c5_b_data[c5_h_k]) {
          c5_guard1 = true;
        }
      }

      if (c5_guard1) {
        c5_b_k0++;
        if (c5_classPred_data_dirtyOnGpu) {
          hipMemcpy(&c5_classPred_data[0], chartInstance->c5_gpu_classPred_data,
                     3136UL, hipMemcpyDeviceToHost);
          c5_classPred_data_dirtyOnGpu = false;
        }

        c5_classPred_data[c5_b_k0] = c5_classPred_data[c5_h_k];
        c5_classPred_data_dirtyOnCpu = true;
      }
    }

    if (c5_d_bboxPred_size[0] == 0) {
      c5_bboxesX1Y1X2Y2_size[0] = 0;
      c5_bboxesX1Y1X2Y2_size[1] = 4;
      c5_bboxesX1Y1X2Y2_size_dirtyOnCpu = true;
      c5_b_scores_size[0] = c5_scorePred_size[0];
      c5_lb_validLaunchParams = mwGetLaunchParameters((real_T)((int64_T)
        (c5_scorePred_size[0] - 1) + 1L), &c5_lb_grid, &c5_lb_block, 1024U,
        65535U);
      if (c5_lb_validLaunchParams) {
        if (c5_scorePred_data_dirtyOnCpu) {
          hipMemcpy(chartInstance->c5_gpu_scorePred_data, &c5_scorePred_data[0],
                     3136UL, hipMemcpyHostToDevice);
        }

        hipMemcpy(chartInstance->c5_gpu_scorePred_size, &c5_scorePred_size[0],
                   4UL, hipMemcpyHostToDevice);
        c5_eML_blk_kernel_kernel66<<<c5_lb_grid, c5_lb_block>>>
          (*chartInstance->c5_gpu_scorePred_data,
           *chartInstance->c5_gpu_scorePred_size,
           *chartInstance->c5_gpu_scores_data);
      }
    } else {
      c5_x1_size[0] = c5_b_i7;
      c5_x1_size_dirtyOnCpu = true;
      if (c5_b_i7 != 0) {
        c5_sortDim = 2;
        if (c5_b_i7 != 1) {
          c5_sortDim = 1;
        }

        c5_inDims[0] = c5_b_i7;
        c5_inDims[1] = 1;
        c5_nb_validLaunchParams = mwGetLaunchParameters((real_T)((int64_T)
          (c5_scorePred_size[0] - 1) + 1L), &c5_nb_grid, &c5_nb_block, 1024U,
          65535U);
        if (c5_nb_validLaunchParams) {
          if (c5_scorePred_data_dirtyOnCpu) {
            hipMemcpy(chartInstance->c5_gpu_scorePred_data, &c5_scorePred_data
                       [0], 3136UL, hipMemcpyHostToDevice);
          }

          hipMemcpy(chartInstance->c5_gpu_scorePred_size, &c5_scorePred_size[0],
                     4UL, hipMemcpyHostToDevice);
          c5_eML_blk_kernel_kernel55<<<c5_nb_grid, c5_nb_block>>>
            (*chartInstance->c5_gpu_scorePred_data,
             *chartInstance->c5_gpu_scorePred_size,
             *chartInstance->c5_gpu_out_data);
        }

        c5_dv2[0] = (uint32_T)c5_b_i7;
        c5_x1_size[0] = c5_b_i7;
        c5_pb_validLaunchParams = mwGetLaunchParameters((real_T)((int64_T)
          ((int32_T)c5_dv2[0] - 1) + 1L), &c5_pb_grid, &c5_pb_block, 1024U,
          65535U);
        if (c5_pb_validLaunchParams) {
          hipMemcpy(chartInstance->c5_gpu_dv2, &c5_dv2[0], 8UL,
                     hipMemcpyHostToDevice);
          c5_eML_blk_kernel_kernel56<<<c5_pb_grid, c5_pb_block>>>
            (*chartInstance->c5_gpu_dv2, *chartInstance->c5_gpu_x1_data);
        }

        thrustSortImplWithIndex(&(*chartInstance->c5_gpu_out_data)[0],
          &(*chartInstance->c5_gpu_x1_data)[0], 2, &c5_inDims[0], c5_sortDim,
          'd', false);
      }

      c5_bboxesX1Y1X2Y2_size[0] = c5_x1_size[0];
      c5_bboxesX1Y1X2Y2_size[1] = 4;
      c5_bboxesX1Y1X2Y2_size_dirtyOnCpu = true;
      c5_mb_validLaunchParams = mwGetLaunchParameters((real_T)(((int64_T)
        (c5_x1_size[0] - 1) + 1L) * 4L), &c5_mb_grid, &c5_mb_block, 1024U,
        65535U);
      if (c5_mb_validLaunchParams) {
        hipMemcpy(chartInstance->c5_gpu_bboxesX1Y1X2Y2_size,
                   &c5_bboxesX1Y1X2Y2_size[0], 8UL, hipMemcpyHostToDevice);
        c5_bboxesX1Y1X2Y2_size_dirtyOnCpu = false;
        hipMemcpy(chartInstance->c5_gpu_bboxPred_size, &c5_d_bboxPred_size[0],
                   8UL, hipMemcpyHostToDevice);
        c5_b_bboxPred_size_dirtyOnCpu = false;
        if (c5_bboxPred_data_dirtyOnCpu) {
          hipMemcpy(chartInstance->c5_c_gpu_bboxPred_data, &c5_b_bboxPred_data
                     [0], 25088UL, hipMemcpyHostToDevice);
          c5_bboxPred_data_dirtyOnCpu = false;
        }

        hipMemcpy(chartInstance->c5_gpu_x1_size, &c5_x1_size[0], 4UL,
                   hipMemcpyHostToDevice);
        c5_x1_size_dirtyOnCpu = false;
        c5_eML_blk_kernel_kernel57<<<c5_mb_grid, c5_mb_block>>>
          (*chartInstance->c5_c_gpu_bboxPred_data,
           *chartInstance->c5_gpu_bboxPred_size, *chartInstance->c5_gpu_x1_data,
           *chartInstance->c5_gpu_bboxesX1Y1X2Y2_size,
           *chartInstance->c5_gpu_x1_size,
           *chartInstance->c5_gpu_bboxesX1Y1X2Y2_data);
        c5_bboxesX1Y1X2Y2_data_dirtyOnGpu = true;
      }

      c5_ob_validLaunchParams = mwGetLaunchParameters((real_T)((int64_T)
        (c5_x1_size[0] - 1) + 1L), &c5_ob_grid, &c5_ob_block, 1024U, 65535U);
      if (c5_ob_validLaunchParams) {
        if (c5_classPred_data_dirtyOnCpu) {
          hipMemcpy(chartInstance->c5_gpu_classPred_data, &c5_classPred_data[0],
                     3136UL, hipMemcpyHostToDevice);
        }

        if (c5_x1_size_dirtyOnCpu) {
          hipMemcpy(chartInstance->c5_gpu_x1_size, &c5_x1_size[0], 4UL,
                     hipMemcpyHostToDevice);
          c5_x1_size_dirtyOnCpu = false;
        }

        c5_eML_blk_kernel_kernel58<<<c5_ob_grid, c5_ob_block>>>
          (*chartInstance->c5_gpu_classPred_data, *chartInstance->c5_gpu_x1_data,
           *chartInstance->c5_gpu_x1_size, *chartInstance->c5_gpu_y1_data);
        c5_y1_data_dirtyOnGpu = true;
      }

      c5_selectedIndex_size[0] = c5_x1_size[0];
      c5_qb_validLaunchParams = mwGetLaunchParameters((real_T)((int64_T)
        (c5_x1_size[0] - 1) + 1L), &c5_qb_grid, &c5_qb_block, 1024U, 65535U);
      if (c5_qb_validLaunchParams) {
        if (c5_x1_size_dirtyOnCpu) {
          hipMemcpy(chartInstance->c5_gpu_x1_size, &c5_x1_size[0], 4UL,
                     hipMemcpyHostToDevice);
        }

        c5_eML_blk_kernel_kernel59<<<c5_qb_grid, c5_qb_block>>>
          (*chartInstance->c5_gpu_x1_size,
           *chartInstance->c5_gpu_selectedIndex_data);
        c5_selectedIndex_data_dirtyOnGpu = true;
      }

      c5_i61 = c5_x1_size[0] - 1;
      c5_rb_validLaunchParams = mwGetLaunchParameters((real_T)((int64_T)c5_i61 +
        1L), &c5_rb_grid, &c5_rb_block, 1024U, 65535U);
      if (c5_rb_validLaunchParams) {
        if (c5_bboxesX1Y1X2Y2_size_dirtyOnCpu) {
          hipMemcpy(chartInstance->c5_gpu_bboxesX1Y1X2Y2_size,
                     &c5_bboxesX1Y1X2Y2_size[0], 8UL, hipMemcpyHostToDevice);
          c5_bboxesX1Y1X2Y2_size_dirtyOnCpu = false;
        }

        c5_eML_blk_kernel_kernel60<<<c5_rb_grid, c5_rb_block>>>
          (*chartInstance->c5_gpu_bboxesX1Y1X2Y2_data,
           *chartInstance->c5_gpu_bboxesX1Y1X2Y2_size, c5_i61,
           *chartInstance->c5_gpu_area_data);
        c5_area_data_dirtyOnGpu = true;
      }

      c5_i64 = c5_x1_size[0] - 1;
      c5_sb_validLaunchParams = mwGetLaunchParameters((real_T)((int64_T)c5_i64 +
        1L), &c5_sb_grid, &c5_sb_block, 1024U, 65535U);
      if (c5_sb_validLaunchParams) {
        if (c5_bboxesX1Y1X2Y2_size_dirtyOnCpu) {
          hipMemcpy(chartInstance->c5_gpu_bboxesX1Y1X2Y2_size,
                     &c5_bboxesX1Y1X2Y2_size[0], 8UL, hipMemcpyHostToDevice);
          c5_bboxesX1Y1X2Y2_size_dirtyOnCpu = false;
        }

        c5_eML_blk_kernel_kernel61<<<c5_sb_grid, c5_sb_block>>>
          (*chartInstance->c5_gpu_bboxesX1Y1X2Y2_size,
           *chartInstance->c5_gpu_bboxesX1Y1X2Y2_data, c5_i64,
           *chartInstance->c5_gpu_x2_data);
        c5_x2_data_dirtyOnGpu = true;
      }

      c5_i66 = c5_x1_size[0] - 1;
      c5_tb_validLaunchParams = mwGetLaunchParameters((real_T)((int64_T)c5_i66 +
        1L), &c5_tb_grid, &c5_tb_block, 1024U, 65535U);
      if (c5_tb_validLaunchParams) {
        if (c5_bboxesX1Y1X2Y2_size_dirtyOnCpu) {
          hipMemcpy(chartInstance->c5_gpu_bboxesX1Y1X2Y2_size,
                     &c5_bboxesX1Y1X2Y2_size[0], 8UL, hipMemcpyHostToDevice);
        }

        c5_eML_blk_kernel_kernel62<<<c5_tb_grid, c5_tb_block>>>
          (*chartInstance->c5_gpu_bboxesX1Y1X2Y2_data,
           *chartInstance->c5_gpu_bboxesX1Y1X2Y2_size, c5_i66,
           *chartInstance->c5_gpu_y2_data);
        c5_y2_data_dirtyOnGpu = true;
      }

      c5_currentBox = -1;
      c5_i11 = c5_x1_size[0];
      for (c5_f_i = 0; c5_f_i < c5_i11; c5_f_i++) {
        c5_currentBox = c5_f_i;
        if (c5_y1_data_dirtyOnGpu) {
          hipMemcpy(&c5_y1_data[0], chartInstance->c5_gpu_y1_data, 6272UL,
                     hipMemcpyDeviceToHost);
          c5_y1_data_dirtyOnGpu = false;
        }

        if (muDoubleScalarIsNaN(c5_y1_data[c5_f_i])) {
          if (c5_selectedIndex_data_dirtyOnGpu) {
            hipMemcpy(&c5_selectedIndex_data[0],
                       chartInstance->c5_gpu_selectedIndex_data, 784UL,
                       hipMemcpyDeviceToHost);
            c5_selectedIndex_data_dirtyOnGpu = false;
          }

          c5_selectedIndex_data[c5_f_i] = false;
          c5_selectedIndex_data_dirtyOnCpu = true;
        } else {
          if (c5_selectedIndex_data_dirtyOnGpu) {
            hipMemcpy(&c5_selectedIndex_data[0],
                       chartInstance->c5_gpu_selectedIndex_data, 784UL,
                       hipMemcpyDeviceToHost);
            c5_selectedIndex_data_dirtyOnGpu = false;
          }

          if (c5_selectedIndex_data[c5_f_i]) {
            c5_b_i14 = (c5_x1_size[0] - c5_f_i) - 2;
            for (c5_b_j = 0; c5_b_j <= c5_b_i14; c5_b_j++) {
              c5_c_j = (c5_f_i + c5_b_j) + 1;
              if (c5_selectedIndex_data[c5_c_j] && (!(c5_y1_data[c5_c_j] !=
                    c5_y1_data[c5_f_i]))) {
                if (c5_bboxesX1Y1X2Y2_data_dirtyOnGpu) {
                  hipMemcpy(&c5_bboxesX1Y1X2Y2_data[0],
                             chartInstance->c5_gpu_bboxesX1Y1X2Y2_data, 25088UL,
                             hipMemcpyDeviceToHost);
                  c5_bboxesX1Y1X2Y2_data_dirtyOnGpu = false;
                }

                if (c5_x2_data_dirtyOnGpu) {
                  hipMemcpy(&c5_x2_data[0], chartInstance->c5_gpu_x2_data,
                             6272UL, hipMemcpyDeviceToHost);
                  c5_x2_data_dirtyOnGpu = false;
                }

                c5_width = muDoubleScalarMin(c5_x2_data[c5_f_i],
                  c5_x2_data[c5_c_j]) - muDoubleScalarMax
                  (c5_bboxesX1Y1X2Y2_data[c5_f_i], c5_bboxesX1Y1X2Y2_data[c5_c_j]);
                if (!(c5_width <= 0.0)) {
                  if (c5_y2_data_dirtyOnGpu) {
                    hipMemcpy(&c5_y2_data[0], chartInstance->c5_gpu_y2_data,
                               6272UL, hipMemcpyDeviceToHost);
                    c5_y2_data_dirtyOnGpu = false;
                  }

                  c5_height = muDoubleScalarMin(c5_y2_data[c5_f_i],
                    c5_y2_data[c5_c_j]) - muDoubleScalarMax
                    (c5_bboxesX1Y1X2Y2_data[c5_f_i + c5_bboxesX1Y1X2Y2_size[0]],
                     c5_bboxesX1Y1X2Y2_data[c5_c_j + c5_bboxesX1Y1X2Y2_size[0]]);
                  if (!(c5_height <= 0.0)) {
                    c5_areaOfIntersect = c5_width * c5_height;
                    if (c5_area_data_dirtyOnGpu) {
                      hipMemcpy(&c5_area_data[0],
                                 chartInstance->c5_gpu_area_data, 6272UL,
                                 hipMemcpyDeviceToHost);
                      c5_area_data_dirtyOnGpu = false;
                    }

                    if (c5_areaOfIntersect / ((c5_area_data[c5_f_i] +
                          c5_area_data[c5_c_j]) - c5_areaOfIntersect) > 0.5) {
                      c5_selectedIndex_data[c5_c_j] = false;
                      c5_selectedIndex_data_dirtyOnCpu = true;
                    }
                  }
                }
              }
            }
          }
        }
      }

      if (c5_currentBox + 2 > c5_selectedIndex_size[0]) {
        c5_i12 = 0;
        c5_i13 = 0;
      } else {
        c5_i12 = c5_currentBox + 1;
        c5_i13 = c5_selectedIndex_size[0];
      }

      c5_iv[1] = c5_i13 - c5_i12;
      c5_ub_validLaunchParams = mwGetLaunchParameters((real_T)((int64_T)(c5_iv[1]
        - 1) + 1L), &c5_ub_grid, &c5_ub_block, 1024U, 65535U);
      if (c5_ub_validLaunchParams) {
        if (c5_selectedIndex_data_dirtyOnCpu) {
          hipMemcpy(chartInstance->c5_gpu_selectedIndex_data,
                     &c5_selectedIndex_data[0], 784UL, hipMemcpyHostToDevice);
          c5_selectedIndex_data_dirtyOnCpu = false;
        }

        hipMemcpy(chartInstance->c5_gpu_iv, &c5_iv[0], 8UL,
                   hipMemcpyHostToDevice);
        c5_eML_blk_kernel_kernel63<<<c5_ub_grid, c5_ub_block>>>(c5_i12,
          *chartInstance->c5_gpu_iv, *chartInstance->c5_gpu_selectedIndex_data);
      }

      c5_index_size[0] = c5_selectedIndex_size[0];
      c5_vb_validLaunchParams = mwGetLaunchParameters((real_T)((int64_T)
        (c5_selectedIndex_size[0] - 1) + 1L), &c5_vb_grid, &c5_vb_block, 1024U,
        65535U);
      if (c5_vb_validLaunchParams) {
        if (c5_selectedIndex_data_dirtyOnCpu) {
          hipMemcpy(chartInstance->c5_gpu_selectedIndex_data,
                     &c5_selectedIndex_data[0], 784UL, hipMemcpyHostToDevice);
        }

        hipMemcpy(chartInstance->c5_gpu_selectedIndex_size,
                   &c5_selectedIndex_size[0], 4UL, hipMemcpyHostToDevice);
        c5_eML_blk_kernel_kernel64<<<c5_vb_grid, c5_vb_block>>>
          (*chartInstance->c5_gpu_selectedIndex_data,
           *chartInstance->c5_gpu_x1_data,
           *chartInstance->c5_gpu_selectedIndex_size,
           *chartInstance->c5_gpu_index_data);
        c5_index_data_dirtyOnGpu = true;
      }

      c5_end = c5_index_size[0] - 1;
      c5_b_trueCount = 0;
      for (c5_g_i = 0; c5_g_i <= c5_end; c5_g_i++) {
        if (c5_index_data_dirtyOnGpu) {
          hipMemcpy(&c5_index_data[0], chartInstance->c5_gpu_index_data, 784UL,
                     hipMemcpyDeviceToHost);
          c5_index_data_dirtyOnGpu = false;
        }

        if (c5_index_data[c5_g_i]) {
          c5_b_trueCount++;
        }
      }

      c5_iv1_size[0] = c5_b_trueCount;
      c5_b_partialTrueCount = 0;
      for (c5_h_i = 0; c5_h_i <= c5_end; c5_h_i++) {
        if (c5_index_data_dirtyOnGpu) {
          hipMemcpy(&c5_index_data[0], chartInstance->c5_gpu_index_data, 784UL,
                     hipMemcpyDeviceToHost);
          c5_index_data_dirtyOnGpu = false;
        }

        if (c5_index_data[c5_h_i]) {
          c5_iv1_data[c5_b_partialTrueCount] = (int16_T)(c5_h_i + 1);
          c5_iv1_data_dirtyOnCpu = true;
          c5_b_partialTrueCount++;
        }
      }

      c5_bboxesX1Y1X2Y2_size[0] = c5_b_trueCount;
      c5_bboxesX1Y1X2Y2_size[1] = 4;
      c5_bboxesX1Y1X2Y2_size_dirtyOnCpu = true;
      c5_wb_validLaunchParams = mwGetLaunchParameters((real_T)(((int64_T)
        (c5_iv1_size[0] - 1) + 1L) * 4L), &c5_wb_grid, &c5_wb_block, 1024U,
        65535U);
      if (c5_wb_validLaunchParams) {
        hipMemcpy(chartInstance->c5_gpu_bboxesX1Y1X2Y2_size,
                   &c5_bboxesX1Y1X2Y2_size[0], 8UL, hipMemcpyHostToDevice);
        c5_bboxesX1Y1X2Y2_size_dirtyOnCpu = false;
        if (c5_b_bboxPred_size_dirtyOnCpu) {
          hipMemcpy(chartInstance->c5_gpu_bboxPred_size, &c5_d_bboxPred_size[0],
                     8UL, hipMemcpyHostToDevice);
        }

        if (c5_bboxPred_data_dirtyOnCpu) {
          hipMemcpy(chartInstance->c5_c_gpu_bboxPred_data, &c5_b_bboxPred_data
                     [0], 25088UL, hipMemcpyHostToDevice);
        }

        if (c5_iv1_data_dirtyOnCpu) {
          hipMemcpy(chartInstance->c5_gpu_iv1_data, &c5_iv1_data[0], 1568UL,
                     hipMemcpyHostToDevice);
        }

        hipMemcpy(chartInstance->c5_gpu_iv1_size, &c5_iv1_size[0], 4UL,
                   hipMemcpyHostToDevice);
        c5_eML_blk_kernel_kernel65<<<c5_wb_grid, c5_wb_block>>>
          (*chartInstance->c5_c_gpu_bboxPred_data,
           *chartInstance->c5_gpu_bboxPred_size, *chartInstance->c5_gpu_iv1_data,
           *chartInstance->c5_gpu_bboxesX1Y1X2Y2_size,
           *chartInstance->c5_gpu_iv1_size,
           *chartInstance->c5_gpu_bboxesX1Y1X2Y2_data);
      }

      c5_b_end = c5_index_size[0] - 1;
      c5_c_trueCount = 0;
      for (c5_i_i = 0; c5_i_i <= c5_b_end; c5_i_i++) {
        if (c5_index_data_dirtyOnGpu) {
          hipMemcpy(&c5_index_data[0], chartInstance->c5_gpu_index_data, 784UL,
                     hipMemcpyDeviceToHost);
          c5_index_data_dirtyOnGpu = false;
        }

        if (c5_index_data[c5_i_i]) {
          c5_c_trueCount++;
        }
      }

      c5_b_scores_size[0] = c5_c_trueCount;
      c5_c_partialTrueCount = 0;
      for (c5_j_i = 0; c5_j_i <= c5_b_end; c5_j_i++) {
        if (c5_index_data_dirtyOnGpu) {
          hipMemcpy(&c5_index_data[0], chartInstance->c5_gpu_index_data, 784UL,
                     hipMemcpyDeviceToHost);
          c5_index_data_dirtyOnGpu = false;
        }

        if (c5_index_data[c5_j_i]) {
          if (c5_scorePred_data_dirtyOnGpu) {
            hipMemcpy(&c5_scorePred_data[0],
                       chartInstance->c5_gpu_scorePred_data, 3136UL,
                       hipMemcpyDeviceToHost);
            c5_scorePred_data_dirtyOnGpu = false;
          }

          c5_c_scores_data[c5_c_partialTrueCount] = c5_scorePred_data[c5_j_i];
          c5_scores_data_dirtyOnCpu = true;
          c5_c_partialTrueCount++;
        }
      }

      c5_b_nx = c5_index_size[0];
      c5_idx = 0;
      c5_ii = 1;
      c5_exitg1 = false;
      while ((!c5_exitg1) && (c5_ii <= c5_b_nx)) {
        if (c5_index_data_dirtyOnGpu) {
          hipMemcpy(&c5_index_data[0], chartInstance->c5_gpu_index_data, 784UL,
                     hipMemcpyDeviceToHost);
          c5_index_data_dirtyOnGpu = false;
        }

        if (c5_index_data[c5_ii - 1]) {
          c5_idx++;
          if (c5_idx >= c5_b_nx) {
            c5_exitg1 = true;
          } else {
            c5_ii++;
          }
        } else {
          c5_ii++;
        }
      }

      if (c5_index_size[0] != 1) {
        c5_iv1[0] = 1;
        if (1 > c5_idx) {
          c5_iv1[1] = 0;
        } else {
          c5_iv1[1] = c5_idx;
        }

        c5_indexShapeCheck(chartInstance, c5_index_size[0], c5_iv1);
      }
    }
  } else {
    c5_bboxesX1Y1X2Y2_size[0] = 0;
    c5_bboxesX1Y1X2Y2_size[1] = 4;
    c5_bboxesX1Y1X2Y2_size_dirtyOnCpu = true;
    c5_b_scores_size[0] = 0;
  }

  c5_bboxes_size[0] = c5_bboxesX1Y1X2Y2_size[0];
  c5_bboxes_size[1] = 4;
  c5_c_validLaunchParams = mwGetLaunchParameters((real_T)((int64_T)
    (c5_bboxesX1Y1X2Y2_size[0] * 4 - 1) + 1L), &c5_c_grid, &c5_c_block, 1024U,
    65535U);
  if (c5_c_validLaunchParams) {
    if (c5_bboxesX1Y1X2Y2_size_dirtyOnCpu) {
      hipMemcpy(chartInstance->c5_gpu_bboxesX1Y1X2Y2_size,
                 &c5_bboxesX1Y1X2Y2_size[0], 8UL, hipMemcpyHostToDevice);
    }

    c5_eML_blk_kernel_kernel67<<<c5_c_grid, c5_c_block>>>
      (*chartInstance->c5_gpu_bboxesX1Y1X2Y2_data,
       *chartInstance->c5_gpu_bboxesX1Y1X2Y2_size,
       chartInstance->c5_gpu_bboxes_data);
    c5_bboxes_data_dirtyOnGpu = true;
  }

  c5_scores_size[0] = c5_b_scores_size[0];
  c5_scores_size[1] = 1;
  c5_e_validLaunchParams = mwGetLaunchParameters((real_T)((int64_T)
    (c5_b_scores_size[0] - 1) + 1L), &c5_e_grid, &c5_e_block, 1024U, 65535U);
  if (c5_e_validLaunchParams) {
    if (c5_scores_data_dirtyOnCpu) {
      hipMemcpy(chartInstance->c5_gpu_scores_data, &c5_c_scores_data[0], 3136UL,
                 hipMemcpyHostToDevice);
    }

    hipMemcpy(chartInstance->c5_gpu_scores_size, &c5_b_scores_size[0], 4UL,
               hipMemcpyHostToDevice);
    c5_eML_blk_kernel_kernel68<<<c5_e_grid, c5_e_block>>>
      (*chartInstance->c5_gpu_scores_data, *chartInstance->c5_gpu_scores_size,
       chartInstance->c5_b_gpu_scores_data);
    c5_scores_data_dirtyOnGpu = true;
  }

  if (c5_bboxes_data_dirtyOnGpu) {
    hipMemcpy(&c5_b_bboxes_data[0], chartInstance->c5_gpu_bboxes_data,
               (uint32_T)(c5_bboxes_size[0] * 4) * sizeof(real_T),
               hipMemcpyDeviceToHost);
  }

  if (c5_scores_data_dirtyOnGpu) {
    hipMemcpy(&c5_b_scores_data[0], chartInstance->c5_b_gpu_scores_data,
               (uint32_T)c5_scores_size[0] * sizeof(real32_T),
               hipMemcpyDeviceToHost);
  }
}

static void c5_DeepLearningNetwork_setup(SFc5_LaneDetectionInstanceStruct
  *chartInstance, c5_yolov2ResNet50VehicleExample0_LaneDetection0 *c5_obj)
{
  c5_obj->setup();
}

static real32_T c5_callFcn(SFc5_LaneDetectionInstanceStruct *chartInstance,
  real32_T c5_input1, real32_T c5_input2)
{
  return muSingleScalarMin(c5_input1, c5_input2);
}

static real32_T c5_b_callFcn(SFc5_LaneDetectionInstanceStruct *chartInstance,
  real32_T c5_input1, real32_T c5_input2)
{
  return muSingleScalarMax(c5_input1, c5_input2);
}

static void c5_DeepLearningNetwork_activations(SFc5_LaneDetectionInstanceStruct *
  chartInstance, c5_yolov2ResNet50VehicleExample0_LaneDetection0 *c5_obj,
  real32_T c5_varargin_1[150528], real32_T c5_b_out[4704])
{
  c5_cell_wrap_18 (*c5_gpu_miniBatchT)[1];
  c5_cell_wrap_18 *c5_gpu_r;
  real32_T (*c5_gpu_varargin_1)[150528];
  real32_T (*c5_c_gpu_out)[4704];
  real32_T (*c5_gpu_outMiniBatch)[4704];
  hipMalloc(&c5_c_gpu_out, 18816UL);
  hipMalloc(&c5_gpu_outMiniBatch, 18816UL);
  hipMalloc(&c5_gpu_miniBatchT, 602112UL);
  hipMalloc(&c5_gpu_r, 602112UL);
  hipMalloc(&c5_gpu_varargin_1, 602112UL);
  hipMemcpy(c5_gpu_varargin_1, &c5_varargin_1[0], 602112UL,
             hipMemcpyHostToDevice);
  c5_DeepLearningNetwork_activations_kernel69<<<dim3(294U, 1U, 1U), dim3(512U,
    1U, 1U)>>>(*c5_gpu_varargin_1, c5_gpu_r);
  c5_DeepLearningNetwork_activations_kernel70<<<dim3(294U, 1U, 1U), dim3(512U,
    1U, 1U)>>>(c5_gpu_r, *c5_gpu_miniBatchT);
  hipMemcpy(c5_obj->getInputDataPointer(0), (*c5_gpu_miniBatchT)[0].f1,
             c5_obj->layers[0]->getOutputTensor(0)->getNumElements() * sizeof
             (real32_T), hipMemcpyDeviceToDevice);
  c5_obj->activations(56);
  hipMemcpy(*c5_gpu_outMiniBatch, c5_obj->getLayerOutput(56, 0), c5_obj->
             layers[56]->getOutputTensor(0)->getNumElements() * sizeof(real32_T),
             hipMemcpyDeviceToDevice);
  c5_DeepLearningNetwork_activations_kernel71<<<dim3(10U, 1U, 1U), dim3(512U, 1U,
    1U)>>>(*c5_gpu_outMiniBatch, *c5_c_gpu_out);
  hipMemcpy(&c5_b_out[0], c5_c_gpu_out, 18816UL, hipMemcpyDeviceToHost);
  hipFree(*c5_gpu_varargin_1);
  hipFree(c5_gpu_r);
  hipFree(*c5_gpu_miniBatchT);
  hipFree(*c5_gpu_outMiniBatch);
  hipFree(*c5_c_gpu_out);
}

static void c5_indexShapeCheck(SFc5_LaneDetectionInstanceStruct *chartInstance,
  int32_T c5_matrixSize, int32_T c5_indexSize[2])
{
}

static void c5_emlrt_marshallIn(SFc5_LaneDetectionInstanceStruct *chartInstance,
  const mxArray *c5_bboxes, const char_T *c5_identifier, real_T c5_y_data[],
  int32_T c5_y_size[2])
{
  emlrtMsgIdentifier c5_thisId;
  c5_thisId.fIdentifier = const_cast<const char_T *>(c5_identifier);
  c5_thisId.fParent = NULL;
  c5_thisId.bParentIsCell = false;
  c5_b_emlrt_marshallIn(chartInstance, sf_mex_dup(c5_bboxes), &c5_thisId,
                        c5_y_data, c5_y_size);
  sf_mex_destroy(&c5_bboxes);
}

static void c5_b_emlrt_marshallIn(SFc5_LaneDetectionInstanceStruct
  *chartInstance, const mxArray *c5_u, const emlrtMsgIdentifier *c5_parentId,
  real_T c5_y_data[], int32_T c5_y_size[2])
{
  real_T c5_dv_data[80];
  int32_T c5_dv_size[2];
  int32_T c5_i;
  int32_T c5_i1;
  int32_T c5_i2;
  uint32_T c5_uv[2];
  boolean_T c5_bv[2];
  for (c5_i = 0; c5_i < 2; c5_i++) {
    c5_uv[c5_i] = (uint32_T)(-16 * c5_i) + 20U;
  }

  c5_dv_size[0] = sf_mex_get_dimension(c5_u, 0);
  c5_dv_size[1] = sf_mex_get_dimension(c5_u, 1);
  for (c5_i1 = 0; c5_i1 < 2; c5_i1++) {
    c5_bv[c5_i1] = true;
  }

  sf_mex_import_vs(c5_parentId, sf_mex_dup(c5_u), &c5_dv_data, 1, 0, 0U, 1, 0U,
                   2, c5_bv, c5_uv, c5_dv_size);
  c5_y_size[0] = c5_dv_size[0];
  c5_y_size[1] = c5_dv_size[1];
  for (c5_i2 = 0; c5_i2 < c5_dv_size[0] * c5_dv_size[1]; c5_i2++) {
    c5_y_data[c5_i2] = c5_dv_data[c5_i2];
  }

  sf_mex_destroy(&c5_u);
}

static void c5_c_emlrt_marshallIn(SFc5_LaneDetectionInstanceStruct
  *chartInstance, const mxArray *c5_scores, const char_T *c5_identifier,
  real32_T c5_y_data[], int32_T c5_y_size[2])
{
  emlrtMsgIdentifier c5_thisId;
  c5_thisId.fIdentifier = const_cast<const char_T *>(c5_identifier);
  c5_thisId.fParent = NULL;
  c5_thisId.bParentIsCell = false;
  c5_d_emlrt_marshallIn(chartInstance, sf_mex_dup(c5_scores), &c5_thisId,
                        c5_y_data, c5_y_size);
  sf_mex_destroy(&c5_scores);
}

static void c5_d_emlrt_marshallIn(SFc5_LaneDetectionInstanceStruct
  *chartInstance, const mxArray *c5_u, const emlrtMsgIdentifier *c5_parentId,
  real32_T c5_y_data[], int32_T c5_y_size[2])
{
  static boolean_T c5_b_bv[2] = { true, false };

  int32_T c5_fv_size[2];
  int32_T c5_b_i;
  int32_T c5_i;
  int32_T c5_i1;
  real32_T c5_fv_data[20];
  uint32_T c5_uv[2];
  boolean_T c5_bv[2];
  for (c5_i = 0; c5_i < 2; c5_i++) {
    c5_uv[c5_i] = (uint32_T)(-19 * c5_i) + 20U;
  }

  c5_fv_size[0] = sf_mex_get_dimension(c5_u, 0);
  c5_fv_size[1] = sf_mex_get_dimension(c5_u, 1);
  for (c5_b_i = 0; c5_b_i < 2; c5_b_i++) {
    c5_bv[c5_b_i] = c5_b_bv[c5_b_i];
  }

  sf_mex_import_vs(c5_parentId, sf_mex_dup(c5_u), &c5_fv_data, 0, 1, 0U, 1, 0U,
                   2, c5_bv, c5_uv, c5_fv_size);
  c5_y_size[0] = c5_fv_size[0];
  c5_y_size[1] = 1;
  for (c5_i1 = 0; c5_i1 < c5_fv_size[0] * c5_fv_size[1]; c5_i1++) {
    c5_y_data[c5_i1] = c5_fv_data[c5_i1];
  }

  sf_mex_destroy(&c5_u);
}

static uint8_T c5_e_emlrt_marshallIn(SFc5_LaneDetectionInstanceStruct
  *chartInstance, const mxArray *c5_b_is_active_c5_LaneDetection, const char_T
  *c5_identifier)
{
  emlrtMsgIdentifier c5_thisId;
  uint8_T c5_y;
  c5_thisId.fIdentifier = const_cast<const char_T *>(c5_identifier);
  c5_thisId.fParent = NULL;
  c5_thisId.bParentIsCell = false;
  c5_y = c5_f_emlrt_marshallIn(chartInstance, sf_mex_dup
    (c5_b_is_active_c5_LaneDetection), &c5_thisId);
  sf_mex_destroy(&c5_b_is_active_c5_LaneDetection);
  return c5_y;
}

static uint8_T c5_f_emlrt_marshallIn(SFc5_LaneDetectionInstanceStruct
  *chartInstance, const mxArray *c5_u, const emlrtMsgIdentifier *c5_parentId)
{
  uint8_T c5_b_u;
  uint8_T c5_y;
  sf_mex_import(c5_parentId, sf_mex_dup(c5_u), &c5_b_u, 1, 3, 0U, 0, 0U, 0);
  c5_y = c5_b_u;
  sf_mex_destroy(&c5_u);
  return c5_y;
}

static void c5_emxConvertDynamicMatrixFromEmx_(SFc5_LaneDetectionInstanceStruct *
  chartInstance, c5_emxArray_real_T_20x4 *c5_emx, real_T c5_data[80], int32_T
  c5_size[2])
{
  memcpy((void *)c5_data, &c5_emx->data, sizeof(real_T) * (uint32_T)
         (c5_emx->size[0] * c5_emx->size[1]));
  memcpy((void *)c5_size, &c5_emx->size, sizeof(int32_T) << 1);
}

static void c5_b_emxConvertDynamicMatrixFromEmx_
  (SFc5_LaneDetectionInstanceStruct *chartInstance, c5_emxArray_real32_T_20x1
   *c5_emx, real32_T c5_data[20], int32_T c5_size[2])
{
  memcpy((void *)c5_data, &c5_emx->data, sizeof(real32_T) * (uint32_T)
         (c5_emx->size[0] * c5_emx->size[1]));
  memcpy((void *)c5_size, &c5_emx->size, sizeof(int32_T) << 1);
}

static __global__ __launch_bounds__(1024, 1) void c5_coder_reduce0(const
  real32_T c5_inputVar[150528], real32_T *c5_outputVar)
{
  real32_T c5_tmpRed0;
  real32_T c5_tmpRed1;
  uint32_T c5_blockStride;
  uint32_T c5_idx;
  uint32_T c5_m;
  uint32_T c5_mask;
  uint32_T c5_numActiveThreads;
  uint32_T c5_numActiveWarps;
  uint32_T c5_thBlkId;
  uint32_T c5_threadId;
  uint32_T c5_threadStride;
  c5_threadStride = (uint32_T)mwGetTotalThreadsLaunched();
  c5_threadId = (uint32_T)mwGetGlobalThreadIndex();
  c5_thBlkId = (uint32_T)mwGetThreadIndexWithinBlock();
  c5_blockStride = (uint32_T)mwGetThreadsPerBlock();
  c5_tmpRed0 = 0.0F;
  c5_tmpRed1 = 0.0F;
  c5_numActiveThreads = c5_blockStride;
  if (mwIsLastBlock()) {
    c5_m = 150527U % c5_blockStride;
    if (c5_m > 0U) {
      c5_numActiveThreads = c5_m;
    }
  }

  c5_numActiveWarps = ((c5_numActiveThreads + warpSize) - 1U) / warpSize;
  if (c5_threadId <= 150526U) {
    c5_tmpRed0 = c5_inputVar[c5_threadId];
    c5_tmpRed1 = c5_tmpRed0;
  }

  c5_mask = __ballot_sync(MAX_uint32_T, c5_threadId <= 150526U);
  for (c5_idx = c5_threadId + c5_threadStride; c5_idx <= 150526U; c5_idx +=
       c5_threadStride) {
    c5_tmpRed0 = c5_b_callFcn_device(c5_tmpRed0, c5_inputVar[c5_idx]);
    c5_tmpRed1 = c5_callFcn_device(c5_tmpRed1, c5_inputVar[c5_idx]);
  }

  c5_tmpRed0 = c5_workGroupReduction(c5_tmpRed0, c5_mask, c5_numActiveWarps);
  c5_tmpRed1 = c5_b_workGroupReduction(c5_tmpRed1, c5_mask, c5_numActiveWarps);
  if (c5_thBlkId == 0U) {
    c5_atomicOpreal32_T(&c5_outputVar[0], c5_tmpRed0);
    c5_b_atomicOpreal32_T(&c5_outputVar[1], c5_tmpRed1);
  }
}

static __device__ real32_T c5_threadGroupReduction(real32_T c5_val, uint32_T
  c5_lane, uint32_T c5_mask)
{
  real32_T c5_other;
  uint32_T c5_activeSize;
  uint32_T c5_offset;
  c5_activeSize = __popc(c5_mask);
  c5_offset = (c5_activeSize + 1U) / 2U;
  while (c5_activeSize > 1U) {
    c5_other = c5_shflDown1(c5_val, c5_offset, c5_mask);
    if (c5_lane + c5_offset < c5_activeSize) {
      c5_val = c5_b_callFcn_device(c5_val, c5_other);
    }

    c5_activeSize = c5_offset;
    c5_offset = (c5_offset + 1U) / 2U;
  }

  return c5_val;
}

static __device__ real32_T c5_shflDown1(real32_T c5_in1, uint32_T c5_offset,
  uint32_T c5_mask)
{
  int32_T *c5_tmp;
  c5_tmp = (int32_T *)&c5_in1;
  *c5_tmp = __shfl_down_sync(c5_mask, *c5_tmp, c5_offset);
  return *(real32_T *)c5_tmp;
}

static __device__ real32_T c5_workGroupReduction(real32_T c5_val, uint32_T
  c5_mask, uint32_T c5_numActiveWarps)
{
  __shared__ real32_T c5_shared[32];
  uint32_T c5_lane;
  uint32_T c5_thBlkId;
  uint32_T c5_widx;
  c5_thBlkId = (uint32_T)mwGetThreadIndexWithinBlock();
  c5_lane = c5_thBlkId % warpSize;
  c5_widx = c5_thBlkId / warpSize;
  c5_val = c5_threadGroupReduction(c5_val, c5_lane, c5_mask);
  if (c5_lane == 0U) {
    c5_shared[c5_widx] = c5_val;
  }

  __syncthreads();
  c5_mask = __ballot_sync(MAX_uint32_T, c5_lane < c5_numActiveWarps);
  c5_val = c5_shared[c5_lane];
  if (c5_widx == 0U) {
    c5_val = c5_threadGroupReduction(c5_val, c5_lane, c5_mask);
  }

  return c5_val;
}

static __device__ real32_T c5_b_threadGroupReduction(real32_T c5_val, uint32_T
  c5_lane, uint32_T c5_mask)
{
  real32_T c5_other;
  uint32_T c5_activeSize;
  uint32_T c5_offset;
  c5_activeSize = __popc(c5_mask);
  c5_offset = (c5_activeSize + 1U) / 2U;
  while (c5_activeSize > 1U) {
    c5_other = c5_shflDown1(c5_val, c5_offset, c5_mask);
    if (c5_lane + c5_offset < c5_activeSize) {
      c5_val = c5_callFcn_device(c5_val, c5_other);
    }

    c5_activeSize = c5_offset;
    c5_offset = (c5_offset + 1U) / 2U;
  }

  return c5_val;
}

static __device__ real32_T c5_b_workGroupReduction(real32_T c5_val, uint32_T
  c5_mask, uint32_T c5_numActiveWarps)
{
  __shared__ real32_T c5_shared[32];
  uint32_T c5_lane;
  uint32_T c5_thBlkId;
  uint32_T c5_widx;
  c5_thBlkId = (uint32_T)mwGetThreadIndexWithinBlock();
  c5_lane = c5_thBlkId % warpSize;
  c5_widx = c5_thBlkId / warpSize;
  c5_val = c5_b_threadGroupReduction(c5_val, c5_lane, c5_mask);
  if (c5_lane == 0U) {
    c5_shared[c5_widx] = c5_val;
  }

  __syncthreads();
  c5_mask = __ballot_sync(MAX_uint32_T, c5_lane < c5_numActiveWarps);
  c5_val = c5_shared[c5_lane];
  if (c5_widx == 0U) {
    c5_val = c5_b_threadGroupReduction(c5_val, c5_lane, c5_mask);
  }

  return c5_val;
}

static __device__ real32_T c5_atomicOpreal32_T(real32_T *c5_address, real32_T
  c5_value)
{
  uint32_T c5_assumed;
  uint32_T c5_old;
  uint32_T *c5_address_as_up;
  c5_address_as_up = (uint32_T *)c5_address;
  c5_old = *c5_address_as_up;
  do {
    c5_assumed = c5_old;
    c5_old = atomicCAS(c5_address_as_up, c5_old, __float_as_uint
                       (c5_b_callFcn_device(c5_value, __uint_as_float(c5_old))));
  } while (c5_assumed != c5_old);

  return __uint_as_float(c5_old);
}

static __device__ real32_T c5_b_atomicOpreal32_T(real32_T *c5_address, real32_T
  c5_value)
{
  uint32_T c5_assumed;
  uint32_T c5_old;
  uint32_T *c5_address_as_up;
  c5_address_as_up = (uint32_T *)c5_address;
  c5_old = *c5_address_as_up;
  do {
    c5_assumed = c5_old;
    c5_old = atomicCAS(c5_address_as_up, c5_old, __float_as_uint
                       (c5_callFcn_device(c5_value, __uint_as_float(c5_old))));
  } while (c5_assumed != c5_old);

  return __uint_as_float(c5_old);
}

static __global__ __launch_bounds__(512, 1) void c5_eML_blk_kernel_kernel1
  (int16_T c5_aux1[960])
{
  int32_T c5_i;
  c5_i = (int32_T)mwGetGlobalThreadIndex();
  if (c5_i < 960) {
    if (c5_i + 1 <= 480) {
      c5_aux1[c5_i] = (int16_T)(c5_i + 1);
    } else {
      c5_aux1[c5_i] = (int16_T)(960 - c5_i);
    }
  }
}

static __global__ __launch_bounds__(512, 1) void c5_eML_blk_kernel_kernel2
  (int16_T c5_aux2[1708])
{
  int32_T c5_i;
  c5_i = (int32_T)mwGetGlobalThreadIndex();
  if (c5_i < 1708) {
    if (c5_i + 1 <= 854) {
      c5_aux2[c5_i] = (int16_T)(c5_i + 1);
    } else {
      c5_aux2[c5_i] = (int16_T)(1708 - c5_i);
    }
  }
}

static __global__ __launch_bounds__(512, 1) void c5_eML_blk_kernel_kernel3(const
  int16_T c5_aux1[960], real_T c5_rowWeights[2016], int16_T c5_ipRowIndices[2016])
{
  real_T c5_absx;
  real_T c5_absx2;
  real_T c5_absx3;
  real_T c5_ipRowIdx;
  uint64_T c5_threadId;
  int32_T c5_k;
  int32_T c5_l;
  int32_T c5_oldIdx;
  int32_T c5_rowIdx;
  int32_T c5_rowIndices;
  c5_threadId = mwGetGlobalThreadIndex();
  c5_k = (int32_T)(c5_threadId % 9UL);
  c5_rowIdx = (int32_T)((c5_threadId - (uint64_T)c5_k) / 9UL);
  if ((c5_rowIdx < 224) && (c5_k < 9)) {
    c5_ipRowIdx = ((real_T)c5_rowIdx + 1.0) / 0.46666666666666667 +
      -0.5714285714285714;
    c5_rowIndices = (int32_T)floor(c5_ipRowIdx - 4.2857142857142856);
    c5_absx = fabs(0.46666666666666667 * (c5_ipRowIdx - ((real_T)(c5_rowIndices
      + c5_k) + 1.0)));
    c5_absx2 = c5_absx * c5_absx;
    c5_absx3 = pow(c5_absx, 3.0);
    c5_oldIdx = (c5_rowIndices + c5_k) + 1;
    if (c5_oldIdx - 1 == 0) {
      c5_l = 0;
    } else {
      c5_l = (int32_T)fmod((real_T)c5_oldIdx - 1.0, 960.0);
      if ((c5_l != 0) && (c5_oldIdx - 1 < 0)) {
        c5_l += 960;
      }
    }

    c5_ipRowIndices[c5_rowIdx + 224 * c5_k] = c5_aux1[c5_l];
    c5_rowWeights[c5_rowIdx + 224 * c5_k] = 0.46666666666666667 * (((1.5 *
      c5_absx3 - 2.5 * c5_absx2) + 1.0) * (real_T)(c5_absx <= 1.0) + (((-0.5 *
      c5_absx3 + 2.5 * c5_absx2) - 4.0 * c5_absx) + 2.0) * (real_T)((1.0 <
      c5_absx) && (c5_absx <= 2.0)));
  }
}

static __global__ __launch_bounds__(512, 1) void c5_eML_blk_kernel_kernel4(const
  int16_T c5_aux2[1708], real_T c5_colWeights[3584], int16_T c5_ipColIndices
  [3584])
{
  real_T c5_absx;
  real_T c5_absx2;
  real_T c5_absx3;
  real_T c5_ipColIdx;
  uint64_T c5_threadId;
  int32_T c5_colIdx;
  int32_T c5_colIndices;
  int32_T c5_k;
  int32_T c5_l;
  int32_T c5_oldIdx;
  c5_threadId = mwGetGlobalThreadIndex();
  c5_k = (int32_T)(c5_threadId % 16UL);
  c5_colIdx = (int32_T)((c5_threadId - (uint64_T)c5_k) / 16UL);
  if ((c5_colIdx < 224) && (c5_k < 16)) {
    c5_ipColIdx = ((real_T)c5_colIdx + 1.0) / 0.26229508196721313 + -1.40625;
    c5_colIndices = (int32_T)floor(c5_ipColIdx - 7.625);
    c5_absx = fabs(0.26229508196721313 * (c5_ipColIdx - ((real_T)(c5_colIndices
      + c5_k) + 1.0)));
    c5_absx2 = c5_absx * c5_absx;
    c5_absx3 = pow(c5_absx, 3.0);
    c5_oldIdx = (c5_colIndices + c5_k) + 1;
    if (c5_oldIdx - 1 == 0) {
      c5_l = 0;
    } else {
      c5_l = (int32_T)fmod((real_T)c5_oldIdx - 1.0, 1708.0);
      if ((c5_l != 0) && (c5_oldIdx - 1 < 0)) {
        c5_l += 1708;
      }
    }

    c5_ipColIndices[c5_colIdx + 224 * c5_k] = c5_aux2[c5_l];
    c5_colWeights[c5_colIdx + 224 * c5_k] = 0.26229508196721313 * (((1.5 *
      c5_absx3 - 2.5 * c5_absx2) + 1.0) * (real_T)(c5_absx <= 1.0) + (((-0.5 *
      c5_absx3 + 2.5 * c5_absx2) - 4.0 * c5_absx) + 2.0) * (real_T)((1.0 <
      c5_absx) && (c5_absx <= 2.0)));
  }
}

static __global__ __launch_bounds__(224, 1) void c5_eML_blk_kernel_kernel5(const
  real_T c5_rowWeights[2016], real_T c5_rowWeightsTotal[224])
{
  int32_T c5_j;
  c5_j = (int32_T)mwGetGlobalThreadIndex();
  if (c5_j < 224) {
    c5_rowWeightsTotal[c5_j] = c5_rowWeights[c5_j];
  }
}

static __global__ __launch_bounds__(224, 1) void c5_eML_blk_kernel_kernel6(const
  real_T c5_rowWeights[2016], const int32_T c5_xoffset, real_T
  c5_rowWeightsTotal[224])
{
  int32_T c5_j;
  c5_j = (int32_T)mwGetGlobalThreadIndex();
  if (c5_j < 224) {
    c5_rowWeightsTotal[c5_j] += c5_rowWeights[c5_xoffset + c5_j];
  }
}

static __global__ __launch_bounds__(224, 1) void c5_eML_blk_kernel_kernel7(const
  real_T c5_colWeights[3584], real_T c5_colWeightsTotal[224])
{
  int32_T c5_j;
  c5_j = (int32_T)mwGetGlobalThreadIndex();
  if (c5_j < 224) {
    c5_colWeightsTotal[c5_j] = c5_colWeights[c5_j];
  }
}

static __global__ __launch_bounds__(224, 1) void c5_eML_blk_kernel_kernel8(const
  real_T c5_colWeights[3584], const int32_T c5_xoffset, real_T
  c5_colWeightsTotal[224])
{
  int32_T c5_j;
  c5_j = (int32_T)mwGetGlobalThreadIndex();
  if (c5_j < 224) {
    c5_colWeightsTotal[c5_j] += c5_colWeights[c5_xoffset + c5_j];
  }
}

static __global__ __launch_bounds__(512, 1) void c5_eML_blk_kernel_kernel9(const
  real_T c5_colWeightsTotal[224], const real_T c5_colWeights[3584], const
  real32_T c5_b_In[1229760], const int16_T c5_ipColIndices[3584], real32_T
  c5_partialResize[322560])
{
  real_T c5_sumVal;
  uint64_T c5_threadId;
  uint64_T c5_tmpIndex;
  int32_T c5_colIdx;
  int32_T c5_dimIdx;
  int32_T c5_l;
  int32_T c5_rowIdx;
  c5_threadId = mwGetGlobalThreadIndex();
  c5_dimIdx = (int32_T)(c5_threadId % 3UL);
  c5_tmpIndex = (c5_threadId - (uint64_T)c5_dimIdx) / 3UL;
  c5_colIdx = (int32_T)(c5_tmpIndex % 224UL);
  c5_tmpIndex = (c5_tmpIndex - (uint64_T)c5_colIdx) / 224UL;
  c5_rowIdx = (int32_T)c5_tmpIndex;
  if ((c5_rowIdx < 480) && (c5_colIdx < 224) && (c5_dimIdx < 3)) {
    c5_sumVal = 0.0;
    for (c5_l = 0; c5_l < 16; c5_l++) {
      c5_sumVal += (real_T)c5_b_In[(c5_rowIdx + 480 * ((int32_T)
        c5_ipColIndices[c5_colIdx + 224 * c5_l] - 1)) + 409920 * c5_dimIdx] *
        (c5_colWeights[c5_colIdx + 224 * c5_l] / c5_colWeightsTotal[c5_colIdx]);
    }

    c5_partialResize[(c5_rowIdx + 480 * c5_colIdx) + 107520 * c5_dimIdx] =
      (real32_T)c5_sumVal;
  }
}

static __global__ __launch_bounds__(512, 1) void c5_eML_blk_kernel_kernel10(
  const real_T c5_rowWeightsTotal[224], const real_T c5_rowWeights[2016], const
  real32_T c5_partialResize[322560], const int16_T c5_ipRowIndices[2016],
  real32_T c5_b_out[150528])
{
  real_T c5_sumVal;
  uint64_T c5_threadId;
  uint64_T c5_tmpIndex;
  int32_T c5_colIdx;
  int32_T c5_dimIdx;
  int32_T c5_l;
  int32_T c5_rowIdx;
  c5_threadId = mwGetGlobalThreadIndex();
  c5_dimIdx = (int32_T)(c5_threadId % 3UL);
  c5_tmpIndex = (c5_threadId - (uint64_T)c5_dimIdx) / 3UL;
  c5_rowIdx = (int32_T)(c5_tmpIndex % 224UL);
  c5_tmpIndex = (c5_tmpIndex - (uint64_T)c5_rowIdx) / 224UL;
  c5_colIdx = (int32_T)c5_tmpIndex;
  if ((c5_colIdx < 224) && (c5_rowIdx < 224) && (c5_dimIdx < 3)) {
    c5_sumVal = 0.0;
    for (c5_l = 0; c5_l < 9; c5_l++) {
      c5_sumVal += (real_T)c5_partialResize[(((int32_T)c5_ipRowIndices[c5_rowIdx
        + 224 * c5_l] + 480 * c5_colIdx) + 107520 * c5_dimIdx) - 1] *
        (c5_rowWeights[c5_rowIdx + 224 * c5_l] / c5_rowWeightsTotal[c5_rowIdx]);
    }

    c5_b_out[(c5_rowIdx + 224 * c5_colIdx) + 50176 * c5_dimIdx] = (real32_T)
      c5_sumVal;
  }
}

static __global__ __launch_bounds__(32, 1) void c5_eML_blk_kernel_kernel11
  (real32_T c5_b_out[150528], real32_T c5_outVal[2])
{
  int32_T c5_indV;
  c5_indV = (int32_T)mwGetGlobalThreadIndex();
  if (c5_indV < 2) {
    c5_outVal[c5_indV] = c5_b_out[150527];
    c5_outVal[c5_indV] = c5_b_out[150527];
  }
}

static __global__ __launch_bounds__(512, 1) void c5_eML_blk_kernel_kernel12(
  const real32_T c5_y, const real32_T c5_outVal, real32_T c5_b_out[150528],
  real32_T c5_c_out[150528])
{
  int32_T c5_i2;
  c5_i2 = (int32_T)mwGetGlobalThreadIndex();
  if (c5_i2 < 150528) {
    c5_c_out[c5_i2] = (c5_b_out[c5_i2] - c5_outVal) / c5_y;
  }
}

static __global__ __launch_bounds__(32, 1) void c5_eML_blk_kernel_kernel13(const
  int8_T c5_dv[8], real_T c5_anchors[8])
{
  int32_T c5_i3;
  c5_i3 = (int32_T)mwGetGlobalThreadIndex();
  if (c5_i3 < 8) {
    c5_anchors[c5_i3] = (real_T)c5_dv[c5_i3];
  }
}

static __global__ __launch_bounds__(32, 1) void c5_eML_blk_kernel_kernel14(const
  real_T c5_dv1[4], real_T c5_anchors[4], real_T c5_b_anchors[8])
{
  int32_T c5_i4;
  c5_i4 = (int32_T)mwGetGlobalThreadIndex();
  if (c5_i4 < 4) {
    c5_b_anchors[c5_i4] = c5_dv1[c5_i4];
    c5_anchors[c5_i4] = c5_b_anchors[c5_i4 + 4] / 16.0;
    c5_b_anchors[c5_i4 + 4] = c5_anchors[c5_i4];
  }
}

static __global__ __launch_bounds__(512, 1) void c5_eML_blk_kernel_kernel15(
  const real_T c5_anchors[8], const real32_T c5_tmpFeatureMap[4704], real32_T
  c5_boxOut[4704])
{
  uint64_T c5_threadId;
  uint64_T c5_tmpIndex;
  int32_T c5_anchorIdx;
  int32_T c5_colIdx;
  int32_T c5_ind;
  int32_T c5_rowIdx;
  real32_T c5_bh;
  real32_T c5_bw;
  real32_T c5_cx;
  real32_T c5_cy;
  c5_threadId = mwGetGlobalThreadIndex();
  c5_rowIdx = (int32_T)(c5_threadId % 14UL);
  c5_tmpIndex = (c5_threadId - (uint64_T)c5_rowIdx) / 14UL;
  c5_colIdx = (int32_T)(c5_tmpIndex % 14UL);
  c5_tmpIndex = (c5_tmpIndex - (uint64_T)c5_colIdx) / 14UL;
  c5_anchorIdx = (int32_T)c5_tmpIndex;
  if ((c5_anchorIdx < 4) && (c5_colIdx < 14) && (c5_rowIdx < 14)) {
    c5_ind = ((((c5_rowIdx * 14) << 2) + (c5_colIdx << 2)) + c5_anchorIdx) + 1;
    c5_cx = (c5_tmpFeatureMap[((c5_rowIdx + 14 * c5_colIdx) + 196 * c5_anchorIdx)
             + 784] + (real32_T)c5_colIdx) * 16.0F;
    c5_cy = (c5_tmpFeatureMap[((c5_rowIdx + 14 * c5_colIdx) + 196 * c5_anchorIdx)
             + 1568] + (real32_T)c5_rowIdx) * 16.0F;
    c5_bw = c5_tmpFeatureMap[((c5_rowIdx + 14 * c5_colIdx) + 196 * c5_anchorIdx)
      + 2352] * (real32_T)c5_anchors[c5_anchorIdx + 4] * 16.0F;
    c5_bh = c5_tmpFeatureMap[((c5_rowIdx + 14 * c5_colIdx) + 196 * c5_anchorIdx)
      + 3136] * (real32_T)c5_anchors[c5_anchorIdx] * 16.0F;
    c5_boxOut[c5_ind - 1] = c5_cx - c5_bw / 2.0F;
    c5_boxOut[c5_ind + 783] = c5_cy - c5_bh / 2.0F;
    c5_boxOut[c5_ind + 1567] = c5_cx + c5_bw / 2.0F;
    c5_boxOut[c5_ind + 2351] = c5_cy + c5_bh / 2.0F;
    c5_boxOut[c5_ind + 3135] = c5_tmpFeatureMap[(c5_rowIdx + 14 * c5_colIdx) +
      196 * c5_anchorIdx] * c5_tmpFeatureMap[((c5_rowIdx + 14 * c5_colIdx) + 196
      * c5_anchorIdx) + 3920];
    c5_boxOut[c5_ind + 3919] = 1.0F;
  }
}

static __global__ __launch_bounds__(512, 1) void c5_eML_blk_kernel_kernel16(
  const real32_T c5_boxOut[4704], boolean_T c5_bv[784])
{
  int32_T c5_i5;
  c5_i5 = (int32_T)mwGetGlobalThreadIndex();
  if (c5_i5 < 784) {
    c5_bv[c5_i5] = (boolean_T)((real_T)c5_boxOut[c5_i5 + 3136] >= 0.2);
  }
}

static __global__ __launch_bounds__(1024, 1) void c5_eML_blk_kernel_kernel17(
  const real32_T c5_boxOut[4704], const int16_T c5_ii_data[784], const int32_T
  c5_thresholdedPrediction_size[2], const int32_T c5_ii_size[1], real32_T
  c5_thresholdedPrediction_data[4704])
{
  int64_T c5_loopEnd;
  uint64_T c5_idx;
  uint64_T c5_threadId;
  uint64_T c5_threadStride;
  uint64_T c5_tmpIndex;
  int32_T c5_i6;
  int32_T c5_i9;
  c5_threadId = mwGetGlobalThreadIndex();
  c5_threadStride = mwGetTotalThreadsLaunched();
  c5_loopEnd = ((int64_T)(c5_ii_size[0] - 1) + 1L) * 6L - 1L;
  for (c5_idx = c5_threadId; c5_idx <= (uint64_T)c5_loopEnd; c5_idx +=
       c5_threadStride) {
    c5_i9 = (int32_T)(c5_idx % ((uint64_T)(c5_ii_size[0] - 1) + 1UL));
    c5_tmpIndex = (c5_idx - (uint64_T)c5_i9) / ((uint64_T)(c5_ii_size[0] - 1) +
      1UL);
    c5_i6 = (int32_T)c5_tmpIndex;
    c5_thresholdedPrediction_data[c5_i9 + c5_thresholdedPrediction_size[0] *
      c5_i6] = c5_boxOut[((int32_T)c5_ii_data[c5_i9] + 784 * c5_i6) - 1];
  }
}

static __global__ __launch_bounds__(1024, 1) void c5_eML_blk_kernel_kernel18(
  const real32_T c5_thresholdedPrediction_data[4704], const int32_T
  c5_thresholdedPrediction_size[2], const int32_T c5_bboxesX1Y1X2Y2_size[2],
  const int32_T c5_i7, real_T c5_bboxesX1Y1X2Y2_data[3136])
{
  int64_T c5_loopEnd;
  uint64_T c5_idx;
  uint64_T c5_threadId;
  uint64_T c5_threadStride;
  int32_T c5_i13;
  int32_T c5_i8;
  c5_threadId = mwGetGlobalThreadIndex();
  c5_threadStride = mwGetTotalThreadsLaunched();
  c5_loopEnd = ((int64_T)c5_i7 + 1L) * 4L - 1L;
  for (c5_idx = c5_threadId; c5_idx <= (uint64_T)c5_loopEnd; c5_idx +=
       c5_threadStride) {
    c5_i13 = (int32_T)(c5_idx % ((uint64_T)c5_i7 + 1UL));
    c5_i8 = (int32_T)((c5_idx - (uint64_T)c5_i13) / ((uint64_T)c5_i7 + 1UL));
    c5_bboxesX1Y1X2Y2_data[c5_i13 + c5_bboxesX1Y1X2Y2_size[0] * c5_i8] = (real_T)
      c5_thresholdedPrediction_data[c5_i13 + c5_thresholdedPrediction_size[0] *
      c5_i8];
  }
}

static __global__ __launch_bounds__(1024, 1) void c5_eML_blk_kernel_kernel19(
  const real_T c5_bboxesX1Y1X2Y2_data[3136], const int32_T c5_i10, real_T
  c5_x1_data[784])
{
  int64_T c5_loopEnd;
  uint64_T c5_idx;
  uint64_T c5_threadId;
  uint64_T c5_threadStride;
  int32_T c5_i12;
  c5_threadId = mwGetGlobalThreadIndex();
  c5_threadStride = mwGetTotalThreadsLaunched();
  c5_loopEnd = (int64_T)c5_i10;
  for (c5_idx = c5_threadId; c5_idx <= (uint64_T)c5_loopEnd; c5_idx +=
       c5_threadStride) {
    c5_i12 = (int32_T)c5_idx;
    c5_x1_data[c5_i12] = c5_bboxesX1Y1X2Y2_data[c5_i12];
  }
}

static __global__ __launch_bounds__(1024, 1) void c5_eML_blk_kernel_kernel20(
  const real_T c5_bboxesX1Y1X2Y2_data[3136], const int32_T
  c5_bboxesX1Y1X2Y2_size[2], const int32_T c5_i14, real_T c5_y1_data[784])
{
  int64_T c5_loopEnd;
  uint64_T c5_idx;
  uint64_T c5_threadId;
  uint64_T c5_threadStride;
  int32_T c5_i16;
  c5_threadId = mwGetGlobalThreadIndex();
  c5_threadStride = mwGetTotalThreadsLaunched();
  c5_loopEnd = (int64_T)c5_i14;
  for (c5_idx = c5_threadId; c5_idx <= (uint64_T)c5_loopEnd; c5_idx +=
       c5_threadStride) {
    c5_i16 = (int32_T)c5_idx;
    c5_y1_data[c5_i16] = c5_bboxesX1Y1X2Y2_data[c5_i16 + c5_bboxesX1Y1X2Y2_size
      [0]];
  }
}

static __global__ __launch_bounds__(1024, 1) void c5_eML_blk_kernel_kernel21(
  const real_T c5_bboxesX1Y1X2Y2_data[3136], const int32_T
  c5_bboxesX1Y1X2Y2_size[2], const int32_T c5_i17, real_T c5_x2_data[784])
{
  int64_T c5_loopEnd;
  uint64_T c5_idx;
  uint64_T c5_threadId;
  uint64_T c5_threadStride;
  int32_T c5_i18;
  c5_threadId = mwGetGlobalThreadIndex();
  c5_threadStride = mwGetTotalThreadsLaunched();
  c5_loopEnd = (int64_T)c5_i17;
  for (c5_idx = c5_threadId; c5_idx <= (uint64_T)c5_loopEnd; c5_idx +=
       c5_threadStride) {
    c5_i18 = (int32_T)c5_idx;
    c5_x2_data[c5_i18] = c5_bboxesX1Y1X2Y2_data[c5_i18 +
      (c5_bboxesX1Y1X2Y2_size[0] << 1)];
  }
}

static __global__ __launch_bounds__(1024, 1) void c5_eML_blk_kernel_kernel22(
  const real_T c5_bboxesX1Y1X2Y2_data[3136], const int32_T
  c5_bboxesX1Y1X2Y2_size[2], const int32_T c5_i19, real_T c5_y2_data[784])
{
  int64_T c5_loopEnd;
  uint64_T c5_idx;
  uint64_T c5_threadId;
  uint64_T c5_threadStride;
  int32_T c5_i20;
  c5_threadId = mwGetGlobalThreadIndex();
  c5_threadStride = mwGetTotalThreadsLaunched();
  c5_loopEnd = (int64_T)c5_i19;
  for (c5_idx = c5_threadId; c5_idx <= (uint64_T)c5_loopEnd; c5_idx +=
       c5_threadStride) {
    c5_i20 = (int32_T)c5_idx;
    c5_y2_data[c5_i20] = c5_bboxesX1Y1X2Y2_data[c5_i20 + c5_bboxesX1Y1X2Y2_size
      [0] * 3];
  }
}

static __global__ __launch_bounds__(1024, 1) void c5_eML_blk_kernel_kernel23(
  const int32_T c5_end, real_T c5_x1_data[784])
{
  int64_T c5_loopEnd;
  uint64_T c5_idx;
  uint64_T c5_threadId;
  uint64_T c5_threadStride;
  int32_T c5_i;
  c5_threadId = mwGetGlobalThreadIndex();
  c5_threadStride = mwGetTotalThreadsLaunched();
  c5_loopEnd = (int64_T)(c5_end - 1);
  for (c5_idx = c5_threadId; c5_idx <= (uint64_T)c5_loopEnd; c5_idx +=
       c5_threadStride) {
    c5_i = (int32_T)c5_idx;
    if (c5_x1_data[c5_i] < 1.0) {
      c5_x1_data[c5_i] = 1.0;
    }
  }
}

static __global__ __launch_bounds__(1024, 1) void c5_eML_blk_kernel_kernel24(
  const int32_T c5_end, real_T c5_y1_data[784])
{
  int64_T c5_loopEnd;
  uint64_T c5_idx;
  uint64_T c5_threadId;
  uint64_T c5_threadStride;
  int32_T c5_i;
  c5_threadId = mwGetGlobalThreadIndex();
  c5_threadStride = mwGetTotalThreadsLaunched();
  c5_loopEnd = (int64_T)(c5_end - 1);
  for (c5_idx = c5_threadId; c5_idx <= (uint64_T)c5_loopEnd; c5_idx +=
       c5_threadStride) {
    c5_i = (int32_T)c5_idx;
    if (c5_y1_data[c5_i] < 1.0) {
      c5_y1_data[c5_i] = 1.0;
    }
  }
}

static __global__ __launch_bounds__(1024, 1) void c5_eML_blk_kernel_kernel25(
  const int32_T c5_end, real_T c5_x2_data[784])
{
  int64_T c5_loopEnd;
  uint64_T c5_idx;
  uint64_T c5_threadId;
  uint64_T c5_threadStride;
  int32_T c5_i;
  c5_threadId = mwGetGlobalThreadIndex();
  c5_threadStride = mwGetTotalThreadsLaunched();
  c5_loopEnd = (int64_T)(c5_end - 1);
  for (c5_idx = c5_threadId; c5_idx <= (uint64_T)c5_loopEnd; c5_idx +=
       c5_threadStride) {
    c5_i = (int32_T)c5_idx;
    if (c5_x2_data[c5_i] > 224.0) {
      c5_x2_data[c5_i] = 224.0;
    }
  }
}

static __global__ __launch_bounds__(1024, 1) void c5_eML_blk_kernel_kernel26(
  const int32_T c5_end, real_T c5_y2_data[784])
{
  int64_T c5_loopEnd;
  uint64_T c5_idx;
  uint64_T c5_threadId;
  uint64_T c5_threadStride;
  int32_T c5_i;
  c5_threadId = mwGetGlobalThreadIndex();
  c5_threadStride = mwGetTotalThreadsLaunched();
  c5_loopEnd = (int64_T)(c5_end - 1);
  for (c5_idx = c5_threadId; c5_idx <= (uint64_T)c5_loopEnd; c5_idx +=
       c5_threadStride) {
    c5_i = (int32_T)c5_idx;
    if (c5_y2_data[c5_i] > 224.0) {
      c5_y2_data[c5_i] = 224.0;
    }
  }
}

static __global__ __launch_bounds__(1024, 1) void c5_eML_blk_kernel_kernel27(
  const real_T c5_x1_data[784], const int32_T c5_loop_ub, real_T
  c5_bboxesX1Y1X2Y2_data[3136])
{
  int64_T c5_loopEnd;
  uint64_T c5_idx;
  uint64_T c5_threadId;
  uint64_T c5_threadStride;
  int32_T c5_i21;
  c5_threadId = mwGetGlobalThreadIndex();
  c5_threadStride = mwGetTotalThreadsLaunched();
  c5_loopEnd = (int64_T)c5_loop_ub;
  for (c5_idx = c5_threadId; c5_idx <= (uint64_T)c5_loopEnd; c5_idx +=
       c5_threadStride) {
    c5_i21 = (int32_T)c5_idx;
    c5_bboxesX1Y1X2Y2_data[c5_i21] = c5_x1_data[c5_i21];
  }
}

static __global__ __launch_bounds__(1024, 1) void c5_eML_blk_kernel_kernel28(
  const real_T c5_y1_data[784], const int32_T c5_bboxesX1Y1X2Y2_size[2], const
  int32_T c5_loop_ub, real_T c5_bboxesX1Y1X2Y2_data[3136])
{
  int64_T c5_loopEnd;
  uint64_T c5_idx;
  uint64_T c5_threadId;
  uint64_T c5_threadStride;
  int32_T c5_i22;
  c5_threadId = mwGetGlobalThreadIndex();
  c5_threadStride = mwGetTotalThreadsLaunched();
  c5_loopEnd = (int64_T)c5_loop_ub;
  for (c5_idx = c5_threadId; c5_idx <= (uint64_T)c5_loopEnd; c5_idx +=
       c5_threadStride) {
    c5_i22 = (int32_T)c5_idx;
    c5_bboxesX1Y1X2Y2_data[c5_i22 + c5_bboxesX1Y1X2Y2_size[0]] =
      c5_y1_data[c5_i22];
  }
}

static __global__ __launch_bounds__(1024, 1) void c5_eML_blk_kernel_kernel29(
  const real_T c5_x2_data[784], const int32_T c5_bboxesX1Y1X2Y2_size[2], const
  int32_T c5_loop_ub, real_T c5_bboxesX1Y1X2Y2_data[3136])
{
  int64_T c5_loopEnd;
  uint64_T c5_idx;
  uint64_T c5_threadId;
  uint64_T c5_threadStride;
  int32_T c5_i23;
  c5_threadId = mwGetGlobalThreadIndex();
  c5_threadStride = mwGetTotalThreadsLaunched();
  c5_loopEnd = (int64_T)c5_loop_ub;
  for (c5_idx = c5_threadId; c5_idx <= (uint64_T)c5_loopEnd; c5_idx +=
       c5_threadStride) {
    c5_i23 = (int32_T)c5_idx;
    c5_bboxesX1Y1X2Y2_data[c5_i23 + (c5_bboxesX1Y1X2Y2_size[0] << 1)] =
      c5_x2_data[c5_i23];
  }
}

static __global__ __launch_bounds__(1024, 1) void c5_eML_blk_kernel_kernel30(
  const real_T c5_y2_data[784], const int32_T c5_bboxesX1Y1X2Y2_size[2], const
  int32_T c5_loop_ub, real_T c5_bboxesX1Y1X2Y2_data[3136])
{
  int64_T c5_loopEnd;
  uint64_T c5_idx;
  uint64_T c5_threadId;
  uint64_T c5_threadStride;
  int32_T c5_i24;
  c5_threadId = mwGetGlobalThreadIndex();
  c5_threadStride = mwGetTotalThreadsLaunched();
  c5_loopEnd = (int64_T)c5_loop_ub;
  for (c5_idx = c5_threadId; c5_idx <= (uint64_T)c5_loopEnd; c5_idx +=
       c5_threadStride) {
    c5_i24 = (int32_T)c5_idx;
    c5_bboxesX1Y1X2Y2_data[c5_i24 + c5_bboxesX1Y1X2Y2_size[0] * 3] =
      c5_y2_data[c5_i24];
  }
}

static __global__ __launch_bounds__(1024, 1) void c5_eML_blk_kernel_kernel31(
  const real_T c5_bboxesX1Y1X2Y2_data[3136], const int32_T c5_loop_ub, real_T
  c5_bboxPred_data[3136])
{
  int64_T c5_loopEnd;
  uint64_T c5_idx;
  uint64_T c5_threadId;
  uint64_T c5_threadStride;
  int32_T c5_i28;
  c5_threadId = mwGetGlobalThreadIndex();
  c5_threadStride = mwGetTotalThreadsLaunched();
  c5_loopEnd = (int64_T)c5_loop_ub;
  for (c5_idx = c5_threadId; c5_idx <= (uint64_T)c5_loopEnd; c5_idx +=
       c5_threadStride) {
    c5_i28 = (int32_T)c5_idx;
    c5_bboxPred_data[c5_i28] = ((c5_bboxesX1Y1X2Y2_data[c5_i28] - 0.5) * 3.8125
      + -1.40625) + 0.5;
  }
}

static __global__ __launch_bounds__(1024, 1) void c5_eML_blk_kernel_kernel32(
  const real_T c5_bboxesX1Y1X2Y2_data[3136], const int32_T
  c5_bboxesX1Y1X2Y2_size[2], const int32_T c5_bboxPred_size[2], const int32_T
  c5_loop_ub, real_T c5_bboxPred_data[3136])
{
  int64_T c5_loopEnd;
  uint64_T c5_idx;
  uint64_T c5_threadId;
  uint64_T c5_threadStride;
  int32_T c5_i29;
  c5_threadId = mwGetGlobalThreadIndex();
  c5_threadStride = mwGetTotalThreadsLaunched();
  c5_loopEnd = (int64_T)c5_loop_ub;
  for (c5_idx = c5_threadId; c5_idx <= (uint64_T)c5_loopEnd; c5_idx +=
       c5_threadStride) {
    c5_i29 = (int32_T)c5_idx;
    c5_bboxPred_data[c5_i29 + c5_bboxPred_size[0]] =
      ((c5_bboxesX1Y1X2Y2_data[c5_i29 + c5_bboxesX1Y1X2Y2_size[0]] - 0.5) *
       2.1428571428571428 + -0.5714285714285714) + 0.5;
  }
}

static __global__ __launch_bounds__(1024, 1) void c5_eML_blk_kernel_kernel33(
  const real_T c5_bboxesX1Y1X2Y2_data[3136], const int32_T
  c5_bboxesX1Y1X2Y2_size[2], const int32_T c5_bboxPred_size[2], const int32_T
  c5_loop_ub, real_T c5_bboxPred_data[3136])
{
  int64_T c5_loopEnd;
  uint64_T c5_idx;
  uint64_T c5_threadId;
  uint64_T c5_threadStride;
  int32_T c5_i30;
  c5_threadId = mwGetGlobalThreadIndex();
  c5_threadStride = mwGetTotalThreadsLaunched();
  c5_loopEnd = (int64_T)c5_loop_ub;
  for (c5_idx = c5_threadId; c5_idx <= (uint64_T)c5_loopEnd; c5_idx +=
       c5_threadStride) {
    c5_i30 = (int32_T)c5_idx;
    c5_bboxPred_data[c5_i30 + (c5_bboxPred_size[0] << 1)] =
      ((c5_bboxesX1Y1X2Y2_data[c5_i30 + (c5_bboxesX1Y1X2Y2_size[0] << 1)] + 0.5)
       * 3.8125 + -1.40625) - 0.5;
  }
}

static __global__ __launch_bounds__(1024, 1) void c5_eML_blk_kernel_kernel34(
  const real_T c5_bboxesX1Y1X2Y2_data[3136], const int32_T
  c5_bboxesX1Y1X2Y2_size[2], const int32_T c5_bboxPred_size[2], const int32_T
  c5_loop_ub, real_T c5_bboxPred_data[3136])
{
  int64_T c5_loopEnd;
  uint64_T c5_idx;
  uint64_T c5_threadId;
  uint64_T c5_threadStride;
  int32_T c5_i31;
  c5_threadId = mwGetGlobalThreadIndex();
  c5_threadStride = mwGetTotalThreadsLaunched();
  c5_loopEnd = (int64_T)c5_loop_ub;
  for (c5_idx = c5_threadId; c5_idx <= (uint64_T)c5_loopEnd; c5_idx +=
       c5_threadStride) {
    c5_i31 = (int32_T)c5_idx;
    c5_bboxPred_data[c5_i31 + c5_bboxPred_size[0] * 3] =
      ((c5_bboxesX1Y1X2Y2_data[c5_i31 + c5_bboxesX1Y1X2Y2_size[0] * 3] + 0.5) *
       2.1428571428571428 + -0.5714285714285714) - 0.5;
  }
}

static __global__ __launch_bounds__(1024, 1) void c5_eML_blk_kernel_kernel35(
  const int32_T c5_nx, real_T c5_bboxPred_data[3136])
{
  int64_T c5_loopEnd;
  uint64_T c5_idx;
  uint64_T c5_threadId;
  uint64_T c5_threadStride;
  int32_T c5_k;
  c5_threadId = mwGetGlobalThreadIndex();
  c5_threadStride = mwGetTotalThreadsLaunched();
  c5_loopEnd = (int64_T)(c5_nx - 1);
  for (c5_idx = c5_threadId; c5_idx <= (uint64_T)c5_loopEnd; c5_idx +=
       c5_threadStride) {
    c5_k = (int32_T)c5_idx;
    c5_bboxPred_data[c5_k] = floor(c5_bboxPred_data[c5_k]);
  }
}

static __global__ __launch_bounds__(1024, 1) void c5_eML_blk_kernel_kernel36(
  const real_T c5_bboxPred_data[3136], const int32_T c5_bboxPred_size[2], const
  int32_T c5_i32, real_T c5_b_bboxPred_data[784])
{
  int64_T c5_loopEnd;
  uint64_T c5_idx;
  uint64_T c5_threadId;
  uint64_T c5_threadStride;
  int32_T c5_i33;
  c5_threadId = mwGetGlobalThreadIndex();
  c5_threadStride = mwGetTotalThreadsLaunched();
  c5_loopEnd = (int64_T)c5_i32;
  for (c5_idx = c5_threadId; c5_idx <= (uint64_T)c5_loopEnd; c5_idx +=
       c5_threadStride) {
    c5_i33 = (int32_T)c5_idx;
    c5_b_bboxPred_data[c5_i33] = (c5_bboxPred_data[c5_i33 + (c5_bboxPred_size[0]
      << 1)] - c5_bboxPred_data[c5_i33]) + 1.0;
  }
}

static __global__ __launch_bounds__(1024, 1) void c5_eML_blk_kernel_kernel37(
  const real_T c5_bboxPred_data[784], const int32_T c5_bboxPred_size[2], const
  int32_T c5_b_bboxPred_size[1], real_T c5_b_bboxPred_data[3136])
{
  int64_T c5_loopEnd;
  uint64_T c5_idx;
  uint64_T c5_threadId;
  uint64_T c5_threadStride;
  int32_T c5_i34;
  c5_threadId = mwGetGlobalThreadIndex();
  c5_threadStride = mwGetTotalThreadsLaunched();
  c5_loopEnd = (int64_T)(c5_b_bboxPred_size[0] - 1);
  for (c5_idx = c5_threadId; c5_idx <= (uint64_T)c5_loopEnd; c5_idx +=
       c5_threadStride) {
    c5_i34 = (int32_T)c5_idx;
    c5_b_bboxPred_data[c5_i34 + (c5_bboxPred_size[0] << 1)] =
      c5_bboxPred_data[c5_i34];
  }
}

static __global__ __launch_bounds__(1024, 1) void c5_eML_blk_kernel_kernel38(
  const real_T c5_bboxPred_data[3136], const int32_T c5_bboxPred_size[2], const
  int32_T c5_i35, real_T c5_b_bboxPred_data[784])
{
  int64_T c5_loopEnd;
  uint64_T c5_idx;
  uint64_T c5_threadId;
  uint64_T c5_threadStride;
  int32_T c5_i36;
  c5_threadId = mwGetGlobalThreadIndex();
  c5_threadStride = mwGetTotalThreadsLaunched();
  c5_loopEnd = (int64_T)c5_i35;
  for (c5_idx = c5_threadId; c5_idx <= (uint64_T)c5_loopEnd; c5_idx +=
       c5_threadStride) {
    c5_i36 = (int32_T)c5_idx;
    c5_b_bboxPred_data[c5_i36] = (c5_bboxPred_data[c5_i36 + c5_bboxPred_size[0] *
      3] - c5_bboxPred_data[c5_i36 + c5_bboxPred_size[0]]) + 1.0;
  }
}

static __global__ __launch_bounds__(1024, 1) void c5_eML_blk_kernel_kernel39(
  const real_T c5_bboxPred_data[784], const int32_T c5_bboxPred_size[2], const
  int32_T c5_b_bboxPred_size[1], real_T c5_b_bboxPred_data[3136])
{
  int64_T c5_loopEnd;
  uint64_T c5_idx;
  uint64_T c5_threadId;
  uint64_T c5_threadStride;
  int32_T c5_i37;
  c5_threadId = mwGetGlobalThreadIndex();
  c5_threadStride = mwGetTotalThreadsLaunched();
  c5_loopEnd = (int64_T)(c5_b_bboxPred_size[0] - 1);
  for (c5_idx = c5_threadId; c5_idx <= (uint64_T)c5_loopEnd; c5_idx +=
       c5_threadStride) {
    c5_i37 = (int32_T)c5_idx;
    c5_b_bboxPred_data[c5_i37 + c5_bboxPred_size[0] * 3] =
      c5_bboxPred_data[c5_i37];
  }
}

static __global__ __launch_bounds__(32, 1) void c5_eML_blk_kernel_kernel40(const
  real_T c5_bboxPred_data[3136], const int32_T c5_bboxPred_size[2], const
  int32_T c5_i, const int32_T c5_b_bboxPred_size[2], const int32_T c5_count,
  real_T c5_b_bboxPred_data[3136])
{
  int32_T c5_i41;
  c5_i41 = (int32_T)mwGetGlobalThreadIndex();
  if (c5_i41 < 4) {
    c5_b_bboxPred_data[c5_count + c5_b_bboxPred_size[0] * c5_i41] =
      c5_bboxPred_data[c5_i + c5_bboxPred_size[0] * c5_i41];
  }
}

static __global__ __launch_bounds__(32, 1) void c5_eML_blk_kernel_kernel41(const
  real32_T c5_thresholdedPrediction_data[4704], const int32_T
  c5_thresholdedPrediction_size[2], const int32_T c5_i, const real_T c5_count,
  real32_T c5_classPred_data[784], real32_T c5_scorePred_data[784])
{
  int32_T c5_tmpIdx;
  c5_tmpIdx = (int32_T)mwGetGlobalThreadIndex();
  if (c5_tmpIdx < 1) {
    c5_scorePred_data[(int32_T)c5_count - 1] =
      c5_thresholdedPrediction_data[c5_i + (c5_thresholdedPrediction_size[0] <<
      2)];
    c5_classPred_data[(int32_T)c5_count - 1] =
      c5_thresholdedPrediction_data[c5_i + c5_thresholdedPrediction_size[0] * 5];
  }
}

static __global__ __launch_bounds__(1024, 1) void c5_eML_blk_kernel_kernel42(
  const int32_T c5_i38, const int32_T c5_i39, int32_T c5_idx_data[784])
{
  int64_T c5_loopEnd;
  uint64_T c5_idx;
  uint64_T c5_threadId;
  uint64_T c5_threadStride;
  int32_T c5_i40;
  c5_threadId = mwGetGlobalThreadIndex();
  c5_threadStride = mwGetTotalThreadsLaunched();
  c5_loopEnd = (int64_T)(c5_i39 - c5_i38);
  for (c5_idx = c5_threadId; c5_idx <= (uint64_T)c5_loopEnd; c5_idx +=
       c5_threadStride) {
    c5_i40 = (int32_T)c5_idx;
    c5_idx_data[c5_i40] = c5_i38 + c5_i40;
  }
}

static __global__ __launch_bounds__(1024, 1) void c5_eML_blk_kernel_kernel43(
  const int32_T c5_bboxPred_size[2], boolean_T c5_b_data[784])
{
  int64_T c5_loopEnd;
  uint64_T c5_idx;
  uint64_T c5_threadId;
  uint64_T c5_threadStride;
  int32_T c5_i42;
  c5_threadId = mwGetGlobalThreadIndex();
  c5_threadStride = mwGetTotalThreadsLaunched();
  c5_loopEnd = (int64_T)(c5_bboxPred_size[0] - 1);
  for (c5_idx = c5_threadId; c5_idx <= (uint64_T)c5_loopEnd; c5_idx +=
       c5_threadStride) {
    c5_i42 = (int32_T)c5_idx;
    c5_b_data[c5_i42] = false;
  }
}

static __global__ __launch_bounds__(1024, 1) void c5_eML_blk_kernel_kernel44(
  const int32_T c5_b_size[2], const boolean_T c5_b_data[784], int32_T c5_i3,
  int32_T *c5_n)
{
  int64_T c5_loopEnd;
  int32_T c5_tmpRed0;
  uint32_T c5_blockStride;
  uint32_T c5_idx;
  uint32_T c5_m;
  uint32_T c5_mask;
  uint32_T c5_numActiveThreads;
  uint32_T c5_numActiveWarps;
  uint32_T c5_thBlkId;
  uint32_T c5_threadId;
  uint32_T c5_threadStride;
  c5_threadStride = (uint32_T)mwGetTotalThreadsLaunched();
  c5_threadId = (uint32_T)mwGetGlobalThreadIndex();
  c5_thBlkId = (uint32_T)mwGetThreadIndexWithinBlock();
  c5_blockStride = (uint32_T)mwGetThreadsPerBlock();
  c5_tmpRed0 = 0;
  c5_loopEnd = (int64_T)(c5_i3 - 1);
  c5_numActiveThreads = c5_blockStride;
  if (mwIsLastBlock()) {
    c5_m = ((int64_T)(c5_i3 - 1) + 1L) % (int64_T)c5_blockStride;
    if (c5_m > 0U) {
      c5_numActiveThreads = c5_m;
    }
  }

  c5_numActiveWarps = (uint32_T)(c5_numActiveThreads + ((int64_T)warpSize - 1L))
    / warpSize;
  if ((int64_T)c5_threadId <= c5_loopEnd) {
    c5_tmpRed0 = (int32_T)c5_b_data[c5_threadId];
  }

  c5_mask = __ballot_sync(MAX_uint32_T, (int64_T)c5_threadId <= c5_loopEnd);
  for (c5_idx = c5_threadId + c5_threadStride; c5_idx <= (uint32_T)c5_loopEnd;
       c5_idx += c5_threadStride) {
    c5_tmpRed0 += (int32_T)c5_b_data[c5_idx];
  }

  c5_tmpRed0 = c5_c_workGroupReduction(c5_tmpRed0, c5_mask, c5_numActiveWarps);
  if (c5_thBlkId == 0U) {
    atomicAdd(&c5_n[0], c5_tmpRed0);
  }
}

static __device__ int32_T c5_c_threadGroupReduction(int32_T c5_val, uint32_T
  c5_lane, uint32_T c5_mask)
{
  int32_T c5_other;
  uint32_T c5_activeSize;
  uint32_T c5_offset;
  c5_activeSize = __popc(c5_mask);
  c5_offset = (c5_activeSize + 1U) / 2U;
  while (c5_activeSize > 1U) {
    c5_other = c5_b_shflDown1(c5_val, c5_offset, c5_mask);
    if (c5_lane + c5_offset < c5_activeSize) {
      c5_val += c5_other;
    }

    c5_activeSize = c5_offset;
    c5_offset = (c5_offset + 1U) / 2U;
  }

  return c5_val;
}

static __device__ int32_T c5_b_shflDown1(int32_T c5_in1, uint32_T c5_offset,
  uint32_T c5_mask)
{
  c5_in1 = __shfl_down_sync(c5_mask, c5_in1, c5_offset);
  return c5_in1;
}

static __device__ int32_T c5_c_workGroupReduction(int32_T c5_val, uint32_T
  c5_mask, uint32_T c5_numActiveWarps)
{
  __shared__ int32_T c5_shared[32];
  uint32_T c5_lane;
  uint32_T c5_thBlkId;
  uint32_T c5_widx;
  c5_thBlkId = (uint32_T)mwGetThreadIndexWithinBlock();
  c5_lane = c5_thBlkId % warpSize;
  c5_widx = c5_thBlkId / warpSize;
  c5_val = c5_c_threadGroupReduction(c5_val, c5_lane, c5_mask);
  if (c5_lane == 0U) {
    c5_shared[c5_widx] = c5_val;
  }

  __syncthreads();
  c5_mask = __ballot_sync(MAX_uint32_T, c5_lane < c5_numActiveWarps);
  c5_val = c5_shared[c5_lane];
  if (c5_widx == 0U) {
    c5_val = c5_c_threadGroupReduction(c5_val, c5_lane, c5_mask);
  }

  return c5_val;
}

static __global__ __launch_bounds__(32, 1) void c5_eML_blk_kernel_kernel45(const
  int32_T *c5_n, const int32_T c5_bboxPred_size[2], int32_T *c5_nrows)
{
  int32_T c5_tmpIdx;
  c5_tmpIdx = (int32_T)mwGetGlobalThreadIndex();
  if (c5_tmpIdx < 1) {
    *c5_nrows = c5_bboxPred_size[0] - *c5_n;
  }
}

static __global__ __launch_bounds__(32, 1) void c5_eML_blk_kernel_kernel46(const
  int32_T c5_bboxPred_size[2], const int32_T *c5_nrows, const int32_T
  c5_idx_data[784], real_T c5_bboxPred_data[3136])
{
  int32_T c5_b_i;
  int32_T c5_i;
  int32_T c5_i2;
  int32_T c5_j;
  c5_j = (int32_T)mwGetGlobalThreadIndex();
  if (c5_j < 4) {
    c5_i2 = c5_idx_data[0];
    for (c5_i = 0; c5_i <= *c5_nrows - c5_i2; c5_i++) {
      c5_b_i = c5_i2 + c5_i;
      c5_bboxPred_data[(c5_b_i + c5_bboxPred_size[0] * c5_j) - 1] =
        c5_bboxPred_data[c5_b_i + c5_bboxPred_size[0] * c5_j];
    }
  }
}

static __global__ __launch_bounds__(1024, 1) void c5_eML_blk_kernel_kernel47(
  const real_T c5_bboxPred_data[3136], const int32_T c5_bboxPred_size[2], const
  int32_T c5_b_bboxPred_size[2], const int32_T c5_i4, real_T c5_b_bboxPred_data
  [3136])
{
  int64_T c5_loopEnd;
  uint64_T c5_idx;
  uint64_T c5_threadId;
  uint64_T c5_threadStride;
  int32_T c5_i43;
  int32_T c5_i46;
  c5_threadId = mwGetGlobalThreadIndex();
  c5_threadStride = mwGetTotalThreadsLaunched();
  c5_loopEnd = ((int64_T)c5_i4 + 1L) * 4L - 1L;
  for (c5_idx = c5_threadId; c5_idx <= (uint64_T)c5_loopEnd; c5_idx +=
       c5_threadStride) {
    c5_i46 = (int32_T)(c5_idx % ((uint64_T)c5_i4 + 1UL));
    c5_i43 = (int32_T)((c5_idx - (uint64_T)c5_i46) / ((uint64_T)c5_i4 + 1UL));
    c5_b_bboxPred_data[c5_i46 + c5_b_bboxPred_size[0] * c5_i43] =
      c5_bboxPred_data[c5_i46 + c5_bboxPred_size[0] * c5_i43];
  }
}

static __global__ __launch_bounds__(1024, 1) void c5_eML_blk_kernel_kernel48(
  const real_T c5_bboxPred_data[3136], const int32_T c5_bboxPred_size[2], real_T
  c5_b_bboxPred_data[3136])
{
  int64_T c5_loopEnd;
  uint64_T c5_idx;
  uint64_T c5_threadId;
  uint64_T c5_threadStride;
  int32_T c5_i44;
  c5_threadId = mwGetGlobalThreadIndex();
  c5_threadStride = mwGetTotalThreadsLaunched();
  c5_loopEnd = (int64_T)(c5_bboxPred_size[0] * 4 - 1);
  for (c5_idx = c5_threadId; c5_idx <= (uint64_T)c5_loopEnd; c5_idx +=
       c5_threadStride) {
    c5_i44 = (int32_T)c5_idx;
    c5_b_bboxPred_data[c5_i44] = c5_bboxPred_data[c5_i44];
  }
}

static __global__ __launch_bounds__(1024, 1) void c5_eML_blk_kernel_kernel49(
  const int32_T c5_i45, const int32_T c5_i47, int32_T c5_idx_data[784])
{
  int64_T c5_loopEnd;
  uint64_T c5_idx;
  uint64_T c5_threadId;
  uint64_T c5_threadStride;
  int32_T c5_i48;
  c5_threadId = mwGetGlobalThreadIndex();
  c5_threadStride = mwGetTotalThreadsLaunched();
  c5_loopEnd = (int64_T)(c5_i47 - c5_i45);
  for (c5_idx = c5_threadId; c5_idx <= (uint64_T)c5_loopEnd; c5_idx +=
       c5_threadStride) {
    c5_i48 = (int32_T)c5_idx;
    c5_idx_data[c5_i48] = c5_i45 + c5_i48;
  }
}

static __global__ __launch_bounds__(1024, 1) void c5_eML_blk_kernel_kernel50(
  const int32_T c5_scorePred_size[1], boolean_T c5_b_data[784])
{
  int64_T c5_loopEnd;
  uint64_T c5_idx;
  uint64_T c5_threadId;
  uint64_T c5_threadStride;
  int32_T c5_i49;
  c5_threadId = mwGetGlobalThreadIndex();
  c5_threadStride = mwGetTotalThreadsLaunched();
  c5_loopEnd = (int64_T)(c5_scorePred_size[0] - 1);
  for (c5_idx = c5_threadId; c5_idx <= (uint64_T)c5_loopEnd; c5_idx +=
       c5_threadStride) {
    c5_i49 = (int32_T)c5_idx;
    c5_b_data[c5_i49] = false;
  }
}

static __global__ __launch_bounds__(1024, 1) void c5_eML_blk_kernel_kernel51(
  const int32_T c5_b_size[2], const boolean_T c5_b_data[784], int32_T c5_i6,
  int32_T *c5_n)
{
  int64_T c5_loopEnd;
  int32_T c5_tmpRed0;
  uint32_T c5_blockStride;
  uint32_T c5_idx;
  uint32_T c5_m;
  uint32_T c5_mask;
  uint32_T c5_numActiveThreads;
  uint32_T c5_numActiveWarps;
  uint32_T c5_thBlkId;
  uint32_T c5_threadId;
  uint32_T c5_threadStride;
  c5_threadStride = (uint32_T)mwGetTotalThreadsLaunched();
  c5_threadId = (uint32_T)mwGetGlobalThreadIndex();
  c5_thBlkId = (uint32_T)mwGetThreadIndexWithinBlock();
  c5_blockStride = (uint32_T)mwGetThreadsPerBlock();
  c5_tmpRed0 = 0;
  c5_loopEnd = (int64_T)(c5_i6 - 1);
  c5_numActiveThreads = c5_blockStride;
  if (mwIsLastBlock()) {
    c5_m = ((int64_T)(c5_i6 - 1) + 1L) % (int64_T)c5_blockStride;
    if (c5_m > 0U) {
      c5_numActiveThreads = c5_m;
    }
  }

  c5_numActiveWarps = (uint32_T)(c5_numActiveThreads + ((int64_T)warpSize - 1L))
    / warpSize;
  if ((int64_T)c5_threadId <= c5_loopEnd) {
    c5_tmpRed0 = (int32_T)c5_b_data[c5_threadId];
  }

  c5_mask = __ballot_sync(MAX_uint32_T, (int64_T)c5_threadId <= c5_loopEnd);
  for (c5_idx = c5_threadId + c5_threadStride; c5_idx <= (uint32_T)c5_loopEnd;
       c5_idx += c5_threadStride) {
    c5_tmpRed0 += (int32_T)c5_b_data[c5_idx];
  }

  c5_tmpRed0 = c5_c_workGroupReduction(c5_tmpRed0, c5_mask, c5_numActiveWarps);
  if (c5_thBlkId == 0U) {
    atomicAdd(&c5_n[0], c5_tmpRed0);
  }
}

static __global__ __launch_bounds__(1024, 1) void c5_eML_blk_kernel_kernel52(
  const int32_T c5_i50, const int32_T c5_i51, int32_T c5_idx_data[784])
{
  int64_T c5_loopEnd;
  uint64_T c5_idx;
  uint64_T c5_threadId;
  uint64_T c5_threadStride;
  int32_T c5_i52;
  c5_threadId = mwGetGlobalThreadIndex();
  c5_threadStride = mwGetTotalThreadsLaunched();
  c5_loopEnd = (int64_T)(c5_i51 - c5_i50);
  for (c5_idx = c5_threadId; c5_idx <= (uint64_T)c5_loopEnd; c5_idx +=
       c5_threadStride) {
    c5_i52 = (int32_T)c5_idx;
    c5_idx_data[c5_i52] = c5_i50 + c5_i52;
  }
}

static __global__ __launch_bounds__(1024, 1) void c5_eML_blk_kernel_kernel53(
  const int32_T c5_classPred_size[1], boolean_T c5_b_data[784])
{
  int64_T c5_loopEnd;
  uint64_T c5_idx;
  uint64_T c5_threadId;
  uint64_T c5_threadStride;
  int32_T c5_i53;
  c5_threadId = mwGetGlobalThreadIndex();
  c5_threadStride = mwGetTotalThreadsLaunched();
  c5_loopEnd = (int64_T)(c5_classPred_size[0] - 1);
  for (c5_idx = c5_threadId; c5_idx <= (uint64_T)c5_loopEnd; c5_idx +=
       c5_threadStride) {
    c5_i53 = (int32_T)c5_idx;
    c5_b_data[c5_i53] = false;
  }
}

static __global__ __launch_bounds__(1024, 1) void c5_eML_blk_kernel_kernel54(
  const int32_T c5_b_size[2], const boolean_T c5_b_data[784], int32_T c5_i9,
  int32_T *c5_n)
{
  int64_T c5_loopEnd;
  int32_T c5_tmpRed0;
  uint32_T c5_blockStride;
  uint32_T c5_idx;
  uint32_T c5_m;
  uint32_T c5_mask;
  uint32_T c5_numActiveThreads;
  uint32_T c5_numActiveWarps;
  uint32_T c5_thBlkId;
  uint32_T c5_threadId;
  uint32_T c5_threadStride;
  c5_threadStride = (uint32_T)mwGetTotalThreadsLaunched();
  c5_threadId = (uint32_T)mwGetGlobalThreadIndex();
  c5_thBlkId = (uint32_T)mwGetThreadIndexWithinBlock();
  c5_blockStride = (uint32_T)mwGetThreadsPerBlock();
  c5_tmpRed0 = 0;
  c5_loopEnd = (int64_T)(c5_i9 - 1);
  c5_numActiveThreads = c5_blockStride;
  if (mwIsLastBlock()) {
    c5_m = ((int64_T)(c5_i9 - 1) + 1L) % (int64_T)c5_blockStride;
    if (c5_m > 0U) {
      c5_numActiveThreads = c5_m;
    }
  }

  c5_numActiveWarps = (uint32_T)(c5_numActiveThreads + ((int64_T)warpSize - 1L))
    / warpSize;
  if ((int64_T)c5_threadId <= c5_loopEnd) {
    c5_tmpRed0 = (int32_T)c5_b_data[c5_threadId];
  }

  c5_mask = __ballot_sync(MAX_uint32_T, (int64_T)c5_threadId <= c5_loopEnd);
  for (c5_idx = c5_threadId + c5_threadStride; c5_idx <= (uint32_T)c5_loopEnd;
       c5_idx += c5_threadStride) {
    c5_tmpRed0 += (int32_T)c5_b_data[c5_idx];
  }

  c5_tmpRed0 = c5_c_workGroupReduction(c5_tmpRed0, c5_mask, c5_numActiveWarps);
  if (c5_thBlkId == 0U) {
    atomicAdd(&c5_n[0], c5_tmpRed0);
  }
}

static __global__ __launch_bounds__(1024, 1) void c5_eML_blk_kernel_kernel55(
  const real32_T c5_scorePred_data[784], const int32_T c5_scorePred_size[1],
  real32_T c5_out_data[784])
{
  int64_T c5_loopEnd;
  uint64_T c5_idx;
  uint64_T c5_threadId;
  uint64_T c5_threadStride;
  int32_T c5_i58;
  c5_threadId = mwGetGlobalThreadIndex();
  c5_threadStride = mwGetTotalThreadsLaunched();
  c5_loopEnd = (int64_T)(c5_scorePred_size[0] - 1);
  for (c5_idx = c5_threadId; c5_idx <= (uint64_T)c5_loopEnd; c5_idx +=
       c5_threadStride) {
    c5_i58 = (int32_T)c5_idx;
    c5_out_data[c5_i58] = c5_scorePred_data[c5_i58];
  }
}

static __global__ __launch_bounds__(1024, 1) void c5_eML_blk_kernel_kernel56(
  const uint32_T c5_dv2[2], real_T c5_x1_data[784])
{
  int64_T c5_loopEnd;
  uint64_T c5_idx;
  uint64_T c5_threadId;
  uint64_T c5_threadStride;
  int32_T c5_i62;
  c5_threadId = mwGetGlobalThreadIndex();
  c5_threadStride = mwGetTotalThreadsLaunched();
  c5_loopEnd = (int64_T)((int32_T)c5_dv2[0] - 1);
  for (c5_idx = c5_threadId; c5_idx <= (uint64_T)c5_loopEnd; c5_idx +=
       c5_threadStride) {
    c5_i62 = (int32_T)c5_idx;
    c5_x1_data[c5_i62] = 0.0;
  }
}

static __global__ __launch_bounds__(1024, 1) void c5_eML_blk_kernel_kernel57(
  const real_T c5_bboxPred_data[3136], const int32_T c5_bboxPred_size[2], const
  real_T c5_x1_data[784], const int32_T c5_bboxesX1Y1X2Y2_size[2], const int32_T
  c5_x1_size[1], real_T c5_bboxesX1Y1X2Y2_data[3136])
{
  int64_T c5_loopEnd;
  uint64_T c5_idx;
  uint64_T c5_threadId;
  uint64_T c5_threadStride;
  uint64_T c5_tmpIndex;
  int32_T c5_i56;
  int32_T c5_i59;
  c5_threadId = mwGetGlobalThreadIndex();
  c5_threadStride = mwGetTotalThreadsLaunched();
  c5_loopEnd = ((int64_T)(c5_x1_size[0] - 1) + 1L) * 4L - 1L;
  for (c5_idx = c5_threadId; c5_idx <= (uint64_T)c5_loopEnd; c5_idx +=
       c5_threadStride) {
    c5_i59 = (int32_T)(c5_idx % ((uint64_T)(c5_x1_size[0] - 1) + 1UL));
    c5_tmpIndex = (c5_idx - (uint64_T)c5_i59) / ((uint64_T)(c5_x1_size[0] - 1) +
      1UL);
    c5_i56 = (int32_T)c5_tmpIndex;
    c5_bboxesX1Y1X2Y2_data[c5_i59 + c5_bboxesX1Y1X2Y2_size[0] * c5_i56] =
      c5_bboxPred_data[((int32_T)c5_x1_data[c5_i59] + c5_bboxPred_size[0] *
                        c5_i56) - 1];
  }
}

static __global__ __launch_bounds__(1024, 1) void c5_eML_blk_kernel_kernel58(
  const real32_T c5_classPred_data[784], const real_T c5_x1_data[784], const
  int32_T c5_x1_size[1], real_T c5_y1_data[784])
{
  int64_T c5_loopEnd;
  uint64_T c5_idx;
  uint64_T c5_threadId;
  uint64_T c5_threadStride;
  int32_T c5_i57;
  c5_threadId = mwGetGlobalThreadIndex();
  c5_threadStride = mwGetTotalThreadsLaunched();
  c5_loopEnd = (int64_T)(c5_x1_size[0] - 1);
  for (c5_idx = c5_threadId; c5_idx <= (uint64_T)c5_loopEnd; c5_idx +=
       c5_threadStride) {
    c5_i57 = (int32_T)c5_idx;
    c5_y1_data[c5_i57] = (real_T)c5_classPred_data[(int32_T)c5_x1_data[c5_i57] -
      1];
  }
}

static __global__ __launch_bounds__(1024, 1) void c5_eML_blk_kernel_kernel59(
  const int32_T c5_x1_size[1], boolean_T c5_selectedIndex_data[784])
{
  int64_T c5_loopEnd;
  uint64_T c5_idx;
  uint64_T c5_threadId;
  uint64_T c5_threadStride;
  int32_T c5_i60;
  c5_threadId = mwGetGlobalThreadIndex();
  c5_threadStride = mwGetTotalThreadsLaunched();
  c5_loopEnd = (int64_T)(c5_x1_size[0] - 1);
  for (c5_idx = c5_threadId; c5_idx <= (uint64_T)c5_loopEnd; c5_idx +=
       c5_threadStride) {
    c5_i60 = (int32_T)c5_idx;
    c5_selectedIndex_data[c5_i60] = true;
  }
}

static __global__ __launch_bounds__(1024, 1) void c5_eML_blk_kernel_kernel60(
  const real_T c5_bboxesX1Y1X2Y2_data[3136], const int32_T
  c5_bboxesX1Y1X2Y2_size[2], const int32_T c5_i61, real_T c5_area_data[784])
{
  int64_T c5_loopEnd;
  uint64_T c5_idx;
  uint64_T c5_threadId;
  uint64_T c5_threadStride;
  int32_T c5_i63;
  c5_threadId = mwGetGlobalThreadIndex();
  c5_threadStride = mwGetTotalThreadsLaunched();
  c5_loopEnd = (int64_T)c5_i61;
  for (c5_idx = c5_threadId; c5_idx <= (uint64_T)c5_loopEnd; c5_idx +=
       c5_threadStride) {
    c5_i63 = (int32_T)c5_idx;
    c5_area_data[c5_i63] = c5_bboxesX1Y1X2Y2_data[c5_i63 +
      (c5_bboxesX1Y1X2Y2_size[0] << 1)] * c5_bboxesX1Y1X2Y2_data[c5_i63 +
      c5_bboxesX1Y1X2Y2_size[0] * 3];
  }
}

static __global__ __launch_bounds__(1024, 1) void c5_eML_blk_kernel_kernel61(
  const int32_T c5_bboxesX1Y1X2Y2_size[2], const real_T c5_bboxesX1Y1X2Y2_data
  [3136], const int32_T c5_i64, real_T c5_x2_data[784])
{
  int64_T c5_loopEnd;
  uint64_T c5_idx;
  uint64_T c5_threadId;
  uint64_T c5_threadStride;
  int32_T c5_i65;
  c5_threadId = mwGetGlobalThreadIndex();
  c5_threadStride = mwGetTotalThreadsLaunched();
  c5_loopEnd = (int64_T)c5_i64;
  for (c5_idx = c5_threadId; c5_idx <= (uint64_T)c5_loopEnd; c5_idx +=
       c5_threadStride) {
    c5_i65 = (int32_T)c5_idx;
    c5_x2_data[c5_i65] = c5_bboxesX1Y1X2Y2_data[c5_i65] +
      c5_bboxesX1Y1X2Y2_data[c5_i65 + (c5_bboxesX1Y1X2Y2_size[0] << 1)];
  }
}

static __global__ __launch_bounds__(1024, 1) void c5_eML_blk_kernel_kernel62(
  const real_T c5_bboxesX1Y1X2Y2_data[3136], const int32_T
  c5_bboxesX1Y1X2Y2_size[2], const int32_T c5_i66, real_T c5_y2_data[784])
{
  int64_T c5_loopEnd;
  uint64_T c5_idx;
  uint64_T c5_threadId;
  uint64_T c5_threadStride;
  int32_T c5_i67;
  c5_threadId = mwGetGlobalThreadIndex();
  c5_threadStride = mwGetTotalThreadsLaunched();
  c5_loopEnd = (int64_T)c5_i66;
  for (c5_idx = c5_threadId; c5_idx <= (uint64_T)c5_loopEnd; c5_idx +=
       c5_threadStride) {
    c5_i67 = (int32_T)c5_idx;
    c5_y2_data[c5_i67] = c5_bboxesX1Y1X2Y2_data[c5_i67 + c5_bboxesX1Y1X2Y2_size
      [0]] + c5_bboxesX1Y1X2Y2_data[c5_i67 + c5_bboxesX1Y1X2Y2_size[0] * 3];
  }
}

static __global__ __launch_bounds__(1024, 1) void c5_eML_blk_kernel_kernel63(
  const int32_T c5_i12, const int32_T c5_iv[2], boolean_T c5_selectedIndex_data
  [784])
{
  int64_T c5_loopEnd;
  uint64_T c5_idx;
  uint64_T c5_threadId;
  uint64_T c5_threadStride;
  int32_T c5_i68;
  c5_threadId = mwGetGlobalThreadIndex();
  c5_threadStride = mwGetTotalThreadsLaunched();
  c5_loopEnd = (int64_T)(c5_iv[1] - 1);
  for (c5_idx = c5_threadId; c5_idx <= (uint64_T)c5_loopEnd; c5_idx +=
       c5_threadStride) {
    c5_i68 = (int32_T)c5_idx;
    c5_selectedIndex_data[c5_i12 + c5_i68] = false;
  }
}

static __global__ __launch_bounds__(1024, 1) void c5_eML_blk_kernel_kernel64(
  const boolean_T c5_selectedIndex_data[784], const real_T c5_x1_data[784],
  const int32_T c5_selectedIndex_size[1], boolean_T c5_index_data[784])
{
  int64_T c5_loopEnd;
  uint64_T c5_idx;
  uint64_T c5_threadId;
  uint64_T c5_threadStride;
  int32_T c5_i69;
  c5_threadId = mwGetGlobalThreadIndex();
  c5_threadStride = mwGetTotalThreadsLaunched();
  c5_loopEnd = (int64_T)(c5_selectedIndex_size[0] - 1);
  for (c5_idx = c5_threadId; c5_idx <= (uint64_T)c5_loopEnd; c5_idx +=
       c5_threadStride) {
    c5_i69 = (int32_T)c5_idx;
    c5_index_data[(int32_T)c5_x1_data[c5_i69] - 1] =
      c5_selectedIndex_data[c5_i69];
  }
}

static __global__ __launch_bounds__(1024, 1) void c5_eML_blk_kernel_kernel65(
  const real_T c5_bboxPred_data[3136], const int32_T c5_bboxPred_size[2], const
  int16_T c5_iv1_data[784], const int32_T c5_bboxesX1Y1X2Y2_size[2], const
  int32_T c5_iv1_size[1], real_T c5_bboxesX1Y1X2Y2_data[3136])
{
  int64_T c5_loopEnd;
  uint64_T c5_idx;
  uint64_T c5_threadId;
  uint64_T c5_threadStride;
  uint64_T c5_tmpIndex;
  int32_T c5_i70;
  int32_T c5_i71;
  c5_threadId = mwGetGlobalThreadIndex();
  c5_threadStride = mwGetTotalThreadsLaunched();
  c5_loopEnd = ((int64_T)(c5_iv1_size[0] - 1) + 1L) * 4L - 1L;
  for (c5_idx = c5_threadId; c5_idx <= (uint64_T)c5_loopEnd; c5_idx +=
       c5_threadStride) {
    c5_i71 = (int32_T)(c5_idx % ((uint64_T)(c5_iv1_size[0] - 1) + 1UL));
    c5_tmpIndex = (c5_idx - (uint64_T)c5_i71) / ((uint64_T)(c5_iv1_size[0] - 1)
      + 1UL);
    c5_i70 = (int32_T)c5_tmpIndex;
    c5_bboxesX1Y1X2Y2_data[c5_i71 + c5_bboxesX1Y1X2Y2_size[0] * c5_i70] =
      c5_bboxPred_data[((int32_T)c5_iv1_data[c5_i71] + c5_bboxPred_size[0] *
                        c5_i70) - 1];
  }
}

static __global__ __launch_bounds__(1024, 1) void c5_eML_blk_kernel_kernel66(
  const real32_T c5_scorePred_data[784], const int32_T c5_scorePred_size[1],
  real32_T c5_b_scores_data[784])
{
  int64_T c5_loopEnd;
  uint64_T c5_idx;
  uint64_T c5_threadId;
  uint64_T c5_threadStride;
  int32_T c5_i55;
  c5_threadId = mwGetGlobalThreadIndex();
  c5_threadStride = mwGetTotalThreadsLaunched();
  c5_loopEnd = (int64_T)(c5_scorePred_size[0] - 1);
  for (c5_idx = c5_threadId; c5_idx <= (uint64_T)c5_loopEnd; c5_idx +=
       c5_threadStride) {
    c5_i55 = (int32_T)c5_idx;
    c5_b_scores_data[c5_i55] = c5_scorePred_data[c5_i55];
  }
}

static __global__ __launch_bounds__(1024, 1) void c5_eML_blk_kernel_kernel67(
  const real_T c5_bboxesX1Y1X2Y2_data[3136], const int32_T
  c5_bboxesX1Y1X2Y2_size[2], real_T c5_b_bboxes_data[])
{
  int64_T c5_loopEnd;
  uint64_T c5_idx;
  uint64_T c5_threadId;
  uint64_T c5_threadStride;
  int32_T c5_i11;
  c5_threadId = mwGetGlobalThreadIndex();
  c5_threadStride = mwGetTotalThreadsLaunched();
  c5_loopEnd = (int64_T)(c5_bboxesX1Y1X2Y2_size[0] * 4 - 1);
  for (c5_idx = c5_threadId; c5_idx <= (uint64_T)c5_loopEnd; c5_idx +=
       c5_threadStride) {
    c5_i11 = (int32_T)c5_idx;
    c5_b_bboxes_data[c5_i11] = c5_bboxesX1Y1X2Y2_data[c5_i11];
  }
}

static __global__ __launch_bounds__(1024, 1) void c5_eML_blk_kernel_kernel68(
  const real32_T c5_b_scores_data[784], const int32_T c5_scores_size[1],
  real32_T c5_c_scores_data[])
{
  int64_T c5_loopEnd;
  uint64_T c5_idx;
  uint64_T c5_threadId;
  uint64_T c5_threadStride;
  int32_T c5_i15;
  c5_threadId = mwGetGlobalThreadIndex();
  c5_threadStride = mwGetTotalThreadsLaunched();
  c5_loopEnd = (int64_T)(c5_scores_size[0] - 1);
  for (c5_idx = c5_threadId; c5_idx <= (uint64_T)c5_loopEnd; c5_idx +=
       c5_threadStride) {
    c5_i15 = (int32_T)c5_idx;
    c5_c_scores_data[c5_i15] = c5_b_scores_data[c5_i15];
  }
}

static __global__ __launch_bounds__(512, 1) void
  c5_DeepLearningNetwork_activations_kernel69(const real32_T c5_varargin_1
  [150528], c5_cell_wrap_18 *c5_r)
{
  int32_T c5_i;
  c5_i = (int32_T)mwGetGlobalThreadIndex();
  if (c5_i < 150528) {
    c5_r->f1[c5_i] = c5_varargin_1[c5_i];
  }
}

static __global__ __launch_bounds__(512, 1) void
  c5_DeepLearningNetwork_activations_kernel70(const c5_cell_wrap_18 *c5_r,
  c5_cell_wrap_18 c5_miniBatchT[1])
{
  uint64_T c5_threadId;
  uint64_T c5_tmpIndex;
  int32_T c5_i1;
  int32_T c5_i2;
  int32_T c5_p;
  c5_threadId = mwGetGlobalThreadIndex();
  c5_i1 = (int32_T)(c5_threadId % 224UL);
  c5_tmpIndex = (c5_threadId - (uint64_T)c5_i1) / 224UL;
  c5_i2 = (int32_T)(c5_tmpIndex % 224UL);
  c5_tmpIndex = (c5_tmpIndex - (uint64_T)c5_i2) / 224UL;
  c5_p = (int32_T)c5_tmpIndex;
  if ((c5_p < 3) && (c5_i2 < 224) && (c5_i1 < 224)) {
    c5_miniBatchT[0].f1[(c5_i1 + 224 * c5_i2) + 50176 * c5_p] = c5_r->f1[(c5_i2
      + 224 * c5_i1) + 50176 * c5_p];
  }
}

static __global__ __launch_bounds__(512, 1) void
  c5_DeepLearningNetwork_activations_kernel71(const real32_T c5_outMiniBatch
  [4704], real32_T c5_b_out[4704])
{
  uint64_T c5_threadId;
  uint64_T c5_tmpIndex;
  int32_T c5_i3;
  int32_T c5_i4;
  int32_T c5_p;
  c5_threadId = mwGetGlobalThreadIndex();
  c5_i3 = (int32_T)(c5_threadId % 14UL);
  c5_tmpIndex = (c5_threadId - (uint64_T)c5_i3) / 14UL;
  c5_i4 = (int32_T)(c5_tmpIndex % 14UL);
  c5_tmpIndex = (c5_tmpIndex - (uint64_T)c5_i4) / 14UL;
  c5_p = (int32_T)c5_tmpIndex;
  if ((c5_p < 24) && (c5_i4 < 14) && (c5_i3 < 14)) {
    c5_b_out[(c5_i3 + 14 * c5_i4) + 196 * c5_p] = c5_outMiniBatch[(c5_i4 + 14 *
      c5_i3) + 196 * c5_p];
  }
}

static __device__ real32_T c5_callFcn_device(real32_T c5_input1, real32_T
  c5_input2)
{
  return fmaxf(c5_input1, c5_input2);
}

static __device__ real32_T c5_b_callFcn_device(real32_T c5_input1, real32_T
  c5_input2)
{
  return fminf(c5_input1, c5_input2);
}

void c5_yolov2ResNet50VehicleExample0_LaneDetection0::allocate()
{
  int32_T c5_idx;
  this->targetImpl->allocate(802816, 4);
  for (c5_idx = 0; c5_idx < 57; c5_idx++) {
    this->layers[c5_idx]->allocate();
  }

  (static_cast<MWTensor<real32_T> *>(this->inputTensors[0]))->setData
    (this->layers[0]->getLayerOutput(0));
}

void c5_yolov2ResNet50VehicleExample0_LaneDetection0::postsetup()
{
  this->targetImpl->postSetup(this->layers, this->numLayers);
}

c5_yolov2ResNet50VehicleExample0_LaneDetection0::
  c5_yolov2ResNet50VehicleExample0_LaneDetection0()
{
  this->numLayers = 57;
  this->isInitialized = false;
  this->targetImpl = 0;
  this->layers[0] = new MWInputLayer;
  this->layers[0]->setName("input_1");
  this->layers[1] = new MWElementwiseAffineLayer;
  this->layers[1]->setName("input_1_normalization");
  this->layers[1]->setInPlaceIndex(0, 0);
  this->layers[2] = new MWFusedConvReLULayer;
  this->layers[2]->setName("conv1_activation_1_relu");
  this->layers[3] = new MWMaxPoolingLayer;
  this->layers[3]->setName("max_pooling2d_1");
  this->layers[4] = new MWFusedConvReLULayer;
  this->layers[4]->setName("res2a_branch2a_activation_2_relu");
  this->layers[5] = new MWFusedConvReLULayer;
  this->layers[5]->setName("res2a_branch2b_activation_3_relu");
  this->layers[6] = new MWConvLayer;
  this->layers[6]->setName("res2a_branch1");
  this->layers[7] = new MWFusedConvReLULayer;
  this->layers[7]->setName("res2a_branch2c_activation_4_relu");
  this->layers[7]->setInPlaceIndex(0, 1);
  this->layers[8] = new MWFusedConvReLULayer;
  this->layers[8]->setName("res2b_branch2a_activation_5_relu");
  this->layers[9] = new MWFusedConvReLULayer;
  this->layers[9]->setName("res2b_branch2b_activation_6_relu");
  this->layers[10] = new MWFusedConvReLULayer;
  this->layers[10]->setName("res2b_branch2c_activation_7_relu");
  this->layers[10]->setInPlaceIndex(0, 1);
  this->layers[11] = new MWFusedConvReLULayer;
  this->layers[11]->setName("res2c_branch2a_activation_8_relu");
  this->layers[12] = new MWFusedConvReLULayer;
  this->layers[12]->setName("res2c_branch2b_activation_9_relu");
  this->layers[13] = new MWMaxPoolingLayer;
  this->layers[13]->setName("downsample_add_3");
  this->layers[14] = new MWFusedConvReLULayer;
  this->layers[14]->setName("res2c_branch2c_activation_10_relu");
  this->layers[14]->setInPlaceIndex(0, 1);
  this->layers[15] = new MWFusedConvReLULayer;
  this->layers[15]->setName("res3a_branch2a_activation_11_relu");
  this->layers[16] = new MWFusedConvReLULayer;
  this->layers[16]->setName("res3a_branch2b_activation_12_relu");
  this->layers[17] = new MWConvLayer;
  this->layers[17]->setName("res3a_branch1");
  this->layers[18] = new MWFusedConvReLULayer;
  this->layers[18]->setName("res3a_branch2c_activation_13_relu");
  this->layers[18]->setInPlaceIndex(0, 1);
  this->layers[19] = new MWFusedConvReLULayer;
  this->layers[19]->setName("res3b_branch2a_activation_14_relu");
  this->layers[20] = new MWFusedConvReLULayer;
  this->layers[20]->setName("res3b_branch2b_activation_15_relu");
  this->layers[21] = new MWFusedConvReLULayer;
  this->layers[21]->setName("res3b_branch2c_activation_16_relu");
  this->layers[21]->setInPlaceIndex(0, 1);
  this->layers[22] = new MWFusedConvReLULayer;
  this->layers[22]->setName("res3c_branch2a_activation_17_relu");
  this->layers[23] = new MWFusedConvReLULayer;
  this->layers[23]->setName("res3c_branch2b_activation_18_relu");
  this->layers[24] = new MWFusedConvReLULayer;
  this->layers[24]->setName("res3c_branch2c_activation_19_relu");
  this->layers[24]->setInPlaceIndex(0, 1);
  this->layers[25] = new MWFusedConvReLULayer;
  this->layers[25]->setName("res3d_branch2a_activation_20_relu");
  this->layers[26] = new MWFusedConvReLULayer;
  this->layers[26]->setName("res3d_branch2b_activation_21_relu");
  this->layers[27] = new MWMaxPoolingLayer;
  this->layers[27]->setName("downsample_add_7");
  this->layers[28] = new MWFusedConvReLULayer;
  this->layers[28]->setName("res3d_branch2c_activation_22_relu");
  this->layers[28]->setInPlaceIndex(0, 1);
  this->layers[29] = new MWFusedConvReLULayer;
  this->layers[29]->setName("res4a_branch2a_activation_23_relu");
  this->layers[30] = new MWFusedConvReLULayer;
  this->layers[30]->setName("res4a_branch2b_activation_24_relu");
  this->layers[31] = new MWConvLayer;
  this->layers[31]->setName("res4a_branch1");
  this->layers[32] = new MWFusedConvReLULayer;
  this->layers[32]->setName("res4a_branch2c_activation_25_relu");
  this->layers[32]->setInPlaceIndex(0, 1);
  this->layers[33] = new MWFusedConvReLULayer;
  this->layers[33]->setName("res4b_branch2a_activation_26_relu");
  this->layers[34] = new MWFusedConvReLULayer;
  this->layers[34]->setName("res4b_branch2b_activation_27_relu");
  this->layers[35] = new MWFusedConvReLULayer;
  this->layers[35]->setName("res4b_branch2c_activation_28_relu");
  this->layers[35]->setInPlaceIndex(0, 1);
  this->layers[36] = new MWFusedConvReLULayer;
  this->layers[36]->setName("res4c_branch2a_activation_29_relu");
  this->layers[37] = new MWFusedConvReLULayer;
  this->layers[37]->setName("res4c_branch2b_activation_30_relu");
  this->layers[38] = new MWFusedConvReLULayer;
  this->layers[38]->setName("res4c_branch2c_activation_31_relu");
  this->layers[38]->setInPlaceIndex(0, 1);
  this->layers[39] = new MWFusedConvReLULayer;
  this->layers[39]->setName("res4d_branch2a_activation_32_relu");
  this->layers[40] = new MWFusedConvReLULayer;
  this->layers[40]->setName("res4d_branch2b_activation_33_relu");
  this->layers[41] = new MWFusedConvReLULayer;
  this->layers[41]->setName("res4d_branch2c_activation_34_relu");
  this->layers[41]->setInPlaceIndex(0, 1);
  this->layers[42] = new MWFusedConvReLULayer;
  this->layers[42]->setName("res4e_branch2a_activation_35_relu");
  this->layers[43] = new MWFusedConvReLULayer;
  this->layers[43]->setName("res4e_branch2b_activation_36_relu");
  this->layers[44] = new MWFusedConvReLULayer;
  this->layers[44]->setName("res4e_branch2c_activation_37_relu");
  this->layers[44]->setInPlaceIndex(0, 1);
  this->layers[45] = new MWFusedConvReLULayer;
  this->layers[45]->setName("res4f_branch2a_activation_38_relu");
  this->layers[46] = new MWFusedConvReLULayer;
  this->layers[46]->setName("res4f_branch2b_activation_39_relu");
  this->layers[47] = new MWFusedConvReLULayer;
  this->layers[47]->setName("res4f_branch2c_activation_40_relu");
  this->layers[47]->setInPlaceIndex(0, 1);
  this->layers[48] = new MWFusedConvReLULayer;
  this->layers[48]->setName("yolov2Conv1_yolov2Relu1");
  this->layers[49] = new MWFusedConvReLULayer;
  this->layers[49]->setName("yolov2Conv2_yolov2Relu2");
  this->layers[50] = new MWConvLayer;
  this->layers[50]->setName("yolov2ClassConv");
  this->layers[51] = new MWYoloExtractionLayer;
  this->layers[51]->setName("YOLOv2ExtractionLayer");
  this->layers[52] = new MWSigmoidLayer;
  this->layers[52]->setName("YOLOSigmoidLayer");
  this->layers[53] = new MWExponentialLayer;
  this->layers[53]->setName("YOLOv2ExponentialLayer");
  this->layers[54] = new MWYoloSoftmaxLayer;
  this->layers[54]->setName("YOLOv2SoftmaxLayer");
  this->layers[55] = new MWConcatenationLayer;
  this->layers[55]->setName("YOLOv2ConcatenationLayer");
  this->layers[56] = new MWOutputLayer;
  this->layers[56]->setName("yolov2OutputLayer");
  this->layers[56]->setInPlaceIndex(0, 0);
  this->targetImpl = new MWTargetNetworkImpl;
  this->inputTensors[0] = new MWTensor<real32_T>;
  this->inputTensors[0]->setHeight(224);
  this->inputTensors[0]->setWidth(224);
  this->inputTensors[0]->setChannels(3);
  this->inputTensors[0]->setBatchSize(1);
  this->inputTensors[0]->setSequenceLength(1);
}

void c5_yolov2ResNet50VehicleExample0_LaneDetection0::deallocate()
{
  int32_T c5_idx;
  this->targetImpl->deallocate();
  for (c5_idx = 0; c5_idx < 57; c5_idx++) {
    this->layers[c5_idx]->deallocate();
  }
}

void c5_yolov2ResNet50VehicleExample0_LaneDetection0::setSize()
{
  int32_T c5_idx;
  for (c5_idx = 0; c5_idx < 57; c5_idx++) {
    this->layers[c5_idx]->propagateSize();
  }

  this->allocate();
  this->postsetup();
}

void c5_yolov2ResNet50VehicleExample0_LaneDetection0::resetState()
{
}

void c5_yolov2ResNet50VehicleExample0_LaneDetection0::setup()
{
  if (this->isInitialized) {
    this->resetState();
  } else {
    this->isInitialized = true;
    this->targetImpl->preSetup();
    this->targetImpl->setAutoTune(true);
    (static_cast<MWInputLayer *>(this->layers[0]))->createInputLayer
      (this->targetImpl, this->inputTensors[0], 224, 224, 3, 0, "", 0);
    (static_cast<MWElementwiseAffineLayer *>(this->layers[1]))
      ->createElementwiseAffineLayer(this->targetImpl, this->layers[0]
      ->getOutputTensor(0), 1, 1, 3, 1, 1, 3, false, 1, 1,
      "/home/hmcl/ADcontrol/slprj/_sfprj/LaneDetection/_self/sfun/src/cnn_yolov2ResNet50VehicleExample0_LaneDetection0_input_1_scale.bi"
      "n",
      "/home/hmcl/ADcontrol/slprj/_sfprj/LaneDetection/_self/sfun/src/cnn_yolov2ResNet50VehicleExample0_LaneDetection0_input_1_offset.b"
      "in", 0);
    (static_cast<MWFusedConvReLULayer *>(this->layers[2]))
      ->createFusedConvReLULayer(this->targetImpl, 1, this->layers[1]
      ->getOutputTensor(0), 7, 7, 3, 64, 2, 2, 3, 3, 3, 3, 1, 1, 1,
      "/home/hmcl/ADcontrol/slprj/_sfprj/LaneDetection/_self/sfun/src/cnn_yolov2ResNet50VehicleExample0_LaneDetection0_conv1_w.bin",
      "/home/hmcl/ADcontrol/slprj/_sfprj/LaneDetection/_self/sfun/src/cnn_yolov2ResNet50VehicleExample0_LaneDetection0_conv1_b.bin",
      1);
    (static_cast<MWMaxPoolingLayer *>(this->layers[3]))->createMaxPoolingLayer
      (this->targetImpl, this->layers[2]->getOutputTensor(0), 3, 3, 2, 2, 0, 0,
       0, 0, 0, 1, 0);
    (static_cast<MWFusedConvReLULayer *>(this->layers[4]))
      ->createFusedConvReLULayer(this->targetImpl, 1, this->layers[3]
      ->getOutputTensor(0), 1, 1, 64, 64, 1, 1, 0, 0, 0, 0, 1, 1, 1,
      "/home/hmcl/ADcontrol/slprj/_sfprj/LaneDetection/_self/sfun/src/cnn_yolov2ResNet50VehicleExample0_LaneDetection0_res2a_branch2a_w"
      ".bin",
      "/home/hmcl/ADcontrol/slprj/_sfprj/LaneDetection/_self/sfun/src/cnn_yolov2ResNet50VehicleExample0_LaneDetection0_res2a_branch2a_b"
      ".bin", 1);
    (static_cast<MWFusedConvReLULayer *>(this->layers[5]))
      ->createFusedConvReLULayer(this->targetImpl, 1, this->layers[4]
      ->getOutputTensor(0), 3, 3, 64, 64, 1, 1, 1, 1, 1, 1, 1, 1, 1,
      "/home/hmcl/ADcontrol/slprj/_sfprj/LaneDetection/_self/sfun/src/cnn_yolov2ResNet50VehicleExample0_LaneDetection0_res2a_branch2b_w"
      ".bin",
      "/home/hmcl/ADcontrol/slprj/_sfprj/LaneDetection/_self/sfun/src/cnn_yolov2ResNet50VehicleExample0_LaneDetection0_res2a_branch2b_b"
      ".bin", 2);
    (static_cast<MWConvLayer *>(this->layers[6]))->createConvLayer
      (this->targetImpl, this->layers[3]->getOutputTensor(0), 1, 1, 64, 256, 1,
       1, 0, 0, 0, 0, 1, 1, 1,
       "/home/hmcl/ADcontrol/slprj/_sfprj/LaneDetection/_self/sfun/src/cnn_yolov2ResNet50VehicleExample0_LaneDetection0_res2a_branch1_w."
       "bin",
       "/home/hmcl/ADcontrol/slprj/_sfprj/LaneDetection/_self/sfun/src/cnn_yolov2ResNet50VehicleExample0_LaneDetection0_res2a_branch1_b."
       "bin", 1);
    (static_cast<MWFusedConvReLULayer *>(this->layers[7]))
      ->createFusedConvReLULayer(this->targetImpl, 2, this->layers[5]
      ->getOutputTensor(0), this->layers[6]->getOutputTensor(0), 1, 1, 64, 256,
      1, 1, 0, 0, 0, 0, 1, 1, 1,
      "/home/hmcl/ADcontrol/slprj/_sfprj/LaneDetection/_self/sfun/src/cnn_yolov2ResNet50VehicleExample0_LaneDetection0_res2a_branch2c_w"
      ".bin",
      "/home/hmcl/ADcontrol/slprj/_sfprj/LaneDetection/_self/sfun/src/cnn_yolov2ResNet50VehicleExample0_LaneDetection0_res2a_branch2c_b"
      ".bin", 1);
    (static_cast<MWFusedConvReLULayer *>(this->layers[8]))
      ->createFusedConvReLULayer(this->targetImpl, 1, this->layers[7]
      ->getOutputTensor(0), 1, 1, 256, 64, 1, 1, 0, 0, 0, 0, 1, 1, 1,
      "/home/hmcl/ADcontrol/slprj/_sfprj/LaneDetection/_self/sfun/src/cnn_yolov2ResNet50VehicleExample0_LaneDetection0_res2b_branch2a_w"
      ".bin",
      "/home/hmcl/ADcontrol/slprj/_sfprj/LaneDetection/_self/sfun/src/cnn_yolov2ResNet50VehicleExample0_LaneDetection0_res2b_branch2a_b"
      ".bin", 0);
    (static_cast<MWFusedConvReLULayer *>(this->layers[9]))
      ->createFusedConvReLULayer(this->targetImpl, 1, this->layers[8]
      ->getOutputTensor(0), 3, 3, 64, 64, 1, 1, 1, 1, 1, 1, 1, 1, 1,
      "/home/hmcl/ADcontrol/slprj/_sfprj/LaneDetection/_self/sfun/src/cnn_yolov2ResNet50VehicleExample0_LaneDetection0_res2b_branch2b_w"
      ".bin",
      "/home/hmcl/ADcontrol/slprj/_sfprj/LaneDetection/_self/sfun/src/cnn_yolov2ResNet50VehicleExample0_LaneDetection0_res2b_branch2b_b"
      ".bin", 2);
    (static_cast<MWFusedConvReLULayer *>(this->layers[10]))
      ->createFusedConvReLULayer(this->targetImpl, 2, this->layers[9]
      ->getOutputTensor(0), this->layers[7]->getOutputTensor(0), 1, 1, 64, 256,
      1, 1, 0, 0, 0, 0, 1, 1, 1,
      "/home/hmcl/ADcontrol/slprj/_sfprj/LaneDetection/_self/sfun/src/cnn_yolov2ResNet50VehicleExample0_LaneDetection0_res2b_branch2c_w"
      ".bin",
      "/home/hmcl/ADcontrol/slprj/_sfprj/LaneDetection/_self/sfun/src/cnn_yolov2ResNet50VehicleExample0_LaneDetection0_res2b_branch2c_b"
      ".bin", 1);
    (static_cast<MWFusedConvReLULayer *>(this->layers[11]))
      ->createFusedConvReLULayer(this->targetImpl, 1, this->layers[10]
      ->getOutputTensor(0), 1, 1, 256, 64, 1, 1, 0, 0, 0, 0, 1, 1, 1,
      "/home/hmcl/ADcontrol/slprj/_sfprj/LaneDetection/_self/sfun/src/cnn_yolov2ResNet50VehicleExample0_LaneDetection0_res2c_branch2a_w"
      ".bin",
      "/home/hmcl/ADcontrol/slprj/_sfprj/LaneDetection/_self/sfun/src/cnn_yolov2ResNet50VehicleExample0_LaneDetection0_res2c_branch2a_b"
      ".bin", 0);
    (static_cast<MWFusedConvReLULayer *>(this->layers[12]))
      ->createFusedConvReLULayer(this->targetImpl, 1, this->layers[11]
      ->getOutputTensor(0), 3, 3, 64, 64, 2, 2, 1, 1, 1, 1, 1, 1, 1,
      "/home/hmcl/ADcontrol/slprj/_sfprj/LaneDetection/_self/sfun/src/cnn_yolov2ResNet50VehicleExample0_LaneDetection0_res2c_branch2b_w"
      ".bin",
      "/home/hmcl/ADcontrol/slprj/_sfprj/LaneDetection/_self/sfun/src/cnn_yolov2ResNet50VehicleExample0_LaneDetection0_res2c_branch2b_b"
      ".bin", 2);
    (static_cast<MWMaxPoolingLayer *>(this->layers[13]))->createMaxPoolingLayer
      (this->targetImpl, this->layers[10]->getOutputTensor(0), 1, 1, 2, 2, 0, 0,
       0, 0, 0, 1, 0);
    (static_cast<MWFusedConvReLULayer *>(this->layers[14]))
      ->createFusedConvReLULayer(this->targetImpl, 2, this->layers[12]
      ->getOutputTensor(0), this->layers[13]->getOutputTensor(0), 1, 1, 64, 256,
      1, 1, 0, 0, 0, 0, 1, 1, 1,
      "/home/hmcl/ADcontrol/slprj/_sfprj/LaneDetection/_self/sfun/src/cnn_yolov2ResNet50VehicleExample0_LaneDetection0_res2c_branch2c_w"
      ".bin",
      "/home/hmcl/ADcontrol/slprj/_sfprj/LaneDetection/_self/sfun/src/cnn_yolov2ResNet50VehicleExample0_LaneDetection0_res2c_branch2c_b"
      ".bin", 0);
    (static_cast<MWFusedConvReLULayer *>(this->layers[15]))
      ->createFusedConvReLULayer(this->targetImpl, 1, this->layers[14]
      ->getOutputTensor(0), 1, 1, 256, 128, 1, 1, 0, 0, 0, 0, 1, 1, 1,
      "/home/hmcl/ADcontrol/slprj/_sfprj/LaneDetection/_self/sfun/src/cnn_yolov2ResNet50VehicleExample0_LaneDetection0_res3a_branch2a_w"
      ".bin",
      "/home/hmcl/ADcontrol/slprj/_sfprj/LaneDetection/_self/sfun/src/cnn_yolov2ResNet50VehicleExample0_LaneDetection0_res3a_branch2a_b"
      ".bin", 1);
    (static_cast<MWFusedConvReLULayer *>(this->layers[16]))
      ->createFusedConvReLULayer(this->targetImpl, 1, this->layers[15]
      ->getOutputTensor(0), 3, 3, 128, 128, 1, 1, 1, 1, 1, 1, 1, 1, 1,
      "/home/hmcl/ADcontrol/slprj/_sfprj/LaneDetection/_self/sfun/src/cnn_yolov2ResNet50VehicleExample0_LaneDetection0_res3a_branch2b_w"
      ".bin",
      "/home/hmcl/ADcontrol/slprj/_sfprj/LaneDetection/_self/sfun/src/cnn_yolov2ResNet50VehicleExample0_LaneDetection0_res3a_branch2b_b"
      ".bin", 2);
    (static_cast<MWConvLayer *>(this->layers[17]))->createConvLayer
      (this->targetImpl, this->layers[14]->getOutputTensor(0), 1, 1, 256, 512, 1,
       1, 0, 0, 0, 0, 1, 1, 1,
       "/home/hmcl/ADcontrol/slprj/_sfprj/LaneDetection/_self/sfun/src/cnn_yolov2ResNet50VehicleExample0_LaneDetection0_res3a_branch1_w."
       "bin",
       "/home/hmcl/ADcontrol/slprj/_sfprj/LaneDetection/_self/sfun/src/cnn_yolov2ResNet50VehicleExample0_LaneDetection0_res3a_branch1_b."
       "bin", 1);
    (static_cast<MWFusedConvReLULayer *>(this->layers[18]))
      ->createFusedConvReLULayer(this->targetImpl, 2, this->layers[16]
      ->getOutputTensor(0), this->layers[17]->getOutputTensor(0), 1, 1, 128, 512,
      1, 1, 0, 0, 0, 0, 1, 1, 1,
      "/home/hmcl/ADcontrol/slprj/_sfprj/LaneDetection/_self/sfun/src/cnn_yolov2ResNet50VehicleExample0_LaneDetection0_res3a_branch2c_w"
      ".bin",
      "/home/hmcl/ADcontrol/slprj/_sfprj/LaneDetection/_self/sfun/src/cnn_yolov2ResNet50VehicleExample0_LaneDetection0_res3a_branch2c_b"
      ".bin", 1);
    (static_cast<MWFusedConvReLULayer *>(this->layers[19]))
      ->createFusedConvReLULayer(this->targetImpl, 1, this->layers[18]
      ->getOutputTensor(0), 1, 1, 512, 128, 1, 1, 0, 0, 0, 0, 1, 1, 1,
      "/home/hmcl/ADcontrol/slprj/_sfprj/LaneDetection/_self/sfun/src/cnn_yolov2ResNet50VehicleExample0_LaneDetection0_res3b_branch2a_w"
      ".bin",
      "/home/hmcl/ADcontrol/slprj/_sfprj/LaneDetection/_self/sfun/src/cnn_yolov2ResNet50VehicleExample0_LaneDetection0_res3b_branch2a_b"
      ".bin", 0);
    (static_cast<MWFusedConvReLULayer *>(this->layers[20]))
      ->createFusedConvReLULayer(this->targetImpl, 1, this->layers[19]
      ->getOutputTensor(0), 3, 3, 128, 128, 1, 1, 1, 1, 1, 1, 1, 1, 1,
      "/home/hmcl/ADcontrol/slprj/_sfprj/LaneDetection/_self/sfun/src/cnn_yolov2ResNet50VehicleExample0_LaneDetection0_res3b_branch2b_w"
      ".bin",
      "/home/hmcl/ADcontrol/slprj/_sfprj/LaneDetection/_self/sfun/src/cnn_yolov2ResNet50VehicleExample0_LaneDetection0_res3b_branch2b_b"
      ".bin", 2);
    (static_cast<MWFusedConvReLULayer *>(this->layers[21]))
      ->createFusedConvReLULayer(this->targetImpl, 2, this->layers[20]
      ->getOutputTensor(0), this->layers[18]->getOutputTensor(0), 1, 1, 128, 512,
      1, 1, 0, 0, 0, 0, 1, 1, 1,
      "/home/hmcl/ADcontrol/slprj/_sfprj/LaneDetection/_self/sfun/src/cnn_yolov2ResNet50VehicleExample0_LaneDetection0_res3b_branch2c_w"
      ".bin",
      "/home/hmcl/ADcontrol/slprj/_sfprj/LaneDetection/_self/sfun/src/cnn_yolov2ResNet50VehicleExample0_LaneDetection0_res3b_branch2c_b"
      ".bin", 1);
    (static_cast<MWFusedConvReLULayer *>(this->layers[22]))
      ->createFusedConvReLULayer(this->targetImpl, 1, this->layers[21]
      ->getOutputTensor(0), 1, 1, 512, 128, 1, 1, 0, 0, 0, 0, 1, 1, 1,
      "/home/hmcl/ADcontrol/slprj/_sfprj/LaneDetection/_self/sfun/src/cnn_yolov2ResNet50VehicleExample0_LaneDetection0_res3c_branch2a_w"
      ".bin",
      "/home/hmcl/ADcontrol/slprj/_sfprj/LaneDetection/_self/sfun/src/cnn_yolov2ResNet50VehicleExample0_LaneDetection0_res3c_branch2a_b"
      ".bin", 0);
    (static_cast<MWFusedConvReLULayer *>(this->layers[23]))
      ->createFusedConvReLULayer(this->targetImpl, 1, this->layers[22]
      ->getOutputTensor(0), 3, 3, 128, 128, 1, 1, 1, 1, 1, 1, 1, 1, 1,
      "/home/hmcl/ADcontrol/slprj/_sfprj/LaneDetection/_self/sfun/src/cnn_yolov2ResNet50VehicleExample0_LaneDetection0_res3c_branch2b_w"
      ".bin",
      "/home/hmcl/ADcontrol/slprj/_sfprj/LaneDetection/_self/sfun/src/cnn_yolov2ResNet50VehicleExample0_LaneDetection0_res3c_branch2b_b"
      ".bin", 2);
    (static_cast<MWFusedConvReLULayer *>(this->layers[24]))
      ->createFusedConvReLULayer(this->targetImpl, 2, this->layers[23]
      ->getOutputTensor(0), this->layers[21]->getOutputTensor(0), 1, 1, 128, 512,
      1, 1, 0, 0, 0, 0, 1, 1, 1,
      "/home/hmcl/ADcontrol/slprj/_sfprj/LaneDetection/_self/sfun/src/cnn_yolov2ResNet50VehicleExample0_LaneDetection0_res3c_branch2c_w"
      ".bin",
      "/home/hmcl/ADcontrol/slprj/_sfprj/LaneDetection/_self/sfun/src/cnn_yolov2ResNet50VehicleExample0_LaneDetection0_res3c_branch2c_b"
      ".bin", 1);
    (static_cast<MWFusedConvReLULayer *>(this->layers[25]))
      ->createFusedConvReLULayer(this->targetImpl, 1, this->layers[24]
      ->getOutputTensor(0), 1, 1, 512, 128, 1, 1, 0, 0, 0, 0, 1, 1, 1,
      "/home/hmcl/ADcontrol/slprj/_sfprj/LaneDetection/_self/sfun/src/cnn_yolov2ResNet50VehicleExample0_LaneDetection0_res3d_branch2a_w"
      ".bin",
      "/home/hmcl/ADcontrol/slprj/_sfprj/LaneDetection/_self/sfun/src/cnn_yolov2ResNet50VehicleExample0_LaneDetection0_res3d_branch2a_b"
      ".bin", 0);
    (static_cast<MWFusedConvReLULayer *>(this->layers[26]))
      ->createFusedConvReLULayer(this->targetImpl, 1, this->layers[25]
      ->getOutputTensor(0), 3, 3, 128, 128, 2, 2, 1, 1, 1, 1, 1, 1, 1,
      "/home/hmcl/ADcontrol/slprj/_sfprj/LaneDetection/_self/sfun/src/cnn_yolov2ResNet50VehicleExample0_LaneDetection0_res3d_branch2b_w"
      ".bin",
      "/home/hmcl/ADcontrol/slprj/_sfprj/LaneDetection/_self/sfun/src/cnn_yolov2ResNet50VehicleExample0_LaneDetection0_res3d_branch2b_b"
      ".bin", 2);
    (static_cast<MWMaxPoolingLayer *>(this->layers[27]))->createMaxPoolingLayer
      (this->targetImpl, this->layers[24]->getOutputTensor(0), 1, 1, 2, 2, 0, 0,
       0, 0, 0, 1, 0);
    (static_cast<MWFusedConvReLULayer *>(this->layers[28]))
      ->createFusedConvReLULayer(this->targetImpl, 2, this->layers[26]
      ->getOutputTensor(0), this->layers[27]->getOutputTensor(0), 1, 1, 128, 512,
      1, 1, 0, 0, 0, 0, 1, 1, 1,
      "/home/hmcl/ADcontrol/slprj/_sfprj/LaneDetection/_self/sfun/src/cnn_yolov2ResNet50VehicleExample0_LaneDetection0_res3d_branch2c_w"
      ".bin",
      "/home/hmcl/ADcontrol/slprj/_sfprj/LaneDetection/_self/sfun/src/cnn_yolov2ResNet50VehicleExample0_LaneDetection0_res3d_branch2c_b"
      ".bin", 0);
    (static_cast<MWFusedConvReLULayer *>(this->layers[29]))
      ->createFusedConvReLULayer(this->targetImpl, 1, this->layers[28]
      ->getOutputTensor(0), 1, 1, 512, 256, 1, 1, 0, 0, 0, 0, 1, 1, 1,
      "/home/hmcl/ADcontrol/slprj/_sfprj/LaneDetection/_self/sfun/src/cnn_yolov2ResNet50VehicleExample0_LaneDetection0_res4a_branch2a_w"
      ".bin",
      "/home/hmcl/ADcontrol/slprj/_sfprj/LaneDetection/_self/sfun/src/cnn_yolov2ResNet50VehicleExample0_LaneDetection0_res4a_branch2a_b"
      ".bin", 1);
    (static_cast<MWFusedConvReLULayer *>(this->layers[30]))
      ->createFusedConvReLULayer(this->targetImpl, 1, this->layers[29]
      ->getOutputTensor(0), 3, 3, 256, 256, 1, 1, 1, 1, 1, 1, 1, 1, 1,
      "/home/hmcl/ADcontrol/slprj/_sfprj/LaneDetection/_self/sfun/src/cnn_yolov2ResNet50VehicleExample0_LaneDetection0_res4a_branch2b_w"
      ".bin",
      "/home/hmcl/ADcontrol/slprj/_sfprj/LaneDetection/_self/sfun/src/cnn_yolov2ResNet50VehicleExample0_LaneDetection0_res4a_branch2b_b"
      ".bin", 2);
    (static_cast<MWConvLayer *>(this->layers[31]))->createConvLayer
      (this->targetImpl, this->layers[28]->getOutputTensor(0), 1, 1, 512, 1024,
       1, 1, 0, 0, 0, 0, 1, 1, 1,
       "/home/hmcl/ADcontrol/slprj/_sfprj/LaneDetection/_self/sfun/src/cnn_yolov2ResNet50VehicleExample0_LaneDetection0_res4a_branch1_w."
       "bin",
       "/home/hmcl/ADcontrol/slprj/_sfprj/LaneDetection/_self/sfun/src/cnn_yolov2ResNet50VehicleExample0_LaneDetection0_res4a_branch1_b."
       "bin", 1);
    (static_cast<MWFusedConvReLULayer *>(this->layers[32]))
      ->createFusedConvReLULayer(this->targetImpl, 2, this->layers[30]
      ->getOutputTensor(0), this->layers[31]->getOutputTensor(0), 1, 1, 256,
      1024, 1, 1, 0, 0, 0, 0, 1, 1, 1,
      "/home/hmcl/ADcontrol/slprj/_sfprj/LaneDetection/_self/sfun/src/cnn_yolov2ResNet50VehicleExample0_LaneDetection0_res4a_branch2c_w"
      ".bin",
      "/home/hmcl/ADcontrol/slprj/_sfprj/LaneDetection/_self/sfun/src/cnn_yolov2ResNet50VehicleExample0_LaneDetection0_res4a_branch2c_b"
      ".bin", 1);
    (static_cast<MWFusedConvReLULayer *>(this->layers[33]))
      ->createFusedConvReLULayer(this->targetImpl, 1, this->layers[32]
      ->getOutputTensor(0), 1, 1, 1024, 256, 1, 1, 0, 0, 0, 0, 1, 1, 1,
      "/home/hmcl/ADcontrol/slprj/_sfprj/LaneDetection/_self/sfun/src/cnn_yolov2ResNet50VehicleExample0_LaneDetection0_res4b_branch2a_w"
      ".bin",
      "/home/hmcl/ADcontrol/slprj/_sfprj/LaneDetection/_self/sfun/src/cnn_yolov2ResNet50VehicleExample0_LaneDetection0_res4b_branch2a_b"
      ".bin", 0);
    (static_cast<MWFusedConvReLULayer *>(this->layers[34]))
      ->createFusedConvReLULayer(this->targetImpl, 1, this->layers[33]
      ->getOutputTensor(0), 3, 3, 256, 256, 1, 1, 1, 1, 1, 1, 1, 1, 1,
      "/home/hmcl/ADcontrol/slprj/_sfprj/LaneDetection/_self/sfun/src/cnn_yolov2ResNet50VehicleExample0_LaneDetection0_res4b_branch2b_w"
      ".bin",
      "/home/hmcl/ADcontrol/slprj/_sfprj/LaneDetection/_self/sfun/src/cnn_yolov2ResNet50VehicleExample0_LaneDetection0_res4b_branch2b_b"
      ".bin", 2);
    (static_cast<MWFusedConvReLULayer *>(this->layers[35]))
      ->createFusedConvReLULayer(this->targetImpl, 2, this->layers[34]
      ->getOutputTensor(0), this->layers[32]->getOutputTensor(0), 1, 1, 256,
      1024, 1, 1, 0, 0, 0, 0, 1, 1, 1,
      "/home/hmcl/ADcontrol/slprj/_sfprj/LaneDetection/_self/sfun/src/cnn_yolov2ResNet50VehicleExample0_LaneDetection0_res4b_branch2c_w"
      ".bin",
      "/home/hmcl/ADcontrol/slprj/_sfprj/LaneDetection/_self/sfun/src/cnn_yolov2ResNet50VehicleExample0_LaneDetection0_res4b_branch2c_b"
      ".bin", 1);
    (static_cast<MWFusedConvReLULayer *>(this->layers[36]))
      ->createFusedConvReLULayer(this->targetImpl, 1, this->layers[35]
      ->getOutputTensor(0), 1, 1, 1024, 256, 1, 1, 0, 0, 0, 0, 1, 1, 1,
      "/home/hmcl/ADcontrol/slprj/_sfprj/LaneDetection/_self/sfun/src/cnn_yolov2ResNet50VehicleExample0_LaneDetection0_res4c_branch2a_w"
      ".bin",
      "/home/hmcl/ADcontrol/slprj/_sfprj/LaneDetection/_self/sfun/src/cnn_yolov2ResNet50VehicleExample0_LaneDetection0_res4c_branch2a_b"
      ".bin", 0);
    (static_cast<MWFusedConvReLULayer *>(this->layers[37]))
      ->createFusedConvReLULayer(this->targetImpl, 1, this->layers[36]
      ->getOutputTensor(0), 3, 3, 256, 256, 1, 1, 1, 1, 1, 1, 1, 1, 1,
      "/home/hmcl/ADcontrol/slprj/_sfprj/LaneDetection/_self/sfun/src/cnn_yolov2ResNet50VehicleExample0_LaneDetection0_res4c_branch2b_w"
      ".bin",
      "/home/hmcl/ADcontrol/slprj/_sfprj/LaneDetection/_self/sfun/src/cnn_yolov2ResNet50VehicleExample0_LaneDetection0_res4c_branch2b_b"
      ".bin", 2);
    (static_cast<MWFusedConvReLULayer *>(this->layers[38]))
      ->createFusedConvReLULayer(this->targetImpl, 2, this->layers[37]
      ->getOutputTensor(0), this->layers[35]->getOutputTensor(0), 1, 1, 256,
      1024, 1, 1, 0, 0, 0, 0, 1, 1, 1,
      "/home/hmcl/ADcontrol/slprj/_sfprj/LaneDetection/_self/sfun/src/cnn_yolov2ResNet50VehicleExample0_LaneDetection0_res4c_branch2c_w"
      ".bin",
      "/home/hmcl/ADcontrol/slprj/_sfprj/LaneDetection/_self/sfun/src/cnn_yolov2ResNet50VehicleExample0_LaneDetection0_res4c_branch2c_b"
      ".bin", 1);
    (static_cast<MWFusedConvReLULayer *>(this->layers[39]))
      ->createFusedConvReLULayer(this->targetImpl, 1, this->layers[38]
      ->getOutputTensor(0), 1, 1, 1024, 256, 1, 1, 0, 0, 0, 0, 1, 1, 1,
      "/home/hmcl/ADcontrol/slprj/_sfprj/LaneDetection/_self/sfun/src/cnn_yolov2ResNet50VehicleExample0_LaneDetection0_res4d_branch2a_w"
      ".bin",
      "/home/hmcl/ADcontrol/slprj/_sfprj/LaneDetection/_self/sfun/src/cnn_yolov2ResNet50VehicleExample0_LaneDetection0_res4d_branch2a_b"
      ".bin", 0);
    (static_cast<MWFusedConvReLULayer *>(this->layers[40]))
      ->createFusedConvReLULayer(this->targetImpl, 1, this->layers[39]
      ->getOutputTensor(0), 3, 3, 256, 256, 1, 1, 1, 1, 1, 1, 1, 1, 1,
      "/home/hmcl/ADcontrol/slprj/_sfprj/LaneDetection/_self/sfun/src/cnn_yolov2ResNet50VehicleExample0_LaneDetection0_res4d_branch2b_w"
      ".bin",
      "/home/hmcl/ADcontrol/slprj/_sfprj/LaneDetection/_self/sfun/src/cnn_yolov2ResNet50VehicleExample0_LaneDetection0_res4d_branch2b_b"
      ".bin", 2);
    (static_cast<MWFusedConvReLULayer *>(this->layers[41]))
      ->createFusedConvReLULayer(this->targetImpl, 2, this->layers[40]
      ->getOutputTensor(0), this->layers[38]->getOutputTensor(0), 1, 1, 256,
      1024, 1, 1, 0, 0, 0, 0, 1, 1, 1,
      "/home/hmcl/ADcontrol/slprj/_sfprj/LaneDetection/_self/sfun/src/cnn_yolov2ResNet50VehicleExample0_LaneDetection0_res4d_branch2c_w"
      ".bin",
      "/home/hmcl/ADcontrol/slprj/_sfprj/LaneDetection/_self/sfun/src/cnn_yolov2ResNet50VehicleExample0_LaneDetection0_res4d_branch2c_b"
      ".bin", 1);
    (static_cast<MWFusedConvReLULayer *>(this->layers[42]))
      ->createFusedConvReLULayer(this->targetImpl, 1, this->layers[41]
      ->getOutputTensor(0), 1, 1, 1024, 256, 1, 1, 0, 0, 0, 0, 1, 1, 1,
      "/home/hmcl/ADcontrol/slprj/_sfprj/LaneDetection/_self/sfun/src/cnn_yolov2ResNet50VehicleExample0_LaneDetection0_res4e_branch2a_w"
      ".bin",
      "/home/hmcl/ADcontrol/slprj/_sfprj/LaneDetection/_self/sfun/src/cnn_yolov2ResNet50VehicleExample0_LaneDetection0_res4e_branch2a_b"
      ".bin", 0);
    (static_cast<MWFusedConvReLULayer *>(this->layers[43]))
      ->createFusedConvReLULayer(this->targetImpl, 1, this->layers[42]
      ->getOutputTensor(0), 3, 3, 256, 256, 1, 1, 1, 1, 1, 1, 1, 1, 1,
      "/home/hmcl/ADcontrol/slprj/_sfprj/LaneDetection/_self/sfun/src/cnn_yolov2ResNet50VehicleExample0_LaneDetection0_res4e_branch2b_w"
      ".bin",
      "/home/hmcl/ADcontrol/slprj/_sfprj/LaneDetection/_self/sfun/src/cnn_yolov2ResNet50VehicleExample0_LaneDetection0_res4e_branch2b_b"
      ".bin", 2);
    (static_cast<MWFusedConvReLULayer *>(this->layers[44]))
      ->createFusedConvReLULayer(this->targetImpl, 2, this->layers[43]
      ->getOutputTensor(0), this->layers[41]->getOutputTensor(0), 1, 1, 256,
      1024, 1, 1, 0, 0, 0, 0, 1, 1, 1,
      "/home/hmcl/ADcontrol/slprj/_sfprj/LaneDetection/_self/sfun/src/cnn_yolov2ResNet50VehicleExample0_LaneDetection0_res4e_branch2c_w"
      ".bin",
      "/home/hmcl/ADcontrol/slprj/_sfprj/LaneDetection/_self/sfun/src/cnn_yolov2ResNet50VehicleExample0_LaneDetection0_res4e_branch2c_b"
      ".bin", 1);
    (static_cast<MWFusedConvReLULayer *>(this->layers[45]))
      ->createFusedConvReLULayer(this->targetImpl, 1, this->layers[44]
      ->getOutputTensor(0), 1, 1, 1024, 256, 1, 1, 0, 0, 0, 0, 1, 1, 1,
      "/home/hmcl/ADcontrol/slprj/_sfprj/LaneDetection/_self/sfun/src/cnn_yolov2ResNet50VehicleExample0_LaneDetection0_res4f_branch2a_w"
      ".bin",
      "/home/hmcl/ADcontrol/slprj/_sfprj/LaneDetection/_self/sfun/src/cnn_yolov2ResNet50VehicleExample0_LaneDetection0_res4f_branch2a_b"
      ".bin", 0);
    (static_cast<MWFusedConvReLULayer *>(this->layers[46]))
      ->createFusedConvReLULayer(this->targetImpl, 1, this->layers[45]
      ->getOutputTensor(0), 3, 3, 256, 256, 1, 1, 1, 1, 1, 1, 1, 1, 1,
      "/home/hmcl/ADcontrol/slprj/_sfprj/LaneDetection/_self/sfun/src/cnn_yolov2ResNet50VehicleExample0_LaneDetection0_res4f_branch2b_w"
      ".bin",
      "/home/hmcl/ADcontrol/slprj/_sfprj/LaneDetection/_self/sfun/src/cnn_yolov2ResNet50VehicleExample0_LaneDetection0_res4f_branch2b_b"
      ".bin", 2);
    (static_cast<MWFusedConvReLULayer *>(this->layers[47]))
      ->createFusedConvReLULayer(this->targetImpl, 2, this->layers[46]
      ->getOutputTensor(0), this->layers[44]->getOutputTensor(0), 1, 1, 256,
      1024, 1, 1, 0, 0, 0, 0, 1, 1, 1,
      "/home/hmcl/ADcontrol/slprj/_sfprj/LaneDetection/_self/sfun/src/cnn_yolov2ResNet50VehicleExample0_LaneDetection0_res4f_branch2c_w"
      ".bin",
      "/home/hmcl/ADcontrol/slprj/_sfprj/LaneDetection/_self/sfun/src/cnn_yolov2ResNet50VehicleExample0_LaneDetection0_res4f_branch2c_b"
      ".bin", 1);
    (static_cast<MWFusedConvReLULayer *>(this->layers[48]))
      ->createFusedConvReLULayer(this->targetImpl, 1, this->layers[47]
      ->getOutputTensor(0), 3, 3, 1024, 1024, 1, 1, 1, 1, 1, 1, 1, 1, 1,
      "/home/hmcl/ADcontrol/slprj/_sfprj/LaneDetection/_self/sfun/src/cnn_yolov2ResNet50VehicleExample0_LaneDetection0_yolov2Conv1_w.bi"
      "n",
      "/home/hmcl/ADcontrol/slprj/_sfprj/LaneDetection/_self/sfun/src/cnn_yolov2ResNet50VehicleExample0_LaneDetection0_yolov2Conv1_b.bi"
      "n", 0);
    (static_cast<MWFusedConvReLULayer *>(this->layers[49]))
      ->createFusedConvReLULayer(this->targetImpl, 1, this->layers[48]
      ->getOutputTensor(0), 3, 3, 1024, 1024, 1, 1, 1, 1, 1, 1, 1, 1, 1,
      "/home/hmcl/ADcontrol/slprj/_sfprj/LaneDetection/_self/sfun/src/cnn_yolov2ResNet50VehicleExample0_LaneDetection0_yolov2Conv2_w.bi"
      "n",
      "/home/hmcl/ADcontrol/slprj/_sfprj/LaneDetection/_self/sfun/src/cnn_yolov2ResNet50VehicleExample0_LaneDetection0_yolov2Conv2_b.bi"
      "n", 1);
    (static_cast<MWConvLayer *>(this->layers[50]))->createConvLayer
      (this->targetImpl, this->layers[49]->getOutputTensor(0), 1, 1, 1024, 24, 1,
       1, 0, 0, 0, 0, 1, 1, 1,
       "/home/hmcl/ADcontrol/slprj/_sfprj/LaneDetection/_self/sfun/src/cnn_yolov2ResNet50VehicleExample0_LaneDetection0_yolov2ClassConv_"
       "w.bin",
       "/home/hmcl/ADcontrol/slprj/_sfprj/LaneDetection/_self/sfun/src/cnn_yolov2ResNet50VehicleExample0_LaneDetection0_yolov2ClassConv_"
       "b.bin", 0);
    (static_cast<MWYoloExtractionLayer *>(this->layers[51]))
      ->createYoloExtractionLayer(this->targetImpl, this->layers[50]
      ->getOutputTensor(0), 4, 1, 2, 3);
    (static_cast<MWSigmoidLayer *>(this->layers[52]))->createSigmoidLayer
      (this->targetImpl, this->layers[51]->getOutputTensor(0), 0);
    (static_cast<MWExponentialLayer *>(this->layers[53]))
      ->createExponentialLayer(this->targetImpl, this->layers[51]
      ->getOutputTensor(1), 1);
    (static_cast<MWYoloSoftmaxLayer *>(this->layers[54]))
      ->createYoloSoftmaxLayer(this->targetImpl, this->layers[51]
      ->getOutputTensor(2), 4, 2);
    (static_cast<MWConcatenationLayer *>(this->layers[55]))
      ->createConcatenationLayer(this->targetImpl, 3, this->layers[52]
      ->getOutputTensor(0), this->layers[53]->getOutputTensor(0), this->layers
      [54]->getOutputTensor(0), 3, 3);
    (static_cast<MWOutputLayer *>(this->layers[56]))->createOutputLayer
      (this->targetImpl, this->layers[55]->getOutputTensor(0), 3);
    this->outputTensors[0] = this->layers[56]->getOutputTensor(0);
    this->setSize();
  }
}

void c5_yolov2ResNet50VehicleExample0_LaneDetection0::predict()
{
  int32_T c5_idx;
  for (c5_idx = 0; c5_idx < 57; c5_idx++) {
    this->layers[c5_idx]->predict();
  }
}

void c5_yolov2ResNet50VehicleExample0_LaneDetection0::activations(int32_T
  c5_layerIdx)
{
  int32_T c5_idx;
  for (c5_idx = 0; c5_idx <= c5_layerIdx; c5_idx++) {
    this->layers[c5_idx]->predict();
  }
}

void c5_yolov2ResNet50VehicleExample0_LaneDetection0::cleanup()
{
  int32_T c5_idx;
  this->deallocate();
  for (c5_idx = 0; c5_idx < 57; c5_idx++) {
    this->layers[c5_idx]->cleanup();
  }

  if (this->targetImpl) {
    this->targetImpl->cleanup();
  }
}

real32_T *c5_yolov2ResNet50VehicleExample0_LaneDetection0::getLayerOutput
  (int32_T c5_layerIndex, int32_T c5_portIndex)
{
  return this->layers[c5_layerIndex]->getLayerOutput(c5_portIndex);
}

real32_T *c5_yolov2ResNet50VehicleExample0_LaneDetection0::getInputDataPointer
  (int32_T c5_index)
{
  return (static_cast<MWTensor<real32_T> *>(this->inputTensors[c5_index]))
    ->getData();
}

real32_T *c5_yolov2ResNet50VehicleExample0_LaneDetection0::getInputDataPointer()
{
  return (static_cast<MWTensor<real32_T> *>(this->inputTensors[0]))->getData();
}

real32_T *c5_yolov2ResNet50VehicleExample0_LaneDetection0::getOutputDataPointer
  (int32_T c5_index)
{
  return (static_cast<MWTensor<real32_T> *>(this->outputTensors[c5_index]))
    ->getData();
}

real32_T *c5_yolov2ResNet50VehicleExample0_LaneDetection0::getOutputDataPointer()
{
  return (static_cast<MWTensor<real32_T> *>(this->outputTensors[0]))->getData();
}

int32_T c5_yolov2ResNet50VehicleExample0_LaneDetection0::getBatchSize()
{
  return this->inputTensors[0]->getBatchSize();
}

c5_yolov2ResNet50VehicleExample0_LaneDetection0::
  ~c5_yolov2ResNet50VehicleExample0_LaneDetection0()
{
  int32_T c5_idx;
  this->cleanup();
  c5_checkCleanupCudaError(hipGetLastError(), __FILE__, __LINE__);
  for (c5_idx = 0; c5_idx < 57; c5_idx++) {
    delete this->layers[c5_idx];
  }

  if (this->targetImpl) {
    delete this->targetImpl;
  }

  delete this->inputTensors[0];
}

static void c5_checkCleanupCudaError(hipError_t c5_errCode, const char_T
  *c5_file, uint32_T c5_line)
{
  emlrtRTEInfo c5_rtInfo;
  if ((c5_errCode != hipSuccess) && (c5_errCode != hipErrorDeinitialized)) {
    c5_rtInfo = c5_createEmlrtInfoStruct(c5_file, c5_line);
    emlrtCUDAWarning(c5_errCode, hipGetErrorName(c5_errCode),
                     hipGetErrorString(c5_errCode), &c5_rtInfo);
  }
}

static emlrtRTEInfo c5_createEmlrtInfoStruct(const char_T *c5_file, uint32_T
  c5_line)
{
  emlrtRTEInfo c5_b_rtInfo;
  uint32_T c5_len;
  char_T *c5_brk;
  char_T *c5_fn;
  char_T *c5_pn;
  c5_len = (uint32_T)strlen(c5_file);
  c5_pn = (char_T *)calloc(c5_len + 1U, 1U);
  c5_fn = (char_T *)calloc(c5_len + 1U, 1U);
  memcpy(c5_pn, c5_file, c5_len);
  memcpy(c5_fn, c5_file, c5_len);
  c5_brk = strrchr(c5_fn, '.');
  *c5_brk = '\x00';
  c5_brk = NULL;
  c5_brk = strrchr(c5_fn, '/');
  if (c5_brk == NULL) {
    c5_brk = strrchr(c5_fn, '\\');
  }

  if (c5_brk == NULL) {
    c5_brk = c5_fn;
  } else {
    c5_brk++;
  }

  c5_b_rtInfo.lineNo = c5_line;
  c5_b_rtInfo.colNo = 0;
  c5_b_rtInfo.fName = c5_brk;
  c5_b_rtInfo.pName = c5_pn;
  return c5_b_rtInfo;
}

static void init_dsm_address_info(SFc5_LaneDetectionInstanceStruct
  *chartInstance)
{
}

static void init_simulink_io_address(SFc5_LaneDetectionInstanceStruct
  *chartInstance)
{
  chartInstance->c5_fEmlrtCtx = (void *)sfrtGetEmlrtCtx(chartInstance->S);
  chartInstance->c5_bboxes_data = (real_T (*)[80])ssGetOutputPortSignal_wrapper
    (chartInstance->S, 1);
  chartInstance->c5_bboxes_sizes = (int32_T (*)[2])
    ssGetCurrentOutputPortDimensions_wrapper(chartInstance->S, 1);
  chartInstance->c5_In = (real32_T (*)[1229760])ssGetInputPortSignal_wrapper
    (chartInstance->S, 0);
  chartInstance->c5_scores_data = (real32_T (*)[20])
    ssGetOutputPortSignal_wrapper(chartInstance->S, 2);
  chartInstance->c5_scores_sizes = (int32_T (*)[2])
    ssGetCurrentOutputPortDimensions_wrapper(chartInstance->S, 2);
}

#include <stdio.h>
#include <stdlib.h>
#include <string.h>

/* SFunction Glue Code */
void sf_c5_LaneDetection_get_check_sum(mxArray *plhs[])
{
  ((real_T *)mxGetPr((plhs[0])))[0] = (real_T)(3571303581U);
  ((real_T *)mxGetPr((plhs[0])))[1] = (real_T)(3804992752U);
  ((real_T *)mxGetPr((plhs[0])))[2] = (real_T)(2354264699U);
  ((real_T *)mxGetPr((plhs[0])))[3] = (real_T)(514449129U);
}

mxArray *sf_c5_LaneDetection_third_party_uses_info(void)
{
  mxArray * mxcell3p = mxCreateCellMatrix(1,1);
  mxSetCell(mxcell3p, 0, mxCreateString("dltargets.cudnn.cudnnApi"));
  return(mxcell3p);
}

mxArray *sf_c5_LaneDetection_jit_fallback_info(void)
{
  const char *infoFields[] = { "fallbackType", "fallbackReason",
    "hiddenFallbackType", "hiddenFallbackReason", "incompatibleSymbol" };

  mxArray *mxInfo = mxCreateStructMatrix(1, 1, 5, infoFields);
  mxArray *fallbackType = mxCreateString("pre");
  mxArray *fallbackReason = mxCreateString("GPUAcceleration");
  mxArray *hiddenFallbackType = mxCreateString("late");
  mxArray *hiddenFallbackReason = mxCreateString("ir_function_calls");
  mxArray *incompatibleSymbol = mxCreateString("#__setup__");
  mxSetField(mxInfo, 0, infoFields[0], fallbackType);
  mxSetField(mxInfo, 0, infoFields[1], fallbackReason);
  mxSetField(mxInfo, 0, infoFields[2], hiddenFallbackType);
  mxSetField(mxInfo, 0, infoFields[3], hiddenFallbackReason);
  mxSetField(mxInfo, 0, infoFields[4], incompatibleSymbol);
  return mxInfo;
}

mxArray *sf_c5_LaneDetection_updateBuildInfo_args_info(void)
{
  mxArray *mxBIArgs = sf_mex_decode(
    "eNrlWMFqwkAQjaUthYJWQatYEASvfkOLVSrVUGpL21NZklWWJrsh2Wh667Gf18/osccem0QTN2N"
    "q8CRsAiHsMtnHy8y8mYlSGI4V/yr691dNUY7954l/F5TldSSsz4X9wP5X2W5/AOwvBfvDFPuiYH"
    "+2WiNdnzDX1vCAGNhZ7n1k4BYA7l0G7gXADdbjp76BTUz5gjj4ajolFI/QO7a7mmVF5/7sib+3I"
    "/+bDNw6wK2H/Aeug/Ueo/N7PHqMuQfnfe+Jt7Uj7yzcU4AbrDVKX5FFIjdL5ecywC2Hfu6paujd"
    "oWkZXc2VML9bALe1Jb+jrxCcG7y0D/6fO/J/zsDtANxOBv9bbFO8igWZ4qAJcJv/6JyYCTLlfw3"
    "g1kL+D8ieYa5ivmD2W0xdJr83AG5jqXuuw5kZOnzA7J57raqx6uehvvshryGOKeKEUSnrewngli"
    "Le83UrJ39+9z2LUV/oCTKEDlYm3hWAWwl5T8jMZERPdO150LUXZrC+x22kJRJbQl1Lj/eA/4RNu"
    "Yk8EO9y9zGbei5rH5Oe77GuJwYZ+fMd6nvu5rgU/usJRqa4rwLc6kadW/te/v9zKXVOCH2Z5vc2"
    "wG1v4y8O77npd8R6n9S/P7KUnlA="
    );
  return mxBIArgs;
}

static const mxArray *sf_get_sim_state_info_c5_LaneDetection(void)
{
  const char *infoFields[] = { "chartChecksum", "varInfo" };

  mxArray *mxInfo = mxCreateStructMatrix(1, 1, 2, infoFields);
  mxArray *mxVarInfo = sf_mex_decode(
    "eNpjYPT0ZQACPiA+wcTAwAakOYCYiQECWKF8RiBmhtIQcRa4uAIQl1QWpILEi4uSPVOAdF5iLpi"
    "fWFrhmZeWDzbfggFhPhsW8xmRzOeEikPAB3vK9Is4oOtnwaKfDUm/AJSflJRfkVoMCR9YOA2cPx"
    "TI9kdxcn4R9fyB6Q7S9EPsDyDgDyk0f4D4mcXxicklmWWp8cmm8T6JeakuqSWpQIH8PIS5IAAA5"
    "lYhew=="
    );
  mxArray *mxChecksum = mxCreateDoubleMatrix(1, 4, mxREAL);
  sf_c5_LaneDetection_get_check_sum(&mxChecksum);
  mxSetField(mxInfo, 0, infoFields[0], mxChecksum);
  mxSetField(mxInfo, 0, infoFields[1], mxVarInfo);
  return mxInfo;
}

static const char* sf_get_instance_specialization(void)
{
  return "sLBlpmjqljmjH7g47RTOyuD";
}

static void sf_opaque_initialize_c5_LaneDetection(void *chartInstanceVar)
{
  initialize_params_c5_LaneDetection((SFc5_LaneDetectionInstanceStruct*)
    chartInstanceVar);
  initialize_c5_LaneDetection((SFc5_LaneDetectionInstanceStruct*)
    chartInstanceVar);
}

static void sf_opaque_enable_c5_LaneDetection(void *chartInstanceVar)
{
  enable_c5_LaneDetection((SFc5_LaneDetectionInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_disable_c5_LaneDetection(void *chartInstanceVar)
{
  disable_c5_LaneDetection((SFc5_LaneDetectionInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_gateway_c5_LaneDetection(void *chartInstanceVar)
{
  sf_gateway_c5_LaneDetection((SFc5_LaneDetectionInstanceStruct*)
    chartInstanceVar);
}

static const mxArray* sf_opaque_get_sim_state_c5_LaneDetection(SimStruct* S)
{
  return get_sim_state_c5_LaneDetection((SFc5_LaneDetectionInstanceStruct *)
    sf_get_chart_instance_ptr(S));     /* raw sim ctx */
}

static void sf_opaque_set_sim_state_c5_LaneDetection(SimStruct* S, const mxArray
  *st)
{
  set_sim_state_c5_LaneDetection((SFc5_LaneDetectionInstanceStruct*)
    sf_get_chart_instance_ptr(S), st);
}

static void sf_opaque_cleanup_runtime_resources_c5_LaneDetection(void
  *chartInstanceVar)
{
  if (chartInstanceVar!=NULL) {
    SimStruct *S = ((SFc5_LaneDetectionInstanceStruct*) chartInstanceVar)->S;
    if (sim_mode_is_rtw_gen(S) || sim_mode_is_external(S)) {
      sf_clear_rtw_identifier(S);
      unload_LaneDetection_optimization_info();
    }

    mdl_cleanup_runtime_resources_c5_LaneDetection
      ((SFc5_LaneDetectionInstanceStruct*) chartInstanceVar);
    ((SFc5_LaneDetectionInstanceStruct*) chartInstanceVar)->
      ~SFc5_LaneDetectionInstanceStruct();
    utFree(chartInstanceVar);
    if (ssGetUserData(S)!= NULL) {
      sf_free_ChartRunTimeInfo(S);
    }

    ssSetUserData(S,NULL);
  }
}

static void sf_opaque_mdl_start_c5_LaneDetection(void *chartInstanceVar)
{
  mdl_start_c5_LaneDetection((SFc5_LaneDetectionInstanceStruct*)
    chartInstanceVar);
}

static void sf_opaque_mdl_terminate_c5_LaneDetection(void *chartInstanceVar)
{
  mdl_terminate_c5_LaneDetection((SFc5_LaneDetectionInstanceStruct*)
    chartInstanceVar);
}

static void sf_opaque_init_subchart_simstructs(void *chartInstanceVar)
{
  initSimStructsc5_LaneDetection((SFc5_LaneDetectionInstanceStruct*)
    chartInstanceVar);
}

extern unsigned int sf_machine_global_initializer_called(void);
static void mdlProcessParameters_c5_LaneDetection(SimStruct *S)
{
  int i;
  for (i=0;i<ssGetNumRunTimeParams(S);i++) {
    if (ssGetSFcnParamTunable(S,i)) {
      ssUpdateDlgParamAsRunTimeParam(S,i);
    }
  }

  sf_warn_if_symbolic_dimension_param_changed(S);
  if (sf_machine_global_initializer_called()) {
    initialize_params_c5_LaneDetection((SFc5_LaneDetectionInstanceStruct*)
      sf_get_chart_instance_ptr(S));
    initSimStructsc5_LaneDetection((SFc5_LaneDetectionInstanceStruct*)
      sf_get_chart_instance_ptr(S));
  }
}

const char* sf_c5_LaneDetection_get_post_codegen_info(void)
{
  int i;
  const char* encStrCodegen [32] = {
    "eNrtXE9vG0UUd9JSUQmqHpCQKiR6AwkJJ7ZDKw7Qxn9US0kTxWkRJ2tmduydeHd2szNrJypCon9",
    "OfAQucCucOHDgyIEDH4Er34CPwJu142zWu3Y3KbGfVUuOs+s3b3/v77yZfevCSnO7AK8b8H7JC4",
    "Vr8Pk2vFcLw9dbo+OV2Ht4/mrho9HxbzBIhu4uCYirClNfkrh8jyvPCbXwZFN2vFQyITs84JIBr",
    "e8FOoubEm7oCNlrhJIZfuorWzC7ZXuhY23CWGLtSOcYuPmh3gU+NRFwphucW9oOvLBrNxzSHSMO",
    "9KBqc9ZToTtNBMV1K/QNLLUdOlr4Dq8fcdaUShNArE6xtTTRvKqPMsU0kqrWCaHn+o4gMlVam6g",
    "W90HBmj/yLfi7E2oQKknGbBLoTW6TPldbohfx9CRP8hQKvqBCEu0Fgjh116magZPYdh3As+1Z3J",
    "miEMC2GXDS8z0hdbb9Ww2QtC4JdXiN07Cbza3FD0Nj/MeCD3iQqbdO1evzgHT5jsy8aKSQ+lFkr",
    "bGXTJJp4fLHJLjPwH6KW5neC56jWgTsxPdhRBYZj4Rsqv1A9EG9mdxCt2k8c1bIhO7Q2GoWWcSt",
    "3ufTrDDm1mCyShxHZZLte/4W73Mn4lojmkwnG3JNp1NKWPseKNi4d3Y0hFKA4UdkVU9aItVc/QR",
    "BlHceQmI5S8lCpT23Cs5b29qa/HqSrCk1DzqE8bQsEBChOOgsUm82N0soY3sgBFQ6gpdGPPSQWV",
    "QF1QllbeAFPdDJlCRyKoKxaCahq7pgS4iERwqCZhqZseUsOkaYzS2TYITDtyFsgDZFJ8qktvsQd",
    "32hj2tcsUD4KVYNIeogDdWNQx37/JHsSW8gG4HntkY5fmgFyAyQw12wwX4UY5IBK6E0pAtxenmL",
    "c3BKEkghu5uQ5oLjBoBMtZiZ99YKp/PeO68w752MS35+HOOzksKnEPtMXvf66vTrrsJ/K6Nx92L",
    "j3k1c52pinKG7Ce+fP/n62z+//+fu5z/9UP+F/PHNRa5/dCVfnXBjdPzBSUIeB1h/wq8N7YMYrq",
    "sp/N+P8b85OlZbm47vHhw6B+7Bgzvdyp29/Z3jsBbx+2t1Ot4rCbwn52+bmQG8MfLjgDWtUQFjj",
    "kk4nNYN/7sxvNdm6OP66Pzw9e+XFxv/3r3k+DR9XUvoyxxT6h2ZiIj77/zkuH1uORTzgtcnxySO",
    "fOOH19+dIcethBy3onqmTUyW5G220d4iEmokzaNyZTLP5I3bN+PejFvkcSvnnIfPO271gvP+ZY2",
    "7qHx565FFo1+bkkcLCfqbCyzHRevE/5v+70K+eu7D0fEX46Vd1RaOlVLlj76GWryT9u2S+Km/N5",
    "3eT9D/PqM++C7h1+a4aHsuL9ouc4r3a7Do0YHnFJXjBwfFtuqYjzNFA5zkTqdoFpBFKF2LTMr2s",
    "ed4/dIeVw+53lh7zG3BzNZVtJ2xdrbmWBsRwxq7X2rTT6mQaPGvtwc58b9I4H8xP/wOUcoIMZYB",
    "K36aA//zBP7nc8APi4pKp00DIpldYqcuhBQ/xYifI9f/GD9Fjx+3/xDk8YvdfwhO/7GQ508Luf9",
    "byP2HIZ+/GHL/Z8vi/xS9/lHip8jjlyKPX4q8/qG447dEcefPOH6U+ifLov9E/P44A7+XwO+9Hv",
    "w6IEJyy9CBBBOIO8ycWB+5CiY9M+R5kpwzTp8l8D+bc5yux3bKF93PS/j8vEwy5tNfZ+B/ksD/Z",
    "A74WeJOBMp5KGcdvHjxiVz/2NeBFHkdlnMd9TSB/+kc8EfPPrTX216no7g+vRGKMu/jrOPLZFnu",
    "IyLdxyHI90E6GflzVn3pJvC7l1JfsjM9I5ji9Lz7xYvQ73KS5xUjDk/0uyyun1Ry+cmi9UUN0Oi",
    "5nKt/aFHikS7LvI9z/65snXP/bn5+vnHGzxd5fkQYjyXk98PK1vLsp1M8++mD8T7jIs+PFEXe2E",
    "BZV9Nl6YNBMw+uY/STUla/Ap79fYpxf7nMlmV/hGCJz8qZ+u/lDJyHCZyHl4LTIppMbBu/kl57C",
    "by9S6o3Pst5/3V+OAfY88QA+fMkFHk/KMr1C/J+0BLyfuLy0uifIsfP0OPH2AdSyerTQoK/vDz4",
    "cdY/BPf8W0Z+H7yMvB+9jPx5gPjzkAPUfVwUef1Gkccvfv1jv/9FsfW3xH/PJe/v0Szg8zwUeT8",
    "A7vzD8t0/9RP4/cvbfx23c6H0E4bcz9Gvc1GusyrI+6YqBGGfRinWp4FQz7ifDyd4+goQ9q1VyL",
    "L8jkOsbvwPObOpPQ==",
    ""
  };

  static char newstr [2269] = "";
  newstr[0] = '\0';
  for (i = 0; i < 32; i++) {
    strcat(newstr, encStrCodegen[i]);
  }

  return newstr;
}

static void mdlSetWorkWidths_c5_LaneDetection(SimStruct *S)
{
  const char* newstr = sf_c5_LaneDetection_get_post_codegen_info();
  sf_set_work_widths(S, newstr);
  ssSetChecksum0(S,(2119773097U));
  ssSetChecksum1(S,(954305476U));
  ssSetChecksum2(S,(1167827258U));
  ssSetChecksum3(S,(2092982698U));
}

static void mdlRTW_c5_LaneDetection(SimStruct *S)
{
  if (sim_mode_is_rtw_gen(S)) {
    ssWriteRTWStrParam(S, "StateflowChartType", "Embedded MATLAB");
  }
}

static void mdlSetupRuntimeResources_c5_LaneDetection(SimStruct *S)
{
  SFc5_LaneDetectionInstanceStruct *chartInstance;
  chartInstance = (SFc5_LaneDetectionInstanceStruct *)utMalloc(sizeof
    (SFc5_LaneDetectionInstanceStruct));
  if (chartInstance==NULL) {
    sf_mex_error_message("Could not allocate memory for chart instance.");
  }

  memset(chartInstance, 0, sizeof(SFc5_LaneDetectionInstanceStruct));
  chartInstance = new (chartInstance) SFc5_LaneDetectionInstanceStruct;
  chartInstance->chartInfo.chartInstance = chartInstance;
  if (ssGetSampleTime(S, 0) == CONTINUOUS_SAMPLE_TIME && ssGetOffsetTime(S, 0) ==
      0 && ssGetNumContStates(ssGetRootSS(S)) > 0) {
    sf_error_out_about_continuous_sample_time_with_persistent_vars(S);
  }

  chartInstance->chartInfo.isEMLChart = 1;
  chartInstance->chartInfo.chartInitialized = 0;
  chartInstance->chartInfo.sFunctionGateway = sf_opaque_gateway_c5_LaneDetection;
  chartInstance->chartInfo.initializeChart =
    sf_opaque_initialize_c5_LaneDetection;
  chartInstance->chartInfo.mdlStart = sf_opaque_mdl_start_c5_LaneDetection;
  chartInstance->chartInfo.mdlTerminate =
    sf_opaque_mdl_terminate_c5_LaneDetection;
  chartInstance->chartInfo.mdlCleanupRuntimeResources =
    sf_opaque_cleanup_runtime_resources_c5_LaneDetection;
  chartInstance->chartInfo.enableChart = sf_opaque_enable_c5_LaneDetection;
  chartInstance->chartInfo.disableChart = sf_opaque_disable_c5_LaneDetection;
  chartInstance->chartInfo.getSimState =
    sf_opaque_get_sim_state_c5_LaneDetection;
  chartInstance->chartInfo.setSimState =
    sf_opaque_set_sim_state_c5_LaneDetection;
  chartInstance->chartInfo.getSimStateInfo =
    sf_get_sim_state_info_c5_LaneDetection;
  chartInstance->chartInfo.zeroCrossings = NULL;
  chartInstance->chartInfo.outputs = NULL;
  chartInstance->chartInfo.derivatives = NULL;
  chartInstance->chartInfo.mdlRTW = mdlRTW_c5_LaneDetection;
  chartInstance->chartInfo.mdlSetWorkWidths = mdlSetWorkWidths_c5_LaneDetection;
  chartInstance->chartInfo.callGetHoverDataForMsg = NULL;
  chartInstance->chartInfo.extModeExec = NULL;
  chartInstance->chartInfo.restoreLastMajorStepConfiguration = NULL;
  chartInstance->chartInfo.restoreBeforeLastMajorStepConfiguration = NULL;
  chartInstance->chartInfo.storeCurrentConfiguration = NULL;
  chartInstance->chartInfo.callAtomicSubchartUserFcn = NULL;
  chartInstance->chartInfo.callAtomicSubchartAutoFcn = NULL;
  chartInstance->chartInfo.callAtomicSubchartEventFcn = NULL;
  chartInstance->chartInfo.chartStateSetterFcn = NULL;
  chartInstance->chartInfo.chartStateGetterFcn = NULL;
  chartInstance->S = S;
  chartInstance->chartInfo.dispatchToExportedFcn = NULL;
  sf_init_ChartRunTimeInfo(S, &(chartInstance->chartInfo), false, 0,
    chartInstance->c5_JITStateAnimation,
    chartInstance->c5_JITTransitionAnimation);
  init_dsm_address_info(chartInstance);
  init_simulink_io_address(chartInstance);
  if (!sim_mode_is_rtw_gen(S)) {
  }

  mdl_setup_runtime_resources_c5_LaneDetection(chartInstance);
}

void c5_LaneDetection_method_dispatcher(SimStruct *S, int_T method, void *data)
{
  switch (method) {
   case SS_CALL_MDL_SETUP_RUNTIME_RESOURCES:
    mdlSetupRuntimeResources_c5_LaneDetection(S);
    break;

   case SS_CALL_MDL_SET_WORK_WIDTHS:
    mdlSetWorkWidths_c5_LaneDetection(S);
    break;

   case SS_CALL_MDL_PROCESS_PARAMETERS:
    mdlProcessParameters_c5_LaneDetection(S);
    break;

   default:
    /* Unhandled method */
    sf_mex_error_message("Stateflow Internal Error:\n"
                         "Error calling c5_LaneDetection_method_dispatcher.\n"
                         "Can't handle method %d.\n", method);
    break;
  }
}
