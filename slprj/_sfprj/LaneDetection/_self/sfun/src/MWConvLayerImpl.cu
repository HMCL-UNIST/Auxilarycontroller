#include "MWConvLayerImpl.hpp"
#include "MWConvLayer.hpp"
#include "MWTargetNetworkImpl.hpp"
#include "cnn_api.hpp"
#include <cassert>
#include <stdio.h>
 MWConvLayerImpl::MWConvLayerImpl(MWCNNLayer* layer, MWTargetNetworkImpl* 
ntwk_impl, int filt_H, int filt_W, int numGrps, int numChnls, int numFilts, int 
FrpxvsDMwwgbpqHXWxmN, int FwLnexHgxHRquTKmNpoa, int 
ClEhcJFlvGCgiavziIag, int CZNYmBcNFSZWvaCklqeM, int 
CufLFODQDXTAPyRqYodN, int DCdZnqpcBnvXVgEsLBnz, int 
AuqaQHxmPQSyYRemQvyX, int AzTsxYcYjIEJsGQbeYHm, const 
char* xHViLEwTujGGrPZZgmbF, const char* JwxFdqOKggeawILBfGgg) : 
MWCNNLayerImpl(layer, ntwk_impl) , vpXxoeEhdEosLSsYXkNG(NULL) , IwKnaBoXVubIRYcxEJLH(NULL) , 
TaAJDyqFVJXfAfCJhOuU(NULL) , BHuHNDGoRwGRouCxeMbw(filt_H) , BLjrjqvCcCommiXWQLjs 
(filt_W) , BuyZFXzwOMxcePIbCLfl (numGrps) , BkwhtPQUCQKchmmimoXs (numChnls) , 
BlRIQPyqJZORKENzSdYf (numFilts) , 
AdmgfUbRAfzFeYHxSnQr(AuqaQHxmPQSyYRemQvyX) , 
AwZQzUhuWVLGrWgLHRuM(AzTsxYcYjIEJsGQbeYHm) , 
CTCbzQMDaLxINPbODdng(ClEhcJFlvGCgiavziIag) , 
CLOUhPjbgggWoXHTtmjC(CZNYmBcNFSZWvaCklqeM) , 
CpMjJjtGOeWOzwxpAAQP(CufLFODQDXTAPyRqYodN) , 
CqtPRJvHlGJFssiPzsOm(DCdZnqpcBnvXVgEsLBnz) , 
FpguQZSermqZCMRiUfML(FrpxvsDMwwgbpqHXWxmN) , 
FshVHIJMRAhtQirYPlZd(FwLnexHgxHRquTKmNpoa) , 
IIiwAtyrOtLzLWAUlTey((CTCbzQMDaLxINPbODdng != CLOUhPjbgggWoXHTtmjC) 
|| (CpMjJjtGOeWOzwxpAAQP != CqtPRJvHlGJFssiPzsOm)) { dJcdBfQQLhIAYHPxwQeg = 
ntwk_impl; CUDNN_CALL(hipdnnCreateConvolutionDescriptor(&NNhshzQGJHLSGjDiVerE)); 
CUDNN_CALL(hipdnnCreateFilterDescriptor(&QVgVGfoCXYiYXzPhvVPX)); 
CUDNN_CALL(hipdnnCreateTensorDescriptor(&JgLfgHrHMEMmMYTettJF));  MWConvLayer* 
convLayer = static_cast<MWConvLayer*>(getLayer()); MWTensorBase* ipTensor = 
convLayer->getInputTensor(0); if (IIiwAtyrOtLzLWAUlTey) { 
TaAJDyqFVJXfAfCJhOuU = new MWTensor<float>(-1, -1, -1, -1, -1, NULL, getLayer(), 0); 
if (!TaAJDyqFVJXfAfCJhOuU) { MWCNNLayerImpl::throwAllocationError(__LINE__ , 
__FILE__); } CUDNN_CALL(hipdnnCreateTensorDescriptor(&XYbzSmRQGatVJtGmDZSo)); } 
else { TaAJDyqFVJXfAfCJhOuU = ipTensor; } assert(TaAJDyqFVJXfAfCJhOuU != NULL); int 
NXruhrCCiguRjAgSNDuz; int NZjOkZPwLzQsdEVkwMcX; if 
(IIiwAtyrOtLzLWAUlTey) { NXruhrCCiguRjAgSNDuz = 0;  
NZjOkZPwLzQsdEVkwMcX = 0; } else { NXruhrCCiguRjAgSNDuz = 
CTCbzQMDaLxINPbODdng; NZjOkZPwLzQsdEVkwMcX = CpMjJjtGOeWOzwxpAAQP; } 
bUVPfnrJhLfHzOLUUrKk = CTCbzQMDaLxINPbODdng; cCXqPFPPcoHzYMDpnUxQ = CpMjJjtGOeWOzwxpAAQP;
#if (CUDNN_MAJOR <= 5)
 { if ((AdmgfUbRAfzFeYHxSnQr != 1) && (AwZQzUhuWVLGrWgLHRuM != 1)){ 
printf("Dilated Convolution only supported for cuDNN 6 or greater "); throw 
std::runtime_error("Unsupported Dilation Factor"); } 
CUDNN_CALL(hipdnnSetConvolution2dDescriptor(NNhshzQGJHLSGjDiVerE, 
NXruhrCCiguRjAgSNDuz, NZjOkZPwLzQsdEVkwMcX, FpguQZSermqZCMRiUfML, 
FshVHIJMRAhtQirYPlZd, 1, 1, HIPDNN_CROSS_CORRELATION));  }
#else
 { CUDNN_CALL(hipdnnSetConvolution2dDescriptor(NNhshzQGJHLSGjDiVerE, 
NXruhrCCiguRjAgSNDuz, NZjOkZPwLzQsdEVkwMcX, FpguQZSermqZCMRiUfML, 
FshVHIJMRAhtQirYPlZd, AdmgfUbRAfzFeYHxSnQr, AwZQzUhuWVLGrWgLHRuM, 
HIPDNN_CROSS_CORRELATION, HIPDNN_DATA_FLOAT)); }
#endif
#if (FP16_ENABLED == 1 && ( CUDNN_MAJOR > 7 || (CUDNN_MAJOR == 7 && CUDNN_MINOR >= 2) ))
 CUDNN_CALL(hipdnnSetConvolutionMathType(NNhshzQGJHLSGjDiVerE, CUDNN_TENSOR_OP_MATH_ALLOW_CONVERSION));
#endif
 if (BuyZFXzwOMxcePIbCLfl > 1){ 
CUDNN_CALL(hipdnnSetConvolutionGroupCount(NNhshzQGJHLSGjDiVerE, 
BuyZFXzwOMxcePIbCLfl)); } int etjQLJVQCaeAXRWYtqOl = 
BkwhtPQUCQKchmmimoXs*BuyZFXzwOMxcePIbCLfl; int fSKMHAqIghbYYgyIpNDw = 
BlRIQPyqJZORKENzSdYf*BuyZFXzwOMxcePIbCLfl; 
CUDNN_CALL(hipdnnSetFilter4dDescriptor(QVgVGfoCXYiYXzPhvVPX, HIPDNN_DATA_FLOAT, 
HIPDNN_TENSOR_NCHW, fSKMHAqIghbYYgyIpNDw, 
etjQLJVQCaeAXRWYtqOl/BuyZFXzwOMxcePIbCLfl, BHuHNDGoRwGRouCxeMbw, 
BLjrjqvCcCommiXWQLjs)); 
CUDNN_CALL(hipdnnSetTensor4dDescriptor(JgLfgHrHMEMmMYTettJF, HIPDNN_TENSOR_NCHW, 
HIPDNN_DATA_FLOAT, 1, fSKMHAqIghbYYgyIpNDw, 1, 1)); int weightSize = 
BkwhtPQUCQKchmmimoXs*fSKMHAqIghbYYgyIpNDw*BHuHNDGoRwGRouCxeMbw*BLjrjqvCcCommiXWQLjs; 
CUDA_CALL(hipMalloc((void**)&vpXxoeEhdEosLSsYXkNG, sizeof(float)*weightSize)); 
CUDA_CALL(hipMalloc((void**)&IwKnaBoXVubIRYcxEJLH, 
sizeof(float)*fSKMHAqIghbYYgyIpNDw)); 
loadWeights(xHViLEwTujGGrPZZgmbF); loadBias(JwxFdqOKggeawILBfGgg); 
createAndAddDescriptor(getLayer()->getOutputTensor()->getSourcePortIndex()); } 
void MWConvLayerImpl::propagateSize() { MWTensorBase* ipTensor = 
getLayer()->getInputTensor(0); int inputH; int inputW; if 
(IIiwAtyrOtLzLWAUlTey) { inputH = ipTensor->getHeight() + 
CTCbzQMDaLxINPbODdng + CLOUhPjbgggWoXHTtmjC; inputW = ipTensor->getWidth() + 
CpMjJjtGOeWOzwxpAAQP + CqtPRJvHlGJFssiPzsOm; } else { inputH = 
ipTensor->getHeight(); inputW = ipTensor->getWidth(); } 
TaAJDyqFVJXfAfCJhOuU->setHeight(inputH); TaAJDyqFVJXfAfCJhOuU->setWidth(inputW); 
TaAJDyqFVJXfAfCJhOuU->setChannels(ipTensor->getChannels()); 
TaAJDyqFVJXfAfCJhOuU->setBatchSize(ipTensor->getBatchSize()); 
TaAJDyqFVJXfAfCJhOuU->setSequenceLength(ipTensor->getSequenceLength()); 
assert(TaAJDyqFVJXfAfCJhOuU->getSequenceLength() == 1); if 
(IIiwAtyrOtLzLWAUlTey) { 
CUDNN_CALL(hipdnnSetTensor4dDescriptor(XYbzSmRQGatVJtGmDZSo, HIPDNN_TENSOR_NCHW, 
HIPDNN_DATA_FLOAT, TaAJDyqFVJXfAfCJhOuU->getBatchSize(), TaAJDyqFVJXfAfCJhOuU->getChannels(), 
TaAJDyqFVJXfAfCJhOuU->getHeight(), TaAJDyqFVJXfAfCJhOuU->getWidth())); } else { 
XYbzSmRQGatVJtGmDZSo = MWCNNLayerImpl::getCuDNNDescriptor(TaAJDyqFVJXfAfCJhOuU); } 
MWTensorBase* opTensor = getLayer()->getOutputTensor(0); 
hipdnnTensorDescriptor_t* desc = getDescriptor(opTensor->getSourcePortIndex()); 
assert(desc); setDescriptor<float>(*desc, static_cast<MWTensor<float>*>(opTensor));
#if (CUDNN_MAJOR < 7)
 { 
CUDNN_CALL(hipdnnGetConvolutionForwardAlgorithm(*dJcdBfQQLhIAYHPxwQeg->getCudnnHandle(), 
XYbzSmRQGatVJtGmDZSo, QVgVGfoCXYiYXzPhvVPX, NNhshzQGJHLSGjDiVerE, *desc, 
HIPDNN_CONVOLUTION_FWD_PREFER_FASTEST, 0, &NDjzAZSYJuWymuKDNZYB)); }
#else
 { const int maxAlgoCount(3); int returnedAlgoCount(-1);  
hipdnnConvolutionFwdAlgoPerf_t perf_results[maxAlgoCount]; 
CUDNN_CALL(cudnnGetConvolutionForwardAlgorithm_v7(*dJcdBfQQLhIAYHPxwQeg->getCudnnHandle(), 
XYbzSmRQGatVJtGmDZSo, QVgVGfoCXYiYXzPhvVPX, NNhshzQGJHLSGjDiVerE, *desc, 
maxAlgoCount, &returnedAlgoCount, perf_results)); NDjzAZSYJuWymuKDNZYB = 
perf_results[0].algo; }
#endif
 size_t tnTPxeDjBsqLAPkJcPJX = 0; 
CUDNN_CALL(hipdnnGetConvolutionForwardWorkspaceSize( 
*dJcdBfQQLhIAYHPxwQeg->getCudnnHandle(), XYbzSmRQGatVJtGmDZSo, 
QVgVGfoCXYiYXzPhvVPX, NNhshzQGJHLSGjDiVerE, *desc, NDjzAZSYJuWymuKDNZYB, 
&tnTPxeDjBsqLAPkJcPJX)); if (tnTPxeDjBsqLAPkJcPJX > 
*dJcdBfQQLhIAYHPxwQeg->getProposedWorkSpaceSize()) { 
dJcdBfQQLhIAYHPxwQeg->setProposedWorkSpaceSize(tnTPxeDjBsqLAPkJcPJX); } } void 
MWConvLayerImpl::allocate() { MWTensorBase* ipTensor = 
getLayer()->getInputTensor(0); if (IIiwAtyrOtLzLWAUlTey) { float* 
newInput; int inputH = ipTensor->getHeight() + CTCbzQMDaLxINPbODdng + 
CLOUhPjbgggWoXHTtmjC; int inputW = ipTensor->getWidth() + 
CpMjJjtGOeWOzwxpAAQP + CqtPRJvHlGJFssiPzsOm; int paddedSize = 
ipTensor->getBatchSize() * ipTensor->getChannels() * inputH * inputW; 
CUDA_CALL(hipMalloc((void**)&newInput, sizeof(float)*paddedSize)); 
CUDA_CALL(hipMemset(newInput, 0, sizeof(float)*paddedSize)); 
static_cast<MWTensor<float>*>(TaAJDyqFVJXfAfCJhOuU)->setData(newInput); } } void 
MWConvLayerImpl::deallocate() { if (TaAJDyqFVJXfAfCJhOuU != 
getLayer()->getInputTensor(0)) { assert(IIiwAtyrOtLzLWAUlTey); 
CUDA_FREE_CALL(static_cast<MWTensor<float>*>(TaAJDyqFVJXfAfCJhOuU)->getData()); 
static_cast<MWTensor<float>*>(TaAJDyqFVJXfAfCJhOuU)->setData((float*)NULL); } } void 
MWConvLayerImpl::predict() { MWConvLayer* convLayer = 
static_cast<MWConvLayer*>(getLayer()); MWTensorBase* ipTensorBase = 
convLayer->getInputTensor(); MWTensorBase* opTensorBase = 
convLayer->getOutputTensor(); MWTensor<float>* ipTensor = 
static_cast<MWTensor<float>*>(ipTensorBase); MWTensor<float>* opTensor = 
static_cast<MWTensor<float>*>(opTensorBase); if (TaAJDyqFVJXfAfCJhOuU != 
convLayer->getInputTensor()) { 
CUDA_CALL(hipMemset(static_cast<MWTensor<float>*>(TaAJDyqFVJXfAfCJhOuU)->getData(), 
0, sizeof(float)*TaAJDyqFVJXfAfCJhOuU->getNumElements()));  
MWCNNLayerImpl::padInput(ipTensor->getData(), ipTensor->getHeight(), 
ipTensor->getWidth(), ipTensor->getChannels(), TaAJDyqFVJXfAfCJhOuU->getHeight(), 
TaAJDyqFVJXfAfCJhOuU->getWidth(), bUVPfnrJhLfHzOLUUrKk, cCXqPFPPcoHzYMDpnUxQ, 
static_cast<MWTensor<float>*>(TaAJDyqFVJXfAfCJhOuU)->getData(), 
ipTensor->getNumElements()); } assert(opTensor->getData() != 
static_cast<MWTensor<float>*>(TaAJDyqFVJXfAfCJhOuU)->getData()); 
hipdnnTensorDescriptor_t* desc = getDescriptor(opTensor->getSourcePortIndex()); 
assert(desc); 
CUDNN_CALL(hipdnnConvolutionForward(*dJcdBfQQLhIAYHPxwQeg->getCudnnHandle(), 
getOnePtr(), XYbzSmRQGatVJtGmDZSo, 
static_cast<MWTensor<float>*>(TaAJDyqFVJXfAfCJhOuU)->getData(), QVgVGfoCXYiYXzPhvVPX, 
vpXxoeEhdEosLSsYXkNG, NNhshzQGJHLSGjDiVerE, NDjzAZSYJuWymuKDNZYB, 
dJcdBfQQLhIAYHPxwQeg->getWorkSpace(), 
*dJcdBfQQLhIAYHPxwQeg->getAllocatedWorkSpaceSize(), getZeroPtr(), *desc, 
opTensor->getData())); 
CUDNN_CALL(hipdnnAddTensor(*dJcdBfQQLhIAYHPxwQeg->getCudnnHandle(), getOnePtr(), 
JgLfgHrHMEMmMYTettJF, IwKnaBoXVubIRYcxEJLH, getOnePtr(), *desc, opTensor->getData())); } 
void MWConvLayerImpl::cleanup() { 
CUDNN_CALL(hipdnnDestroyConvolutionDescriptor(NNhshzQGJHLSGjDiVerE)); 
CUDNN_CALL(hipdnnDestroyFilterDescriptor(QVgVGfoCXYiYXzPhvVPX)); if 
(vpXxoeEhdEosLSsYXkNG) { CUDA_FREE_CALL(vpXxoeEhdEosLSsYXkNG); vpXxoeEhdEosLSsYXkNG = NULL; } 
CUDNN_CALL(hipdnnDestroyTensorDescriptor(JgLfgHrHMEMmMYTettJF)); if 
(IwKnaBoXVubIRYcxEJLH) { CUDA_FREE_CALL(IwKnaBoXVubIRYcxEJLH); IwKnaBoXVubIRYcxEJLH = NULL; } if 
(TaAJDyqFVJXfAfCJhOuU != getLayer()->getInputTensor(0)) { 
assert(IIiwAtyrOtLzLWAUlTey); 
CUDNN_CALL(hipdnnDestroyTensorDescriptor(XYbzSmRQGatVJtGmDZSo)); } } void 
MWConvLayerImpl::loadWeights(const char* QMgBqCuvjnbWHWiVPEwn) { MWConvLayer* 
convLayer = static_cast<MWConvLayer*>(getLayer()); FILE* QhTesEEIHwhNmHSeYbRR = 
MWCNNLayer::openBinaryFile(QMgBqCuvjnbWHWiVPEwn); assert(QhTesEEIHwhNmHSeYbRR); int 
dMxIKDGTITyhdLqIHBLA = 
BkwhtPQUCQKchmmimoXs*BlRIQPyqJZORKENzSdYf*BuyZFXzwOMxcePIbCLfl*BHuHNDGoRwGRouCxeMbw*BLjrjqvCcCommiXWQLjs; 
 float* KZWeXiYFmdpQdsgidKeG = MALLOC_CALL(sizeof(float)*dMxIKDGTITyhdLqIHBLA); 
call_fread(KZWeXiYFmdpQdsgidKeG, sizeof(float), dMxIKDGTITyhdLqIHBLA, QhTesEEIHwhNmHSeYbRR, 
QMgBqCuvjnbWHWiVPEwn); CUDA_CALL(hipMemcpy(vpXxoeEhdEosLSsYXkNG, KZWeXiYFmdpQdsgidKeG, 
sizeof(float)*dMxIKDGTITyhdLqIHBLA, hipMemcpyHostToDevice)); fclose(QhTesEEIHwhNmHSeYbRR); 
free(KZWeXiYFmdpQdsgidKeG); } void MWConvLayerImpl::loadBias(const char* 
QMgBqCuvjnbWHWiVPEwn) { MWConvLayer* convLayer = 
static_cast<MWConvLayer*>(getLayer()); FILE* QhTesEEIHwhNmHSeYbRR = 
MWCNNLayer::openBinaryFile(QMgBqCuvjnbWHWiVPEwn);  assert(QhTesEEIHwhNmHSeYbRR); int 
dMxIKDGTITyhdLqIHBLA = BlRIQPyqJZORKENzSdYf*BuyZFXzwOMxcePIbCLfl;  float* 
KZWeXiYFmdpQdsgidKeG = MALLOC_CALL(sizeof(float)*dMxIKDGTITyhdLqIHBLA); 
call_fread(KZWeXiYFmdpQdsgidKeG, sizeof(float), dMxIKDGTITyhdLqIHBLA, QhTesEEIHwhNmHSeYbRR, 
QMgBqCuvjnbWHWiVPEwn); CUDA_CALL(hipMemcpy(IwKnaBoXVubIRYcxEJLH, KZWeXiYFmdpQdsgidKeG, 
sizeof(float)*dMxIKDGTITyhdLqIHBLA, hipMemcpyHostToDevice)); 
free(KZWeXiYFmdpQdsgidKeG); fclose(QhTesEEIHwhNmHSeYbRR); } void 
MWConvLayerImpl::postSetup() { if (dJcdBfQQLhIAYHPxwQeg->getAutoTune()) { 
getConvAlgoTuned(); } else { getConvAlgoWorkSpaceLimit(); } } void 
MWConvLayerImpl::getConvAlgoTuned() { MWTensorBase* opTensorBase = 
getLayer()->getOutputTensor(0); MWTensor<float>* opTensor = 
static_cast<MWTensor<float>*>(opTensorBase); hipdnnConvolutionFwdAlgoPerf_t 
perf_results[3]; hipdnnTensorDescriptor_t* desc = 
getDescriptor(getLayer()->getOutputTensor()->getSourcePortIndex()); 
assert(desc); int returnedAlgoCount; 
CUDNN_CALL(hipdnnFindConvolutionForwardAlgorithmEx(*dJcdBfQQLhIAYHPxwQeg->getCudnnHandle(), 
XYbzSmRQGatVJtGmDZSo, static_cast<MWTensor<float>*>(TaAJDyqFVJXfAfCJhOuU)->getData(), 
QVgVGfoCXYiYXzPhvVPX, vpXxoeEhdEosLSsYXkNG, NNhshzQGJHLSGjDiVerE, *desc, 
opTensor->getData(), 3, &returnedAlgoCount, &perf_results[0], 
dJcdBfQQLhIAYHPxwQeg->getWorkSpace(), 
*dJcdBfQQLhIAYHPxwQeg->getAllocatedWorkSpaceSize())); NDjzAZSYJuWymuKDNZYB = 
perf_results[0].algo; } void MWConvLayerImpl::getConvAlgoWorkSpaceLimit() { 
hipdnnTensorDescriptor_t* desc = 
getDescriptor(getLayer()->getOutputTensor()->getSourcePortIndex()); assert(desc);
#if (CUDNN_MAJOR < 8)
 
CUDNN_CALL(hipdnnGetConvolutionForwardAlgorithm(*dJcdBfQQLhIAYHPxwQeg->getCudnnHandle(), 
XYbzSmRQGatVJtGmDZSo, QVgVGfoCXYiYXzPhvVPX, NNhshzQGJHLSGjDiVerE, *desc, 
HIPDNN_CONVOLUTION_FWD_SPECIFY_WORKSPACE_LIMIT, 
*dJcdBfQQLhIAYHPxwQeg->getAllocatedWorkSpaceSize(), &NDjzAZSYJuWymuKDNZYB));
#else
 int maxAlgoCount(-1); 
CUDNN_CALL(cudnnGetConvolutionForwardAlgorithmMaxCount(*dJcdBfQQLhIAYHPxwQeg->getCudnnHandle(), 
&maxAlgoCount)); int returnedAlgoCount(-1); 
std::vector<hipdnnConvolutionFwdAlgoPerf_t> perf_results(maxAlgoCount);  
CUDNN_CALL(cudnnGetConvolutionForwardAlgorithm_v7(*dJcdBfQQLhIAYHPxwQeg->getCudnnHandle(), 
XYbzSmRQGatVJtGmDZSo, QVgVGfoCXYiYXzPhvVPX, NNhshzQGJHLSGjDiVerE, *desc, 
maxAlgoCount, &returnedAlgoCount, &perf_results[0])); 
hipdnnConvolutionFwdAlgoPerf_t nextFastest; bool algoFound(false); for (int i = 
0; i < returnedAlgoCount; ++i) { nextFastest = perf_results[i]; if 
(nextFastest.memory <= *dJcdBfQQLhIAYHPxwQeg->getAllocatedWorkSpaceSize()) { 
NDjzAZSYJuWymuKDNZYB = nextFastest.algo; algoFound = true; break; } } assert(algoFound);
#endif
 }