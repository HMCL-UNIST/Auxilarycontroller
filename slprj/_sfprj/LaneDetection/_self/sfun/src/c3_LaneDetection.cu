#include "hip/hip_runtime.h"
/* Include files */

#include "LaneDetection_sfun.h"
#include "c3_LaneDetection.h"
#include "MWCudaDimUtility.hpp"
#include "mwmathutil.h"
#ifdef utFree
#undef utFree
#endif

#ifdef utMalloc
#undef utMalloc
#endif

#ifdef __cplusplus

extern "C" void *utMalloc(size_t size);
extern "C" void utFree(void*);

#else

extern void *utMalloc(size_t size);
extern void utFree(void*);

#endif

/* Type Definitions */

/* Named Constants */
const int32_T CALL_EVENT = -1;

/* Variable Declarations */

/* Variable Definitions */
static real_T _sfTime_;
static __device__ real_T c3_gpu_vehicleXPoints[28];
static __device__ real_T c3_gpu_laneCoeffMeans[6];
static __device__ real_T c3_gpu_laneCoeffStds[6];

/* Function Declarations */
static void initialize_c3_LaneDetection(SFc3_LaneDetectionInstanceStruct
  *chartInstance);
static void initialize_params_c3_LaneDetection(SFc3_LaneDetectionInstanceStruct *
  chartInstance);
static void enable_c3_LaneDetection(SFc3_LaneDetectionInstanceStruct
  *chartInstance);
static void disable_c3_LaneDetection(SFc3_LaneDetectionInstanceStruct
  *chartInstance);
static void c3_do_animation_call_c3_LaneDetection
  (SFc3_LaneDetectionInstanceStruct *chartInstance);
static void ext_mode_exec_c3_LaneDetection(SFc3_LaneDetectionInstanceStruct
  *chartInstance);
static const mxArray *get_sim_state_c3_LaneDetection
  (SFc3_LaneDetectionInstanceStruct *chartInstance);
static void set_sim_state_c3_LaneDetection(SFc3_LaneDetectionInstanceStruct
  *chartInstance, const mxArray *c3_st);
static void sf_gateway_c3_LaneDetection(SFc3_LaneDetectionInstanceStruct
  *chartInstance);
static void mdl_start_c3_LaneDetection(SFc3_LaneDetectionInstanceStruct
  *chartInstance);
static void mdl_terminate_c3_LaneDetection(SFc3_LaneDetectionInstanceStruct
  *chartInstance);
static void mdl_setup_runtime_resources_c3_LaneDetection
  (SFc3_LaneDetectionInstanceStruct *chartInstance);
static void mdl_cleanup_runtime_resources_c3_LaneDetection
  (SFc3_LaneDetectionInstanceStruct *chartInstance);
static void initSimStructsc3_LaneDetection(SFc3_LaneDetectionInstanceStruct
  *chartInstance);
static void c3_eML_blk_kernel(SFc3_LaneDetectionInstanceStruct *chartInstance,
  real32_T c3_b_laneNetOut[6], boolean_T *c3_b_laneFound, real32_T c3_b_ltPts[56],
  real32_T c3_b_rtPts[56]);
static boolean_T c3_emlrt_marshallIn(SFc3_LaneDetectionInstanceStruct
  *chartInstance, const mxArray *c3_b_laneFound, const char_T *c3_identifier);
static boolean_T c3_b_emlrt_marshallIn(SFc3_LaneDetectionInstanceStruct
  *chartInstance, const mxArray *c3_u, const emlrtMsgIdentifier *c3_parentId);
static void c3_c_emlrt_marshallIn(SFc3_LaneDetectionInstanceStruct
  *chartInstance, const mxArray *c3_b_ltPts, const char_T *c3_identifier,
  real32_T c3_y[56]);
static void c3_d_emlrt_marshallIn(SFc3_LaneDetectionInstanceStruct
  *chartInstance, const mxArray *c3_u, const emlrtMsgIdentifier *c3_parentId,
  real32_T c3_y[56]);
static void c3_e_emlrt_marshallIn(SFc3_LaneDetectionInstanceStruct
  *chartInstance, const mxArray *c3_b_laneCoeffMeans, const char_T
  *c3_identifier, boolean_T *c3_svPtr, real_T c3_y[6]);
static void c3_f_emlrt_marshallIn(SFc3_LaneDetectionInstanceStruct
  *chartInstance, const mxArray *c3_u, const emlrtMsgIdentifier *c3_parentId,
  boolean_T *c3_svPtr, real_T c3_y[6]);
static void c3_g_emlrt_marshallIn(SFc3_LaneDetectionInstanceStruct
  *chartInstance, const mxArray *c3_b_vehicleXPoints, const char_T
  *c3_identifier, boolean_T *c3_svPtr, real_T c3_y[28]);
static void c3_h_emlrt_marshallIn(SFc3_LaneDetectionInstanceStruct
  *chartInstance, const mxArray *c3_u, const emlrtMsgIdentifier *c3_parentId,
  boolean_T *c3_svPtr, real_T c3_y[28]);
static uint8_T c3_i_emlrt_marshallIn(SFc3_LaneDetectionInstanceStruct
  *chartInstance, const mxArray *c3_b_is_active_c3_LaneDetection, const char_T
  *c3_identifier);
static uint8_T c3_j_emlrt_marshallIn(SFc3_LaneDetectionInstanceStruct
  *chartInstance, const mxArray *c3_u, const emlrtMsgIdentifier *c3_parentId);
static __global__ void c3_eML_blk_kernel_kernel1(const real_T c3_dv[6]);
static __global__ void c3_eML_blk_kernel_kernel2(const real_T c3_dv1[6]);
static __global__ void c3_eML_blk_kernel_kernel3(const real32_T c3_b_laneNetOut
  [6], real32_T c3_params[6]);
static __global__ void c3_eML_blk_kernel_kernel4();
static __global__ void c3_eML_blk_kernel_kernel5(const real32_T c3_params[6],
  real32_T c3_rt_y[28]);
static __global__ void c3_eML_blk_kernel_kernel6(const real32_T c3_params,
  real32_T c3_rt_y[28]);
static __global__ void c3_eML_blk_kernel_kernel7(const real32_T c3_params[6],
  real32_T c3_lt_y[28]);
static __global__ void c3_eML_blk_kernel_kernel8(const real32_T c3_params,
  real32_T c3_lt_y[28]);
static __global__ void c3_eML_blk_kernel_kernel9(const real_T c3_Tinv[9], real_T
  c3_x[9]);
static __global__ void c3_eML_blk_kernel_kernel10(const real_T c3_Tinv[9],
  real_T c3_x[9]);
static __global__ void c3_eML_blk_kernel_kernel11(const real_T c3_Tinv[9],
  real_T c3_x[9]);
static __global__ void c3_eML_blk_kernel_kernel12(const real_T c3_t3, const
  real_T c3_t2, const real_T c3_x[9], const int32_T c3_p1, real_T c3_Tinv[9]);
static __global__ void c3_eML_blk_kernel_kernel13(const real_T c3_t3, const
  real_T c3_t2, const real_T c3_x[9], const int32_T c3_p2, real_T c3_Tinv[9]);
static __global__ void c3_eML_blk_kernel_kernel14(const real_T c3_t3, const
  real_T c3_t2, const real_T c3_x[9], const int32_T c3_p3, real_T c3_Tinv[9]);
static __global__ void c3_eML_blk_kernel_kernel15(const real32_T c3_lt_y[28],
  real32_T c3_fv[84]);
static __global__ void c3_eML_blk_kernel_kernel16(const real_T c3_Tinv[9], const
  real32_T c3_fv[84], real32_T c3_U[84]);
static __global__ void c3_eML_blk_kernel_kernel17(real32_T c3_U[56], real32_T
  c3_b_U[84], real32_T c3_b[56]);
static __global__ void c3_eML_blk_kernel_kernel18(const real32_T c3_U[84],
  real32_T c3_b_ltPts[56]);
static __global__ void c3_eML_blk_kernel_kernel19(const real_T c3_Tinv[9],
  real_T c3_x[9]);
static __global__ void c3_eML_blk_kernel_kernel20(const real_T c3_Tinv[9],
  real_T c3_x[9]);
static __global__ void c3_eML_blk_kernel_kernel21(const real_T c3_Tinv[9],
  real_T c3_x[9]);
static __global__ void c3_eML_blk_kernel_kernel22(const real_T c3_t3, const
  real_T c3_t2, const int32_T c3_p3, real_T c3_Tinv[9]);
static __global__ void c3_eML_blk_kernel_kernel23(const real32_T c3_rt_y[28],
  real32_T c3_fv1[84]);
static __global__ void c3_eML_blk_kernel_kernel24(const real_T c3_Tinv[9], const
  real32_T c3_fv1[84], real32_T c3_U[84]);
static __global__ void c3_eML_blk_kernel_kernel25(real32_T c3_U[56], real32_T
  c3_b_U[84], real32_T c3_b[56]);
static __global__ void c3_eML_blk_kernel_kernel26(const real32_T c3_U[84],
  real32_T c3_b_rtPts[56]);
static void init_dsm_address_info(SFc3_LaneDetectionInstanceStruct
  *chartInstance);
static void init_simulink_io_address(SFc3_LaneDetectionInstanceStruct
  *chartInstance);

/* Function Definitions */
static void initialize_c3_LaneDetection(SFc3_LaneDetectionInstanceStruct
  *chartInstance)
{
  emlrtLicenseCheckR2012b(chartInstance->c3_fEmlrtCtx,
    "distrib_computing_toolbox", 2);
  emlrtLicenseCheckR2012b(chartInstance->c3_fEmlrtCtx, "image_toolbox", 2);
  sim_mode_is_external(chartInstance->S);
  _sfTime_ = sf_get_time(chartInstance->S);
  chartInstance->c3_laneCoeffMeans_not_empty = false;
  chartInstance->c3_laneCoeffStds_not_empty = false;
  chartInstance->c3_vehicleXPoints_not_empty = false;
  chartInstance->c3_is_active_c3_LaneDetection = 0U;
  hipGetLastError();
  hipMalloc(&chartInstance->c3_gpu_params, 4UL);
  hipMalloc(&chartInstance->c3_gpu_fv, 336UL);
  hipMalloc(&chartInstance->c3_gpu_x, 72UL);
  hipMalloc(&chartInstance->c3_b_gpu_t3, 8UL);
  hipMalloc(&chartInstance->c3_b_gpu_params, 4UL);
  hipMalloc(&chartInstance->c3_gpu_t2, 8UL);
  hipMalloc(&chartInstance->c3_gpu_Tinv, 72UL);
  hipMalloc(&chartInstance->c3_gpu_dv, 48UL);
  hipMalloc(&chartInstance->c3_gpu_ltPts, 224UL);
  hipMalloc(&chartInstance->c3_gpu_dv1, 48UL);
  hipMalloc(&chartInstance->c3_c_gpu_U, 336UL);
  hipMalloc(&chartInstance->c3_gpu_b, 224UL);
  hipMalloc(&chartInstance->c3_c_gpu_params, 24UL);
  hipMalloc(&chartInstance->c3_b_gpu_U, 224UL);
  hipMalloc(&chartInstance->c3_b_gpu_Tinv, 72UL);
  hipMalloc(&chartInstance->c3_gpu_p3, 4UL);
  hipMalloc(&chartInstance->c3_gpu_U, 224UL);
  hipMalloc(&chartInstance->c3_gpu_p1, 4UL);
  hipMalloc(&chartInstance->c3_gpu_laneNetOut, 24UL);
  hipMalloc(&chartInstance->c3_gpu_rtPts, 224UL);
  hipMalloc(&chartInstance->c3_gpu_p2, 4UL);
  hipMalloc(&chartInstance->c3_gpu_t3, 8UL);
  hipMalloc(&chartInstance->c3_b_gpu_t2, 8UL);
  hipMalloc(&chartInstance->c3_gpu_lt_y, 112UL);
  hipMalloc(&chartInstance->c3_gpu_fv1, 336UL);
  hipMalloc(&chartInstance->c3_gpu_rt_y, 112UL);
  hipMalloc(&chartInstance->c3_b_gpu_p3, 4UL);
}

static void initialize_params_c3_LaneDetection(SFc3_LaneDetectionInstanceStruct *
  chartInstance)
{
}

static void enable_c3_LaneDetection(SFc3_LaneDetectionInstanceStruct
  *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
}

static void disable_c3_LaneDetection(SFc3_LaneDetectionInstanceStruct
  *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
}

static void c3_do_animation_call_c3_LaneDetection
  (SFc3_LaneDetectionInstanceStruct *chartInstance)
{
  sfDoAnimationWrapper(chartInstance->S, false, true);
  sfDoAnimationWrapper(chartInstance->S, false, false);
}

static void ext_mode_exec_c3_LaneDetection(SFc3_LaneDetectionInstanceStruct
  *chartInstance)
{
}

static const mxArray *get_sim_state_c3_LaneDetection
  (SFc3_LaneDetectionInstanceStruct *chartInstance)
{
  const mxArray *c3_b_y = NULL;
  const mxArray *c3_c_y = NULL;
  const mxArray *c3_d_y = NULL;
  const mxArray *c3_e_y = NULL;
  const mxArray *c3_f_y = NULL;
  const mxArray *c3_g_y = NULL;
  const mxArray *c3_h_y = NULL;
  const mxArray *c3_st;
  const mxArray *c3_y = NULL;
  c3_st = NULL;
  c3_y = NULL;
  sf_mex_assign(&c3_y, sf_mex_createcellmatrix(7, 1), false);
  c3_b_y = NULL;
  sf_mex_assign(&c3_b_y, sf_mex_create("y", chartInstance->c3_laneFound, 11, 0U,
    0U, 0U, 0), false);
  sf_mex_setcell(c3_y, 0, c3_b_y);
  c3_c_y = NULL;
  sf_mex_assign(&c3_c_y, sf_mex_create("y", *chartInstance->c3_ltPts, 1, 0U, 1U,
    0U, 2, 28, 2), false);
  sf_mex_setcell(c3_y, 1, c3_c_y);
  c3_d_y = NULL;
  sf_mex_assign(&c3_d_y, sf_mex_create("y", *chartInstance->c3_rtPts, 1, 0U, 1U,
    0U, 2, 28, 2), false);
  sf_mex_setcell(c3_y, 2, c3_d_y);
  c3_e_y = NULL;
  if (!chartInstance->c3_laneCoeffMeans_not_empty) {
    sf_mex_assign(&c3_e_y, sf_mex_create("y", NULL, 0, 0U, 1U, 0U, 2, 0, 0),
                  false);
  } else {
    sf_mex_assign(&c3_e_y, sf_mex_create("y", chartInstance->c3_laneCoeffMeans,
      0, 0U, 1U, 0U, 2, 1, 6), false);
  }

  sf_mex_setcell(c3_y, 3, c3_e_y);
  c3_f_y = NULL;
  if (!chartInstance->c3_laneCoeffMeans_not_empty) {
    sf_mex_assign(&c3_f_y, sf_mex_create("y", NULL, 0, 0U, 1U, 0U, 2, 0, 0),
                  false);
  } else {
    sf_mex_assign(&c3_f_y, sf_mex_create("y", chartInstance->c3_laneCoeffStds, 0,
      0U, 1U, 0U, 2, 1, 6), false);
  }

  sf_mex_setcell(c3_y, 4, c3_f_y);
  c3_g_y = NULL;
  if (!chartInstance->c3_vehicleXPoints_not_empty) {
    sf_mex_assign(&c3_g_y, sf_mex_create("y", NULL, 0, 0U, 1U, 0U, 2, 0, 0),
                  false);
  } else {
    sf_mex_assign(&c3_g_y, sf_mex_create("y", chartInstance->c3_vehicleXPoints,
      0, 0U, 1U, 0U, 2, 1, 28), false);
  }

  sf_mex_setcell(c3_y, 5, c3_g_y);
  c3_h_y = NULL;
  sf_mex_assign(&c3_h_y, sf_mex_create("y",
    &chartInstance->c3_is_active_c3_LaneDetection, 3, 0U, 0U, 0U, 0), false);
  sf_mex_setcell(c3_y, 6, c3_h_y);
  sf_mex_assign(&c3_st, c3_y, false);
  return c3_st;
}

static void set_sim_state_c3_LaneDetection(SFc3_LaneDetectionInstanceStruct
  *chartInstance, const mxArray *c3_st)
{
  const mxArray *c3_u;
  c3_u = sf_mex_dup(c3_st);
  *chartInstance->c3_laneFound = c3_emlrt_marshallIn(chartInstance, sf_mex_dup
    (sf_mex_getcell(c3_u, 0)), "laneFound");
  c3_c_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getcell(c3_u, 1)),
                        "ltPts", *chartInstance->c3_ltPts);
  c3_c_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getcell(c3_u, 2)),
                        "rtPts", *chartInstance->c3_rtPts);
  c3_e_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getcell(c3_u, 3)),
                        "laneCoeffMeans",
                        &chartInstance->c3_laneCoeffMeans_not_empty,
                        chartInstance->c3_laneCoeffMeans);
  c3_e_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getcell(c3_u, 4)),
                        "laneCoeffStds",
                        &chartInstance->c3_laneCoeffStds_not_empty,
                        chartInstance->c3_laneCoeffStds);
  c3_g_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getcell(c3_u, 5)),
                        "vehicleXPoints",
                        &chartInstance->c3_vehicleXPoints_not_empty,
                        chartInstance->c3_vehicleXPoints);
  chartInstance->c3_is_active_c3_LaneDetection = c3_i_emlrt_marshallIn
    (chartInstance, sf_mex_dup(sf_mex_getcell(c3_u, 6)),
     "is_active_c3_LaneDetection");
  sf_mex_destroy(&c3_u);
  sf_mex_destroy(&c3_st);
}

static void sf_gateway_c3_LaneDetection(SFc3_LaneDetectionInstanceStruct
  *chartInstance)
{
  int32_T c3_i;
  int32_T c3_i1;
  int32_T c3_i2;
  real32_T c3_fv1[56];
  real32_T c3_fv2[56];
  real32_T c3_fv[6];
  boolean_T c3_b;
  chartInstance->c3_JITTransitionAnimation[0] = 0U;
  _sfTime_ = sf_get_time(chartInstance->S);
  for (c3_i = 0; c3_i < 6; c3_i++) {
    c3_fv[c3_i] = (*chartInstance->c3_laneNetOut)[c3_i];
  }

  c3_eML_blk_kernel(chartInstance, c3_fv, &c3_b, c3_fv1, c3_fv2);
  for (c3_i1 = 0; c3_i1 < 56; c3_i1++) {
    (*chartInstance->c3_rtPts)[c3_i1] = c3_fv2[c3_i1];
  }

  for (c3_i2 = 0; c3_i2 < 56; c3_i2++) {
    (*chartInstance->c3_ltPts)[c3_i2] = c3_fv1[c3_i2];
  }

  *chartInstance->c3_laneFound = c3_b;
  c3_do_animation_call_c3_LaneDetection(chartInstance);
}

static void mdl_start_c3_LaneDetection(SFc3_LaneDetectionInstanceStruct
  *chartInstance)
{
}

static void mdl_terminate_c3_LaneDetection(SFc3_LaneDetectionInstanceStruct
  *chartInstance)
{
  hipError_t c3_errCode;
  hipFree(*chartInstance->c3_gpu_dv);
  hipFree(*chartInstance->c3_gpu_U);
  hipFree(*chartInstance->c3_gpu_Tinv);
  hipFree(*chartInstance->c3_gpu_dv1);
  hipFree(*chartInstance->c3_gpu_lt_y);
  hipFree(*chartInstance->c3_gpu_ltPts);
  hipFree(chartInstance->c3_gpu_t3);
  hipFree(*chartInstance->c3_gpu_fv);
  hipFree(chartInstance->c3_b_gpu_t2);
  hipFree(chartInstance->c3_b_gpu_t3);
  hipFree(chartInstance->c3_gpu_p3);
  hipFree(*chartInstance->c3_gpu_x);
  hipFree(chartInstance->c3_b_gpu_params);
  hipFree(chartInstance->c3_gpu_params);
  hipFree(chartInstance->c3_gpu_t2);
  hipFree(chartInstance->c3_gpu_p1);
  hipFree(*chartInstance->c3_gpu_rt_y);
  hipFree(chartInstance->c3_b_gpu_p3);
  hipFree(*chartInstance->c3_c_gpu_params);
  hipFree(*chartInstance->c3_c_gpu_U);
  hipFree(*chartInstance->c3_b_gpu_Tinv);
  hipFree(*chartInstance->c3_gpu_rtPts);
  hipFree(*chartInstance->c3_gpu_fv1);
  hipFree(chartInstance->c3_gpu_p2);
  hipFree(*chartInstance->c3_gpu_laneNetOut);
  hipFree(*chartInstance->c3_gpu_b);
  hipFree(*chartInstance->c3_b_gpu_U);
  c3_errCode = hipGetLastError();
  if (c3_errCode != hipSuccess) {
    emlrtThinCUDAError(c3_errCode, hipGetErrorName(c3_errCode),
                       hipGetErrorString(c3_errCode), "SimGPUErrorChecks",
                       chartInstance->c3_fEmlrtCtx);
  }
}

static void mdl_setup_runtime_resources_c3_LaneDetection
  (SFc3_LaneDetectionInstanceStruct *chartInstance)
{
  setLegacyDebuggerFlag(chartInstance->S, false);
  setDebuggerFlag(chartInstance->S, false);
  sim_mode_is_external(chartInstance->S);
}

static void mdl_cleanup_runtime_resources_c3_LaneDetection
  (SFc3_LaneDetectionInstanceStruct *chartInstance)
{
}

static void initSimStructsc3_LaneDetection(SFc3_LaneDetectionInstanceStruct
  *chartInstance)
{
}

const mxArray *sf_c3_LaneDetection_get_eml_resolved_functions_info()
{
  const mxArray *c3_nameCaptureInfo = NULL;
  const char_T *c3_data[9] = {
    "789ced5a5d6fd25018ee0c331ae336135de22ebcf62319730c36ae0c857543c6c6e4633063a0b487b65bdb036df9f2c65d7aa55e68a2ffc04ba33f40fd27fe07"
    "fd0152a083568fdd2c1ea4f424e4f0e6a1e779fabe3d4fce399498892767088298eb7cde3d23880f61a2dbaef63a62a1df5f20cccd8acff4fb2b96d868b384cf",
    "749d8ebfeef0bdecc70c9435d0d27a814c4be0f44a164a824ccb5aa65d05840254283600db452a8208328204d2c3c1ae1e49d410741ae890fe3dca03e6385d97"
    "088557070ac5e1a09b0fbd9510f7ebb3c987b559f361fd9dc177f2977cc6f84b367c062ed23228b240038c2640b9c840a8b09d346b4035eb2939d47311a9a787",
    "a882cc8960c0f7d621df1a92cf8c3fde7ce2e7a104fcbcc488fe484c7ffe1428fad1795996f4ebecf2327f469dd67ef0fb4bddfee69b1b659c7c5f1e7c5dc3c9"
    "67b471f1b510e39df5395b44f02d58702a9bdc0f54ca91607d23980e366118c4f81c39d091b2e1b1d34120625ce34fca7c755aefeb363a0d9c8152b5ae0112d6",
    "659656da49c802715847c9a18ef3fae92b877ccb483e338eaacfeff2a15706d7bcfff8d957d4fb69f135dc7cb87cf4612bb192a5225a41dd6532895aabcaa4c9"
    "fcb6e7a3b87db4ea50272a2f0b969e035a51ab4045cac0b84473605ceb71a7fe79d786cfc05175b1e6a1570d7cf3fdfdb745acebd0e6526d16279fd1dcee9f01",
    "21a4b6ea548ae657e95a4838ca6f516a36e61efffcdfe7a9d33a5fb3d167e09a42cbaa48eb269eeb78395408b38e92431d76eb4f16d6cba35c7fde43f2997154"
    "5d7ec947bf30b8e6fb77ccfef9e247e5324e3ea3b9dd3fe307692e7a44c7e285687b434db2f7b983ed38e5f9e7b8e7e959f5f92cf1405f0f51a076384abef3fa",
    "e48943be5b483e338ecabf7eff38f7e59fbc7df93fe5c3e58b3b3bb2120ba99b646d25d3e077727c424a673d5f74932fe647c93781be98f77cd13d7cb87c917b"
    "ba4785b29140392529d9649bd94ab5c230eaf922aefdb6d373ca394b6cd567e01d7fe89a7392d614a1353e9f7cee90ef3692cf8cffc12787f2b02c797ee90e3e",
    "5c7eb92a938904fda8093699507d6d2f1c09e6a2828bfedf99b4f939c2f56341efa778fd58f0d68feee1c3b67ecc372a5c9ae49a3149aba8d461637d7f25efa2"
    "f786a6d80fbd7346cf0f5dc3e79d338e66fc29f643ef7cd1f343d7f079e78ba319dfede78bf396d8aacfc0858e1c415605a67fc038a9eff1dc41f29971543d2c",
    "7930cae1f9a64bf870f9e69e466e45c934bf2f05e4f5c8ea7128c208d005fbea9f237e868d",
    "" };

  c3_nameCaptureInfo = NULL;
  emlrtNameCaptureMxArrayR2016a(&c3_data[0], 14792U, &c3_nameCaptureInfo);
  return c3_nameCaptureInfo;
}

static void c3_eML_blk_kernel(SFc3_LaneDetectionInstanceStruct *chartInstance,
  real32_T c3_b_laneNetOut[6], boolean_T *c3_b_laneFound, real32_T c3_b_ltPts[56],
  real32_T c3_b_rtPts[56])
{
  static real_T c3_dv[6] = { -0.0002, 0.0002, 1.474, -0.0002, 0.0045, -1.3787 };

  static real_T c3_dv1[6] = { 0.003, 0.0766, 0.6313, 0.0026, 0.0736, 0.9846 };

  real_T c3_Tinv[9];
  real_T c3_x[9];
  real_T c3_absx11;
  real_T c3_absx21;
  real_T c3_absx31;
  real_T c3_b_absx11;
  real_T c3_b_absx21;
  real_T c3_b_absx31;
  real_T c3_b_t1;
  real_T c3_b_t2;
  real_T c3_b_t3;
  real_T c3_b_z;
  real_T c3_c_z;
  real_T c3_d_z;
  real_T c3_e_z;
  real_T c3_f_z;
  real_T c3_t1;
  real_T c3_t2;
  real_T c3_t3;
  real_T c3_z;
  int32_T c3_b_itmp;
  int32_T c3_b_k;
  int32_T c3_b_p1;
  int32_T c3_b_p2;
  int32_T c3_b_p3;
  int32_T c3_itmp;
  int32_T c3_k;
  int32_T c3_p1;
  int32_T c3_p2;
  int32_T c3_p3;
  real32_T c3_params[6];
  boolean_T c3_Tinv_dirtyOnCpu;
  boolean_T c3_c_laneFound;
  boolean_T c3_ltPts_dirtyOnGpu;
  boolean_T c3_params_dirtyOnGpu;
  boolean_T c3_rtPts_dirtyOnGpu;
  boolean_T c3_x_dirtyOnCpu;
  c3_rtPts_dirtyOnGpu = false;
  c3_ltPts_dirtyOnGpu = false;
  if (!chartInstance->c3_laneCoeffMeans_not_empty) {
    hipMemcpy(chartInstance->c3_gpu_dv, &c3_dv[0], 48UL, hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(c3_gpu_laneCoeffMeans), chartInstance->c3_laneCoeffMeans,
                       48UL, 0UL, hipMemcpyHostToDevice);
    c3_eML_blk_kernel_kernel1<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
      (*chartInstance->c3_gpu_dv);
    hipMemcpyFromSymbol(chartInstance->c3_laneCoeffMeans, HIP_SYMBOL(c3_gpu_laneCoeffMeans),
                         48UL, 0UL, hipMemcpyDeviceToHost);
    chartInstance->c3_laneCoeffMeans_not_empty = true;
  }

  if (!chartInstance->c3_laneCoeffStds_not_empty) {
    hipMemcpy(chartInstance->c3_gpu_dv1, &c3_dv1[0], 48UL,
               hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(c3_gpu_laneCoeffStds), chartInstance->c3_laneCoeffStds,
                       48UL, 0UL, hipMemcpyHostToDevice);
    c3_eML_blk_kernel_kernel2<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
      (*chartInstance->c3_gpu_dv1);
    hipMemcpyFromSymbol(chartInstance->c3_laneCoeffStds, HIP_SYMBOL(c3_gpu_laneCoeffStds),
                         48UL, 0UL, hipMemcpyDeviceToHost);
    chartInstance->c3_laneCoeffStds_not_empty = true;
  }

  hipMemcpy(chartInstance->c3_gpu_laneNetOut, &c3_b_laneNetOut[0], 24UL,
             hipMemcpyHostToDevice);
  hipMemcpyToSymbol(HIP_SYMBOL(c3_gpu_laneCoeffMeans), chartInstance->c3_laneCoeffMeans,
                     48UL, 0UL, hipMemcpyHostToDevice);
  hipMemcpyToSymbol(HIP_SYMBOL(c3_gpu_laneCoeffStds), chartInstance->c3_laneCoeffStds, 48UL,
                     0UL, hipMemcpyHostToDevice);
  c3_eML_blk_kernel_kernel3<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
    (*chartInstance->c3_gpu_laneNetOut, *chartInstance->c3_c_gpu_params);
  if (!chartInstance->c3_vehicleXPoints_not_empty) {
    hipMemcpyToSymbol(HIP_SYMBOL(c3_gpu_vehicleXPoints), chartInstance->c3_vehicleXPoints,
                       224UL, 0UL, hipMemcpyHostToDevice);
    c3_eML_blk_kernel_kernel4<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>();
    hipMemcpyFromSymbol(chartInstance->c3_vehicleXPoints, HIP_SYMBOL(c3_gpu_vehicleXPoints),
                         224UL, 0UL, hipMemcpyDeviceToHost);
    chartInstance->c3_vehicleXPoints_not_empty = true;
  }

  hipMemcpy(&c3_params[0], chartInstance->c3_c_gpu_params, 24UL,
             hipMemcpyDeviceToHost);
  c3_params_dirtyOnGpu = false;
  if (muSingleScalarAbs(c3_params[5]) > 0.5F) {
    if (c3_params_dirtyOnGpu) {
      hipMemcpy(&c3_params[0], chartInstance->c3_c_gpu_params, 24UL,
                 hipMemcpyDeviceToHost);
    }

    if (muSingleScalarAbs(c3_params[2]) > 0.5F) {
      c3_eML_blk_kernel_kernel5<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
        (*chartInstance->c3_c_gpu_params, *chartInstance->c3_gpu_rt_y);
      for (c3_k = 0; c3_k < 2; c3_k++) {
        hipMemcpyToSymbol(HIP_SYMBOL(c3_gpu_vehicleXPoints),
                           chartInstance->c3_vehicleXPoints, 224UL, 0UL,
                           hipMemcpyHostToDevice);
        c3_eML_blk_kernel_kernel6<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
          (c3_params[c3_k + 4], *chartInstance->c3_gpu_rt_y);
      }

      c3_eML_blk_kernel_kernel7<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
        (*chartInstance->c3_c_gpu_params, *chartInstance->c3_gpu_lt_y);
      for (c3_b_k = 0; c3_b_k < 2; c3_b_k++) {
        hipMemcpyToSymbol(HIP_SYMBOL(c3_gpu_vehicleXPoints),
                           chartInstance->c3_vehicleXPoints, 224UL, 0UL,
                           hipMemcpyHostToDevice);
        c3_eML_blk_kernel_kernel8<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
          (c3_params[c3_b_k + 1], *chartInstance->c3_gpu_lt_y);
      }

      c3_Tinv[3] = -0.0032316839464807288;
      c3_Tinv[4] = -1.2852132429203174E-19;
      c3_Tinv[5] = 1.0305949982581226;
      c3_Tinv[6] = 1.9916790026632809E-35;
      c3_Tinv[7] = 0.0012931719938928032;
      c3_Tinv[8] = -0.22205377950113064;
      c3_Tinv[0] = -1.9788357004567556E-19;
      c3_Tinv[1] = -0.00070281981464454381;
      c3_Tinv[2] = 1.1512965678044422;
      hipMemcpy(chartInstance->c3_gpu_Tinv, &c3_Tinv[0], 72UL,
                 hipMemcpyHostToDevice);
      c3_Tinv_dirtyOnCpu = false;
      c3_eML_blk_kernel_kernel9<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
        (*chartInstance->c3_gpu_Tinv, *chartInstance->c3_gpu_x);
      c3_p1 = 0;
      c3_p2 = 3;
      c3_p3 = 6;
      c3_absx11 = muDoubleScalarAbs(c3_Tinv[0]);
      c3_absx21 = muDoubleScalarAbs(c3_Tinv[1]);
      c3_absx31 = muDoubleScalarAbs(c3_Tinv[2]);
      if ((c3_absx21 > c3_absx11) && (c3_absx21 > c3_absx31)) {
        c3_p1 = 3;
        c3_p2 = 0;
        if (c3_Tinv_dirtyOnCpu) {
          hipMemcpy(chartInstance->c3_gpu_Tinv, &c3_Tinv[0], 72UL,
                     hipMemcpyHostToDevice);
          c3_Tinv_dirtyOnCpu = false;
        }

        c3_eML_blk_kernel_kernel11<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
          (*chartInstance->c3_gpu_Tinv, *chartInstance->c3_gpu_x);
      } else {
        if (c3_absx31 > c3_absx11) {
          c3_p1 = 6;
          c3_p3 = 0;
          if (c3_Tinv_dirtyOnCpu) {
            hipMemcpy(chartInstance->c3_gpu_Tinv, &c3_Tinv[0], 72UL,
                       hipMemcpyHostToDevice);
            c3_Tinv_dirtyOnCpu = false;
          }

          c3_eML_blk_kernel_kernel10<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
            (*chartInstance->c3_gpu_Tinv, *chartInstance->c3_gpu_x);
        }
      }

      hipMemcpy(&c3_x[0], chartInstance->c3_gpu_x, 72UL, hipMemcpyDeviceToHost);
      c3_z = c3_x[1] / c3_x[0];
      c3_x[1] /= c3_x[0];
      c3_b_z = c3_x[2] / c3_x[0];
      c3_x[2] /= c3_x[0];
      c3_x[4] -= c3_z * c3_x[3];
      c3_x[5] -= c3_b_z * c3_x[3];
      c3_x[7] -= c3_z * c3_x[6];
      c3_x[8] -= c3_b_z * c3_x[6];
      if (muDoubleScalarAbs(c3_x[5]) > muDoubleScalarAbs(c3_x[4])) {
        c3_itmp = c3_p2;
        c3_p2 = c3_p3;
        c3_p3 = c3_itmp;
        c3_x[1] = c3_b_z;
        c3_x[2] = c3_z;
        c3_t1 = c3_x[4];
        c3_x[4] = c3_x[5];
        c3_x[5] = c3_t1;
        c3_t1 = c3_x[7];
        c3_x[7] = c3_x[8];
        c3_x[8] = c3_t1;
      }

      c3_c_z = c3_x[5] / c3_x[4];
      c3_x[5] /= c3_x[4];
      c3_x[8] -= c3_c_z * c3_x[7];
      c3_t3 = (c3_x[5] * c3_x[1] - c3_x[2]) / c3_x[8];
      c3_t2 = -(c3_x[1] + c3_x[7] * c3_t3) / c3_x[4];
      hipMemcpy(chartInstance->c3_gpu_x, &c3_x[0], 72UL, hipMemcpyHostToDevice);
      c3_x_dirtyOnCpu = false;
      c3_eML_blk_kernel_kernel12<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>(c3_t3,
        c3_t2, *chartInstance->c3_gpu_x, c3_p1, *chartInstance->c3_b_gpu_Tinv);
      c3_t3 = -c3_x[5] / c3_x[8];
      c3_t2 = (1.0 - c3_x[7] * c3_t3) / c3_x[4];
      if (c3_x_dirtyOnCpu) {
        hipMemcpy(chartInstance->c3_gpu_x, &c3_x[0], 72UL,
                   hipMemcpyHostToDevice);
      }

      c3_eML_blk_kernel_kernel13<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>(c3_t3,
        c3_t2, *chartInstance->c3_gpu_x, c3_p2, *chartInstance->c3_b_gpu_Tinv);
      c3_t3 = 1.0 / c3_x[8];
      c3_t2 = -c3_x[7] * c3_t3 / c3_x[4];
      c3_eML_blk_kernel_kernel14<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>(c3_t3,
        c3_t2, *chartInstance->c3_gpu_x, c3_p3, *chartInstance->c3_b_gpu_Tinv);
      hipMemcpyToSymbol(HIP_SYMBOL(c3_gpu_vehicleXPoints), chartInstance->c3_vehicleXPoints,
                         224UL, 0UL, hipMemcpyHostToDevice);
      c3_eML_blk_kernel_kernel15<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
        (*chartInstance->c3_gpu_lt_y, *chartInstance->c3_gpu_fv);
      c3_eML_blk_kernel_kernel16<<<dim3(1U, 1U, 1U), dim3(96U, 1U, 1U)>>>
        (*chartInstance->c3_b_gpu_Tinv, *chartInstance->c3_gpu_fv,
         *chartInstance->c3_c_gpu_U);
      c3_eML_blk_kernel_kernel17<<<dim3(1U, 1U, 1U), dim3(64U, 1U, 1U)>>>
        (*chartInstance->c3_gpu_U, *chartInstance->c3_c_gpu_U,
         *chartInstance->c3_gpu_b);
      c3_eML_blk_kernel_kernel18<<<dim3(1U, 1U, 1U), dim3(64U, 1U, 1U)>>>
        (*chartInstance->c3_c_gpu_U, *chartInstance->c3_gpu_ltPts);
      c3_ltPts_dirtyOnGpu = true;
      if (c3_Tinv_dirtyOnCpu) {
        hipMemcpy(chartInstance->c3_gpu_Tinv, &c3_Tinv[0], 72UL,
                   hipMemcpyHostToDevice);
      }

      c3_eML_blk_kernel_kernel19<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
        (*chartInstance->c3_gpu_Tinv, *chartInstance->c3_gpu_x);
      c3_b_p1 = 0;
      c3_b_p2 = 3;
      c3_b_p3 = 6;
      c3_b_absx11 = muDoubleScalarAbs(c3_Tinv[0]);
      c3_b_absx21 = muDoubleScalarAbs(c3_Tinv[1]);
      c3_b_absx31 = muDoubleScalarAbs(c3_Tinv[2]);
      if ((c3_b_absx21 > c3_b_absx11) && (c3_b_absx21 > c3_b_absx31)) {
        c3_b_p1 = 3;
        c3_b_p2 = 0;
        c3_eML_blk_kernel_kernel21<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
          (*chartInstance->c3_gpu_Tinv, *chartInstance->c3_gpu_x);
      } else {
        if (c3_b_absx31 > c3_b_absx11) {
          c3_b_p1 = 6;
          c3_b_p3 = 0;
          c3_eML_blk_kernel_kernel20<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
            (*chartInstance->c3_gpu_Tinv, *chartInstance->c3_gpu_x);
        }
      }

      hipMemcpy(&c3_x[0], chartInstance->c3_gpu_x, 72UL, hipMemcpyDeviceToHost);
      c3_d_z = c3_x[1] / c3_x[0];
      c3_x[1] /= c3_x[0];
      c3_e_z = c3_x[2] / c3_x[0];
      c3_x[2] /= c3_x[0];
      c3_x[4] -= c3_d_z * c3_x[3];
      c3_x[5] -= c3_e_z * c3_x[3];
      c3_x[7] -= c3_d_z * c3_x[6];
      c3_x[8] -= c3_e_z * c3_x[6];
      if (muDoubleScalarAbs(c3_x[5]) > muDoubleScalarAbs(c3_x[4])) {
        c3_b_itmp = c3_b_p2;
        c3_b_p2 = c3_b_p3;
        c3_b_p3 = c3_b_itmp;
        c3_x[1] = c3_e_z;
        c3_x[2] = c3_d_z;
        c3_b_t1 = c3_x[4];
        c3_x[4] = c3_x[5];
        c3_x[5] = c3_b_t1;
        c3_b_t1 = c3_x[7];
        c3_x[7] = c3_x[8];
        c3_x[8] = c3_b_t1;
      }

      c3_f_z = c3_x[5] / c3_x[4];
      c3_x[5] /= c3_x[4];
      c3_x[8] -= c3_f_z * c3_x[7];
      c3_b_t3 = (c3_x[5] * c3_x[1] - c3_x[2]) / c3_x[8];
      c3_b_t2 = -(c3_x[1] + c3_x[7] * c3_b_t3) / c3_x[4];
      c3_Tinv[c3_b_p1] = ((1.0 - c3_x[3] * c3_b_t2) - c3_x[6] * c3_b_t3) / c3_x
        [0];
      c3_Tinv[c3_b_p1 + 1] = c3_b_t2;
      c3_Tinv[c3_b_p1 + 2] = c3_b_t3;
      c3_b_t3 = -c3_x[5] / c3_x[8];
      c3_b_t2 = (1.0 - c3_x[7] * c3_b_t3) / c3_x[4];
      c3_Tinv[c3_b_p2] = -(c3_x[3] * c3_b_t2 + c3_x[6] * c3_b_t3) / c3_x[0];
      c3_Tinv[c3_b_p2 + 1] = c3_b_t2;
      c3_Tinv[c3_b_p2 + 2] = c3_b_t3;
      c3_b_t3 = 1.0 / c3_x[8];
      c3_b_t2 = -c3_x[7] * c3_b_t3 / c3_x[4];
      c3_Tinv[c3_b_p3] = -(c3_x[3] * c3_b_t2 + c3_x[6] * c3_b_t3) / c3_x[0];
      hipMemcpy(chartInstance->c3_gpu_Tinv, &c3_Tinv[0], 72UL,
                 hipMemcpyHostToDevice);
      c3_eML_blk_kernel_kernel22<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
        (c3_b_t3, c3_b_t2, c3_b_p3, *chartInstance->c3_gpu_Tinv);
      hipMemcpyToSymbol(HIP_SYMBOL(c3_gpu_vehicleXPoints), chartInstance->c3_vehicleXPoints,
                         224UL, 0UL, hipMemcpyHostToDevice);
      c3_eML_blk_kernel_kernel23<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
        (*chartInstance->c3_gpu_rt_y, *chartInstance->c3_gpu_fv1);
      c3_eML_blk_kernel_kernel24<<<dim3(1U, 1U, 1U), dim3(96U, 1U, 1U)>>>
        (*chartInstance->c3_gpu_Tinv, *chartInstance->c3_gpu_fv1,
         *chartInstance->c3_c_gpu_U);
      c3_eML_blk_kernel_kernel25<<<dim3(1U, 1U, 1U), dim3(64U, 1U, 1U)>>>
        (*chartInstance->c3_b_gpu_U, *chartInstance->c3_c_gpu_U,
         *chartInstance->c3_gpu_b);
      c3_eML_blk_kernel_kernel26<<<dim3(1U, 1U, 1U), dim3(64U, 1U, 1U)>>>
        (*chartInstance->c3_c_gpu_U, *chartInstance->c3_gpu_rtPts);
      c3_rtPts_dirtyOnGpu = true;
      c3_c_laneFound = true;
    } else {
      c3_c_laneFound = false;
    }
  } else {
    c3_c_laneFound = false;
  }

  *c3_b_laneFound = c3_c_laneFound;
  if (c3_ltPts_dirtyOnGpu) {
    hipMemcpy(&c3_b_ltPts[0], chartInstance->c3_gpu_ltPts, 224UL,
               hipMemcpyDeviceToHost);
  }

  if (c3_rtPts_dirtyOnGpu) {
    hipMemcpy(&c3_b_rtPts[0], chartInstance->c3_gpu_rtPts, 224UL,
               hipMemcpyDeviceToHost);
  }
}

static boolean_T c3_emlrt_marshallIn(SFc3_LaneDetectionInstanceStruct
  *chartInstance, const mxArray *c3_b_laneFound, const char_T *c3_identifier)
{
  emlrtMsgIdentifier c3_thisId;
  boolean_T c3_y;
  c3_thisId.fIdentifier = const_cast<const char_T *>(c3_identifier);
  c3_thisId.fParent = NULL;
  c3_thisId.bParentIsCell = false;
  c3_y = c3_b_emlrt_marshallIn(chartInstance, sf_mex_dup(c3_b_laneFound),
    &c3_thisId);
  sf_mex_destroy(&c3_b_laneFound);
  return c3_y;
}

static boolean_T c3_b_emlrt_marshallIn(SFc3_LaneDetectionInstanceStruct
  *chartInstance, const mxArray *c3_u, const emlrtMsgIdentifier *c3_parentId)
{
  boolean_T c3_b;
  boolean_T c3_y;
  sf_mex_import(c3_parentId, sf_mex_dup(c3_u), &c3_b, 1, 11, 0U, 0, 0U, 0);
  c3_y = c3_b;
  sf_mex_destroy(&c3_u);
  return c3_y;
}

static void c3_c_emlrt_marshallIn(SFc3_LaneDetectionInstanceStruct
  *chartInstance, const mxArray *c3_b_ltPts, const char_T *c3_identifier,
  real32_T c3_y[56])
{
  emlrtMsgIdentifier c3_thisId;
  c3_thisId.fIdentifier = const_cast<const char_T *>(c3_identifier);
  c3_thisId.fParent = NULL;
  c3_thisId.bParentIsCell = false;
  c3_d_emlrt_marshallIn(chartInstance, sf_mex_dup(c3_b_ltPts), &c3_thisId, c3_y);
  sf_mex_destroy(&c3_b_ltPts);
}

static void c3_d_emlrt_marshallIn(SFc3_LaneDetectionInstanceStruct
  *chartInstance, const mxArray *c3_u, const emlrtMsgIdentifier *c3_parentId,
  real32_T c3_y[56])
{
  int32_T c3_i;
  real32_T c3_fv[56];
  sf_mex_import(c3_parentId, sf_mex_dup(c3_u), c3_fv, 0, 1, 0U, 1, 0U, 2, 28, 2);
  for (c3_i = 0; c3_i < 56; c3_i++) {
    c3_y[c3_i] = c3_fv[c3_i];
  }

  sf_mex_destroy(&c3_u);
}

static void c3_e_emlrt_marshallIn(SFc3_LaneDetectionInstanceStruct
  *chartInstance, const mxArray *c3_b_laneCoeffMeans, const char_T
  *c3_identifier, boolean_T *c3_svPtr, real_T c3_y[6])
{
  emlrtMsgIdentifier c3_thisId;
  c3_thisId.fIdentifier = const_cast<const char_T *>(c3_identifier);
  c3_thisId.fParent = NULL;
  c3_thisId.bParentIsCell = false;
  c3_f_emlrt_marshallIn(chartInstance, sf_mex_dup(c3_b_laneCoeffMeans),
                        &c3_thisId, c3_svPtr, c3_y);
  sf_mex_destroy(&c3_b_laneCoeffMeans);
}

static void c3_f_emlrt_marshallIn(SFc3_LaneDetectionInstanceStruct
  *chartInstance, const mxArray *c3_u, const emlrtMsgIdentifier *c3_parentId,
  boolean_T *c3_svPtr, real_T c3_y[6])
{
  real_T c3_dv[6];
  int32_T c3_i;
  if (mxIsEmpty(c3_u)) {
    *c3_svPtr = false;
  } else {
    *c3_svPtr = true;
    sf_mex_import(c3_parentId, sf_mex_dup(c3_u), c3_dv, 1, 0, 0U, 1, 0U, 2, 1, 6);
    for (c3_i = 0; c3_i < 6; c3_i++) {
      c3_y[c3_i] = c3_dv[c3_i];
    }
  }

  sf_mex_destroy(&c3_u);
}

static void c3_g_emlrt_marshallIn(SFc3_LaneDetectionInstanceStruct
  *chartInstance, const mxArray *c3_b_vehicleXPoints, const char_T
  *c3_identifier, boolean_T *c3_svPtr, real_T c3_y[28])
{
  emlrtMsgIdentifier c3_thisId;
  c3_thisId.fIdentifier = const_cast<const char_T *>(c3_identifier);
  c3_thisId.fParent = NULL;
  c3_thisId.bParentIsCell = false;
  c3_h_emlrt_marshallIn(chartInstance, sf_mex_dup(c3_b_vehicleXPoints),
                        &c3_thisId, c3_svPtr, c3_y);
  sf_mex_destroy(&c3_b_vehicleXPoints);
}

static void c3_h_emlrt_marshallIn(SFc3_LaneDetectionInstanceStruct
  *chartInstance, const mxArray *c3_u, const emlrtMsgIdentifier *c3_parentId,
  boolean_T *c3_svPtr, real_T c3_y[28])
{
  real_T c3_dv[28];
  int32_T c3_i;
  if (mxIsEmpty(c3_u)) {
    *c3_svPtr = false;
  } else {
    *c3_svPtr = true;
    sf_mex_import(c3_parentId, sf_mex_dup(c3_u), c3_dv, 1, 0, 0U, 1, 0U, 2, 1,
                  28);
    for (c3_i = 0; c3_i < 28; c3_i++) {
      c3_y[c3_i] = c3_dv[c3_i];
    }
  }

  sf_mex_destroy(&c3_u);
}

static uint8_T c3_i_emlrt_marshallIn(SFc3_LaneDetectionInstanceStruct
  *chartInstance, const mxArray *c3_b_is_active_c3_LaneDetection, const char_T
  *c3_identifier)
{
  emlrtMsgIdentifier c3_thisId;
  uint8_T c3_y;
  c3_thisId.fIdentifier = const_cast<const char_T *>(c3_identifier);
  c3_thisId.fParent = NULL;
  c3_thisId.bParentIsCell = false;
  c3_y = c3_j_emlrt_marshallIn(chartInstance, sf_mex_dup
    (c3_b_is_active_c3_LaneDetection), &c3_thisId);
  sf_mex_destroy(&c3_b_is_active_c3_LaneDetection);
  return c3_y;
}

static uint8_T c3_j_emlrt_marshallIn(SFc3_LaneDetectionInstanceStruct
  *chartInstance, const mxArray *c3_u, const emlrtMsgIdentifier *c3_parentId)
{
  uint8_T c3_b_u;
  uint8_T c3_y;
  sf_mex_import(c3_parentId, sf_mex_dup(c3_u), &c3_b_u, 1, 3, 0U, 0, 0U, 0);
  c3_y = c3_b_u;
  sf_mex_destroy(&c3_u);
  return c3_y;
}

static __global__ __launch_bounds__(32, 1) void c3_eML_blk_kernel_kernel1(const
  real_T c3_dv[6])
{
  int32_T c3_i4;
  c3_i4 = (int32_T)mwGetGlobalThreadIndex();
  if (c3_i4 < 6) {
    c3_gpu_laneCoeffMeans[c3_i4] = c3_dv[c3_i4];
  }
}

static __global__ __launch_bounds__(32, 1) void c3_eML_blk_kernel_kernel2(const
  real_T c3_dv1[6])
{
  int32_T c3_i5;
  c3_i5 = (int32_T)mwGetGlobalThreadIndex();
  if (c3_i5 < 6) {
    c3_gpu_laneCoeffStds[c3_i5] = c3_dv1[c3_i5];
  }
}

static __global__ __launch_bounds__(32, 1) void c3_eML_blk_kernel_kernel3(const
  real32_T c3_b_laneNetOut[6], real32_T c3_params[6])
{
  int32_T c3_i6;
  c3_i6 = (int32_T)mwGetGlobalThreadIndex();
  if (c3_i6 < 6) {
    c3_params[c3_i6] = c3_b_laneNetOut[c3_i6] * (real32_T)
      c3_gpu_laneCoeffStds[c3_i6] + (real32_T)c3_gpu_laneCoeffMeans[c3_i6];
  }
}

static __global__ __launch_bounds__(32, 1) void c3_eML_blk_kernel_kernel4()
{
  int32_T c3_i7;
  c3_i7 = (int32_T)mwGetGlobalThreadIndex();
  if (c3_i7 < 28) {
    c3_gpu_vehicleXPoints[c3_i7] = (real_T)c3_i7 + 3.0;
  }
}

static __global__ __launch_bounds__(32, 1) void c3_eML_blk_kernel_kernel5(const
  real32_T c3_params[6], real32_T c3_rt_y[28])
{
  int32_T c3_i8;
  c3_i8 = (int32_T)mwGetGlobalThreadIndex();
  if (c3_i8 < 28) {
    c3_rt_y[c3_i8] = c3_params[3];
  }
}

static __global__ __launch_bounds__(32, 1) void c3_eML_blk_kernel_kernel6(const
  real32_T c3_params, real32_T c3_rt_y[28])
{
  int32_T c3_i10;
  c3_i10 = (int32_T)mwGetGlobalThreadIndex();
  if (c3_i10 < 28) {
    c3_rt_y[c3_i10] = (real32_T)c3_gpu_vehicleXPoints[c3_i10] * c3_rt_y[c3_i10]
      + c3_params;
  }
}

static __global__ __launch_bounds__(32, 1) void c3_eML_blk_kernel_kernel7(const
  real32_T c3_params[6], real32_T c3_lt_y[28])
{
  int32_T c3_i9;
  c3_i9 = (int32_T)mwGetGlobalThreadIndex();
  if (c3_i9 < 28) {
    c3_lt_y[c3_i9] = c3_params[0];
  }
}

static __global__ __launch_bounds__(32, 1) void c3_eML_blk_kernel_kernel8(const
  real32_T c3_params, real32_T c3_lt_y[28])
{
  int32_T c3_i11;
  c3_i11 = (int32_T)mwGetGlobalThreadIndex();
  if (c3_i11 < 28) {
    c3_lt_y[c3_i11] = (real32_T)c3_gpu_vehicleXPoints[c3_i11] * c3_lt_y[c3_i11]
      + c3_params;
  }
}

static __global__ __launch_bounds__(32, 1) void c3_eML_blk_kernel_kernel9(const
  real_T c3_Tinv[9], real_T c3_x[9])
{
  int32_T c3_i12;
  c3_i12 = (int32_T)mwGetGlobalThreadIndex();
  if (c3_i12 < 9) {
    c3_x[c3_i12] = c3_Tinv[c3_i12];
  }
}

static __global__ __launch_bounds__(32, 1) void c3_eML_blk_kernel_kernel10(const
  real_T c3_Tinv[9], real_T c3_x[9])
{
  int32_T c3_tmpIdx;
  c3_tmpIdx = (int32_T)mwGetGlobalThreadIndex();
  if (c3_tmpIdx < 1) {
    c3_x[0] = c3_Tinv[2];
    c3_x[2] = c3_Tinv[0];
    c3_x[3] = c3_Tinv[5];
    c3_x[5] = c3_Tinv[3];
    c3_x[6] = c3_Tinv[8];
    c3_x[8] = c3_Tinv[6];
  }
}

static __global__ __launch_bounds__(32, 1) void c3_eML_blk_kernel_kernel11(const
  real_T c3_Tinv[9], real_T c3_x[9])
{
  int32_T c3_tmpIdx;
  c3_tmpIdx = (int32_T)mwGetGlobalThreadIndex();
  if (c3_tmpIdx < 1) {
    c3_x[0] = c3_Tinv[1];
    c3_x[1] = c3_Tinv[0];
    c3_x[3] = c3_Tinv[4];
    c3_x[4] = c3_Tinv[3];
    c3_x[6] = c3_Tinv[7];
    c3_x[7] = c3_Tinv[6];
  }
}

static __global__ __launch_bounds__(32, 1) void c3_eML_blk_kernel_kernel12(const
  real_T c3_t3, const real_T c3_t2, const real_T c3_x[9], const int32_T c3_p1,
  real_T c3_Tinv[9])
{
  int32_T c3_tmpIdx;
  c3_tmpIdx = (int32_T)mwGetGlobalThreadIndex();
  if (c3_tmpIdx < 1) {
    c3_Tinv[c3_p1] = ((1.0 - c3_x[3] * c3_t2) - c3_x[6] * c3_t3) / c3_x[0];
    c3_Tinv[c3_p1 + 1] = c3_t2;
    c3_Tinv[c3_p1 + 2] = c3_t3;
  }
}

static __global__ __launch_bounds__(32, 1) void c3_eML_blk_kernel_kernel13(const
  real_T c3_t3, const real_T c3_t2, const real_T c3_x[9], const int32_T c3_p2,
  real_T c3_Tinv[9])
{
  int32_T c3_tmpIdx;
  c3_tmpIdx = (int32_T)mwGetGlobalThreadIndex();
  if (c3_tmpIdx < 1) {
    c3_Tinv[c3_p2] = -(c3_x[3] * c3_t2 + c3_x[6] * c3_t3) / c3_x[0];
    c3_Tinv[c3_p2 + 1] = c3_t2;
    c3_Tinv[c3_p2 + 2] = c3_t3;
  }
}

static __global__ __launch_bounds__(32, 1) void c3_eML_blk_kernel_kernel14(const
  real_T c3_t3, const real_T c3_t2, const real_T c3_x[9], const int32_T c3_p3,
  real_T c3_Tinv[9])
{
  int32_T c3_tmpIdx;
  c3_tmpIdx = (int32_T)mwGetGlobalThreadIndex();
  if (c3_tmpIdx < 1) {
    c3_Tinv[c3_p3] = -(c3_x[3] * c3_t2 + c3_x[6] * c3_t3) / c3_x[0];
    c3_Tinv[c3_p3 + 1] = c3_t2;
    c3_Tinv[c3_p3 + 2] = c3_t3;
  }
}

static __global__ __launch_bounds__(32, 1) void c3_eML_blk_kernel_kernel15(const
  real32_T c3_lt_y[28], real32_T c3_fv[84])
{
  int32_T c3_i13;
  c3_i13 = (int32_T)mwGetGlobalThreadIndex();
  if (c3_i13 < 28) {
    c3_fv[c3_i13] = (real32_T)c3_gpu_vehicleXPoints[c3_i13];
    c3_fv[c3_i13 + 28] = c3_lt_y[c3_i13];
    c3_fv[c3_i13 + 56] = 1.0F;
  }
}

static __global__ __launch_bounds__(96, 1) void c3_eML_blk_kernel_kernel16(const
  real_T c3_Tinv[9], const real32_T c3_fv[84], real32_T c3_U[84])
{
  uint64_T c3_threadId;
  int32_T c3_i14;
  int32_T c3_i15;
  int32_T c3_i16;
  c3_threadId = mwGetGlobalThreadIndex();
  c3_i16 = (int32_T)(c3_threadId % 3UL);
  c3_i14 = (int32_T)((c3_threadId - (uint64_T)c3_i16) / 3UL);
  if ((c3_i14 < 28) && (c3_i16 < 3)) {
    c3_U[c3_i14 + 28 * c3_i16] = 0.0F;
    for (c3_i15 = 0; c3_i15 < 3; c3_i15++) {
      c3_U[c3_i14 + 28 * c3_i16] += c3_fv[c3_i14 + 28 * c3_i15] * (real32_T)
        c3_Tinv[c3_i15 + 3 * c3_i16];
    }
  }
}

static __global__ __launch_bounds__(64, 1) void c3_eML_blk_kernel_kernel17
  (real32_T c3_U[56], real32_T c3_b_U[84], real32_T c3_b[56])
{
  uint64_T c3_threadId;
  int32_T c3_jtilecol;
  int32_T c3_k;
  c3_threadId = mwGetGlobalThreadIndex();
  c3_k = (int32_T)(c3_threadId % 28UL);
  c3_jtilecol = (int32_T)((c3_threadId - (uint64_T)c3_k) / 28UL);
  if ((c3_jtilecol < 2) && (c3_k < 28)) {
    c3_b[c3_jtilecol * 28 + c3_k] = c3_b_U[c3_k + 56];
    c3_U[c3_k + 28 * c3_jtilecol] = c3_b_U[c3_k + 28 * c3_jtilecol] / c3_b[c3_k
      + 28 * c3_jtilecol];
    c3_b_U[c3_k + 28 * c3_jtilecol] = c3_U[c3_k + 28 * c3_jtilecol];
  }
}

static __global__ __launch_bounds__(64, 1) void c3_eML_blk_kernel_kernel18(const
  real32_T c3_U[84], real32_T c3_b_ltPts[56])
{
  uint64_T c3_threadId;
  int32_T c3_i17;
  int32_T c3_i19;
  c3_threadId = mwGetGlobalThreadIndex();
  c3_i19 = (int32_T)(c3_threadId % 28UL);
  c3_i17 = (int32_T)((c3_threadId - (uint64_T)c3_i19) / 28UL);
  if ((c3_i17 < 2) && (c3_i19 < 28)) {
    c3_b_ltPts[c3_i19 + 28 * c3_i17] = c3_U[c3_i19 + 28 * c3_i17];
  }
}

static __global__ __launch_bounds__(32, 1) void c3_eML_blk_kernel_kernel19(const
  real_T c3_Tinv[9], real_T c3_x[9])
{
  int32_T c3_i18;
  c3_i18 = (int32_T)mwGetGlobalThreadIndex();
  if (c3_i18 < 9) {
    c3_x[c3_i18] = c3_Tinv[c3_i18];
  }
}

static __global__ __launch_bounds__(32, 1) void c3_eML_blk_kernel_kernel20(const
  real_T c3_Tinv[9], real_T c3_x[9])
{
  int32_T c3_tmpIdx;
  c3_tmpIdx = (int32_T)mwGetGlobalThreadIndex();
  if (c3_tmpIdx < 1) {
    c3_x[0] = c3_Tinv[2];
    c3_x[2] = c3_Tinv[0];
    c3_x[3] = c3_Tinv[5];
    c3_x[5] = c3_Tinv[3];
    c3_x[6] = c3_Tinv[8];
    c3_x[8] = c3_Tinv[6];
  }
}

static __global__ __launch_bounds__(32, 1) void c3_eML_blk_kernel_kernel21(const
  real_T c3_Tinv[9], real_T c3_x[9])
{
  int32_T c3_tmpIdx;
  c3_tmpIdx = (int32_T)mwGetGlobalThreadIndex();
  if (c3_tmpIdx < 1) {
    c3_x[0] = c3_Tinv[1];
    c3_x[1] = c3_Tinv[0];
    c3_x[3] = c3_Tinv[4];
    c3_x[4] = c3_Tinv[3];
    c3_x[6] = c3_Tinv[7];
    c3_x[7] = c3_Tinv[6];
  }
}

static __global__ __launch_bounds__(32, 1) void c3_eML_blk_kernel_kernel22(const
  real_T c3_t3, const real_T c3_t2, const int32_T c3_p3, real_T c3_Tinv[9])
{
  int32_T c3_tmpIdx;
  c3_tmpIdx = (int32_T)mwGetGlobalThreadIndex();
  if (c3_tmpIdx < 1) {
    c3_Tinv[c3_p3 + 1] = c3_t2;
    c3_Tinv[c3_p3 + 2] = c3_t3;
  }
}

static __global__ __launch_bounds__(32, 1) void c3_eML_blk_kernel_kernel23(const
  real32_T c3_rt_y[28], real32_T c3_fv1[84])
{
  int32_T c3_i20;
  c3_i20 = (int32_T)mwGetGlobalThreadIndex();
  if (c3_i20 < 28) {
    c3_fv1[c3_i20] = (real32_T)c3_gpu_vehicleXPoints[c3_i20];
    c3_fv1[c3_i20 + 28] = c3_rt_y[c3_i20];
    c3_fv1[c3_i20 + 56] = 1.0F;
  }
}

static __global__ __launch_bounds__(96, 1) void c3_eML_blk_kernel_kernel24(const
  real_T c3_Tinv[9], const real32_T c3_fv1[84], real32_T c3_U[84])
{
  uint64_T c3_threadId;
  int32_T c3_i21;
  int32_T c3_i22;
  int32_T c3_i23;
  c3_threadId = mwGetGlobalThreadIndex();
  c3_i23 = (int32_T)(c3_threadId % 3UL);
  c3_i21 = (int32_T)((c3_threadId - (uint64_T)c3_i23) / 3UL);
  if ((c3_i21 < 28) && (c3_i23 < 3)) {
    c3_U[c3_i21 + 28 * c3_i23] = 0.0F;
    for (c3_i22 = 0; c3_i22 < 3; c3_i22++) {
      c3_U[c3_i21 + 28 * c3_i23] += c3_fv1[c3_i21 + 28 * c3_i22] * (real32_T)
        c3_Tinv[c3_i22 + 3 * c3_i23];
    }
  }
}

static __global__ __launch_bounds__(64, 1) void c3_eML_blk_kernel_kernel25
  (real32_T c3_U[56], real32_T c3_b_U[84], real32_T c3_b[56])
{
  uint64_T c3_threadId;
  int32_T c3_jtilecol;
  int32_T c3_k;
  c3_threadId = mwGetGlobalThreadIndex();
  c3_k = (int32_T)(c3_threadId % 28UL);
  c3_jtilecol = (int32_T)((c3_threadId - (uint64_T)c3_k) / 28UL);
  if ((c3_jtilecol < 2) && (c3_k < 28)) {
    c3_b[c3_jtilecol * 28 + c3_k] = c3_b_U[c3_k + 56];
    c3_U[c3_k + 28 * c3_jtilecol] = c3_b_U[c3_k + 28 * c3_jtilecol] / c3_b[c3_k
      + 28 * c3_jtilecol];
    c3_b_U[c3_k + 28 * c3_jtilecol] = c3_U[c3_k + 28 * c3_jtilecol];
  }
}

static __global__ __launch_bounds__(64, 1) void c3_eML_blk_kernel_kernel26(const
  real32_T c3_U[84], real32_T c3_b_rtPts[56])
{
  uint64_T c3_threadId;
  int32_T c3_i24;
  int32_T c3_i25;
  c3_threadId = mwGetGlobalThreadIndex();
  c3_i25 = (int32_T)(c3_threadId % 28UL);
  c3_i24 = (int32_T)((c3_threadId - (uint64_T)c3_i25) / 28UL);
  if ((c3_i24 < 2) && (c3_i25 < 28)) {
    c3_b_rtPts[c3_i25 + 28 * c3_i24] = c3_U[c3_i25 + 28 * c3_i24];
  }
}

static void init_dsm_address_info(SFc3_LaneDetectionInstanceStruct
  *chartInstance)
{
}

static void init_simulink_io_address(SFc3_LaneDetectionInstanceStruct
  *chartInstance)
{
  chartInstance->c3_fEmlrtCtx = (void *)sfrtGetEmlrtCtx(chartInstance->S);
  chartInstance->c3_laneNetOut = (real32_T (*)[6])ssGetInputPortSignal_wrapper
    (chartInstance->S, 0);
  chartInstance->c3_laneFound = (boolean_T *)ssGetOutputPortSignal_wrapper
    (chartInstance->S, 1);
  chartInstance->c3_ltPts = (real32_T (*)[56])ssGetOutputPortSignal_wrapper
    (chartInstance->S, 2);
  chartInstance->c3_rtPts = (real32_T (*)[56])ssGetOutputPortSignal_wrapper
    (chartInstance->S, 3);
}

#include <stdio.h>
#include <stdlib.h>
#include <string.h>

/* SFunction Glue Code */
void sf_c3_LaneDetection_get_check_sum(mxArray *plhs[])
{
  ((real_T *)mxGetPr((plhs[0])))[0] = (real_T)(2051119261U);
  ((real_T *)mxGetPr((plhs[0])))[1] = (real_T)(4225097776U);
  ((real_T *)mxGetPr((plhs[0])))[2] = (real_T)(1871082434U);
  ((real_T *)mxGetPr((plhs[0])))[3] = (real_T)(2287114376U);
}

mxArray *sf_c3_LaneDetection_third_party_uses_info(void)
{
  mxArray * mxcell3p = mxCreateCellMatrix(1,0);
  return(mxcell3p);
}

mxArray *sf_c3_LaneDetection_jit_fallback_info(void)
{
  const char *infoFields[] = { "fallbackType", "fallbackReason",
    "hiddenFallbackType", "hiddenFallbackReason", "incompatibleSymbol" };

  mxArray *mxInfo = mxCreateStructMatrix(1, 1, 5, infoFields);
  mxArray *fallbackType = mxCreateString("pre");
  mxArray *fallbackReason = mxCreateString("GPUAcceleration");
  mxArray *hiddenFallbackType = mxCreateString("none");
  mxArray *hiddenFallbackReason = mxCreateString("");
  mxArray *incompatibleSymbol = mxCreateString("");
  mxSetField(mxInfo, 0, infoFields[0], fallbackType);
  mxSetField(mxInfo, 0, infoFields[1], fallbackReason);
  mxSetField(mxInfo, 0, infoFields[2], hiddenFallbackType);
  mxSetField(mxInfo, 0, infoFields[3], hiddenFallbackReason);
  mxSetField(mxInfo, 0, infoFields[4], incompatibleSymbol);
  return mxInfo;
}

mxArray *sf_c3_LaneDetection_updateBuildInfo_args_info(void)
{
  mxArray *mxBIArgs = sf_mex_decode(
    "eNpjYPT0ZQACPiDm4GRgYAPRQMzIAAGsSHxOJHGQ+h8M+NUzoal3QFLPgkU9H5J6ASg/MSUlOL+"
    "0KDnVLTMntRgi1kDAXkY0ewMI2CuDZi+I7xvumpOam5pXUp5ZnOqYlpaZl+qTWJlapJdcUAAz98"
    "MA+b+CRP97ELBXAs1eCbD/3UqLU1Oc8/PKglJ9QuF+B5n3YoD8XUCivwnZy41mL4ifnJcXn1iQC"
    "YvmYRXPgmj2CoLj2dnPDxy7nrkFOXrJpcMwf8uj2SuPJ3/DQgFkLkjTQPi/g0T/RxCwVxXNXlUC"
    "/vdOLcpLhaaF4ZQOpNHslcZRziHnhOGU/8XQ7BUD+z8ksSg9tcQvtaQ8vygb7vXhFO+SaPZKQsq"
    "90uKS/FxwhLvlFzmXuvj5wUt9BgYA8ZCiqg=="
    );
  return mxBIArgs;
}

static const mxArray *sf_get_sim_state_info_c3_LaneDetection(void)
{
  const char *infoFields[] = { "chartChecksum", "varInfo" };

  mxArray *mxInfo = mxCreateStructMatrix(1, 1, 2, infoFields);
  mxArray *mxVarInfo = sf_mex_decode(
    "eNrtlkFLwzAUx9OyTXtY6cGDiId9Aot48WbEIRQcFLx4qyFNaaFNapsNveoX2Efxo/kRTNps1lA"
    "suOImGng83oP/S3/hn1BgeDMgli3i1QJgJPK+CBPUa6hqQ8SeynV/sO5PRPCnnMh+WWAvFJmirK"
    "rR/NGjEavmn4OP+aOW+UZjvqX69Xq72Ex/AKUeNvSDFr3V0DuqThEl12xOQ9W3G3l7PIdQ17fxD"
    "DUeWafc5+XOcBx/m6PolcOBm+nr/bv8ZWsctvLXFSNRNCOIlorD+PoeGto9NFUtNSlIQN7yPW08"
    "psbjrGkCWCWIYB/+iDvO5VQ7F1m7McuIG2c4dS+nmFFesNSVhxWEhBPME0YDzFgRJhRxUp5kv8g"
    "HY4133PTBLQ9Xvt66D5Z57YP7h38f/NB7sCBxglNy57OE8l15D5bPtQ/Ayx/zwaQXH/gdvEcar6"
    "yTMkCCakECfBbcCM7pCvPzf8g7obOKrw=="
    );
  mxArray *mxChecksum = mxCreateDoubleMatrix(1, 4, mxREAL);
  sf_c3_LaneDetection_get_check_sum(&mxChecksum);
  mxSetField(mxInfo, 0, infoFields[0], mxChecksum);
  mxSetField(mxInfo, 0, infoFields[1], mxVarInfo);
  return mxInfo;
}

static const char* sf_get_instance_specialization(void)
{
  return "sc88SziHwZbSUvEPWN0mVPE";
}

static void sf_opaque_initialize_c3_LaneDetection(void *chartInstanceVar)
{
  initialize_params_c3_LaneDetection((SFc3_LaneDetectionInstanceStruct*)
    chartInstanceVar);
  initialize_c3_LaneDetection((SFc3_LaneDetectionInstanceStruct*)
    chartInstanceVar);
}

static void sf_opaque_enable_c3_LaneDetection(void *chartInstanceVar)
{
  enable_c3_LaneDetection((SFc3_LaneDetectionInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_disable_c3_LaneDetection(void *chartInstanceVar)
{
  disable_c3_LaneDetection((SFc3_LaneDetectionInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_gateway_c3_LaneDetection(void *chartInstanceVar)
{
  sf_gateway_c3_LaneDetection((SFc3_LaneDetectionInstanceStruct*)
    chartInstanceVar);
}

static const mxArray* sf_opaque_get_sim_state_c3_LaneDetection(SimStruct* S)
{
  return get_sim_state_c3_LaneDetection((SFc3_LaneDetectionInstanceStruct *)
    sf_get_chart_instance_ptr(S));     /* raw sim ctx */
}

static void sf_opaque_set_sim_state_c3_LaneDetection(SimStruct* S, const mxArray
  *st)
{
  set_sim_state_c3_LaneDetection((SFc3_LaneDetectionInstanceStruct*)
    sf_get_chart_instance_ptr(S), st);
}

static void sf_opaque_cleanup_runtime_resources_c3_LaneDetection(void
  *chartInstanceVar)
{
  if (chartInstanceVar!=NULL) {
    SimStruct *S = ((SFc3_LaneDetectionInstanceStruct*) chartInstanceVar)->S;
    if (sim_mode_is_rtw_gen(S) || sim_mode_is_external(S)) {
      sf_clear_rtw_identifier(S);
      unload_LaneDetection_optimization_info();
    }

    mdl_cleanup_runtime_resources_c3_LaneDetection
      ((SFc3_LaneDetectionInstanceStruct*) chartInstanceVar);
    ((SFc3_LaneDetectionInstanceStruct*) chartInstanceVar)->
      ~SFc3_LaneDetectionInstanceStruct();
    utFree(chartInstanceVar);
    if (ssGetUserData(S)!= NULL) {
      sf_free_ChartRunTimeInfo(S);
    }

    ssSetUserData(S,NULL);
  }
}

static void sf_opaque_mdl_start_c3_LaneDetection(void *chartInstanceVar)
{
  mdl_start_c3_LaneDetection((SFc3_LaneDetectionInstanceStruct*)
    chartInstanceVar);
}

static void sf_opaque_mdl_terminate_c3_LaneDetection(void *chartInstanceVar)
{
  mdl_terminate_c3_LaneDetection((SFc3_LaneDetectionInstanceStruct*)
    chartInstanceVar);
}

static void sf_opaque_init_subchart_simstructs(void *chartInstanceVar)
{
  initSimStructsc3_LaneDetection((SFc3_LaneDetectionInstanceStruct*)
    chartInstanceVar);
}

extern unsigned int sf_machine_global_initializer_called(void);
static void mdlProcessParameters_c3_LaneDetection(SimStruct *S)
{
  int i;
  for (i=0;i<ssGetNumRunTimeParams(S);i++) {
    if (ssGetSFcnParamTunable(S,i)) {
      ssUpdateDlgParamAsRunTimeParam(S,i);
    }
  }

  sf_warn_if_symbolic_dimension_param_changed(S);
  if (sf_machine_global_initializer_called()) {
    initialize_params_c3_LaneDetection((SFc3_LaneDetectionInstanceStruct*)
      sf_get_chart_instance_ptr(S));
    initSimStructsc3_LaneDetection((SFc3_LaneDetectionInstanceStruct*)
      sf_get_chart_instance_ptr(S));
  }
}

const char* sf_c3_LaneDetection_get_post_codegen_info(void)
{
  int i;
  const char* encStrCodegen [35] = {
    "eNrtXE9vG0UU34S2EESjHJBAVSV6g1Od2GnVE6TxHzVS0lp1kiKE5M7OjrNT785sdmbthJ7atPd",
    "eERcOIHGsBAdOfAYuSIgrX4CPwMx64zjjXbt2QuxnasnZrP1m9vfe/N6bN7Nvbc1tbFnqtajerz",
    "zLuqKO76n3vNV5XU7O53renc8vWZ8m57+oRizyqyhEvrAGvhjyyUMiuBdJytkGa/BUMcoaJCQMK",
    "9mAhzKrN0H9yKOsWYkY1v2JRy7Fbs3lkeesq7bIecC8Q9VbEMmq6qdEQ4JlhRBHuiGP9tyKh/a6",
    "iEPZLroEN0XkD1JBEFmLAg1LbEWepIFHygcEbzAhkUIsTrDVJJKkKA8y1dSaitqxIPcDjyKWqq2",
    "LRI0EysCS7ASO+vsgkkopUwy7KJTrxEUtIjZpM+6TM2L2SYX6wqYMSR5S5JV9r6gb9mOregrPFn",
    "eIN8AgCtt6SFAz4JTJ7PGvVZSmZYZsj5SIHe1l91Yj+5Ee/F1K2iTMtFujyFskRHvkAcu8aGyQ8",
    "kE8Wl2W9ItJ6pNdFN7FavwEcTLZq5gjakiNE9lWLbLESKzkhtgOaUuZN7O3yN/QzBzmMpHfGWwx",
    "TCzurdwig0ah21sFsyLyPJEpts2DTdIiXtxrCUk0WKzTa7qcENTZ5srAmt7Z3hAxqgY+ESty5tD",
    "U4WoZAnHcua8Cy2lJHAnJ/aIib2lzs//rfrENJknYQJikRYEQUUGUzWLzZvfmUKHHXgkqVDKGly",
    "bcYcgwKUs0IlZq87CpbDIgiJyooEc0U9AXe2oslSfsCOU0g8T0WA6Twwi7xNEBhnpkS7mNkk2xi",
    "dCh7a7yuxaVhyUicEiDlFGNlNepMFTWhDoMyA5rMt5mlZD7tSTGd0ZBRQYVw301BtuxjzGsuqJC",
    "qnBBTy7vEKJIiUJG2d66CnPhYUWBTB0xPe8tWyfz3gdvMO8dtzOPn/X0M5fSj9VzNK+7MD/4uvP",
    "qv7mk3VpPu6vGdS4Z7bTcknr//Nu3t4++++HJ7398vYR//Ovvs1z/8fuj5QmLyfn144DcdbBWH6",
    "+17L0eXJdS+v+op/+l5FzgO3dq39B77a/s2k6rXH10f9nfrZbj/l4vDMb7roH3+PMbemZQbIx5H",
    "OINJ0lg9DmKOtO67v9OD94rQ+yxkHzeef3zxdnaf7hm8iHNXguGvfS5mthJhUfMSeHx5PT5eM1s",
    "n6bPZUMffe7JajzrTYce18fWIzxXPZbWzta+P96k6bFo6LGY8KvISaOxRZAO9jGOudHixnxyrtt",
    "4FrWCFDxp+swb+ix1tamvxYc1tHYe/HCH2GXFsIs+z7ncJznXx17ubklNXjLkXk4bq+4QSeL8tI",
    "45Dx2dnRNx0wfEg6uGvld7eVCTzjGvJ86DV0GHB4/33/LgguJBi6iluUe+rCaLxOngwfMOD6yj/",
    "xkPbpwLD6pD9L1m6HstXrfXkV4NkDou1DeVnqVjNfvzkHHz01HbWW/bvW13Ae3mxlxvjtvunTOu",
    "by+q3Vn1G3XdPW3yywPiqGXIL02xHmfdD/mv5f+0Rss3PknOP+9uYRZd6jkpu1nJ15sENdK+nRG",
    "eBg8HyweG/K9D8oNnBq+fZeVDwgvCJ7m6aOjDqaRBfUi8Rk5vlOZEiHOYsfoh93gr/5CI+0TeWt",
    "7tJJ7lg3jbfvl0zrGcCBc5a+Xr9k2bMrD4V+rtEfG/NPC/nBx+DwmhlejqABW/PQL+Fwb+FxPAH",
    "xKx2qjbIWLYzeMTCgHFb0PET4Dbv4vfBo8fNn8QcP+Fzh8Ekz8O8PjpAOe/A5w/GPj8hYHzH88K",
    "/23w9geJ3wbuvzZw/7WB5z82bP/N27DjZy9+kPZHs2J/w3+/H4KfG/j5+eCXIaKMOFpOadCHuIH",
    "1BysJVSDZGQOPk2hMPz0y8B9N2E9XenbKp53neXg8L6CM+fT1EPxPDfxPJ4AfG3ciQM5DI+bB0+",
    "efwO0PfR1oA8/DRlxHPTfwP58A/vgZv/pKnTcagsiTG6Eg4z7MPL6AZuU+ItB9HAR8H6SRET+H5",
    "Ze+gd+/kPwSn6oZgeSn4+4XT0O9y3GcFxh5xKh3mV6erI7Ek2mri2qDsXNhpPqhafFHe1bmfZj7",
    "dwVnzP27yfH81imeT/P8CNAf88DvhxWc2dlPt+Hsp7e7+4zTPD/aIOLGLZB5tT0rdTBg5sEViDz",
    "JZ9UrwNnftyHuLxfwrOyPICj+uXoq//tpCM59A+f+heB0kER928ZvZNemgbd5QfnG7RHvv04OZx",
    "t6nGgDf57EBl4PCnL9ArweNA+8nrgwM/a3gePH4PFDrANZzarTAoK/MDv4YeY/CPb8WwB+H7wAv",
    "B69APx5gN7nIdug67hs4PmbDdx/4dsf+v0vG1p9S+/vuYz6ezRT+DyPDbweAHb8waPdPw0M/MHF",
    "7b92y7lA8gQD5zn4dS7IddYq8LqpVQSwTiPfU6cB0M6wnw9HcOoKANatraJZ+R2HnrzxX2KiExs",
    "=",
    ""
  };

  static char newstr [2477] = "";
  newstr[0] = '\0';
  for (i = 0; i < 35; i++) {
    strcat(newstr, encStrCodegen[i]);
  }

  return newstr;
}

static void mdlSetWorkWidths_c3_LaneDetection(SimStruct *S)
{
  const char* newstr = sf_c3_LaneDetection_get_post_codegen_info();
  sf_set_work_widths(S, newstr);
  ssSetChecksum0(S,(916045491U));
  ssSetChecksum1(S,(1789172611U));
  ssSetChecksum2(S,(274519504U));
  ssSetChecksum3(S,(3822757219U));
}

static void mdlRTW_c3_LaneDetection(SimStruct *S)
{
  if (sim_mode_is_rtw_gen(S)) {
    ssWriteRTWStrParam(S, "StateflowChartType", "Embedded MATLAB");
  }
}

static void mdlSetupRuntimeResources_c3_LaneDetection(SimStruct *S)
{
  SFc3_LaneDetectionInstanceStruct *chartInstance;
  chartInstance = (SFc3_LaneDetectionInstanceStruct *)utMalloc(sizeof
    (SFc3_LaneDetectionInstanceStruct));
  if (chartInstance==NULL) {
    sf_mex_error_message("Could not allocate memory for chart instance.");
  }

  memset(chartInstance, 0, sizeof(SFc3_LaneDetectionInstanceStruct));
  chartInstance = new (chartInstance) SFc3_LaneDetectionInstanceStruct;
  chartInstance->chartInfo.chartInstance = chartInstance;
  if (ssGetSampleTime(S, 0) == CONTINUOUS_SAMPLE_TIME && ssGetOffsetTime(S, 0) ==
      0 && ssGetNumContStates(ssGetRootSS(S)) > 0) {
    sf_error_out_about_continuous_sample_time_with_persistent_vars(S);
  }

  chartInstance->chartInfo.isEMLChart = 1;
  chartInstance->chartInfo.chartInitialized = 0;
  chartInstance->chartInfo.sFunctionGateway = sf_opaque_gateway_c3_LaneDetection;
  chartInstance->chartInfo.initializeChart =
    sf_opaque_initialize_c3_LaneDetection;
  chartInstance->chartInfo.mdlStart = sf_opaque_mdl_start_c3_LaneDetection;
  chartInstance->chartInfo.mdlTerminate =
    sf_opaque_mdl_terminate_c3_LaneDetection;
  chartInstance->chartInfo.mdlCleanupRuntimeResources =
    sf_opaque_cleanup_runtime_resources_c3_LaneDetection;
  chartInstance->chartInfo.enableChart = sf_opaque_enable_c3_LaneDetection;
  chartInstance->chartInfo.disableChart = sf_opaque_disable_c3_LaneDetection;
  chartInstance->chartInfo.getSimState =
    sf_opaque_get_sim_state_c3_LaneDetection;
  chartInstance->chartInfo.setSimState =
    sf_opaque_set_sim_state_c3_LaneDetection;
  chartInstance->chartInfo.getSimStateInfo =
    sf_get_sim_state_info_c3_LaneDetection;
  chartInstance->chartInfo.zeroCrossings = NULL;
  chartInstance->chartInfo.outputs = NULL;
  chartInstance->chartInfo.derivatives = NULL;
  chartInstance->chartInfo.mdlRTW = mdlRTW_c3_LaneDetection;
  chartInstance->chartInfo.mdlSetWorkWidths = mdlSetWorkWidths_c3_LaneDetection;
  chartInstance->chartInfo.callGetHoverDataForMsg = NULL;
  chartInstance->chartInfo.extModeExec = NULL;
  chartInstance->chartInfo.restoreLastMajorStepConfiguration = NULL;
  chartInstance->chartInfo.restoreBeforeLastMajorStepConfiguration = NULL;
  chartInstance->chartInfo.storeCurrentConfiguration = NULL;
  chartInstance->chartInfo.callAtomicSubchartUserFcn = NULL;
  chartInstance->chartInfo.callAtomicSubchartAutoFcn = NULL;
  chartInstance->chartInfo.callAtomicSubchartEventFcn = NULL;
  chartInstance->chartInfo.chartStateSetterFcn = NULL;
  chartInstance->chartInfo.chartStateGetterFcn = NULL;
  chartInstance->S = S;
  chartInstance->chartInfo.dispatchToExportedFcn = NULL;
  sf_init_ChartRunTimeInfo(S, &(chartInstance->chartInfo), false, 0,
    chartInstance->c3_JITStateAnimation,
    chartInstance->c3_JITTransitionAnimation);
  init_dsm_address_info(chartInstance);
  init_simulink_io_address(chartInstance);
  if (!sim_mode_is_rtw_gen(S)) {
  }

  mdl_setup_runtime_resources_c3_LaneDetection(chartInstance);
}

void c3_LaneDetection_method_dispatcher(SimStruct *S, int_T method, void *data)
{
  switch (method) {
   case SS_CALL_MDL_SETUP_RUNTIME_RESOURCES:
    mdlSetupRuntimeResources_c3_LaneDetection(S);
    break;

   case SS_CALL_MDL_SET_WORK_WIDTHS:
    mdlSetWorkWidths_c3_LaneDetection(S);
    break;

   case SS_CALL_MDL_PROCESS_PARAMETERS:
    mdlProcessParameters_c3_LaneDetection(S);
    break;

   default:
    /* Unhandled method */
    sf_mex_error_message("Stateflow Internal Error:\n"
                         "Error calling c3_LaneDetection_method_dispatcher.\n"
                         "Can't handle method %d.\n", method);
    break;
  }
}
