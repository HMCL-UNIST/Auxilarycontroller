#include "hip/hip_runtime.h"
/* Include files */

#include "LaneDetection_sfun.h"
#include "c3_LaneDetection.h"
#include "MWCudaDimUtility.hpp"
#include "mwmathutil.h"
#ifdef utFree
#undef utFree
#endif

#ifdef utMalloc
#undef utMalloc
#endif

#ifdef __cplusplus

extern "C" void *utMalloc(size_t size);
extern "C" void utFree(void*);

#else

extern void *utMalloc(size_t size);
extern void utFree(void*);

#endif

/* Type Definitions */

/* Named Constants */
const int32_T CALL_EVENT = -1;

/* Variable Declarations */

/* Variable Definitions */
static real_T _sfTime_;
static __device__ real_T c3_gpu_vehicleXPoints[28];
static __device__ real_T c3_gpu_laneCoeffStds[6];
static __device__ real_T c3_gpu_laneCoeffMeans[6];

/* Function Declarations */
static void initialize_c3_LaneDetection(SFc3_LaneDetectionInstanceStruct
  *chartInstance);
static void initialize_params_c3_LaneDetection(SFc3_LaneDetectionInstanceStruct *
  chartInstance);
static void enable_c3_LaneDetection(SFc3_LaneDetectionInstanceStruct
  *chartInstance);
static void disable_c3_LaneDetection(SFc3_LaneDetectionInstanceStruct
  *chartInstance);
static void c3_do_animation_call_c3_LaneDetection
  (SFc3_LaneDetectionInstanceStruct *chartInstance);
static void ext_mode_exec_c3_LaneDetection(SFc3_LaneDetectionInstanceStruct
  *chartInstance);
static const mxArray *get_sim_state_c3_LaneDetection
  (SFc3_LaneDetectionInstanceStruct *chartInstance);
static void set_sim_state_c3_LaneDetection(SFc3_LaneDetectionInstanceStruct
  *chartInstance, const mxArray *c3_st);
static void sf_gateway_c3_LaneDetection(SFc3_LaneDetectionInstanceStruct
  *chartInstance);
static void mdl_start_c3_LaneDetection(SFc3_LaneDetectionInstanceStruct
  *chartInstance);
static void mdl_terminate_c3_LaneDetection(SFc3_LaneDetectionInstanceStruct
  *chartInstance);
static void mdl_setup_runtime_resources_c3_LaneDetection
  (SFc3_LaneDetectionInstanceStruct *chartInstance);
static void mdl_cleanup_runtime_resources_c3_LaneDetection
  (SFc3_LaneDetectionInstanceStruct *chartInstance);
static void initSimStructsc3_LaneDetection(SFc3_LaneDetectionInstanceStruct
  *chartInstance);
static void c3_eML_blk_kernel(SFc3_LaneDetectionInstanceStruct *chartInstance,
  real32_T c3_b_laneNetOut[6], boolean_T *c3_b_laneFound, real32_T c3_b_ltPts[56],
  real32_T c3_b_rtPts[56]);
static boolean_T c3_emlrt_marshallIn(SFc3_LaneDetectionInstanceStruct
  *chartInstance, const mxArray *c3_b_laneFound, const char_T *c3_identifier);
static boolean_T c3_b_emlrt_marshallIn(SFc3_LaneDetectionInstanceStruct
  *chartInstance, const mxArray *c3_u, const emlrtMsgIdentifier *c3_parentId);
static void c3_c_emlrt_marshallIn(SFc3_LaneDetectionInstanceStruct
  *chartInstance, const mxArray *c3_b_ltPts, const char_T *c3_identifier,
  real32_T c3_y[56]);
static void c3_d_emlrt_marshallIn(SFc3_LaneDetectionInstanceStruct
  *chartInstance, const mxArray *c3_u, const emlrtMsgIdentifier *c3_parentId,
  real32_T c3_y[56]);
static void c3_e_emlrt_marshallIn(SFc3_LaneDetectionInstanceStruct
  *chartInstance, const mxArray *c3_b_laneCoeffMeans, const char_T
  *c3_identifier, boolean_T *c3_svPtr, real_T c3_y[6]);
static void c3_f_emlrt_marshallIn(SFc3_LaneDetectionInstanceStruct
  *chartInstance, const mxArray *c3_u, const emlrtMsgIdentifier *c3_parentId,
  boolean_T *c3_svPtr, real_T c3_y[6]);
static void c3_g_emlrt_marshallIn(SFc3_LaneDetectionInstanceStruct
  *chartInstance, const mxArray *c3_b_vehicleXPoints, const char_T
  *c3_identifier, boolean_T *c3_svPtr, real_T c3_y[28]);
static void c3_h_emlrt_marshallIn(SFc3_LaneDetectionInstanceStruct
  *chartInstance, const mxArray *c3_u, const emlrtMsgIdentifier *c3_parentId,
  boolean_T *c3_svPtr, real_T c3_y[28]);
static uint8_T c3_i_emlrt_marshallIn(SFc3_LaneDetectionInstanceStruct
  *chartInstance, const mxArray *c3_b_is_active_c3_LaneDetection, const char_T
  *c3_identifier);
static uint8_T c3_j_emlrt_marshallIn(SFc3_LaneDetectionInstanceStruct
  *chartInstance, const mxArray *c3_u, const emlrtMsgIdentifier *c3_parentId);
static __global__ void c3_eML_blk_kernel_kernel1(const real_T c3_dv[6]);
static __global__ void c3_eML_blk_kernel_kernel2(const real_T c3_dv1[6]);
static __global__ void c3_eML_blk_kernel_kernel3(const real32_T c3_b_laneNetOut
  [6], real32_T c3_params[6]);
static __global__ void c3_eML_blk_kernel_kernel4();
static __global__ void c3_eML_blk_kernel_kernel5(const real32_T c3_params[6],
  real32_T c3_rt_y[28]);
static __global__ void c3_eML_blk_kernel_kernel6(const real32_T c3_params,
  real32_T c3_rt_y[28]);
static __global__ void c3_eML_blk_kernel_kernel7(const real32_T c3_params[6],
  real32_T c3_lt_y[28]);
static __global__ void c3_eML_blk_kernel_kernel8(const real32_T c3_params,
  real32_T c3_lt_y[28]);
static __global__ void c3_eML_blk_kernel_kernel9(const real_T c3_T[9], real_T
  c3_x[9]);
static __global__ void c3_eML_blk_kernel_kernel10(const real_T c3_Tinv[9],
  real_T c3_x[9]);
static __global__ void c3_eML_blk_kernel_kernel11(const real_T c3_Tinv[9],
  real_T c3_x[9]);
static __global__ void c3_eML_blk_kernel_kernel12(const real_T c3_Tinv[9],
  real_T c3_x[9]);
static __global__ void c3_eML_blk_kernel_kernel13(const real_T c3_t3, const
  real_T c3_t2, const real_T c3_x[9], const int32_T c3_p1, real_T c3_Tinv[9]);
static __global__ void c3_eML_blk_kernel_kernel14(const real_T c3_t3, const
  real_T c3_t2, const real_T c3_x[9], const int32_T c3_p2, real_T c3_Tinv[9]);
static __global__ void c3_eML_blk_kernel_kernel15(const real_T c3_t3, const
  real_T c3_t2, const real_T c3_x[9], const int32_T c3_p3, real_T c3_Tinv[9]);
static __global__ void c3_eML_blk_kernel_kernel16(const real32_T c3_lt_y[28],
  real32_T c3_fv[84]);
static __global__ void c3_eML_blk_kernel_kernel17(const real_T c3_Tinv[9], const
  real32_T c3_fv[84], real32_T c3_U[84]);
static __global__ void c3_eML_blk_kernel_kernel18(real32_T c3_U[56], real32_T
  c3_b_U[84], real32_T c3_b[56]);
static __global__ void c3_eML_blk_kernel_kernel19(const real32_T c3_U[84],
  real32_T c3_b_ltPts[56]);
static __global__ void c3_eML_blk_kernel_kernel20(const real_T c3_Tinv[9],
  real_T c3_x[9]);
static __global__ void c3_eML_blk_kernel_kernel21(const real_T c3_Tinv[9],
  real_T c3_x[9]);
static __global__ void c3_eML_blk_kernel_kernel22(const real_T c3_Tinv[9],
  real_T c3_x[9]);
static __global__ void c3_eML_blk_kernel_kernel23(const real_T c3_t3, const
  real_T c3_t2, const int32_T c3_p3, real_T c3_Tinv[9]);
static __global__ void c3_eML_blk_kernel_kernel24(const real32_T c3_rt_y[28],
  real32_T c3_fv1[84]);
static __global__ void c3_eML_blk_kernel_kernel25(const real_T c3_Tinv[9], const
  real32_T c3_fv1[84], real32_T c3_U[84]);
static __global__ void c3_eML_blk_kernel_kernel26(real32_T c3_U[56], real32_T
  c3_b_U[84], real32_T c3_b[56]);
static __global__ void c3_eML_blk_kernel_kernel27(const real32_T c3_U[84],
  real32_T c3_b_rtPts[56]);
static void init_dsm_address_info(SFc3_LaneDetectionInstanceStruct
  *chartInstance);
static void init_simulink_io_address(SFc3_LaneDetectionInstanceStruct
  *chartInstance);

/* Function Definitions */
static void initialize_c3_LaneDetection(SFc3_LaneDetectionInstanceStruct
  *chartInstance)
{
  emlrtLicenseCheckR2012b(chartInstance->c3_fEmlrtCtx,
    "distrib_computing_toolbox", 2);
  emlrtLicenseCheckR2012b(chartInstance->c3_fEmlrtCtx, "image_toolbox", 2);
  sim_mode_is_external(chartInstance->S);
  _sfTime_ = sf_get_time(chartInstance->S);
  chartInstance->c3_laneCoeffMeans_not_empty = false;
  chartInstance->c3_laneCoeffStds_not_empty = false;
  chartInstance->c3_vehicleXPoints_not_empty = false;
  chartInstance->c3_is_active_c3_LaneDetection = 0U;
  hipGetLastError();
  hipMalloc(&chartInstance->c3_gpu_params, 4UL);
  hipMalloc(&chartInstance->c3_c_gpu_params, 4UL);
  hipMalloc(&chartInstance->c3_gpu_p2, 4UL);
  hipMalloc(&chartInstance->c3_b_gpu_t2, 8UL);
  hipMalloc(&chartInstance->c3_gpu_lt_y, 112UL);
  hipMalloc(&chartInstance->c3_gpu_t3, 8UL);
  hipMalloc(&chartInstance->c3_gpu_rtPts, 224UL);
  hipMalloc(&chartInstance->c3_gpu_laneNetOut, 24UL);
  hipMalloc(&chartInstance->c3_gpu_Tinv, 72UL);
  hipMalloc(&chartInstance->c3_gpu_rt_y, 112UL);
  hipMalloc(&chartInstance->c3_gpu_dv1, 48UL);
  hipMalloc(&chartInstance->c3_gpu_t2, 8UL);
  hipMalloc(&chartInstance->c3_b_gpu_params, 24UL);
  hipMalloc(&chartInstance->c3_gpu_dv, 48UL);
  hipMalloc(&chartInstance->c3_b_gpu_p3, 4UL);
  hipMalloc(&chartInstance->c3_gpu_U, 224UL);
  hipMalloc(&chartInstance->c3_b_gpu_Tinv, 72UL);
  hipMalloc(&chartInstance->c3_gpu_fv, 336UL);
  hipMalloc(&chartInstance->c3_gpu_ltPts, 224UL);
  hipMalloc(&chartInstance->c3_b_gpu_U, 336UL);
  hipMalloc(&chartInstance->c3_c_gpu_U, 224UL);
  hipMalloc(&chartInstance->c3_gpu_p1, 4UL);
  hipMalloc(&chartInstance->c3_gpu_fv1, 336UL);
  hipMalloc(&chartInstance->c3_gpu_T, 72UL);
  hipMalloc(&chartInstance->c3_b_gpu_t3, 8UL);
  hipMalloc(&chartInstance->c3_gpu_b, 224UL);
  hipMalloc(&chartInstance->c3_gpu_x, 72UL);
  hipMalloc(&chartInstance->c3_gpu_p3, 4UL);
}

static void initialize_params_c3_LaneDetection(SFc3_LaneDetectionInstanceStruct *
  chartInstance)
{
}

static void enable_c3_LaneDetection(SFc3_LaneDetectionInstanceStruct
  *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
}

static void disable_c3_LaneDetection(SFc3_LaneDetectionInstanceStruct
  *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
}

static void c3_do_animation_call_c3_LaneDetection
  (SFc3_LaneDetectionInstanceStruct *chartInstance)
{
  sfDoAnimationWrapper(chartInstance->S, false, true);
  sfDoAnimationWrapper(chartInstance->S, false, false);
}

static void ext_mode_exec_c3_LaneDetection(SFc3_LaneDetectionInstanceStruct
  *chartInstance)
{
}

static const mxArray *get_sim_state_c3_LaneDetection
  (SFc3_LaneDetectionInstanceStruct *chartInstance)
{
  const mxArray *c3_b_y = NULL;
  const mxArray *c3_c_y = NULL;
  const mxArray *c3_d_y = NULL;
  const mxArray *c3_e_y = NULL;
  const mxArray *c3_f_y = NULL;
  const mxArray *c3_g_y = NULL;
  const mxArray *c3_h_y = NULL;
  const mxArray *c3_st;
  const mxArray *c3_y = NULL;
  c3_st = NULL;
  c3_y = NULL;
  sf_mex_assign(&c3_y, sf_mex_createcellmatrix(7, 1), false);
  c3_b_y = NULL;
  sf_mex_assign(&c3_b_y, sf_mex_create("y", chartInstance->c3_laneFound, 11, 0U,
    0U, 0U, 0), false);
  sf_mex_setcell(c3_y, 0, c3_b_y);
  c3_c_y = NULL;
  sf_mex_assign(&c3_c_y, sf_mex_create("y", *chartInstance->c3_ltPts, 1, 0U, 1U,
    0U, 2, 28, 2), false);
  sf_mex_setcell(c3_y, 1, c3_c_y);
  c3_d_y = NULL;
  sf_mex_assign(&c3_d_y, sf_mex_create("y", *chartInstance->c3_rtPts, 1, 0U, 1U,
    0U, 2, 28, 2), false);
  sf_mex_setcell(c3_y, 2, c3_d_y);
  c3_e_y = NULL;
  if (!chartInstance->c3_laneCoeffMeans_not_empty) {
    sf_mex_assign(&c3_e_y, sf_mex_create("y", NULL, 0, 0U, 1U, 0U, 2, 0, 0),
                  false);
  } else {
    sf_mex_assign(&c3_e_y, sf_mex_create("y", chartInstance->c3_laneCoeffMeans,
      0, 0U, 1U, 0U, 2, 1, 6), false);
  }

  sf_mex_setcell(c3_y, 3, c3_e_y);
  c3_f_y = NULL;
  if (!chartInstance->c3_laneCoeffMeans_not_empty) {
    sf_mex_assign(&c3_f_y, sf_mex_create("y", NULL, 0, 0U, 1U, 0U, 2, 0, 0),
                  false);
  } else {
    sf_mex_assign(&c3_f_y, sf_mex_create("y", chartInstance->c3_laneCoeffStds, 0,
      0U, 1U, 0U, 2, 1, 6), false);
  }

  sf_mex_setcell(c3_y, 4, c3_f_y);
  c3_g_y = NULL;
  if (!chartInstance->c3_vehicleXPoints_not_empty) {
    sf_mex_assign(&c3_g_y, sf_mex_create("y", NULL, 0, 0U, 1U, 0U, 2, 0, 0),
                  false);
  } else {
    sf_mex_assign(&c3_g_y, sf_mex_create("y", chartInstance->c3_vehicleXPoints,
      0, 0U, 1U, 0U, 2, 1, 28), false);
  }

  sf_mex_setcell(c3_y, 5, c3_g_y);
  c3_h_y = NULL;
  sf_mex_assign(&c3_h_y, sf_mex_create("y",
    &chartInstance->c3_is_active_c3_LaneDetection, 3, 0U, 0U, 0U, 0), false);
  sf_mex_setcell(c3_y, 6, c3_h_y);
  sf_mex_assign(&c3_st, c3_y, false);
  return c3_st;
}

static void set_sim_state_c3_LaneDetection(SFc3_LaneDetectionInstanceStruct
  *chartInstance, const mxArray *c3_st)
{
  const mxArray *c3_u;
  c3_u = sf_mex_dup(c3_st);
  *chartInstance->c3_laneFound = c3_emlrt_marshallIn(chartInstance, sf_mex_dup
    (sf_mex_getcell(c3_u, 0)), "laneFound");
  c3_c_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getcell(c3_u, 1)),
                        "ltPts", *chartInstance->c3_ltPts);
  c3_c_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getcell(c3_u, 2)),
                        "rtPts", *chartInstance->c3_rtPts);
  c3_e_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getcell(c3_u, 3)),
                        "laneCoeffMeans",
                        &chartInstance->c3_laneCoeffMeans_not_empty,
                        chartInstance->c3_laneCoeffMeans);
  c3_e_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getcell(c3_u, 4)),
                        "laneCoeffStds",
                        &chartInstance->c3_laneCoeffStds_not_empty,
                        chartInstance->c3_laneCoeffStds);
  c3_g_emlrt_marshallIn(chartInstance, sf_mex_dup(sf_mex_getcell(c3_u, 5)),
                        "vehicleXPoints",
                        &chartInstance->c3_vehicleXPoints_not_empty,
                        chartInstance->c3_vehicleXPoints);
  chartInstance->c3_is_active_c3_LaneDetection = c3_i_emlrt_marshallIn
    (chartInstance, sf_mex_dup(sf_mex_getcell(c3_u, 6)),
     "is_active_c3_LaneDetection");
  sf_mex_destroy(&c3_u);
  sf_mex_destroy(&c3_st);
}

static void sf_gateway_c3_LaneDetection(SFc3_LaneDetectionInstanceStruct
  *chartInstance)
{
  int32_T c3_i;
  int32_T c3_i1;
  int32_T c3_i2;
  real32_T c3_fv1[56];
  real32_T c3_fv2[56];
  real32_T c3_fv[6];
  boolean_T c3_b;
  chartInstance->c3_JITTransitionAnimation[0] = 0U;
  _sfTime_ = sf_get_time(chartInstance->S);
  for (c3_i = 0; c3_i < 6; c3_i++) {
    c3_fv[c3_i] = (*chartInstance->c3_laneNetOut)[c3_i];
  }

  c3_eML_blk_kernel(chartInstance, c3_fv, &c3_b, c3_fv1, c3_fv2);
  for (c3_i1 = 0; c3_i1 < 56; c3_i1++) {
    (*chartInstance->c3_rtPts)[c3_i1] = c3_fv2[c3_i1];
  }

  for (c3_i2 = 0; c3_i2 < 56; c3_i2++) {
    (*chartInstance->c3_ltPts)[c3_i2] = c3_fv1[c3_i2];
  }

  *chartInstance->c3_laneFound = c3_b;
  c3_do_animation_call_c3_LaneDetection(chartInstance);
}

static void mdl_start_c3_LaneDetection(SFc3_LaneDetectionInstanceStruct
  *chartInstance)
{
}

static void mdl_terminate_c3_LaneDetection(SFc3_LaneDetectionInstanceStruct
  *chartInstance)
{
  hipError_t c3_errCode;
  hipFree(chartInstance->c3_c_gpu_params);
  hipFree(*chartInstance->c3_gpu_dv);
  hipFree(*chartInstance->c3_gpu_ltPts);
  hipFree(*chartInstance->c3_gpu_rt_y);
  hipFree(*chartInstance->c3_c_gpu_U);
  hipFree(*chartInstance->c3_b_gpu_U);
  hipFree(chartInstance->c3_gpu_t2);
  hipFree(*chartInstance->c3_gpu_b);
  hipFree(*chartInstance->c3_b_gpu_params);
  hipFree(*chartInstance->c3_gpu_lt_y);
  hipFree(chartInstance->c3_gpu_params);
  hipFree(chartInstance->c3_b_gpu_t3);
  hipFree(*chartInstance->c3_b_gpu_Tinv);
  hipFree(chartInstance->c3_b_gpu_p3);
  hipFree(chartInstance->c3_b_gpu_t2);
  hipFree(chartInstance->c3_gpu_p1);
  hipFree(*chartInstance->c3_gpu_Tinv);
  hipFree(*chartInstance->c3_gpu_T);
  hipFree(*chartInstance->c3_gpu_rtPts);
  hipFree(chartInstance->c3_gpu_t3);
  hipFree(chartInstance->c3_gpu_p3);
  hipFree(*chartInstance->c3_gpu_dv1);
  hipFree(*chartInstance->c3_gpu_U);
  hipFree(*chartInstance->c3_gpu_fv);
  hipFree(*chartInstance->c3_gpu_laneNetOut);
  hipFree(*chartInstance->c3_gpu_x);
  hipFree(*chartInstance->c3_gpu_fv1);
  hipFree(chartInstance->c3_gpu_p2);
  c3_errCode = hipGetLastError();
  if (c3_errCode != hipSuccess) {
    emlrtThinCUDAError(c3_errCode, hipGetErrorName(c3_errCode),
                       hipGetErrorString(c3_errCode), "SimGPUErrorChecks",
                       chartInstance->c3_fEmlrtCtx);
  }
}

static void mdl_setup_runtime_resources_c3_LaneDetection
  (SFc3_LaneDetectionInstanceStruct *chartInstance)
{
  setLegacyDebuggerFlag(chartInstance->S, false);
  setDebuggerFlag(chartInstance->S, false);
  sim_mode_is_external(chartInstance->S);
}

static void mdl_cleanup_runtime_resources_c3_LaneDetection
  (SFc3_LaneDetectionInstanceStruct *chartInstance)
{
}

static void initSimStructsc3_LaneDetection(SFc3_LaneDetectionInstanceStruct
  *chartInstance)
{
}

const mxArray *sf_c3_LaneDetection_get_eml_resolved_functions_info()
{
  const mxArray *c3_nameCaptureInfo = NULL;
  const char_T *c3_data[9] = {
    "789ced5adf6fd2401cef0c1a13e386899ab807df4cd4258063307952ca8fb93136b6026133064a3968b35e8fb50763beb847dff4c5e85f618cfe01d3bfc007ff"
    "16dfa5408556cf6e160f29bd841cdf7ce87d3efd7e7b9fdc1d65e6d6b3730cc3cc773fdf5e30cc8718d36b57fb1de31ff4171873b3e27383fe8a2536da45c667",
    "ba4ec7bf76f95e0d6201291874703f5078087e5e594350527805e78f9b80518186e436a8f590ba2483bc0401371a6ce9114c8f403f031dd2bf2744201c702dc8"
    "a8a23654288f06bd7ce8ad42b85f9f4d3eaccd9a0febef0cbe93bfe433c65fb4e13370995740b9063010b08494b280905aeba61903cdaca7e250cf25a29e3ea2",
    "494a430643be770ef956887c66fc69ea59504410044528c8c178527ffe542407c9790940fd3abbbc2c9c51a7b51ffefe72afbff5e6469526dfe7475f5668f219"
    "6d527c1dc278677dce6e12f8fc163c5dc8ee84ebd578a4f530c2458e500c24c5223bd491b3e1b1d3c110625ae34fcb7c755aefeb363a0d5c40b0d9c280452da5",
    "c6abc7595403f2a88e8a431de7f5d3d70ef902443e334eaacfeff2a15786d6bcff78ea2bebfdacf81a6d3e5a3ebad1c9840ae938ded3b6847ce6b0d31438b6f4"
    "c4f351da3eda74a8939417bfa56f005cc675a4c23c5a877c034c6a3deed43fefdbf01938a92ed63cf4ab416fbe7fa7bc0ef5374f0334f98ce676ff5485629b13",
    "0ee4dd940852da46683f510c2fa7dce39ffffb3c755ae76b36fa0c1cabbca2c9bc6ee2c5ae97239531eba838d461b7feaca156759cebcf25229f1927d5e5977c"
    "0c0a436bbebf7f4bd73f1f2f2ddea1c96734b7fb673215c225362356b9503aa53d1022cd6a6e37e9f9e7a4e7e959f5f92cf1505f1f5111de1f27df797df2c421",
    "df6d229f1927e55fbf7f9afbf24fdebefc9ff2d1f2c5cd4d454d46b5147b18cab7c5cda298815c21edf9a28b7cb1344ebe29f4c592e78beee1a3e58b8de7dbe9"
    "68211eaee6a05ac81e0b6bb94e0c253c5fa4b5df767a4e396f89adfa0cbceb0f3d73cef258953a93f3c9970ef9ee12f9ccf81f7c72240f01e8f9a53bf868f9e5",
    "b2c26632fcee114809d1d6ca762c1e29262417fdbf336df3738cebc73dbd9fe1f5e39eb77e740f1fb5f563a95d6f706ce32809715d4befb75777422517bd3734"
    "c37ee89d337a7ee81a3eef9c713ce3cfb01f7ae78b9e1fba86cf3b5f1ccff86e3f5f5cb0c4567d062e75e5488a26098303c6697d8fe71e91cf8c93ea61c98351",
    "0ecf375dc247cb37b731bb96603971078695d5f8f241342e48c805fbea1f34f98517", "" };

  c3_nameCaptureInfo = NULL;
  emlrtNameCaptureMxArrayR2016a(&c3_data[0], 14792U, &c3_nameCaptureInfo);
  return c3_nameCaptureInfo;
}

static void c3_eML_blk_kernel(SFc3_LaneDetectionInstanceStruct *chartInstance,
  real32_T c3_b_laneNetOut[6], boolean_T *c3_b_laneFound, real32_T c3_b_ltPts[56],
  real32_T c3_b_rtPts[56])
{
  static real_T c3_T[9] = { 318.9034, -309.4362, 1594.5169999999998, 257.5352,
    -1.5769482917388672E-14, 2037.9982547799998, 1.0, -6.123233995736766E-17,
    5.0 };

  static real_T c3_dv[6] = { -0.0002, 0.0002, 1.474, -0.0002, 0.0045, -1.3787 };

  static real_T c3_dv1[6] = { 0.003, 0.0766, 0.6313, 0.0026, 0.0736, 0.9846 };

  real_T c3_Tinv[9];
  real_T c3_x[9];
  real_T c3_absx11;
  real_T c3_absx21;
  real_T c3_absx31;
  real_T c3_b_absx11;
  real_T c3_b_absx21;
  real_T c3_b_absx31;
  real_T c3_b_t1;
  real_T c3_b_t2;
  real_T c3_b_t3;
  real_T c3_b_z;
  real_T c3_c_t1;
  real_T c3_c_t2;
  real_T c3_c_t3;
  real_T c3_c_z;
  real_T c3_d_z;
  real_T c3_e_z;
  real_T c3_f_z;
  real_T c3_g_z;
  real_T c3_h_z;
  real_T c3_t1;
  real_T c3_t2;
  real_T c3_t3;
  real_T c3_z;
  int32_T c3_b_itmp;
  int32_T c3_b_k;
  int32_T c3_b_p1;
  int32_T c3_b_p2;
  int32_T c3_b_p3;
  int32_T c3_c_p2;
  int32_T c3_c_p3;
  int32_T c3_itmp;
  int32_T c3_k;
  int32_T c3_p1;
  int32_T c3_p2;
  int32_T c3_p3;
  real32_T c3_params[6];
  boolean_T c3_Tinv_dirtyOnCpu;
  boolean_T c3_c_laneFound;
  boolean_T c3_ltPts_dirtyOnGpu;
  boolean_T c3_params_dirtyOnGpu;
  boolean_T c3_rtPts_dirtyOnGpu;
  boolean_T c3_x_dirtyOnCpu;
  boolean_T c3_x_dirtyOnGpu;
  c3_rtPts_dirtyOnGpu = false;
  c3_ltPts_dirtyOnGpu = false;
  if (!chartInstance->c3_laneCoeffMeans_not_empty) {
    hipMemcpy(chartInstance->c3_gpu_dv, &c3_dv[0], 48UL, hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(c3_gpu_laneCoeffMeans), chartInstance->c3_laneCoeffMeans,
                       48UL, 0UL, hipMemcpyHostToDevice);
    c3_eML_blk_kernel_kernel1<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
      (*chartInstance->c3_gpu_dv);
    hipMemcpyFromSymbol(chartInstance->c3_laneCoeffMeans, HIP_SYMBOL(c3_gpu_laneCoeffMeans),
                         48UL, 0UL, hipMemcpyDeviceToHost);
    chartInstance->c3_laneCoeffMeans_not_empty = true;
  }

  if (!chartInstance->c3_laneCoeffStds_not_empty) {
    hipMemcpy(chartInstance->c3_gpu_dv1, &c3_dv1[0], 48UL,
               hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(c3_gpu_laneCoeffStds), chartInstance->c3_laneCoeffStds,
                       48UL, 0UL, hipMemcpyHostToDevice);
    c3_eML_blk_kernel_kernel2<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
      (*chartInstance->c3_gpu_dv1);
    hipMemcpyFromSymbol(chartInstance->c3_laneCoeffStds, HIP_SYMBOL(c3_gpu_laneCoeffStds),
                         48UL, 0UL, hipMemcpyDeviceToHost);
    chartInstance->c3_laneCoeffStds_not_empty = true;
  }

  hipMemcpy(chartInstance->c3_gpu_laneNetOut, &c3_b_laneNetOut[0], 24UL,
             hipMemcpyHostToDevice);
  hipMemcpyToSymbol(HIP_SYMBOL(c3_gpu_laneCoeffStds), chartInstance->c3_laneCoeffStds, 48UL,
                     0UL, hipMemcpyHostToDevice);
  hipMemcpyToSymbol(HIP_SYMBOL(c3_gpu_laneCoeffMeans), chartInstance->c3_laneCoeffMeans,
                     48UL, 0UL, hipMemcpyHostToDevice);
  c3_eML_blk_kernel_kernel3<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
    (*chartInstance->c3_gpu_laneNetOut, *chartInstance->c3_b_gpu_params);
  if (!chartInstance->c3_vehicleXPoints_not_empty) {
    hipMemcpyToSymbol(HIP_SYMBOL(c3_gpu_vehicleXPoints), chartInstance->c3_vehicleXPoints,
                       224UL, 0UL, hipMemcpyHostToDevice);
    c3_eML_blk_kernel_kernel4<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>();
    hipMemcpyFromSymbol(chartInstance->c3_vehicleXPoints, HIP_SYMBOL(c3_gpu_vehicleXPoints),
                         224UL, 0UL, hipMemcpyDeviceToHost);
    chartInstance->c3_vehicleXPoints_not_empty = true;
  }

  hipMemcpy(&c3_params[0], chartInstance->c3_b_gpu_params, 24UL,
             hipMemcpyDeviceToHost);
  c3_params_dirtyOnGpu = false;
  if (muSingleScalarAbs(c3_params[5]) > 0.5F) {
    if (c3_params_dirtyOnGpu) {
      hipMemcpy(&c3_params[0], chartInstance->c3_b_gpu_params, 24UL,
                 hipMemcpyDeviceToHost);
    }

    if (muSingleScalarAbs(c3_params[2]) > 0.5F) {
      c3_eML_blk_kernel_kernel5<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
        (*chartInstance->c3_b_gpu_params, *chartInstance->c3_gpu_rt_y);
      for (c3_k = 0; c3_k < 2; c3_k++) {
        hipMemcpyToSymbol(HIP_SYMBOL(c3_gpu_vehicleXPoints),
                           chartInstance->c3_vehicleXPoints, 224UL, 0UL,
                           hipMemcpyHostToDevice);
        c3_eML_blk_kernel_kernel6<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
          (c3_params[c3_k + 4], *chartInstance->c3_gpu_rt_y);
      }

      c3_eML_blk_kernel_kernel7<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
        (*chartInstance->c3_b_gpu_params, *chartInstance->c3_gpu_lt_y);
      for (c3_b_k = 0; c3_b_k < 2; c3_b_k++) {
        hipMemcpyToSymbol(HIP_SYMBOL(c3_gpu_vehicleXPoints),
                           chartInstance->c3_vehicleXPoints, 224UL, 0UL,
                           hipMemcpyHostToDevice);
        c3_eML_blk_kernel_kernel8<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
          (c3_params[c3_b_k + 1], *chartInstance->c3_gpu_lt_y);
      }

      hipMemcpy(chartInstance->c3_gpu_T, &c3_T[0], 72UL, hipMemcpyHostToDevice);
      c3_eML_blk_kernel_kernel9<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
        (*chartInstance->c3_gpu_T, *chartInstance->c3_gpu_x);
      c3_p2 = 3;
      c3_p3 = 0;
      hipMemcpy(&c3_x[0], chartInstance->c3_gpu_x, 72UL, hipMemcpyDeviceToHost);
      c3_x_dirtyOnGpu = false;
      c3_z = c3_x[1] / 1594.5169999999998;
      if (c3_x_dirtyOnGpu) {
        hipMemcpy(&c3_x[0], chartInstance->c3_gpu_x, 72UL,
                   hipMemcpyDeviceToHost);
      }

      c3_x[1] /= 1594.5169999999998;
      c3_x[2] = 0.2;
      c3_x[4] -= c3_z * 2037.9982547799998;
      c3_x[5] = -150.06445095600003;
      c3_x[7] -= c3_z * 5.0;
      c3_x[8] = 0.0;
      if (150.06445095600003 > muDoubleScalarAbs(c3_x[4])) {
        c3_p2 = 0;
        c3_p3 = 3;
        c3_x[1] = 0.2;
        c3_x[2] = c3_z;
        c3_t1 = c3_x[4];
        c3_x[4] = -150.06445095600003;
        c3_x[5] = c3_t1;
        c3_t1 = c3_x[7];
        c3_x[7] = 0.0;
        c3_x[8] = c3_t1;
      }

      c3_b_z = c3_x[5] / c3_x[4];
      c3_x[5] /= c3_x[4];
      c3_x[8] -= c3_b_z * c3_x[7];
      c3_t3 = (c3_x[5] * c3_x[1] - c3_x[2]) / c3_x[8];
      c3_t2 = -(c3_x[1] + c3_x[7] * c3_t3) / c3_x[4];
      c3_Tinv[6] = ((1.0 - 2037.9982547799998 * c3_t2) - 5.0 * c3_t3) /
        1594.5169999999998;
      c3_Tinv[7] = c3_t2;
      c3_Tinv[8] = c3_t3;
      c3_t3 = -c3_x[5] / c3_x[8];
      c3_t2 = (1.0 - c3_x[7] * c3_t3) / c3_x[4];
      c3_Tinv[c3_p2] = -(2037.9982547799998 * c3_t2 + 5.0 * c3_t3) /
        1594.5169999999998;
      c3_Tinv[c3_p2 + 1] = c3_t2;
      c3_Tinv[c3_p2 + 2] = c3_t3;
      c3_t3 = 1.0 / c3_x[8];
      c3_t2 = -c3_x[7] * c3_t3 / c3_x[4];
      c3_Tinv[c3_p3] = -(2037.9982547799998 * c3_t2 + 5.0 * c3_t3) /
        1594.5169999999998;
      c3_Tinv[c3_p3 + 1] = c3_t2;
      c3_Tinv[c3_p3 + 2] = c3_t3;
      hipMemcpy(chartInstance->c3_gpu_Tinv, &c3_Tinv[0], 72UL,
                 hipMemcpyHostToDevice);
      c3_Tinv_dirtyOnCpu = false;
      c3_eML_blk_kernel_kernel10<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
        (*chartInstance->c3_gpu_Tinv, *chartInstance->c3_gpu_x);
      c3_p1 = 0;
      c3_b_p2 = 3;
      c3_b_p3 = 6;
      c3_absx11 = muDoubleScalarAbs(c3_Tinv[0]);
      c3_absx21 = muDoubleScalarAbs(c3_Tinv[1]);
      c3_absx31 = muDoubleScalarAbs(c3_Tinv[2]);
      if ((c3_absx21 > c3_absx11) && (c3_absx21 > c3_absx31)) {
        c3_p1 = 3;
        c3_b_p2 = 0;
        if (c3_Tinv_dirtyOnCpu) {
          hipMemcpy(chartInstance->c3_gpu_Tinv, &c3_Tinv[0], 72UL,
                     hipMemcpyHostToDevice);
          c3_Tinv_dirtyOnCpu = false;
        }

        c3_eML_blk_kernel_kernel12<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
          (*chartInstance->c3_gpu_Tinv, *chartInstance->c3_gpu_x);
      } else {
        if (c3_absx31 > c3_absx11) {
          c3_p1 = 6;
          c3_b_p3 = 0;
          if (c3_Tinv_dirtyOnCpu) {
            hipMemcpy(chartInstance->c3_gpu_Tinv, &c3_Tinv[0], 72UL,
                       hipMemcpyHostToDevice);
            c3_Tinv_dirtyOnCpu = false;
          }

          c3_eML_blk_kernel_kernel11<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
            (*chartInstance->c3_gpu_Tinv, *chartInstance->c3_gpu_x);
        }
      }

      hipMemcpy(&c3_x[0], chartInstance->c3_gpu_x, 72UL, hipMemcpyDeviceToHost);
      c3_c_z = c3_x[1] / c3_x[0];
      c3_x[1] /= c3_x[0];
      c3_d_z = c3_x[2] / c3_x[0];
      c3_x[2] /= c3_x[0];
      c3_x[4] -= c3_c_z * c3_x[3];
      c3_x[5] -= c3_d_z * c3_x[3];
      c3_x[7] -= c3_c_z * c3_x[6];
      c3_x[8] -= c3_d_z * c3_x[6];
      if (muDoubleScalarAbs(c3_x[5]) > muDoubleScalarAbs(c3_x[4])) {
        c3_itmp = c3_b_p2;
        c3_b_p2 = c3_b_p3;
        c3_b_p3 = c3_itmp;
        c3_x[1] = c3_d_z;
        c3_x[2] = c3_c_z;
        c3_b_t1 = c3_x[4];
        c3_x[4] = c3_x[5];
        c3_x[5] = c3_b_t1;
        c3_b_t1 = c3_x[7];
        c3_x[7] = c3_x[8];
        c3_x[8] = c3_b_t1;
      }

      c3_e_z = c3_x[5] / c3_x[4];
      c3_x[5] /= c3_x[4];
      c3_x[8] -= c3_e_z * c3_x[7];
      c3_b_t3 = (c3_x[5] * c3_x[1] - c3_x[2]) / c3_x[8];
      c3_b_t2 = -(c3_x[1] + c3_x[7] * c3_b_t3) / c3_x[4];
      hipMemcpy(chartInstance->c3_gpu_x, &c3_x[0], 72UL, hipMemcpyHostToDevice);
      c3_x_dirtyOnCpu = false;
      c3_eML_blk_kernel_kernel13<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
        (c3_b_t3, c3_b_t2, *chartInstance->c3_gpu_x, c3_p1,
         *chartInstance->c3_b_gpu_Tinv);
      c3_b_t3 = -c3_x[5] / c3_x[8];
      c3_b_t2 = (1.0 - c3_x[7] * c3_b_t3) / c3_x[4];
      if (c3_x_dirtyOnCpu) {
        hipMemcpy(chartInstance->c3_gpu_x, &c3_x[0], 72UL,
                   hipMemcpyHostToDevice);
      }

      c3_eML_blk_kernel_kernel14<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
        (c3_b_t3, c3_b_t2, *chartInstance->c3_gpu_x, c3_b_p2,
         *chartInstance->c3_b_gpu_Tinv);
      c3_b_t3 = 1.0 / c3_x[8];
      c3_b_t2 = -c3_x[7] * c3_b_t3 / c3_x[4];
      c3_eML_blk_kernel_kernel15<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
        (c3_b_t3, c3_b_t2, *chartInstance->c3_gpu_x, c3_b_p3,
         *chartInstance->c3_b_gpu_Tinv);
      hipMemcpyToSymbol(HIP_SYMBOL(c3_gpu_vehicleXPoints), chartInstance->c3_vehicleXPoints,
                         224UL, 0UL, hipMemcpyHostToDevice);
      c3_eML_blk_kernel_kernel16<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
        (*chartInstance->c3_gpu_lt_y, *chartInstance->c3_gpu_fv);
      c3_eML_blk_kernel_kernel17<<<dim3(1U, 1U, 1U), dim3(96U, 1U, 1U)>>>
        (*chartInstance->c3_b_gpu_Tinv, *chartInstance->c3_gpu_fv,
         *chartInstance->c3_b_gpu_U);
      c3_eML_blk_kernel_kernel18<<<dim3(1U, 1U, 1U), dim3(64U, 1U, 1U)>>>
        (*chartInstance->c3_gpu_U, *chartInstance->c3_b_gpu_U,
         *chartInstance->c3_gpu_b);
      c3_eML_blk_kernel_kernel19<<<dim3(1U, 1U, 1U), dim3(64U, 1U, 1U)>>>
        (*chartInstance->c3_b_gpu_U, *chartInstance->c3_gpu_ltPts);
      c3_ltPts_dirtyOnGpu = true;
      if (c3_Tinv_dirtyOnCpu) {
        hipMemcpy(chartInstance->c3_gpu_Tinv, &c3_Tinv[0], 72UL,
                   hipMemcpyHostToDevice);
      }

      c3_eML_blk_kernel_kernel20<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
        (*chartInstance->c3_gpu_Tinv, *chartInstance->c3_gpu_x);
      c3_b_p1 = 0;
      c3_c_p2 = 3;
      c3_c_p3 = 6;
      c3_b_absx11 = muDoubleScalarAbs(c3_Tinv[0]);
      c3_b_absx21 = muDoubleScalarAbs(c3_Tinv[1]);
      c3_b_absx31 = muDoubleScalarAbs(c3_Tinv[2]);
      if ((c3_b_absx21 > c3_b_absx11) && (c3_b_absx21 > c3_b_absx31)) {
        c3_b_p1 = 3;
        c3_c_p2 = 0;
        c3_eML_blk_kernel_kernel22<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
          (*chartInstance->c3_gpu_Tinv, *chartInstance->c3_gpu_x);
      } else {
        if (c3_b_absx31 > c3_b_absx11) {
          c3_b_p1 = 6;
          c3_c_p3 = 0;
          c3_eML_blk_kernel_kernel21<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
            (*chartInstance->c3_gpu_Tinv, *chartInstance->c3_gpu_x);
        }
      }

      hipMemcpy(&c3_x[0], chartInstance->c3_gpu_x, 72UL, hipMemcpyDeviceToHost);
      c3_f_z = c3_x[1] / c3_x[0];
      c3_x[1] /= c3_x[0];
      c3_g_z = c3_x[2] / c3_x[0];
      c3_x[2] /= c3_x[0];
      c3_x[4] -= c3_f_z * c3_x[3];
      c3_x[5] -= c3_g_z * c3_x[3];
      c3_x[7] -= c3_f_z * c3_x[6];
      c3_x[8] -= c3_g_z * c3_x[6];
      if (muDoubleScalarAbs(c3_x[5]) > muDoubleScalarAbs(c3_x[4])) {
        c3_b_itmp = c3_c_p2;
        c3_c_p2 = c3_c_p3;
        c3_c_p3 = c3_b_itmp;
        c3_x[1] = c3_g_z;
        c3_x[2] = c3_f_z;
        c3_c_t1 = c3_x[4];
        c3_x[4] = c3_x[5];
        c3_x[5] = c3_c_t1;
        c3_c_t1 = c3_x[7];
        c3_x[7] = c3_x[8];
        c3_x[8] = c3_c_t1;
      }

      c3_h_z = c3_x[5] / c3_x[4];
      c3_x[5] /= c3_x[4];
      c3_x[8] -= c3_h_z * c3_x[7];
      c3_c_t3 = (c3_x[5] * c3_x[1] - c3_x[2]) / c3_x[8];
      c3_c_t2 = -(c3_x[1] + c3_x[7] * c3_c_t3) / c3_x[4];
      c3_Tinv[c3_b_p1] = ((1.0 - c3_x[3] * c3_c_t2) - c3_x[6] * c3_c_t3) / c3_x
        [0];
      c3_Tinv[c3_b_p1 + 1] = c3_c_t2;
      c3_Tinv[c3_b_p1 + 2] = c3_c_t3;
      c3_c_t3 = -c3_x[5] / c3_x[8];
      c3_c_t2 = (1.0 - c3_x[7] * c3_c_t3) / c3_x[4];
      c3_Tinv[c3_c_p2] = -(c3_x[3] * c3_c_t2 + c3_x[6] * c3_c_t3) / c3_x[0];
      c3_Tinv[c3_c_p2 + 1] = c3_c_t2;
      c3_Tinv[c3_c_p2 + 2] = c3_c_t3;
      c3_c_t3 = 1.0 / c3_x[8];
      c3_c_t2 = -c3_x[7] * c3_c_t3 / c3_x[4];
      c3_Tinv[c3_c_p3] = -(c3_x[3] * c3_c_t2 + c3_x[6] * c3_c_t3) / c3_x[0];
      hipMemcpy(chartInstance->c3_gpu_Tinv, &c3_Tinv[0], 72UL,
                 hipMemcpyHostToDevice);
      c3_eML_blk_kernel_kernel23<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
        (c3_c_t3, c3_c_t2, c3_c_p3, *chartInstance->c3_gpu_Tinv);
      hipMemcpyToSymbol(HIP_SYMBOL(c3_gpu_vehicleXPoints), chartInstance->c3_vehicleXPoints,
                         224UL, 0UL, hipMemcpyHostToDevice);
      c3_eML_blk_kernel_kernel24<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
        (*chartInstance->c3_gpu_rt_y, *chartInstance->c3_gpu_fv1);
      c3_eML_blk_kernel_kernel25<<<dim3(1U, 1U, 1U), dim3(96U, 1U, 1U)>>>
        (*chartInstance->c3_gpu_Tinv, *chartInstance->c3_gpu_fv1,
         *chartInstance->c3_b_gpu_U);
      c3_eML_blk_kernel_kernel26<<<dim3(1U, 1U, 1U), dim3(64U, 1U, 1U)>>>
        (*chartInstance->c3_c_gpu_U, *chartInstance->c3_b_gpu_U,
         *chartInstance->c3_gpu_b);
      c3_eML_blk_kernel_kernel27<<<dim3(1U, 1U, 1U), dim3(64U, 1U, 1U)>>>
        (*chartInstance->c3_b_gpu_U, *chartInstance->c3_gpu_rtPts);
      c3_rtPts_dirtyOnGpu = true;
      c3_c_laneFound = true;
    } else {
      c3_c_laneFound = false;
    }
  } else {
    c3_c_laneFound = false;
  }

  *c3_b_laneFound = c3_c_laneFound;
  if (c3_ltPts_dirtyOnGpu) {
    hipMemcpy(&c3_b_ltPts[0], chartInstance->c3_gpu_ltPts, 224UL,
               hipMemcpyDeviceToHost);
  }

  if (c3_rtPts_dirtyOnGpu) {
    hipMemcpy(&c3_b_rtPts[0], chartInstance->c3_gpu_rtPts, 224UL,
               hipMemcpyDeviceToHost);
  }
}

static boolean_T c3_emlrt_marshallIn(SFc3_LaneDetectionInstanceStruct
  *chartInstance, const mxArray *c3_b_laneFound, const char_T *c3_identifier)
{
  emlrtMsgIdentifier c3_thisId;
  boolean_T c3_y;
  c3_thisId.fIdentifier = const_cast<const char_T *>(c3_identifier);
  c3_thisId.fParent = NULL;
  c3_thisId.bParentIsCell = false;
  c3_y = c3_b_emlrt_marshallIn(chartInstance, sf_mex_dup(c3_b_laneFound),
    &c3_thisId);
  sf_mex_destroy(&c3_b_laneFound);
  return c3_y;
}

static boolean_T c3_b_emlrt_marshallIn(SFc3_LaneDetectionInstanceStruct
  *chartInstance, const mxArray *c3_u, const emlrtMsgIdentifier *c3_parentId)
{
  boolean_T c3_b;
  boolean_T c3_y;
  sf_mex_import(c3_parentId, sf_mex_dup(c3_u), &c3_b, 1, 11, 0U, 0, 0U, 0);
  c3_y = c3_b;
  sf_mex_destroy(&c3_u);
  return c3_y;
}

static void c3_c_emlrt_marshallIn(SFc3_LaneDetectionInstanceStruct
  *chartInstance, const mxArray *c3_b_ltPts, const char_T *c3_identifier,
  real32_T c3_y[56])
{
  emlrtMsgIdentifier c3_thisId;
  c3_thisId.fIdentifier = const_cast<const char_T *>(c3_identifier);
  c3_thisId.fParent = NULL;
  c3_thisId.bParentIsCell = false;
  c3_d_emlrt_marshallIn(chartInstance, sf_mex_dup(c3_b_ltPts), &c3_thisId, c3_y);
  sf_mex_destroy(&c3_b_ltPts);
}

static void c3_d_emlrt_marshallIn(SFc3_LaneDetectionInstanceStruct
  *chartInstance, const mxArray *c3_u, const emlrtMsgIdentifier *c3_parentId,
  real32_T c3_y[56])
{
  int32_T c3_i;
  real32_T c3_fv[56];
  sf_mex_import(c3_parentId, sf_mex_dup(c3_u), c3_fv, 0, 1, 0U, 1, 0U, 2, 28, 2);
  for (c3_i = 0; c3_i < 56; c3_i++) {
    c3_y[c3_i] = c3_fv[c3_i];
  }

  sf_mex_destroy(&c3_u);
}

static void c3_e_emlrt_marshallIn(SFc3_LaneDetectionInstanceStruct
  *chartInstance, const mxArray *c3_b_laneCoeffMeans, const char_T
  *c3_identifier, boolean_T *c3_svPtr, real_T c3_y[6])
{
  emlrtMsgIdentifier c3_thisId;
  c3_thisId.fIdentifier = const_cast<const char_T *>(c3_identifier);
  c3_thisId.fParent = NULL;
  c3_thisId.bParentIsCell = false;
  c3_f_emlrt_marshallIn(chartInstance, sf_mex_dup(c3_b_laneCoeffMeans),
                        &c3_thisId, c3_svPtr, c3_y);
  sf_mex_destroy(&c3_b_laneCoeffMeans);
}

static void c3_f_emlrt_marshallIn(SFc3_LaneDetectionInstanceStruct
  *chartInstance, const mxArray *c3_u, const emlrtMsgIdentifier *c3_parentId,
  boolean_T *c3_svPtr, real_T c3_y[6])
{
  real_T c3_dv[6];
  int32_T c3_i;
  if (mxIsEmpty(c3_u)) {
    *c3_svPtr = false;
  } else {
    *c3_svPtr = true;
    sf_mex_import(c3_parentId, sf_mex_dup(c3_u), c3_dv, 1, 0, 0U, 1, 0U, 2, 1, 6);
    for (c3_i = 0; c3_i < 6; c3_i++) {
      c3_y[c3_i] = c3_dv[c3_i];
    }
  }

  sf_mex_destroy(&c3_u);
}

static void c3_g_emlrt_marshallIn(SFc3_LaneDetectionInstanceStruct
  *chartInstance, const mxArray *c3_b_vehicleXPoints, const char_T
  *c3_identifier, boolean_T *c3_svPtr, real_T c3_y[28])
{
  emlrtMsgIdentifier c3_thisId;
  c3_thisId.fIdentifier = const_cast<const char_T *>(c3_identifier);
  c3_thisId.fParent = NULL;
  c3_thisId.bParentIsCell = false;
  c3_h_emlrt_marshallIn(chartInstance, sf_mex_dup(c3_b_vehicleXPoints),
                        &c3_thisId, c3_svPtr, c3_y);
  sf_mex_destroy(&c3_b_vehicleXPoints);
}

static void c3_h_emlrt_marshallIn(SFc3_LaneDetectionInstanceStruct
  *chartInstance, const mxArray *c3_u, const emlrtMsgIdentifier *c3_parentId,
  boolean_T *c3_svPtr, real_T c3_y[28])
{
  real_T c3_dv[28];
  int32_T c3_i;
  if (mxIsEmpty(c3_u)) {
    *c3_svPtr = false;
  } else {
    *c3_svPtr = true;
    sf_mex_import(c3_parentId, sf_mex_dup(c3_u), c3_dv, 1, 0, 0U, 1, 0U, 2, 1,
                  28);
    for (c3_i = 0; c3_i < 28; c3_i++) {
      c3_y[c3_i] = c3_dv[c3_i];
    }
  }

  sf_mex_destroy(&c3_u);
}

static uint8_T c3_i_emlrt_marshallIn(SFc3_LaneDetectionInstanceStruct
  *chartInstance, const mxArray *c3_b_is_active_c3_LaneDetection, const char_T
  *c3_identifier)
{
  emlrtMsgIdentifier c3_thisId;
  uint8_T c3_y;
  c3_thisId.fIdentifier = const_cast<const char_T *>(c3_identifier);
  c3_thisId.fParent = NULL;
  c3_thisId.bParentIsCell = false;
  c3_y = c3_j_emlrt_marshallIn(chartInstance, sf_mex_dup
    (c3_b_is_active_c3_LaneDetection), &c3_thisId);
  sf_mex_destroy(&c3_b_is_active_c3_LaneDetection);
  return c3_y;
}

static uint8_T c3_j_emlrt_marshallIn(SFc3_LaneDetectionInstanceStruct
  *chartInstance, const mxArray *c3_u, const emlrtMsgIdentifier *c3_parentId)
{
  uint8_T c3_b_u;
  uint8_T c3_y;
  sf_mex_import(c3_parentId, sf_mex_dup(c3_u), &c3_b_u, 1, 3, 0U, 0, 0U, 0);
  c3_y = c3_b_u;
  sf_mex_destroy(&c3_u);
  return c3_y;
}

static __global__ __launch_bounds__(32, 1) void c3_eML_blk_kernel_kernel1(const
  real_T c3_dv[6])
{
  int32_T c3_i4;
  c3_i4 = (int32_T)mwGetGlobalThreadIndex();
  if (c3_i4 < 6) {
    c3_gpu_laneCoeffMeans[c3_i4] = c3_dv[c3_i4];
  }
}

static __global__ __launch_bounds__(32, 1) void c3_eML_blk_kernel_kernel2(const
  real_T c3_dv1[6])
{
  int32_T c3_i5;
  c3_i5 = (int32_T)mwGetGlobalThreadIndex();
  if (c3_i5 < 6) {
    c3_gpu_laneCoeffStds[c3_i5] = c3_dv1[c3_i5];
  }
}

static __global__ __launch_bounds__(32, 1) void c3_eML_blk_kernel_kernel3(const
  real32_T c3_b_laneNetOut[6], real32_T c3_params[6])
{
  int32_T c3_i6;
  c3_i6 = (int32_T)mwGetGlobalThreadIndex();
  if (c3_i6 < 6) {
    c3_params[c3_i6] = c3_b_laneNetOut[c3_i6] * (real32_T)
      c3_gpu_laneCoeffStds[c3_i6] + (real32_T)c3_gpu_laneCoeffMeans[c3_i6];
  }
}

static __global__ __launch_bounds__(32, 1) void c3_eML_blk_kernel_kernel4()
{
  int32_T c3_i7;
  c3_i7 = (int32_T)mwGetGlobalThreadIndex();
  if (c3_i7 < 28) {
    c3_gpu_vehicleXPoints[c3_i7] = (real_T)c3_i7 + 3.0;
  }
}

static __global__ __launch_bounds__(32, 1) void c3_eML_blk_kernel_kernel5(const
  real32_T c3_params[6], real32_T c3_rt_y[28])
{
  int32_T c3_i8;
  c3_i8 = (int32_T)mwGetGlobalThreadIndex();
  if (c3_i8 < 28) {
    c3_rt_y[c3_i8] = c3_params[3];
  }
}

static __global__ __launch_bounds__(32, 1) void c3_eML_blk_kernel_kernel6(const
  real32_T c3_params, real32_T c3_rt_y[28])
{
  int32_T c3_i10;
  c3_i10 = (int32_T)mwGetGlobalThreadIndex();
  if (c3_i10 < 28) {
    c3_rt_y[c3_i10] = (real32_T)c3_gpu_vehicleXPoints[c3_i10] * c3_rt_y[c3_i10]
      + c3_params;
  }
}

static __global__ __launch_bounds__(32, 1) void c3_eML_blk_kernel_kernel7(const
  real32_T c3_params[6], real32_T c3_lt_y[28])
{
  int32_T c3_i9;
  c3_i9 = (int32_T)mwGetGlobalThreadIndex();
  if (c3_i9 < 28) {
    c3_lt_y[c3_i9] = c3_params[0];
  }
}

static __global__ __launch_bounds__(32, 1) void c3_eML_blk_kernel_kernel8(const
  real32_T c3_params, real32_T c3_lt_y[28])
{
  int32_T c3_i11;
  c3_i11 = (int32_T)mwGetGlobalThreadIndex();
  if (c3_i11 < 28) {
    c3_lt_y[c3_i11] = (real32_T)c3_gpu_vehicleXPoints[c3_i11] * c3_lt_y[c3_i11]
      + c3_params;
  }
}

static __global__ __launch_bounds__(32, 1) void c3_eML_blk_kernel_kernel9(const
  real_T c3_T[9], real_T c3_x[9])
{
  int32_T c3_i12;
  c3_i12 = (int32_T)mwGetGlobalThreadIndex();
  if (c3_i12 < 9) {
    c3_x[c3_i12] = c3_T[c3_i12];
  }
}

static __global__ __launch_bounds__(32, 1) void c3_eML_blk_kernel_kernel10(const
  real_T c3_Tinv[9], real_T c3_x[9])
{
  int32_T c3_i13;
  c3_i13 = (int32_T)mwGetGlobalThreadIndex();
  if (c3_i13 < 9) {
    c3_x[c3_i13] = c3_Tinv[c3_i13];
  }
}

static __global__ __launch_bounds__(32, 1) void c3_eML_blk_kernel_kernel11(const
  real_T c3_Tinv[9], real_T c3_x[9])
{
  int32_T c3_tmpIdx;
  c3_tmpIdx = (int32_T)mwGetGlobalThreadIndex();
  if (c3_tmpIdx < 1) {
    c3_x[0] = c3_Tinv[2];
    c3_x[2] = c3_Tinv[0];
    c3_x[3] = c3_Tinv[5];
    c3_x[5] = c3_Tinv[3];
    c3_x[6] = c3_Tinv[8];
    c3_x[8] = c3_Tinv[6];
  }
}

static __global__ __launch_bounds__(32, 1) void c3_eML_blk_kernel_kernel12(const
  real_T c3_Tinv[9], real_T c3_x[9])
{
  int32_T c3_tmpIdx;
  c3_tmpIdx = (int32_T)mwGetGlobalThreadIndex();
  if (c3_tmpIdx < 1) {
    c3_x[0] = c3_Tinv[1];
    c3_x[1] = c3_Tinv[0];
    c3_x[3] = c3_Tinv[4];
    c3_x[4] = c3_Tinv[3];
    c3_x[6] = c3_Tinv[7];
    c3_x[7] = c3_Tinv[6];
  }
}

static __global__ __launch_bounds__(32, 1) void c3_eML_blk_kernel_kernel13(const
  real_T c3_t3, const real_T c3_t2, const real_T c3_x[9], const int32_T c3_p1,
  real_T c3_Tinv[9])
{
  int32_T c3_tmpIdx;
  c3_tmpIdx = (int32_T)mwGetGlobalThreadIndex();
  if (c3_tmpIdx < 1) {
    c3_Tinv[c3_p1] = ((1.0 - c3_x[3] * c3_t2) - c3_x[6] * c3_t3) / c3_x[0];
    c3_Tinv[c3_p1 + 1] = c3_t2;
    c3_Tinv[c3_p1 + 2] = c3_t3;
  }
}

static __global__ __launch_bounds__(32, 1) void c3_eML_blk_kernel_kernel14(const
  real_T c3_t3, const real_T c3_t2, const real_T c3_x[9], const int32_T c3_p2,
  real_T c3_Tinv[9])
{
  int32_T c3_tmpIdx;
  c3_tmpIdx = (int32_T)mwGetGlobalThreadIndex();
  if (c3_tmpIdx < 1) {
    c3_Tinv[c3_p2] = -(c3_x[3] * c3_t2 + c3_x[6] * c3_t3) / c3_x[0];
    c3_Tinv[c3_p2 + 1] = c3_t2;
    c3_Tinv[c3_p2 + 2] = c3_t3;
  }
}

static __global__ __launch_bounds__(32, 1) void c3_eML_blk_kernel_kernel15(const
  real_T c3_t3, const real_T c3_t2, const real_T c3_x[9], const int32_T c3_p3,
  real_T c3_Tinv[9])
{
  int32_T c3_tmpIdx;
  c3_tmpIdx = (int32_T)mwGetGlobalThreadIndex();
  if (c3_tmpIdx < 1) {
    c3_Tinv[c3_p3] = -(c3_x[3] * c3_t2 + c3_x[6] * c3_t3) / c3_x[0];
    c3_Tinv[c3_p3 + 1] = c3_t2;
    c3_Tinv[c3_p3 + 2] = c3_t3;
  }
}

static __global__ __launch_bounds__(32, 1) void c3_eML_blk_kernel_kernel16(const
  real32_T c3_lt_y[28], real32_T c3_fv[84])
{
  int32_T c3_i14;
  c3_i14 = (int32_T)mwGetGlobalThreadIndex();
  if (c3_i14 < 28) {
    c3_fv[c3_i14] = (real32_T)c3_gpu_vehicleXPoints[c3_i14];
    c3_fv[c3_i14 + 28] = c3_lt_y[c3_i14];
    c3_fv[c3_i14 + 56] = 1.0F;
  }
}

static __global__ __launch_bounds__(96, 1) void c3_eML_blk_kernel_kernel17(const
  real_T c3_Tinv[9], const real32_T c3_fv[84], real32_T c3_U[84])
{
  uint64_T c3_threadId;
  int32_T c3_i15;
  int32_T c3_i16;
  int32_T c3_i17;
  c3_threadId = mwGetGlobalThreadIndex();
  c3_i17 = (int32_T)(c3_threadId % 3UL);
  c3_i15 = (int32_T)((c3_threadId - (uint64_T)c3_i17) / 3UL);
  if ((c3_i15 < 28) && (c3_i17 < 3)) {
    c3_U[c3_i15 + 28 * c3_i17] = 0.0F;
    for (c3_i16 = 0; c3_i16 < 3; c3_i16++) {
      c3_U[c3_i15 + 28 * c3_i17] += c3_fv[c3_i15 + 28 * c3_i16] * (real32_T)
        c3_Tinv[c3_i16 + 3 * c3_i17];
    }
  }
}

static __global__ __launch_bounds__(64, 1) void c3_eML_blk_kernel_kernel18
  (real32_T c3_U[56], real32_T c3_b_U[84], real32_T c3_b[56])
{
  uint64_T c3_threadId;
  int32_T c3_jtilecol;
  int32_T c3_k;
  c3_threadId = mwGetGlobalThreadIndex();
  c3_k = (int32_T)(c3_threadId % 28UL);
  c3_jtilecol = (int32_T)((c3_threadId - (uint64_T)c3_k) / 28UL);
  if ((c3_jtilecol < 2) && (c3_k < 28)) {
    c3_b[c3_jtilecol * 28 + c3_k] = c3_b_U[c3_k + 56];
    c3_U[c3_k + 28 * c3_jtilecol] = c3_b_U[c3_k + 28 * c3_jtilecol] / c3_b[c3_k
      + 28 * c3_jtilecol];
    c3_b_U[c3_k + 28 * c3_jtilecol] = c3_U[c3_k + 28 * c3_jtilecol];
  }
}

static __global__ __launch_bounds__(64, 1) void c3_eML_blk_kernel_kernel19(const
  real32_T c3_U[84], real32_T c3_b_ltPts[56])
{
  uint64_T c3_threadId;
  int32_T c3_i18;
  int32_T c3_i20;
  c3_threadId = mwGetGlobalThreadIndex();
  c3_i20 = (int32_T)(c3_threadId % 28UL);
  c3_i18 = (int32_T)((c3_threadId - (uint64_T)c3_i20) / 28UL);
  if ((c3_i18 < 2) && (c3_i20 < 28)) {
    c3_b_ltPts[c3_i20 + 28 * c3_i18] = c3_U[c3_i20 + 28 * c3_i18];
  }
}

static __global__ __launch_bounds__(32, 1) void c3_eML_blk_kernel_kernel20(const
  real_T c3_Tinv[9], real_T c3_x[9])
{
  int32_T c3_i19;
  c3_i19 = (int32_T)mwGetGlobalThreadIndex();
  if (c3_i19 < 9) {
    c3_x[c3_i19] = c3_Tinv[c3_i19];
  }
}

static __global__ __launch_bounds__(32, 1) void c3_eML_blk_kernel_kernel21(const
  real_T c3_Tinv[9], real_T c3_x[9])
{
  int32_T c3_tmpIdx;
  c3_tmpIdx = (int32_T)mwGetGlobalThreadIndex();
  if (c3_tmpIdx < 1) {
    c3_x[0] = c3_Tinv[2];
    c3_x[2] = c3_Tinv[0];
    c3_x[3] = c3_Tinv[5];
    c3_x[5] = c3_Tinv[3];
    c3_x[6] = c3_Tinv[8];
    c3_x[8] = c3_Tinv[6];
  }
}

static __global__ __launch_bounds__(32, 1) void c3_eML_blk_kernel_kernel22(const
  real_T c3_Tinv[9], real_T c3_x[9])
{
  int32_T c3_tmpIdx;
  c3_tmpIdx = (int32_T)mwGetGlobalThreadIndex();
  if (c3_tmpIdx < 1) {
    c3_x[0] = c3_Tinv[1];
    c3_x[1] = c3_Tinv[0];
    c3_x[3] = c3_Tinv[4];
    c3_x[4] = c3_Tinv[3];
    c3_x[6] = c3_Tinv[7];
    c3_x[7] = c3_Tinv[6];
  }
}

static __global__ __launch_bounds__(32, 1) void c3_eML_blk_kernel_kernel23(const
  real_T c3_t3, const real_T c3_t2, const int32_T c3_p3, real_T c3_Tinv[9])
{
  int32_T c3_tmpIdx;
  c3_tmpIdx = (int32_T)mwGetGlobalThreadIndex();
  if (c3_tmpIdx < 1) {
    c3_Tinv[c3_p3 + 1] = c3_t2;
    c3_Tinv[c3_p3 + 2] = c3_t3;
  }
}

static __global__ __launch_bounds__(32, 1) void c3_eML_blk_kernel_kernel24(const
  real32_T c3_rt_y[28], real32_T c3_fv1[84])
{
  int32_T c3_i21;
  c3_i21 = (int32_T)mwGetGlobalThreadIndex();
  if (c3_i21 < 28) {
    c3_fv1[c3_i21] = (real32_T)c3_gpu_vehicleXPoints[c3_i21];
    c3_fv1[c3_i21 + 28] = c3_rt_y[c3_i21];
    c3_fv1[c3_i21 + 56] = 1.0F;
  }
}

static __global__ __launch_bounds__(96, 1) void c3_eML_blk_kernel_kernel25(const
  real_T c3_Tinv[9], const real32_T c3_fv1[84], real32_T c3_U[84])
{
  uint64_T c3_threadId;
  int32_T c3_i22;
  int32_T c3_i23;
  int32_T c3_i24;
  c3_threadId = mwGetGlobalThreadIndex();
  c3_i24 = (int32_T)(c3_threadId % 3UL);
  c3_i22 = (int32_T)((c3_threadId - (uint64_T)c3_i24) / 3UL);
  if ((c3_i22 < 28) && (c3_i24 < 3)) {
    c3_U[c3_i22 + 28 * c3_i24] = 0.0F;
    for (c3_i23 = 0; c3_i23 < 3; c3_i23++) {
      c3_U[c3_i22 + 28 * c3_i24] += c3_fv1[c3_i22 + 28 * c3_i23] * (real32_T)
        c3_Tinv[c3_i23 + 3 * c3_i24];
    }
  }
}

static __global__ __launch_bounds__(64, 1) void c3_eML_blk_kernel_kernel26
  (real32_T c3_U[56], real32_T c3_b_U[84], real32_T c3_b[56])
{
  uint64_T c3_threadId;
  int32_T c3_jtilecol;
  int32_T c3_k;
  c3_threadId = mwGetGlobalThreadIndex();
  c3_k = (int32_T)(c3_threadId % 28UL);
  c3_jtilecol = (int32_T)((c3_threadId - (uint64_T)c3_k) / 28UL);
  if ((c3_jtilecol < 2) && (c3_k < 28)) {
    c3_b[c3_jtilecol * 28 + c3_k] = c3_b_U[c3_k + 56];
    c3_U[c3_k + 28 * c3_jtilecol] = c3_b_U[c3_k + 28 * c3_jtilecol] / c3_b[c3_k
      + 28 * c3_jtilecol];
    c3_b_U[c3_k + 28 * c3_jtilecol] = c3_U[c3_k + 28 * c3_jtilecol];
  }
}

static __global__ __launch_bounds__(64, 1) void c3_eML_blk_kernel_kernel27(const
  real32_T c3_U[84], real32_T c3_b_rtPts[56])
{
  uint64_T c3_threadId;
  int32_T c3_i25;
  int32_T c3_i26;
  c3_threadId = mwGetGlobalThreadIndex();
  c3_i26 = (int32_T)(c3_threadId % 28UL);
  c3_i25 = (int32_T)((c3_threadId - (uint64_T)c3_i26) / 28UL);
  if ((c3_i25 < 2) && (c3_i26 < 28)) {
    c3_b_rtPts[c3_i26 + 28 * c3_i25] = c3_U[c3_i26 + 28 * c3_i25];
  }
}

static void init_dsm_address_info(SFc3_LaneDetectionInstanceStruct
  *chartInstance)
{
}

static void init_simulink_io_address(SFc3_LaneDetectionInstanceStruct
  *chartInstance)
{
  chartInstance->c3_fEmlrtCtx = (void *)sfrtGetEmlrtCtx(chartInstance->S);
  chartInstance->c3_laneNetOut = (real32_T (*)[6])ssGetInputPortSignal_wrapper
    (chartInstance->S, 0);
  chartInstance->c3_laneFound = (boolean_T *)ssGetOutputPortSignal_wrapper
    (chartInstance->S, 1);
  chartInstance->c3_ltPts = (real32_T (*)[56])ssGetOutputPortSignal_wrapper
    (chartInstance->S, 2);
  chartInstance->c3_rtPts = (real32_T (*)[56])ssGetOutputPortSignal_wrapper
    (chartInstance->S, 3);
}

#include <stdio.h>
#include <stdlib.h>
#include <string.h>

/* SFunction Glue Code */
void sf_c3_LaneDetection_get_check_sum(mxArray *plhs[])
{
  ((real_T *)mxGetPr((plhs[0])))[0] = (real_T)(2051119261U);
  ((real_T *)mxGetPr((plhs[0])))[1] = (real_T)(4225097776U);
  ((real_T *)mxGetPr((plhs[0])))[2] = (real_T)(1871082434U);
  ((real_T *)mxGetPr((plhs[0])))[3] = (real_T)(2287114376U);
}

mxArray *sf_c3_LaneDetection_third_party_uses_info(void)
{
  mxArray * mxcell3p = mxCreateCellMatrix(1,0);
  return(mxcell3p);
}

mxArray *sf_c3_LaneDetection_jit_fallback_info(void)
{
  const char *infoFields[] = { "fallbackType", "fallbackReason",
    "hiddenFallbackType", "hiddenFallbackReason", "incompatibleSymbol" };

  mxArray *mxInfo = mxCreateStructMatrix(1, 1, 5, infoFields);
  mxArray *fallbackType = mxCreateString("pre");
  mxArray *fallbackReason = mxCreateString("GPUAcceleration");
  mxArray *hiddenFallbackType = mxCreateString("none");
  mxArray *hiddenFallbackReason = mxCreateString("");
  mxArray *incompatibleSymbol = mxCreateString("");
  mxSetField(mxInfo, 0, infoFields[0], fallbackType);
  mxSetField(mxInfo, 0, infoFields[1], fallbackReason);
  mxSetField(mxInfo, 0, infoFields[2], hiddenFallbackType);
  mxSetField(mxInfo, 0, infoFields[3], hiddenFallbackReason);
  mxSetField(mxInfo, 0, infoFields[4], incompatibleSymbol);
  return mxInfo;
}

mxArray *sf_c3_LaneDetection_updateBuildInfo_args_info(void)
{
  mxArray *mxBIArgs = sf_mex_decode(
    "eNpjYPT0ZQACPiDm4GRgYAPRQMzIAAGsSHxOJHGQ+h8M+NUzoal3QFLPgkU9H5J6ASg/MSUlOL+"
    "0KDnVLTMntRgi1kDAXkY0ewMI2CuDZi+I7xvumpOam5pXUp5ZnOqYlpaZl+qTWJlapJdcUAAz98"
    "MA+b+CRP97ELBXAs1eCbD/3UqLU1Oc8/PKglJ9QuF+B5n3YoD8XUCivwnZy41mL4ifnJcXn1iQC"
    "YvmYRXPgmj2CoLj2dnPDxy7nrkFOXrJpcMwf8uj2SuPJ3/DQgFkLkjTQPi/g0T/RxCwVxXNXlUC"
    "/vdOLcpLhaaF4ZQOpNHslcZRziHnhOGU/8XQ7BUD+z8ksSg9tcQvtaQ8vygb7vXhFO+SaPZKQsq"
    "90uKS/FxwhLvlFzmXuvj5wUt9BgYA8ZCiqg=="
    );
  return mxBIArgs;
}

static const mxArray *sf_get_sim_state_info_c3_LaneDetection(void)
{
  const char *infoFields[] = { "chartChecksum", "varInfo" };

  mxArray *mxInfo = mxCreateStructMatrix(1, 1, 2, infoFields);
  mxArray *mxVarInfo = sf_mex_decode(
    "eNrtlkFLwzAUx9OyTXtY6cGDiId9Aot48WbEIRQcFLx4qyFNaaFNapsNveoX2Efxo/kRTNps1lA"
    "suOImGng83oP/S3/hn1BgeDMgli3i1QJgJPK+CBPUa6hqQ8SeynV/sO5PRPCnnMh+WWAvFJmirK"
    "rR/NGjEavmn4OP+aOW+UZjvqX69Xq72Ex/AKUeNvSDFr3V0DuqThEl12xOQ9W3G3l7PIdQ17fxD"
    "DUeWafc5+XOcBx/m6PolcOBm+nr/bv8ZWsctvLXFSNRNCOIlorD+PoeGto9NFUtNSlIQN7yPW08"
    "psbjrGkCWCWIYB/+iDvO5VQ7F1m7McuIG2c4dS+nmFFesNSVhxWEhBPME0YDzFgRJhRxUp5kv8g"
    "HY4133PTBLQ9Xvt66D5Z57YP7h38f/NB7sCBxglNy57OE8l15D5bPtQ/Ayx/zwaQXH/gdvEcar6"
    "yTMkCCakECfBbcCM7pCvPzf8g7obOKrw=="
    );
  mxArray *mxChecksum = mxCreateDoubleMatrix(1, 4, mxREAL);
  sf_c3_LaneDetection_get_check_sum(&mxChecksum);
  mxSetField(mxInfo, 0, infoFields[0], mxChecksum);
  mxSetField(mxInfo, 0, infoFields[1], mxVarInfo);
  return mxInfo;
}

static const char* sf_get_instance_specialization(void)
{
  return "sc88SziHwZbSUvEPWN0mVPE";
}

static void sf_opaque_initialize_c3_LaneDetection(void *chartInstanceVar)
{
  initialize_params_c3_LaneDetection((SFc3_LaneDetectionInstanceStruct*)
    chartInstanceVar);
  initialize_c3_LaneDetection((SFc3_LaneDetectionInstanceStruct*)
    chartInstanceVar);
}

static void sf_opaque_enable_c3_LaneDetection(void *chartInstanceVar)
{
  enable_c3_LaneDetection((SFc3_LaneDetectionInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_disable_c3_LaneDetection(void *chartInstanceVar)
{
  disable_c3_LaneDetection((SFc3_LaneDetectionInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_gateway_c3_LaneDetection(void *chartInstanceVar)
{
  sf_gateway_c3_LaneDetection((SFc3_LaneDetectionInstanceStruct*)
    chartInstanceVar);
}

static const mxArray* sf_opaque_get_sim_state_c3_LaneDetection(SimStruct* S)
{
  return get_sim_state_c3_LaneDetection((SFc3_LaneDetectionInstanceStruct *)
    sf_get_chart_instance_ptr(S));     /* raw sim ctx */
}

static void sf_opaque_set_sim_state_c3_LaneDetection(SimStruct* S, const mxArray
  *st)
{
  set_sim_state_c3_LaneDetection((SFc3_LaneDetectionInstanceStruct*)
    sf_get_chart_instance_ptr(S), st);
}

static void sf_opaque_cleanup_runtime_resources_c3_LaneDetection(void
  *chartInstanceVar)
{
  if (chartInstanceVar!=NULL) {
    SimStruct *S = ((SFc3_LaneDetectionInstanceStruct*) chartInstanceVar)->S;
    if (sim_mode_is_rtw_gen(S) || sim_mode_is_external(S)) {
      sf_clear_rtw_identifier(S);
      unload_LaneDetection_optimization_info();
    }

    mdl_cleanup_runtime_resources_c3_LaneDetection
      ((SFc3_LaneDetectionInstanceStruct*) chartInstanceVar);
    ((SFc3_LaneDetectionInstanceStruct*) chartInstanceVar)->
      ~SFc3_LaneDetectionInstanceStruct();
    utFree(chartInstanceVar);
    if (ssGetUserData(S)!= NULL) {
      sf_free_ChartRunTimeInfo(S);
    }

    ssSetUserData(S,NULL);
  }
}

static void sf_opaque_mdl_start_c3_LaneDetection(void *chartInstanceVar)
{
  mdl_start_c3_LaneDetection((SFc3_LaneDetectionInstanceStruct*)
    chartInstanceVar);
}

static void sf_opaque_mdl_terminate_c3_LaneDetection(void *chartInstanceVar)
{
  mdl_terminate_c3_LaneDetection((SFc3_LaneDetectionInstanceStruct*)
    chartInstanceVar);
}

static void sf_opaque_init_subchart_simstructs(void *chartInstanceVar)
{
  initSimStructsc3_LaneDetection((SFc3_LaneDetectionInstanceStruct*)
    chartInstanceVar);
}

extern unsigned int sf_machine_global_initializer_called(void);
static void mdlProcessParameters_c3_LaneDetection(SimStruct *S)
{
  int i;
  for (i=0;i<ssGetNumRunTimeParams(S);i++) {
    if (ssGetSFcnParamTunable(S,i)) {
      ssUpdateDlgParamAsRunTimeParam(S,i);
    }
  }

  sf_warn_if_symbolic_dimension_param_changed(S);
  if (sf_machine_global_initializer_called()) {
    initialize_params_c3_LaneDetection((SFc3_LaneDetectionInstanceStruct*)
      sf_get_chart_instance_ptr(S));
    initSimStructsc3_LaneDetection((SFc3_LaneDetectionInstanceStruct*)
      sf_get_chart_instance_ptr(S));
  }
}

const char* sf_c3_LaneDetection_get_post_codegen_info(void)
{
  int i;
  const char* encStrCodegen [25] = {
    "eNrtWM1v40QUd0t3oYhWFVoJtFqJnoDTph+7qCdImw81Uj+iTdtFXLJT+zkeMp5xZ8ZJizjBAY7",
    "8CRw5cuA/4MYFcYP/ghMnDrxx3DR17KRpS5pALaXu2L/3/Hvv/ebN2NZMZdfCYxF/y+9b1kM8v4",
    "G/WatzPIjHMz2/zvU564N4/BMa8dCvEkl8ZQ08OPHhBSjBQk0Fr3BXpMIod0ECtxEbCKmzvCnqh",
    "4zyZjnktvGnXnrU9mqeCJmzhbbE2efsDL0Foa6inyKVYOsygKM9KcKGV2ak0WUsdbvggd1UoT8o",
    "BAW6FgaGltoNmaYBg9Ip2BWuNEHG6oJbTRMNBX2aGaaJVNXOgcIPGCU8NVqPqBoEmGANh4GDf/d",
    "DjUElYbZHpN4Cj7RA7dBm5FNwSPqkCm8cU060kJSwks8KxrCfW5Uhn13hABuQEOS2JYE0A0G5zq",
    "5/rYyRljg5ZlCE47CR7a0GJ6Ep/hGFNsjMvLkF0QJJGrDPMx8aJaR0GlWrq5J+mKY+HBG5aWP9F",
    "DiZ6kXlqBrBOsEBWmTBIAqyog4kbWF6M72FfsUoc9iUCf1OsdUwWOSt1IJBVeh6K9u8QBhTmbAD",
    "EexAC1jktUg0GQzreE3HKUWdA4EJNvLOng0hp1j4GFYQ3KGp5WolAFHf2cPGchlph0oLv4DiLe7",
    "s9N/uh1W4BukSG9K6gCRUAeYsSm+2N4cqU3sEIisd0UsDdxQyDGUpN+TFtpBNzMmAJnIRgqloJt",
    "BXDawlzoRDhZNmEMzUchjOJrYHjmkwlMEuThvEpuREmda2ifOuRfVZEZQtaZBS1RBnHbahkhHUW",
    "QCHvMlFm5el8Gtxj+9UATsD9nAfa3AQzTFuoyuqNLYLevF4BwBFSSSnvLGFbU6elZFkasXMurdi",
    "Xax7b11h3Tu3S54/7PEzk+LH6jknnzs/O/i5s/jfTGyX77FbSDxnLmFncEvmWX/88tejX7e//Pb",
    "n539/89uLjZs8/9Wbo+0TFuPxk/OG3J1grT5dG+x2D6+5FP/v9PhfisfK3tiofUG3258d1w5bpe",
    "rLvRX/qFqK/P04P5jv6wm+59eXzcqAaox0LO2KE29gzJiEnWXd+N/o4ftwSD7m4+ud489Pbmb/K",
    "J/UQ1q+5hP5MmNc2KEsQu6k6Pju4nk3n7RPi+dBIh4zZroarXqTEceTa8chbzWOpfzN7Pv7TVoc",
    "i4k4FmN9FQS47i4Q0+wjHjOj9Y3ZeGxsmEWtIIVPWjyziXiWutHU89EpT/K3oQ9vSF5WE3kx45w",
    "nfMh5vs1ym0VcvLQULGeSVXdAQ7Q/rdtCSMfszkE99adIBwuJeBd6dVDTzrmu71wH3wUdHbw6ud",
    "fBmPpBC/DVnMGn1fglcTJ08FVHB9bX/zMdLN+KDqpD4n2ciPdx9N5eJ+ZtAOr2en0H4yyeh9m/D",
    "7nu/nRUO+ve7t5uDHYz13zfvK7dazd8vx2X3U3jG/W9e9LwKwP6qJXAL01wHDf9HvJv43+3Rttv",
    "vBePP+5+wix4lDkpX7Pi2ztA3LS7/xGdmkk5CP92Av/9kP2BSOhaZO2HFAvk57m6cs3p0qYBLwJ",
    "zc+ZDaU5JO2dzXteSUA6Owe2BXrm8y1ipu7a5sFZvPz2mfAp4Hl+Zp5/g6Y+FJzptrcfZtCac52",
    "qczUnnuTYlPFdHqnuQ4BmMhadDNKkrmzAwVCc7n8+mpO7rU8Lz+ZT0pbWReN7lerQ6wrp5l3WfD",
    "n0+u1T3H4bwPEnwPBlf/xSuq0DHXK+6v2sm+DbHpNOPutWfdJ7tkXje7bzv5PQfX5E00Q==",
    ""
  };

  static char newstr [1797] = "";
  newstr[0] = '\0';
  for (i = 0; i < 25; i++) {
    strcat(newstr, encStrCodegen[i]);
  }

  return newstr;
}

static void mdlSetWorkWidths_c3_LaneDetection(SimStruct *S)
{
  const char* newstr = sf_c3_LaneDetection_get_post_codegen_info();
  sf_set_work_widths(S, newstr);
  ssSetChecksum0(S,(4140489008U));
  ssSetChecksum1(S,(2085146388U));
  ssSetChecksum2(S,(4231381896U));
  ssSetChecksum3(S,(944951687U));
}

static void mdlRTW_c3_LaneDetection(SimStruct *S)
{
  if (sim_mode_is_rtw_gen(S)) {
    ssWriteRTWStrParam(S, "StateflowChartType", "Embedded MATLAB");
  }
}

static void mdlSetupRuntimeResources_c3_LaneDetection(SimStruct *S)
{
  SFc3_LaneDetectionInstanceStruct *chartInstance;
  chartInstance = (SFc3_LaneDetectionInstanceStruct *)utMalloc(sizeof
    (SFc3_LaneDetectionInstanceStruct));
  if (chartInstance==NULL) {
    sf_mex_error_message("Could not allocate memory for chart instance.");
  }

  memset(chartInstance, 0, sizeof(SFc3_LaneDetectionInstanceStruct));
  chartInstance = new (chartInstance) SFc3_LaneDetectionInstanceStruct;
  chartInstance->chartInfo.chartInstance = chartInstance;
  if (ssGetSampleTime(S, 0) == CONTINUOUS_SAMPLE_TIME && ssGetOffsetTime(S, 0) ==
      0 && ssGetNumContStates(ssGetRootSS(S)) > 0) {
    sf_error_out_about_continuous_sample_time_with_persistent_vars(S);
  }

  chartInstance->chartInfo.isEMLChart = 1;
  chartInstance->chartInfo.chartInitialized = 0;
  chartInstance->chartInfo.sFunctionGateway = sf_opaque_gateway_c3_LaneDetection;
  chartInstance->chartInfo.initializeChart =
    sf_opaque_initialize_c3_LaneDetection;
  chartInstance->chartInfo.mdlStart = sf_opaque_mdl_start_c3_LaneDetection;
  chartInstance->chartInfo.mdlTerminate =
    sf_opaque_mdl_terminate_c3_LaneDetection;
  chartInstance->chartInfo.mdlCleanupRuntimeResources =
    sf_opaque_cleanup_runtime_resources_c3_LaneDetection;
  chartInstance->chartInfo.enableChart = sf_opaque_enable_c3_LaneDetection;
  chartInstance->chartInfo.disableChart = sf_opaque_disable_c3_LaneDetection;
  chartInstance->chartInfo.getSimState =
    sf_opaque_get_sim_state_c3_LaneDetection;
  chartInstance->chartInfo.setSimState =
    sf_opaque_set_sim_state_c3_LaneDetection;
  chartInstance->chartInfo.getSimStateInfo =
    sf_get_sim_state_info_c3_LaneDetection;
  chartInstance->chartInfo.zeroCrossings = NULL;
  chartInstance->chartInfo.outputs = NULL;
  chartInstance->chartInfo.derivatives = NULL;
  chartInstance->chartInfo.mdlRTW = mdlRTW_c3_LaneDetection;
  chartInstance->chartInfo.mdlSetWorkWidths = mdlSetWorkWidths_c3_LaneDetection;
  chartInstance->chartInfo.callGetHoverDataForMsg = NULL;
  chartInstance->chartInfo.extModeExec = NULL;
  chartInstance->chartInfo.restoreLastMajorStepConfiguration = NULL;
  chartInstance->chartInfo.restoreBeforeLastMajorStepConfiguration = NULL;
  chartInstance->chartInfo.storeCurrentConfiguration = NULL;
  chartInstance->chartInfo.callAtomicSubchartUserFcn = NULL;
  chartInstance->chartInfo.callAtomicSubchartAutoFcn = NULL;
  chartInstance->chartInfo.callAtomicSubchartEventFcn = NULL;
  chartInstance->chartInfo.chartStateSetterFcn = NULL;
  chartInstance->chartInfo.chartStateGetterFcn = NULL;
  chartInstance->S = S;
  chartInstance->chartInfo.dispatchToExportedFcn = NULL;
  sf_init_ChartRunTimeInfo(S, &(chartInstance->chartInfo), false, 0,
    chartInstance->c3_JITStateAnimation,
    chartInstance->c3_JITTransitionAnimation);
  init_dsm_address_info(chartInstance);
  init_simulink_io_address(chartInstance);
  if (!sim_mode_is_rtw_gen(S)) {
  }

  mdl_setup_runtime_resources_c3_LaneDetection(chartInstance);
}

void c3_LaneDetection_method_dispatcher(SimStruct *S, int_T method, void *data)
{
  switch (method) {
   case SS_CALL_MDL_SETUP_RUNTIME_RESOURCES:
    mdlSetupRuntimeResources_c3_LaneDetection(S);
    break;

   case SS_CALL_MDL_SET_WORK_WIDTHS:
    mdlSetWorkWidths_c3_LaneDetection(S);
    break;

   case SS_CALL_MDL_PROCESS_PARAMETERS:
    mdlProcessParameters_c3_LaneDetection(S);
    break;

   default:
    /* Unhandled method */
    sf_mex_error_message("Stateflow Internal Error:\n"
                         "Error calling c3_LaneDetection_method_dispatcher.\n"
                         "Can't handle method %d.\n", method);
    break;
  }
}
