#include "hip/hip_runtime.h"
#include <cstdlib>
#include <cassert>
#include <stdio.h>
#include <stdexcept>
#include "MWCNNLayerImpl.hpp"
#include "MWTargetNetworkImpl.hpp"
#include "cnn_api.hpp"
#ifdef RANDOM
#include <hiprand.h>
 hiprandGenerator_t RAtlBpdedvgxUsgDTsch; void 
curand_call_line_file(hiprandStatus_t rlQsibXJSWJVnUVpdNeL, const int 
atVCyzqXZAZxwlkRLBRA, const char *QMNXyOvXaZDsCpiIJPsn) { if (rlQsibXJSWJVnUVpdNeL != 
HIPRAND_STATUS_SUCCESS) { char buffer[100]; int numElem = sprintf(buffer, 
"%d at line: %d, file: %s\n", rlQsibXJSWJVnUVpdNeL, atVCyzqXZAZxwlkRLBRA, 
QMNXyOvXaZDsCpiIJPsn); throw std::runtime_error(buffer); } }
#endif
 float* malloc_call_line_file(size_t msize, const int atVCyzqXZAZxwlkRLBRA, const 
char *QMNXyOvXaZDsCpiIJPsn) { float * mem = (float*)malloc(msize); if (!mem) { char 
buffer[100]; int numElem = sprintf(buffer, "%s at line: %d, file: %s\n", 
"Memory allocation failed. ", atVCyzqXZAZxwlkRLBRA, QMNXyOvXaZDsCpiIJPsn); throw 
std::runtime_error(buffer); } return mem; } void 
cuda_call_line_file(hipError_t rlQsibXJSWJVnUVpdNeL, const int atVCyzqXZAZxwlkRLBRA, 
const char *QMNXyOvXaZDsCpiIJPsn) { if (rlQsibXJSWJVnUVpdNeL != hipSuccess) { 
throw_cuda_error(rlQsibXJSWJVnUVpdNeL, atVCyzqXZAZxwlkRLBRA, QMNXyOvXaZDsCpiIJPsn);  } } 
void throw_cuda_error(hipError_t rlQsibXJSWJVnUVpdNeL, const int atVCyzqXZAZxwlkRLBRA, 
const char *QMNXyOvXaZDsCpiIJPsn) { char buffer[100]; int numElem = sprintf(buffer, 
"Cuda Error %d(%s) at line: %d, file: %s\n", rlQsibXJSWJVnUVpdNeL, 
hipGetErrorString(rlQsibXJSWJVnUVpdNeL), atVCyzqXZAZxwlkRLBRA, QMNXyOvXaZDsCpiIJPsn); 
rlQsibXJSWJVnUVpdNeL = hipGetLastError();  throw std::runtime_error(buffer);  } 
void cudnn_call_line_file(hipdnnStatus_t rlQsibXJSWJVnUVpdNeL, const int 
atVCyzqXZAZxwlkRLBRA, const char *QMNXyOvXaZDsCpiIJPsn) { if (rlQsibXJSWJVnUVpdNeL != 
HIPDNN_STATUS_SUCCESS) { char buffer[100]; int numElem = sprintf(buffer, 
"CuDNN Error %d(%s) at line: %d, file: %s\n", rlQsibXJSWJVnUVpdNeL, 
hipdnnGetErrorString(rlQsibXJSWJVnUVpdNeL), atVCyzqXZAZxwlkRLBRA, QMNXyOvXaZDsCpiIJPsn); 
throw std::runtime_error(buffer); } } const char* 
cublasGetErrorString(hipblasStatus_t rlQsibXJSWJVnUVpdNeL) { 
switch(rlQsibXJSWJVnUVpdNeL) { case HIPBLAS_STATUS_SUCCESS: return 
"HIPBLAS_STATUS_SUCCESS"; case HIPBLAS_STATUS_NOT_INITIALIZED: return 
"HIPBLAS_STATUS_NOT_INITIALIZED"; case HIPBLAS_STATUS_ALLOC_FAILED: return 
"HIPBLAS_STATUS_ALLOC_FAILED"; case HIPBLAS_STATUS_INVALID_VALUE: return 
"HIPBLAS_STATUS_INVALID_VALUE";  case HIPBLAS_STATUS_ARCH_MISMATCH: return 
"HIPBLAS_STATUS_ARCH_MISMATCH";  case HIPBLAS_STATUS_MAPPING_ERROR: return 
"HIPBLAS_STATUS_MAPPING_ERROR"; case HIPBLAS_STATUS_EXECUTION_FAILED: return 
"HIPBLAS_STATUS_EXECUTION_FAILED";  case HIPBLAS_STATUS_INTERNAL_ERROR: return 
"HIPBLAS_STATUS_INTERNAL_ERROR";  case HIPBLAS_STATUS_NOT_SUPPORTED: return 
"HIPBLAS_STATUS_NOT_SUPPORTED";  case HIPBLAS_STATUS_UNKNOWN: return 
"HIPBLAS_STATUS_UNKNOWN";  } return "unknown error"; } void 
cublas_call_line_file(hipblasStatus_t rlQsibXJSWJVnUVpdNeL, const int 
atVCyzqXZAZxwlkRLBRA, const char *QMNXyOvXaZDsCpiIJPsn) { if (rlQsibXJSWJVnUVpdNeL != 
HIPBLAS_STATUS_SUCCESS) { char buffer[100]; int numElem = sprintf(buffer, 
"CuBlas Error %d(%s) at line: %d, file: %s\n", rlQsibXJSWJVnUVpdNeL, 
cublasGetErrorString(rlQsibXJSWJVnUVpdNeL), atVCyzqXZAZxwlkRLBRA, QMNXyOvXaZDsCpiIJPsn); 
throw std::runtime_error(buffer); } } 
MWCNNLayerImpl::MWCNNLayerImpl(MWCNNLayer* layer, MWTargetNetworkImpl* 
ntwk_impl) : PVBPDNaynqYkBlDZgXgj(0.0), OwortPcLToImGdYFtbSF(1.0), 
OumvfgWXDdmsQaciHMHx(-1.0), aPzBTLIjCXEQZUlbxayX(layer), 
dJcdBfQQLhIAYHPxwQeg(ntwk_impl) { } MWCNNLayerImpl::~MWCNNLayerImpl() { 
for(std::map<int, hipdnnTensorDescriptor_t*>::iterator it = 
kqftrrQBBOgGsrDSkIUk.begin(); it != kqftrrQBBOgGsrDSkIUk.end(); ++it) { 
CUDNN_CALL(hipdnnDestroyTensorDescriptor(*it->second)); delete it->second; 
it->second = 0; } } template <class T> void 
MWCNNLayerImpl::allocateOutputData(int outIdx) { MWTensorBase* opTensorBase = 
getLayer()->getOutputTensor(outIdx); bool bufferReuse = 
opTensorBase->getopBufIndex() >= 0; if (bufferReuse) { 
assert(opTensorBase->isFloat()); MWTensor<float>* opTensor = 
static_cast<MWTensor<float>*>(opTensorBase); 
opTensor->setData(dJcdBfQQLhIAYHPxwQeg->memBuffer[opTensor->getopBufIndex()]); } 
else { int inIdx = getLayer()->getInPlaceIndex(outIdx); if (inIdx != -1) { 
MWTensor<T>* ipTensor = 
static_cast<MWTensor<T>*>(getLayer()->getInputTensor(inIdx)); MWTensor<T>* 
opTensor = static_cast<MWTensor<T>*>(opTensorBase); T* ipData = 
ipTensor->getData(); assert(ipData); opTensor->setData(ipData); } else { 
MWTensor<T>* opTensor = static_cast<MWTensor<T>*>(opTensorBase); T* 
OAKPrVDonUthXHZkRzEc;  CUDA_CALL(hipMalloc((void**)&OAKPrVDonUthXHZkRzEc, 
sizeof(T)*opTensor->getNumElements())); opTensor->setData(OAKPrVDonUthXHZkRzEc); } } 
} template void MWCNNLayerImpl::allocateOutputData<float>(int); template void 
MWCNNLayerImpl::allocateOutputData<signed char>(int); template <class T> void 
MWCNNLayerImpl::deallocateOutputData(int outIdx) { 
if(getLayer()->getInPlaceIndex(outIdx) == -1) { MWTensor<T>* opTensor = 
static_cast<MWTensor<T>*>(getLayer()->getOutputTensor(outIdx)); T* data = 
opTensor->getData(); CUDA_FREE_CALL(data); } } template void 
MWCNNLayerImpl::deallocateOutputData<float>(int); template void 
MWCNNLayerImpl::deallocateOutputData<signed char>(int); float* 
MWCNNLayerImpl::getZeroPtr() { return &PVBPDNaynqYkBlDZgXgj; } float* 
MWCNNLayerImpl::getOnePtr() { return &OwortPcLToImGdYFtbSF; } float* 
MWCNNLayerImpl::getNegOnePtr() { return &OumvfgWXDdmsQaciHMHx; } 
hipdnnTensorDescriptor_t* MWCNNLayerImpl::createAndAddDescriptor(int index) { 
std::map<int, hipdnnTensorDescriptor_t*>::iterator it = 
kqftrrQBBOgGsrDSkIUk.find(index); assert(it == kqftrrQBBOgGsrDSkIUk.end()); 
hipdnnTensorDescriptor_t* newDescriptor = new hipdnnTensorDescriptor_t; if 
(!newDescriptor) { MWCNNLayerImpl::throwAllocationError(__LINE__ , __FILE__); } 
kqftrrQBBOgGsrDSkIUk[index] = newDescriptor; 
CUDNN_CALL(hipdnnCreateTensorDescriptor(newDescriptor)); return newDescriptor; } 
hipdnnTensorDescriptor_t* MWCNNLayerImpl::getDescriptor(int index) {  
std::map<int, hipdnnTensorDescriptor_t*>::iterator it = 
kqftrrQBBOgGsrDSkIUk.find(index); if (it != kqftrrQBBOgGsrDSkIUk.end()) { 
return it->second; } else { return NULL; } } template <class T> void 
MWCNNLayerImpl::setDescriptor(hipdnnTensorDescriptor_t& desc, MWTensor<T>* 
tensor) { if (tensor->getSequenceLength() == 1) { 
CUDNN_CALL(hipdnnSetTensor4dDescriptor(desc, HIPDNN_TENSOR_NCHW, 
MWCNNLayerImpl::getCuDNNDataType<T>(), tensor->getBatchSize(),  
tensor->getChannels(),  tensor->getHeight(),  tensor->getWidth()));  } else { 
int dims[5] = {tensor->getSequenceLength(), tensor->getBatchSize(), 
tensor->getChannels(), tensor->getHeight(), tensor->getWidth()}; int 
strides[5]; MWTensorBase::getStrides(dims, 5, strides); 
CUDNN_CALL(hipdnnSetTensorNdDescriptor(desc, 
MWCNNLayerImpl::getCuDNNDataType<T>(), 5, dims, strides)); }  } template void 
MWCNNLayerImpl::setDescriptor<float>(hipdnnTensorDescriptor_t&, 
MWTensor<float>*); template void MWCNNLayerImpl::setDescriptor<signed 
char>(hipdnnTensorDescriptor_t&, MWTensor<signed char>*); template <> 
hipdnnDataType_t MWCNNLayerImpl::getCuDNNDataType<float>() { return 
HIPDNN_DATA_FLOAT; } template <> hipdnnDataType_t 
MWCNNLayerImpl::getCuDNNDataType<signed char>() { return HIPDNN_DATA_INT8; } 
hipdnnTensorDescriptor_t MWCNNLayerImpl::getCuDNNDescriptor(MWTensorBase* 
tensor) { MWCNNLayer* layer = tensor->getOwner(); MWCNNLayerImpl* impl = 
layer->getImpl(); if(impl) { hipdnnTensorDescriptor_t* desc = 
impl->getDescriptor(tensor->getSourcePortIndex()); if (desc == NULL) { 
impl->createAndAddDescriptor(tensor->getSourcePortIndex()); desc = 
impl->getDescriptor(tensor->getSourcePortIndex()); assert(desc);  } if 
(tensor->isFloat()) { MWCNNLayerImpl::setDescriptor<float>(*desc, 
static_cast<MWTensor<float>*>(tensor)); } else { assert(tensor->isInt8()); 
MWCNNLayerImpl::setDescriptor<signed char>(*desc, static_cast<MWTensor<signed 
char>*>(tensor)); } return *desc; } else { hipdnnTensorDescriptor_t 
tmpDescriptor; CUDNN_CALL(hipdnnCreateTensorDescriptor(&tmpDescriptor)); if 
(tensor->isFloat()) { MWCNNLayerImpl::setDescriptor<float>(tmpDescriptor, 
static_cast<MWTensor<float>*>(tensor)); } else { assert(tensor->isInt8()); 
MWCNNLayerImpl::setDescriptor<signed char>(tmpDescriptor, 
static_cast<MWTensor<signed char>*>(tensor)); } return tmpDescriptor; } } void 
__global__ __launch_bounds__(1024) padInputImpl(float* in, int inputH, int 
inputW, int inputCh, int outputH, int outputW, int offsetH, int offsetW, float* 
out, int inputElems) { for(int i = blockDim.x * blockIdx.x + threadIdx.x; i < 
inputElems; i+= blockDim.x*gridDim.x) { int idxB = i/(inputH*inputW*inputCh); 
int rem = (i - idxB*(inputH*inputW*inputCh)); int idxCh = rem/(inputH*inputW); 
int rem1 = rem - idxCh*(inputH*inputW); int idxH = rem1/inputW; int idxCol = 
rem1 - idxH*inputW; if ((idxH < inputH) && (idxCol < inputW)) { int outputR = 
idxH + offsetH; int outputCol = idxCol + offsetW; int outputCh = inputCh; 
out[idxB*(outputH*outputW*outputCh) + idxCh*(outputH*outputW) + 
outputR*(outputW) + outputCol] = in[i]; } } } void 
MWCNNLayerImpl::padInput(float* TaAJDyqFVJXfAfCJhOuU, int VFKMunbyHoAmpHUSkuUn, int 
WIxRBCJtmETvfxpuRuus, int VCbcPxtPsBLTrHYdEvqn, int lHtftnmGBvlSSoGOXVui, int 
lkGLRakytrdNuJCcpYWt, int gTcJMwtYuwiqqUmqvKhT, int gzSTokDHvkXefhiGDcWL, float* 
jmcFOAbZArjGDNhshSro, int enPbWLzEmxYCBmzGJutZ) { int tGsvtyAVkrDznETdweDC = 
(enPbWLzEmxYCBmzGJutZ + 31)/32 * 32; tGsvtyAVkrDznETdweDC = 
(tGsvtyAVkrDznETdweDC < 1024) ? tGsvtyAVkrDznETdweDC : 1024; int 
KHClOltUSuqFVVErSxVb = (enPbWLzEmxYCBmzGJutZ + tGsvtyAVkrDznETdweDC - 
1)/tGsvtyAVkrDznETdweDC; padInputImpl<<<KHClOltUSuqFVVErSxVb, 
tGsvtyAVkrDznETdweDC>>>(TaAJDyqFVJXfAfCJhOuU, VFKMunbyHoAmpHUSkuUn, 
WIxRBCJtmETvfxpuRuus, VCbcPxtPsBLTrHYdEvqn, lHtftnmGBvlSSoGOXVui, lkGLRakytrdNuJCcpYWt, 
gTcJMwtYuwiqqUmqvKhT, gzSTokDHvkXefhiGDcWL, jmcFOAbZArjGDNhshSro, enPbWLzEmxYCBmzGJutZ); } 
void __global__ __launch_bounds__(1024) fillOutputBufferImpl(signed char* in, 
int inputH, int inputW, int inputCh, int outputH, int outputW, int offsetH, int 
offsetW, signed char* out, int inputElems, int outputCh) { for(int i = 
blockDim.x * blockIdx.x + threadIdx.x; i < inputElems; i+= 
blockDim.x*gridDim.x) { int idxB = i/(inputH*inputW*inputCh); int rem = (i - 
idxB*(inputH*inputW*inputCh)); int idxCh = rem/(inputH*inputW); int rem1 = rem 
- idxCh*(inputH*inputW); int idxH = rem1/inputW; int idxCol = rem1 - 
idxH*inputW; if ((idxH < inputH) && (idxCol < inputW)) { int outputR = idxH + 
offsetH; int outputCol = idxCol + offsetW; *(out + 
idxB*(outputH*outputW*outputCh) + idxCh*(outputH*outputW) + outputR*(outputW) + 
outputCol) = *(in + i); } } } void MWCNNLayerImpl::fillOutputBuffer(signed 
char* TaAJDyqFVJXfAfCJhOuU, int VFKMunbyHoAmpHUSkuUn, int WIxRBCJtmETvfxpuRuus, int 
VCbcPxtPsBLTrHYdEvqn, int lHtftnmGBvlSSoGOXVui, int lkGLRakytrdNuJCcpYWt, int 
gTcJMwtYuwiqqUmqvKhT, int gzSTokDHvkXefhiGDcWL, signed char* jmcFOAbZArjGDNhshSro, int 
enPbWLzEmxYCBmzGJutZ, int kkqTyvjYvRFtTOyQUwrF) { int tGsvtyAVkrDznETdweDC 
= (enPbWLzEmxYCBmzGJutZ < 1024) ? enPbWLzEmxYCBmzGJutZ : 1024; int 
KHClOltUSuqFVVErSxVb = (enPbWLzEmxYCBmzGJutZ + tGsvtyAVkrDznETdweDC - 
1)/tGsvtyAVkrDznETdweDC; fillOutputBufferImpl<<<KHClOltUSuqFVVErSxVb, 
tGsvtyAVkrDznETdweDC>>>(TaAJDyqFVJXfAfCJhOuU, VFKMunbyHoAmpHUSkuUn, 
WIxRBCJtmETvfxpuRuus, VCbcPxtPsBLTrHYdEvqn, lHtftnmGBvlSSoGOXVui, lkGLRakytrdNuJCcpYWt, 
gTcJMwtYuwiqqUmqvKhT, gzSTokDHvkXefhiGDcWL, jmcFOAbZArjGDNhshSro, enPbWLzEmxYCBmzGJutZ, 
kkqTyvjYvRFtTOyQUwrF); } void MWCNNLayerImpl::throwAllocationError(const int 
line, const char * file) { char buffer[200]; int numElem = sprintf(buffer, 
"Failed to allocate memory at %d, file %s\n", line, file); throw 
std::runtime_error(buffer); } MWReLULayerImpl::MWReLULayerImpl(MWCNNLayer* 
layer, MWTargetNetworkImpl* ntwk_impl)  : MWCNNLayerImpl(layer, ntwk_impl) { 
CUDNN_CALL(hipdnnCreateActivationDescriptor(&oJUVMnJggjhEdQLWzIUC)); 
createAndAddDescriptor(getLayer()->getOutputTensor(0)->getSourcePortIndex()); } 
MWReLULayerImpl::~MWReLULayerImpl() { } void MWReLULayerImpl::propagateSize() { 
MWTensorBase* opTensor = getLayer()->getOutputTensor(0); 
hipdnnTensorDescriptor_t* desc = getDescriptor(opTensor->getSourcePortIndex()); 
assert(desc); setDescriptor<float>(*desc, 
static_cast<MWTensor<float>*>(opTensor)); 
CUDNN_CALL(hipdnnSetActivationDescriptor(oJUVMnJggjhEdQLWzIUC, 
HIPDNN_ACTIVATION_RELU, HIPDNN_NOT_PROPAGATE_NAN, 0));  } void 
MWReLULayerImpl::predict() { MWReLULayer* reluLayer = 
static_cast<MWReLULayer*>(getLayer()); MWTensorBase* ipTensorBase = 
reluLayer->getInputTensor(0); MWTensorBase* opTensorBase = 
reluLayer->getOutputTensor(0); MWTensor<float>* ipTensor = 
static_cast<MWTensor<float>*>(ipTensorBase); MWTensor<float>* opTensor = 
static_cast<MWTensor<float>*>(opTensorBase); hipdnnTensorDescriptor_t* desc = 
getDescriptor(opTensor->getSourcePortIndex()); assert(desc); 
hipdnnTensorDescriptor_t ipDesc = 
MWCNNLayerImpl::getCuDNNDescriptor(ipTensorBase); 
CUDNN_CALL(hipdnnActivationForward(*dJcdBfQQLhIAYHPxwQeg->getCudnnHandle(), 
oJUVMnJggjhEdQLWzIUC, getOnePtr(), ipDesc, ipTensor->getData(), getZeroPtr(), 
*desc, opTensor->getData())); } void MWReLULayerImpl::cleanup() { 
CUDNN_CALL(hipdnnDestroyActivationDescriptor(oJUVMnJggjhEdQLWzIUC)); } 
MWNormLayerImpl::MWNormLayerImpl(MWCNNLayer* layer, MWTargetNetworkImpl* 
ntwk_impl, unsigned GDRXdUDklKFEYEfifhIH,  double AFQBkxwYGKLsACiDKwRM,  
double AHqhysOOIgbDpWZoPUFT,  double BUOdotSvmFyUWQKMUdra) : MWCNNLayerImpl(layer, 
ntwk_impl)  { CUDNN_CALL(hipdnnCreateLRNDescriptor(&dAGMlbhOYuZqhuDGCqih)); 
createAndAddDescriptor(getLayer()->getOutputTensor(0)->getSourcePortIndex()); 
CUDNN_CALL(hipdnnSetLRNDescriptor(dAGMlbhOYuZqhuDGCqih, 
GDRXdUDklKFEYEfifhIH, AFQBkxwYGKLsACiDKwRM, AHqhysOOIgbDpWZoPUFT, 
BUOdotSvmFyUWQKMUdra)); } MWNormLayerImpl::~MWNormLayerImpl() { } void 
MWNormLayerImpl::propagateSize() { MWTensorBase* opTensor = 
getLayer()->getOutputTensor(0); hipdnnTensorDescriptor_t* desc = 
getDescriptor(opTensor->getSourcePortIndex()); assert(desc); 
setDescriptor<float>(*desc, static_cast<MWTensor<float>*>(opTensor));  } void 
MWNormLayerImpl::predict() { MWNormLayer* normLayer = 
static_cast<MWNormLayer*>(getLayer()); MWTensorBase* ipTensorBase = 
normLayer->getInputTensor();  MWTensorBase* opTensorBase = 
normLayer->getOutputTensor(); MWTensor<float>* ipTensor = 
static_cast<MWTensor<float>*>(ipTensorBase); MWTensor<float>* opTensor = 
static_cast<MWTensor<float>*>(opTensorBase); hipdnnTensorDescriptor_t* desc = 
getDescriptor(opTensor->getSourcePortIndex()); assert(desc); 
hipdnnTensorDescriptor_t ipDesc = 
MWCNNLayerImpl::getCuDNNDescriptor(ipTensorBase); 
CUDNN_CALL(hipdnnLRNCrossChannelForward(*dJcdBfQQLhIAYHPxwQeg->getCudnnHandle(), 
dAGMlbhOYuZqhuDGCqih, HIPDNN_LRN_CROSS_CHANNEL, getOnePtr(), ipDesc, 
ipTensor->getData(), getZeroPtr(), *desc, opTensor->getData())); } void 
MWNormLayerImpl::cleanup() { 
CUDNN_CALL(hipdnnDestroyLRNDescriptor(dAGMlbhOYuZqhuDGCqih)); } void __global__ 
MWSetDyForBackPropImpl(float * PQjbchiGbyJfmpiqPpOC, const int fDqxEdcpBDmVQxZEmQxm); 
void __global__ doMWMaxPoolingLayerImpl(float * UWAGLbDcvybdWBtshhsr, 
float * UVzBVEOIylFjkSgHwFMp, const int BdqURaHPmdnfzvtUvocl); 
MWMaxPoolingLayerImpl::MWMaxPoolingLayerImpl(MWCNNLayer* layer, 
MWTargetNetworkImpl* ntwk_impl, int DSsxcjIrUgZCKZovyNQf,  int 
EfvWctmlsWAPsxXgdKWf,  int FrpxvsDMwwgbpqHXWxmN,  int 
FwLnexHgxHRquTKmNpoa, int ClEhcJFlvGCgiavziIag, int 
CZNYmBcNFSZWvaCklqeM,  int CufLFODQDXTAPyRqYodN, int 
DCdZnqpcBnvXVgEsLBnz, bool GIbahSoBBDrvvZduPEqU, int fOpFYwKNwIfWjnPzNuob) 
: MWCNNLayerImpl(layer, ntwk_impl) , 
BRSPqxNffoBYKqpSVHne(GIbahSoBBDrvvZduPEqU) , UWAGLbDcvybdWBtshhsr(0) 
, PQjbchiGbyJfmpiqPpOC(0) , DRzwhbNPpftRRIXXfHzd(DSsxcjIrUgZCKZovyNQf) , 
ECTnqgWHyHCHCLBZlffd(EfvWctmlsWAPsxXgdKWf) , 
DGzdAcREJHGXjyRzNjJV(DSsxcjIrUgZCKZovyNQf) , 
DqxLTLaJwwgQqmrtCDuu(EfvWctmlsWAPsxXgdKWf) , 
CTCbzQMDaLxINPbODdng(ClEhcJFlvGCgiavziIag) , 
CLOUhPjbgggWoXHTtmjC(CZNYmBcNFSZWvaCklqeM) , 
CpMjJjtGOeWOzwxpAAQP(CufLFODQDXTAPyRqYodN) , 
CqtPRJvHlGJFssiPzsOm(DCdZnqpcBnvXVgEsLBnz) , 
FpguQZSermqZCMRiUfML(FrpxvsDMwwgbpqHXWxmN) , 
FshVHIJMRAhtQirYPlZd(FwLnexHgxHRquTKmNpoa) , 
fSbUUBgjKRbNXrHrlOLo(fOpFYwKNwIfWjnPzNuob) {  
CUDNN_CALL(hipdnnCreatePoolingDescriptor(&mtolGPkUMBYDlSSqrRzc)); 
createAndAddDescriptor(getLayer()->getOutputTensor(0)->getSourcePortIndex()); } 
MWMaxPoolingLayerImpl::~MWMaxPoolingLayerImpl() { } void 
MWMaxPoolingLayerImpl::propagateSize() {  MWTensorBase* ipTensor = 
getLayer()->getInputTensor(0); MWTensorBase* opTensor = 
getLayer()->getOutputTensor(0); if ((DRzwhbNPpftRRIXXfHzd == -1) && 
(ECTnqgWHyHCHCLBZlffd == -1)) { DGzdAcREJHGXjyRzNjJV = ipTensor->getHeight(); 
DqxLTLaJwwgQqmrtCDuu = ipTensor->getWidth(); } int muwRQxtWMMXAPxSuMYBw = 
CTCbzQMDaLxINPbODdng; int nDsbARncmIrIaLubvLVZ = 
CpMjJjtGOeWOzwxpAAQP; 
CUDNN_CALL(hipdnnSetPooling2dDescriptor(mtolGPkUMBYDlSSqrRzc, HIPDNN_POOLING_MAX, 
HIPDNN_NOT_PROPAGATE_NAN, DGzdAcREJHGXjyRzNjJV, DqxLTLaJwwgQqmrtCDuu, 
muwRQxtWMMXAPxSuMYBw, nDsbARncmIrIaLubvLVZ, FpguQZSermqZCMRiUfML, 
FshVHIJMRAhtQirYPlZd)); hipdnnTensorDescriptor_t* desc = 
getDescriptor(opTensor->getSourcePortIndex()); assert(desc); 
setDescriptor<float>(*desc, static_cast<MWTensor<float>*>(opTensor)); } void 
MWMaxPoolingLayerImpl::allocate() { MWMaxPoolingLayer* maxpoolLayer = 
static_cast<MWMaxPoolingLayer*>(getLayer()); MWTensorBase* ipTensor = 
maxpoolLayer->getInputTensor(0); MWTensorBase* opTensor = 
maxpoolLayer->getOutputTensor(0); if (BRSPqxNffoBYKqpSVHne){ const int 
eVAFqeShtGZAZluKdMvQ = ipTensor->getNumElements(); 
CUDA_CALL(hipMalloc((void**)&UWAGLbDcvybdWBtshhsr, 
sizeof(float)*eVAFqeShtGZAZluKdMvQ)); const int fDqxEdcpBDmVQxZEmQxm = 
opTensor->getNumElements(); CUDA_CALL(hipMalloc((void**)&PQjbchiGbyJfmpiqPpOC, 
sizeof(float)*fDqxEdcpBDmVQxZEmQxm)); int tGsvtyAVkrDznETdweDC = 
(fDqxEdcpBDmVQxZEmQxm < 1024) ? fDqxEdcpBDmVQxZEmQxm : 1024; int 
KHClOltUSuqFVVErSxVb = (fDqxEdcpBDmVQxZEmQxm + tGsvtyAVkrDznETdweDC - 
1)/tGsvtyAVkrDznETdweDC; 
MWSetDyForBackPropImpl<<<KHClOltUSuqFVVErSxVb, 
tGsvtyAVkrDznETdweDC>>>( PQjbchiGbyJfmpiqPpOC, fDqxEdcpBDmVQxZEmQxm); } } void 
MWMaxPoolingLayerImpl::deallocate() { if (UWAGLbDcvybdWBtshhsr){ 
CUDA_FREE_CALL(UWAGLbDcvybdWBtshhsr); UWAGLbDcvybdWBtshhsr = 
NULL; } if (PQjbchiGbyJfmpiqPpOC){ CUDA_FREE_CALL(PQjbchiGbyJfmpiqPpOC); PQjbchiGbyJfmpiqPpOC = 
NULL; }  } void MWMaxPoolingLayerImpl::predict() { MWMaxPoolingLayer* 
maxpoolLayer = static_cast<MWMaxPoolingLayer*>(getLayer()); MWTensorBase* 
ipTensorBase = maxpoolLayer->getInputTensor(0); MWTensorBase* opTensorBase = 
maxpoolLayer->getOutputTensor(0); MWTensor<float>* ipTensor = 
static_cast<MWTensor<float>*>(ipTensorBase); MWTensor<float>* opTensor = 
static_cast<MWTensor<float>*>(opTensorBase); hipdnnTensorDescriptor_t* desc = 
getDescriptor(opTensor->getSourcePortIndex()); assert(desc); 
hipdnnTensorDescriptor_t XYbzSmRQGatVJtGmDZSo = 
MWCNNLayerImpl::getCuDNNDescriptor(ipTensorBase); 
CUDNN_CALL(hipdnnPoolingForward(*dJcdBfQQLhIAYHPxwQeg->getCudnnHandle(), 
mtolGPkUMBYDlSSqrRzc, getOnePtr(), XYbzSmRQGatVJtGmDZSo, ipTensor->getData(), 
getZeroPtr(), *desc, opTensor->getData())); if (BRSPqxNffoBYKqpSVHne) { 
CUDNN_CALL(hipdnnPoolingBackward(*dJcdBfQQLhIAYHPxwQeg->getCudnnHandle(), 
mtolGPkUMBYDlSSqrRzc, getOnePtr(), *desc, opTensor->getData(), *desc, 
PQjbchiGbyJfmpiqPpOC, XYbzSmRQGatVJtGmDZSo, ipTensor->getData(), getZeroPtr(), 
XYbzSmRQGatVJtGmDZSo, UWAGLbDcvybdWBtshhsr)); int eVAFqeShtGZAZluKdMvQ = 
ipTensor->getNumElements(); int tGsvtyAVkrDznETdweDC = 
(eVAFqeShtGZAZluKdMvQ < 1024) ? eVAFqeShtGZAZluKdMvQ : 1024; int 
KHClOltUSuqFVVErSxVb = (eVAFqeShtGZAZluKdMvQ + tGsvtyAVkrDznETdweDC - 
1)/tGsvtyAVkrDznETdweDC; 
doMWMaxPoolingLayerImpl<<<KHClOltUSuqFVVErSxVb, 
tGsvtyAVkrDznETdweDC>>>( UWAGLbDcvybdWBtshhsr, 
static_cast<MWTensor<float>*>(maxpoolLayer->getOutputTensor(1))->getData(), 
eVAFqeShtGZAZluKdMvQ); } return; } void MWMaxPoolingLayerImpl::cleanup() { 
CUDNN_CALL(hipdnnDestroyPoolingDescriptor(mtolGPkUMBYDlSSqrRzc));  } float* 
MWMaxPoolingLayerImpl::getIndexData()  { return 
static_cast<MWTensor<float>*>(getLayer()->getOutputTensor(1))->getData(); } 
void __global__ __launch_bounds__(1024) MWSetDyForBackPropImpl(float * 
PQjbchiGbyJfmpiqPpOC, const int fDqxEdcpBDmVQxZEmQxm) { for(int i = blockDim.x * 
blockIdx.x + threadIdx.x; i < fDqxEdcpBDmVQxZEmQxm; i+= blockDim.x*gridDim.x) { 
PQjbchiGbyJfmpiqPpOC[i] = i+1; } } void __global__ __launch_bounds__(1024) 
doMWMaxPoolingLayerImpl(float * UWAGLbDcvybdWBtshhsr, float * 
UVzBVEOIylFjkSgHwFMp, const int BdqURaHPmdnfzvtUvocl) { for(int i = blockDim.x * 
blockIdx.x + threadIdx.x; i < BdqURaHPmdnfzvtUvocl; i+= blockDim.x*gridDim.x) { if 
(static_cast<int>(UWAGLbDcvybdWBtshhsr[i]) != 0){ 
UVzBVEOIylFjkSgHwFMp[static_cast<int>(UWAGLbDcvybdWBtshhsr[i])-1] = 
i; } } } MWFCLayerImpl::MWFCLayerImpl(MWCNNLayer* layer, MWTargetNetworkImpl* 
ntwk_impl, int XLJXOFXdnZOyJvtltbyr, int lsqeARVLtpJTWezgnTkg, const char* 
xHViLEwTujGGrPZZgmbF,  const char* JwxFdqOKggeawILBfGgg) : 
MWCNNLayerImpl(layer, ntwk_impl)  , 
CCKWXUFWgrbBMjwfpOBN(XLJXOFXdnZOyJvtltbyr) , 
CDJtexcMbXMWAmnNZsNf(lsqeARVLtpJTWezgnTkg) , vpXxoeEhdEosLSsYXkNG(NULL) , 
wJyXsrUCMgxdIKVIJSyx(NULL) , IwKnaBoXVubIRYcxEJLH(NULL) , 
xHiBGayUfxIpXKkCTDNU(false) { 
CUDNN_CALL(hipdnnCreateTensorDescriptor(&JgLfgHrHMEMmMYTettJF)); 
createAndAddDescriptor(getLayer()->getOutputTensor(0)->getSourcePortIndex()); 
CUDA_CALL(hipMalloc((void**)&vpXxoeEhdEosLSsYXkNG, 
sizeof(float)*CCKWXUFWgrbBMjwfpOBN*CDJtexcMbXMWAmnNZsNf)); 
CUDA_CALL(hipMalloc((void**)&IwKnaBoXVubIRYcxEJLH, 
sizeof(float)*CDJtexcMbXMWAmnNZsNf)); wJyXsrUCMgxdIKVIJSyx = 
MALLOC_CALL(sizeof(float)*CCKWXUFWgrbBMjwfpOBN*CDJtexcMbXMWAmnNZsNf); 
loadWeights(xHViLEwTujGGrPZZgmbF); loadBias(JwxFdqOKggeawILBfGgg); } 
MWFCLayerImpl::~MWFCLayerImpl() { } void MWFCLayerImpl::propagateSize() { 
MWFCLayer* fcLayer = static_cast<MWFCLayer*>(getLayer()); MWTensorBase* 
opTensor = fcLayer->getOutputTensor(0); hipdnnTensorDescriptor_t* desc = 
getDescriptor(opTensor->getSourcePortIndex()); assert(desc); 
setDescriptor<float>(*desc, static_cast<MWTensor<float>*>(opTensor)); if 
(opTensor->getSequenceLength() == 1) { 
CUDNN_CALL(hipdnnSetTensor4dDescriptor(JgLfgHrHMEMmMYTettJF, HIPDNN_TENSOR_NCHW, 
HIPDNN_DATA_FLOAT, 1, CDJtexcMbXMWAmnNZsNf, 1, 1)); } else { int dims[5] 
= {1, 1, CDJtexcMbXMWAmnNZsNf, 1, 1}; int strides[5]; 
MWTensorBase::getStrides(dims, 5, strides); 
CUDNN_CALL(hipdnnSetTensorNdDescriptor(JgLfgHrHMEMmMYTettJF, HIPDNN_DATA_FLOAT, 5, 
dims, strides)); } } void MWFCLayerImpl::loadWeights(const char* 
QMgBqCuvjnbWHWiVPEwn) {  FILE* QhTesEEIHwhNmHSeYbRR = 
MWCNNLayer::openBinaryFile(QMgBqCuvjnbWHWiVPEwn); assert(QhTesEEIHwhNmHSeYbRR); int 
dMxIKDGTITyhdLqIHBLA = CCKWXUFWgrbBMjwfpOBN*CDJtexcMbXMWAmnNZsNf;  
call_fread(wJyXsrUCMgxdIKVIJSyx, sizeof(float), dMxIKDGTITyhdLqIHBLA, 
QhTesEEIHwhNmHSeYbRR, QMgBqCuvjnbWHWiVPEwn); fclose(QhTesEEIHwhNmHSeYbRR); } void 
MWFCLayerImpl::prepareWeights() { if (!xHiBGayUfxIpXKkCTDNU) { int 
dMxIKDGTITyhdLqIHBLA = CCKWXUFWgrbBMjwfpOBN*CDJtexcMbXMWAmnNZsNf; 
MWFCLayer* fcLayer = static_cast<MWFCLayer*>(getLayer()); MWTensorBase* 
ipTensor = fcLayer->getInputTensor(0); if( ipTensor->getHeight() != 1 && 
ipTensor->getWidth() != 1 ) { float* KZWeXiYFmdpQdsgidKeG = 
MALLOC_CALL(sizeof(float)*ipTensor->getHeight()*ipTensor->getWidth()); for(int 
k=0; k<dMxIKDGTITyhdLqIHBLA/ipTensor->getHeight()/ipTensor->getWidth(); k++) { 
for(int i=0; i<ipTensor->getHeight()*ipTensor->getWidth(); i++) 
KZWeXiYFmdpQdsgidKeG[i]=wJyXsrUCMgxdIKVIJSyx[k*ipTensor->getHeight()*ipTensor->getWidth()+i]; 
for(int j=0; j<ipTensor->getHeight(); j++) for(int i=0; i<ipTensor->getWidth(); 
i++) 
wJyXsrUCMgxdIKVIJSyx[k*ipTensor->getHeight()*ipTensor->getWidth()+j*ipTensor->getWidth()+i]=KZWeXiYFmdpQdsgidKeG[j+i*ipTensor->getHeight()]; 
} free(KZWeXiYFmdpQdsgidKeG); } CUDA_CALL(hipMemcpy(vpXxoeEhdEosLSsYXkNG, 
wJyXsrUCMgxdIKVIJSyx, sizeof(float)*dMxIKDGTITyhdLqIHBLA, 
hipMemcpyHostToDevice)); free(wJyXsrUCMgxdIKVIJSyx); 
wJyXsrUCMgxdIKVIJSyx = NULL; xHiBGayUfxIpXKkCTDNU = true; } } void 
MWFCLayerImpl::loadBias(const char* QMgBqCuvjnbWHWiVPEwn) { MWFCLayer* fcLayer = 
static_cast<MWFCLayer*>(getLayer()); MWTensorBase* opTensor = 
fcLayer->getOutputTensor(0); FILE* QhTesEEIHwhNmHSeYbRR = 
MWCNNLayer::openBinaryFile(QMgBqCuvjnbWHWiVPEwn); assert(QhTesEEIHwhNmHSeYbRR); int 
dMxIKDGTITyhdLqIHBLA = CDJtexcMbXMWAmnNZsNf;  float* KZWeXiYFmdpQdsgidKeG = 
MALLOC_CALL(sizeof(float)*dMxIKDGTITyhdLqIHBLA); call_fread(KZWeXiYFmdpQdsgidKeG, 
sizeof(float), dMxIKDGTITyhdLqIHBLA, QhTesEEIHwhNmHSeYbRR, QMgBqCuvjnbWHWiVPEwn); 
CUDA_CALL(hipMemcpy(IwKnaBoXVubIRYcxEJLH, KZWeXiYFmdpQdsgidKeG, 
sizeof(float)*dMxIKDGTITyhdLqIHBLA, hipMemcpyHostToDevice)); 
free(KZWeXiYFmdpQdsgidKeG); fclose(QhTesEEIHwhNmHSeYbRR); } void 
MWFCLayerImpl::postSetup() { prepareWeights(); } void MWFCLayerImpl::predict() 
{ MWFCLayer* fcLayer = static_cast<MWFCLayer*>(getLayer()); MWTensorBase* 
ipTensorBase = fcLayer->getInputTensor(0); MWTensorBase* opTensorBase = 
fcLayer->getOutputTensor(0); MWTensor<float>* ipTensor = 
static_cast<MWTensor<float>*>(ipTensorBase); MWTensor<float>* opTensor = 
static_cast<MWTensor<float>*>(opTensorBase); int numOutputRows = 
opTensor->getChannels(); int numOutputCols = 
ipTensor->getBatchSize()*ipTensor->getSequenceLength(); int innerDimension = 
ipTensor->getHeight()*ipTensor->getWidth()*ipTensor->getChannels(); int 
UKtMXCCqdjeyaVHabkxg=1; int URgvgDXnZskIYGdtimcU=1; if(opTensor->getBatchSize() == 1 && 
opTensor->getSequenceLength() == 1) { CUDA_CALL(hipMemcpy(opTensor->getData(), 
IwKnaBoXVubIRYcxEJLH, sizeof(float)*numOutputRows, hipMemcpyDeviceToDevice)); 
CUBLAS_CALL(hipblasSgemv(*dJcdBfQQLhIAYHPxwQeg->getCublasHandle(), HIPBLAS_OP_T, 
innerDimension, numOutputRows, getOnePtr(), vpXxoeEhdEosLSsYXkNG, innerDimension, 
ipTensor->getData(), UKtMXCCqdjeyaVHabkxg, getOnePtr(), opTensor->getData(), 
URgvgDXnZskIYGdtimcU)); } else { 
CUBLAS_CALL(hipblasSgemm(*dJcdBfQQLhIAYHPxwQeg->getCublasHandle(), HIPBLAS_OP_T, 
HIPBLAS_OP_N, numOutputRows, numOutputCols, innerDimension, getOnePtr(), 
vpXxoeEhdEosLSsYXkNG, innerDimension, ipTensor->getData(), innerDimension, 
getZeroPtr(), opTensor->getData(), numOutputRows)); hipdnnTensorDescriptor_t* 
desc = getDescriptor(opTensor->getSourcePortIndex()); assert(desc); 
CUDNN_CALL(hipdnnAddTensor(*dJcdBfQQLhIAYHPxwQeg->getCudnnHandle(), getOnePtr(), 
JgLfgHrHMEMmMYTettJF, IwKnaBoXVubIRYcxEJLH, getOnePtr(), *desc, opTensor->getData())); } 
return; } void MWFCLayerImpl::cleanup() { if (vpXxoeEhdEosLSsYXkNG) { 
CUDA_FREE_CALL(vpXxoeEhdEosLSsYXkNG); vpXxoeEhdEosLSsYXkNG = NULL; } 
CUDNN_CALL(hipdnnDestroyTensorDescriptor(JgLfgHrHMEMmMYTettJF)); if 
(IwKnaBoXVubIRYcxEJLH) { CUDA_FREE_CALL(IwKnaBoXVubIRYcxEJLH); IwKnaBoXVubIRYcxEJLH = NULL; } } 
MWSoftmaxLayerImpl::MWSoftmaxLayerImpl(MWCNNLayer* layer, MWTargetNetworkImpl* 
ntwk_impl) : MWCNNLayerImpl(layer, ntwk_impl)  { 
CUDNN_CALL(hipdnnCreateTensorDescriptor(&shEncNmxJsMuJKwbrwok)); 
CUDNN_CALL(hipdnnCreateTensorDescriptor(&sjLjZacPSDNBEjAccrGU)); } 
MWSoftmaxLayerImpl::~MWSoftmaxLayerImpl() { } void 
MWSoftmaxLayerImpl::propagateSize() { MWSoftmaxLayer* sfmxLayer = 
static_cast<MWSoftmaxLayer*>(getLayer()); MWTensorBase* ipTensor = 
sfmxLayer->getInputTensor(0); MWTensorBase* opTensor = 
sfmxLayer->getOutputTensor(0); 
CUDNN_CALL(hipdnnSetTensor4dDescriptor(shEncNmxJsMuJKwbrwok, HIPDNN_TENSOR_NCHW, 
HIPDNN_DATA_FLOAT, ipTensor->getSequenceLength()*ipTensor->getBatchSize(), 
ipTensor->getChannels(), ipTensor->getHeight(), ipTensor->getWidth())); 
CUDNN_CALL(hipdnnSetTensor4dDescriptor(sjLjZacPSDNBEjAccrGU, HIPDNN_TENSOR_NCHW, 
HIPDNN_DATA_FLOAT, opTensor->getSequenceLength()*opTensor->getBatchSize(), 
opTensor->getChannels(), opTensor->getHeight(), opTensor->getWidth())); } void 
MWSoftmaxLayerImpl::predict() { MWSoftmaxLayer* sfmxLayer = 
static_cast<MWSoftmaxLayer*>(getLayer()); MWTensorBase* ipTensorBase = 
sfmxLayer->getInputTensor(0); MWTensorBase* opTensorBase = 
sfmxLayer->getOutputTensor(0); MWTensor<float>* ipTensor = 
static_cast<MWTensor<float>*>(ipTensorBase); MWTensor<float>* opTensor = 
static_cast<MWTensor<float>*>(opTensorBase); 
CUDNN_CALL(hipdnnSoftmaxForward(*dJcdBfQQLhIAYHPxwQeg->getCudnnHandle(), 
HIPDNN_SOFTMAX_ACCURATE, HIPDNN_SOFTMAX_MODE_CHANNEL, getOnePtr(), 
shEncNmxJsMuJKwbrwok, ipTensor->getData(), getZeroPtr(), 
sjLjZacPSDNBEjAccrGU, opTensor->getData())); } void 
MWSoftmaxLayerImpl::cleanup() { 
CUDNN_CALL(hipdnnDestroyTensorDescriptor(shEncNmxJsMuJKwbrwok)); 
CUDNN_CALL(hipdnnDestroyTensorDescriptor(sjLjZacPSDNBEjAccrGU)); } 
MWAvgPoolingLayerImpl::MWAvgPoolingLayerImpl(MWCNNLayer* layer, 
MWTargetNetworkImpl* ntwk_impl, int DSsxcjIrUgZCKZovyNQf,  int 
EfvWctmlsWAPsxXgdKWf,  int FrpxvsDMwwgbpqHXWxmN,  int 
FwLnexHgxHRquTKmNpoa,  int ClEhcJFlvGCgiavziIag,  int 
CZNYmBcNFSZWvaCklqeM, int CufLFODQDXTAPyRqYodN, int 
DCdZnqpcBnvXVgEsLBnz) : MWCNNLayerImpl(layer, ntwk_impl) , 
TaAJDyqFVJXfAfCJhOuU(NULL) , DRzwhbNPpftRRIXXfHzd(DSsxcjIrUgZCKZovyNQf) , 
ECTnqgWHyHCHCLBZlffd(EfvWctmlsWAPsxXgdKWf) , 
DGzdAcREJHGXjyRzNjJV(DSsxcjIrUgZCKZovyNQf) , 
DqxLTLaJwwgQqmrtCDuu(EfvWctmlsWAPsxXgdKWf) , 
FpguQZSermqZCMRiUfML(FrpxvsDMwwgbpqHXWxmN) , 
FshVHIJMRAhtQirYPlZd(FwLnexHgxHRquTKmNpoa) , 
CTCbzQMDaLxINPbODdng(ClEhcJFlvGCgiavziIag) , 
CLOUhPjbgggWoXHTtmjC(CZNYmBcNFSZWvaCklqeM) , 
CpMjJjtGOeWOzwxpAAQP(CufLFODQDXTAPyRqYodN) , 
CqtPRJvHlGJFssiPzsOm(DCdZnqpcBnvXVgEsLBnz) , 
IIiwAtyrOtLzLWAUlTey((CTCbzQMDaLxINPbODdng != CLOUhPjbgggWoXHTtmjC) 
|| (CpMjJjtGOeWOzwxpAAQP != CqtPRJvHlGJFssiPzsOm)) , 
muwRQxtWMMXAPxSuMYBw(ClEhcJFlvGCgiavziIag) , 
nDsbARncmIrIaLubvLVZ(CufLFODQDXTAPyRqYodN) { 
CUDNN_CALL(hipdnnCreatePoolingDescriptor(&mtolGPkUMBYDlSSqrRzc)); MWTensorBase* 
ipTensor = getLayer()->getInputTensor(0); if (IIiwAtyrOtLzLWAUlTey) {  
muwRQxtWMMXAPxSuMYBw = 0;  nDsbARncmIrIaLubvLVZ = 0; 
TaAJDyqFVJXfAfCJhOuU = new MWTensor<float>(-1, -1, -1, -1, -1, NULL, getLayer(), 0); 
if (!TaAJDyqFVJXfAfCJhOuU) { MWCNNLayerImpl::throwAllocationError(__LINE__ , 
__FILE__); } CUDNN_CALL(hipdnnCreateTensorDescriptor(&XYbzSmRQGatVJtGmDZSo));  } 
else { TaAJDyqFVJXfAfCJhOuU = ipTensor;  } assert(TaAJDyqFVJXfAfCJhOuU != NULL); 
MWAvgPoolingLayer* avgpoolLayer = static_cast<MWAvgPoolingLayer*>(getLayer()); 
MWTensorBase* opTensor = avgpoolLayer->getOutputTensor(0); 
createAndAddDescriptor(opTensor->getSourcePortIndex()); } 
MWAvgPoolingLayerImpl::~MWAvgPoolingLayerImpl() { } void 
MWAvgPoolingLayerImpl::propagateSize() { MWTensorBase* ipTensor = 
getLayer()->getInputTensor(0); if ((DRzwhbNPpftRRIXXfHzd == -1) && 
(ECTnqgWHyHCHCLBZlffd == -1)) { DGzdAcREJHGXjyRzNjJV = ipTensor->getHeight(); 
DqxLTLaJwwgQqmrtCDuu = ipTensor->getWidth(); } int inputH; int inputW; if 
(IIiwAtyrOtLzLWAUlTey) { inputH = ipTensor->getHeight() + 
CTCbzQMDaLxINPbODdng + CLOUhPjbgggWoXHTtmjC;  inputW = ipTensor->getWidth() + 
CpMjJjtGOeWOzwxpAAQP + CqtPRJvHlGJFssiPzsOm; } else { inputH = 
ipTensor->getHeight(); inputW = ipTensor->getWidth(); } 
TaAJDyqFVJXfAfCJhOuU->setHeight(inputH); TaAJDyqFVJXfAfCJhOuU->setWidth(inputW); 
TaAJDyqFVJXfAfCJhOuU->setChannels(ipTensor->getChannels()); 
TaAJDyqFVJXfAfCJhOuU->setBatchSize(ipTensor->getBatchSize()); 
TaAJDyqFVJXfAfCJhOuU->setSequenceLength(ipTensor->getSequenceLength()); 
assert(TaAJDyqFVJXfAfCJhOuU->getSequenceLength() == 1); if 
(IIiwAtyrOtLzLWAUlTey) { 
CUDNN_CALL(hipdnnSetTensor4dDescriptor(XYbzSmRQGatVJtGmDZSo, HIPDNN_TENSOR_NCHW, 
HIPDNN_DATA_FLOAT, TaAJDyqFVJXfAfCJhOuU->getBatchSize(), TaAJDyqFVJXfAfCJhOuU->getChannels(), 
TaAJDyqFVJXfAfCJhOuU->getHeight(), TaAJDyqFVJXfAfCJhOuU->getWidth())); } else { 
XYbzSmRQGatVJtGmDZSo = MWCNNLayerImpl::getCuDNNDescriptor(TaAJDyqFVJXfAfCJhOuU); } 
MWTensorBase* opTensor = getLayer()->getOutputTensor(0); 
CUDNN_CALL(hipdnnSetPooling2dDescriptor(mtolGPkUMBYDlSSqrRzc,  
HIPDNN_POOLING_AVERAGE_COUNT_INCLUDE_PADDING,  HIPDNN_NOT_PROPAGATE_NAN,  
DGzdAcREJHGXjyRzNjJV,  DqxLTLaJwwgQqmrtCDuu,  muwRQxtWMMXAPxSuMYBw,  
nDsbARncmIrIaLubvLVZ,  FpguQZSermqZCMRiUfML,  FshVHIJMRAhtQirYPlZd)); 
hipdnnTensorDescriptor_t* desc = getDescriptor(opTensor->getSourcePortIndex()); 
assert(desc); setDescriptor<float>(*desc, 
static_cast<MWTensor<float>*>(opTensor)); } void 
MWAvgPoolingLayerImpl::allocate() { MWTensorBase* ipTensor = 
getLayer()->getInputTensor(0); if (IIiwAtyrOtLzLWAUlTey) { float* 
newInput; int inputH = ipTensor->getHeight() + CTCbzQMDaLxINPbODdng + 
CLOUhPjbgggWoXHTtmjC;  int inputW = ipTensor->getWidth() + 
CpMjJjtGOeWOzwxpAAQP + CqtPRJvHlGJFssiPzsOm; int paddedSize = 
ipTensor->getBatchSize() * ipTensor->getChannels() * inputH * inputW; 
CUDA_CALL(hipMalloc((void**)&newInput, sizeof(float)*paddedSize)); 
CUDA_CALL(hipMemset(newInput, 0, sizeof(float)*paddedSize)); 
static_cast<MWTensor<float>*>(TaAJDyqFVJXfAfCJhOuU)->setData(newInput); } } void 
MWAvgPoolingLayerImpl::deallocate() { if (TaAJDyqFVJXfAfCJhOuU != 
getLayer()->getInputTensor(0)) { assert(IIiwAtyrOtLzLWAUlTey); 
CUDA_FREE_CALL(static_cast<MWTensor<float>*>(TaAJDyqFVJXfAfCJhOuU)->getData()); 
static_cast<MWTensor<float>*>(TaAJDyqFVJXfAfCJhOuU)->setData((float*)NULL); } } void 
MWAvgPoolingLayerImpl::predict() { MWAvgPoolingLayer* avgpoolLayer = 
static_cast<MWAvgPoolingLayer*>(getLayer()); MWTensorBase* opTensorBase = 
avgpoolLayer->getOutputTensor(0); MWTensorBase* ipTensorBase = 
avgpoolLayer->getInputTensor(0); MWTensor<float>* ipTensor = 
static_cast<MWTensor<float>*>(ipTensorBase); MWTensor<float>* opTensor = 
static_cast<MWTensor<float>*>(opTensorBase); if (TaAJDyqFVJXfAfCJhOuU != 
avgpoolLayer->getInputTensor()) { 
CUDA_CALL(hipMemset(static_cast<MWTensor<float>*>(TaAJDyqFVJXfAfCJhOuU)->getData(), 
0, sizeof(float)*TaAJDyqFVJXfAfCJhOuU->getNumElements()));  
MWCNNLayerImpl::padInput(ipTensor->getData(), ipTensor->getHeight(), 
ipTensor->getWidth(), ipTensor->getChannels(), TaAJDyqFVJXfAfCJhOuU->getHeight(), 
TaAJDyqFVJXfAfCJhOuU->getWidth(), CTCbzQMDaLxINPbODdng, CpMjJjtGOeWOzwxpAAQP, 
static_cast<MWTensor<float>*>(TaAJDyqFVJXfAfCJhOuU)->getData(), 
ipTensor->getNumElements()); } assert(opTensor->getData() != 
static_cast<MWTensor<float>*>(TaAJDyqFVJXfAfCJhOuU)->getData()); 
hipdnnTensorDescriptor_t* desc = getDescriptor(opTensor->getSourcePortIndex()); 
assert(desc); 
CUDNN_CALL(hipdnnPoolingForward(*dJcdBfQQLhIAYHPxwQeg->getCudnnHandle(), 
mtolGPkUMBYDlSSqrRzc, getOnePtr(), XYbzSmRQGatVJtGmDZSo, 
static_cast<MWTensor<float>*>(TaAJDyqFVJXfAfCJhOuU)->getData(), getZeroPtr(), *desc, 
opTensor->getData())); } void MWAvgPoolingLayerImpl::cleanup() { 
CUDNN_CALL(hipdnnDestroyPoolingDescriptor(mtolGPkUMBYDlSSqrRzc)); if 
(TaAJDyqFVJXfAfCJhOuU != getLayer()->getInputTensor(0)) { 
assert(IIiwAtyrOtLzLWAUlTey); 
CUDNN_CALL(hipdnnDestroyTensorDescriptor(XYbzSmRQGatVJtGmDZSo)); } } 
MWOutputLayerImpl::MWOutputLayerImpl(MWCNNLayer* layer, MWTargetNetworkImpl* 
ntwk_impl) : MWCNNLayerImpl(layer, ntwk_impl) { } 
MWOutputLayerImpl::~MWOutputLayerImpl() { } void 
MWOutputLayerImpl::propagateSize() { } void 
MWOutputLayerImpl::deallocateOutputData(int) { } void 
MWOutputLayerImpl::predict() { } void MWOutputLayerImpl::cleanup() { }