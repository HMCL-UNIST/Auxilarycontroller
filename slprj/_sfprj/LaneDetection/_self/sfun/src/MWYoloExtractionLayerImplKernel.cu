#include "hip/hip_runtime.h"
#include "MWKernelHeaders.hpp"
 void __global__ __launch_bounds__(1024) YoloExtractionImpl(float* inputBuffer, 
float* outputBuffer_iouxy, float* outputBuffer_wh, float* outputBuffer_cscores, 
int dkLDkRwCBjeybwDHbKiE, int YOWMnLKOMqAODXiVNoGy, long int 
YNmJhGSUszJKxsodxiuV, long int YNDVziqpDddiXQKYZZhX, long int 
jHzoRQWaHafftmrmuvHO, long int jLyhrFjMmVnNjoeDJCwH, 
long int iwclITrbVyVrJaArrXNr, const long int BdqURaHPmdnfzvtUvocl) { 
for (int idx = blockDim.x * blockIdx.x + threadIdx.x; idx < BdqURaHPmdnfzvtUvocl; idx 
+= blockDim.x * gridDim.x) { int cwCXkgHfZmFQRzNVUlCO = idx / 
YNDVziqpDddiXQKYZZhX; long int FOcStuqCptsGIZXskVpC = idx - 
(YNDVziqpDddiXQKYZZhX * cwCXkgHfZmFQRzNVUlCO); int LklYEpYUjaLTgcFFAaJX = 
static_cast<int>(FOcStuqCptsGIZXskVpC / YNmJhGSUszJKxsodxiuV); long 
int FeVcBgtQmTLtmnNcJGMY = FOcStuqCptsGIZXskVpC - 
(YNmJhGSUszJKxsodxiuV * LklYEpYUjaLTgcFFAaJX); int zFaEOIBQYqPoaerpaixN = 
static_cast<int>(FeVcBgtQmTLtmnNcJGMY % YOWMnLKOMqAODXiVNoGy); int 
yCdIUfwoZFngCRRRkCTg = static_cast<int>(FeVcBgtQmTLtmnNcJGMY / YOWMnLKOMqAODXiVNoGy); 
if (LklYEpYUjaLTgcFFAaJX < 3 * dkLDkRwCBjeybwDHbKiE) { long int opIdx = cwCXkgHfZmFQRzNVUlCO * 
jHzoRQWaHafftmrmuvHO + LklYEpYUjaLTgcFFAaJX * 
YNmJhGSUszJKxsodxiuV + yCdIUfwoZFngCRRRkCTg * YOWMnLKOMqAODXiVNoGy + 
zFaEOIBQYqPoaerpaixN; outputBuffer_iouxy[opIdx] = inputBuffer[idx]; } else if 
(LklYEpYUjaLTgcFFAaJX >= 3 * dkLDkRwCBjeybwDHbKiE && LklYEpYUjaLTgcFFAaJX < 5 * 
dkLDkRwCBjeybwDHbKiE) { int LtEgcYoEYjkrWuohutgw = LklYEpYUjaLTgcFFAaJX - (3 * 
dkLDkRwCBjeybwDHbKiE); long int opIdx = cwCXkgHfZmFQRzNVUlCO * 
jLyhrFjMmVnNjoeDJCwH + LtEgcYoEYjkrWuohutgw * 
YNmJhGSUszJKxsodxiuV + yCdIUfwoZFngCRRRkCTg * YOWMnLKOMqAODXiVNoGy + 
zFaEOIBQYqPoaerpaixN; outputBuffer_wh[opIdx] = inputBuffer[idx]; } else { int 
LtEgcYoEYjkrWuohutgw = LklYEpYUjaLTgcFFAaJX - (5 * dkLDkRwCBjeybwDHbKiE); long int opIdx = 
cwCXkgHfZmFQRzNVUlCO * iwclITrbVyVrJaArrXNr + LtEgcYoEYjkrWuohutgw * 
YNmJhGSUszJKxsodxiuV + yCdIUfwoZFngCRRRkCTg * YOWMnLKOMqAODXiVNoGy + 
zFaEOIBQYqPoaerpaixN; outputBuffer_cscores[opIdx] = inputBuffer[idx]; } } }